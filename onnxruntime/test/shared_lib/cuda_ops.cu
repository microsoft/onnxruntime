// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_runtime.h>

#include <cstdint>

using namespace std;

__global__ void cuda_add_impl(int64_t N, float* O, const float* X, const float* Y) {
  auto offset = threadIdx.x;
  if (offset < N) {
    O[offset] = Y[offset] + X[offset];
  }
}

void cuda_add(int64_t N, float* O, const float* X, const float* Y) {
  cuda_add_impl<<<1, 256>>>(N, O, X, Y);
}

template<typename T>
__global__ void cuda_slice_impl(const T* X , int64_t from, int64_t to, T* Y) {
  auto offset = threadIdx.x;
  if (offset >= from && offset < to) {
    Y[offset - from] = X[offset];
  }
}

template<typename T>
void cuda_slice(const T* X, int64_t from, int64_t to, T* Y) {
    cuda_slice_impl<T><<<1, 256>>>(X, from, to, Y);
}

template void cuda_slice(const float*, int64_t, int64_t, float*);
template void cuda_slice(const double*, int64_t, int64_t, double*);
