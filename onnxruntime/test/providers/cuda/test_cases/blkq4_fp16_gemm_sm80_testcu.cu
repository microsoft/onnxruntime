/**
 * Copyright (c) Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License.
 *
 * Module Name:
 *    blkq4_fp16_gemm_sm80_testcu.cu
 *
 * Abstract:
 *   Test code for invoking block-wise quantized 4b GEMM kernels.
 *   This part requires CUTLASS header files, which do not play
 *   well with gtest headers.
 */

#include "blkq4_fp16_gemm_sm80.h"

#include <random>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "core/mickey/blk_q4/f16_gemm_sm80.h"
#include "core/mickey/gemm/device/quant_b4_gemm.h"

#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"

#include "core/common/common.h"

namespace onnxruntime {
namespace cuda {
namespace test {

Status sm80_supported() {
  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::ostringstream ss;
    ss << "Unable to obtain GPU device properties: " << hipGetErrorString(error);
    return Status(common::ONNXRUNTIME, common::ENGINE_ERROR, ss.str());
  }

  if (!((props.major * 10 + props.minor) >= 80)) {
    std::ostringstream ss;
    ss << "Device compute capability mismatch, desired 8.0, actual " << props.major << "." << props.minor;
    return Status(common::ONNXRUNTIME, common::ENGINE_ERROR, ss.str());
  }
  return Status::OK();
}

/**
 * @brief Reference implementation of GEMM
 *        Copied directly from cutlass util/reference/device/gemm.h
 *        for the strange reason that compiler insists on asking
 *        for explicit stream argument in kernel launch.
 */
template <
    typename ElementA,
    typename LayoutA,
    typename ElementB,
    typename LayoutB,
    typename ElementC,
    typename LayoutC,
    typename ScalarType,
    typename AccumulatorType>
void compute_gemm_ref(
    cutlass::gemm::GemmCoord problem_size,
    ScalarType alpha,
    cutlass::TensorRef<ElementA, LayoutA> tensor_a,
    cutlass::TensorRef<ElementB, LayoutB> tensor_b,
    ScalarType beta,
    cutlass::TensorRef<ElementC, LayoutC> tensor_c,
    cutlass::TensorRef<ElementC, LayoutC> tensor_d,
    AccumulatorType initial_accum = AccumulatorType(0)) {
  // Blocking structure potentially improves performance of reference implementation
  // with a minor increase in complexity.
  //
  // Note, this reference implementation is NOT expected to approach peak performance.
  using OutputTile = cutlass::MatrixShape<4, 4>;

  dim3 block(16, 8);

  dim3 grid(
      (problem_size.m() + block.x * OutputTile::kRow - 1) / (block.x * OutputTile::kRow),
      (problem_size.n() + block.y * OutputTile::kColumn - 1) / (block.y * OutputTile::kColumn));

  // Launch a GEMM kernel
  cutlass::reference::device::kernel::Gemm<
      cutlass::TensorRef<ElementA, LayoutA>,
      cutlass::TensorRef<ElementB, LayoutB>,
      cutlass::TensorRef<ElementC, LayoutC>,
      ScalarType,
      AccumulatorType,
      OutputTile,
      cutlass::multiply_add<AccumulatorType>,
      cutlass::NumericConverter<ElementC, ScalarType>><<<grid, block, 0, 0>>>(
      problem_size,
      alpha,
      tensor_a,
      tensor_b,
      beta,
      tensor_c,
      tensor_d,
      initial_accum);
}
////////////////////////////////////////////////////////////////////////////////////////////////////

//
// Converting cutlass tensor to MatrixRef
//

template <
    typename Element,
    typename LayoutCutlass,
    typename Layout = std::conditional_t<std::is_same<LayoutCutlass,
                                                      cutlass::layout::ColumnMajor>::value,
                                         ColumnMajorLayout, RowMajorLayout>>
__forceinline__
    MatrixRef<Element, Layout, true>
    make_MatrixRef(cutlass::HostTensor<Element, LayoutCutlass> const& tensor) {
  static_assert(std::is_same<LayoutCutlass, cutlass::layout::ColumnMajor>::value ||
                std::is_same<LayoutCutlass, cutlass::layout::RowMajor>::value);
  auto shape = make_Position(tensor.extent().row(), tensor.extent().column());
  auto* ptr = const_cast<typename std::remove_const<Element>::type*>(tensor.host_data());
  return MatrixRef<Element, Layout, true>(ptr, tensor.capacity(), shape);
}

template <
    typename Element,
    typename LayoutCutlass,
    typename Layout = std::conditional_t<std::is_same<LayoutCutlass, cutlass::layout::ColumnMajor>::value,
                                         ColumnMajorLayout, RowMajorLayout>>
__forceinline__
    MatrixRef<Element const, Layout, true>
    make_ConstMatrixRef(cutlass::HostTensor<Element, LayoutCutlass> const& tensor) {
  static_assert(std::is_same<LayoutCutlass, cutlass::layout::ColumnMajor>::value ||
                std::is_same<LayoutCutlass, cutlass::layout::RowMajor>::value);
  auto shape = make_Position(tensor.extent().row(), tensor.extent().column());
  return MatrixRef<Element const, Layout, true>(tensor.host_data(), tensor.capacity(), shape);
}

//
// Invoking the kernel
//

template <
    int block_size,
    bool column_wise_blocking,
    bool small_m,
    bool has_offsets>
void run_blkq4_gemm(int m, int n, int k) {
  unsigned int seed = 28571;  // Replace with desired seed value
  std::seed_seq seq{seed};
  std::mt19937 gen(seq);
  std::uniform_int_distribution<> dis(0, 8192);

  using ElementDequant = cutlass::half_t;
  using QuantBlocking =
      typename std::conditional<column_wise_blocking,
                                cutlass::MatrixShape<block_size, 1>,
                                cutlass::MatrixShape<1, block_size>>::type;

  using GemmRunner = BlkQ4F16GemmImpl<ElementDequant, QuantBlocking, small_m, has_offsets>;

  using ElementAccumulator = typename GemmRunner::ElementAccumulator;
  using ElementComputeEpilogue = typename GemmRunner::ElementComputeEpilogue;
  using ElementInputA = typename GemmRunner::ElementInputA;
  using ElementOutput = typename GemmRunner::ElementOutput;
  using ElementW = typename GemmRunner::ElementW;
  using ElementWPack = typename GemmRunner::ElementWPack;
  using ElementQScale = typename GemmRunner::ElementQScale;
  using ElementQOffset = typename GemmRunner::ElementQOffset;

  using LayoutInputA = typename GemmRunner::LayoutInputA;
  using LayoutOutput = typename GemmRunner::LayoutOutput;
  using LayoutInputWPack = typename GemmRunner::LayoutInputWPack;
  using LayoutInputQScale = typename GemmRunner::LayoutInputQScale;

  const cutlass::gemm::GemmCoord problem_size = {m, n, k};
  const auto q_weight_shape = cutlass::make_Coord(problem_size.k() / 2, problem_size.n());
  const auto meta_shape = cutlass::make_Coord(problem_size.k() / QuantBlocking::kRow, problem_size.n() /
                                                                                          QuantBlocking::kColumn);

  //
  // Generate quantized and dequantizeed input matrix B [K, N]
  //
  static_assert(std::is_same<LayoutInputWPack, cutlass::layout::ColumnMajor>::value);
  thrust::host_vector<ElementW> q_weights;
  thrust::host_vector<ElementQScale> q_scales;
  thrust::host_vector<ElementQOffset> q_zp;
  thrust::host_vector<ElementDequant> dequants;
  onnxruntime::cuda::test::blkq4_weights_gen<ElementDequant, block_size, column_wise_blocking, has_offsets>(
      problem_size.k(), problem_size.n(), dequants, q_weights, q_scales, q_zp);

  using PrepackT = onnxruntime::cuda::BlockwiseQuantization<
      ElementDequant,
      block_size,
      4,
      column_wise_blocking>;

  thrust::host_vector<ElementW> packed_w(q_weight_shape.product());
  PrepackT::prepack_weights(problem_size.k(), problem_size.n(), q_weights, packed_w);
  thrust::host_vector<ElementQScale> packed_scales(meta_shape.product());
  PrepackT::prepack_quant_scales(problem_size.k(), problem_size.n(), q_scales, packed_scales);
  thrust::host_vector<ElementQOffset> packed_zp;
  if constexpr (has_offsets) {
    packed_zp.resize(meta_shape.product());
    PrepackT::prepack_quant_offsets(problem_size.k(), problem_size.n(), q_zp, packed_zp);
  }

  cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(
      problem_size.mk());  // <- Create matrix A with dimensions M x K
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_c(
      problem_size.mn());  // <- Create matrix C with dimensions M x N
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d(
      problem_size.mn());  // <- Create matrix D with dimensions M x N used to store output from
                           // CUTLASS kernel

  // Fill input and output matrices on host using CUTLASS helper functions
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_a.host_view(),
      1,
      ElementInputA(4),
      ElementInputA(-4),
      2);  // <- Fill matrix A on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_c.host_view(),
      1,
      ElementOutput(4),
      ElementOutput(-4),
      0);  // <- Fill matrix C on host with uniform-distribution random data
  cutlass::reference::host::TensorFill(
      tensor_d.host_view());  // <- fill matrix D on host with zeros

  //
  // Copy data from host to GPU...
  //
  thrust::device_vector<ElementW> d_packed_w(packed_w);
  cutlass::TensorRef<ElementWPack const, LayoutInputWPack> ref_W(
      reinterpret_cast<ElementWPack const*>(d_packed_w.data().get()),
      LayoutInputWPack::packed({problem_size.k() / 2, problem_size.n() / 2}));

  thrust::device_vector<ElementQScale> d_packed_scales(packed_scales);
  cutlass::TensorRef<ElementQScale const, LayoutInputQScale> ref_scales(
      d_packed_scales.data().get(), LayoutInputQScale::packed(meta_shape));

  thrust::device_vector<ElementQOffset> d_packed_zp(packed_zp);
  cutlass::TensorRef<ElementQOffset const, LayoutInputQScale> ref_zp(
      d_packed_zp.data().get(), LayoutInputQScale::packed(meta_shape));

  tensor_a.sync_device();
  tensor_c.sync_device();
  tensor_d.sync_device();

  // run GEMM
  cutlass::Status status;
  if constexpr (has_offsets) {
    status = GemmRunner::run(
        nullptr, problem_size, tensor_a.device_ref(), ref_W,
        ref_scales, ref_zp,
        tensor_c.device_ref(), tensor_d.device_ref());
  } else {
    status = GemmRunner::run(
        nullptr, problem_size, tensor_a.device_ref(), ref_W,
        ref_scales,
        tensor_c.device_ref(), tensor_d.device_ref());
  }
  ORT_ENFORCE(status == cutlass::Status::kSuccess, "Kernel execution failed: ", cutlassGetStatusString(status));

  // Running reference kernel
  using ElementInputB = ElementInputA;
  using LayoutInputB = cutlass::layout::ColumnMajor;
  thrust::device_vector<ElementInputB> d_dequants(dequants);
  cutlass::TensorRef<ElementInputB, LayoutInputB> ref_B(
      d_dequants.data().get(), LayoutInputB::packed(problem_size.kn()));
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_ref_d(
      problem_size.mn());  // <- Create matrix D with dimensions M x N used to store output from
                           // reference kernel

  cutlass::reference::host::TensorFill(
      tensor_ref_d.host_view());  // <- fill matrix D for reference on host with zeros
  tensor_ref_d.sync_device();

  // Initialize alpha and beta for dot product computation
  ElementComputeEpilogue alpha = ElementComputeEpilogue(1);
  ElementComputeEpilogue beta = ElementComputeEpilogue(0);

  compute_gemm_ref<ElementInputA, LayoutInputA,
                   ElementInputB, LayoutInputB,
                   ElementOutput, LayoutOutput,
                   ElementComputeEpilogue, ElementAccumulator>(
      problem_size,
      alpha,
      tensor_a.device_ref(),
      ref_B,
      beta,
      tensor_c.device_ref(),
      tensor_ref_d.device_ref());

  //// Wait for kernels to finish
  hipDeviceSynchronize();

  //// Copy output data from CUTLASS and reference kernel to host for comparison
  tensor_d.sync_host();
  tensor_ref_d.sync_host();

  //// Check if output from CUTLASS kernel and reference kernel are equal or not
  bool passed = cutlass::reference::host::TensorEquals(
      tensor_d.host_view(),
      tensor_ref_d.host_view());
  ORT_ENFORCE(passed, "Gemm kernel result wrong!");
}

template void run_blkq4_gemm<16, true, false, true>(int m, int n, int k);
template void run_blkq4_gemm<16, true, false, false>(int m, int n, int k);
template void run_blkq4_gemm<32, true, false, true>(int m, int n, int k);
template void run_blkq4_gemm<32, true, false, false>(int m, int n, int k);
template void run_blkq4_gemm<64, true, false, true>(int m, int n, int k);
template void run_blkq4_gemm<64, true, false, false>(int m, int n, int k);
template void run_blkq4_gemm<16, false, false, true>(int m, int n, int k);
template void run_blkq4_gemm<16, false, false, false>(int m, int n, int k);
template void run_blkq4_gemm<32, false, false, true>(int m, int n, int k);
template void run_blkq4_gemm<32, false, false, false>(int m, int n, int k);
template void run_blkq4_gemm<64, false, false, true>(int m, int n, int k);
template void run_blkq4_gemm<64, false, false, false>(int m, int n, int k);
template void run_blkq4_gemm<16, true, true, true>(int m, int n, int k);
template void run_blkq4_gemm<16, true, true, false>(int m, int n, int k);
template void run_blkq4_gemm<32, true, true, true>(int m, int n, int k);
template void run_blkq4_gemm<32, true, true, false>(int m, int n, int k);
template void run_blkq4_gemm<64, true, true, true>(int m, int n, int k);
template void run_blkq4_gemm<64, true, true, false>(int m, int n, int k);
template void run_blkq4_gemm<16, false, true, true>(int m, int n, int k);
template void run_blkq4_gemm<16, false, true, false>(int m, int n, int k);
template void run_blkq4_gemm<32, false, true, true>(int m, int n, int k);
template void run_blkq4_gemm<32, false, true, false>(int m, int n, int k);
template void run_blkq4_gemm<64, false, true, true>(int m, int n, int k);
template void run_blkq4_gemm<64, false, true, false>(int m, int n, int k);



/// @brief Testing small tile GEMM impl
template <
    int block_size,
    bool column_wise_blocking,
    bool has_offsets>
void run_blkq4_small_gemm(int m, int n, int k) {
  unsigned int seed = 28571;  // Replace with desired seed value
  std::seed_seq seq{seed};
  std::mt19937 gen(seq);
  std::uniform_int_distribution<> dis(0, 8192);

  using PrepackT = onnxruntime::cuda::BlockwiseQuantization<
      cutlass::half_t,
      block_size,
      4,
      column_wise_blocking,
      true>;
  using QuantBlocking = cutlass::MatrixShape<PrepackT::QuantBlocking::kRow, PrepackT::QuantBlocking::kColumn>;
  using LayoutQmeta = typename std::conditional<std::is_same<typename PrepackT::LayoutQmeta, RowMajorLayout>::value,
                                            cutlass::layout::RowMajor, cutlass::layout::ColumnMajor>::type;

  using WarpShape = cutlass::gemm::GemmShape<16, 16, 64>;
  // change split k to 1 to help debug in case of test failure
  using GemmRunner = mickey::gemm::device::QuantB4Gemm<QuantBlocking, has_offsets, WarpShape, 4, 3>;

  using ElementW = uint8_t;
  using ElementWPack = cutlass::half_t;
  using ElementQScale = cutlass::half_t;
  using ElementQOffset = uint8_t;

  using LayoutInputA = cutlass::layout::RowMajor;
  using LayoutInputB = cutlass::layout::ColumnMajor;
  using LayoutOutput = cutlass::layout::RowMajor;

  const cutlass::gemm::GemmCoord problem_size = {m, n, k};
  const auto q_weight_shape = cutlass::make_Coord(problem_size.k() / 2, problem_size.n());
  const auto meta_shape = cutlass::make_Coord(problem_size.k() / QuantBlocking::kRow,
                                              problem_size.n() / QuantBlocking::kColumn);
  if ((problem_size.k() % QuantBlocking::kRow != 0) ||
    (problem_size.n() % QuantBlocking::kColumn) != 0){
    ORT_THROW("Test case setup fail: partial quantization block not supported!");
  }

  //
  // Generate quantized and dequantizeed input matrix B [K, N]
  //
  thrust::host_vector<ElementW> q_weights;
  thrust::host_vector<ElementQScale> q_scales;
  thrust::host_vector<ElementQOffset> q_zp;
  thrust::host_vector<cutlass::half_t> dequants;
  onnxruntime::cuda::test::blkq4_weights_gen<cutlass::half_t, block_size, column_wise_blocking, has_offsets>(
      problem_size.k(), problem_size.n(), dequants, q_weights, q_scales, q_zp);

  thrust::host_vector<ElementW> packed_w(q_weight_shape.product());
  PrepackT::prepack_weights(problem_size.k(), problem_size.n(), q_weights, packed_w);
  thrust::host_vector<ElementQScale> packed_scales(meta_shape.product());
  PrepackT::prepack_quant_scales(problem_size.k(), problem_size.n(), q_scales, packed_scales);
  thrust::host_vector<ElementQOffset> packed_zp;
  if constexpr (has_offsets) {
    packed_zp.resize(meta_shape.product());
    PrepackT::prepack_quant_offsets(problem_size.k(), problem_size.n(), q_zp, packed_zp);
  }

  cutlass::HostTensor<cutlass::half_t, LayoutInputA> tensor_a(
      problem_size.mk());  // <- Create matrix A with dimensions M x K
  cutlass::HostTensor<cutlass::half_t, LayoutOutput> tensor_c(
      problem_size.mn());  // <- Create matrix C with dimensions M x N
  cutlass::HostTensor<cutlass::half_t, LayoutOutput> tensor_d(
      problem_size.mn());  // <- Create matrix D with dimensions M x N used to store output from
                           // CUTLASS kernel

  // Fill input and output matrices on host using CUTLASS helper functions
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_a.host_view(),
      1,
      cutlass::half_t(1.25),
      cutlass::half_t(-1.0),
      5);  // <- Fill matrix A on host with uniform-distribution random data
//   std::cout << "==========  A:  ============ \n" << tensor_a.host_view() << std::endl;
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_c.host_view(),
      1,
      cutlass::half_t(1.25),
      cutlass::half_t(-1.0),
      0);  // <- Fill matrix C on host with uniform-distribution random data
  cutlass::reference::host::TensorFill(
      tensor_d.host_view());  // <- fill matrix D on host with zeros

  //
  // Copy data from host to GPU...
  //
  thrust::device_vector<ElementW> d_packed_w(packed_w);
  cutlass::TensorRef<ElementW const, cutlass::layout::ColumnMajor> ref_W(
      d_packed_w.data().get(),
      cutlass::layout::ColumnMajor::packed({problem_size.k(), problem_size.n() / 2}));

  thrust::device_vector<ElementQScale> d_packed_scales(packed_scales);
  cutlass::TensorRef<ElementQScale const, LayoutQmeta> ref_scales(
      d_packed_scales.data().get(), LayoutQmeta::packed(meta_shape));

  thrust::device_vector<ElementQOffset> d_packed_zp(packed_zp);
  cutlass::TensorRef<ElementQOffset const, LayoutQmeta> ref_zp(
      d_packed_zp.data().get(), LayoutQmeta::packed(meta_shape));

  tensor_a.sync_device();
  tensor_c.sync_device();
  tensor_d.sync_device();

  // run GEMM
  const void* ptr_zp = has_offsets ? thrust::raw_pointer_cast(d_packed_zp.data()) : nullptr;
  size_t zp_byte_stride = has_offsets ? ref_zp.stride(0) * sizeof(ElementQOffset) : size_t(0);
  cutlass::Status status = GemmRunner::run(
      nullptr, problem_size,
      tensor_d.device_data(), tensor_d.stride(0) * sizeof(cutlass::half_t),
      tensor_a.device_data(), tensor_a.stride(0) * sizeof(cutlass::half_t),
      thrust::raw_pointer_cast(d_packed_w.data()), problem_size.k() * sizeof(ElementW),
      thrust::raw_pointer_cast(d_packed_scales.data()), ref_scales.stride(0) * sizeof(ElementQScale),
      ptr_zp, zp_byte_stride);
  ORT_ENFORCE(status == cutlass::Status::kSuccess, "Kernel execution failed: ", cutlassGetStatusString(status));

  // Running reference kernel
  thrust::device_vector<cutlass::half_t> d_dequants(dequants);
  cutlass::TensorRef<cutlass::half_t, LayoutInputB> ref_B(
      d_dequants.data().get(), LayoutInputB::packed(problem_size.kn()));
  cutlass::HostTensor<cutlass::half_t, LayoutOutput> tensor_ref_d(
      problem_size.mn());  // <- Create matrix D with dimensions M x N used to store output from
                           // reference kernel

  cutlass::reference::host::TensorFill(
      tensor_ref_d.host_view());  // <- fill matrix D for reference on host with zeros
  tensor_ref_d.sync_device();

  // Initialize alpha and beta for dot product computation
  float alpha = 1.0f;
  float beta = 0.0f;

  compute_gemm_ref<cutlass::half_t, LayoutInputA,
                   cutlass::half_t, LayoutInputB,
                   cutlass::half_t, LayoutOutput,
                   float, float>(
      problem_size,
      alpha,
      tensor_a.device_ref(),
      ref_B,
      beta,
      tensor_c.device_ref(),
      tensor_ref_d.device_ref());

  //// Wait for kernels to finish
  hipDeviceSynchronize();

  //// Copy output data from CUTLASS and reference kernel to host for comparison
  tensor_d.sync_host();
  tensor_ref_d.sync_host();

  //// Check if output from CUTLASS kernel and reference kernel are equal or not
  for (int row = 0; row < problem_size.m(); ++row) {
    for (int col = 0; col < problem_size.n(); ++col) {
      if (tensor_d.at({row, col}) != tensor_ref_d.at({row, col})) {
        std::cout << "Mismatch at (" << row << ", " << col << "): "
                  << tensor_d.at({row, col}) << " vs " << tensor_ref_d.at({row, col}) << std::endl;
      }
    }
  }
  bool passed = cutlass::reference::host::TensorEquals(
      tensor_d.host_view(),
      tensor_ref_d.host_view());
  ORT_ENFORCE(passed, "Gemm kernel result wrong!");
}

template void run_blkq4_small_gemm<16, true, true>(int m, int n, int k);
template void run_blkq4_small_gemm<16, true, false>(int m, int n, int k);
template void run_blkq4_small_gemm<32, true, true>(int m, int n, int k);
template void run_blkq4_small_gemm<32, true, false>(int m, int n, int k);
template void run_blkq4_small_gemm<64, true, true>(int m, int n, int k);
template void run_blkq4_small_gemm<64, true, false>(int m, int n, int k);
template void run_blkq4_small_gemm<128, true, true>(int m, int n, int k);
template void run_blkq4_small_gemm<128, true, false>(int m, int n, int k);
template void run_blkq4_small_gemm<16, false, true>(int m, int n, int k);
template void run_blkq4_small_gemm<16, false, false>(int m, int n, int k);
template void run_blkq4_small_gemm<32, false, true>(int m, int n, int k);
template void run_blkq4_small_gemm<32, false, false>(int m, int n, int k);
template void run_blkq4_small_gemm<64, false, true>(int m, int n, int k);
template void run_blkq4_small_gemm<64, false, false>(int m, int n, int k);
template void run_blkq4_small_gemm<128, false, true>(int m, int n, int k);
template void run_blkq4_small_gemm<128, false, false>(int m, int n, int k);


}  // namespace test
}  // namespace cuda
}  // namespace onnxruntime
