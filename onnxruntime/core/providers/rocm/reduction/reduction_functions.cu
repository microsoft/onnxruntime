#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/rocm/reduction/reduction_functions.h"

#include <algorithm>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "core/common/common.h"
#include "core/providers/rocm/atomic/common.cuh"
#include "core/providers/rocm/cu_inc/common.cuh"
#include "core/providers/rocm/shared_inc/rocm_utils.h"
#include "core/providers/rocm/reduction/reduction_utils.cuh"

namespace onnxruntime {
namespace rocm {

namespace detail {
constexpr auto MAX_NUM_ELEMENTS_PER_THREAD = 4;
constexpr auto MAX_NUM_WARPS_PER_BLOCK = 8;
constexpr auto MAX_NUM_BLOCKS_IN_GRID_ROW = 256;
constexpr auto MAX_NUM_GRID_ROWS = 32768;

dim3 compute_block_dim(int num_cols) {
  const int x = GPU_WARP_SIZE;
  const int y = std::min(MAX_NUM_WARPS_PER_BLOCK, std::max(1, num_cols / (MAX_NUM_ELEMENTS_PER_THREAD * x)));
  return dim3(x, y);
}

std::pair<dim3, dim3> compute_grid_and_block_dims(int num_rows, int num_cols) {
  const auto block_dim = compute_block_dim(num_cols);
  const auto grid_x =
      std::min<int>(
          MAX_NUM_BLOCKS_IN_GRID_ROW,
          std::max<int>(1, num_cols / (MAX_NUM_ELEMENTS_PER_THREAD * block_dim.x * block_dim.y)));
  const auto grid_y = std::min(MAX_NUM_GRID_ROWS, num_rows);
  const dim3 grid_dim(grid_x, grid_y);
  return {grid_dim, block_dim};
}

uintptr_t round_up_to_aligned(uintptr_t original, size_t alignment) {
  assert((alignment & (alignment - 1)) == 0);
  const size_t alignment_mask = ~(alignment - 1);
  return (original + alignment - 1) & alignment_mask;
}

/**
 * call_reduce_matrix_columns() intermediate buffer layout
 *
 * Given buffer element type TBuf, the intermediate buffer layout looks like this:
 *
 * -----
 * m * num_blocks_per_row * sizeof(TBuf) bytes for block reductions per row
 * alignment padding bytes as needed
 * m * sizeof(int) bytes for block done counts per row
 * -----
 */

size_t compute_reduce_matrix_columns_intermediate_buffer_size(
    int element_size, int num_rows, int num_cols) {
  ORT_ENFORCE(element_size >= 0 && num_rows >= 0 && num_cols >= 0);

  const auto grid_dim = compute_grid_and_block_dims(num_rows, num_cols).first;

  size_t buffer_size{};

  // at the beginning, for sizing purposes, assume we are aligned
  buffer_size += static_cast<size_t>(num_rows) * grid_dim.x * element_size;

  buffer_size = round_up_to_aligned(buffer_size, alignof(int));
  buffer_size += static_cast<size_t>(num_rows) * sizeof(int);

  // add padding to give us room to align
  buffer_size += alignof(max_align_t) - 1;

  return buffer_size;
}

template <typename TBuf>
Status get_reduction_buffers(
    int num_rows, int num_cols, void* buffer, size_t buffer_size,
    TBuf*& block_reductions_buffer, int*& block_done_counts_buffer) {
  const auto grid_dim = compute_grid_and_block_dims(num_rows, num_cols).first;

  const uintptr_t begin_addr = reinterpret_cast<uintptr_t>(buffer);
  const uintptr_t block_reductions_addr =
      round_up_to_aligned(begin_addr, alignof(TBuf));
  const uintptr_t block_done_counts_buffer_addr =
      round_up_to_aligned(
          block_reductions_addr + static_cast<size_t>(num_rows) * grid_dim.x * sizeof(TBuf), alignof(int));
  const uintptr_t end_addr =
      block_done_counts_buffer_addr + static_cast<size_t>(num_rows) * sizeof(int);
  const size_t required_size = end_addr - begin_addr;

  ORT_RETURN_IF_NOT(
      required_size <= buffer_size,
      "Buffer size is too small (", buffer_size, " bytes). ",
      "At least ", required_size, " bytes are needed from the given base address (", buffer, ").");

  block_reductions_buffer = reinterpret_cast<TBuf*>(block_reductions_addr);
  block_done_counts_buffer = reinterpret_cast<int*>(block_done_counts_buffer_addr);

  return Status::OK();
}

template <typename TIn, typename TOut, typename TBuf, typename TOp, typename TFinalOp, bool DivideResultBySize>
__device__ void reduce_all(
    const int num_elements, const TIn* const input, TOut* const output,
    TBuf* const block_reductions_buffer, int* const block_done_count_buffer) {
  HIP_DYNAMIC_SHARED( unsigned char, shared_memory_bytes)
  TBuf* shared_memory = reinterpret_cast<TBuf*>(shared_memory_bytes);
  // Thread-level indices:
  // Linear index of thread in block.
  const int tid_in_block = threadIdx.y * blockDim.x + threadIdx.x;
  // Total number of threads in a 2-D block.
  const int num_threads_in_block = blockDim.x * blockDim.y;

  // Warp-level indices:
  // Warp index of thread.
  const int wid_in_block = tid_in_block / GPU_WARP_SIZE;
  // Lane index of thread.
  const int lid_in_block = tid_in_block % GPU_WARP_SIZE;
  // Warp count per block.
  const int num_warps_in_block = num_threads_in_block / GPU_WARP_SIZE;

  // Grid-level indices:
  // Linear index of block in grid row.
  const int bid_in_grid_row = blockIdx.x;
  // Linear index of thread in grid row.
  const int tid_in_grid_row = bid_in_grid_row * (blockDim.x * blockDim.y) + tid_in_block;
  // Total number of blocks in a grid row.
  const int num_blocks_in_grid_row = gridDim.x;
  // Total number of threads in a grid row with 2-D blocks.
  const int num_threads_in_grid_row = num_blocks_in_grid_row * num_threads_in_block;

  const auto write_result = [&output, &num_elements](const TOut result) {
    // Compilation time if-else branch controlled by template argument can be
    // optimized out, so there will be no branch in real computation phase.
    if (DivideResultBySize) {
      output[0] = TFinalOp()(result / TOut(num_elements));
    } else {
      output[0] = TFinalOp()(result);
    }
  };

  // Thread-level reduction (storage change: global memory -> register).
  // One thread reduces MAX_NUM_ELEMENTS_PER_THREAD elements to a thread register
  // in one iteration.
  TBuf value = 0;
  for (int id = tid_in_grid_row; id < num_elements; id += MAX_NUM_ELEMENTS_PER_THREAD * num_threads_in_grid_row) {
    TIn v[MAX_NUM_ELEMENTS_PER_THREAD];

#pragma unroll
    for (int i = 0; i < MAX_NUM_ELEMENTS_PER_THREAD; i++) {
      const int offset = id + i * num_threads_in_grid_row;
      if (offset < num_elements) {
        v[i] = input[offset];
      }
    }

#pragma unroll
    for (int i = 0; i < MAX_NUM_ELEMENTS_PER_THREAD; i++) {
      const int offset = id + i * num_threads_in_grid_row;
      if (offset < num_elements) {
        value += TOp()(TBuf(v[i]));
      }
    }
  }

  __syncthreads();

  // Warp-level reduction (storage change: register -> register).
  // The values in a warp will be summed up to a scalar. After warp-level
  // reduction, each block holds num_warps_in_block values in the shared memory.
#pragma unroll
  for (int stride = GPU_WARP_SIZE / 2; stride > 0; stride /= 2) {
    value += WARP_SHFL_DOWN(value, stride);
  }

  // Return early if only one warp is used for reduction.
  // Given a fixed amount of threads, we prefer threads over warps over blocks so that we never have cases such as
  // 1. two blocks and each of them has only 1 warp (32 threads).
  // 2. two warps and each of them has only 2 threads.
  if (num_warps_in_block == 1) {
    if (tid_in_grid_row == 0) {
      write_result(value);
    }
    return;
  }

  if (lid_in_block == 0) {
    shared_memory[wid_in_block] = value;
  }

  __syncthreads();

  // Block-level reduction (storage change: shared memory -> global memory).
  // The values in a block will be summed up to a scalar.
  // Note that the values are stored in the shared memory.
  // Here we assume that the size of shared_memory is smaller
  // than num_warps_in_block, so we just keep halving the number
  // of threads in each iteration. Our assumption is always true because
  // the size of shared_memory equals to the number of warps.
#pragma unroll
  for (int stride = MAX_NUM_WARPS_PER_BLOCK / 2; stride > 0; stride /= 2) {
    if (tid_in_block + stride < num_warps_in_block) {
      shared_memory[tid_in_block] += shared_memory[tid_in_block + stride];
    }
    __syncthreads();
  }

  // Return early if only one block is used for reduction.
  if (num_blocks_in_grid_row == 1) {
    if (tid_in_grid_row == 0) {
      write_result(shared_memory[0]);
    }
    return;
  }

  if (tid_in_block == 0) {
    block_reductions_buffer[bid_in_grid_row] = shared_memory[0];
  }

  __threadfence();
  __syncthreads();

  // Grid-level reduction. We use the last block to sum up values
  // stored in the global block_reductions_buffer.
  __shared__ bool is_last_block_done;

  if (tid_in_block == 0) {
    const int count = atomicAdd(block_done_count_buffer, 1);
    is_last_block_done = (count == (num_blocks_in_grid_row - 1));
  }

  // All threads in each block see if they belong the last active block
  // (i.e., the value of is_last_block_done).
  __syncthreads();

  // Only the block which saw that count equals to num_blocks_in_grid_row - 1 can
  // enter the following block.
  if (is_last_block_done) {
    const int pow2_bound = least_pow2_bound(num_blocks_in_grid_row);
    for (int stride = pow2_bound / 2; stride > 0; stride /= 2) {
      if (tid_in_block < stride && tid_in_block + stride < num_blocks_in_grid_row) {
        block_reductions_buffer[tid_in_block] += block_reductions_buffer[tid_in_block + stride];
      }
      __syncthreads();
    }

    // The first thread in the last block assigns the final output.
    if (tid_in_block == 0) {
      write_result(block_reductions_buffer[0]);
    }
  }
}

template <typename TIn, typename TOut, typename TBuf, typename TOp, typename TFinalOp, bool DivideResultBySize>
__global__ void reduce_matrix_columns_kernel(
    const int num_rows, const int num_cols, const TIn* const input, TOut* const output,
    TBuf* const block_reductions_buffer, int* const block_done_counts_buffer) {
  const int num_blocks_in_grid_row = gridDim.x;
  const int row_id_in_grid = blockIdx.y;
  const int num_grid_rows = gridDim.y;

  // one row per iteration
  // row_id is int64_t to avoid int overflow in offset calculations
  for (int64_t row_id = row_id_in_grid; row_id < num_rows; row_id += num_grid_rows) {
    const TIn* const row_data = input + row_id * num_cols;
    TOut* const row_output = output + row_id;
    TBuf* const row_block_reductions_buffer = block_reductions_buffer + row_id * num_blocks_in_grid_row;
    int* const row_block_done_counts_buffer = block_done_counts_buffer + row_id;

    reduce_all<TIn, TOut, TBuf, TOp, TFinalOp, DivideResultBySize>(
        num_cols, row_data, row_output,
        row_block_reductions_buffer, row_block_done_counts_buffer);
  }
}

template <typename TIn, typename TOut, typename TOp, typename TFinalOp, bool DivideResultBySize>
Status call_reduce_matrix_columns(
    const TIn* input, TOut* output, const int num_rows, const int num_cols, void* buffer, size_t buffer_size) {
  ORT_ENFORCE(num_rows >= 0 && num_cols >= 0);

  using TBuf = AccumulationType_t<TIn>;

  const auto grid_and_block_dims = compute_grid_and_block_dims(num_rows, num_cols);
  const dim3& grid_dim = grid_and_block_dims.first;
  const dim3& block_dim = grid_and_block_dims.second;

  TBuf* block_reductions_buffer;
  int* block_done_counts_buffer;
  ORT_RETURN_IF_ERROR(get_reduction_buffers(
      num_rows, num_cols, buffer, buffer_size,
      block_reductions_buffer, block_done_counts_buffer));

  // If more than one block is used per grid row, then inter-block reduction is needed.
  if (grid_dim.x > 1) {
    HIP_RETURN_IF_ERROR(hipMemsetAsync(block_done_counts_buffer, 0, num_rows * sizeof(int)));
  }

  const int shared_mem_size = sizeof(TBuf) * block_dim.x * block_dim.y / GPU_WARP_SIZE;
  hipLaunchKernelGGL(HIP_KERNEL_NAME(reduce_matrix_columns_kernel<TIn, TOut, TBuf, TOp, TFinalOp, DivideResultBySize>), dim3(grid_dim), dim3(block_dim), shared_mem_size, 0, 
          num_rows, num_cols, input, output, block_reductions_buffer, block_done_counts_buffer);

  return Status::OK();
}
}  // namespace detail

template <typename TIn, typename TOut>
Status reduce_sum(
    const TIn* input, TOut* output, int size, void* buffer, size_t buffer_size) {
  return detail::call_reduce_matrix_columns<TIn, TOut, Identity, Identity, false>(
      input, output, 1, size, buffer, buffer_size);
}

template <typename TIn, typename TOut>
Status reduce_square_sum(
    const TIn* input, TOut* output, int size, void* buffer, size_t buffer_size) {
  return detail::call_reduce_matrix_columns<TIn, TOut, Square, Identity, false>(
      input, output, 1, size, buffer, buffer_size);
}

template <typename TIn, typename TOut>
Status reduce_l2_norm(
    const TIn* input, TOut* output, int size, void* buffer, size_t buffer_size) {
  return detail::call_reduce_matrix_columns<TIn, TOut, Square, Sqrt, false>(
      input, output, 1, size, buffer, buffer_size);
}

template <typename TIn, typename TOut>
Status reduce_mean(
    const TIn* input, TOut* output, int size, void* buffer, size_t buffer_size) {
  return detail::call_reduce_matrix_columns<TIn, TOut, Identity, Identity, true>(
      input, output, 1, size, buffer, buffer_size);
}

#define INSTANTIATE_REDUCE_SUM(TIn, TOut) \
  template Status reduce_sum<TIn, TOut>(const TIn* input, TOut* output, int size, void* buffer, size_t buffer_size)
INSTANTIATE_REDUCE_SUM(half, float);
INSTANTIATE_REDUCE_SUM(float, float);
INSTANTIATE_REDUCE_SUM(double, double);
#undef INSTANTIATE_REDUCE_SUM

#define INSTANTIATE_REDUCE_SQUARE_SUM(TIn, TOut) \
  template Status reduce_square_sum<TIn, TOut>(const TIn* input, TOut* output, int size, void* buffer, size_t buffer_size)
INSTANTIATE_REDUCE_SQUARE_SUM(half, float);
INSTANTIATE_REDUCE_SQUARE_SUM(float, float);
INSTANTIATE_REDUCE_SQUARE_SUM(double, double);
#undef INSTANTIATE_REDUCE_SQUARE_SUM

#define INSTANTIATE_REDUCE_L2_NORM(TIn, TOut) \
  template Status reduce_l2_norm<TIn, TOut>(const TIn* input, TOut* output, int size, void* buffer, size_t buffer_size)
INSTANTIATE_REDUCE_L2_NORM(half, float);
INSTANTIATE_REDUCE_L2_NORM(float, float);
INSTANTIATE_REDUCE_L2_NORM(double, double);
#undef INSTANTIATE_REDUCE_L2_NORM

#define INSTANTIATE_REDUCE_MEAN(TIn, TOut) \
  template Status reduce_mean<TIn, TOut>(const TIn* input, TOut* output, int size, void* buffer, size_t buffer_size)
INSTANTIATE_REDUCE_MEAN(half, float);
INSTANTIATE_REDUCE_MEAN(float, float);
INSTANTIATE_REDUCE_MEAN(double, double);
#undef INSTANTIATE_REDUCE_MEAN

namespace detail {
template <typename TIn, typename TOut, typename TBuf>
__global__ void reduce_matrix_rows_kernel(const TIn* input, TOut* output, int m, int n) {
  constexpr int x_load_count_per_thread = 1;
  constexpr int y_load_count_per_thread = 4;
  const int t_count_x_in_grid = blockDim.x * gridDim.x;
  const int t_count_y_in_grid = blockDim.y * gridDim.y;
  const int x_grid_stride = t_count_x_in_grid * x_load_count_per_thread;
  const int y_grid_stride = t_count_y_in_grid * y_load_count_per_thread;
  const int tid_x_in_grid = threadIdx.x + blockDim.x * blockIdx.x;
  const int tid_y_in_grid = threadIdx.y + blockDim.y * blockIdx.y;
  const int tid_in_block = threadIdx.x + blockDim.x * threadIdx.y;

  // Shape is blockDim.y-by-blockDim.x and element type is TBuf.
  HIP_DYNAMIC_SHARED( unsigned char, shared_memory_bytes)
  TBuf* shared_memory = reinterpret_cast<TBuf*>(shared_memory_bytes);

  // to prevent int overflow in index calculation for input size m*n
  const int64_t n_int64 = static_cast<int64_t>(n);

  for (int col = tid_x_in_grid; col < n; col += x_grid_stride) {
    shared_memory[tid_in_block] = TBuf(0.0f);
    TBuf sum = TBuf(0.0f);
    // This loops load multiple blockDim.y-by-blockDim.x sub-tensors from the input.
    for (int row = tid_y_in_grid; row < m; row += y_grid_stride) {
      // Thread-level reduction. Each thread loads y_load_count_per_thread values
      // and aggregrate them.
#pragma unroll(y_load_count_per_thread)
      for (int row_inner = 0; row_inner < y_load_count_per_thread; ++row_inner) {
        int row_final = row + row_inner * t_count_y_in_grid;
        int col_final = col;
        if (row_final < m && col_final < n) {
          sum += TBuf(input[row_final * n_int64 + col_final]);
        }
      }
    }
    // Write thread-level reduction result into shared memory.
    shared_memory[tid_in_block] = sum;

    // Wait all threads to finish their thread-level reductions.
    __syncthreads();

// This loop conducts reduction on elements stored in shared memory.
// Each block reduces blockDim.y-by-blockDim.x tensor to 1-by-blockDim.x tensor.
#pragma unroll(4)
    for (int stride = blockDim.y / 2; stride > 0; stride /= 2) {
      if (threadIdx.y < stride) {
        shared_memory[tid_in_block] += shared_memory[tid_in_block + stride * blockDim.x];
      }
      __syncthreads();
    }

    if (threadIdx.y == 0) {
      atomic_add(output + col, TOut(shared_memory[threadIdx.x]));
    }
  }
}

template <typename TIn, typename TOut, typename TBuf>
Status call_reduce_matrix_rows(const TIn* input, TOut* output, int m, int n, bool reset_initial_output) {
  ORT_ENFORCE(m >= 0 && n >= 0);

  if (reset_initial_output) {
    HIP_RETURN_IF_ERROR(hipMemsetAsync(output, 0, n * sizeof(TOut)));
  }

  constexpr int max_num_threads_in_block = 512;
  constexpr int max_num_blocks_in_grid = 512;
  constexpr int load_count_per_thread = 4;

  const int block_x_dim = least_pow2_bound(std::max(1, std::min(n, GPU_WARP_SIZE)));
  const int block_y_dim = least_pow2_bound(std::max(1, std::min(max_num_threads_in_block / block_x_dim, m / load_count_per_thread)));
  const int grid_x_dim = std::max(1, std::min(n / block_x_dim, max_num_blocks_in_grid));
  const int grid_y_dim = std::max(1, std::min(max_num_blocks_in_grid / grid_x_dim, m / block_y_dim / 4));

  const dim3 grid(grid_x_dim, grid_y_dim, 1);
  const dim3 block(block_x_dim, block_y_dim, 1);

  hipLaunchKernelGGL(HIP_KERNEL_NAME(reduce_matrix_rows_kernel<TIn, TOut, TBuf>), dim3(grid), dim3(block), block.y * block.x * sizeof(TBuf), 0, 
      input, output, m, n);

  return Status::OK();
}
}  // namespace detail

template <typename TIn, typename TOut>
Status reduce_matrix_rows(const TIn* input, TOut* output, int m, int n, bool reset_initial_output) {
  using TBuf = AccumulationType_t<TIn>;
  return detail::call_reduce_matrix_rows<TIn, TOut, TBuf>(input, output, m, n, reset_initial_output);
}

#define INSTANTIATE_REDUCE_MATRIX_ROWS(T) \
  template Status reduce_matrix_rows<T, T>(const T* input, T* output, int m, int n, bool reset_initial_output)
INSTANTIATE_REDUCE_MATRIX_ROWS(half);
INSTANTIATE_REDUCE_MATRIX_ROWS(float);
INSTANTIATE_REDUCE_MATRIX_ROWS(double);
#undef INSTANTIATE_REDUCE_MATRIX_ROWS

template <typename TIn, typename TOut>
Status reduce_matrix_columns(const TIn* input, TOut* output, int m, int n, void* buffer, size_t buffer_size) {
  return detail::call_reduce_matrix_columns<TIn, TOut, Identity, Identity, false>(
      input, output, m, n, buffer, buffer_size);
}

#define INSTANTIATE_REDUCE_MATRIX_COLUMNS(T) \
  template Status reduce_matrix_columns<T, T>(const T* input, T* output, int m, int n, void* buffer, size_t buffer_size)
INSTANTIATE_REDUCE_MATRIX_COLUMNS(half);
INSTANTIATE_REDUCE_MATRIX_COLUMNS(float);
INSTANTIATE_REDUCE_MATRIX_COLUMNS(double);
#undef INSTANTIATE_REDUCE_MATRIX_COLUMNS

}  // namespace rocm
}  // namespace onnxruntime
