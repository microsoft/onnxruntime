#include "hip/hip_runtime.h"
/**
* Copyright (c) 2016-present, Facebook, Inc.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

/* Modifications Copyright (c) Microsoft. */

// The code below is mostly copied from Pytorch PersistentSoftmax.cuh
#include "hip/hip_runtime.h"
#include "core/providers/rocm/cu_inc/common.cuh"
#include "core/providers/rocm/math/softmax_impl.cuh"
#include "core/providers/rocm/math/softmax.h"

#include <limits>

namespace onnxruntime {
namespace rocm {

// The softmax_warp_* methods perform softmax forward and backward propagation on samples spanning the fast dimension.
// Each sample contains element_count scalar elements. element_count can be any integer value <= 1024.
// The template arguments have the following meaning:
// One "WARP" works on one "BATCH". One "BATCH" contains "WARP_BATCH" samples.
// WARP_BATCH is equal to 1 when element_count is large, and > 1 when element_count is small.
// A "WARP" contains "GPU_WARP_SIZE" threads, these treads are guaranteed to belong to the same warp.
// This is important because it means only __shfl_ instructions are required for reductions.
// Note that this means WARP_SIZE must be a power of two and <= architecture warp size.
// ROCM warp size is 32 for all existing GPU architecures, but there is no guarantee this will not change for future arch.
// is_log_softmax is a flag indicating whether SoftMax or LogSoftMax should be computed.
// The template can be instantiated with any floating point type for the type arguments input_t, output_t and acc_t.
// This allows SoftMax to be fused with a cast immediately following the SoftMax.
// For instance:
// input_t=half,  acc_t=float, output_t=half  => read half tensor, float accumulators, write half tensor.
// input_t=half,  acc_t=float, output_t=float => read half tensor, float accumulators, write float tensor.
// input_t_float, acc_t=float, output_t=half  => read float tensor, float accumulators, write half tensor.

template <typename input_t, typename output_t, typename acc_t, int log2_elements, bool is_log_softmax>
__global__ void softmax_warp_forward(output_t* dst, const input_t* src, int batch_size, int stride, int element_count) {
  // WARP_SIZE and WARP_BATCH must match the return values batches_per_warp and warp_size of method warp_softmax_forward_kernel.
  constexpr int next_power_of_two = 1 << log2_elements;
  constexpr int WARP_SIZE = (next_power_of_two < GPU_WARP_SIZE) ? next_power_of_two : GPU_WARP_SIZE;
  constexpr int WARP_ITERATIONS = next_power_of_two / WARP_SIZE;
  // constexpr int WARP_BATCH = (next_power_of_two <= 128) ? 2 : 1;
  constexpr int WARP_BATCH = 1;

  int first_batch = (blockDim.y * blockIdx.x + threadIdx.y) * WARP_BATCH;

  // batch_size might not be a multiple of WARP_BATCH. Check how
  // many batches have to computed within this WARP.
  int local_batches = batch_size - first_batch;
  if (local_batches > WARP_BATCH)
    local_batches = WARP_BATCH;

  // there might be multiple batches per warp. compute the index within the batch
  int local_idx = threadIdx.x;

  src += first_batch * stride + local_idx;
  dst += first_batch * stride + local_idx;

  // The nested loops over WARP_BATCH and then WARP_ITERATIONS can be simplified to one loop,
  // but I think doing so would obfuscate the logic of the algorithm, thus I chose to keep
  // the nested loops.
  // This should have no impact on performance because the loops are unrolled anyway.

  // load data from global memory
  acc_t elements[WARP_BATCH][WARP_ITERATIONS];
  for (int i = 0; i < WARP_BATCH; ++i) {
    int batch_element_count = (i >= local_batches) ? 0 : element_count;
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      int element_index = local_idx + it * WARP_SIZE;
      if (element_index < batch_element_count) {
        elements[i][it] = src[i * element_count + it * WARP_SIZE];
      } else {
        elements[i][it] = -std::numeric_limits<acc_t>::infinity();
      }
    }
  }

  // compute max_value
  acc_t max_value[WARP_BATCH];
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
    max_value[i] = elements[i][0];
#pragma unroll
    for (int it = 1; it < WARP_ITERATIONS; ++it) {
      max_value[i] = (max_value[i] > elements[i][it]) ? max_value[i] : elements[i][it];
    }
  }
  warp_reduce<acc_t, WARP_BATCH, WARP_SIZE, Max>(max_value);

  acc_t sum[WARP_BATCH]{0.0f};
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
#pragma unroll
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      if (is_log_softmax) {
        sum[i] += expf((float)(elements[i][it] - max_value[i]));
      } else {
        elements[i][it] = expf((float)(elements[i][it] - max_value[i]));
        sum[i] += elements[i][it];
      }
    }
  }
  warp_reduce<acc_t, WARP_BATCH, WARP_SIZE, Add>(sum);

// store result
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
    if (i >= local_batches)
      break;
    if (is_log_softmax) sum[i] = max_value[i] + logf((float)(sum[i]));
#pragma unroll
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      int element_index = local_idx + it * WARP_SIZE;
      if (element_index < element_count) {
        if (is_log_softmax) {
          dst[i * element_count + it * WARP_SIZE] = elements[i][it] - sum[i];
        } else {
          dst[i * element_count + it * WARP_SIZE] = elements[i][it] / sum[i];
        }
      } else {
        break;
      }
    }
  }
}

template <typename input_t, typename output_t, typename acc_t, bool is_log_softmax>
void dispatch_softmax_forward(output_t* dst, const input_t* src, int softmax_elements, int softmax_elements_stride, int batch_count) {
  if (softmax_elements == 0) {
    return;
  } else {
    int log2_elements = log2_ceil(softmax_elements);
    const int next_power_of_two = 1 << log2_elements;

    // This value must match the WARP_SIZE constexpr value computed inside softmax_warp_forward.
    int warp_size = (next_power_of_two < GPU_WARP_SIZE) ? next_power_of_two : GPU_WARP_SIZE;

    // This value must match the WARP_BATCH constexpr value computed inside softmax_warp_forward.
    // int batches_per_warp = (next_power_of_two <= 128) ? 2 : 1;
    int batches_per_warp = 1;

    // use 128 threads per block to maximimize gpu utilization
    constexpr int threads_per_block = 256;

    int warps_per_block = (threads_per_block / warp_size);
    int batches_per_block = warps_per_block * batches_per_warp;
    int blocks = (batch_count + batches_per_block - 1) / batches_per_block;
    dim3 threads(warp_size, warps_per_block, 1);
    // Launch code would be more elegant if C++ supported FOR CONSTEXPR
    switch (log2_elements) {
      case 0:  // 1
        hipLaunchKernelGGL(HIP_KERNEL_NAME(softmax_warp_forward<input_t, output_t, acc_t, 0, is_log_softmax>), dim3(blocks), dim3(threads), 0, 0, dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 1:  // 2
        hipLaunchKernelGGL(HIP_KERNEL_NAME(softmax_warp_forward<input_t, output_t, acc_t, 1, is_log_softmax>), dim3(blocks), dim3(threads), 0, 0, dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 2:  // 4
        hipLaunchKernelGGL(HIP_KERNEL_NAME(softmax_warp_forward<input_t, output_t, acc_t, 2, is_log_softmax>), dim3(blocks), dim3(threads), 0, 0, dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 3:  // 8
        hipLaunchKernelGGL(HIP_KERNEL_NAME(softmax_warp_forward<input_t, output_t, acc_t, 3, is_log_softmax>), dim3(blocks), dim3(threads), 0, 0, dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 4:  // 16
        hipLaunchKernelGGL(HIP_KERNEL_NAME(softmax_warp_forward<input_t, output_t, acc_t, 4, is_log_softmax>), dim3(blocks), dim3(threads), 0, 0, dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 5:  // 32
        hipLaunchKernelGGL(HIP_KERNEL_NAME(softmax_warp_forward<input_t, output_t, acc_t, 5, is_log_softmax>), dim3(blocks), dim3(threads), 0, 0, dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 6:  // 64
        hipLaunchKernelGGL(HIP_KERNEL_NAME(softmax_warp_forward<input_t, output_t, acc_t, 6, is_log_softmax>), dim3(blocks), dim3(threads), 0, 0, dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 7:  // 128
        hipLaunchKernelGGL(HIP_KERNEL_NAME(softmax_warp_forward<input_t, output_t, acc_t, 7, is_log_softmax>), dim3(blocks), dim3(threads), 0, 0, dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 8:  // 256
        hipLaunchKernelGGL(HIP_KERNEL_NAME(softmax_warp_forward<input_t, output_t, acc_t, 8, is_log_softmax>), dim3(blocks), dim3(threads), 0, 0, dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 9:  // 512
        hipLaunchKernelGGL(HIP_KERNEL_NAME(softmax_warp_forward<input_t, output_t, acc_t, 9, is_log_softmax>), dim3(blocks), dim3(threads), 0, 0, dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 10:  // 1024
        hipLaunchKernelGGL(HIP_KERNEL_NAME(softmax_warp_forward<input_t, output_t, acc_t, 10, is_log_softmax>), dim3(blocks), dim3(threads), 0, 0, dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      default:
        break;
    }
  }
}

#define SPECIALIZED_SOFTMAX_IMPL(input_t, output_t, acc_t) \
template void dispatch_softmax_forward<input_t, output_t, acc_t, false>(output_t * dst, const input_t* src, int softmax_elements, int softmax_elements_stride, int batch_count); \
template void dispatch_softmax_forward<input_t, output_t, acc_t, true>(output_t * dst, const input_t* src, int softmax_elements, int softmax_elements_stride, int batch_count);

SPECIALIZED_SOFTMAX_IMPL(float, float, float)
SPECIALIZED_SOFTMAX_IMPL(half, half, float)
SPECIALIZED_SOFTMAX_IMPL(double, double, double)

}
}