#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/rocm/cu_inc/common.cuh"
#include "core/providers/rocm/tensor/transpose_impl.h"

namespace onnxruntime {
namespace rocm {

template <typename T>
// __global__ void _TransposeKernel(int32_t shape_rank, const TArray<int64_t> input_strides,
//                                  const T* __restrict__ input_data, const TArray<fast_divmod> output_strides, T* __restrict__ output_data, HIP_LONG N) {
__global__ void _TransposeKernel(int32_t shape_rank, const int64_t* input_strides,
  const T* __restrict__ input_data, const fast_divmod* output_strides, T* __restrict__ output_data, HIP_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  HIP_LONG input_index = 0;
  HIP_LONG output_index = id;

  // #pragma unroll
  // for (auto dim = 0; dim < input_strides.GetCapacity(); ++dim) {
  //   if (dim >= shape_rank) {
  //     break;
  //   }
  for (auto dim = 0; dim < shape_rank; ++dim) {
    int out_coord, r;
    output_strides[dim].divmod(output_index, out_coord, r);
    output_index = r;
    input_index += input_strides[dim] * out_coord;
  }
  output_data[id] = input_data[input_index];
}

Status TransposeImpl(size_t element_size, int32_t shape_rank, const int64_t* input_strides,
  const void* input_data, const fast_divmod* fdm_output_strides, void* output_data, int64_t N) {
// Status TransposeImpl(size_t element_size, int32_t shape_rank, const TArray<int64_t>& input_strides,
//                      const void* input_data, const TArray<fast_divmod>& fdm_output_strides, void* output_data, int64_t N) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  switch (element_size) {
    case sizeof(int8_t):
      hipLaunchKernelGGL(_TransposeKernel<int8_t>, dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, 
          shape_rank, input_strides,
          reinterpret_cast<const ToHipType<int8_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToHipType<int8_t>::MappedType*>(output_data),
          N);
      break;
    case sizeof(int16_t):
      hipLaunchKernelGGL(_TransposeKernel<int16_t>, dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, 
          shape_rank, input_strides,
          reinterpret_cast<const ToHipType<int16_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToHipType<int16_t>::MappedType*>(output_data),
          N);
      break;
    case sizeof(int32_t):
      hipLaunchKernelGGL(_TransposeKernel<int32_t>, dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, 
          shape_rank, input_strides,
          reinterpret_cast<const ToHipType<int32_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToHipType<int32_t>::MappedType*>(output_data),
          N);
      break;
    case sizeof(int64_t):
      hipLaunchKernelGGL(_TransposeKernel<int64_t>, dim3(blocksPerGrid), dim3(GridDim::maxThreadsPerBlock), 0, 0, 
          shape_rank, input_strides,
          reinterpret_cast<const ToHipType<int64_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToHipType<int64_t>::MappedType*>(output_data),
          N);
      break;
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for transpose on HIP. Element size was ",
                             element_size);
  }

  return Status::OK();
}

}  // namespace rocm
}  // namespace onnxruntime
