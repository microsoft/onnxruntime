#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

// Thrust code needs to be compiled with nvcc
#include <memory>
#include "core/providers/cuda/shared_inc/cuda_utils.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "cudnn_common.h"

namespace onnxruntime {
namespace cuda {

template <typename T, int NumThreadsPerBlock, int NumElementsPerThread>
__global__ void _Fill(
    T* output_data,
    T val,
    CUDA_LONG N) {
  CUDA_LONG id = NumElementsPerThread * blockDim.x * blockIdx.x + threadIdx.x;

#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      output_data[id] = val;
      id += blockDim.x;
    }
  }
}

template <typename T>
void Fill(T* output, T value, int64_t count) {
  int blocksPerGrid = static_cast<int>(CeilDiv(count, GridDim::maxThreadsPerBlock * GridDim::maxElementsPerThread));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _Fill<T, GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread>
      <<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(output, value, N);
}
template <typename T>
class ConstantBufferImpl : public IConstantBuffer<T> {
 public:
  ConstantBufferImpl(T val) : val_(val), buffer_(nullptr), count_(0) {
  }
  ~ConstantBufferImpl() {
    if (buffer_)
      hipFree(buffer_);
  }

  virtual const T* GetBuffer(size_t count) {
    if (count > count_) {
      if (buffer_) {
        hipFree(buffer_);
        buffer_ = nullptr;
      }
      CUDA_CALL_THROW(hipMalloc(&buffer_, count * sizeof(T)));
      count_ = count;

      Fill(buffer_, val_, count);
    }
    return buffer_;
  }

 private:
  T* buffer_;
  size_t count_;
  T val_;
};

template <typename T>
std::unique_ptr<IConstantBuffer<T>> CreateConstantOnes() {
  return onnxruntime::make_unique<ConstantBufferImpl<T>>(Consts<T>::One);
}

template std::unique_ptr<IConstantBuffer<float>> CreateConstantOnes<float>();
template std::unique_ptr<IConstantBuffer<double>> CreateConstantOnes<double>();
template std::unique_ptr<IConstantBuffer<half>> CreateConstantOnes<half>();
#if CUDA_VERSION >= 11000 && (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
template std::unique_ptr<IConstantBuffer<hip_bfloat16>> CreateConstantOnes<hip_bfloat16>();
#endif

#define SPECIALIZED_FILL(T) \
  template void Fill<T>(T * output, T value, int64_t count);

SPECIALIZED_FILL(int8_t)
SPECIALIZED_FILL(int16_t)
SPECIALIZED_FILL(int32_t)
SPECIALIZED_FILL(int64_t)
SPECIALIZED_FILL(float)
SPECIALIZED_FILL(double)
SPECIALIZED_FILL(__half)
#if CUDA_VERSION >= 11000 && (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
SPECIALIZED_FILL(hip_bfloat16)
#endif

}  // namespace cuda
}  // namespace onnxruntime
