#include "hip/hip_runtime.h"
//
// Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
// Licensed under the MIT license. See LICENSE.md file in the project root for full license information.
//

// Make generic operators for floating point types
/* This file contains:
   Generalized library calls
   kernels to be called for not supported data type
*/
// NV_TODO: optimize speed -- pass things needed in, optimize kernel speed, add half2
// NV_TODO: investigate cub support for half

#include "core/providers/cuda/cu_inc/common.cuh"
#include <hiprand/hiprand_kernel.h>

#define TRANS_TILE_DIM 32
#define BLOCK_ROWS 8
#define COPY_TILE_DIM 1024
#define COPY_BLOCK_DIM 256

// kernel(s) for half functions with no library support
namespace {

// TODO - refactor the function with similar logic in Transpose3DKernel using 16x16 Tile
__global__ void transposeNoOverlap(half* odata, const half* idata, const int m, const int n) {
  __shared__ half tile[TRANS_TILE_DIM][TRANS_TILE_DIM + 1];

  int x = blockIdx.x * TRANS_TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TRANS_TILE_DIM + threadIdx.y;

  if (x < m) {
    for (int j = 0; j < TRANS_TILE_DIM; j += BLOCK_ROWS) {
      if (j >= (n - y)) continue;
      tile[threadIdx.y + j][threadIdx.x] = idata[(y + j) * m + x];
    }
  }

  __syncthreads();

  x = blockIdx.y * TRANS_TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TRANS_TILE_DIM + threadIdx.y;

  if (x >= n) return;

  for (int j = 0; j < TRANS_TILE_DIM; j += BLOCK_ROWS) {
    if ((y + j) >= m) return;
    odata[(y + j) * n + x] = tile[threadIdx.x][threadIdx.y + j];
  }
}
// set up hiprand state, need to move up layer to remove calling for each generate call
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
  hiprand_init(seed, 0, 0, state);
}

__global__ void GenerateUniformHalf(hiprandState* state, half* result, int n) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id >= n) return;

  hiprandState localState = *state;

  float x;
  skipahead(id, &localState);
  x = hiprand_uniform(&localState);

  result[id] = x;
  if (id == n - 1) *state = localState;
}

__global__ void GenerateNormalHalf(hiprandState* state, half* result, int n, half mean, half stddev) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id >= n) return;

  hiprandState localState = *state;

  float x;
  skipahead(id, &localState);
  x = hiprand_normal(&localState);

  result[id] = (float)mean + (float)stddev * x;
  if (id == n - 1) *state = localState;
}

// kernels can convert matrix between half and float. speed currently not optimized, may need to add half2
/*
__global__ void copyHalf2Float(float *odata, const half *idata, const int n)
{
    float tmp[COPY_TILE_DIM/COPY_BLOCK_DIM];

    int x = blockIdx.x * COPY_TILE_DIM + threadIdx.x;

    for (int j = 0; j < COPY_TILE_DIM/COPY_BLOCK_DIM; j++)
        tmp[j] = (float) idata[x + j*COPY_BLOCK_DIM];

    for (int j = 0; j < COPY_TILE_DIM/COPY_BLOCK_DIM; j++)
        if(x + j*COPY_BLOCK_DIM < n) odata[x + j*COPY_BLOCK_DIM] = tmp[j];
}

__global__ void copyFloat2Half(half *odata, const float *idata, const int n)
{
    float tmp[COPY_TILE_DIM/COPY_BLOCK_DIM];

    int x = blockIdx.x * COPY_TILE_DIM + threadIdx.x;

    for (int j = 0; j < COPY_TILE_DIM/COPY_BLOCK_DIM; j++)
        tmp[j] = idata[x + j*COPY_BLOCK_DIM];

    for (int j = 0; j < COPY_TILE_DIM/COPY_BLOCK_DIM; j++)
        if(x + j*COPY_BLOCK_DIM < n) odata[x + j*COPY_BLOCK_DIM] = tmp[j];
}
*/

__global__ void CopyVectorHalf(const half* x, int incx, half* y, int incy, int n) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id >= n) return;
  y[id * incy] = x[id * incx];
}

}  // namespace

hipblasStatus_t cublasTransposeHelper(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int m, int n, const half*, const half* A, int, const half*, const half*, int, half* C, int) {
  if (C != A) {
    dim3 dimGrid((n + TRANS_TILE_DIM - 1) / TRANS_TILE_DIM, (m + TRANS_TILE_DIM - 1) / TRANS_TILE_DIM, 1);
    dim3 dimBlock(TRANS_TILE_DIM, BLOCK_ROWS, 1);

    transposeNoOverlap<<<dimGrid, dimBlock>>>(C, A, n, m);
  } else {
    return HIPBLAS_STATUS_NOT_SUPPORTED;
  }
  return HIPBLAS_STATUS_SUCCESS;
}

hipblasStatus_t cublasCopyHelper(hipblasHandle_t, int n, const half* x, int incx, half* y, int incy) {
  dim3 dimGrid((unsigned int)(n + COPY_BLOCK_DIM - 1) / COPY_BLOCK_DIM, 1, 1);
  dim3 dimBlock(COPY_BLOCK_DIM, 1, 1);
  CopyVectorHalf<<<dimGrid, dimBlock>>>(x, incx, y, incy, n);
  return HIPBLAS_STATUS_SUCCESS;
}

hiprandStatus_t curandGenerateUniformHelper(hiprandGenerator_t, half* outputPtr, size_t num) {
  hiprandState* devStates;
  hipMalloc((void**)&devStates, sizeof(hiprandState));
  setup_state<<<1, 1>>>(devStates, time(NULL));  // What does hiprandGenerateUniform actually doing? should also pass in state here

  dim3 dimGrid((unsigned int)(num + COPY_BLOCK_DIM - 1) / COPY_BLOCK_DIM, 1, 1);
  dim3 dimBlock(COPY_BLOCK_DIM, 1, 1);
  GenerateUniformHalf<<<dimGrid, dimBlock>>>(devStates, outputPtr, (int)num);

  return (hiprandStatus_t)0;
}

hiprandStatus_t curandGenerateNormalHelper(hiprandGenerator_t, half* outputPtr, size_t n, half mean, half stddev) {
  hiprandState* devStates;
  hipMalloc((void**)&devStates, sizeof(hiprandState));
  setup_state<<<1, 1>>>(devStates, time(NULL));  // What does hiprandGenerateUniform actually doing? should also pass in state here

  dim3 dimGrid((unsigned int)(n + COPY_BLOCK_DIM - 1) / COPY_BLOCK_DIM, 1, 1);
  dim3 dimBlock(COPY_BLOCK_DIM, 1, 1);
  GenerateNormalHalf<<<dimGrid, dimBlock>>>(devStates, outputPtr, (int)n, mean, stddev);

  return (hiprandStatus_t)0;
}
