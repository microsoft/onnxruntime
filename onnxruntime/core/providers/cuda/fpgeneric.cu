#include "hip/hip_runtime.h"
//
// Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
// Licensed under the MIT license. See LICENSE.md file in the project root for full license information.
//

// Make generic operators for floating point types
/* This file contains:
   Generalized library calls
   kernels to be called for not supported data type
*/
// NV_TODO: optimize speed -- pass things needed in, optimize kernel speed, add half2
// NV_TODO: investigate cub support for half

#include "core/providers/cuda/cu_inc/common.cuh"
#include <hiprand/hiprand_kernel.h>

#define TRANS_TILE_DIM 32
#define BLOCK_ROWS 8
#define COPY_TILE_DIM 1024
#define COPY_BLOCK_DIM 256

// kernel(s) for half functions with no library support
namespace {

// TODO - refactor the function with similar logic in Transpose3DKernel using 16x16 Tile
__global__ void transposeNoOverlap(half* odata, const half* idata, const int m, const int n) {
  __shared__ half tile[TRANS_TILE_DIM][TRANS_TILE_DIM + 1];

  int x = blockIdx.x * TRANS_TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TRANS_TILE_DIM + threadIdx.y;

  if (x < m) {
    for (int j = 0; j < TRANS_TILE_DIM; j += BLOCK_ROWS) {
      if (j >= (n - y)) continue;
      tile[threadIdx.y + j][threadIdx.x] = idata[(y + j) * m + x];
    }
  }

  __syncthreads();

  x = blockIdx.y * TRANS_TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TRANS_TILE_DIM + threadIdx.y;

  if (x >= n) return;

  for (int j = 0; j < TRANS_TILE_DIM; j += BLOCK_ROWS) {
    if ((y + j) >= m) return;
    odata[(y + j) * n + x] = tile[threadIdx.x][threadIdx.y + j];
  }
}

__global__ void CopyVectorHalf(const half* x, int incx, half* y, int incy, int n) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id >= n) return;
  y[id * incy] = x[id * incx];
}

#if CUDA_VERSION >= 11000
__global__ void CopyVectorBFloat16(const hip_bfloat16* x, int incx, hip_bfloat16* y, int incy, int n) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id >= n) return;
  y[id * incy] = x[id * incx];
}
#endif

}  // namespace

hipblasStatus_t cublasTransposeHelper(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int m, int n, const half*, const half* A, int, const half*, const half*, int, half* C, int) {
  if (C != A) {
    dim3 dimGrid((n + TRANS_TILE_DIM - 1) / TRANS_TILE_DIM, (m + TRANS_TILE_DIM - 1) / TRANS_TILE_DIM, 1);
    dim3 dimBlock(TRANS_TILE_DIM, BLOCK_ROWS, 1);

    transposeNoOverlap<<<dimGrid, dimBlock>>>(C, A, n, m);
  } else {
    return HIPBLAS_STATUS_NOT_SUPPORTED;
  }
  return HIPBLAS_STATUS_SUCCESS;
}

hipblasStatus_t cublasCopyHelper(hipblasHandle_t, int n, const half* x, int incx, half* y, int incy) {
  dim3 dimGrid((unsigned int)(n + COPY_BLOCK_DIM - 1) / COPY_BLOCK_DIM, 1, 1);
  dim3 dimBlock(COPY_BLOCK_DIM, 1, 1);
  CopyVectorHalf<<<dimGrid, dimBlock>>>(x, incx, y, incy, n);
  return HIPBLAS_STATUS_SUCCESS;
}

#if CUDA_VERSION >= 11000
hipblasStatus_t cublasCopyHelper(hipblasHandle_t, int n, const hip_bfloat16* x, int incx, hip_bfloat16* y, int incy) {
  dim3 dimGrid((unsigned int)(n + COPY_BLOCK_DIM - 1) / COPY_BLOCK_DIM, 1, 1);
  dim3 dimBlock(COPY_BLOCK_DIM, 1, 1);
  CopyVectorBFloat16<<<dimGrid, dimBlock>>>(x, incx, y, incy, n);
  return HIPBLAS_STATUS_SUCCESS;
}


#endif
