#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_runtime.h>
#include "activations_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cu_inc/unary_elementwise_impl.cuh"

namespace onnxruntime {
namespace cuda {

template <typename T>
struct OP_Elu : public CtxElu {
  __device__ __inline__ T operator()(const T& a) const {
    return a > (T)0 ? a : (T)alpha * (_Exp(a) - (T)1);
  }
};

template <typename T>
struct OP_HardSigmoid : public CtxHardSigmoid {
  __device__ __inline__ T operator()(const T& a) const {
    return _Max(_Min((T)alpha * a + (T)beta, (T)1), (T)0);
  }
};

template <typename T>
struct OP_LeakyRelu : public CtxLeakyRelu {
  __device__ __inline__ T operator()(const T& a) const {
    return a > (T)0 ? a : (T)alpha * a;
  }
};

template <typename T>
struct OP_Relu : public CtxRelu {
  __device__ __inline__ T operator()(const T& a) const {
    return _Max(a, (T)0);
  }
};

template <typename T>
struct OP_Selu : public CtxSelu {
  __device__ __inline__ T operator()(const T& a) const {
    return (T)gamma * (_Max(a, (T)0) + _Min((T)alpha * (_Exp(a) - (T)1), (T)0));
  }
};

template <typename T>
struct OP_Sigmoid : public CtxSigmoid {
  __device__ __inline__ T operator()(const T& a) const {
    return a > T(0) ? (T)1 / ((T)1. + _Exp(-_Abs(a))) : (T)1 - (T)1 / ((T)1 + _Exp(-_Abs(a)));
  }
};

template <typename T>
struct OP_Softplus : public CtxSoftplus {
  __device__ __inline__ T operator()(const T& a) const {
    if (a > (T)0)
      return a + _Log(_Exp(-a) + (T)1);
    else
      return _Log(_Exp(a) + (T)1);
  }
};

template <typename T>
struct OP_Softsign : public CtxSoftsign {
  __device__ __inline__ T operator()(const T& a) const {
    return a / ((T)1. + _Abs(a));
  }
};

template <typename T>
struct OP_Tanh : public CtxTanh {
  __device__ __inline__ T operator()(const T& a) const {
    return _Tanh(a);
  }
};

template <typename T>
struct OP_ThresholdedRelu : public CtxThresholdedRelu {
  __device__ __inline__ T operator()(const T& a) const {
    return a > (T)alpha ? a : (T)0;
  }
};

#define UNARY_ACTIVATION_IMPL(name)                                        \
  UNARY_ACTIVATION_IMPL_DECLARATION(name) {                                \
    UnaryElementWiseImpl(input_data,                                       \
                         output_data,                                      \
                         *reinterpret_cast<const OP_##name<T>*>(func_ctx), \
                         count);                                           \
  }

#define SPECIALIZED_UNARY_ACTIVATION_IMPL(name, T) \
  template void Impl_##name<T>(const T* input_data, T* output_data, const Ctx##name* func_ctx, size_t count);

#if CUDA_VERSION >= 11000 && (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
#define SPECIALIZED_UNARY_ACTIVATION_IMPL_BF16(name) SPECIALIZED_UNARY_ACTIVATION_IMPL(name, hip_bfloat16)
#else
#define SPECIALIZED_UNARY_ACTIVATION_IMPL_BF16(name)
#endif

#define SPECIALIZED_UNARY_ACTIVATIONL_HFD(name)   \
  SPECIALIZED_UNARY_ACTIVATION_IMPL(name, half)   \
  SPECIALIZED_UNARY_ACTIVATION_IMPL_BF16(name)    \
  SPECIALIZED_UNARY_ACTIVATION_IMPL(name, float)  \
  SPECIALIZED_UNARY_ACTIVATION_IMPL(name, double)

#define UNARY_ACTIVATION_OP_NAME(name) \
  UNARY_ACTIVATION_IMPL(name);         \
  SPECIALIZED_UNARY_ACTIVATIONL_HFD(name)

UNARY_ACTIVATION_OPS()
#undef UNARY_ACTIVATION_OP_NAME

}  // namespace cuda
}  // namespace onnxruntime
