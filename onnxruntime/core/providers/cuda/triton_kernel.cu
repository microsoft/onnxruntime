// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/triton_kernel.h"
#include "core/framework/tunable.h"
#include <fstream>
#include <thread>

#ifdef USE_TRITON_KERNEL
#include <dlfcn.h>
#include "triton_kernel_infos.h"
#endif

#define ORT_TRITON_CHECK(status, msg)                \
  if ((status) != hipSuccess) {                      \
      ORT_RETURN_IF(true, msg);                      \
  }

#define ORT_TRITON_THROW(status, msg)          \
  if ((status) != hipSuccess) {                \
      ORT_THROW(msg);                          \
  }

namespace onnxruntime {
namespace cuda {
namespace {

// a vector of kernel metadata
static std::vector<TritonKernelMetaData> ort_triton_kernel_metadata;

// store group_name -> [kernel metadata id vector]
static std::unordered_map<std::string, std::vector<int>> ort_triton_kernel_group_map;

#ifdef USE_TRITON_KERNEL

// store func_name -> kernel metadata id
static std::unordered_map<std::string, int> ort_triton_kernel_map;

const int GPU_WARP_SIZE = 32;

Status GetSymbolFromLibrary(const std::string& symbol_name, void** symbol) {
  dlerror();  // clear any old error str

  // USe RTLD_DEFAULT for search current lib.so
  // value of RTLD_DEFAULT differs across posix platforms (-2 on macos, 0 on linux).
  void* handle = RTLD_DEFAULT;
  *symbol = dlsym(handle, symbol_name.c_str());

  char* error_str = dlerror();
  if (error_str) {
    Status status = ORT_MAKE_STATUS(ONNXRUNTIME, INVALID_ARGUMENT,
		    "Failed to get symbol " + symbol_name + " with error: " + error_str);
    return status;
  }
  // it's possible to get a NULL symbol in our case when Schemas are not custom.
  return Status::OK();
}
#endif


/*
 *  Try to load HIP kernels that compiled by triton.
 *  They are in hsaco/cubin format, and should use hipModuleLoad to load these kernels.
 */
void TryToLoadKernel() {
  auto status = Status::OK();

#ifdef USE_TRITON_KERNEL
  // get all kernel symbols from curret lib.so
  size_t size = sizeof(kernel_infos) / sizeof(kernel_infos[0]);

  for (int i = 0; i < size; ++i) {
    auto k_i = kernel_infos[i];

    void *buff;
    ORT_THROW_IF_ERROR(GetSymbolFromLibrary(k_i.name_start, &buff));

    // try to load module and get function
    hipModule_t module;
    ORT_TRITON_THROW(hipModuleLoadData(&module, buff), "load module data failed.");

    hipFunction_t function;
    ORT_TRITON_THROW(hipModuleGetFunction(&function, module, k_i.func_name), "get funcion from module failed.");

    // setup kernel metadata
    TritonKernelMetaData metadata;
    metadata.num_warps = k_i.num_warps;
    metadata.shared_mem_size = k_i.shared;
    metadata.func = function;
    std::string fname = k_i.name;  // name is not same as func_name
    metadata.name = fname;
    std::string group_name = k_i.group_name;

    // pass constants
    for (auto &kv : k_i.constants) {
      metadata.constants[kv.first] = kv.second;
    }

    auto idx = ort_triton_kernel_metadata.size();
    ort_triton_kernel_metadata.push_back(metadata);
    ort_triton_kernel_map[fname] = idx;
    ort_triton_kernel_group_map[group_name].push_back(idx);
    LOGS_DEFAULT(VERBOSE) << "loaded ort triton kernel: " << fname << " idx: " << idx;
  }
#endif

  ORT_THROW_IF_ERROR(status);
}

static std::once_flag load_ort_triton_kernel_flag;

}  // end of namespace

void LoadOrtTritonKernel() {
  // load kernel should be called only once
  std::call_once(load_ort_triton_kernel_flag, TryToLoadKernel);
}

Status LaunchTritonKernel(hipStream_t stream, std::string fname, int grid0, int grid1, int grid2, void* args, size_t args_size) {
#ifdef USE_TRITON_KERNEL
  if (ort_triton_kernel_map.count(fname) == 0) {
    // return unsupported status when not found function name in registry
    // this error status will be used by tunableOp
    std::ostringstream message_stream;
    message_stream << "can't find ort triton kernel name: " << fname;
    std::string message = message_stream.str();
    TUNABLE_OP_RETURN_UNSUPPORTED_ARGUMENT_IF(true, message);
  }
  auto idx = ort_triton_kernel_map[fname];
  auto metadata = ort_triton_kernel_metadata[idx];

  void* config[] = {HIP_LAUNCH_PARAM_BUFFER_POINTER, args, HIP_LAUNCH_PARAM_BUFFER_SIZE, &args_size,
                    HIP_LAUNCH_PARAM_END};

  ORT_TRITON_CHECK(hipModuleLaunchKernel(metadata.func,
                                  grid0, grid1, grid2,
                                  GPU_WARP_SIZE * metadata.num_warps, 1, 1,
                                  metadata.shared_mem_size,
                                  stream,
                                  nullptr,
                                  (void**)&config), "launch kernel failed.");
#endif

  return Status::OK();
}

Status LaunchTritonKernel(hipStream_t stream, size_t idx, int grid0, int grid1, int grid2, void* args, size_t args_size) {
#ifdef USE_TRITON_KERNEL
  if (idx >= ort_triton_kernel_metadata.size()) {
    // return unsupported status when not found function name in registry
    // this error status will be used by tunableOp
    std::ostringstream message_stream;
    message_stream << "can't find ort triton kernel idx: " << idx;
    std::string message = message_stream.str();
    TUNABLE_OP_RETURN_UNSUPPORTED_ARGUMENT_IF(true, message);
  }
  auto metadata = ort_triton_kernel_metadata[idx];

  void* config[] = {HIP_LAUNCH_PARAM_BUFFER_POINTER, args, HIP_LAUNCH_PARAM_BUFFER_SIZE, &args_size,
                    HIP_LAUNCH_PARAM_END};

  ORT_TRITON_CHECK(hipModuleLaunchKernel(metadata.func,
                                  grid0, grid1, grid2,
                                  GPU_WARP_SIZE * metadata.num_warps, 1, 1,
                                  metadata.shared_mem_size,
                                  stream,
                                  nullptr,
                                  (void**)&config), "launch kernel failed.");
#endif

  return Status::OK();
}

const TritonKernelMetaData* GetOrtTritonKernelMetadata(size_t idx) {
  if (idx >= ort_triton_kernel_metadata.size()) {
    return nullptr;
  }
  return &ort_triton_kernel_metadata[idx];
}

const std::vector<int>* GetOrtTritonKernelByGroup(std::string group_name) {
  if (ort_triton_kernel_group_map.count(group_name) == 0) {
    return nullptr;
  }
  return &ort_triton_kernel_group_map.at(group_name);
}

}  // namespace cuda
}  // namespace onnxruntime
