#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "transpose_impl.h"

namespace onnxruntime {
namespace cuda {

constexpr unsigned int NUM_ELE_PER_THREAD = 4;

template <typename T, unsigned int TILE_DIM>
__global__ void Transpose3DKernel(const TArray<int64_t> input_shape, const TArray<int64_t> input_strides,
                                  const T* input_data, T* output_data) {
  __shared__ T tile[TILE_DIM][TILE_DIM + 1];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;

#pragma unroll
  for (unsigned int i = 0; i < TILE_DIM; i += (TILE_DIM / NUM_ELE_PER_THREAD)) {
    tile[threadIdx.y + i][threadIdx.x] = input_data[blockIdx.z * input_strides[0] + (y + i) * input_shape[2] + x];
  }
  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;
  y = blockIdx.x * TILE_DIM + threadIdx.y;

#pragma unroll
  for (unsigned int i = 0; i < TILE_DIM; i += (TILE_DIM / NUM_ELE_PER_THREAD)) {
    output_data[blockIdx.z * input_strides[0] + (y + i) * input_shape[1] + x] = tile[threadIdx.x][threadIdx.y + i];
  }
}

bool CanDoTranspose3D(const hipDeviceProp_t& prop, size_t rank, const gsl::span<const int64_t>& input_dims,
                      const gsl::span<const size_t>& permutations, dim3& grid_size, dim3& block_size) {
  // Permutation is done in the last two dimensions and the last two dimensions are aligned with TILE_DIM.
  if (rank == 3 && permutations[rank - 2] == (rank - 1) && permutations[rank - 1] == (rank - 2)) {
    unsigned int tile_dim = 0;
    if (input_dims[rank - 2] % 32 == 0 && input_dims[rank - 1] % 32 == 0) {
      tile_dim = 32;
    } else if (input_dims[rank - 2] % 16 == 0 && input_dims[rank - 1] % 16 == 0) {
      tile_dim = 16;
    } else {
      return false;
    }

    int grid_size_x = static_cast<int>(input_dims[2] / tile_dim);
    int grid_size_y = static_cast<int>(input_dims[1] / tile_dim);
    int grid_size_z = static_cast<int>(input_dims[0]);

    if (grid_size_x <= prop.maxGridSize[0] && grid_size_y <= prop.maxGridSize[1] &&
        grid_size_z <= prop.maxGridSize[2]) {
      block_size = dim3(tile_dim, tile_dim / NUM_ELE_PER_THREAD);
      grid_size = dim3(static_cast<unsigned int>(grid_size_x), static_cast<unsigned int>(grid_size_y),
                       static_cast<unsigned int>(grid_size_z));
      return true;
    } else {
      return false;
    }
  }
  return false;
}

#define CALL_TRANSPOSE_3D(type, tile_dim)                                                            \
  Transpose3DKernel<type, tile_dim><<<grid_size, block_size, 0, stream>>>(                           \
      input_shape, input_strides, reinterpret_cast<const ToCudaType<type>::MappedType*>(input_data), \
      reinterpret_cast<ToCudaType<type>::MappedType*>(output_data))

#define HANDLE_TRANSPOSE_3D_TILE_DIM(type) \
  case sizeof(type): {                     \
    if (block_size.x == 32) {              \
      CALL_TRANSPOSE_3D(type, 32);         \
    } else {                               \
      CALL_TRANSPOSE_3D(type, 16);         \
    }                                      \
  } break

Status Transpose3DImpl(hipStream_t stream, size_t element_size, const TArray<int64_t>& input_shape,
                       const TArray<int64_t>& input_strides, const void* input_data, void* output_data, int64_t N,
                       const dim3& grid_size, const dim3& block_size) {
  switch (element_size) {
    HANDLE_TRANSPOSE_3D_TILE_DIM(int8_t);
    HANDLE_TRANSPOSE_3D_TILE_DIM(int16_t);
    HANDLE_TRANSPOSE_3D_TILE_DIM(int32_t);
    HANDLE_TRANSPOSE_3D_TILE_DIM(int64_t);
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for transpose on CUDA. Element size was ",
                             element_size);
  }

  return Status::OK();
}

template <int element_size>
__global__ void Transpose4DKernelParallelizeMultipleElementsPerThreadInInnermostDim(
    const TArray<int64_t> input_strides, const void* input_data,
    const TArray<int64_t> output_strides, void* output_data,
    int64_t input_shape_2, CUDA_LONG N) {
  // coordinates will be: [d0, d1, d2, d3]
  CUDA_LONG d0 = blockIdx.z;
  CUDA_LONG d1 = blockIdx.y;
  CUDA_LONG d2 = threadIdx.y + blockIdx.x * blockDim.y;
  CUDA_LONG d3 = threadIdx.x;

  CUDA_LONG input_index = (d0 * input_strides[0] +
                           d1 * input_strides[1] +
                           d2 * input_strides[2]) /
                              (4 * sizeof(int) / element_size) +
                          d3 * input_strides[3];

  CUDA_LONG output_index = (d0 * output_strides[0] +
                            d1 * output_strides[1] +
                            d2 * output_strides[2]) /
                               (4 * sizeof(int) / element_size) +
                           d3 * output_strides[3];

  const int4* v_input = reinterpret_cast<const int4*>(input_data);
  int4* v_output = reinterpret_cast<int4*>(output_data);

  if (input_index < N && output_index < N && d2 < input_shape_2) {
    v_output[output_index] = v_input[input_index];
  }
}

bool CanDoTranspose4DParallelizeMultipleElementsPerThreadInInnermostDim(const hipDeviceProp_t& prop,
                                                                        size_t element_size,
                                                                        int32_t rank,
                                                                        const gsl::span<const int64_t>& input_dims,
                                                                        const gsl::span<const size_t>& permutations,
                                                                        dim3& grid_size, dim3& block_size) {
  if (rank == 4 &&
      // the permutations is not on the last dimension.
      permutations[3] == 3) {
    unsigned int num_elements_per_thread = 4 * sizeof(int) / static_cast<unsigned int>(element_size);  // int4 is used in the kernel to access data.

    // dims[3]: block.x
    // dims[2]: block.y + grid.x
    // dims[1]: grid.y
    // dims[0]: grid.z
    if (input_dims[3] / num_elements_per_thread <= prop.maxThreadsPerBlock &&
        (input_dims[3] % num_elements_per_thread) == 0 &&
        input_dims[1] <= prop.maxGridSize[1] &&
        input_dims[0] <= prop.maxGridSize[2]) {
      // There are 2 constrains when luanching the kernels
      // 1. block_size_x * block_size_y <= prop.maxThreadsPerBlock
      // 2. block_size_y * num_block_ext >= input_dims[2]
      int64_t block_size_x = input_dims[3] / num_elements_per_thread;
      int64_t max_block_size_y = prop.maxThreadsPerBlock / block_size_x;
      int64_t block_size_y = min(input_dims[2], max_block_size_y);
      int64_t num_block_ext = CeilDiv(input_dims[2], block_size_y);

      if (num_block_ext <= prop.maxGridSize[0]) {
        block_size = dim3(static_cast<unsigned int>(block_size_x), static_cast<unsigned int>(block_size_y));
        grid_size = dim3(static_cast<unsigned int>(num_block_ext),
                         static_cast<unsigned int>(input_dims[1]),
                         static_cast<unsigned int>(input_dims[0]));
        return true;
      } else {
        return false;
      }
    }
  }
  return false;
}

Status Transpose4DParallelizeMultipleElementsPerThreadInInnermostDim(
    hipStream_t stream, size_t element_size,
    const TArray<int64_t>& input_shape, const TArray<int64_t>& input_strides,
    const void* input_data, const TArray<int64_t>& output_strides,
    void* output_data, int N, const dim3& grid_size, const dim3& block_size) {
  unsigned int num_elements_per_thread = 4 * sizeof(int) / static_cast<unsigned int>(element_size);  // int4 is used in the kernel to access data.

  switch (element_size) {
    case sizeof(int8_t):
      Transpose4DKernelParallelizeMultipleElementsPerThreadInInnermostDim<sizeof(int8_t)>
          <<<grid_size, block_size, 0, stream>>>(
              input_strides, input_data,
              output_strides, output_data,
              input_shape[2],
              N / num_elements_per_thread);
      break;
    case sizeof(int16_t):
      Transpose4DKernelParallelizeMultipleElementsPerThreadInInnermostDim<sizeof(int16_t)>
          <<<grid_size, block_size, 0, stream>>>(
              input_strides, input_data,
              output_strides, output_data,
              input_shape[2],
              N / num_elements_per_thread);
      break;
    case sizeof(int32_t):
      Transpose4DKernelParallelizeMultipleElementsPerThreadInInnermostDim<sizeof(int32_t)>
          <<<grid_size, block_size, 0, stream>>>(
              input_strides, input_data,
              output_strides, output_data,
              input_shape[2],
              N / num_elements_per_thread);
      break;
    case sizeof(int64_t):
      Transpose4DKernelParallelizeMultipleElementsPerThreadInInnermostDim<sizeof(int64_t)>
          <<<grid_size, block_size, 0, stream>>>(
              input_strides, input_data,
              output_strides, output_data,
              input_shape[2],
              N / num_elements_per_thread);
      break;
    default:
      // User will not hit this as this kernel is for fixed element size tensors only
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for transpose on CUDA. Element size was ",
                             element_size);
  }

  return Status::OK();
}

__global__ void Transpose4DKernelParallelizeOneElementPerThread(
    const TArray<int64_t> input_strides, const int8_t* input_data,
    const TArray<int64_t> output_strides, int8_t* output_data,
    size_t element_size, int64_t input_shape_2, CUDA_LONG N) {
  // coordinates will be: [d0, d1, d2, d3]
  CUDA_LONG d0 = blockIdx.z;
  CUDA_LONG d1 = blockIdx.y;
  CUDA_LONG d2 = threadIdx.y + blockIdx.x * blockDim.y;
  CUDA_LONG d3 = threadIdx.x;

  CUDA_LONG input_index = d0 * input_strides[0] +
                          d1 * input_strides[1] +
                          d2 * input_strides[2] +
                          d3 * input_strides[3];

  CUDA_LONG output_index = d0 * output_strides[0] +
                           d1 * output_strides[1] +
                           d2 * output_strides[2] +
                           d3 * output_strides[3];

  if (input_index < N && output_index < N && d2 < input_shape_2) {
    const int8_t* input_data_to_be_copied = input_data + (input_index * element_size);
    int8_t* output_data_to_be_copied = output_data + (output_index * element_size);

    // copy over the bytes
    for (size_t iter = 0; iter < element_size; ++iter) {
      *output_data_to_be_copied++ = *input_data_to_be_copied++;
    }
  }
}

bool CanDoTranspose4DParallelizeOneElementPerThread(const hipDeviceProp_t& prop,
                                                    size_t element_size,
                                                    int32_t rank,
                                                    const gsl::span<const int64_t>& input_dims,
                                                    const gsl::span<const size_t>& permutations,
                                                    dim3& grid_size, dim3& block_size) {
  if (rank == 4) {
    // dims[3]: block.x
    // dims[2]: block.y + grid.x
    // dims[1]: grid.y
    // dims[0]: grid.z
    if (input_dims[3] <= prop.maxThreadsPerBlock &&
        input_dims[1] <= prop.maxGridSize[1] &&
        input_dims[0] <= prop.maxGridSize[2]) {
      // There are 2 constrains when luanching the kernels
      // 1. block_size_x * block_size_y <= prop.maxThreadsPerBlock
      // 2. block_size_y * num_block_ext >= input_dims[2]
      int64_t block_size_x = input_dims[3];
      int64_t max_block_size_y = prop.maxThreadsPerBlock / block_size_x;
      int64_t block_size_y = std::min(input_dims[2], max_block_size_y);
      int64_t num_block_ext = CeilDiv(input_dims[2], block_size_y);

      if (num_block_ext <= prop.maxGridSize[0]) {
        block_size = dim3(static_cast<unsigned int>(block_size_x), static_cast<unsigned int>(block_size_y));
        grid_size = dim3(static_cast<unsigned int>(num_block_ext),
                         static_cast<unsigned int>(input_dims[1]),
                         static_cast<unsigned int>(input_dims[0]));
        return true;
      } else {
        return false;
      }
    }
  }
  return false;
}

Status Transpose4DParallelizeOneElementPerThread(
    hipStream_t stream, size_t element_size,
    const TArray<int64_t>& input_shape, const TArray<int64_t>& input_strides,
    const void* input_data, const TArray<int64_t>& output_strides,
    void* output_data, int N, const dim3& grid_size, const dim3& block_size) {
  if (element_size != sizeof(int8_t) &&
      element_size != sizeof(int16_t) &&
      element_size != sizeof(int32_t) &&
      element_size != sizeof(int64_t)) {
    // User will not hit this as this kernel is for fixed element size tensors only
    return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for transpose on CUDA. Element size was ",
                           element_size);
  }

  Transpose4DKernelParallelizeOneElementPerThread<<<grid_size, block_size, 0, stream>>>(
      input_strides, reinterpret_cast<const int8_t*>(input_data),
      output_strides, reinterpret_cast<int8_t*>(output_data),
      element_size, input_shape[2], N);

  return Status::OK();
}

template <typename T>
__global__ void TransposeKernel(int32_t shape_rank, const TArray<int64_t> input_strides,
                                const T* input_data, const TArray<fast_divmod> output_strides, T* output_data, CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  CUDA_LONG input_index = 0;
  CUDA_LONG output_index = id;

#pragma unroll
  for (auto dim = 0; dim < input_strides.Capacity(); ++dim) {
    if (dim >= shape_rank) {
      break;
    }
    int out_coord, r;
    output_strides[dim].divmod(output_index, out_coord, r);
    output_index = r;
    input_index += input_strides[dim] * out_coord;
  }
  output_data[id] = input_data[input_index];
}

Status TransposeImpl(hipStream_t stream, size_t element_size, int32_t shape_rank, const TArray<int64_t>& input_strides,
                     const void* input_data, const TArray<fast_divmod>& fdm_output_strides, void* output_data, int N) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  switch (element_size) {
    case sizeof(int8_t):
      TransposeKernel<int8_t><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          shape_rank, input_strides,
          reinterpret_cast<const ToCudaType<int8_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToCudaType<int8_t>::MappedType*>(output_data),
          N);
      break;
    case sizeof(int16_t):
      TransposeKernel<int16_t><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          shape_rank, input_strides,
          reinterpret_cast<const ToCudaType<int16_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToCudaType<int16_t>::MappedType*>(output_data),
          N);
      break;
    case sizeof(int32_t):
      TransposeKernel<int32_t><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          shape_rank, input_strides,
          reinterpret_cast<const ToCudaType<int32_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToCudaType<int32_t>::MappedType*>(output_data),
          N);
      break;
    case sizeof(int64_t):
      TransposeKernel<int64_t><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          shape_rank, input_strides,
          reinterpret_cast<const ToCudaType<int64_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToCudaType<int64_t>::MappedType*>(output_data),
          N);
      break;
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for transpose on CUDA. Element size was ",
                             element_size);
  }

  return Status::OK();
}

}  // namespace cuda
}  // namespace onnxruntime
