#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "transpose_impl.h"

namespace onnxruntime {
namespace cuda {

constexpr unsigned int TILE_DIM = 16;

template <typename T>
__global__ void Transpose3DKernel(const TArray<int64_t> input_shape,
                                  const TArray<int64_t> input_strides,
                                  const T* input_data, T* output_data) {
  __shared__ T tile[TILE_DIM * (TILE_DIM + 1)];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;

  tile[threadIdx.y * TILE_DIM + threadIdx.x] = input_data[blockIdx.z * input_strides[0] + y * input_shape[2] + x];
  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  output_data[blockIdx.z * input_strides[0] + y * input_shape[1] + x] = tile[threadIdx.x * TILE_DIM + threadIdx.y];
}

bool CanDoTranspose3D(int32_t rank,
                      const std::vector<int64_t>& input_dims,
                      const std::vector<size_t>& permutations) {
  if (rank == 3 &&
      // permutation is done in the last two dimensions.
      permutations[rank - 2] == (rank - 1) && permutations[rank - 1] == (rank - 2) &&
      // the last two dimensions are aligned with TILE_DIM.
      input_dims[rank - 2] % TILE_DIM == 0 && input_dims[rank - 1] % TILE_DIM == 0) {
    return true;
  }
  return false;
}

Status Transpose3DImpl(size_t element_size,
                       const TArray<int64_t>& input_shape, const TArray<int64_t>& input_strides,
                       const void* input_data, void* output_data, int64_t N) {
  dim3 block_size(TILE_DIM, TILE_DIM);
  dim3 grid_size(static_cast<unsigned int>(input_shape[2] / TILE_DIM), static_cast<unsigned int>(input_shape[1] / TILE_DIM), static_cast<unsigned int>(input_shape[0]));

  switch (element_size) {
    case sizeof(int8_t):
      Transpose3DKernel<int8_t><<<grid_size, block_size, 0>>>(
          input_shape, input_strides,
          reinterpret_cast<const ToCudaType<int8_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int8_t>::MappedType*>(output_data));
      break;
    case sizeof(int16_t):
      Transpose3DKernel<int16_t><<<grid_size, block_size, 0>>>(
          input_shape, input_strides,
          reinterpret_cast<const ToCudaType<int16_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int16_t>::MappedType*>(output_data));
      break;
    case sizeof(int32_t):
      Transpose3DKernel<int32_t><<<grid_size, block_size, 0>>>(
          input_shape, input_strides,
          reinterpret_cast<const ToCudaType<int32_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int32_t>::MappedType*>(output_data));
      break;
    case sizeof(int64_t):
      Transpose3DKernel<int64_t><<<grid_size, block_size, 0>>>(
          input_shape, input_strides,
          reinterpret_cast<const ToCudaType<int64_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int64_t>::MappedType*>(output_data));
      break;
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for transpose on CUDA. Element size was ",
                             element_size);
  }

  return Status::OK();
}

template <int element_size>
__global__ void Transpose4DKernel(const TArray<int64_t> input_strides, const void* input_data,
                                  const TArray<int64_t> output_strides, void* output_data,
                                  CUDA_LONG N) {
  // output coordinates will be: blockIdx.y, blockIdx.x, threadIdx.y, threadIdx.x
  CUDA_LONG input_index = (blockIdx.y * input_strides[0] +
                           blockIdx.x * input_strides[1] +
                           threadIdx.y * input_strides[2]) /
                              (4 * sizeof(int) / element_size) +
                          threadIdx.x * input_strides[3];

  CUDA_LONG output_index = (blockIdx.y * output_strides[0] +
                            blockIdx.x * output_strides[1] +
                            threadIdx.y * output_strides[2]) /
                               (4 * sizeof(int) / element_size) +
                           threadIdx.x * output_strides[3];

  const int4* v_input = reinterpret_cast<const int4*>(input_data);
  int4* v_output = reinterpret_cast<int4*>(output_data);

  if (input_index < N && output_index < N) {
    v_output[output_index] = v_input[input_index];
  }
}

bool CanDoTranspose4D(const hipDeviceProp_t& prop,
                      size_t element_size,
                      int32_t rank,
                      const std::vector<int64_t>& input_dims,
                      const std::vector<size_t>& permutations) {
  if (rank == 4 &&
      // the permutations is not on the last dimension.
      permutations[rank - 1] == (rank - 1)) {
    // The block size will be set based on the last two dimensions of 4D tensor.
    // the number threads per block will be calculated as below.
    unsigned int num_elements_per_thread = 4 * sizeof(int) / static_cast<unsigned int>(element_size);  // int4 is used in the kernel to access data.
    int64_t num_elements_in_last_two_dimensions = input_dims[rank - 2] * input_dims[rank - 1];
    int64_t num_threads_per_block = num_elements_in_last_two_dimensions / num_elements_per_thread;

    if (((num_elements_in_last_two_dimensions & (num_elements_per_thread - 1)) == 0) &&
        num_threads_per_block <= prop.maxThreadsPerBlock &&
        num_threads_per_block >= prop.warpSize &&
        // num_threads_per_block must be aligned with warp size: 32
        ((num_threads_per_block & (prop.warpSize - 1)) == 0)) {
      return true;
    }
  }
  return false;
}

Status Transpose4DImpl(size_t element_size, const TArray<int64_t>& input_shape, const TArray<int64_t>& input_strides, const void* input_data,
                       const TArray<int64_t>& output_strides, void* output_data, int N) {
  unsigned int num_elements_per_thread = 4 * sizeof(int) / static_cast<unsigned int>(element_size);  // int4 is used in the kernel to access data.
  dim3 block_size(static_cast<unsigned int>(input_shape[3] / num_elements_per_thread), static_cast<unsigned int>(input_shape[2]));
  dim3 grid_size(static_cast<unsigned int>(input_shape[1]), static_cast<unsigned int>(input_shape[0]));

  switch (element_size) {
    case sizeof(int8_t):
      Transpose4DKernel<sizeof(int8_t)><<<grid_size, block_size, 0>>>(
          input_strides, input_data,
          output_strides, output_data, N / num_elements_per_thread);
      break;
    case sizeof(int16_t):
      Transpose4DKernel<sizeof(int16_t)><<<grid_size, block_size, 0>>>(
          input_strides, input_data,
          output_strides, output_data, N / num_elements_per_thread);
      break;
    case sizeof(int32_t):
      Transpose4DKernel<sizeof(int32_t)><<<grid_size, block_size, 0>>>(
          input_strides, input_data,
          output_strides, output_data, N / num_elements_per_thread);
      break;
    case sizeof(int64_t):
      Transpose4DKernel<sizeof(int64_t)><<<grid_size, block_size, 0>>>(
          input_strides, input_data,
          output_strides, output_data, N / num_elements_per_thread);
      break;
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for transpose on CUDA. Element size was ",
                             element_size);
  }

  return Status::OK();
}

template <typename T>
__global__ void TransposeKernel(int32_t shape_rank, const TArray<int64_t> input_strides,
                                const T* input_data, const TArray<fast_divmod> output_strides, T* output_data, CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  CUDA_LONG input_index = 0;
  CUDA_LONG output_index = id;

#pragma unroll
  for (auto dim = 0; dim < input_strides.Capacity(); ++dim) {
    if (dim >= shape_rank) {
      break;
    }
    int out_coord, r;
    output_strides[dim].divmod(output_index, out_coord, r);
    output_index = r;
    input_index += input_strides[dim] * out_coord;
  }
  output_data[id] = input_data[input_index];
}

Status TransposeImpl(size_t element_size, int32_t shape_rank, const TArray<int64_t>& input_strides,
                     const void* input_data, const TArray<fast_divmod>& fdm_output_strides, void* output_data, int N) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  switch (element_size) {
    case sizeof(int8_t):
      TransposeKernel<int8_t><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          shape_rank, input_strides,
          reinterpret_cast<const ToCudaType<int8_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToCudaType<int8_t>::MappedType*>(output_data),
          N);
      break;
    case sizeof(int16_t):
      TransposeKernel<int16_t><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          shape_rank, input_strides,
          reinterpret_cast<const ToCudaType<int16_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToCudaType<int16_t>::MappedType*>(output_data),
          N);
      break;
    case sizeof(int32_t):
      TransposeKernel<int32_t><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          shape_rank, input_strides,
          reinterpret_cast<const ToCudaType<int32_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToCudaType<int32_t>::MappedType*>(output_data),
          N);
      break;
    case sizeof(int64_t):
      TransposeKernel<int64_t><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          shape_rank, input_strides,
          reinterpret_cast<const ToCudaType<int64_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToCudaType<int64_t>::MappedType*>(output_data),
          N);
      break;
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for transpose on CUDA. Element size was ",
                             element_size);
  }

  return Status::OK();
}

}  // namespace cuda
}  // namespace onnxruntime
