#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "quantize_linear.cuh"

#include <limits>

#include "core/providers/cuda/cu_inc/common.cuh"

namespace onnxruntime {
namespace cuda {

template <typename T>
struct Round;

template <>
struct Round<float> {
  __device__ __forceinline__ int operator()(float v) const {
    return __float2int_rn(v);
  }
};

template <>
struct Round<half> {
  __device__ __forceinline__ int operator()(half v) const {
    return __half2int_rn(v);
  }
};

template <int NumThreadsPerBlock, int NumElementsPerThread, typename OutT, typename InT>
__global__ void QuantizeLinearKernel(const InT* input, OutT* output, const InT* scale_ptr, const OutT* zero_point_ptr, CUDA_LONG N, Round<InT> round) {
  CUDA_LONG id = NumElementsPerThread * NumThreadsPerBlock * blockIdx.x + threadIdx.x;

  InT scale = *scale_ptr;
  OutT zero_point = zero_point_ptr != nullptr ? *zero_point_ptr : 0;
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      int value = round(input[id] / scale) + zero_point;
      output[id] = static_cast<OutT>(max(std::numeric_limits<OutT>::min(), min(std::numeric_limits<OutT>::max(), value)));
      id += NumThreadsPerBlock;
    }
  }
}

template <class OutT, class InT>
Status CudaQuantizeLinear(const InT* input, OutT* output, const InT* scale, const OutT* zero_point, size_t num_of_element) {
  if (num_of_element <= 0)
    return Status::OK();

  int blocksPerGrid = static_cast<int>(CeilDiv(num_of_element, GridDim::maxThreadsPerBlock * GridDim::maxElementsPerThread));
  QuantizeLinearKernel<GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread, OutT, InT><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      input,
      output,
      scale,
      zero_point,
      static_cast<int>(num_of_element),
      Round<InT>());
  return Status::OK();
}

template <class InT, class OutT, int NumThreadsPerBlock, int NumElementsPerThread>
__global__ void DequantizeLinearKernel(const InT* input, OutT* output, const OutT* scale_ptr, const InT* zero_point_ptr, CUDA_LONG N) {
  CUDA_LONG id = NumElementsPerThread * NumThreadsPerBlock * blockIdx.x + threadIdx.x;

  OutT scale = *scale_ptr;
  InT zero_point = zero_point_ptr != nullptr ? *zero_point_ptr : 0;
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      output[id] = static_cast<OutT>(input[id] - zero_point) * scale;
      id += NumThreadsPerBlock;
    }
  }
}

template <class InT, class OutT>
Status CudaDequantizeLinear(const InT* input, OutT* output, const OutT* scale, const InT* zero_point, size_t num_of_element) {
  if (num_of_element <= 0)
    return Status::OK();

  int blocksPerGrid = static_cast<int>(CeilDiv(num_of_element, GridDim::maxThreadsPerBlock * GridDim::maxElementsPerThread));
  DequantizeLinearKernel<InT, OutT, GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      input,
      output,
      scale,
      zero_point,
      static_cast<int>(num_of_element));
  return Status::OK();
}

template Status CudaQuantizeLinear<int8_t, float>(const float* input, int8_t* output, const float* scale, const int8_t* zero_point, size_t num_of_element);
template Status CudaQuantizeLinear<uint8_t, float>(const float* input, uint8_t* output, const float* scale, const uint8_t* zero_point, size_t num_of_element);
template Status CudaQuantizeLinear<int8_t, half>(const half* input, int8_t* output, const half* scale, const int8_t* zero_point, size_t num_of_element);
template Status CudaQuantizeLinear<uint8_t, half>(const half* input, uint8_t* output, const half* scale, const uint8_t* zero_point, size_t num_of_element);

template Status CudaDequantizeLinear<int8_t, float>(const int8_t* input, float* output, const float* scale, const int8_t* zero_point, size_t num_of_element);
template Status CudaDequantizeLinear<uint8_t, float>(const uint8_t* input, float* output, const float* scale, const uint8_t* zero_point, size_t num_of_element);
template Status CudaDequantizeLinear<int8_t, half>(const int8_t* input, half* output, const half* scale, const int8_t* zero_point, size_t num_of_element);
template Status CudaDequantizeLinear<uint8_t, half>(const uint8_t* input, half* output, const half* scale, const uint8_t* zero_point, size_t num_of_element);

}  // namespace cuda
}  // namespace onnxruntime
