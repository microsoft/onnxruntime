#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "tile_impl.h"

namespace onnxruntime {
namespace cuda {

#ifdef USE_ROCM
constexpr int num_elements_per_thread = 2;
constexpr int num_threads_per_block = 512;
#else
constexpr int num_elements_per_thread = GridDim::maxElementsPerThread;
constexpr int num_threads_per_block = GridDim::maxThreadsPerBlock;
#endif

template <typename T>
__global__ void _UnRolledTileKernel(const size_t shape_rank, const TArray<fast_divmod> fdm_input_shape,
                                    const TArray<int64_t> input_strides, const T* input_data,
                                    const TArray<fast_divmod> fdm_output_strides, T* output_data, const CUDA_LONG N) {
  CUDA_LONG start = num_elements_per_thread * num_threads_per_block * blockIdx.x + threadIdx.x;
  T value[num_elements_per_thread];
  CUDA_LONG id = start;
#pragma unroll
  for (int i = 0; i < num_elements_per_thread; ++i) {
    if (id < N) {
      CUDA_LONG input_index = 0;
      CUDA_LONG offset = id;
#pragma unroll
      for (auto dim = 0; dim < fdm_output_strides.Capacity(); ++dim) {
        if (dim >= shape_rank) {
          break;
        }

        int out_coord, r;
        fdm_output_strides[dim].divmod(offset, out_coord, r);
        int in_coord = fdm_input_shape[dim].mod(out_coord);
        input_index += input_strides[dim] * in_coord;
        offset = r;
      }

      value[i] = input_data[input_index];
      id += num_threads_per_block;
    }
  }

  id = start;
#pragma unroll
  for (int i = 0; i < num_elements_per_thread; ++i) {
    if (id < N) {
      output_data[id] = value[i];
      id += num_threads_per_block;
    }
  }
}

template <typename T>
void TileImpl(hipStream_t stream, const size_t shape_rank, const TArray<fast_divmod>& fdm_input_shape,
              const TArray<int64_t>& input_stride, const T* input_data, const TArray<fast_divmod>& fdm_output_strides,
              T* output_data, const size_t N) {
  int blocksPerGrid = static_cast<int>(CeilDiv(N, num_threads_per_block * num_elements_per_thread));
  _UnRolledTileKernel<T><<<blocksPerGrid, num_threads_per_block, 0, stream>>>(shape_rank, fdm_input_shape, input_stride,
                                                                              input_data, fdm_output_strides,
                                                                              output_data, static_cast<CUDA_LONG>(N));
}

template <typename T>
__global__ void _TileMemcpyKernelFromOutput(const T* input_data, T* output_data,
                                            const fast_divmod divmod_num_input_elements, const CUDA_LONG N) {
  CUDA_LONG start = num_elements_per_thread * num_threads_per_block * blockIdx.x + threadIdx.x;
  T value[num_elements_per_thread];
  CUDA_LONG id = start;
#pragma unroll
  for (int i = 0; i < num_elements_per_thread; ++i) {
    if (id < N) {
      value[i] = input_data[divmod_num_input_elements.mod(id)];
      id += num_threads_per_block;
    }
  }

  id = start;
#pragma unroll
  for (int i = 0; i < num_elements_per_thread; ++i) {
    if (id < N) {
      output_data[id] = value[i];
      id += num_threads_per_block;
    }
  }
}

template <typename T>
__global__ void _TileMemcpyKernelFromInput(const T* input_data, T* output_data, const CUDA_LONG N,
                                           const size_t repeats) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  T input_val = input_data[id];
  for (size_t i = 0; i < repeats; ++i) {
    output_data[id] = input_val;
    id += N;
  }
}

template <typename T>
size_t GetVectorizedSize(size_t num_input_elements, size_t num_elements_per_batch, uint64_t address_input,
                         uint64_t address_output, CUDA_LONG& N, int& blocksPerGrid) {
  constexpr int vec4_alignment = std::alignment_of<aligned_vector<T, 4>>::value;
  constexpr int vec2_alignment = std::alignment_of<aligned_vector<T, 2>>::value;
  N = static_cast<CUDA_LONG>(num_input_elements);
  size_t vectorized_size = 1;
  if (num_elements_per_batch % 4 == 0 && address_input % vec4_alignment == 0 && address_output % vec4_alignment == 0) {
    N /= 4;
    vectorized_size = 4;
  } else if (num_elements_per_batch % 2 == 0 && address_input % vec2_alignment == 0 &&
             address_output % vec2_alignment == 0) {
    N /= 2;
    vectorized_size = 2;
  }
  blocksPerGrid = CeilDiv(N, num_threads_per_block);
  return vectorized_size;
}

template <typename T>
void TileMemcpyImpl(hipStream_t stream, const T* input_data, T* output_data, const size_t num_input_elements,
                    const size_t repeats) {
  // If the block number from input size is too small to fill all streaming multiprocessors,
  // it won't have perf gain to launch from inputs. In this case we will use the output based kernel.
  CUDA_LONG N;
  int blocksPerGrid;
  size_t vectorized_size =
      GetVectorizedSize<T>(num_input_elements, num_input_elements, reinterpret_cast<uint64_t>(input_data),
                           reinterpret_cast<uint64_t>(output_data), N, blocksPerGrid);
  if (blocksPerGrid < 128) {
    N = static_cast<CUDA_LONG>(num_input_elements * repeats);
    blocksPerGrid = CeilDiv(N, num_threads_per_block * num_elements_per_thread);
    _TileMemcpyKernelFromOutput<<<blocksPerGrid, num_threads_per_block, 0, stream>>>(
        input_data, output_data, fast_divmod(static_cast<int>(num_input_elements)), N);
    return;
  }

  if (vectorized_size == 4) {
    using Vec4T = aligned_vector<T, 4>;
    _TileMemcpyKernelFromInput<<<blocksPerGrid, num_threads_per_block, 0, stream>>>(
        reinterpret_cast<const Vec4T*>(input_data), reinterpret_cast<Vec4T*>(output_data), N, repeats);
    return;
  } else if (vectorized_size == 2) {
    using Vec2T = aligned_vector<T, 2>;
    _TileMemcpyKernelFromInput<<<blocksPerGrid, num_threads_per_block, 0, stream>>>(
        reinterpret_cast<const Vec2T*>(input_data), reinterpret_cast<Vec2T*>(output_data), N, repeats);
    return;
  }

  _TileMemcpyKernelFromInput<<<blocksPerGrid, num_threads_per_block, 0, stream>>>(input_data, output_data, N, repeats);
}

template <typename T>
__global__ void _TileBatchedMemcpyKernelFromOutput(const T* input_data, T* output_data,
                                                   const fast_divmod divmod_size_output_row,
                                                   const size_t size_input_row, const fast_divmod divmod_batch,
                                                   const fast_divmod divmod_size_input_row, const CUDA_LONG N) {
  CUDA_LONG start = num_elements_per_thread * num_threads_per_block * blockIdx.x + threadIdx.x;
  T value[num_elements_per_thread];
  CUDA_LONG id = start;
#pragma unroll
  for (int i = 0; i < num_elements_per_thread; ++i) {
    if (id < N) {
      int batch_idx, element_idx;
      divmod_size_output_row.divmod(id, batch_idx, element_idx);
      value[i] = input_data[divmod_batch.mod(batch_idx) * size_input_row + divmod_size_input_row.mod(element_idx)];
      id += num_threads_per_block;
    }
  }

  id = start;
#pragma unroll
  for (int i = 0; i < num_elements_per_thread; ++i) {
    if (id < N) {
      output_data[id] = value[i];
      id += num_threads_per_block;
    }
  }
}

// Input size is [batch, data], output size is [batch * batch_repeats, data * repeats_per_batch].
// Here size_input_row = data, size_output_row = data * repeats_per_batch,
// size_output_batch = batch * data * repeats_per_batch
template <typename T>
__global__ void _TileBatchedMemcpyKernelFromInput(const T* input_data, T* output_data,
                                                  const fast_divmod divmod_size_input_row,
                                                  const CUDA_LONG size_input_row, const CUDA_LONG size_output_row,
                                                  const CUDA_LONG size_output_batch, const size_t batch_repeats,
                                                  const size_t repeats_per_batch, const CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  T input_val = input_data[id];
  CUDA_LONG q, r;
  divmod_size_input_row.divmod(id, q, r);
  CUDA_LONG batch_offset = q * size_output_row + r;
  for (size_t i = 0; i < batch_repeats; ++i) {
    CUDA_LONG offset = batch_offset;
    for (size_t j = 0; j < repeats_per_batch; ++j) {
      output_data[offset] = input_val;
      offset += size_input_row;
    }
    batch_offset += size_output_batch;
  }
}

// Input size is [batch, data], output size is [batch * batch_repeats, data * repeats_per_batch].
// Here size_input_row = data, num_input_elements = batch * data
template <typename T>
void TileBatchedMemcpyImpl(hipStream_t stream, const T* input_data, T* output_data, const size_t size_input_row,
                           const size_t num_input_elements, const size_t batch_repeats,
                           const size_t repeats_per_batch) {
  // If the block number from input size is too small to fill all streaming multiprocessors,
  // it won't have perf gain to launch from inputs. In this case we will use the output based kernel.
  CUDA_LONG N;
  int blocksPerGrid;
  size_t vectorized_size =
      GetVectorizedSize<T>(num_input_elements, size_input_row, reinterpret_cast<uint64_t>(input_data),
                           reinterpret_cast<uint64_t>(output_data), N, blocksPerGrid);
  if (blocksPerGrid < 128) {
    N = static_cast<CUDA_LONG>(num_input_elements * batch_repeats * repeats_per_batch);
    blocksPerGrid = CeilDiv(N, num_threads_per_block * num_elements_per_thread);
    _TileBatchedMemcpyKernelFromOutput<<<blocksPerGrid, num_threads_per_block, 0, stream>>>(
        input_data, output_data, fast_divmod(static_cast<int>(size_input_row * repeats_per_batch)), size_input_row,
        fast_divmod(static_cast<int>(num_input_elements / size_input_row)),
        fast_divmod(static_cast<int>(size_input_row)), N);
    return;
  }

  CUDA_LONG size_input_row_vec = static_cast<CUDA_LONG>(size_input_row);
  if (vectorized_size == 4) {
    using Vec4T = aligned_vector<T, 4>;
    size_input_row_vec /= 4;
    _TileBatchedMemcpyKernelFromInput<<<blocksPerGrid, num_threads_per_block, 0, stream>>>(
        reinterpret_cast<const Vec4T*>(input_data), reinterpret_cast<Vec4T*>(output_data),
        fast_divmod(size_input_row_vec), size_input_row_vec,
        size_input_row_vec * static_cast<CUDA_LONG>(repeats_per_batch), N * static_cast<CUDA_LONG>(repeats_per_batch),
        batch_repeats, repeats_per_batch, N);
    return;
  } else if (vectorized_size == 2) {
    using Vec2T = aligned_vector<T, 2>;
    size_input_row_vec /= 2;
    _TileBatchedMemcpyKernelFromInput<<<blocksPerGrid, num_threads_per_block, 0, stream>>>(
        reinterpret_cast<const Vec2T*>(input_data), reinterpret_cast<Vec2T*>(output_data),
        fast_divmod(size_input_row_vec), size_input_row_vec,
        size_input_row_vec * static_cast<CUDA_LONG>(repeats_per_batch), N * static_cast<CUDA_LONG>(repeats_per_batch),
        batch_repeats, repeats_per_batch, N);
    return;
  }

  _TileBatchedMemcpyKernelFromInput<<<blocksPerGrid, num_threads_per_block, 0, stream>>>(
      input_data, output_data, fast_divmod(size_input_row_vec), size_input_row_vec,
      size_input_row_vec * static_cast<CUDA_LONG>(repeats_per_batch), N * static_cast<CUDA_LONG>(repeats_per_batch),
      batch_repeats, repeats_per_batch, N);
}

#define SPECIALIZED_IMPL(T)                                                                                           \
  template void TileImpl<T>(hipStream_t stream, const size_t shape_rank, const TArray<fast_divmod>& fdm_input_shape, \
                            const TArray<int64_t>& input_stride, const T* input_data,                                 \
                            const TArray<fast_divmod>& fdm_output_strides, T* output_data, const size_t N);           \
  template void TileMemcpyImpl<T>(hipStream_t stream, const T* input_data, T* output_data,                           \
                                  const size_t num_input_elements, const size_t repeats);                             \
  template void TileBatchedMemcpyImpl<T>(hipStream_t stream, const T* input_data, T* output_data,                    \
                                         const size_t size_input_row, const size_t num_input_elements,                \
                                         const size_t batch_repeats, const size_t repeats_per_batch);

SPECIALIZED_IMPL(float)
SPECIALIZED_IMPL(double)
SPECIALIZED_IMPL(half)

}  // namespace cuda
}  // namespace onnxruntime
