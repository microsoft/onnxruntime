#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/tensor/scatter_nd_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/atomic/common.cuh"

namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void _ScatterNDKernel(
    T* output_data,
    const size_t num_indices,
    const int64_t* indices_data,
    const int64_t last_index_dimension,
    ElementCountsAndInputDimsSpanOrGpu element_counts_and_input_dims,
    const T* updates_data,
    const size_t num_updates_elements) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, num_indices);

  // Compute the base offset into the output data
  int64_t data_offset = 0;

  size_t indices_start = last_index_dimension * id;
  size_t indices_end = indices_start + last_index_dimension;
  for (size_t i = indices_start; i < indices_end; ++i) {
    int64_t index = indices_data[i];

    int64_t element_count_dim = element_counts_and_input_dims.gpu_ptr == nullptr
                                    ? element_counts_and_input_dims.stack_ptr[i - indices_start]
                                    : element_counts_and_input_dims.gpu_ptr[i - indices_start];
    int64_t dim_value = element_counts_and_input_dims.gpu_ptr == nullptr
                            ? element_counts_and_input_dims.stack_ptr[i - indices_start + last_index_dimension]
                            : element_counts_and_input_dims.gpu_ptr[i - indices_start + last_index_dimension];

    // Clamp the index if out of range
    // This would have been an error in the CPU kernel, but throwing in the CUDA EP
    // is hard. This is the approach taken by other frameworks for out of bound indices
    // in their corresponding GPU backends as well.
    // index >= -dim_value && index < dim_value

    if (index >= 0) {
      if (index >= dim_value) {
        index = dim_value - 1;
      }
    } else {
      if (index < -dim_value) {
        index = 0;
      } else {
        index += dim_value;
      }
    }

    data_offset += (index * element_count_dim);
  }

  const T* updates_data_base = updates_data + num_updates_elements * id;
  T* output_data_base = output_data + data_offset;

  for (size_t i = 0; i < num_updates_elements; ++i) {
    output_data_base[i] = updates_data_base[i];
  }
}

Status ScatterNDImpl(
    hipStream_t stream,
    void* output_data,
    const size_t element_size,
    const size_t num_indices,
    const int64_t* indices_data,
    const int64_t last_index_dimension,
    const ElementCountsAndInputDimsSpanOrGpu& element_counts_and_input_dims,
    const void* updates_data,
    const size_t num_updates_elements) {
  if (num_indices == 0)
    return Status::OK();

  // Parallelize on number of indices
  int blocksPerGrid = static_cast<int>(ceil(static_cast<float>(num_indices) / GridDim::maxThreadsPerBlock));

  switch (element_size) {
    case sizeof(int8_t):
      _ScatterNDKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          reinterpret_cast<int8_t*>(output_data),
          num_indices,
          indices_data,
          last_index_dimension,
          element_counts_and_input_dims,
          reinterpret_cast<const int8_t*>(updates_data),
          num_updates_elements);
      break;

    case sizeof(int16_t):
      _ScatterNDKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          reinterpret_cast<int16_t*>(output_data),
          num_indices,
          indices_data,
          last_index_dimension,
          element_counts_and_input_dims,
          reinterpret_cast<const int16_t*>(updates_data),
          num_updates_elements);
      break;

    case sizeof(int32_t):
      _ScatterNDKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          reinterpret_cast<int32_t*>(output_data),
          num_indices,
          indices_data,
          last_index_dimension,
          element_counts_and_input_dims,
          reinterpret_cast<const int32_t*>(updates_data),
          num_updates_elements);
      break;

    case sizeof(int64_t):
      _ScatterNDKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          reinterpret_cast<int64_t*>(output_data),
          num_indices,
          indices_data,
          last_index_dimension,
          element_counts_and_input_dims,
          reinterpret_cast<const int64_t*>(updates_data),
          num_updates_elements);
      break;

    default:
      // Shouldn't hit this
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for ScatterND operator");
  }

  return Status::OK();
}

template <class T>
struct FuncAdd {
  __device__ __inline__ void operator()(T* start_addr, T value) const {
    atomic_add(start_addr, value);
  }
};

template <class T>
struct FuncMul {
  __device__ __inline__ void operator()(T* start_addr, T value) const {
    atomic_mul(start_addr, value);
  }
};

template <class T>
struct FuncMax {
  __device__ __inline__ void operator()(T* start_addr, T value) const {
    atomic_max(start_addr, value);
  }
};

template <class T>
struct FuncMin {
  __device__ __inline__ void operator()(T* start_addr, T value) const {
    atomic_min(start_addr, value);
  }
};

template <typename T, typename TFunc>
__global__ void _ScatterNDKernelReduction(
    T* output_data,
    const size_t num_indices,
    const int64_t* indices_data,
    const int64_t last_index_dimension,
    ElementCountsAndInputDimsSpanOrGpu element_counts_and_input_dims,
    const T* updates_data,
    const size_t num_updates_elements,
    const TFunc func) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, num_indices);

  // Compute the base offset into the output data
  int64_t data_offset = 0;

  size_t indices_start = last_index_dimension * id;
  size_t indices_end = indices_start + last_index_dimension;
  for (size_t i = indices_start; i < indices_end; ++i) {
    int64_t index = indices_data[i];

    int64_t element_count_dim = element_counts_and_input_dims.gpu_ptr == nullptr
                                    ? element_counts_and_input_dims.stack_ptr[i - indices_start]
                                    : element_counts_and_input_dims.gpu_ptr[i - indices_start];
    int64_t dim_value = element_counts_and_input_dims.gpu_ptr == nullptr
                            ? element_counts_and_input_dims.stack_ptr[i - indices_start + last_index_dimension]
                            : element_counts_and_input_dims.gpu_ptr[i - indices_start + last_index_dimension];

    // Clamp the index if out of range
    // This would have been an error in the CPU kernel, but throwing in the CUDA EP
    // is hard. This is the approach taken by other frameworks for out of bound indices
    // in their corresponding GPU backends as well.
    // index >= -dim_value && index < dim_value

    if (index >= 0) {
      if (index >= dim_value) {
        index = dim_value - 1;
      }
    } else {
      if (index < -dim_value) {
        index = 0;
      } else {
        index += dim_value;
      }
    }

    data_offset += (index * element_count_dim);
  }

  const T* updates_data_base = updates_data + num_updates_elements * id;
  T* output_data_base = output_data + data_offset;

  for (size_t i = 0; i < num_updates_elements; ++i) {
    func(output_data_base + i, updates_data_base[i]);
  }
}

template <typename T>
Status _ScatterNDType(
    hipStream_t stream,
    T* output_data,
    const size_t num_indices,
    const int64_t* indices_data,
    const int64_t last_index_dimension,
    const ElementCountsAndInputDimsSpanOrGpu& element_counts_and_input_dims,
    const T* updates_data,
    const size_t num_updates_elements,
    ScatterNDReduction reduction) {
  // Parallelize on number of indices
  int blocksPerGrid = static_cast<int>(ceil(static_cast<float>(num_indices) / GridDim::maxThreadsPerBlock));

  switch (reduction) {
    case ScatterNDReduction::Add:
      _ScatterNDKernelReduction<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          output_data,
          num_indices,
          indices_data,
          last_index_dimension,
          element_counts_and_input_dims,
          updates_data,
          num_updates_elements,
          FuncAdd<T>());
      break;
    case ScatterNDReduction::Mul:
      _ScatterNDKernelReduction<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          output_data,
          num_indices,
          indices_data,
          last_index_dimension,
          element_counts_and_input_dims,
          updates_data,
          num_updates_elements,
          FuncMul<T>());
      break;
    case ScatterNDReduction::Min:
      _ScatterNDKernelReduction<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          output_data,
          num_indices,
          indices_data,
          last_index_dimension,
          element_counts_and_input_dims,
          updates_data,
          num_updates_elements,
          FuncMin<T>());
      break;
    case ScatterNDReduction::Max:
      _ScatterNDKernelReduction<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          output_data,
          num_indices,
          indices_data,
          last_index_dimension,
          element_counts_and_input_dims,
          updates_data,
          num_updates_elements,
          FuncMax<T>());
      break;
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Reduction ", static_cast<int>(reduction), " not implemented for ScatterND operator.");
  }

  return Status::OK();
}

Status ScatterNDImplReduction(
    hipStream_t stream,
    void* output_data,
    const int32_t element_type,
    const size_t num_indices,
    const int64_t* indices_data,
    const int64_t last_index_dimension,
    const ElementCountsAndInputDimsSpanOrGpu& element_counts_and_input_dims,
    const void* updates_data,
    const size_t num_updates_elements,
    ScatterNDReduction reduction) {
  if (num_indices == 0)
    return Status::OK();

  switch (element_type) {
    case ONNX_NAMESPACE::TensorProto_DataType_FLOAT:
      return _ScatterNDType<float>(
          stream,
          reinterpret_cast<float*>(output_data),
          num_indices,
          indices_data,
          last_index_dimension,
          element_counts_and_input_dims,
          reinterpret_cast<const float*>(updates_data),
          num_updates_elements,
          reduction);
    case ONNX_NAMESPACE::TensorProto_DataType_FLOAT16:
      return _ScatterNDType<half>(
          stream,
          reinterpret_cast<half*>(output_data),
          num_indices,
          indices_data,
          last_index_dimension,
          element_counts_and_input_dims,
          reinterpret_cast<const half*>(updates_data),
          num_updates_elements,
          reduction);
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "element_type ", static_cast<int>(element_type), " not implemented for ScatterND operator.");
  }
}

}  // namespace cuda
}  // namespace onnxruntime
