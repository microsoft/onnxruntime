#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/tensor/scatter_nd_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/atomic/common.cuh"

namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void _ScatterNDKernel(
    T* output_data,
    const size_t num_indices,
    const int64_t* indices_data,
    const int64_t last_index_dimension,
    const int64_t* element_counts_and_input_dims,
    const T* updates_data,
    const size_t num_updates_elements) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, num_indices);

  // Compute the base offset into the output data
  int64_t data_offset = 0;

  size_t indices_start = last_index_dimension * id;
  size_t indices_end = indices_start + last_index_dimension;
  for (size_t i = indices_start; i < indices_end; ++i) {
    int64_t index = indices_data[i];

    int64_t element_count_dim = element_counts_and_input_dims[i - indices_start];
    int64_t dim_value = element_counts_and_input_dims[i - indices_start + last_index_dimension];

    // Clamp the index if out of range
    // This would have been an error in the CPU kernel, but throwing in the CUDA EP
    // is hard. This is the approach taken by other frameworks for out of bound indices
    // in their corresponding GPU backends as well.
    if (index < 0)
      index = 0;

    else if (index >= dim_value)
      index = dim_value - 1;

    data_offset += (index * element_count_dim);
  }

  const T* updates_data_base = updates_data + num_updates_elements * id;
  T* output_data_base = output_data + data_offset;

  for (size_t i = 0; i < num_updates_elements; ++i) {
    output_data_base[i] = updates_data_base[i];
  }
}

Status ScatterNDImpl(
    hipStream_t stream,
    void* output_data,
    const size_t element_size,
    const size_t num_indices,
    const int64_t* indices_data,
    const int64_t last_index_dimension,
    const int64_t* element_counts_and_input_dims,
    const void* updates_data,
    const size_t num_updates_elements) {
  if (num_indices == 0)
    return Status::OK();

  // Parallelize on number of indices
  int blocksPerGrid = static_cast<int>(ceil(static_cast<float>(num_indices) / GridDim::maxThreadsPerBlock));

  switch (element_size) {
    case sizeof(int8_t):
      _ScatterNDKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          reinterpret_cast<int8_t*>(output_data),
          num_indices,
          indices_data,
          last_index_dimension,
          element_counts_and_input_dims,
          reinterpret_cast<const int8_t*>(updates_data),
          num_updates_elements);
      break;

    case sizeof(int16_t):
      _ScatterNDKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          reinterpret_cast<int16_t*>(output_data),
          num_indices,
          indices_data,
          last_index_dimension,
          element_counts_and_input_dims,
          reinterpret_cast<const int16_t*>(updates_data),
          num_updates_elements);
      break;

    case sizeof(int32_t):
      _ScatterNDKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          reinterpret_cast<int32_t*>(output_data),
          num_indices,
          indices_data,
          last_index_dimension,
          element_counts_and_input_dims,
          reinterpret_cast<const int32_t*>(updates_data),
          num_updates_elements);
      break;

    case sizeof(int64_t):
      _ScatterNDKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          reinterpret_cast<int64_t*>(output_data),
          num_indices,
          indices_data,
          last_index_dimension,
          element_counts_and_input_dims,
          reinterpret_cast<const int64_t*>(updates_data),
          num_updates_elements);
      break;

    default:
      // Shouldn't hit this
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for ScatterND operator");
  }

  return Status::OK();
}

}  // namespace cuda
}  // namespace onnxruntime
