#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/atomic/common.cuh"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "scatter_elements_impl.h"
#ifdef ENABLE_TRAINING
#include "orttraining/training_ops/cuda/tensor/gather_elements_grad_impl.h"
#endif

namespace onnxruntime {
namespace cuda {

template <typename T, typename Tin, bool OUTERAXIS, typename FuncT>
__global__ void _ScatterElementsKernel2D(
    const int max_dim,  // max dim on the scattered axis
    const T* input_data,
    const Tin* indices_data,
    const int64_t indices_size,
    const fast_divmod indices_stride_row,
    const T* updates,
    const int64_t output_row_size,
    T* output_data,
    const FuncT& func) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(indices_index, indices_size);

  int row, col, data_idx;
  indices_stride_row.divmod(indices_index, row, col);
  int dim = (int)(indices_data[indices_index]);
  if (dim >= -max_dim && dim < max_dim) {
    if (dim < 0) dim += max_dim;
    if (OUTERAXIS) {
      data_idx = dim * output_row_size + col;
    } else {
      data_idx = row * output_row_size + dim;
    }

    func(output_data + data_idx, updates + indices_index);
  }
  // else invalid index
}

template <typename T, typename Tin, typename FuncT>
__global__ void _ScatterElementsKernel(
    const int rank,
    const T* input_data,
    const TArray<int64_t> input_dims,
    const TArray<int64_t> input_strides,
    const Tin* indices_data,
    const int64_t indices_size,
    const TArray<int64_t> indices_dims,
    const TArray<fast_divmod> indices_strides,
    const T* updates,
    const int axis,
    T* output_data,
    const FuncT& func) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(indices_index, indices_size);
  int dim, remain = indices_index;
  size_t data_idx = 0;
  for (int i = 0; i < rank; ++i) {
    indices_strides[i].divmod(remain, dim, remain);
    if (i == axis) {
      dim = (int)(indices_data[indices_index]);
      if (dim < -input_dims[i] || dim >= input_dims[i]) {
        return;  // Invalid index
      }
      if (dim < 0) dim += input_dims[i];
    }
    data_idx += input_strides[i] * dim;
  }
  func(output_data + data_idx, updates + indices_index);
}

// From the innermost axis (largest) check equality of dim value of input and indices.
// If same, merge it and continue. Otherwise, copy remaining. The scatter axis need
// to be keep.
static int CompactInputIndicesDims(
    int rank, int axis, int64_t* input_dims, int64_t* indices_dims,
    std::vector<int64_t>& eff_input_dims,
    std::vector<int64_t>& eff_indices_dims) {
  eff_input_dims.clear();
  eff_indices_dims.clear();

  bool could_continue_merge = true;
  if (axis < rank - 1) {
    eff_input_dims.push_back(1);
    eff_indices_dims.push_back(1);
    int i = rank - 1;
    for (; i > axis; --i) {
      if (input_dims[i] == indices_dims[i]) {
        eff_input_dims.back() *= input_dims[i];
        eff_indices_dims.back() *= indices_dims[i];
      } else {
        could_continue_merge = false;
        break;
      }
    }
    if (eff_input_dims.back() == 1) {
      eff_input_dims.pop_back();
      eff_indices_dims.pop_back();
    }
    if (!could_continue_merge) {
      for (; i > axis; --i) {
        eff_input_dims.push_back(input_dims[i]);
        eff_indices_dims.push_back(indices_dims[i]);
      }
    }
  }
  could_continue_merge = could_continue_merge && (input_dims[axis] == indices_dims[axis]);
  eff_input_dims.push_back(input_dims[axis]);
  eff_indices_dims.push_back(indices_dims[axis]);
  int new_axis = (int)(eff_input_dims.size());
  if (axis > 0) {
    if (!could_continue_merge) {
      eff_input_dims.push_back(1);
      eff_indices_dims.push_back(1);
      could_continue_merge = true;
    }
    int i = axis - 1;
    for (; i >= 0; --i) {
      if (input_dims[i] == indices_dims[i]) {
        eff_input_dims.back() *= input_dims[i];
        eff_indices_dims.back() *= indices_dims[i];
      } else {
        could_continue_merge = false;
        break;
      }
    }
    if (new_axis < (int)eff_indices_dims.size() && eff_input_dims.back() == 1) {
      eff_input_dims.pop_back();
      eff_indices_dims.pop_back();
    }
    if (!could_continue_merge) {
      for (; i >= 0; --i) {
        eff_input_dims.push_back(input_dims[i]);
        eff_indices_dims.push_back(indices_dims[i]);
      }
    }
  }
  new_axis = static_cast<int>(eff_input_dims.size()) - new_axis;
  std::reverse(eff_input_dims.begin(), eff_input_dims.end());
  std::reverse(eff_indices_dims.begin(), eff_indices_dims.end());
  return new_axis;
}

template <typename T, typename Tin, typename FuncT>
Status ScatterElementsImpl2D(
    const T* input_data,
    const std::vector<int64_t>& input_dims,
    const Tin* indices_data,
    const int64_t indices_size,
    const std::vector<int64_t>& indices_dims,
    const T* updates,
    const int axis,
    T* output_data,
    const FuncT& func) {
  int blocksPerGrid = gsl::narrow_cast<int>(CeilDiv(indices_size, GridDim::maxThreadsPerBlock));
  fast_divmod indices_stride_row(static_cast<int>(indices_dims[1]));
  if (axis == 0) {
    _ScatterElementsKernel2D<T, Tin, true, FuncT><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
        gsl::narrow_cast<int>(input_dims[0]), input_data,
        indices_data, indices_size, indices_stride_row,
        updates, input_dims[1], output_data, func);
  } else {
    _ScatterElementsKernel2D<T, Tin, false, FuncT><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
        gsl::narrow_cast<int>(input_dims[1]), input_data,
        indices_data, indices_size, indices_stride_row,
        updates, input_dims[1], output_data, func);
  }
  return Status::OK();
}

template <typename T, typename Tin, typename FuncT>
Status ScatterElementsImplInternal(
    const int rank,
    const T* input_data,
    const int64_t input_size,
    TArray<int64_t>& buffer_input_dims,
    TArray<int64_t>& buffer_input_strides,
    const Tin* indices_data,
    const int64_t indices_size,
    TArray<int64_t>& buffer_indices_dims,
    TArray<fast_divmod>& fdm_indices_strides,
    const T* updates,
    const int axis,
    T* output_data,
    const FuncT& func) {
  if (input_data != output_data) {
    CUDA_RETURN_IF_ERROR(hipMemcpyAsync(output_data, input_data, input_size * sizeof(T), hipMemcpyDeviceToDevice, 0));
  }

  if (indices_size > 0) {
    std::vector<int64_t> eff_input_dims;
    std::vector<int64_t> eff_indices_dims;
    int new_axis = CompactInputIndicesDims(
        rank, axis, buffer_input_dims.Data(), buffer_indices_dims.Data(), eff_input_dims, eff_indices_dims);
    if (eff_input_dims.size() == 2) {
      return ScatterElementsImpl2D(
          input_data, eff_input_dims, indices_data, indices_size, eff_indices_dims, updates, new_axis, output_data,
          func);
    }

    int blocksPerGrid = gsl::narrow_cast<int>(CeilDiv(indices_size, GridDim::maxThreadsPerBlock));
    _ScatterElementsKernel<T, Tin><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
        rank, input_data, buffer_input_dims, buffer_input_strides,
        indices_data, indices_size, buffer_indices_dims, fdm_indices_strides,
        updates, axis, output_data, func);
  }
  return Status::OK();
}

template <class T>
struct Func_Assignment {
  __device__ __inline__ void operator()(T* a, const T* b) const {
    *a = *b;
  }
};

template <typename T, typename Tin>
Status ScatterElementsImpl(
    const int rank,
    const T* input_data,
    const int64_t input_size,
    TArray<int64_t>& buffer_input_dims,
    TArray<int64_t>& buffer_input_strides,
    const Tin* indices_data,
    const int64_t indices_size,
    TArray<int64_t>& buffer_indices_dims,
    TArray<fast_divmod>& fdm_indices_strides,
    const T* updates,
    const int axis,
    T* output_data) {
  return ScatterElementsImplInternal(rank, input_data, input_size, buffer_input_dims,
                                     buffer_input_strides, indices_data, indices_size, buffer_indices_dims, fdm_indices_strides,
                                     updates, axis, output_data, Func_Assignment<T>());
}

#define SCATTER_ELEMENTS_SPECIALIZED_TINDEX_IMPL(T, TIndex) \
  template Status ScatterElementsImpl<T, TIndex>(           \
      const int rank,                                       \
      const T* input_data,                                  \
      const int64_t input_size,                             \
      TArray<int64_t>& buffer_input_dims,                   \
      TArray<int64_t>& buffer_input_strides,                \
      const TIndex* indices_data,                           \
      const int64_t indices_size,                           \
      TArray<int64_t>& buffer_indices_dims,                 \
      TArray<fast_divmod>& indices_strides,                 \
      const T* updates,                                     \
      const int axis,                                       \
      T* output_data)

#define SCATTER_ELEMENTS_SPECIALIZED_IMPL(T)            \
  SCATTER_ELEMENTS_SPECIALIZED_TINDEX_IMPL(T, int32_t); \
  SCATTER_ELEMENTS_SPECIALIZED_TINDEX_IMPL(T, int64_t);

SCATTER_ELEMENTS_SPECIALIZED_IMPL(int8_t)
SCATTER_ELEMENTS_SPECIALIZED_IMPL(int16_t)
SCATTER_ELEMENTS_SPECIALIZED_IMPL(int32_t)
SCATTER_ELEMENTS_SPECIALIZED_IMPL(int64_t)
SCATTER_ELEMENTS_SPECIALIZED_IMPL(uint8_t)
SCATTER_ELEMENTS_SPECIALIZED_IMPL(uint16_t)
SCATTER_ELEMENTS_SPECIALIZED_IMPL(uint32_t)
SCATTER_ELEMENTS_SPECIALIZED_IMPL(uint64_t)
SCATTER_ELEMENTS_SPECIALIZED_IMPL(half)
SCATTER_ELEMENTS_SPECIALIZED_IMPL(float)
SCATTER_ELEMENTS_SPECIALIZED_IMPL(double)
SCATTER_ELEMENTS_SPECIALIZED_IMPL(bool)

#ifdef ENABLE_TRAINING

template <class T>
struct Func_AtomicAdd {
  __device__ __inline__ void operator()(T* a, const T* b) const {
    atomic_add(a, *b);
  }
};

template <typename T, typename Tin>
Status GatherElementsGradImpl(
    const int rank,
    TArray<int64_t>& buffer_input_dims,
    TArray<int64_t>& buffer_input_strides,
    const Tin* indices_data,
    const int64_t indices_size,
    TArray<int64_t>& buffer_indices_dims,
    TArray<fast_divmod>& fdm_indices_strides,
    const T* updates,
    const int axis,
    T* output_data) {
  // Give output_data as the input_data parameter by intention,
  // to skip input_data copy, which is not applicable for GatherElementsGrad.
  return ScatterElementsImplInternal(rank, output_data, 0,
                                     buffer_input_dims, buffer_input_strides, indices_data,
                                     indices_size, buffer_indices_dims, fdm_indices_strides,
                                     updates, axis, output_data, Func_AtomicAdd<T>());
}

#define GATHER_ELEMENTS_GRAD_SPECIALIZED_TINDEX_IMPL(T, TIndex) \
  template Status GatherElementsGradImpl<T, TIndex>(            \
      const int rank,                                           \
      TArray<int64_t>& buffer_input_dims,                       \
      TArray<int64_t>& buffer_input_strides,                    \
      const TIndex* indices_data,                               \
      const int64_t indices_size,                               \
      TArray<int64_t>& buffer_indices_dims,                     \
      TArray<fast_divmod>& indices_strides,                     \
      const T* updates,                                         \
      const int axis,                                           \
      T* output_data)

#define GATHER_ELEMENTS_GRAD_SPECIALIZED_SCATTER_ADD_IMPL(T) \
  GATHER_ELEMENTS_GRAD_SPECIALIZED_TINDEX_IMPL(T, int32_t);  \
  GATHER_ELEMENTS_GRAD_SPECIALIZED_TINDEX_IMPL(T, int64_t);

GATHER_ELEMENTS_GRAD_SPECIALIZED_SCATTER_ADD_IMPL(half)
GATHER_ELEMENTS_GRAD_SPECIALIZED_SCATTER_ADD_IMPL(float)
GATHER_ELEMENTS_GRAD_SPECIALIZED_SCATTER_ADD_IMPL(double)

#endif

}  // namespace cuda
}  // namespace onnxruntime
