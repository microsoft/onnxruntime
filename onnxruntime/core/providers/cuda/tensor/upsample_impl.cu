#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "upsample_impl.h"

namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void _UpampleNearestKernel(const size_t rank,
                                      const TArray<int64_t> input_pitches,
                                      const TArray<fast_divmod> output_div_pitches,
                                      const TArray<fast_divmod> scales_div,
                                      const T* input_data,
                                      T* output_data,
                                      const size_t N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  CUDA_LONG input_index = 0;
  CUDA_LONG output_index = id;

  int div, mod;
  for (int dim = 0; dim < rank; ++dim) {
    output_div_pitches[dim].divmod(output_index, div, mod);
    output_index = mod;
    if (scales_div[dim].d_ != 1 && div > 0) {
      scales_div[dim].divmod(div, div, mod); 
    }
    input_index += input_pitches[dim] * div;
  }
  output_data[id] = input_data[input_index];
}

// The following method supports a 4-D input in 'Linear mode' 
// that amounts to 'Bilinear' Upsampling/Resizing in the sense that it assumes
// the scale values for the outermost 2 dimensions are 1.
// This is the common use-case where the 4-D input (batched multi-channel images) 
// is usually of shape [N, C, H, W] and the scales are [1.0, 1.0, height_scale, width_scale]
template <typename T>
__global__ void _UpampleBilinear4DInputKernel(const int64_t input_dim2,
                                       const TArray<int64_t> input_pitches,
                                       const TArray<fast_divmod> output_div_pitches,
                                       const TArray<fast_divmod> scales_div,
                                       const T* input_data,
                                       T* output_data,
                                       const size_t N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  CUDA_LONG input_index = 0;

  // For bilinear mode, scales[0]=scales[1]=1
  int mod;
  int index_of_dim0, index_of_dim1, index_of_dim2, index_of_dim3;
  output_div_pitches[0].divmod(id, index_of_dim0, mod);
  output_div_pitches[1].divmod(mod, index_of_dim1, mod);
  output_div_pitches[2].divmod(mod, index_of_dim2, mod);
  index_of_dim3 = mod;
  int index_of_input_dim2, index_of_input_dim3, x_offset, y_offset;
  scales_div[2].divmod(index_of_dim2, index_of_input_dim2, y_offset);
  scales_div[3].divmod(index_of_dim3, index_of_input_dim3, x_offset);

  input_index = index_of_dim0 * input_pitches[0] +
                index_of_dim1 * input_pitches[1] +
                index_of_input_dim2 * input_pitches[2] +
                index_of_input_dim3;
  
  T x00 = input_data[input_index];
  T x10, x01, x11;

  bool end_of_dim2 = false;
  if (index_of_input_dim2 == (input_dim2 - 1)) {
    // It's the end in dimension 2
    x01 = x00;
    end_of_dim2 = true;
  } else {
    x01 = input_data[input_index + input_pitches[2]];
  }

  if (index_of_input_dim3 == (input_pitches[2] - 1)) {
    // It's the end in dimension 3
    x10 = x00;
    x11 = x01;
  }
  else {
    x10 = input_data[input_index + 1];
    x11 = end_of_dim2 ? x10 : input_data[input_index + input_pitches[2] + 1];
  }

  T y_offset_T = static_cast<T>(y_offset);
  T x_offset_T = static_cast<T>(x_offset);
  T scales_div2_T = static_cast<T>(scales_div[2].d_);
  T scales_div3_T = static_cast<T>(scales_div[3].d_);
  T y0 = x00 + static_cast<T>(y_offset_T * (x01 - x00) / scales_div2_T);
  T y1 = x10 + static_cast<T>(y_offset_T * (x11 - x10) / scales_div2_T);

  output_data[id] = y0 + static_cast<T>(x_offset_T * (y1 - y0) / scales_div3_T);
}

// The following method supports a 2-D input in 'Linear mode'
template <typename T>
__global__ void _UpampleBilinear2DInputKernel(const int64_t input_dim0,
                                              const TArray<int64_t> input_pitches,
                                              const TArray<fast_divmod> output_div_pitches,
                                              const TArray<fast_divmod> scales_div,
                                              const T* input_data,
                                              T* output_data,
                                              const size_t N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  CUDA_LONG input_index = 0;

  int mod;
  int index_of_dim0, index_of_dim1;
  output_div_pitches[0].divmod(id, index_of_dim0, mod);
  index_of_dim1 = mod;
  int index_of_input_dim0, index_of_input_dim1, x_offset, y_offset;
  scales_div[0].divmod(index_of_dim0, index_of_input_dim0, y_offset);
  scales_div[1].divmod(index_of_dim1, index_of_input_dim1, x_offset);

  input_index = index_of_input_dim0 * input_pitches[0] + index_of_input_dim1;

  T x00 = input_data[input_index];
  T x10, x01, x11;

  bool end_of_dim0 = false;
  if (index_of_input_dim0 == (input_dim0 - 1)) {
    // It's the end in dimension 0
    x01 = x00;
    end_of_dim0 = true;
  } else {
    x01 = input_data[input_index + input_pitches[0]];
  }

  if (index_of_input_dim1 == (input_pitches[0] - 1)) {
    // It's the end in dimension 1
    x10 = x00;
    x11 = x01;
  } else {
    x10 = input_data[input_index + 1];
    x11 = end_of_dim0 ? x10 : input_data[input_index + input_pitches[0] + 1];
  }

  T y_offset_T = static_cast<T>(y_offset);
  T x_offset_T = static_cast<T>(x_offset);
  T scales_div0_T = static_cast<T>(scales_div[0].d_);
  T scales_div1_T = static_cast<T>(scales_div[1].d_);
  T y0 = x00 + static_cast<T>(y_offset_T * (x01 - x00) / scales_div0_T);
  T y1 = x10 + static_cast<T>(y_offset_T * (x11 - x10) / scales_div0_T);

  output_data[id] = y0 + static_cast<T>(x_offset_T * (y1 - y0) / scales_div1_T);
}

template <typename T>
void UpampleImpl(hipStream_t stream,
                 const onnxruntime::UpsampleMode upsample_mode,
                 const size_t rank,
                 const int64_t input_dim2,
                 const TArray<int64_t>& input_pitches,
                 const TArray<fast_divmod>& output_div_pitches,
                 const TArray<fast_divmod>& scales_div,
                 const T* input_data,
                 T* output_data,
                 const size_t N) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  if (onnxruntime::UpsampleMode::NN == upsample_mode) {
    _UpampleNearestKernel<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
        rank, input_pitches, output_div_pitches, scales_div,
        input_data, output_data, N);
  } else if (onnxruntime::UpsampleMode::LINEAR == upsample_mode && rank == 4) {
    _UpampleBilinear4DInputKernel<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
        input_dim2, input_pitches, output_div_pitches, scales_div,
        input_data, output_data, N);
  } else if (onnxruntime::UpsampleMode::LINEAR == upsample_mode && rank == 2) {
    _UpampleBilinear2DInputKernel<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
        input_dim2, input_pitches, output_div_pitches, scales_div,
        input_data, output_data, N);
  }
}

#define SPECIALIZED_IMPL(T)                                                     \
  template void UpampleImpl<T>(hipStream_t stream,                       \
                               const onnxruntime::UpsampleMode upsample_mode,   \
                               const size_t rank,                               \
                               const int64_t input_dim2,                        \
                               const TArray<int64_t>& input_pitches,                    \
                               const TArray<fast_divmod>& output_div_pitches,           \
                               const TArray<fast_divmod>& scales_div,                   \
                               const T* input_data,                             \
                               T* output_data,                                  \
                               const size_t N);

SPECIALIZED_IMPL(float)
SPECIALIZED_IMPL(double)
SPECIALIZED_IMPL(half)
SPECIALIZED_IMPL(int32_t)
SPECIALIZED_IMPL(uint8_t)

}  // namespace cuda
}  // namespace onnxruntime
