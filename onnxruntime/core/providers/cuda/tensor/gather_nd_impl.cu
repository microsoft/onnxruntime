#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/tensor/gather_nd_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/atomic/common.cuh"

namespace onnxruntime {
namespace cuda {

template <typename TIndex>
__global__ void _ComputeSliceOffsetsKernel(
    const int64_t batch_dims,
    const TArray<int64_t> input_dims,
    const size_t num_slices,
    const size_t num_slices_per_batch,
    const size_t input_batch_stride,
    const size_t num_slice_dims,
    const int64_t* const sizes_from_slice_dims_data,  // num_slice_dims elements
    const TIndex* const indices_data,                 // num_slices * num_slice_dims elements
    int64_t* const input_slice_offsets_data) {        // num_slices elements
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(slice_idx, num_slices)

  const size_t batch_idx = slice_idx / num_slices_per_batch;
  const size_t base_offset = batch_idx * input_batch_stride;

  const TIndex* const slice_indices = indices_data + slice_idx * num_slice_dims;
  size_t relative_slice_offset = 0;
  for (size_t dim_idx = 0; dim_idx < num_slice_dims; ++dim_idx) {
    int64_t index = static_cast<int64_t>(slice_indices[dim_idx]);
    const size_t input_dim_idx = batch_dims + dim_idx;
    CUDA_KERNEL_ASSERT(index >= -input_dims[input_dim_idx] && index < input_dims[input_dim_idx]);
    if (index < 0) index += input_dims[input_dim_idx];

    relative_slice_offset += index * sizes_from_slice_dims_data[dim_idx];
  }

  input_slice_offsets_data[slice_idx] = base_offset + relative_slice_offset;
}

template <typename T>
__global__ void _GatherNDKernel(
    const size_t num_slices,
    const T* input_data,
    T* output_data,
    const size_t slice_size,
    const int64_t* slice_offsets) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(i, num_slices * slice_size)
  uint64_t slice_offset = slice_offsets[i / slice_size];
  output_data[i] = input_data[slice_offset + i % slice_size];
};

template <typename TIndex>
void ComputeSliceOffsetsImpl(
    const int64_t batch_dims,
    const TArray<int64_t> input_dims,
    const size_t num_slices,
    const size_t num_slices_per_batch,
    const size_t input_batch_stride,
    const size_t num_slice_dims,
    const int64_t* const sizes_from_slice_dims_data,  // num_slice_dims elements
    const TIndex* const indices_data,                 // num_slices * num_slice_dims elements
    int64_t* const input_slice_offsets_data) {        // num_slices elements
  const auto blocks_per_grid = CeilDiv(num_slices, GridDim::maxThreadsPerBlock);
  _ComputeSliceOffsetsKernel<<<blocks_per_grid, GridDim::maxThreadsPerBlock>>>(
      batch_dims,
      input_dims,
      num_slices,
      num_slices_per_batch,
      input_batch_stride,
      num_slice_dims,
      sizes_from_slice_dims_data,
      indices_data,
      input_slice_offsets_data);
}

template <typename T>
void GatherNDImpl(
    const size_t num_slices,
    const void* input_data,
    void* output_data,
    const size_t slice_size,
    const int64_t* input_slice_offsets_data) {
  const auto blocks_per_grid = CeilDiv(num_slices * slice_size, GridDim::maxThreadsPerBlock);
  _GatherNDKernel<T><<<blocks_per_grid, GridDim::maxThreadsPerBlock, 0>>>(
      num_slices, static_cast<const T*>(input_data), static_cast<T*>(output_data), slice_size, input_slice_offsets_data);
}

#define SPECIALIZED_COMPUTE_SLICE_OFFSETS_IMPL(TIndex) \
  template void ComputeSliceOffsetsImpl<TIndex>(       \
      const int64_t batch_dims,                        \
      const TArray<int64_t> input_dims,                \
      const size_t num_slices,                         \
      const size_t num_slices_per_batch,               \
      const size_t input_batch_stride,                 \
      const size_t num_slice_dims,                     \
      const int64_t* const sizes_from_slice_dims_data, \
      const TIndex* const indices_data,                \
      int64_t* const input_slice_offsets_data);

#define SPECIALIZED_IMPL(T) \
  template void GatherNDImpl<T>(const size_t num_slices, const void* input_data, void* output_data, const size_t slice_size, const int64_t* input_slice_offsets_data);

SPECIALIZED_COMPUTE_SLICE_OFFSETS_IMPL(int32_t)
SPECIALIZED_COMPUTE_SLICE_OFFSETS_IMPL(int64_t)

SPECIALIZED_IMPL(float)
SPECIALIZED_IMPL(int64_t)
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
SPECIALIZED_IMPL(half)
SPECIALIZED_IMPL(double)
#endif
#if CUDA_VERSION >= 11000 && (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
SPECIALIZED_IMPL(hip_bfloat16)
#endif

}  // namespace cuda
}  // namespace onnxruntime
