#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "gather_impl.h"

namespace onnxruntime {
namespace cuda {

__host__ __device__ inline int64_t GetIndexValue(const void* index_data, size_t index_element_size, size_t offset) {
  switch (index_element_size) {
    case sizeof(int32_t):
      return *(reinterpret_cast<const int32_t*>(index_data) + offset);
      break;
    case sizeof(int64_t):
      return *(reinterpret_cast<const int64_t*>(index_data) + offset);
      break;
    default:
      break;
  }
  // What is a sensible thing to do here?
  assert(false);
  return std::numeric_limits<int64_t>::max();
}

template <typename T>
__global__ void _GatherKernel(
    const int64_t input_block_size,
    const int64_t indices_max,
    const fast_divmod output_block_size,
    const fast_divmod block_size,
    const void* indices_data,
    const size_t index_element_size,
    const T* input_data,
    T* output_data,
    const CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  CUDA_LONG input_index = 0;
  int input_block_index, block_offset;
  output_block_size.divmod(id, input_block_index, block_offset);
  int indices_index, offset;
  block_size.divmod(block_offset, indices_index, offset);
  int64_t idx = GetIndexValue(indices_data, index_element_size, indices_index);
  idx = idx < 0 ? idx + indices_max : idx;
  if (idx < 0 || idx >= indices_max) {
    output_data[id] = 0;
    return;
  }

  input_index = input_block_index * input_block_size + idx * block_size.d_ + offset;
  output_data[id] = input_data[input_index];
}

void GatherImpl(
    const int64_t input_block_size,
    const int64_t indices_max,
    const fast_divmod& output_block_size,
    const fast_divmod& block_size,
    const void* indices_data,
    size_t index_element_size,
    const void* input_data,
    size_t element_size,
    void* output_data,
    const size_t N) {

  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));

  switch (element_size) {
    case sizeof(int8_t): {
      using CudaType = typename ToCudaType<int8_t>::MappedType;
      _GatherKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          input_block_size, indices_max, output_block_size, block_size, indices_data, index_element_size,
          reinterpret_cast<const CudaType*>(input_data), reinterpret_cast<CudaType*>(output_data), (CUDA_LONG)N);

    } break;
    case sizeof(int16_t): {
      using CudaType = typename ToCudaType<int16_t>::MappedType;
      _GatherKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          input_block_size, indices_max, output_block_size, block_size, indices_data, index_element_size,
          reinterpret_cast<const CudaType*>(input_data), reinterpret_cast<CudaType*>(output_data), (CUDA_LONG)N);

    } break;
    case sizeof(int32_t): {
      using CudaType = typename ToCudaType<int32_t>::MappedType;
      _GatherKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          input_block_size, indices_max, output_block_size, block_size, indices_data, index_element_size,
          reinterpret_cast<const CudaType*>(input_data), reinterpret_cast<CudaType*>(output_data), (CUDA_LONG)N);

    } break;
    case sizeof(int64_t): {
      using CudaType = typename ToCudaType<int64_t>::MappedType;
      _GatherKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          input_block_size, indices_max, output_block_size, block_size, indices_data, index_element_size,
          reinterpret_cast<const CudaType*>(input_data), reinterpret_cast<CudaType*>(output_data), (CUDA_LONG)N);

    } break;

    default:
      ORT_THROW("Unsupported element size by the Gather CUDA kernel");
  }
}

}  // namespace cuda
}  // namespace onnxruntime
