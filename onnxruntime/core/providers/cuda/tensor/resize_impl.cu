#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/tensor/resize_impl.h"

namespace onnxruntime {
namespace cuda {

using onnxruntime::ResizeCoordinateTransformationMode;
using onnxruntime::ResizeNearestMode;
using onnxruntime::UpsampleMode;

struct NearestPixel_SIMPLE {
  __device__ __forceinline__ int operator() (float x_original, bool is_down_sampling) const {
    if (is_down_sampling) {
      return static_cast<int>(_Ceil(x_original));
    }
    return static_cast<int>(x_original);
  }
};

struct NearestPixel_ROUND_PREFER_FLOOR {
  __device__ __forceinline__ int operator() (float x_original, bool) const {
    if (x_original == static_cast<int>(x_original) + 0.5f) {
      return static_cast<int>(_Floor(x_original));
    }
    return static_cast<int>(roundf(x_original));
  }
};

struct NearestPixel_ROUND_PREFER_CEIL {
  __device__ __forceinline__ int operator() (float x_original, bool) const {
    return static_cast<int>(roundf(x_original));
  }
};

struct NearestPixel_FLOOR {
  __device__ __forceinline__ int operator() (float x_original, bool) const {
    return static_cast<int>(_Floor(x_original));
  }
};

struct NearestPixel_CEIL {
  __device__ __forceinline__ int operator() (float x_original, bool) const {
    return static_cast<int>(_Ceil(x_original));
  }
};

struct TransformCoordinate_ASYMMETRIC {
  __device__ __forceinline__ float operator() (float x_resized, float x_scale, float, float, float, float) const {
    return x_resized / x_scale;
  }
};

struct TransformCoordinate_HALF_PIXEL {
  __device__ __forceinline__ float operator() (float x_resized, float x_scale, float, float, float, float) const {
    return ((x_resized + 0.5f) / x_scale) - 0.5f;
  }
};

struct TransformCoordinate_PYTORCH_HALF_PIXEL {
  __device__ __forceinline__ float operator() (float x_resized, float x_scale, float length_resized, float, float, float) const {
    return length_resized > 1 ? (x_resized + 0.5f) / x_scale - 0.5f : 0.0f;
  }
};

struct TransformCoordinate_TF_HALF_PIXEL_FOR_NN {
  __device__ __forceinline__ float operator() (float x_resized, float x_scale, float, float, float, float) const {
    return (x_resized + 0.5f) / x_scale;
  }
};

struct TransformCoordinate_ALIGN_CORNERS {
  __device__ __forceinline__ float operator() (float x_resized, float, float length_resized, float length_original, float, float) const {
    return length_resized == 1 ? 0 : x_resized * (length_original - 1) / (length_resized - 1);
  }
};

struct TransformCoordinate_TF_CROP_AND_RESIZE {
  __device__ __forceinline__ float operator() (float x_resized, float, float length_resized, float length_original, float roi_start, float roi_end) const {
    auto orig = length_resized > 1
      ? roi_start * (length_original - 1) + (x_resized * (roi_end - roi_start) * (length_original - 1)) / (length_resized - 1)
      : 0.5 * (roi_start + roi_end) * (length_original - 1);
    return static_cast<float>(orig);
  }
};

#define CASE_TYPE_USING_HINT(enum_type, type, HINT, ...) \
  case enum_type: {                                      \
    using HINT = type;                                   \
    return __VA_ARGS__();                                \
  }

#define CASE_TYPE_COORD(enum_type, type, ...) \
  CASE_TYPE_USING_HINT(enum_type, type, coord_t, __VA_ARGS__)

#define DISPATCH_RESIZE_COORDINATE_TRANSFORMATION_MODE(TYPE, ...)                                                                      \
  [&] {                                                                                                                                \
    const auto& the_type = TYPE;                                                                                                       \
    /* don't use TYPE again in case it is an expensive or side-effect op */                                                            \
    switch (the_type) {                                                                                                                \
      CASE_TYPE_COORD(ResizeCoordinateTransformationMode::HALF_PIXEL,           TransformCoordinate_HALF_PIXEL, __VA_ARGS__)           \
      CASE_TYPE_COORD(ResizeCoordinateTransformationMode::ASYMMETRIC,           TransformCoordinate_ASYMMETRIC, __VA_ARGS__)           \
      CASE_TYPE_COORD(ResizeCoordinateTransformationMode::PYTORCH_HALF_PIXEL,   TransformCoordinate_PYTORCH_HALF_PIXEL, __VA_ARGS__)   \
      CASE_TYPE_COORD(ResizeCoordinateTransformationMode::ALIGN_CORNERS,        TransformCoordinate_ALIGN_CORNERS, __VA_ARGS__)        \
      CASE_TYPE_COORD(ResizeCoordinateTransformationMode::TF_HALF_PIXEL_FOR_NN, TransformCoordinate_TF_HALF_PIXEL_FOR_NN, __VA_ARGS__) \
      CASE_TYPE_COORD(ResizeCoordinateTransformationMode::TF_CROP_AND_RESIZE,   TransformCoordinate_TF_CROP_AND_RESIZE, __VA_ARGS__)   \
      default:                                                                                                                         \
        ORT_THROW("unknown ResizeCoordinateTransformationMode");                                                                       \
    }                                                                                                                                  \
  }()

#define CASE_TYPE_NEAREST(enum_type, type, ...) \
  CASE_TYPE_USING_HINT(enum_type, type, nearest_t, __VA_ARGS__)

#define DISPATCH_RESIZE_NEAREST_MODE(TYPE, ...)                                                              \
  [&] {                                                                                                      \
    const auto& the_type = TYPE;                                                                             \
    /* don't use TYPE again in case it is an expensive or side-effect op */                                  \
    switch (the_type) {                                                                                      \
      CASE_TYPE_NEAREST(ResizeNearestMode::SIMPLE,             NearestPixel_SIMPLE, __VA_ARGS__)             \
      CASE_TYPE_NEAREST(ResizeNearestMode::ROUND_PREFER_FLOOR, NearestPixel_ROUND_PREFER_FLOOR, __VA_ARGS__) \
      CASE_TYPE_NEAREST(ResizeNearestMode::ROUND_PREFER_CEIL,  NearestPixel_ROUND_PREFER_CEIL, __VA_ARGS__)  \
      CASE_TYPE_NEAREST(ResizeNearestMode::FLOOR,              NearestPixel_FLOOR, __VA_ARGS__)              \
      CASE_TYPE_NEAREST(ResizeNearestMode::CEIL,               NearestPixel_CEIL, __VA_ARGS__)               \
      default:                                                                                               \
        ORT_THROW("unknown ResizeNearestMode");                                                              \
    }                                                                                                        \
  }()

struct NearestMappingInfo {
  int origin_;
  int extrapolate_;
};

template <typename T, typename CudaFunctionOriginalCoordinate, typename CudaFunctionNearestPixel>
__global__ void _ResizeNearestMappingKernel2D(
    const int input_height, const int input_width,
    const int output_height, const int output_width,
    const float scales_height, const float scales_width,
    const float roi_start_height, const float roi_end_height,
    const float roi_start_width, const float roi_end_width,
    const bool extrapolation_enabled,
    const CudaFunctionOriginalCoordinate& transform_coordinate,
    const CudaFunctionNearestPixel& calc_nearest_pixel,
    NearestMappingInfo* dims_mapping) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, output_height + output_width);
  if (id >= 0 && id < output_height) {  // for Height
    int dim = id;

    // only apply co-ordinate transformation if scale != 1.0
    if (scales_height == 1.0f) {
        dims_mapping[id].extrapolate_ = 0;
    } else {
      float orig_coord = transform_coordinate(static_cast<float>(dim), scales_height, static_cast<float>(output_height),
                                              static_cast<float>(input_height), roi_start_height, roi_end_height);
      dims_mapping[id].extrapolate_ = static_cast<int>(
          extrapolation_enabled && (orig_coord < 0.f || orig_coord > static_cast<float>(input_height - 1)));
      dim = calc_nearest_pixel(orig_coord, scales_height < 1);
      if (dim >= input_height) dim = input_height - 1;
      if (dim < 0) dim = 0;
    }

    dims_mapping[id].origin_ = dim;
  } else {
    int dim = id - output_height;

    // only apply co-ordinate transformation if scale != 1.0
    if (scales_width == 1.0f) {
      dims_mapping[id].extrapolate_ = 0;
    } else {
      float orig_coord = transform_coordinate(static_cast<float>(dim), scales_width, static_cast<float>(output_width),
                                              static_cast<float>(input_width), roi_start_width, roi_end_width);
      dims_mapping[id].extrapolate_ = static_cast<int>(
          extrapolation_enabled && (orig_coord < 0.f || orig_coord > static_cast<float>(input_width - 1)));
      dim = calc_nearest_pixel(orig_coord, scales_width < 1);
      if (dim >= input_width) dim = input_width - 1;
      if (dim < 0) dim = 0;
    }

    dims_mapping[id].origin_ = dim;
    return;
  }
}

template <typename T, typename CudaFunctionOriginalCoordinate, typename CudaFunctionNearestPixel>
__global__ void _ResizeNearestMappingKernel(
    const size_t rank,
    const TArray<int64_t> input_shape,
    const TArray<int64_t> output_shape,
    const TArray<float> scales,
    const TArray<float, 10> roi,
    const size_t total_dim_sum,
    bool extrapolation_enabled,
    const CudaFunctionOriginalCoordinate& transform_coordinate,
    const CudaFunctionNearestPixel& calc_nearest_pixel,
    int64_t* prefix_dim_sum,
    NearestMappingInfo* dims_mapping) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, total_dim_sum);
  int64_t dim_sum = 0;
  for (int axis = 0; axis < rank; ++axis) {
    if (id == dim_sum) {
      prefix_dim_sum[axis] = dim_sum;
    }
    if (id >= dim_sum && id < dim_sum + output_shape[axis]) {
      int dim = id - dim_sum;

      // only apply co-ordinate transformation if scale != 1.0
      if (scales[axis] == 1.0f) {
        dims_mapping[id].extrapolate_ = 0;
      } else {
        float orig_coord = transform_coordinate(static_cast<float>(dim), scales[axis], static_cast<float>(output_shape[axis]),
                                                static_cast<float>(input_shape[axis]), roi[axis], roi[axis + rank]);
        dims_mapping[id].extrapolate_ = static_cast<int>(extrapolation_enabled && (orig_coord < 0.f || orig_coord > static_cast<float>(input_shape[axis] - 1)));
        dim = calc_nearest_pixel(orig_coord, scales[axis] < 1);
        if (dim >= input_shape[axis]) dim = input_shape[axis] - 1;
        if (dim < 0) dim = 0;
      }

      dims_mapping[id].origin_ = dim;
      return;
    }
    dim_sum += output_shape[axis];
  }
}

template <typename T, bool UseExtrapolation>
__global__ void _ResizeNearestKernel2D(
    const int64_t output_height, const int64_t output_width,
    const int64_t input_stride_image, const int input_stride_row,
    const fast_divmod output_stride_image, const fast_divmod output_stride_row,
    const T* input_data, T* output_data, const size_t N,
    const T extrapolation_value, const NearestMappingInfo* dims_mapping) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);

  int imageid, h, w, output_index;
  output_stride_image.divmod(static_cast<int>(id), imageid, output_index);
  output_stride_row.divmod(output_index, h, w);
  if (UseExtrapolation) {
    if (dims_mapping[h].extrapolate_ + dims_mapping[output_height + w].extrapolate_) {
      output_data[id] = extrapolation_value;
      return;
    }
  }
  int input_index = input_stride_image * imageid +
                    input_stride_row * dims_mapping[h].origin_ +
                    dims_mapping[output_height + w].origin_;
  output_data[id] = input_data[input_index];
}

template <typename T>
__global__ void _ResizeNearestKernel(
    const int rank,
    const TArray<int64_t> input_strides,
    const TArray<fast_divmod> output_div_pitches,
    const T* input_data,
    T* output_data,
    const size_t N,
    const T extrapolation_value,
    const int64_t* prefix_dim_sum,
    const NearestMappingInfo* dims_mapping) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);

  int output_index = static_cast<int>(id);
  int input_index = 0;
  int extrapolation_occured = 0;
  for (int axis = 0; axis < rank; ++axis) {
    int dim = 0;
    output_div_pitches[axis].divmod(output_index, dim, output_index);
    const NearestMappingInfo& mi = dims_mapping[prefix_dim_sum[axis] + dim];
    extrapolation_occured += mi.extrapolate_;
    input_index += input_strides[axis] * mi.origin_;
  }
  output_data[id] = extrapolation_occured ? extrapolation_value : input_data[input_index];
}

struct LinearMappingInfo {
  int origin_;
  float weight_;
  int extrapolate_;
};

template <typename T, typename CudaFunctionOriginalCoordinate>
__global__ void _ResizeBilinearCoordinateMapping(
    int64_t input_height, int64_t input_width,
    int64_t output_height, int64_t output_width,
    float scale_height, float scale_width,
    float roi_height_start, float roi_height_end,
    float roi_width_start, float roi_width_end,
    const size_t SumHW, bool extrapolation_enabled,
    const CudaFunctionOriginalCoordinate& transform_coordinate,
    LinearMappingInfo* dims_mapping) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, SumHW);
  if (id < output_height) {  //  y = id
    float input_y = scale_height == 1 ? static_cast<float>(id) :
                                        transform_coordinate(static_cast<float>(id), scale_height,
                                        static_cast<float>(output_height), static_cast<float>(input_height),
                                        roi_height_start, roi_height_end);
    dims_mapping[id].extrapolate_ = (int)(extrapolation_enabled && (input_y < 0 || input_y > static_cast<float>(input_height - 1)));
    input_y = max(0.0f, min(input_y, static_cast<float>(input_height - 1)));
    int y_int = static_cast<int>(input_y);
    dims_mapping[id].origin_ = y_int;
    dims_mapping[id].weight_ = (y_int >= input_height - 1) ? 0.5f : input_y - y_int;
  } else {  //x = id - output_height
    float input_x = scale_width == 1 ? static_cast<float>(id - output_height) :
                                       transform_coordinate(static_cast<float>(id - output_height), scale_width,
                                       static_cast<float>(output_width), static_cast<float>(input_width),
                                       roi_width_start, roi_width_end);
    dims_mapping[id].extrapolate_ = (int)(extrapolation_enabled && (input_x < 0 || input_x > static_cast<float>(input_width - 1)));
    input_x = max(0.0f, min(input_x, static_cast<float>(input_width - 1)));
    int x_int = static_cast<int>(input_x);
    dims_mapping[id].origin_ = x_int;
    dims_mapping[id].weight_ = (x_int >= input_width - 1) ? 0.5f : input_x - x_int;
  }
}

// The following method supports a 2-D or 4-D input in 'Linear mode'. Last two dimension is [H, W].
// the scale values for the outer dimensions except last two are 1.
template <typename T>
__global__ void _ResizeBilinearKernel(
    int64_t input_height, int64_t input_width,
    int64_t output_height, int64_t output_width,
    fast_divmod div_output_width, fast_divmod div_output_image,
    const T* input_data, T* output_data, const size_t N,
    const T extrapolation_value,
    LinearMappingInfo* dims_mapping) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  int bxc, output_image_index;
  div_output_image.divmod(id, bxc, output_image_index);
  CUDA_LONG input_index = bxc * input_height * input_width;
  int output_y, output_x;
  div_output_width.divmod(output_image_index, output_y, output_x);

  if (dims_mapping[output_y].extrapolate_ || dims_mapping[output_x + output_height].extrapolate_) {
    output_data[id] = extrapolation_value;
    return;
  }
  float y_offset_0 = dims_mapping[output_y].weight_;
  int y_int = dims_mapping[output_y].origin_;
  float x_offset_0 = dims_mapping[output_x + output_height].weight_;
  int x_int = dims_mapping[output_x + output_height].origin_;
  input_index += y_int * input_width + x_int;

  T x00 = input_data[input_index];
  bool end_of_h = (y_int >= input_height - 1);
  bool end_of_w = (x_int >= input_width - 1);
  T x10 = end_of_w ? x00 : input_data[input_index + 1];
  T x01 = end_of_h ? x00 : input_data[input_index + input_width];
  T x11 = end_of_w ? x01 : (end_of_h ? x10 : input_data[input_index + input_width + 1]);

  float y_offset_1 = 1.0f - y_offset_0;
  float x_offset_1 = 1.0f - x_offset_0;
  output_data[id] =
      x00 * static_cast<T>(y_offset_1 * x_offset_1) +
      x01 * static_cast<T>(y_offset_0 * x_offset_1) +
      x10 * static_cast<T>(y_offset_1 * x_offset_0) +
      x11 * static_cast<T>(y_offset_0 * x_offset_0);
}

template <typename T, typename CudaFunctionOriginalCoordinate>
__global__ void _ResizeTrilinearCoordinateMapping(
    int64_t input_depth, int64_t input_height, int64_t input_width,
    int64_t output_depth, int64_t output_height, int64_t output_width,
    float scale_depth, float scale_height, float scale_width,
    float roi_depth_start, float roi_depth_end,
    float roi_height_start, float roi_height_end,
    float roi_width_start, float roi_width_end,
    const size_t SumDHW, bool extrapolation_enabled,
    const CudaFunctionOriginalCoordinate& transform_coordinate,
    LinearMappingInfo* dims_mapping) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, SumDHW);
  if (id < output_depth) {  //  z = id
    float input_z = scale_depth == 1 ? static_cast<float>(id)  :
                                       transform_coordinate(static_cast<float>(id), scale_depth,
                                       static_cast<float>(output_depth), static_cast<float>(input_depth),
                                       roi_depth_start, roi_depth_end);
    dims_mapping[id].extrapolate_ = (int)(extrapolation_enabled && (input_z < 0 || input_z > static_cast<float>(input_depth - 1)));
    input_z = max(0.0f, min(input_z, static_cast<float>(input_depth - 1)));
    int z_int = static_cast<int>(input_z);
    dims_mapping[id].origin_ = z_int;
    dims_mapping[id].weight_ = (z_int >= input_depth - 1) ? 0.5f : input_z - z_int;
  } else if (id >= output_depth && id < (output_depth + output_height)) {  //  y = id - output_depth
    float input_y = scale_height == 1 ? static_cast<float>(id - output_depth) :
                                        transform_coordinate(static_cast<float>(id - output_depth), scale_height,
                                        static_cast<float>(output_height), static_cast<float>(input_height),
                                        roi_height_start, roi_height_end);

    dims_mapping[id].extrapolate_ = (int)(extrapolation_enabled && (input_y < 0 || input_y > static_cast<float>(input_height - 1)));
    input_y = max(0.0f, min(input_y, static_cast<float>(input_height - 1)));
    int y_int = static_cast<int>(input_y);
    dims_mapping[id].origin_ = y_int;
    dims_mapping[id].weight_ = (y_int >= input_height - 1) ? 0.5f : input_y - y_int;
  } else {  //x = id - output_depth - output_height
    float input_x = scale_width == 1 ? static_cast<float>(id - output_depth - output_height) :
                                       transform_coordinate(static_cast<float>(id - output_depth - output_height), scale_width,
                                       static_cast<float>(output_width), static_cast<float>(input_width),
                                       roi_width_start, roi_width_end);
    dims_mapping[id].extrapolate_ = (int)(extrapolation_enabled && (input_x < 0 || input_x > static_cast<float>(input_width - 1)));
    input_x = max(0.0f, min(input_x, static_cast<float>(input_width - 1)));
    int x_int = static_cast<int>(input_x);
    dims_mapping[id].origin_ = x_int;
    dims_mapping[id].weight_ = (x_int >= input_width - 1) ? 0.5f : input_x - x_int;
  }
}

// The following method supports a 3-D or 5-D input in 'Linear mode'. Last two dimension is [D, sH, W].
// the scale values for the outer dimensions except last two are 1.
template <typename T>
__global__ void _ResizeTrilinearKernel(
    int64_t input_depth, int64_t input_height, int64_t input_width,
    int64_t output_depth, int64_t output_height, int64_t output_width,
    fast_divmod div_output_height, fast_divmod div_output_width, fast_divmod div_output_image,
    const T* input_data, T* output_data, const size_t N,
    const T extrapolation_value,
    LinearMappingInfo* dims_mapping) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  int bxc, output_image_index;
  div_output_image.divmod(id, bxc, output_image_index);
  CUDA_LONG input_index = bxc * input_depth * input_height * input_width;
  int output_z, output_y, output_x, temp;

  div_output_height.divmod(output_image_index, output_z, temp);
  div_output_width.divmod(temp, output_y, output_x);

  if (dims_mapping[output_z].extrapolate_ ||
      dims_mapping[output_y + output_depth].extrapolate_ ||
      dims_mapping[output_x + output_depth + output_height].extrapolate_) {
    output_data[id] = extrapolation_value;
    return;
  }

  float z_offset_0 = dims_mapping[output_z].weight_;
  int z_int = dims_mapping[output_z].origin_;

  float y_offset_0 = dims_mapping[output_y + output_depth].weight_;
  int y_int = dims_mapping[output_y + output_depth].origin_;

  float x_offset_0 = dims_mapping[output_x + output_depth + output_height].weight_;
  int x_int = dims_mapping[output_x + output_depth + output_height].origin_;

  input_index += z_int * input_height * input_width + y_int * input_width + x_int;

  T x000 = input_data[input_index];

  bool end_of_h = (y_int >= input_height - 1);
  bool end_of_w = (x_int >= input_width - 1);

  T x100 = end_of_w ? x000 : input_data[input_index + 1];
  T x010 = end_of_h ? x000 : input_data[input_index + input_width];
  T x110 = end_of_w ? x010 : (end_of_h ? x100 : input_data[input_index + input_width + 1]);

  bool end_of_d = (z_int >= input_depth - 1);
  if (!end_of_d) {
    input_index = input_index + input_height * input_width;
  }

  T x001 = end_of_d ? x000 : input_data[input_index];

  T x101 = end_of_w ? x001 : input_data[input_index + 1];
  T x011 = end_of_h ? x001 : input_data[input_index + input_width];
  T x111 = end_of_w ? x011 : (end_of_h ? x101 : input_data[input_index + input_width + 1]);

  float z_offset_1 = 1.0f - z_offset_0;
  float y_offset_1 = 1.0f - y_offset_0;
  float x_offset_1 = 1.0f - x_offset_0;
  output_data[id] =
      x000 * static_cast<T>(z_offset_1 * y_offset_1 * x_offset_1) +
      x010 * static_cast<T>(z_offset_1 * y_offset_0 * x_offset_1) +
      x100 * static_cast<T>(z_offset_1 * y_offset_1 * x_offset_0) +
      x110 * static_cast<T>(z_offset_1 * y_offset_0 * x_offset_0) +

      x001 * static_cast<T>(z_offset_0 * y_offset_1 * x_offset_1) +
      x011 * static_cast<T>(z_offset_0 * y_offset_0 * x_offset_1) +
      x101 * static_cast<T>(z_offset_0 * y_offset_1 * x_offset_0) +
      x111 * static_cast<T>(z_offset_0 * y_offset_0 * x_offset_0);
}

template <typename T>
__device__ __forceinline__ float CubicInterpolationRowwise(
    const T* image, int x, int y, int input_height, int input_width,
    float coeff0, float coeff1, float coeff2, float coeff3) {
  int row_index = max(0, min(y, input_height - 1)) * input_width;
  return coeff0 * static_cast<float>(image[row_index + max(0, min(x - 1, input_width - 1))]) +
         coeff1 * static_cast<float>(image[row_index + max(0, min(x, input_width - 1))]) +
         coeff2 * static_cast<float>(image[row_index + max(0, min(x + 1, input_width - 1))]) +
         coeff3 * static_cast<float>(image[row_index + max(0, min(x + 2, input_width - 1))]);
}

struct CubicMappingInfo {
  int origin_;
  int extrapolate_;
  float coeff0_;
  float coeff1_;
  float coeff2_;
  float coeff3_;
};

template <typename T, typename CudaFunctionOriginalCoordinate>
__global__ void _ResizeCubicCoordinateMapping(
    int64_t input_height, int64_t input_width,
    int64_t output_height, int64_t output_width,
    float scale_height, float scale_width,
    float roi_height_start, float roi_height_end,
    float roi_width_start, float roi_width_end,
    const size_t SumHW, bool extrapolation_enabled,
    float cubic_coeff_a, bool exclude_outside,
    const CudaFunctionOriginalCoordinate& transform_coordinate,
    CubicMappingInfo* dims_mapping) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, SumHW);
  auto& dm = dims_mapping[id];
  bool is_y_axis = (id < output_height);
  int max_input_coord = static_cast<int>(is_y_axis ? input_height : input_width);

  float scale = is_y_axis ? scale_height : scale_width;
  float input_coordinat = scale == 1 ? (is_y_axis ? id : id - output_height) :
      transform_coordinate(
      static_cast<float>(is_y_axis ? id : id - output_height),
      scale,
      static_cast<float>(is_y_axis ? output_height : output_width),
      static_cast<float>(max_input_coord),
      (is_y_axis ? roi_height_start : roi_width_start),
      (is_y_axis ? roi_height_end : roi_width_end));
  int coord_int = static_cast<int>(_Floor(input_coordinat));
  float s_coord = abs(input_coordinat - coord_int);
  float coeff_sum = 1.0f;
  float coeff_0 = static_cast<float>(((cubic_coeff_a * (s_coord + 1) - 5 * cubic_coeff_a) * (s_coord + 1) + 8 * cubic_coeff_a) * (s_coord + 1) - 4 * cubic_coeff_a);
  float coeff_1 = static_cast<float>(((cubic_coeff_a + 2) * s_coord - (cubic_coeff_a + 3)) * s_coord * s_coord + 1);
  float coeff_2 = static_cast<float>(((cubic_coeff_a + 2) * (1 - s_coord) - (cubic_coeff_a + 3)) * (1 - s_coord) * (1 - s_coord) + 1);
  float coeff_3 = static_cast<float>(((cubic_coeff_a * (2 - s_coord) - 5 * cubic_coeff_a) * (2 - s_coord) + 8 * cubic_coeff_a) * (2 - s_coord) - 4 * cubic_coeff_a);
  if (exclude_outside) {
    coeff_0 = (coord_int - 1 < 0 || coord_int - 1 >= max_input_coord) ? 0.0 : coeff_0;
    coeff_1 = (coord_int + 0 < 0 || coord_int + 0 >= max_input_coord) ? 0.0 : coeff_1;
    coeff_2 = (coord_int + 1 < 0 || coord_int + 1 >= max_input_coord) ? 0.0 : coeff_2;
    coeff_3 = (coord_int + 2 < 0 || coord_int + 2 >= max_input_coord) ? 0.0 : coeff_3;
    coeff_sum = coeff_0 + coeff_1 + coeff_2 + coeff_3;
  }
  dm.origin_ = coord_int;
  dm.coeff0_ = coeff_0 / coeff_sum;
  dm.coeff1_ = coeff_1 / coeff_sum;
  dm.coeff2_ = coeff_2 / coeff_sum;
  dm.coeff3_ = coeff_3 / coeff_sum;
  dm.extrapolate_ = (int)(extrapolation_enabled && (input_coordinat < 0 || input_coordinat > static_cast<float>(max_input_coord - 1)));
}

template <typename T>
__global__ void _ResizeBiCubicKernel(
    int64_t input_height, int64_t input_width, int64_t output_height, int64_t output_width,
    fast_divmod div_output_width, fast_divmod div_output_image,
    const T* input_data, T* output_data, const size_t N, const T extrapolation_value,
    CubicMappingInfo* dims_mapping) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  int bxc, output_image_index, output_x, output_y;
  div_output_image.divmod(id, bxc, output_image_index);
  CUDA_LONG input_index = bxc * input_height * input_width;
  div_output_width.divmod(output_image_index, output_y, output_x);

  CubicMappingInfo& y_info = dims_mapping[output_y];
  CubicMappingInfo& x_info = dims_mapping[output_x + output_height];
  if (y_info.extrapolate_ || x_info.extrapolate_) {
    output_data[id] = extrapolation_value;
    return;
  }

  float w0 = x_info.coeff0_;
  float w1 = x_info.coeff1_;
  float w2 = x_info.coeff2_;
  float w3 = x_info.coeff3_;
  int x_int = x_info.origin_;
  int y_int = y_info.origin_;
  const T* image = input_data + input_index;
  output_data[id] = y_info.coeff0_ * CubicInterpolationRowwise(image, x_int, y_int - 1, input_height, input_width, w0, w1, w2, w3) +
                    y_info.coeff1_ * CubicInterpolationRowwise(image, x_int, y_int, input_height, input_width, w0, w1, w2, w3) +
                    y_info.coeff2_ * CubicInterpolationRowwise(image, x_int, y_int + 1, input_height, input_width, w0, w1, w2, w3) +
                    y_info.coeff3_ * CubicInterpolationRowwise(image, x_int, y_int + 2, input_height, input_width, w0, w1, w2, w3);
}

size_t CalcResizeBufferSize(const onnxruntime::UpsampleMode upsample_mode,
                            const gsl::span<const int64_t>& output_dims) {
  switch (upsample_mode) {
    case UpsampleMode::NN:
      return sizeof(int64_t) * output_dims.size() + sizeof(NearestMappingInfo) * static_cast<size_t>(std::accumulate(output_dims.begin(), output_dims.end(), (int64_t)0));
    case UpsampleMode::LINEAR:
      return sizeof(LinearMappingInfo) * static_cast<size_t>(std::accumulate(output_dims.rbegin(), output_dims.rbegin() + 2, (int64_t)0));
    case UpsampleMode::CUBIC:
      return sizeof(CubicMappingInfo) * static_cast<size_t>(std::accumulate(output_dims.rbegin(), output_dims.rbegin() + 2, (int64_t)0));
  }
  return 0;
}

template <typename T>
void ResizeNearestImpl(
    hipStream_t stream,
    const int rank,
    TArray<int64_t>& input_shape,
    TArray<int64_t>& output_shape,
    TArray<int64_t>& input_strides,
    TArray<fast_divmod>& output_div_pitches,
    TArray<float>& scales_vals,
    TArray<float, 10>& roi_vals,
    const T* input_data,
    T* output_data,
    const size_t N,
    bool extrapolation_enabled,
    const T extrapolation_value,
    float cubic_coeff_a,
    ResizeCoordinateTransformationMode transform_coordinate,
    ResizeNearestMode calc_nearest_pixel,
    int64_t* /* prefix_dim_sum */,
    NearestMappingInfo* dims_mapping) {
  unsigned int blocksPerGrid = static_cast<unsigned int>(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));

  bool could2d = rank >= 2 &&
                 transform_coordinate != ResizeCoordinateTransformationMode::TF_CROP_AND_RESIZE &&
                 std::all_of(scales_vals.Data(), scales_vals.Data() + (rank - 2), [](float v) { return v == 1.0; });
  if (could2d) {
    int64_t output_height = output_shape[rank - 2];
    int64_t output_width = output_shape[rank - 1];
    fast_divmod div_output_image = (rank > 2) ? output_div_pitches[rank - 3] : fast_divmod(static_cast<int>(output_height * output_width));
    int blocksPerDimsMappingGrid = static_cast<int>(ceil((output_height + output_width) / 32.0));

    DISPATCH_RESIZE_COORDINATE_TRANSFORMATION_MODE(transform_coordinate, [&]() {
      DISPATCH_RESIZE_NEAREST_MODE(calc_nearest_pixel, [&]() {
        _ResizeNearestMappingKernel2D<T><<<blocksPerDimsMappingGrid, 32, 0, stream>>>(
            static_cast<int>(input_shape[rank - 2]), static_cast<int>(input_shape[rank - 1]),
            static_cast<int>(output_height), static_cast<int>(output_width),
            scales_vals[rank - 2], scales_vals[rank - 1],
            roi_vals[rank - 2], roi_vals[rank - 2 + rank],
            roi_vals[rank - 1], roi_vals[rank - 1 + rank],
            extrapolation_enabled, coord_t(), nearest_t(),
            dims_mapping);
      });
    });
    if (extrapolation_enabled) {
      _ResizeNearestKernel2D<T, true><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          output_height, output_width,
          input_shape[rank - 2] * input_shape[rank - 1], static_cast<int>(input_shape[rank - 1]),
          div_output_image, output_div_pitches[rank - 2],
          input_data, output_data, N,
          extrapolation_value,
          dims_mapping);
    } else {
      _ResizeNearestKernel2D<T, false><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          output_height, output_width,
          input_shape[rank - 2] * input_shape[rank - 1], static_cast<int>(input_shape[rank - 1]),
          div_output_image, output_div_pitches[rank - 2],
          input_data, output_data, N,
          extrapolation_value,
          dims_mapping);
    }
    return;
  }

  int64_t total_dim_sum = std::accumulate(output_shape.Data(), output_shape.Data() + rank, (int64_t)0);
  int blocksPerDimsMappingGrid = (int)(ceil(static_cast<double>(total_dim_sum) / 32));
  DISPATCH_RESIZE_COORDINATE_TRANSFORMATION_MODE(transform_coordinate, [&]() {
    DISPATCH_RESIZE_NEAREST_MODE(calc_nearest_pixel, [&]() {
      _ResizeNearestMappingKernel<T><<<blocksPerDimsMappingGrid, 32, 0, stream>>>(
          rank, input_shape, output_shape,
          scales_vals, roi_vals,
          total_dim_sum, extrapolation_enabled,
          coord_t(), nearest_t(),
          reinterpret_cast<int64_t*>(dims_mapping),
          reinterpret_cast<NearestMappingInfo*>(reinterpret_cast<int64_t*>(dims_mapping) + rank));
    });
  });
  _ResizeNearestKernel<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      rank, input_strides, output_div_pitches,
      input_data, output_data, N,
      extrapolation_value,
      reinterpret_cast<const int64_t*>(dims_mapping),
      reinterpret_cast<const NearestMappingInfo*>(reinterpret_cast<int64_t*>(dims_mapping) + rank));
  return;
}

template <typename T>
void ResizeImpl(
    hipStream_t stream,
    const UpsampleMode upsample_mode,
    const int rank,
    TArray<int64_t>& input_shape,
    TArray<int64_t>& output_shape,
    TArray<int64_t>& input_strides,
    TArray<fast_divmod>& output_div_pitches,
    TArray<float>& scales_vals,
    TArray<float, 10>& roi_vals,
    const T* input_data,
    T* output_data,
    const size_t N,
    bool extrapolation_enabled,
    const T extrapolation_value,
    float cubic_coeff_a,
    bool exclude_outside,
    ResizeCoordinateTransformationMode coordinate_transform_mode,
    ResizeNearestMode nearest_mode,
    void* dims_mapping) {
  bool isSame = std::all_of(scales_vals.Data(), scales_vals.Data() + rank, [](float v) { return v == 1.0f; }) &&
                (coordinate_transform_mode != ResizeCoordinateTransformationMode::TF_CROP_AND_RESIZE);
  if (isSame) {
    CUDA_CALL_THROW(hipMemcpyAsync(output_data, input_data, N * sizeof(T), hipMemcpyDeviceToDevice, stream));
    return;
  }

  if (upsample_mode == UpsampleMode::NN) {
    ResizeNearestImpl(
        stream, rank, input_shape, output_shape, input_strides, output_div_pitches,
        scales_vals, roi_vals, input_data, output_data, N,
        extrapolation_enabled, extrapolation_value, cubic_coeff_a,
        coordinate_transform_mode, nearest_mode,
        reinterpret_cast<int64_t*>(dims_mapping),
        reinterpret_cast<NearestMappingInfo*>(reinterpret_cast<int64_t*>(dims_mapping) + rank));
    return;
  }

  // We support a special case of bilinear or bicubic if the input data is 4D with the outer 2 scales being 1.0
  // We would have validated the outer scale values by the time execution reaches this
  bool is_2D = (rank == 2 || rank == 4);

  // We support a special case of trilinear or tricubic if the input data is 5D with the outer 2 scales being 1.0
  // We would have validated the outer scale values by the time execution reaches this
  bool is_3D = (rank == 3 || rank == 5);

  // Should not hit this as we have already validated input rank/scales and we provide verbose error messages
  // to the user.
  ORT_ENFORCE(is_2D || is_3D, "Only bilinear/trilinear and bicubic modes are supported in Resize");

  int blocksPerGrid = static_cast<int>(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  fast_divmod div_output_image;
  if (is_2D) {
    div_output_image = (rank > 2) ? output_div_pitches[rank - 3] : fast_divmod(gsl::narrow_cast<int>(N));
  } else if (is_3D) {
    div_output_image = (rank > 3) ? output_div_pitches[rank - 4] : fast_divmod(gsl::narrow_cast<int>(N));
  }

  int64_t output_depth = is_3D ? output_shape[rank - 3] : 0;
  int64_t output_height = output_shape[rank - 2];
  int64_t output_width = output_shape[rank - 1];
  int blocksPerDimsMappingGrid =
      static_cast<int>(ceil((output_depth + output_height + output_width) / 32.0));

  switch (upsample_mode) {
    case UpsampleMode::LINEAR:
      if (is_2D) {
        DISPATCH_RESIZE_COORDINATE_TRANSFORMATION_MODE(coordinate_transform_mode, [&]() {
          _ResizeBilinearCoordinateMapping<T><<<blocksPerDimsMappingGrid, 32, 0, stream>>>(
              input_shape[rank - 2], input_shape[rank - 1],
              output_height, output_width,
              scales_vals[rank - 2], scales_vals[rank - 1],
              roi_vals[rank - 2], roi_vals[rank - 2 + rank],
              roi_vals[rank - 1], roi_vals[rank - 1 + rank],
              output_height + output_width, extrapolation_enabled, coord_t(),
              reinterpret_cast<LinearMappingInfo*>(dims_mapping));
        });
        _ResizeBilinearKernel<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
            input_shape[rank - 2], input_shape[rank - 1],
            output_height, output_width,
            output_div_pitches[rank - 2], div_output_image,
            input_data, output_data, N, extrapolation_value,
            reinterpret_cast<LinearMappingInfo*>(dims_mapping));
        return;
      } else if (is_3D) {
        DISPATCH_RESIZE_COORDINATE_TRANSFORMATION_MODE(coordinate_transform_mode, [&]() {
          _ResizeTrilinearCoordinateMapping<T><<<blocksPerDimsMappingGrid, 32, 0, stream>>>(
              input_shape[rank - 3] , input_shape[rank - 2], input_shape[rank - 1],
              output_depth, output_height, output_width,
              scales_vals[rank - 3], scales_vals[rank - 2], scales_vals[rank - 1],
              roi_vals[rank - 3], roi_vals[rank - 3 + rank],
              roi_vals[rank - 2], roi_vals[rank - 2 + rank],
              roi_vals[rank - 1], roi_vals[rank - 1 + rank],
              output_depth + output_height + output_width, extrapolation_enabled, coord_t(),
              reinterpret_cast<LinearMappingInfo*>(dims_mapping));
        });
        _ResizeTrilinearKernel<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
            input_shape[rank - 3], input_shape[rank - 2], input_shape[rank - 1],
            output_depth, output_height, output_width,
            output_div_pitches[rank - 3], output_div_pitches[rank - 2], div_output_image,
            input_data, output_data, N, extrapolation_value,
            reinterpret_cast<LinearMappingInfo*>(dims_mapping));
        return;
      }
      ORT_THROW("Only bilinear/trilinear and bicubic modes are supported in Resize");
      break;
    case UpsampleMode::CUBIC:
      if (is_2D) {
        DISPATCH_RESIZE_COORDINATE_TRANSFORMATION_MODE(coordinate_transform_mode, [&]() {
          _ResizeCubicCoordinateMapping<T><<<blocksPerDimsMappingGrid, 32, 0, stream>>>(
              input_shape[rank - 2], input_shape[rank - 1],
              output_height, output_width,
              scales_vals[rank - 2], scales_vals[rank - 1],
              roi_vals[rank - 2], roi_vals[rank - 2 + rank],
              roi_vals[rank - 1], roi_vals[rank - 1 + rank],
              output_height + output_width, extrapolation_enabled,
              cubic_coeff_a, exclude_outside, coord_t(),
              reinterpret_cast<CubicMappingInfo*>(dims_mapping));
        });
        _ResizeBiCubicKernel<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
            input_shape[rank - 2], input_shape[rank - 1],
            output_height, output_width,
            output_div_pitches[rank - 2], div_output_image,
            input_data, output_data, N, extrapolation_value,
            reinterpret_cast<CubicMappingInfo*>(dims_mapping));
        return;
      }
      ORT_THROW("Only bilinear/trilinear and bicubic modes are supported in Resize");
    case UpsampleMode::NN:
      ORT_THROW("Only bilinear/trilinear and bicubic modes are supported in Resize");
  }
}

#define SPECIALIZED_IMPL(T)                                         \
  template void ResizeImpl<T>(                                      \
      hipStream_t stream,                                    \
      const UpsampleMode upsample_mode,                             \
      const int rank,                                               \
      TArray<int64_t>& input_shape,                                 \
      TArray<int64_t>& output_shape,                                \
      TArray<int64_t>& input_strides,                               \
      TArray<fast_divmod>& output_div_pitches,                      \
      TArray<float>& scales_vals,                                   \
      TArray<float, 10>& roi_vals,                                  \
      const T* input_data,                                          \
      T* output_data,                                               \
      const size_t N,                                               \
      bool extrapolation_enabled,                                   \
      const T extrapolation_value,                                  \
      float cubic_coeff_a,                                          \
      bool exclude_outside,                                         \
      ResizeCoordinateTransformationMode coordinate_transform_mode, \
      ResizeNearestMode nearest_mode,                               \
      void* dims_mapping);

SPECIALIZED_IMPL(float)
SPECIALIZED_IMPL(double)
SPECIALIZED_IMPL(half)
SPECIALIZED_IMPL(int32_t)
SPECIALIZED_IMPL(uint8_t)

}  // namespace cuda
}  // namespace onnxruntime
