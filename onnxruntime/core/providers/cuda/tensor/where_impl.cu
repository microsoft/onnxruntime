#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <stdint.h>
#include "core/providers/cuda/shared_inc/cuda_utils.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "where_impl.h"

namespace onnxruntime {
namespace cuda {

// broadcast by computing output coordinate from offset, using fast_divmod
template <typename T, bool cond_need_compute, bool x_need_compute, bool y_need_compute>
__global__ void _TenaryElementWise(
    size_t output_rank,
    const TArray<int64_t> cond_padded_strides,
    const bool* cond_data,
    const TArray<int64_t> x_padded_strides,
    const T* x_data,
    const TArray<int64_t> y_padded_strides,
    const T* y_data,
    const TArray<fast_divmod> fdm_output_strides,
    T* output_data,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  CUDA_LONG cond_index = (cond_need_compute ? 0 : id);
  CUDA_LONG x_index = (x_need_compute ? 0 : id);
  CUDA_LONG y_index = (y_need_compute ? 0 : id);

  // compute indexes with broadcasting rules: https://github.com/onnx/onnx/blob/master/docs/Broadcasting.md
  CUDA_LONG offset = id;
  for (auto dim = 0; dim < fdm_output_strides.GetCapacity(); dim++) {
    if (dim >= output_rank) {
      break;
    }

    int q, r;
    fdm_output_strides[dim].divmod(offset, q, r);

    if (cond_need_compute) {
      cond_index += static_cast<int>(cond_padded_strides[dim]) * q;
    }

    if (x_need_compute) {
      x_index += static_cast<int>(x_padded_strides[dim]) * q;
    }

    if (y_need_compute) {
      y_index += static_cast<int>(y_padded_strides[dim]) * q;
    }

    offset = r;
  }

  output_data[id] = cond_data[cond_index] ? x_data[x_index] : y_data[y_index];
}

// for scalar broadcast or non-broadcast case
template <typename T>
__global__ void _TenaryElementWiseSimple(
    const bool* cond_data,
    const T* x_data,
    const T* y_data,
    T* output_data,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  output_data[id] = cond_data[id] ? x_data[id] : y_data[id];
}

template <typename T>
void WhereImpl(
    size_t output_rank_or_simple_broadcast,
    const TArray<int64_t>& cond_padded_strides,
    const bool* cond_data,
    const TArray<int64_t>& x_padded_strides,
    const T* x_data,
    const TArray<int64_t>& y_padded_strides,
    const T* y_data,
    const TArray<fast_divmod>& fdm_output_strides,
    T* output_data,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);

  if (output_rank_or_simple_broadcast == static_cast<size_t>(SimpleBroadcast::NoBroadcast)) {
    _TenaryElementWiseSimple<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
        cond_data,
        x_data,
        y_data,
        output_data,
        N);
  } else {
    if (cond_padded_strides.size_ && x_padded_strides.size_ && y_padded_strides.size_) {
      _TenaryElementWise<T, true, true, true><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          output_rank_or_simple_broadcast,
          cond_padded_strides,
          cond_data,
          x_padded_strides,
          x_data,
          y_padded_strides,
          y_data,
          fdm_output_strides,
          output_data,
          N);
    } else if (cond_padded_strides.size_ && x_padded_strides.size_ && !y_padded_strides.size_) {
      _TenaryElementWise<T, true, true, false><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          output_rank_or_simple_broadcast,
          cond_padded_strides,
          cond_data,
          x_padded_strides,
          x_data,
          y_padded_strides,
          y_data,
          fdm_output_strides,
          output_data,
          N);
    } else if (cond_padded_strides.size_ && !x_padded_strides.size_ && y_padded_strides.size_) {
      _TenaryElementWise<T, true, false, true><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          output_rank_or_simple_broadcast,
          cond_padded_strides,
          cond_data,
          x_padded_strides,
          x_data,
          y_padded_strides,
          y_data,
          fdm_output_strides,
          output_data,
          N);
    } else if (!cond_padded_strides.size_ && x_padded_strides.size_ && y_padded_strides.size_) {
      _TenaryElementWise<T, false, true, true><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          output_rank_or_simple_broadcast,
          cond_padded_strides,
          cond_data,
          x_padded_strides,
          x_data,
          y_padded_strides,
          y_data,
          fdm_output_strides,
          output_data,
          N);
    } else if (cond_padded_strides.size_ && !x_padded_strides.size_ && !y_padded_strides.size_) {
      _TenaryElementWise<T, true, false, false><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          output_rank_or_simple_broadcast,
          cond_padded_strides,
          cond_data,
          x_padded_strides,
          x_data,
          y_padded_strides,
          y_data,
          fdm_output_strides,
          output_data,
          N);
      } else if (!cond_padded_strides.size_ && x_padded_strides.size_ && !y_padded_strides.size_) {
        _TenaryElementWise<T, false, true, false><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
            output_rank_or_simple_broadcast,
            cond_padded_strides,
            cond_data,
            x_padded_strides,
            x_data,
            y_padded_strides,
            y_data,
            fdm_output_strides,
            output_data,
            N);
      } else if (!cond_padded_strides.size_ && !x_padded_strides.size_ && y_padded_strides.size_) {
        _TenaryElementWise<T, false, false, true><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
            output_rank_or_simple_broadcast,
            cond_padded_strides,
            cond_data,
            x_padded_strides,
            x_data,
            y_padded_strides,
            y_data,
            fdm_output_strides,
            output_data,
            N);
      } else {
        _TenaryElementWise<T, false, false, false><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
            output_rank_or_simple_broadcast,
            cond_padded_strides,
            cond_data,
            x_padded_strides,
            x_data,
            y_padded_strides,
            y_data,
            fdm_output_strides,
            output_data,
            N);
      }
  }
}

#define SPECIALIZED_IMPL(T)                                          \
  template void WhereImpl<T>(size_t output_rank_or_simple_broadcast, \
                             const TArray<int64_t>& cond_padded_strides,     \
                             const bool* cond_data,                  \
                             const TArray<int64_t>& x_padded_strides,        \
                             const T* x_data,                        \
                             const TArray<int64_t>& y_padded_strides,        \
                             const T* y_data,                        \
                             const TArray<fast_divmod>& fdm_output_strides,  \
                             T* output_data,                         \
                             size_t count);

SPECIALIZED_IMPL(uint8_t)
SPECIALIZED_IMPL(int32_t)
SPECIALIZED_IMPL(int64_t)
SPECIALIZED_IMPL(float)
SPECIALIZED_IMPL(half)

}  // namespace cuda
}  // namespace onnxruntime