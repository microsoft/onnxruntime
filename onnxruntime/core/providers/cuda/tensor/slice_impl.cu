#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/tensor/slice_impl.h"

namespace onnxruntime {
namespace cuda {

template <bool is_grad, typename T>
__global__ void _SliceKernel(const int32_t dimension_count,
                             const TArray<int64_t> starts,
                             const TArray<int64_t> steps,
                             const TArray<int64_t> input_strides,
                             const TArray<fast_divmod> output_strides,
                             const T* input_data,
                             T* output_data,
                             const CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  CUDA_LONG input_index = 0;
  int div;
  int mod = id;
  int value = id;
  int dim = 0;
#pragma unroll
  for (; dim < starts.Capacity(); ++dim) {
    if (dim >= dimension_count - 1) {
      break;
    }

    output_strides[dim].divmod(value, div, mod);
    input_index += (starts[dim] + div * steps[dim]) * input_strides[dim];
    value = mod;
  }
  input_index += starts[dim] + mod * steps[dim];
  if (is_grad)
    output_data[input_index] = input_data[id];
  else
    output_data[id] = input_data[input_index];
}

Status SliceImpl(const size_t element_size,
                 const int32_t dimension_count,
                 const TArray<int64_t>& starts,
                 const TArray<int64_t>& steps,
                 const TArray<int64_t>& input_strides,
                 const TArray<fast_divmod>& output_strides,
                 const void* input_data,
                 void* output_data,
                 const size_t N) {
  return SliceImplEx<false>(element_size, dimension_count, starts, steps, input_strides, output_strides, input_data,
                            output_data, N);
}

Status SliceImplGrad(const size_t element_size,
                     const int32_t dimension_count,
                     const TArray<int64_t>& starts,
                     const TArray<int64_t>& steps,
                     const TArray<int64_t>& input_strides,
                     const TArray<fast_divmod>& output_strides,
                     const void* input_data,
                     void* output_data,
                     const size_t N) {
  return SliceImplEx<true>(element_size, dimension_count, starts, steps, input_strides, output_strides, input_data,
                           output_data, N);
}

template <bool is_grad>
Status SliceImplEx(const size_t element_size,
                   const int32_t dimension_count,
                   const TArray<int64_t>& starts,
                   const TArray<int64_t>& steps,
                   const TArray<int64_t>& input_strides,
                   const TArray<fast_divmod>& output_strides,
                   const void* input_data,
                   void* output_data,
                   const size_t N) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));

  switch (element_size) {
    case sizeof(int8_t):
      _SliceKernel<is_grad><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          dimension_count, starts, steps, input_strides, output_strides,
          reinterpret_cast<const ToCudaType<int8_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int8_t>::MappedType*>(output_data),
          (CUDA_LONG)N);
      break;
    case sizeof(int16_t):
      _SliceKernel<is_grad><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          dimension_count, starts, steps, input_strides, output_strides,
          reinterpret_cast<const ToCudaType<int16_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int16_t>::MappedType*>(output_data),
          (CUDA_LONG)N);
      break;
    case sizeof(int32_t):
      _SliceKernel<is_grad><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          dimension_count, starts, steps, input_strides, output_strides,
          reinterpret_cast<const ToCudaType<int32_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int32_t>::MappedType*>(output_data),
          (CUDA_LONG)N);
      break;
    case sizeof(int64_t):
      _SliceKernel<is_grad><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          dimension_count, starts, steps, input_strides, output_strides,
          reinterpret_cast<const ToCudaType<int64_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int64_t>::MappedType*>(output_data),
          (CUDA_LONG)N);
      break;
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for Slice operator");
  }

  return Status::OK();
}

}  // namespace cuda
}  // namespace onnxruntime
