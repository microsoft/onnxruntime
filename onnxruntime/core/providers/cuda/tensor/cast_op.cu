#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <limits>

#include "core/providers/cuda/cu_inc/common.cuh"

#if defined(CUDA_VERSION) && CUDA_VERSION >= 11080
#include "hip/hip_fp8.h"
#endif

namespace onnxruntime {
namespace cuda {

template <typename OutT, typename InT>
struct CastStd;

template <typename OutT, typename InT>
struct CastSat;

template <typename OutT, typename InT>
struct CastNoSat;

#if !defined(DISABLE_FLOAT8_TYPES)

#if defined(CUDA_VERSION) && CUDA_VERSION >= 11080

template <>
struct CastStd<float, Float8E4M3FN> {
  __device__ __forceinline__ float operator()(Float8E4M3FN v) const {
    return __half2float(__hip_cvt_fp8_to_halfraw(v.val, __HIP_E4M3_FNUZ));
  }
};

template <>
struct CastStd<half, Float8E4M3FN> {
  __device__ __forceinline__ half operator()(Float8E4M3FN v) const {
    return __hip_cvt_fp8_to_halfraw(v.val, __HIP_E4M3_FNUZ);
  }
};

template <>
struct CastStd<float, Float8E5M2> {
  __device__ __forceinline__ float operator()(Float8E5M2 v) const {
    return __half2float(__hip_cvt_fp8_to_halfraw(v.val, __HIP_E5M2_FNUZ));
  }
};

template <>
struct CastStd<half, Float8E5M2> {
  __device__ __forceinline__ half operator()(Float8E5M2 v) const {
    return __hip_cvt_fp8_to_halfraw(v.val, __HIP_E5M2_FNUZ);
  }
};

template <>
struct CastSat<Float8E4M3FN, float> {
  __device__ __forceinline__ Float8E4M3FN operator()(float v, bool saturate) const {
    return Float8E4M3FN(static_cast<unsigned char>(__hip_cvt_float_to_fp8(v, saturate ? __HIP_SATFINITE : __HIP_NOSAT, __HIP_E4M3_FNUZ)), Float8E4M3FN::FromBits());
  }
};

template <>
struct CastSat<Float8E4M3FN, half> {
  __device__ __forceinline__ Float8E4M3FN operator()(half v, bool saturate) const {
    return Float8E4M3FN(static_cast<unsigned char>(__hip_cvt_halfraw_to_fp8(v, saturate ? __HIP_SATFINITE : __HIP_NOSAT, __HIP_E4M3_FNUZ)), Float8E4M3FN::FromBits());
  }
};

template <>
struct CastSat<Float8E5M2, float> {
  __device__ __forceinline__ Float8E5M2 operator()(float v, bool saturate) const {
    return Float8E5M2(static_cast<unsigned char>(__hip_cvt_float_to_fp8(v, saturate ? __HIP_SATFINITE : __HIP_NOSAT, __HIP_E4M3_FNUZ)), Float8E5M2::FromBits());
  }
};

template <>
struct CastSat<Float8E5M2, half> {
  __device__ __forceinline__ Float8E5M2 operator()(half v, bool saturate) const {
    return Float8E5M2(static_cast<unsigned char>(__hip_cvt_halfraw_to_fp8(v, saturate ? __HIP_SATFINITE : __HIP_NOSAT, __HIP_E4M3_FNUZ)), Float8E5M2::FromBits());
  }
};

#else

template <>
struct CastStd<float, Float8E4M3FN> {
  __device__ __forceinline__ float operator()(Float8E4M3FN v) const {
    return v.ToFloat();
  }
};

template <>
struct CastStd<half, Float8E4M3FN> {
  __device__ __forceinline__ half operator()(Float8E4M3FN v) const {
    return __float2half(v.ToFloat());
  }
};

template <>
struct CastStd<float, Float8E5M2> {
  __device__ __forceinline__ float operator()(Float8E5M2 v) const {
    return v.ToFloat();
  }
};

template <>
struct CastStd<half, Float8E5M2> {
  __device__ __forceinline__ half operator()(Float8E5M2 v) const {
    return __float2half(v.ToFloat());
  }
};

template <>
struct CastSat<Float8E4M3FN, float> {
  __device__ __forceinline__ Float8E4M3FN operator()(float v, bool saturate) const {
    return Float8E4M3FN(v, saturate);
  }
};

template <>
struct CastSat<Float8E4M3FN, half> {
  __device__ __forceinline__ Float8E4M3FN operator()(half v, bool saturate) const {
    return Float8E4M3FN(__half2float(v), saturate);
  }
};

template <>
struct CastSat<Float8E5M2, float> {
  __device__ __forceinline__ Float8E5M2 operator()(float v, bool saturate) const {
    return Float8E5M2(v, saturate);
  }
};

template <>
struct CastSat<Float8E5M2, half> {
  __device__ __forceinline__ Float8E5M2 operator()(half v, bool saturate) const {
    return Float8E5M2(__half2float(v), saturate);
  }
};

#endif

#endif

template <int NumThreadsPerBlock, int NumElementsPerThread, typename OutT, typename InT>
__global__ void CastKernelStd(const InT* input, OutT* output, CUDA_LONG N, CastStd<OutT, InT> cast) {
  CUDA_LONG id = NumElementsPerThread * NumThreadsPerBlock * blockIdx.x + threadIdx.x;

#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      output[id] = cast(input[id]);
      id += NumThreadsPerBlock;
    }
  }
}

template <class OutT, class InT>
Status CudaCastStd(hipStream_t stream, const InT* input, OutT* output, size_t num_of_element) {
  if (num_of_element <= 0)
    return Status::OK();

  int blocksPerGrid = static_cast<int>(CeilDiv(num_of_element, GridDim::maxThreadsPerBlock * GridDim::maxElementsPerThread));
  CastKernelStd<GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread, OutT, InT><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      input,
      output,
      static_cast<int>(num_of_element),
      CastStd<OutT, InT>()
      );
  return Status::OK();
}

#if !defined(DISABLE_FLOAT8_TYPES)

template <int NumThreadsPerBlock, int NumElementsPerThread, typename OutT, typename InT>
__global__ void CastKernelSat(const InT* input, OutT* output, CUDA_LONG N, CastSat<OutT, InT> cast, bool saturate) {
  CUDA_LONG id = NumElementsPerThread * NumThreadsPerBlock * blockIdx.x + threadIdx.x;

#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      output[id] = cast(input[id], saturate);
      id += NumThreadsPerBlock;
    }
  }
}

template <class OutT, class InT>
Status CudaCastSat(hipStream_t stream, const InT* input, OutT* output, size_t num_of_element, bool saturate) {
  if (num_of_element <= 0)
    return Status::OK();

  int blocksPerGrid = static_cast<int>(CeilDiv(num_of_element, GridDim::maxThreadsPerBlock * GridDim::maxElementsPerThread));
  CastKernelSat<GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread, OutT, InT><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      input,
      output,
      static_cast<int>(num_of_element),
      CastSat<OutT, InT>(),
      saturate
      );
  return Status::OK();
}

template Status CudaCastStd<float, Float8E4M3FN>(hipStream_t stream, const Float8E4M3FN* input, float* output, size_t num_of_element);
template Status CudaCastStd<half, Float8E4M3FN>(hipStream_t stream, const Float8E4M3FN* input, half* output, size_t num_of_element);

template Status CudaCastSat<Float8E4M3FN, float>(hipStream_t stream, const float* input, Float8E4M3FN* output, size_t num_of_element, bool saturate);
template Status CudaCastSat<Float8E4M3FN, half>(hipStream_t stream, const half* input, Float8E4M3FN* output, size_t num_of_element, bool saturate);

template Status CudaCastStd<float, Float8E5M2>(hipStream_t stream, const Float8E5M2* input, float* output, size_t num_of_element);
template Status CudaCastStd<half, Float8E5M2>(hipStream_t stream, const Float8E5M2* input, half* output, size_t num_of_element);

template Status CudaCastSat<Float8E5M2, float>(hipStream_t stream, const float* input, Float8E5M2* output, size_t num_of_element, bool saturate);
template Status CudaCastSat<Float8E5M2, half>(hipStream_t stream, const half* input, Float8E5M2* output, size_t num_of_element, bool saturate);

#endif

}  // namespace cuda
}  // namespace onnxruntime
