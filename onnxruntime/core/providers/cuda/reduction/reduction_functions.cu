#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/atomic/common.cuh"
#include "reduction_functions.h"
#include "reduction_utils.cuh"

#define NUM_ELEMENTS_PER_THREAD 4
#define NUM_THREADS_PER_WARP 32
#define NUM_WARPS_PER_BLOCK 8
#define MAX_NUM_BLOCKS 256

#define ALL_ONE_MASK 0xFFFFFFFF
#define ONE_MASK 0x00000001

namespace onnxruntime {
namespace cuda {

std::pair<int, int> compute_block_size(int size) {
  int x = NUM_THREADS_PER_WARP;
  int y = std::min(NUM_WARPS_PER_BLOCK, std::max(1, size / (NUM_ELEMENTS_PER_THREAD * NUM_THREADS_PER_WARP)));
  return std::make_pair(x, y);
}

int compute_grid_size(int size) {
  const auto block = compute_block_size(size);
  return std::min(MAX_NUM_BLOCKS, std::max(1, size / (NUM_ELEMENTS_PER_THREAD * block.first * block.second)));
}

int compute_reduction_buffer_size(int element_size, int size) {
  const int num_blocks = compute_grid_size(size);
  return static_cast<int>(num_blocks * element_size + sizeof(int));
}

template<typename TIn, typename TOut, typename TOp, typename TFinalOp, bool DivideResultBySize>
__global__ void reduce_all_kernel(const int size, const TIn * data, TOut* output, TOut* buffer) {
  extern __shared__ unsigned char shared_memory_[];
  TOut* shared_memory = reinterpret_cast<TOut*>(shared_memory_);
  // Thread-level indexes:
  // Linear index of thread in block.
  const int tid_in_block = threadIdx.y * blockDim.x + threadIdx.x;
  // Total number of threads in a 2-D block.
  const int num_threads_in_block = blockDim.x * blockDim.y;

  // Warp-level indexes:
  // Warp index of thread.
  const int wid_in_block = tid_in_block / NUM_THREADS_PER_WARP;
  // Lane index of thread.
  const int lid_in_block = tid_in_block % NUM_THREADS_PER_WARP;
  // Warp count per block.
  const int num_warps_in_block = num_threads_in_block / NUM_THREADS_PER_WARP;

  // Grid-level indexes:
  // Linear index of block in grid.
  const int bid_in_grid = blockIdx.x + blockIdx.y * gridDim.x;
  // Linear index of thread in grid.
  const int tid_in_grid = bid_in_grid * (blockDim.x * blockDim.y) + tid_in_block;
  // Total number of blocks in a 2-D grid.
  const int num_blocks_in_grid = gridDim.x * gridDim.y;
  // Total number of threads in a 2-D grid with 2-D blocks.
  const int num_threads_in_grid = num_blocks_in_grid * num_threads_in_block;

  // Thread-level reduction (storage change: global memory -> register).
  // One thread reduces NUM_ELEMENTS_PER_THREAD elements to a thread register
  // in one iteration.
  TOut value = 0;
  for (int id = tid_in_grid; id < size; id += NUM_ELEMENTS_PER_THREAD * num_threads_in_grid) {
    TOut v[NUM_ELEMENTS_PER_THREAD];

    #pragma unroll
    for (int i = 0; i < NUM_ELEMENTS_PER_THREAD; i++) {
      int offset = id + i * num_threads_in_grid;
      if (offset < size) {
        v[i] = TOut(TOp()(data[offset]));
      } else {
        v[i] = TOut(0.0f);
      }
    }

    #pragma unroll
    for (int i = 0; i < NUM_ELEMENTS_PER_THREAD; i++) {
      value += v[i];
    }
  }

#if __CUDA_ARCH__ >= 700
  __syncwarp();
#else
  __syncthreads();
#endif

  // Warp-level reduction (storage change: register -> register).
  // The values in a warp will be summed up to a scalar. After warp-level
  // reduction, each block holds num_warps_in_block values in the shared memory.
  TOut value_ = value;
#pragma unroll
  for (int stride = NUM_THREADS_PER_WARP / 2; stride > 0; stride /= 2) {
    value_ += WARP_SHFL_DOWN(value_, stride);
  }

  // Return early if only one warp is used for reduction.
  // Given a fixed amount of threads, we perfer threads over warps over blocks so that we never have cases such as
  // 1. two blocks and each of them has only 1 warp (32 threads).
  // 2. two warps and each of them has only 2 threads.
  if (num_warps_in_block == 1) {
    if (tid_in_grid == 0) {
      // Compilation time if-else branch controlled by template argument can be
      // optimized out, so there will be no branch in real computation phase.
      if (DivideResultBySize) {
        output[0] = TFinalOp()(value_ / TOut(size));
      } else {
        output[0] = TFinalOp()(value_);
      }
    }
    return;
  }

  if (lid_in_block == 0) {
    shared_memory[wid_in_block] = value_;
  }

  __syncthreads();

  // Block-level reduction (storage change: shared memory -> global memory).
  // The values in a block will be summed up to a scalar.
  // Note that the values are stored in the shared memory.
  // Here we assume that the size of shared_memory is smaller
  // than num_warps_in_block, so we just keep halving the number
  // of threads in each iteartion. Our assumption is always true because
  // the size of shared_memory equals to the number of warps.
#pragma unroll
  for (int stride = NUM_WARPS_PER_BLOCK / 2; stride > 0; stride /= 2) {
    if (tid_in_block + stride < num_warps_in_block) {
      shared_memory[tid_in_block] += shared_memory[tid_in_block + stride];
    }
    __syncthreads();
  }

  // Return early if only one block is used for reduction.
  if (num_blocks_in_grid == 1) {
    if (tid_in_grid == 0) {
      // Compilation time if-else branch controlled by template argument can be
      // optimized out, so there will be no branch in real computation phase.
      if (DivideResultBySize) {
        output[0] = TFinalOp()(shared_memory[0] / TOut(size));
      } else {
        output[0] = TFinalOp()(shared_memory[0]);
      }
    }
    return;
  }

  if (tid_in_block == 0) {
    buffer[bid_in_grid] = shared_memory[0];
  }

  __threadfence();
  __syncthreads();

  // Grid-level reduciton. We use the last block to sum up values
  // stored in the global buffer.
  __shared__ bool is_last_block_done;

  if (tid_in_block == 0) {
    int* p_lock = reinterpret_cast<int*>(buffer + num_blocks_in_grid);
    int count = atomicAdd(p_lock, 1);
    is_last_block_done = (count == (num_blocks_in_grid - 1));
  }

  // All threads in each block see if they belong the last active block
  // (i.e., the value of is_last_block_done).
  __syncthreads();

  // Only the block which saw that count equals to num_blocks_in_grid - 1 can
  // enter the following block.
  if (is_last_block_done) {
    const int pow2_bound = least_pow2_bound(num_blocks_in_grid);
    for (int stride = pow2_bound / 2; stride > 0; stride /= 2) {
      if (tid_in_block < stride && tid_in_block + stride < num_blocks_in_grid) {
        buffer[tid_in_block] += buffer[tid_in_block + stride];
      }
      __syncthreads();
    }

    // The first thread in the last block assigns the final output.
    if (tid_in_block == 0) {
      // Compilation time if-else branch controlled by template argument can be
      // optimized out, so there will be no branch in real computation phase.
      if (DivideResultBySize) {
        output[0] = TFinalOp()(buffer[0] / TOut(size));
      } else {
        output[0] = TFinalOp()(buffer[0]);
      }
    }
  }
}

template<typename TIn, typename TOut, typename TOp, typename TFinalOp, bool DivideResultBySize>
void call_reduce_all_kernel(const TIn *data, TOut *output, int size, TOut *buffer) {
  const auto block_size = compute_block_size(size);
  const int num_blocks = compute_grid_size(size);
  const dim3 block(block_size.first, block_size.second, 1);
  const dim3 grid(num_blocks, 1, 1);

  // If more than one blocks are used, then inter-blocks reduction is needed.
  if (num_blocks != 1) {
    hipMemset(buffer + num_blocks, 0, sizeof(int));
  }

  const int shared_mem_size = sizeof(TOut) * block_size.first * block_size.second / NUM_THREADS_PER_WARP;
  reduce_all_kernel<TIn, TOut, TOp, TFinalOp, DivideResultBySize><<<grid, block, shared_mem_size>>>(size, data, output, buffer);
}

template <typename TIn, typename TOut>
void reduce_sum(const TIn* data, TOut* output, int size, TOut* buffer) {
  call_reduce_all_kernel<TIn, TOut, Cast<TOut, TIn>, Identity<TOut>, false>(
      data, output, size, buffer);
}

template <typename TIn, typename TOut>
void reduce_square_sum(const TIn* data, TOut* output, int size, TOut* buffer) {
  call_reduce_all_kernel<TIn, TOut, Square<TOut, TIn>, Identity<TOut>, false>(
      data, output, size, buffer);
}

template <typename TIn, typename TOut>
void reduce_l2_norm(const TIn* data, TOut* output, int size, TOut* buffer) {
  call_reduce_all_kernel<TIn, TOut, Square<TOut, TIn>, Sqrt<TOut>, false>(
      data, output, size, buffer);
}

template <typename TIn, typename TOut>
void reduce_mean(const TIn* data, TOut* output, int size, TOut* buffer) {
  call_reduce_all_kernel<TIn, TOut, Cast<TOut, TIn>, Identity<TOut>, true>(
      data, output, size, buffer);
}

template void reduce_sum<half, float>(
    const half* data, float* output, int size, float* buffer);
template void reduce_sum<float, float>(
    const float* data, float* output, int size, float* buffer);
template void reduce_sum<double, double>(
    const double* data, double* output, int size, double* buffer);

template void reduce_square_sum<half, float>(
    const half* data, float* output, int size, float* buffer);
template void reduce_square_sum<float, float>(
    const float* data, float* output, int size, float* buffer);
template void reduce_square_sum<double, double>(
    const double* data, double* output, int size, double* buffer);

template void reduce_l2_norm<half, float>(
    const half* data, float* output, int size, float* buffer);
template void reduce_l2_norm<float, float>(
    const float* data, float* output, int size, float* buffer);
template void reduce_l2_norm<double, double>(
    const double* data, double* output, int size, double* buffer);

template void reduce_mean<half, float>(
    const half* data, float* output, int size, float* buffer);
template void reduce_mean<float, float>(
    const float* data, float* output, int size, float* buffer);
template void reduce_mean<double, double>(
    const double* data, double* output, int size, double* buffer);

bool is_matrix_row_reduction(
    const hipdnnReduceTensorOp_t cudnn_reduce_op,
    const int m,
    const int n,
    const size_t rank,
    std::vector<int64_t> axes) {
  if (m < 1)
    return false;

  if (n < 1)
    return false;

  if (rank < 2)
    return false;

  if (cudnn_reduce_op != HIPDNN_REDUCE_TENSOR_ADD)
    return false;

  //empty axes, default reduction
  if (axes.size() < 1)
    return false;

  return true;
}

template <typename TIn, typename TOut, typename TBuf>
__global__ void reduce_matrix_rows_kernel(const TIn* input, TOut* output, int m, int n) {
  constexpr int x_load_count_per_thread = 1;
  constexpr int y_load_count_per_thread = 4;
  const int t_count_x_in_grid = blockDim.x * gridDim.x;
  const int t_count_y_in_grid = blockDim.y * gridDim.y;
  const int x_grid_stride = t_count_x_in_grid * x_load_count_per_thread;
  const int y_grid_stride = t_count_y_in_grid * y_load_count_per_thread;
  const int tid_x_in_grid = threadIdx.x + blockDim.x * blockIdx.x;
  const int tid_y_in_grid = threadIdx.y + blockDim.y * blockIdx.y;
  const int tid_in_block = threadIdx.x + blockDim.x * threadIdx.y;

  // Shape is blockDim.y-by-blockDim.x and element type is TBuf.
  extern __shared__ unsigned char shared_memory_[];
  TBuf* shared_memory = reinterpret_cast<TBuf*>(shared_memory_);

  // to prevent int overflow in index calculation for input size m*n
  const int64_t n_int64 = static_cast<int64_t>(n);

  for (int col = tid_x_in_grid; col < n; col += x_grid_stride) {
    shared_memory[tid_in_block] = TBuf(0.0f);

    // This loops load multiple blockDim.y-by-blockDim.x sub-tensors from the input.
    for (int row = tid_y_in_grid; row < m; row += y_grid_stride) {
      TBuf sum = 0.0f;
// Thread-level reduction. Each thread loads y_load_count_per_thread values
// and aggregrate them.
#pragma unroll(y_load_count_per_thread)
      for (int row_inner = 0; row_inner < y_load_count_per_thread; ++row_inner) {
        int row_final = row + row_inner * t_count_y_in_grid;
        int col_final = col;
        if (row_final < m && col_final < n) {
          sum += TBuf(input[row_final * n_int64 + col_final]);
        }
      }
      // Write thread-level reduction result into shared memory.
      shared_memory[tid_in_block] += sum;
    }

    // Wait all threads to finish their thread-level reductions.
    __syncthreads();

// This loop conducts reduction on elements stored in shared memory.
// Each block reduces blockDim.y-by-blockDim.x tensor to 1-by-blockDim.x tensor.
#pragma unroll(4)
    for (int stride = blockDim.y / 2; stride > 0; stride /= 2) {
      if (threadIdx.y < stride) {
        shared_memory[tid_in_block] += shared_memory[tid_in_block + stride * blockDim.x];
      }
      __syncthreads();
    }

    if (threadIdx.y == 0) {
      atomic_add(output + col, TOut(shared_memory[threadIdx.x]));
    }

    // Make sure all values in shared memory have been written into the output memory.
    __syncthreads();
  }
}

// This function reduces the given input tensor along all but the last axis.
// For example, [N, C, H, W]-tensor may lead to a output [W]-tensor.
// It's implementation is in reduction_ops.cu and called in reduction_ops.cc.
template <typename TIn, typename TOut, typename TBuf>
void call_reduce_matrix_rows(const TIn* input, TOut* output, int m, int n) {
  constexpr int max_num_threads_in_block = 512;
  constexpr int max_num_blocks_in_grid = 512;
  constexpr int load_count_per_thread = 4;

  const int block_x_dim = least_pow2_bound(std::max(1, std::min(n, GPU_WARP_SIZE)));
  const int block_y_dim = least_pow2_bound(std::max(1, std::min(max_num_threads_in_block / block_x_dim, m / load_count_per_thread)));
  const int grid_x_dim = std::max(1, std::min(n / block_x_dim, max_num_blocks_in_grid));
  const int grid_y_dim = std::max(1, std::min(max_num_blocks_in_grid / grid_x_dim, m / block_y_dim / 4));

  const dim3 grid(grid_x_dim, grid_y_dim, 1);
  const dim3 block(block_x_dim, block_y_dim, 1);

  reduce_matrix_rows_kernel<TIn, TOut, TBuf><<<grid, block, block.y * block.x * sizeof(TBuf)>>>(
      input, output, m, n);
}

template <typename TIn, typename TOut>
void reduce_matrix_rows(const TIn* data, TOut* output, int m, int n) {
  call_reduce_matrix_rows<TIn, TOut, TOut>(data, output, m, n);
}

template <>
void reduce_matrix_rows<half, half>(const half* data, half* output, int m, int n) {
  call_reduce_matrix_rows<half, half, float>(data, output, m, n);
}

template void reduce_matrix_rows<float, float>(
    const float* data, float* output, int m, int n);
template void reduce_matrix_rows<double, double>(
    const double* data, double* output, int m, int n);

}  // namespace cuda
}  // namespace onnxruntime
