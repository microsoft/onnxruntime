#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_runtime.h>
#include "unary_elementwise_ops_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cu_inc/unary_elementwise_impl.cuh"

#if defined(CUDA_VERSION) && CUDA_VERSION >= 11080
#include "hip/hip_fp8.h"
#endif

namespace onnxruntime {
namespace cuda {

#define OP(name, expr)                                     \
  template <typename T>                                    \
  struct OP_##name {                                       \
    __device__ __inline__ T operator()(const T& a) const { \
      return expr;                                         \
    }                                                      \
  };

#define UNARY_ELEMENTWISE_IMPL(name)         \
  UNARY_ELEMENTWISE_IMPL_DECLARATION(name) { \
    UnaryElementWiseImpl(stream,             \
                         input_data,         \
                         output_data,        \
                         OP_##name<T>(),     \
                         count);             \
  }

#define SPECIALIZED_UNARY_ELEMENTWISE_IMPL(name, T) \
  template void Impl_##name<T>(hipStream_t stream, const T* input_data, T* output_data, size_t count);

#define UNARY_OP_NAME_EXPR(name, expr) \
  OP(name, expr)                       \
  UNARY_ELEMENTWISE_IMPL(name)

UNARY_OPS()
#undef UNARY_OP_NAME_EXPR

// the postfix of means the types supported by the op:
// B: uint8_t
// W: uint16_t
// U: uint32_t
// Z: uint64_t
// C: int8_t
// S: int16_t
// I: int32_t
// L: int64_t
// H: float16
// F: float
// D: double
// O: bool

#define SPECIALIZED_UNARY_ELEMENTWISE_IMPL_HFD(name) \
  SPECIALIZED_UNARY_ELEMENTWISE_IMPL(name, half)     \
  SPECIALIZED_UNARY_ELEMENTWISE_IMPL(name, float)    \
  SPECIALIZED_UNARY_ELEMENTWISE_IMPL(name, double)

#define SPECIALIZED_UNARY_ELEMENTWISE_IMPL_HFDB(name) \
  SPECIALIZED_UNARY_ELEMENTWISE_IMPL_HFD(name)        \
  SPECIALIZED_UNARY_ELEMENTWISE_IMPL(name, BFloat16)

#define SPECIALIZED_UNARY_ELEMENTWISE_IMPL_CSILHFD(name) \
  SPECIALIZED_UNARY_ELEMENTWISE_IMPL(name, int8_t)       \
  SPECIALIZED_UNARY_ELEMENTWISE_IMPL(name, int16_t)      \
  SPECIALIZED_UNARY_ELEMENTWISE_IMPL(name, int32_t)      \
  SPECIALIZED_UNARY_ELEMENTWISE_IMPL(name, int64_t)      \
  SPECIALIZED_UNARY_ELEMENTWISE_IMPL_HFD(name)

#define SPECIALIZED_UNARY_ELEMENTWISE_IMPL_BWUZCSILHFD(name) \
  SPECIALIZED_UNARY_ELEMENTWISE_IMPL(name, uint8_t)          \
  SPECIALIZED_UNARY_ELEMENTWISE_IMPL(name, uint16_t)         \
  SPECIALIZED_UNARY_ELEMENTWISE_IMPL(name, uint32_t)         \
  SPECIALIZED_UNARY_ELEMENTWISE_IMPL(name, uint64_t)         \
  SPECIALIZED_UNARY_ELEMENTWISE_IMPL_CSILHFD(name)

SPECIALIZED_UNARY_ELEMENTWISE_IMPL_BWUZCSILHFD(Abs)
SPECIALIZED_UNARY_ELEMENTWISE_IMPL_CSILHFD(Neg)
SPECIALIZED_UNARY_ELEMENTWISE_IMPL_HFD(Floor)
SPECIALIZED_UNARY_ELEMENTWISE_IMPL_HFD(Ceil)
SPECIALIZED_UNARY_ELEMENTWISE_IMPL_HFD(Reciprocal)
SPECIALIZED_UNARY_ELEMENTWISE_IMPL_HFD(Sqrt)
SPECIALIZED_UNARY_ELEMENTWISE_IMPL_HFDB(Log)
SPECIALIZED_UNARY_ELEMENTWISE_IMPL_HFDB(Exp)
SPECIALIZED_UNARY_ELEMENTWISE_IMPL_HFD(Erf)
SPECIALIZED_UNARY_ELEMENTWISE_IMPL_HFD(Round)
SPECIALIZED_UNARY_ELEMENTWISE_IMPL_HFD(Sin)
SPECIALIZED_UNARY_ELEMENTWISE_IMPL_HFD(Cos)
SPECIALIZED_UNARY_ELEMENTWISE_IMPL(Not, bool)
SPECIALIZED_UNARY_ELEMENTWISE_IMPL_BWUZCSILHFD(Sign)

// When casting, half needs to be converted via float type from most other types
template <typename T>
struct ViaTypeMap {
  typedef T ViaT;
};

template <>
struct ViaTypeMap<half> {
  typedef float ViaT;
};

template <>
struct ViaTypeMap<BFloat16> {
  typedef float ViaT;
};

template <typename InT, typename OutT>
struct OP_Cast {
  __device__ __inline__ OutT operator()(const InT& a) const {
    const bool any_float16 = std::is_same<half, InT>::value || std::is_same<half, OutT>::value;
    const bool any_bf16 = std::is_same<BFloat16, InT>::value || std::is_same<BFloat16, OutT>::value;
    typedef typename std::conditional<any_bf16, BFloat16, OutT>::type T1;
    typedef typename std::conditional<any_float16, half, T1>::type T;
    typedef typename ViaTypeMap<T>::ViaT ViaT;
    return (OutT)((ViaT)a);
  }
};

#define IMPL_CAST_IMPL(InT, OutT)                                                                        \
  void Explicit_Impl_Cast(hipStream_t stream, const InT* input_data, OutT* output_data, size_t count) { \
    UnaryElementWiseImpl(stream, input_data, output_data, OP_Cast<InT, OutT>(), count);                  \
  }

#define IMPL_CAST_IMPL_THROW(InT, OutT)                                                                  \
  void Explicit_Impl_Cast(hipStream_t stream, const InT* input_data, OutT* output_data, size_t count) { \
    ORT_THROW("Cast from " #InT " to " #OutT " must define saturate.");                                  \
  }

#if !defined(DISABLE_FLOAT8_TYPES)

#define IMPL_CAST_IMPL_FROM(T)            \
  IMPL_CAST_IMPL(T, half)                 \
  IMPL_CAST_IMPL(T, float)                \
  IMPL_CAST_IMPL(T, double)               \
  IMPL_CAST_IMPL(T, int8_t)               \
  IMPL_CAST_IMPL(T, int16_t)              \
  IMPL_CAST_IMPL(T, int32_t)              \
  IMPL_CAST_IMPL(T, int64_t)              \
  IMPL_CAST_IMPL(T, uint8_t)              \
  IMPL_CAST_IMPL(T, uint16_t)             \
  IMPL_CAST_IMPL(T, uint32_t)             \
  IMPL_CAST_IMPL(T, uint64_t)             \
  IMPL_CAST_IMPL(T, bool)                 \
  IMPL_CAST_IMPL(T, BFloat16)             \
  IMPL_CAST_IMPL_THROW(T, Float8E4M3FN)   \
  IMPL_CAST_IMPL_THROW(T, Float8E5M2)     \
  IMPL_CAST_IMPL_THROW(T, Float8E4M3FNUZ) \
  IMPL_CAST_IMPL_THROW(T, Float8E5M2FNUZ)

#else

#define IMPL_CAST_IMPL_FROM(T) \
  IMPL_CAST_IMPL(T, half)      \
  IMPL_CAST_IMPL(T, float)     \
  IMPL_CAST_IMPL(T, double)    \
  IMPL_CAST_IMPL(T, int8_t)    \
  IMPL_CAST_IMPL(T, int16_t)   \
  IMPL_CAST_IMPL(T, int32_t)   \
  IMPL_CAST_IMPL(T, int64_t)   \
  IMPL_CAST_IMPL(T, uint8_t)   \
  IMPL_CAST_IMPL(T, uint16_t)  \
  IMPL_CAST_IMPL(T, uint32_t)  \
  IMPL_CAST_IMPL(T, uint64_t)  \
  IMPL_CAST_IMPL(T, bool)      \
  IMPL_CAST_IMPL(T, BFloat16)

#endif

IMPL_CAST_IMPL_FROM(half)
IMPL_CAST_IMPL_FROM(float)
IMPL_CAST_IMPL_FROM(double)
IMPL_CAST_IMPL_FROM(int8_t)
IMPL_CAST_IMPL_FROM(int16_t)
IMPL_CAST_IMPL_FROM(int32_t)
IMPL_CAST_IMPL_FROM(int64_t)
IMPL_CAST_IMPL_FROM(uint8_t)
IMPL_CAST_IMPL_FROM(uint16_t)
IMPL_CAST_IMPL_FROM(uint32_t)
IMPL_CAST_IMPL_FROM(uint64_t)
IMPL_CAST_IMPL_FROM(bool)
IMPL_CAST_IMPL_FROM(BFloat16)
#if !defined(DISABLE_FLOAT8_TYPES)
IMPL_CAST_IMPL_FROM(Float8E4M3FN)
IMPL_CAST_IMPL_FROM(Float8E5M2)
#endif

template <typename InT, typename OutT>
struct OP_CastSat {
  __device__ __inline__ OutT operator()(const InT& a) const;
};

template <typename InT, typename OutT>
struct OP_CastNoSat {
  __device__ __inline__ OutT operator()(const InT& a) const;
};

#if defined(CUDA_VERSION) && CUDA_VERSION >= 11080

#define OP_CAST(T, NVT)                                                                                     \
  template <>                                                                                               \
  struct OP_CastSat<half, T> {                                                                              \
    __device__ __inline__ T operator()(const half& v) const {                                               \
      return T(static_cast<unsigned char>(__hip_cvt_halfraw_to_fp8(v, __HIP_SATFINITE, NVT)), T::FromBits()); \
    }                                                                                                       \
  };                                                                                                        \
  template <>                                                                                               \
  struct OP_CastNoSat<half, T> {                                                                            \
    __device__ __inline__ T operator()(const half& v) const {                                               \
      return T(static_cast<unsigned char>(__hip_cvt_halfraw_to_fp8(v, __HIP_NOSAT, NVT)), T::FromBits());     \
    }                                                                                                       \
  };                                                                                                        \
  template <>                                                                                               \
  struct OP_CastSat<float, T> {                                                                             \
    __device__ __inline__ T operator()(const float& v) const {                                              \
      return T(static_cast<unsigned char>(__hip_cvt_float_to_fp8(v, __HIP_SATFINITE, NVT)), T::FromBits());   \
    }                                                                                                       \
  };                                                                                                        \
  template <>                                                                                               \
  struct OP_CastNoSat<float, T> {                                                                           \
    __device__ __inline__ T operator()(const float& v) const {                                              \
      return T(static_cast<unsigned char>(__hip_cvt_float_to_fp8(v, __HIP_NOSAT, NVT)), T::FromBits());       \
    }                                                                                                       \
  };

#else

#define OP_CAST(T, NVT)                                        \
  template <>                                                  \
  struct OP_CastSat<half, T> {                                 \
    __device__ __inline__ T operator()(const half& v) const {  \
      return T(__half2float(v), true);                         \
    }                                                          \
  };                                                           \
  template <>                                                  \
  struct OP_CastNoSat<half, T> {                               \
    __device__ __inline__ T operator()(const half& v) const {  \
      return T(__half2float(v), false);                        \
    }                                                          \
  };                                                           \
  template <>                                                  \
  struct OP_CastSat<float, T> {                                \
    __device__ __inline__ T operator()(const float& v) const { \
      return T(v, true);                                       \
    }                                                          \
  };                                                           \
  template <>                                                  \
  struct OP_CastNoSat<float, T> {                              \
    __device__ __inline__ T operator()(const float& v) const { \
      return T(v, false);                                      \
    }                                                          \
  };

#endif

#if !defined(DISABLE_FLOAT8_TYPES)

OP_CAST(Float8E4M3FN, __HIP_E4M3_FNUZ)
OP_CAST(Float8E5M2, __HIP_E5M2_FNUZ)

#define EXPLICIT_IMPL_CASTSAT(InT, OutT)                                                                                   \
  void Explicit_Impl_CastSat(hipStream_t stream, const InT* input_data, OutT* output_data, size_t count, bool saturate) { \
    if (saturate) {                                                                                                        \
      UnaryElementWiseImpl(stream, input_data, output_data, OP_CastSat<InT, OutT>(), count);                               \
    } else {                                                                                                               \
      UnaryElementWiseImpl(stream, input_data, output_data, OP_CastNoSat<InT, OutT>(), count);                             \
    }                                                                                                                      \
  }

EXPLICIT_IMPL_CASTSAT(float, Float8E4M3FN)
EXPLICIT_IMPL_CASTSAT(half, Float8E4M3FN)
EXPLICIT_IMPL_CASTSAT(float, Float8E5M2)
EXPLICIT_IMPL_CASTSAT(half, Float8E5M2)

// TODO: enable bfloat16 in another PR.
/*
EXPLICIT_IMPL_CASTSAT(__hip_bfloat16, Float8E4M3FN)
EXPLICIT_IMPL_CASTSAT(__hip_bfloat16, Float8E5M2)
*/

#endif

}  // namespace cuda
}  // namespace onnxruntime
