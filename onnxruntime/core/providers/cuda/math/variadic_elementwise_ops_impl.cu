#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/math/variadic_elementwise_ops_impl.h"

#include "core/providers/cuda/cu_inc/variadic_elementwise_impl.cuh"
#include "core/providers/cuda/math/binary_elementwise_ops_impl.h"
#include "core/providers/cuda/math/binary_elementwise_ops_impl_functors.cuh"
#include "core/providers/cuda/math/variadic_elementwise_ops_tags.h"

namespace onnxruntime {
namespace cuda {

template <typename T, typename VariadicElementwiseOpTag>
struct VariadicElementwiseOpTraits;

#define DEFINE_TRAITS(VariadicElementwiseOpTag, ImplName)           \
  template <typename T>                                             \
  struct VariadicElementwiseOpTraits<T, VariadicElementwiseOpTag> { \
    using ScalarComputeFunctor = OP_##ImplName<T, T, T>;            \
                                                                    \
    static void ComputeFn(                                          \
        int32_t output_rank_or_simple_broadcast,                    \
        const TArray<int64_t>* lhs_padded_strides,                  \
        const T* lhs_data,                                          \
        const TArray<int64_t>* rhs_padded_strides,                  \
        const T* rhs_data,                                          \
        const TArray<fast_divmod>* fdm_output_strides,              \
        const fast_divmod& fdm_H,                                   \
        const fast_divmod& fdm_C,                                   \
        T* output_data,                                             \
        size_t count) {                                             \
      Impl_##ImplName(                                              \
          output_rank_or_simple_broadcast,                          \
          lhs_padded_strides,                                       \
          lhs_data,                                                 \
          rhs_padded_strides,                                       \
          rhs_data,                                                 \
          fdm_output_strides,                                       \
          fdm_H,                                                    \
          fdm_C,                                                    \
          output_data,                                              \
          count);                                                   \
    }                                                               \
  };

DEFINE_TRAITS(variadic_elementwise_ops::Sum, Add)
DEFINE_TRAITS(variadic_elementwise_ops::Min, Min)
DEFINE_TRAITS(variadic_elementwise_ops::Max, Max)

#undef DEFINE_TRAITS

template <typename T, typename VariadicElementwiseOpTag>
void Impl_General(
    int32_t output_rank_or_simple_broadcast,
    const TArray<int64_t>* lhs_padded_strides,
    const T* lhs_data,
    const TArray<int64_t>* rhs_padded_strides,
    const T* rhs_data,
    const TArray<fast_divmod>* fdm_output_strides,
    const fast_divmod& fdm_H,
    const fast_divmod& fdm_C,
    T* output_data,
    size_t count) {
  VariadicElementwiseOpTraits<T, VariadicElementwiseOpTag>::ComputeFn(
      output_rank_or_simple_broadcast,
      lhs_padded_strides,
      lhs_data,
      rhs_padded_strides,
      rhs_data,
      fdm_output_strides,
      fdm_H,
      fdm_C,
      output_data,
      count);
}

template <typename T, typename VariadicElementwiseOpTag>
void Impl_NoBroadcastInputBatch(
    InputBatchArray<T> input_data_batch,
    T* output_data,
    size_t count) {
  VariadicElementWiseNoBroadcastInputBatchImpl<
      T, typename VariadicElementwiseOpTraits<T, VariadicElementwiseOpTag>::ScalarComputeFunctor,
      k_max_input_batch_size>(
      typename VariadicElementwiseOpTraits<T, VariadicElementwiseOpTag>::ScalarComputeFunctor{},
      count,
      input_data_batch,
      output_data);
}

#define SPECIALIZE_IMPL(T, VariadicElementwiseOpTag)                     \
  template void Impl_General<T, VariadicElementwiseOpTag>(               \
      int32_t output_rank_or_simple_broadcast,                           \
      const TArray<int64_t>* lhs_padded_strides,                         \
      const T* lhs_data,                                                 \
      const TArray<int64_t>* rhs_padded_strides,                         \
      const T* rhs_data,                                                 \
      const TArray<fast_divmod>* fdm_output_strides,                     \
      const fast_divmod& fdm_H,                                          \
      const fast_divmod& fdm_C,                                          \
      T* output_data,                                                    \
      size_t count);                                                     \
                                                                         \
  template void Impl_NoBroadcastInputBatch<T, VariadicElementwiseOpTag>( \
      InputBatchArray<T> input_data_batch,                               \
      T * output_data,                                                   \
      size_t count);

// the postfix means the types supported by the op:
// B: uint8_t
// W: uint16_t
// U: uint32_t
// Z: uint64_t
// C: int8_t
// S: int16_t
// I: int32_t
// L: int64_t
// H: float16
// F: float
// D: double
// O: bool

#if CUDA_VERSION >= 11000 && (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
#define SPECIALIZE_IMPL_BF16(VariadicElementwiseOpTag) SPECIALIZE_IMPL(hip_bfloat16, VariadicElementwiseOpTag)
#else
#define SPECIALIZE_IMPL_BF16(VariadicElementwiseOpTag)
#endif

#define SPECIALIZE_IMPL_HFD(VariadicElementwiseOpTag) \
  SPECIALIZE_IMPL(half, VariadicElementwiseOpTag)     \
  SPECIALIZE_IMPL_BF16(VariadicElementwiseOpTag)      \
  SPECIALIZE_IMPL(float, VariadicElementwiseOpTag)    \
  SPECIALIZE_IMPL(double, VariadicElementwiseOpTag)

#define SPECIALIZE_IMPL_UZILHFD(VariadicElementwiseOpTag) \
  SPECIALIZE_IMPL(uint32_t, VariadicElementwiseOpTag)     \
  SPECIALIZE_IMPL(uint64_t, VariadicElementwiseOpTag)     \
  SPECIALIZE_IMPL(int32_t, VariadicElementwiseOpTag)      \
  SPECIALIZE_IMPL(int64_t, VariadicElementwiseOpTag)      \
  SPECIALIZE_IMPL_HFD(VariadicElementwiseOpTag)

SPECIALIZE_IMPL_HFD(variadic_elementwise_ops::Sum)
SPECIALIZE_IMPL_UZILHFD(variadic_elementwise_ops::Min)
SPECIALIZE_IMPL_UZILHFD(variadic_elementwise_ops::Max)

#undef SPECIALIZE_IMPL_UZILHFD
#undef SPECIALIZE_IMPL_HFD
#undef SPECIALIZE_IMPL

}  // namespace cuda
}  // namespace onnxruntime
