#include "hip/hip_runtime.h"
/**
* Copyright (c) 2016-present, Facebook, Inc.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

/* Modifications Copyright (c) Microsoft. */

// The code below is mostly copied from Pytorch PersistentSoftmax.cuh

#include "core/providers/cuda/cu_inc/common.cuh"
#include "softmax_warpwise_impl.cuh"
#include "softmax_blockwise_impl.cuh"
#include "softmax.h"

#include <limits>

namespace onnxruntime {
namespace cuda {


template <typename input_t, typename output_t, typename acc_t, bool is_log_softmax>
void dispatch_warpwise_softmax_forward(hipStream_t stream, output_t* dst, const input_t* src, int softmax_elements, int softmax_elements_stride, int batch_count) {
  if (softmax_elements == 0) {
    return;
  } else {
    int log2_elements = log2_ceil(softmax_elements);
    const int next_power_of_two = 1 << log2_elements;

    // This value must match the WARP_SIZE constexpr value computed inside softmax_warp_forward.
    int warp_size = (next_power_of_two < GPU_WARP_SIZE) ? next_power_of_two : GPU_WARP_SIZE;

    // This value must match the WARP_BATCH constexpr value computed inside softmax_warp_forward.
    int batches_per_warp = (next_power_of_two <= 128) ? 2 : 1;

    // use 128 threads per block to maximimize gpu utilization
    constexpr int threads_per_block = 128;

    int warps_per_block = (threads_per_block / warp_size);
    int batches_per_block = warps_per_block * batches_per_warp;
    int blocks = (batch_count + batches_per_block - 1) / batches_per_block;
    dim3 threads(warp_size, warps_per_block, 1);
    // Launch code would be more elegant if C++ supported FOR CONSTEXPR
    switch (log2_elements) {
      case 0:  // 1
        softmax_warp_forward<input_t, output_t, acc_t, 0, is_log_softmax>
            <<<blocks, threads, 0, stream>>>(dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 1:  // 2
        softmax_warp_forward<input_t, output_t, acc_t, 1, is_log_softmax>
            <<<blocks, threads, 0, stream>>>(dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 2:  // 4
        softmax_warp_forward<input_t, output_t, acc_t, 2, is_log_softmax>
            <<<blocks, threads, 0, stream>>>(dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 3:  // 8
        softmax_warp_forward<input_t, output_t, acc_t, 3, is_log_softmax>
            <<<blocks, threads, 0, stream>>>(dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 4:  // 16
        softmax_warp_forward<input_t, output_t, acc_t, 4, is_log_softmax>
            <<<blocks, threads, 0, stream>>>(dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 5:  // 32
        softmax_warp_forward<input_t, output_t, acc_t, 5, is_log_softmax>
            <<<blocks, threads, 0, stream>>>(dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 6:  // 64
        softmax_warp_forward<input_t, output_t, acc_t, 6, is_log_softmax>
            <<<blocks, threads, 0, stream>>>(dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 7:  // 128
        softmax_warp_forward<input_t, output_t, acc_t, 7, is_log_softmax>
            <<<blocks, threads, 0, stream>>>(dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 8:  // 256
        softmax_warp_forward<input_t, output_t, acc_t, 8, is_log_softmax>
            <<<blocks, threads, 0, stream>>>(dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 9:  // 512
        softmax_warp_forward<input_t, output_t, acc_t, 9, is_log_softmax>
            <<<blocks, threads, 0, stream>>>(dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      case 10:  // 1024
        softmax_warp_forward<input_t, output_t, acc_t, 10, is_log_softmax>
            <<<blocks, threads, 0, stream>>>(dst, src, batch_count, softmax_elements_stride, softmax_elements);
        break;
      default:
        break;
    }
  }
}

#define SPECIALIZED_WRAPWISE_SOFTMAX_IMPL(input_t, output_t, acc_t) \
template void dispatch_warpwise_softmax_forward<input_t, output_t, acc_t, false>(hipStream_t stream, output_t * dst, const input_t* src, int softmax_elements, int softmax_elements_stride, int batch_count); \
template void dispatch_warpwise_softmax_forward<input_t, output_t, acc_t, true>(hipStream_t stream, output_t * dst, const input_t* src, int softmax_elements, int softmax_elements_stride, int batch_count);

SPECIALIZED_WRAPWISE_SOFTMAX_IMPL(float, float, float)
SPECIALIZED_WRAPWISE_SOFTMAX_IMPL(half, half, float)
SPECIALIZED_WRAPWISE_SOFTMAX_IMPL(double, double, double)
SPECIALIZED_WRAPWISE_SOFTMAX_IMPL(BFloat16, BFloat16, float)

template <typename input_t, typename output_t, typename acc_t, bool is_log_softmax>
void dispatch_blockwise_softmax_forward(hipStream_t stream, output_t* output, const input_t* input, int softmax_elements, int softmax_elements_stride, int batch_count) {
  dim3 grid(batch_count);
  constexpr int ILP = sizeof(float4) / sizeof(input_t);
  dim3 block = SoftMax_getBlockSize(ILP, softmax_elements);
  if (is_log_softmax) {
    softmax_block_forward<ILP, input_t, acc_t, output_t, LogSoftMaxForwardEpilogue>
      <<<grid, block, block.x * sizeof(acc_t), stream>>>(output, const_cast<input_t*>(input), softmax_elements);
  } else {
    softmax_block_forward<ILP, input_t, acc_t, output_t, SoftMaxForwardEpilogue>
      <<<grid, block, block.x * sizeof(acc_t), stream>>>(output, const_cast<input_t*>(input), softmax_elements);
  }
}

#define SPECIALIZED_BLOCKWISE_SOFTMAX_IMPL(input_t, output_t, acc_t)                                                 \
  template void dispatch_blockwise_softmax_forward<input_t, output_t, acc_t, false>(                                 \
      hipStream_t stream, output_t * output, const input_t* src, int softmax_elements, int softmax_elements_stride, \
      int batch_count);                                                                                              \
  template void dispatch_blockwise_softmax_forward<input_t, output_t, acc_t, true>(                                  \
      hipStream_t stream, output_t * output, const input_t* src, int softmax_elements, int softmax_elements_stride, \
      int batch_count);

SPECIALIZED_BLOCKWISE_SOFTMAX_IMPL(float, float, float)
SPECIALIZED_BLOCKWISE_SOFTMAX_IMPL(half, half, float)
SPECIALIZED_BLOCKWISE_SOFTMAX_IMPL(double, double, double)
SPECIALIZED_BLOCKWISE_SOFTMAX_IMPL(BFloat16, BFloat16, float)

#ifndef DISABLE_CONTRIB_OPS
SPECIALIZED_BLOCKWISE_SOFTMAX_IMPL(half, float, float) // used by BeamSearch op
#endif

}
}
