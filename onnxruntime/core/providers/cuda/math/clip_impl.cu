#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/math/clip_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"

namespace onnxruntime {
namespace cuda {
template <typename T>
__global__ void _Clip(const T* input, T* output, T min, T max, size_t N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  output[id] = (input[id] < min) ? min : ((input[id] > max) ? max : input[id]);
}

template <typename T>
void ClipImpl(hipStream_t stream, const T* input_data, T* output_data, T min, T max, size_t count) {
  typedef typename ToCudaType<T>::MappedType CudaT;

  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  _Clip<CudaT><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(reinterpret_cast<const CudaT*>(input_data),
                                                                  reinterpret_cast<CudaT*>(output_data),
                                                                  *reinterpret_cast<CudaT*>(&min),
                                                                  *reinterpret_cast<CudaT*>(&max),
                                                                  count);
}

template void ClipImpl<float>(hipStream_t stream, const float* input_data, float* output_data, float min, float max, size_t count);
template void ClipImpl<double>(hipStream_t stream, const double* input_data, double* output_data, double min, double max, size_t count);
template void ClipImpl<MLFloat16>(hipStream_t stream, const MLFloat16* input_data, MLFloat16* output_data, MLFloat16 min, MLFloat16 max, size_t count);
template void ClipImpl<int8_t>(hipStream_t stream, const int8_t* input_data, int8_t* output_data, int8_t min, int8_t max, size_t count);
template void ClipImpl<uint8_t>(hipStream_t stream, const uint8_t* input_data, uint8_t* output_data, uint8_t min, uint8_t max, size_t count);
template void ClipImpl<int64_t>(hipStream_t stream, const int64_t* input_data, int64_t* output_data, int64_t min, int64_t max, size_t count);
template void ClipImpl<uint64_t>(hipStream_t stream, const uint64_t* input_data, uint64_t* output_data, uint64_t min, uint64_t max, size_t count);

}  // namespace cuda
}  // namespace onnxruntime
