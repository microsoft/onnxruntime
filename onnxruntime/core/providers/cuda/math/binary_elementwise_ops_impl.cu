#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_runtime.h>
#include "core/providers/cuda/math/binary_elementwise_ops_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cu_inc/binary_elementwise_impl.cuh"
#include "core/providers/cuda/math/binary_elementwise_ops_impl_functors.cuh"

namespace onnxruntime {
namespace cuda {

#define BINARY_ELEMENTWISE_IMPL(name)                      \
  BINARY_ELEMENTWISE_IMPL_DECLARATION(name) {              \
    BinaryElementWiseImpl(output_rank_or_simple_broadcast, \
                          lhs_padded_strides,              \
                          lhs_data,                        \
                          rhs_padded_strides,              \
                          rhs_data,                        \
                          fdm_output_strides,              \
                          fdm_H,                           \
                          fdm_C,                           \
                          output_data,                     \
                          OP_##name<T, T, T>(),            \
                          count);                          \
  }

#define BINARY_ELEMENTWISE_IMPL_T1(name)                   \
  BINARY_ELEMENTWISE_IMPL_DECLARATION_T1(name) {           \
    BinaryElementWiseImpl(output_rank_or_simple_broadcast, \
                          lhs_padded_strides,              \
                          lhs_data,                        \
                          rhs_padded_strides,              \
                          rhs_data,                        \
                          fdm_output_strides,              \
                          fdm_H,                           \
                          fdm_C,                           \
                          output_data,                     \
                          OP_##name<T, T, T1>(),           \
                          count);                          \
  }

#define BINARY_ELEMENTWISE_IMPL_T2(name)                   \
  BINARY_ELEMENTWISE_IMPL_DECLARATION_T2(name) {           \
    BinaryElementWiseImpl(output_rank_or_simple_broadcast, \
                          lhs_padded_strides,              \
                          lhs_data,                        \
                          rhs_padded_strides,              \
                          rhs_data,                        \
                          fdm_output_strides,              \
                          fdm_H,                           \
                          fdm_C,                           \
                          output_data,                     \
                          OP_##name<T, T1, T2>(),          \
                          count);                          \
  }

#define SPECIALIZED_BINARY_ELEMENTWISE_IMPL(x, T)                                         \
  template void Impl_##x<T>(int32_t output_rank,                                          \
                            const TArray<int64_t>* lhs_padded_strides, const T* lhs_data, \
                            const TArray<int64_t>* rhs_padded_strides, const T* rhs_data, \
                            const TArray<fast_divmod>* fdm_output_strides, const fast_divmod& fdm_H, const fast_divmod& fdm_C, T* output_data, size_t count);

#define SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T1(x, T, T1)                                         \
  template void ImplT1_##x<T, T1>(int32_t output_rank,                                           \
                                  const TArray<int64_t>* lhs_padded_strides, const T* lhs_data,  \
                                  const TArray<int64_t>* rhs_padded_strides, const T1* rhs_data, \
                                  const TArray<fast_divmod>* fdm_output_strides, const fast_divmod& fdm_H, const fast_divmod& fdm_C, T* output_data, size_t count);

#define SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T2(x, T, T1, T2)                                         \
  template void ImplT2_##x<T, T1, T2>(int32_t output_rank,                                           \
                                      const TArray<int64_t>* lhs_padded_strides, const T1* lhs_data, \
                                      const TArray<int64_t>* rhs_padded_strides, const T2* rhs_data, \
                                      const TArray<fast_divmod>* fdm_output_strides, const fast_divmod& fdm_H, const fast_divmod& fdm_C, T* output_data, size_t count);

#if CUDA_VERSION >= 11000 && (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
#define SPECIALIZED_BINARY_ELEMENTWISE_IMPL_BF16(x) SPECIALIZED_BINARY_ELEMENTWISE_IMPL(x, hip_bfloat16)
#define SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T2_BF16(name) SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T2(name, bool, hip_bfloat16, hip_bfloat16)
#else
#define SPECIALIZED_BINARY_ELEMENTWISE_IMPL_BF16(x)
#define SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T2_BF16(name)
#endif

#define SPECIALIZED_BINARY_ELEMENTWISE_IMPL_UZILHFD(x) \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL(x, uint32_t)     \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL(x, uint64_t)     \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL(x, int32_t)      \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL(x, int64_t)      \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL(x, half)         \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL_BF16(x)          \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL(x, float)        \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL(x, double)

#define SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T1_ILHFD(x, T) \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T1(x, T, int32_t)    \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T1(x, T, int64_t)    \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T1(x, T, half)       \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T1(x, T, float)      \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T1(x, T, double)

#define SPECIALIZED_BINARY_ELEMENTWISE_IMPL_OIL(x) \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL(x, bool)     \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL(x, int32_t)  \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL(x, int64_t)

#define SPECIALIZED_BINARY_ELEMENTWISE_IMPL_HFD(x) \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL(x, half)     \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL_BF16(x)      \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL(x, float)    \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL(x, double)

// create declarations for impl
#define BINARY_OP_NAME_EXPR(name, expr) \
  BINARY_ELEMENTWISE_IMPL(name)

BINARY_OPS()
#undef BINARY_OP_NAME_EXPR

// create specialized impl
// the postfix of means the types supported by the op:
// B: uint8_t
// W: uint16_t
// U: uint32_t
// Z: uint64_t
// C: int8_t
// S: int16_t
// I: int32_t
// L: int64_t
// H: float16
// F: float
// D: double
// O: bool

SPECIALIZED_BINARY_ELEMENTWISE_IMPL_UZILHFD(Add)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL(Add, bool)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL_UZILHFD(Sub)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL_UZILHFD(Mul)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL_UZILHFD(Div)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL_HFD(Pow_7)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL(And, bool)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL(Or, bool)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL(Xor, bool)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL_HFD(PRelu)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL_UZILHFD(Max)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL_UZILHFD(Min)

// create declarations for impl for Pow
BINARY_ELEMENTWISE_IMPL_T1(Pow)

SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T1_ILHFD(Pow, int32_t)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T1_ILHFD(Pow, int64_t)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T1_ILHFD(Pow, float)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T1_ILHFD(Pow, double)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T1_ILHFD(Pow, half)

// create declarations for impl2
#define BINARY_OP_NAME_EXPR2(name, expr) \
  BINARY_ELEMENTWISE_IMPL_T2(name)

BINARY_OPS2()
#undef BINARY_OP_NAME_EXPR2

#define SPECIALIZED_BINARY_ELEMENTWISE_IMPL_UZILHFD2(name)               \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T2(name, bool, uint32_t, uint32_t) \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T2(name, bool, uint64_t, uint64_t) \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T2(name, bool, int32_t, int32_t)   \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T2(name, bool, int64_t, int64_t)   \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T2(name, bool, half, half)         \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T2_BF16(name)                      \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T2(name, bool, float, float)       \
  SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T2(name, bool, double, double)

SPECIALIZED_BINARY_ELEMENTWISE_IMPL_UZILHFD2(Greater)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL_UZILHFD2(Equal)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL_T2(Equal, bool, bool, bool)
SPECIALIZED_BINARY_ELEMENTWISE_IMPL_UZILHFD2(Less)

}  // namespace cuda
}  // namespace onnxruntime
