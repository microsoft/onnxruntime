#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "rnn_impl.h"

namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void _ReverseBySequenceKernel(const int32_t max_seq_length,
                                         const int32_t* seq_lengths,
                                         const int32_t block_size,
                                         const fast_divmod div_batch_block,
                                         const fast_divmod div_input_or_hidden_size,
                                         const T* data,
                                         T* reversed_data,
                                         const CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  int seq_id, offset;
  div_batch_block.divmod(id, seq_id, offset);
  int batch, batch_offset;
  div_input_or_hidden_size.divmod(offset, batch, batch_offset);
  int seq_id_org = seq_lengths[batch] - seq_id - 1;
  if (seq_id_org >= 0) {
    int org_id = seq_id_org * block_size + offset;
    reversed_data[id] = data[org_id];
  } else {
    reversed_data[id] = T{};
  }
}

template <typename T>
void ReverseBySequence(hipStream_t stream,
                       const int32_t max_seq_length,
                       const int32_t *seq_lengths,
                       const int32_t batch_size,
                       const int32_t input_or_hidden_size,
                       const T* data,
                       T* reversed_data,
                       const size_t N) {
  // kerneral
  int32_t block_size = batch_size * input_or_hidden_size;
  fast_divmod div_batch_block(block_size);
  fast_divmod div_input_or_hidden_size(input_or_hidden_size);
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  _ReverseBySequenceKernel<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      max_seq_length, seq_lengths, block_size, div_batch_block, div_input_or_hidden_size, data, reversed_data, (CUDA_LONG)N);
}

template <typename T>
__global__ void _BidirectionalDataKernel(const int32_t seq_length,
                                         const int32_t batch_size,
                                         const int32_t hidden_size,
                                         const int32_t seq_block_size,
                                         const fast_divmod div_seq_block,
                                         const fast_divmod div_output_block,
                                         const T* data,
                                         T* reordered_data,
                                         const CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  int seq_id, seq_offset, output_id, offset;
  div_seq_block.divmod(id, seq_id, seq_offset);
  div_output_block.divmod(seq_offset, output_id, offset);
  int org_output_id = 0;
  if (output_id < batch_size) {
    org_output_id = 2 * output_id;
  } else {
    org_output_id = (output_id - batch_size) * 2 + 1;
  }
  int org_id = seq_id * seq_block_size + org_output_id * hidden_size + offset;
  reordered_data[id] = data[org_id];
}

template <typename T>
void ReorderBidirectionalDataInSequence(hipStream_t stream,
                                        const int32_t seq_length,
                                        const int32_t batch_size,
                                        const int32_t hidden_size,
                                        const T* data,
                                        T* reordered_data,
                                        const size_t N) {
  // The cudnn Y output is organize like [Y1, YB1] [Y2, YB2] ... 
  // need to reorganize it to [Y1, Y2, ...] [YB1, YB2, ...]
  int32_t seq_block_size = 2 * batch_size * hidden_size;
  fast_divmod div_seq_block(seq_block_size);
  fast_divmod div_output_block(hidden_size);
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));

  _BidirectionalDataKernel<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      seq_length, batch_size, hidden_size, seq_block_size,
      div_seq_block, div_output_block,
      data, reordered_data, (CUDA_LONG)N);
}

template <typename T>
__global__ void _MaskZeroSequences(const int32_t hidden_size,
                                   T* y_output_data,
                                   T* y_h_output_data,
                                   T* y_c_output_data,
                                   const int32_t* zeor_seq_index_cache,
                                   const CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);

  int32_t zero_seq_offset = zeor_seq_index_cache[id] * hidden_size;

  if (y_output_data != nullptr) {
    for (int i = 0; i < hidden_size; ++i) {
      y_output_data[zero_seq_offset + i] = 0;
    }
  }

  if (y_h_output_data != nullptr) {
    for (int i = 0; i < hidden_size; ++i) {
      y_h_output_data[zero_seq_offset + i] = 0;
    }
  }

  if (y_c_output_data != nullptr) {
    for (int i = 0; i < hidden_size; ++i) {
      y_c_output_data[zero_seq_offset + i] = 0;
    }
  }
}

template <typename T> 
void MaskZeroSequences(hipStream_t stream,
                       const int32_t hidden_size,
                       T* y_output_data,
                       T* y_h_output_data,
                       T* y_c_output_data,
                       const int32_t* zeor_seq_index_cache,
                       const size_t N) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  _MaskZeroSequences<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      hidden_size, y_output_data, y_h_output_data, y_c_output_data, zeor_seq_index_cache, (CUDA_LONG)N);
}

#define SPECIALIZED_RNN_IMPL(T)                                                 \
  template void ReverseBySequence<T>(hipStream_t stream,                       \
                                     const int32_t max_seq_length,              \
                                     const int32_t* seq_lengths,                \
                                     const int32_t batch_size,                  \
                                     const int32_t hidden_size,                 \
                                     const T* data,                             \
                                     T* reversed_data,                          \
                                     const size_t N);                           \
  template void ReorderBidirectionalDataInSequence<T>(hipStream_t stream,\
                                                      const int32_t seq_length, \
                                                      const int32_t batch_size, \
                                                      const int32_t hidden_size,\
                                                      const T* data,            \
                                                      T* reordered_data,        \
                                                     const size_t N);           \
template void MaskZeroSequences<T>(hipStream_t stream,                         \
                                   const int32_t hidden_size,                   \
                                   T* y_output_data,                            \
                                   T* y_h_output_data,                          \
                                   T* y_c_output_data,                          \
                                   const int32_t* zeor_seq_index_cache,         \
                                   const size_t N);

SPECIALIZED_RNN_IMPL(half)
SPECIALIZED_RNN_IMPL(float)
SPECIALIZED_RNN_IMPL(double)

}  // namespace cuda
}  // namespace onnxruntime
