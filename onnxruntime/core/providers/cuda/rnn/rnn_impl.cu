#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "rnn_impl.h"

namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void _ReverseBySequenceKernel(const int32_t seq_length,
                                         const int32_t block_size,
                                         const fast_divmod div_batch_block,
                                         const T* data,
                                         T* reversed_data,
                                         const CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  int seq_id, offset;
  div_batch_block.divmod(id, seq_id, offset);
  int org_id = (seq_length - seq_id - 1) * block_size + offset;
  reversed_data[id] = data[org_id];
}

template <typename T>
void ReverseBySequence(hipStream_t stream,
                       const int32_t seq_length,
                       const int32_t batch_size,
                       const int32_t input_or_hidden_size,
                       const T* data,
                       T* reversed_data,
                       const size_t N) {
  // kerneral
  int32_t block_size = batch_size * input_or_hidden_size;
  fast_divmod div_batch_block(block_size);
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  _ReverseBySequenceKernel<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      seq_length, block_size, div_batch_block, data, reversed_data, (CUDA_LONG)N);
}

template <typename T>
__global__ void _BidirectionalDataKernel(const int32_t seq_length,
                                         const int32_t batch_size,
                                         const int32_t hidden_size,
                                         const int32_t seq_block_size,
                                         const fast_divmod div_seq_block,
                                         const fast_divmod div_output_block,
                                         const T* data,
                                         T* reordered_data,
                                         const CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  int seq_id, seq_offset, output_id, offset;
  div_seq_block.divmod(id, seq_id, seq_offset);
  div_output_block.divmod(seq_offset, output_id, offset);
  int org_output_id = 0;
  if (output_id < batch_size) {
    org_output_id = 2 * output_id;
  } else {
    org_output_id = (output_id - batch_size) * 2 + 1;
  }
  int org_id = seq_id * seq_block_size + org_output_id * hidden_size + offset;
  reordered_data[id] = data[org_id];
}

template <typename T>
void ReorderBidirectionalDataInSequence(hipStream_t stream,
                                        const int32_t seq_length,
                                        const int32_t batch_size,
                                        const int32_t hidden_size,
                                        const T* data,
                                        T* reordered_data,
                                        const size_t N) {
  // The cudnn Y output is organize like [Y1, YB1] [Y2, YB2] ... 
  // need to reorganize it to [Y1, Y2, ...] [YB1, YB2, ...]
  int32_t seq_block_size = 2 * batch_size * hidden_size;
  fast_divmod div_seq_block(seq_block_size);
  fast_divmod div_output_block(hidden_size);
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));

  _BidirectionalDataKernel<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      seq_length, batch_size, hidden_size, seq_block_size,
      div_seq_block, div_output_block,
      data, reordered_data, (CUDA_LONG)N);
}

template <typename T>
__global__ void _RnnMaskKernel(const int32_t seq_length,
                               const int32_t batch_size,
                               const int32_t hidden_size,
                               const int32_t* sequence_lens,
                               const fast_divmod div_seq_block,
                               const fast_divmod div_dir_block,
                               const fast_divmod div_batch_block,
                               T* y_output_data,
                               T* y_h_output_data,
                               const CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);

  int seq_id, direction_id, batch_id, offset;
  div_seq_block.divmod(id, seq_id, offset);
  div_dir_block.divmod(offset, direction_id, offset);
  div_batch_block.divmod(offset, batch_id, offset);
  int32_t batch_seq_length = sequence_lens[batch_id];

  if (batch_id >= batch_size || batch_seq_length == seq_length) {
    return;
  }

  if (seq_id >= batch_seq_length) {
    y_output_data[id] = 0;
    return;
  }

  if ((y_h_output_data != nullptr) && 
      ((direction_id == 0 && (seq_id + 1) == batch_seq_length) || (direction_id == 1 && seq_id == 0))) {
    int hy_idx = direction_id * batch_size * hidden_size + batch_id * hidden_size + offset;
    y_h_output_data[hy_idx] = y_output_data[id];
  }
}

template <typename T>
void RnnMaskImpl(hipStream_t stream,
                 const int32_t num_directions,
                 const int32_t seq_length,
                 const int32_t batch_size,
                 const int32_t hidden_size,
                 const int32_t* sequence_lens,
                 T* y_output_data,
                 T* y_h_output_data,
                 const size_t N) {
  fast_divmod div_seq_block(batch_size * hidden_size * num_directions);
  fast_divmod div_dir_block(batch_size * hidden_size);
  fast_divmod div_batch_block(hidden_size);
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  _RnnMaskKernel<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      seq_length, batch_size, hidden_size, sequence_lens, div_seq_block,
      div_dir_block, div_batch_block, y_output_data, y_h_output_data, (CUDA_LONG)N);
}

template <typename T>
__global__ void _MaskZeroSequences(const int32_t hidden_size,
                                   T* y_output_data,
                                   T* y_h_output_data,
                                   T* y_c_output_data,
                                   const int32_t* zeor_seq_index_cache,
                                   const CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);

  int32_t zero_seq_offset = zeor_seq_index_cache[id] * hidden_size;

  if (y_output_data != nullptr) {
    for (int i = 0; i < hidden_size; ++i) {
      y_output_data[zero_seq_offset + i] = 0;
    }
  }

  if (y_h_output_data != nullptr) {
    for (int i = 0; i < hidden_size; ++i) {
      y_h_output_data[zero_seq_offset + i] = 0;
    }
  }

  if (y_c_output_data != nullptr) {
    for (int i = 0; i < hidden_size; ++i) {
      y_c_output_data[zero_seq_offset + i] = 0;
    }
  }
}

template <typename T> 
void MaskZeroSequences(hipStream_t stream,
                       const int32_t hidden_size,
                       T* y_output_data,
                       T* y_h_output_data,
                       T* y_c_output_data,
                       const int32_t* zeor_seq_index_cache,
                       const size_t N) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  _MaskZeroSequences<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      hidden_size, y_output_data, y_h_output_data, y_c_output_data, zeor_seq_index_cache, (CUDA_LONG)N);
}

#define SPECIALIZED_RNN_IMPL(T)                                                 \
  template void RnnMaskImpl<T>(hipStream_t stream,                       \
                               const int32_t num_directions,                    \
                               const int32_t seq_length,                        \
                               const int32_t batch_size,                        \
                               const int32_t hidden_size,                       \
                               const int32_t* sequence_lens,                    \
                               T* y_output_data,                                \
                               T* y_h_output_data,                              \
                               const size_t N);                                 \
  template void ReverseBySequence<T>(hipStream_t stream,                 \
                                     const int32_t seq_length,                  \
                                     const int32_t batch_size,                  \
                                     const int32_t hidden_size,                 \
                                     const T* data,                             \
                                     T* reversed_data,                          \
                                     const size_t N);                           \
  template void ReorderBidirectionalDataInSequence<T>(hipStream_t stream,\
                                                      const int32_t seq_length, \
                                                      const int32_t batch_size, \
                                                      const int32_t hidden_size,\
                                                      const T* data,            \
                                                      T* reordered_data,        \
                                                     const size_t N);           \
template void MaskZeroSequences<T>(hipStream_t stream,                   \
                                   const int32_t hidden_size,                   \
                                   T* y_output_data,                            \
                                   T* y_h_output_data,                          \
                                   T* y_c_output_data,                          \
                                   const int32_t* zeor_seq_index_cache,         \
                                   const size_t N);

SPECIALIZED_RNN_IMPL(half)
SPECIALIZED_RNN_IMPL(float)
SPECIALIZED_RNN_IMPL(double)

}  // namespace cuda
}  // namespace onnxruntime
