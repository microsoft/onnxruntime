#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <math.h>
#include <sstream>

// Ignore CUTLASS warnings about type punning
#ifdef __GNUC__
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wstrict-aliasing"
#endif

#include "cutlass/array.h"
#include "cutlass/numeric_conversion.h"

#ifdef __GNUC__
#pragma GCC diagnostic pop
#endif

#include "moe_kernel.h"

#if CUDA_VERSION >= 11000
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <cub/util_type.cuh>
#else
#include "hipcub/hipcub.hpp"
#include "cub/device/device_radix_sort.cuh"
#include "cub/util_type.cuh"
#endif

namespace ort_fastertransformer {
static constexpr int WARP_SIZE = 32;

// ====================== Softmax things ===============================
// We have our own implementation of softmax here so we can support transposing the output
// in the softmax kernel when we extend this module to support expert-choice routing.
template <typename T, int TPB>
__launch_bounds__(TPB) __global__
    void moe_softmax(const T* input, const bool* finished, T* output, const int num_cols) {
  using BlockReduce = hipcub::BlockReduce<float, TPB>;
  __shared__ typename BlockReduce::TempStorage tmpStorage;

  __shared__ float normalizing_factor;
  __shared__ float float_max;

  const int thread_row_offset = blockIdx.x * num_cols;

  hipcub::Sum sum;
  float threadData(-FLT_MAX);

  // Don't touch finished rows.
  if ((finished != nullptr) && finished[blockIdx.x]) {
    return;
  }

  for (int ii = threadIdx.x; ii < num_cols; ii += TPB) {
    const int idx = thread_row_offset + ii;
    threadData = max(static_cast<float>(input[idx]), threadData);
  }

  const float maxElem = BlockReduce(tmpStorage).Reduce(threadData, hipcub::Max());
  if (threadIdx.x == 0) {
    float_max = maxElem;
  }
  __syncthreads();

  threadData = 0;

  for (int ii = threadIdx.x; ii < num_cols; ii += TPB) {
    const int idx = thread_row_offset + ii;
    threadData += exp((static_cast<float>(input[idx]) - float_max));
  }

  const auto Z = BlockReduce(tmpStorage).Reduce(threadData, sum);

  if (threadIdx.x == 0) {
    normalizing_factor = 1.f / Z;
  }
  __syncthreads();

  for (int ii = threadIdx.x; ii < num_cols; ii += TPB) {
    const int idx = thread_row_offset + ii;
    const float val = exp((static_cast<float>(input[idx]) - float_max)) * normalizing_factor;
    output[idx] = T(val);
  }
}

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 530
template <typename T, int TPB>
__launch_bounds__(TPB) __global__ void moe_top_k(const T*, const bool*, T*, int*, int*, int, int, bool) {
  // Does not support pre-Kepler architectures
  ;
}
#else
template <typename T, int TPB>
__launch_bounds__(TPB) __global__
    void moe_top_k(const T* inputs_after_softmax, const bool* finished, T* output, int* indices, int* source_rows,
                   int num_experts, int k, bool normalize_routing_weights) {
  using cub_kvp = hipcub::KeyValuePair<int, T>;
  using BlockReduce = hipcub::BlockReduce<cub_kvp, TPB>;
  __shared__ typename BlockReduce::TempStorage tmpStorage;

  cub_kvp thread_kvp;
  hipcub::ArgMax arg_max;

  int num_rows = gridDim.x;
  const int block_row = blockIdx.x;

  const bool should_process_row = finished ? !finished[block_row] : true;
  const int thread_read_offset = blockIdx.x * num_experts;
  float output_row_sum = 0.f;
  for (int k_idx = 0; k_idx < k; ++k_idx) {
    thread_kvp.key = 0;
    thread_kvp.value = T(-1.f);  // This is OK because inputs are probabilities

    cub_kvp inp_kvp;
    for (int expert = threadIdx.x; expert < num_experts; expert += TPB) {
      const int idx = thread_read_offset + expert;
      inp_kvp.key = expert;
      inp_kvp.value = inputs_after_softmax[idx];

      for (int prior_k = 0; prior_k < k_idx; ++prior_k) {
        const int prior_winning_expert = indices[k * block_row + prior_k];

        if (prior_winning_expert == expert) {
          inp_kvp = thread_kvp;
        }
      }

      thread_kvp = arg_max(inp_kvp, thread_kvp);
    }

    const cub_kvp result_kvp = BlockReduce(tmpStorage).Reduce(thread_kvp, arg_max);
    if (threadIdx.x == 0) {
      const int idx = k * block_row + k_idx;
      output[idx] = result_kvp.value;
      indices[idx] = should_process_row ? result_kvp.key : num_experts;
      source_rows[idx] = k_idx * num_rows + block_row;

      if (normalize_routing_weights && k_idx == k - 1) {
#pragma unroll
        for (int ki = 0; ki < k; ++ki) {
          output[idx - ki] = T(static_cast<float>(output[idx - ki]) / output_row_sum);
        }
      }
    }
    __syncthreads();
  }
}
#endif

// ====================== TopK softmax things ===============================

/*
  A Top-K gating softmax written to exploit when the number of experts in the MoE layers
  are a small power of 2. This allows us to cleanly share the rows among the threads in
  a single warp and eliminate communication between warps (so no need to use shared mem).

  It fuses the softmax, max and argmax into a single kernel.

  Limitations:
  1) This implementation is intended for when the number of experts is a small power of 2.
  2) This implementation assumes k is small, but will work for any k.
*/

template <typename T, int VPT, int NUM_EXPERTS, int WARPS_PER_CTA, int BYTES_PER_LDG>
__launch_bounds__(WARPS_PER_CTA* WARP_SIZE) __global__
    void topk_gating_softmax(const T* input, const bool* finished, T* output, int num_rows, int* indices,
                             int* source_rows, int k, bool normalize_routing_weights) {
  // We begin by enforcing compile time assertions and setting up compile time constants.
  static_assert(VPT == (VPT & -VPT), "VPT must be power of 2");
  static_assert(NUM_EXPERTS == (NUM_EXPERTS & -NUM_EXPERTS), "NUM_EXPERTS must be power of 2");
  static_assert(BYTES_PER_LDG == (BYTES_PER_LDG & -BYTES_PER_LDG), "BYTES_PER_LDG must be power of 2");
  static_assert(BYTES_PER_LDG <= 16, "BYTES_PER_LDG must be leq 16");

  // Number of bytes each thread pulls in per load
  static constexpr int ELTS_PER_LDG = BYTES_PER_LDG / sizeof(T);
  static constexpr int ELTS_PER_ROW = NUM_EXPERTS;
  static constexpr int THREADS_PER_ROW = ELTS_PER_ROW / VPT;
  static constexpr int LDG_PER_THREAD = VPT / ELTS_PER_LDG;

  // Restrictions based on previous section.
  static_assert(VPT % ELTS_PER_LDG == 0, "The elements per thread must be a multiple of the elements per ldg");
  static_assert(WARP_SIZE % THREADS_PER_ROW == 0, "The threads per row must cleanly divide the threads per warp");
  static_assert(THREADS_PER_ROW == (THREADS_PER_ROW & -THREADS_PER_ROW), "THREADS_PER_ROW must be power of 2");
  static_assert(THREADS_PER_ROW <= WARP_SIZE, "THREADS_PER_ROW can be at most warp size");

  // We have NUM_EXPERTS elements per row. We specialize for small #experts
  static constexpr int ELTS_PER_WARP = WARP_SIZE * VPT;
  static constexpr int ROWS_PER_WARP = ELTS_PER_WARP / ELTS_PER_ROW;
  static constexpr int ROWS_PER_CTA = WARPS_PER_CTA * ROWS_PER_WARP;

  // Restrictions for previous section.
  static_assert(ELTS_PER_WARP % ELTS_PER_ROW == 0, "The elts per row must cleanly divide the total elt per warp");

  // ===================== From this point, we finally start computing run-time variables. ========================

  // Compute CTA and warp rows. We pack multiple rows into a single warp, and a block contains WARPS_PER_CTA warps.
  // This, each block processes a chunk of rows. We start by computing the start row for each block.
  const int cta_base_row = blockIdx.x * ROWS_PER_CTA;

  // Now, using the base row per thread block, we compute the base row per warp.
  const int warp_base_row = cta_base_row + threadIdx.y * ROWS_PER_WARP;

  // The threads in a warp are split into sub-groups that will work on a row.
  // We compute row offset for each thread sub-group
  const int thread_row_in_warp = threadIdx.x / THREADS_PER_ROW;
  const int thread_row = warp_base_row + thread_row_in_warp;

  // Threads with indices out of bounds should early exit here.
  if (thread_row >= num_rows) return;
  const bool should_process_row = finished ? !finished[thread_row] : true;

  // We finally start setting up the read pointers for each thread. First, each thread jumps to the start of the
  // row it will read.
  const T* thread_row_ptr = input + thread_row * ELTS_PER_ROW;

  // Now, we compute the group each thread belong to in order to determine the first column to start loads.
  const int thread_group_idx = threadIdx.x % THREADS_PER_ROW;
  const int first_elt_read_by_thread = thread_group_idx * ELTS_PER_LDG;
  const T* thread_read_ptr = thread_row_ptr + first_elt_read_by_thread;

  // Determine the pointer type to use to read in the data depending on the BYTES_PER_LDG template param. In theory,
  // this can support all powers of 2 up to 16.
  using AccessType = cutlass::AlignedArray<T, ELTS_PER_LDG>;

  // Finally, we pull in the data from global mem
  cutlass::Array<T, VPT> row_chunk_input;
  AccessType* row_chunk_vec_ptr = reinterpret_cast<AccessType*>(&row_chunk_input);
  const AccessType* vec_thread_read_ptr = reinterpret_cast<const AccessType*>(thread_read_ptr);
#pragma unroll
  for (int ii = 0; ii < LDG_PER_THREAD; ++ii) {
    row_chunk_vec_ptr[ii] = vec_thread_read_ptr[ii * THREADS_PER_ROW];
  }

  using ComputeType = float;
  using Converter = cutlass::NumericArrayConverter<ComputeType, T, VPT>;
  Converter compute_type_converter;
  cutlass::Array<ComputeType, VPT> row_chunk = compute_type_converter(row_chunk_input);

  // First, we perform a max reduce within the thread. We can do the max in fp16 safely (I think) and just
  // convert to float afterwards for the exp + sum reduction.
  ComputeType thread_max = row_chunk[0];
#pragma unroll
  for (int ii = 1; ii < VPT; ++ii) {
    thread_max = max(thread_max, row_chunk[ii]);
  }

// Now, we find the max within the thread group and distribute among the threads. We use a butterfly reduce.
#pragma unroll
  for (int mask = THREADS_PER_ROW / 2; mask > 0; mask /= 2) {
    thread_max = max(thread_max, __shfl_xor_sync(0xFFFFFFFF, thread_max, mask, THREADS_PER_ROW));
  }

  // From this point, thread max in all the threads have the max within the row.
  // Now, we subtract the max from each element in the thread and take the exp. We also compute the thread local sum.
  float row_sum = 0;
#pragma unroll
  for (int ii = 0; ii < VPT; ++ii) {
    row_chunk[ii] = expf(row_chunk[ii] - thread_max);
    row_sum += row_chunk[ii];
  }

// Now, we perform the sum reduce within each thread group. Similar to the max reduce, we use a bufferfly pattern.
#pragma unroll
  for (int mask = THREADS_PER_ROW / 2; mask > 0; mask /= 2) {
    row_sum += __shfl_xor_sync(0xFFFFFFFF, row_sum, mask, THREADS_PER_ROW);
  }

  // From this point, all threads have the max and the sum for their rows in the thread_max and thread_sum variables
  // respectively. Finally, we can scale the rows for the softmax. Technically, for top-k gating we don't need to
  // compute the entire softmax row. We can likely look at the maxes and only compute for the top-k values in the row.
  // However, this kernel will likely not be a bottle neck and it seems better to closer match torch and find the
  // argmax after computing the softmax.
  const float reciprocal_row_sum = 1.f / row_sum;

#pragma unroll
  for (int ii = 0; ii < VPT; ++ii) {
    row_chunk[ii] = row_chunk[ii] * reciprocal_row_sum;
  }

  // Now, softmax_res contains the softmax of the row chunk. Now, I want to find the topk elements in each row, along
  // with the max index.​
  int start_col = first_elt_read_by_thread;
  static constexpr int COLS_PER_GROUP_LDG = ELTS_PER_LDG * THREADS_PER_ROW;

  float output_row_sum = 0.f;
  for (int k_idx = 0; k_idx < k; ++k_idx) {
    // First, each thread does the local argmax
    float max_val = row_chunk[0];
    int expert = start_col;
#pragma unroll
    for (int ldg = 0, col = start_col; ldg < LDG_PER_THREAD; ++ldg, col += COLS_PER_GROUP_LDG) {
#pragma unroll
      for (int ii = 0; ii < ELTS_PER_LDG; ++ii) {
        float val = row_chunk[ldg * ELTS_PER_LDG + ii];

        // No check on the experts here since columns with the smallest index are processed first and only
        // updated if > (not >=)
        if (val > max_val) {
          max_val = val;
          expert = col + ii;
        }
      }
    }

// Now, we perform the argmax reduce. We use the butterfly pattern so threads reach consensus about the max.
// This will be useful for K > 1 so that the threads can agree on "who" had the max value. That thread can
// then blank out their max with -inf and the warp can run more iterations...
#pragma unroll
    for (int mask = THREADS_PER_ROW / 2; mask > 0; mask /= 2) {
      float other_max = __shfl_xor_sync(0xFFFFFFFF, max_val, mask, THREADS_PER_ROW);
      int other_expert = __shfl_xor_sync(0xFFFFFFFF, expert, mask, THREADS_PER_ROW);

      // We want lower indices to "win" in every thread so we break ties this way
      if (other_max > max_val || (other_max == max_val && other_expert < expert)) {
        max_val = other_max;
        expert = other_expert;
      }
    }

    // Write the max for this k iteration to global memory.
    if (thread_group_idx == 0) {
      // The lead thread from each sub-group will write out the final results to global memory. (This will be a
      // single) thread per row of the input/output matrices.
      const int idx = k * thread_row + k_idx;
      output[idx] = T(max_val);
      output_row_sum = output_row_sum + static_cast<float>(max_val);
      indices[idx] = should_process_row ? expert : NUM_EXPERTS;
      source_rows[idx] = k_idx * num_rows + thread_row;

      if (normalize_routing_weights && k_idx == k - 1) {
#pragma unroll
        for (int ki = 0; ki < k; ++ki) {
          output[idx - ki] = T(static_cast<float>(output[idx - ki]) / output_row_sum);
        }
      }
    }

    // Finally, we clear the value in the thread with the current max if there is another iteration to run.
    if (k_idx + 1 < k) {
      const int ldg_group_for_expert = expert / COLS_PER_GROUP_LDG;
      const int thread_to_clear_in_group = (expert / ELTS_PER_LDG) % THREADS_PER_ROW;

      // Only the thread in the group which produced the max will reset the "winning" value to -inf.
      if (thread_group_idx == thread_to_clear_in_group) {
        const int offset_for_expert = expert % ELTS_PER_LDG;
        // Safe to set to any negative value since row_chunk values must be between 0 and 1.
        row_chunk[ldg_group_for_expert * ELTS_PER_LDG + offset_for_expert] = ComputeType(-10000.f);
      }
    }
  }
}

namespace detail {
// Constructs some constants needed to partition the work across threads at compile time.
template <typename T, int EXPERTS, int BYTES_PER_LDG>
struct TopkConstants {
  static constexpr int ELTS_PER_LDG = BYTES_PER_LDG / sizeof(T);
  static_assert(EXPERTS / (ELTS_PER_LDG * WARP_SIZE) == 0 || EXPERTS % (ELTS_PER_LDG * WARP_SIZE) == 0, "");
  static constexpr int VECs_PER_THREAD = std::max(1, (int)EXPERTS / (ELTS_PER_LDG * WARP_SIZE));
  static constexpr int VPT = VECs_PER_THREAD * ELTS_PER_LDG;
  static constexpr int THREADS_PER_ROW = EXPERTS / VPT;
  static constexpr int ROWS_PER_WARP = WARP_SIZE / THREADS_PER_ROW;
};
}  // namespace detail

template <typename T, int EXPERTS, int WARPS_PER_TB>
void topk_gating_softmax_launcher_helper(const T* input, const bool* finished, T* output, int* indices, int* source_row,
                                         int num_rows, int /*num_experts*/, int k, bool normalize_routing_weights,
                                         hipStream_t stream) {
  static constexpr unsigned long MAX_BYTES_PER_LDG = 16;

  static constexpr int BYTES_PER_LDG = std::min((int)MAX_BYTES_PER_LDG, (int)sizeof(T) * EXPERTS);
  using Constants = detail::TopkConstants<T, EXPERTS, BYTES_PER_LDG>;
  static constexpr int VPT = Constants::VPT;
  static constexpr int ROWS_PER_WARP = Constants::ROWS_PER_WARP;
  const int num_warps = (num_rows + ROWS_PER_WARP - 1) / ROWS_PER_WARP;
  const int num_blocks = (num_warps + WARPS_PER_TB - 1) / WARPS_PER_TB;

  dim3 block_dim(WARP_SIZE, WARPS_PER_TB);
  topk_gating_softmax<T, VPT, EXPERTS, WARPS_PER_TB, BYTES_PER_LDG><<<num_blocks, block_dim, 0, stream>>>(
      input, finished, output, num_rows, indices, source_row, k, normalize_routing_weights);
}

template <typename T>
void topk_gating_softmax_kernelLauncher(const T* input, const bool* finished, T* output, T* softmax_temp_output,
                                        int* indices, int* source_row, int num_rows, int num_experts, int k,
                                        bool normalize_routing_weights, hipStream_t stream) {
  static constexpr int WARPS_PER_TB = 4;

  switch (num_experts) {
    case 2: {
      topk_gating_softmax_launcher_helper<T, 2, WARPS_PER_TB>(input, finished, output, indices, source_row, num_rows,
                                                              num_experts, k, normalize_routing_weights, stream);
      break;
    }
    case 4: {
      topk_gating_softmax_launcher_helper<T, 4, WARPS_PER_TB>(input, finished, output, indices, source_row, num_rows,
                                                              num_experts, k, normalize_routing_weights, stream);
      break;
    }
    case 8: {
      topk_gating_softmax_launcher_helper<T, 8, WARPS_PER_TB>(input, finished, output, indices, source_row, num_rows,
                                                              num_experts, k, normalize_routing_weights, stream);
      break;
    }
    case 16: {
      topk_gating_softmax_launcher_helper<T, 16, WARPS_PER_TB>(input, finished, output, indices, source_row, num_rows,
                                                               num_experts, k, normalize_routing_weights, stream);
      break;
    }
    case 32: {
      topk_gating_softmax_launcher_helper<T, 32, WARPS_PER_TB>(input, finished, output, indices, source_row, num_rows,
                                                               num_experts, k, normalize_routing_weights, stream);
      break;
    }
    case 64: {
      topk_gating_softmax_launcher_helper<T, 64, WARPS_PER_TB>(input, finished, output, indices, source_row, num_rows,
                                                               num_experts, k, normalize_routing_weights, stream);
      break;
    }
    case 128: {
      topk_gating_softmax_launcher_helper<T, 128, WARPS_PER_TB>(input, finished, output, indices, source_row, num_rows,
                                                                num_experts, k, normalize_routing_weights, stream);
      break;
    }
    case 256: {
      topk_gating_softmax_launcher_helper<T, 256, WARPS_PER_TB>(input, finished, output, indices, source_row, num_rows,
                                                                num_experts, k, normalize_routing_weights, stream);
      break;
    }
    default: {
      static constexpr int TPB = 256;
      moe_softmax<T, TPB><<<num_rows, TPB, 0, stream>>>(input, finished, softmax_temp_output, num_experts);
      moe_top_k<T, TPB><<<num_rows, TPB, 0, stream>>>(softmax_temp_output, finished, output, indices, source_row,
                                                      num_experts, k, normalize_routing_weights);
    }
  }
}

// ========================== CUB Sorting things ====================================
CubKeyValueSorter::CubKeyValueSorter() : num_experts_(0), num_bits_(sizeof(int) * 8) {}

CubKeyValueSorter::CubKeyValueSorter(int num_experts)
    : num_experts_(num_experts), num_bits_((int)log2(num_experts) + 1) {}

void CubKeyValueSorter::update_num_experts(int num_experts) {
  num_experts_ = num_experts;
  num_bits_ = (int)log2(num_experts) + 1;
}

size_t CubKeyValueSorter::getWorkspaceSize(const size_t num_key_value_pairs) {
  num_key_value_pairs_ = num_key_value_pairs;
  size_t required_storage = 0;
  int* null_int = nullptr;
  hipcub::DeviceRadixSort::SortPairs(NULL, required_storage, null_int, null_int, null_int, null_int,
                                  (int)num_key_value_pairs, 0, num_bits_);
  return required_storage;
}

void CubKeyValueSorter::run(void* workspace, const size_t workspace_size, const int* keys_in, int* keys_out,
                            const int* values_in, int* values_out, const size_t num_key_value_pairs,
                            hipStream_t stream) {
  size_t expected_ws_size = getWorkspaceSize(num_key_value_pairs);
  size_t actual_ws_size = workspace_size;

  if (expected_ws_size > workspace_size) {
    ORT_THROW("Error. The allocated workspace is too small to run this problem. Expected workspace size of at least ",
              expected_ws_size, " but got problem size ", workspace_size, "\n");
  }
  hipcub::DeviceRadixSort::SortPairs(workspace, actual_ws_size, keys_in, keys_out, values_in, values_out,
                                  (int)num_key_value_pairs, 0, num_bits_, stream);
}

// ============================== Infer GEMM sizes =================================
__device__ inline int find_total_elts_leq_target(const int* sorted_indices, const int arr_length, const int target) {
  int64_t low = 0, high = arr_length - 1, target_location = -1;
  while (low <= high) {
    int64_t mid = (low + high) / 2;

    if (sorted_indices[mid] > target) {
      high = mid - 1;
    } else {
      low = mid + 1;
      target_location = mid;
    }
  }
  return target_location + 1;
}

// Sets up the gemm assuming the inputs, experts and outputs are stored in row major order.
// Assumes we want to perform output = matmul(inputs, experts) + bias
__global__ void compute_total_rows_before_expert_kernel(const int* sorted_experts, const int sorted_experts_len,
                                                        const int64_t num_experts, int64_t* total_rows_before_expert) {
  // First, compute the global tid. We only need 1 thread per expert.
  const int expert = blockIdx.x * blockDim.x + threadIdx.x;
  if (expert >= num_experts) return;

  // This should construct the last index where each expert occurs.
  total_rows_before_expert[expert] = find_total_elts_leq_target(sorted_experts, sorted_experts_len, expert);
}

__global__ void dispatch_activations_kernel(int64_t* total_rows_before_expert, int num_experts, int local_num_experts,
                                            int local_experts_start_index) {
  const int expert = blockIdx.x * blockDim.x + threadIdx.x;
  const int local_experts_end_index = local_experts_start_index + local_num_experts - 1;

  int total_past_rows = 0;
  if (local_experts_start_index > 0) {
    total_past_rows = total_rows_before_expert[local_experts_start_index - 1];
  }

  if (expert < local_experts_start_index || expert > local_experts_end_index) {
    return;
  }

  total_rows_before_expert[expert] -= total_past_rows;
}

template <typename T, typename WeightType, typename Enable>
CutlassMoeFCRunner<T, WeightType, Enable>::CutlassMoeFCRunner(int sm_version, bool has_fc3,
                                                              bool normalize_routing_weights)
    : has_fc3_(has_fc3),
      total_past_rows_(0),
      total_covered_rows_(0),
      normalize_routing_weights_(normalize_routing_weights) {
  moe_gemm_runner_.initialize(sm_version);
}

template <typename T, typename WeightType, typename Enable>
size_t CutlassMoeFCRunner<T, WeightType, Enable>::getWorkspaceSize(size_t num_rows, const size_t hidden_size,
                                                                   const size_t inter_size, size_t num_experts,
                                                                   size_t k) {
  total_covered_rows_ = k * num_rows;

  const size_t buf_size = pad_to_multiple_of_16(k * num_rows * hidden_size);
  const size_t interbuf_size = pad_to_multiple_of_16(k * num_rows * inter_size);
  const size_t padded_experts = pad_to_multiple_of_16(num_experts);
  const size_t num_moe_inputs = pad_to_multiple_of_16(k * num_rows);
  size_t num_softmax_outs = 0;

  const bool is_pow_2 = (num_experts != 0) && ((num_experts & (num_experts - 1)) == 0);
  if (!is_pow_2 || num_experts > 256) {
    num_softmax_outs = pad_to_multiple_of_16(num_rows * num_experts);
  }

  // softmax output, permuted_rows and permuted_experts have moved to outside of moe kernel, allocate them
  // in Encoder or Decoder before invoking FfnLayer forward.
  size_t total_ws_bytes = 3 * num_moe_inputs * sizeof(int);  // source_rows_, permuted_rows_, permuted_experts_
  total_ws_bytes += buf_size * sizeof(T);                    // permuted_data
  total_ws_bytes += padded_experts * sizeof(int64_t);        // Hold total_rows_before_expert_
  total_ws_bytes += num_softmax_outs * sizeof(T);
  const size_t bytes_for_fc1_result = has_fc3_ ? 2 * interbuf_size * sizeof(T) : interbuf_size * sizeof(T);
  const size_t sorter_ws_size_bytes = pad_to_multiple_of_16(sorter_.getWorkspaceSize(num_rows));
  sorter_.update_num_experts(static_cast<int>(num_experts));

  size_t bytes_for_intermediate_and_sorting = bytes_for_fc1_result;
  if (sorter_ws_size_bytes > bytes_for_fc1_result) {
    size_t remaining_bytes = pad_to_multiple_of_16(sorter_ws_size_bytes - bytes_for_fc1_result);
    bytes_for_intermediate_and_sorting += remaining_bytes;
  }

  total_ws_bytes += bytes_for_intermediate_and_sorting;  // intermediate (fc1) output + cub sorting workspace
  return total_ws_bytes;
}

template <typename T, typename WeightType, typename Enable>
void CutlassMoeFCRunner<T, WeightType, Enable>::configure_ws_ptrs(char* ws_ptr, size_t num_rows,
                                                                  const size_t hidden_size, const size_t inter_size,
                                                                  size_t num_experts, size_t k) {
  const size_t buf_size = pad_to_multiple_of_16(k * num_rows * hidden_size);
  const size_t interbuf_size = pad_to_multiple_of_16(k * num_rows * inter_size);
  const size_t padded_experts = pad_to_multiple_of_16(num_experts);
  const size_t num_moe_inputs = pad_to_multiple_of_16(k * num_rows);

  source_rows_ = reinterpret_cast<int*>(ws_ptr);
  permuted_rows_ = source_rows_ + num_moe_inputs;
  permuted_experts_ = permuted_rows_ + num_moe_inputs;
  permuted_data_ = reinterpret_cast<T*>(permuted_experts_ + num_moe_inputs);

  total_rows_before_expert_ = reinterpret_cast<int64_t*>(permuted_data_ + buf_size);

  if (has_fc3_) {
    fc3_result_ = reinterpret_cast<T*>(total_rows_before_expert_ + padded_experts);
    fc1_result_ = reinterpret_cast<T*>(fc3_result_ + interbuf_size);
  } else {
    fc1_result_ = reinterpret_cast<T*>(total_rows_before_expert_ + padded_experts);
  }

  const bool is_pow_2 = (num_experts != 0) && ((num_experts & (num_experts - 1)) == 0);
  if (!is_pow_2 || num_experts > 256) {
    softmax_out_ = reinterpret_cast<T*>(fc1_result_ + interbuf_size);
  } else {
    softmax_out_ = nullptr;
  }
}

namespace {

struct __align__(8) Half4 {
  half2 x;
  half2 y;
};

// TODO(wy): move to common header
template <typename T>
struct T4;
template <>
struct T4<float> {
  using Type = float4;
};
template <>
struct T4<half> {
  using Type = Half4;
};

template <typename T>
struct T2;
template <>
struct T2<float> {
  using Type = float2;
};
template <>
struct T2<half> {
  using Type = half2;
};

inline __device__ float2 operator*(const float2 a, const float2 b) { return make_float2(a.x * b.x, a.y * b.y); }

inline __device__ float4 operator*(const float4 a, const float4 b) {
  return make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w);
}

// TODO(wy): use cuda common header and investigate pipeline build issue.
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 530 && \
    ((__CUDACC_VER_MAJOR__ < 12) || ((__CUDACC_VER_MAJOR__ == 12) && (__CUDACC_VER_MINOR__ < 2)))
inline __device__ half operator*(const half a, const half b) {
  return __float2half(__half2float(a) * __half2float(b));
}

inline __device__ half2 operator*(const half2 a, const half2 b) {
  return make_half2(a.x * b.x, a.y * b.y);
}
#endif

// TODO(wy): use cuda common header and investigate pipeline build issue.
inline __device__ Half4 operator*(const Half4 a, const Half4 b) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 530 && \
    ((__CUDACC_VER_MAJOR__ < 12) || ((__CUDACC_VER_MAJOR__ == 12) && (__CUDACC_VER_MINOR__ < 2)))
  Half4 result;
  result.x = a.x * b.x;
  result.y = a.y * b.y;
  return result;
#else
  return Half4{__hmul2(a.x, b.x), __hmul2(a.y, b.y)};
#endif
}

}  // anonymous namespace

template <typename T>
__global__ void elementWiseMulKernel(T* output, T const* input, size_t inter_size) {
  int const tid = threadIdx.x;
  int const token = blockIdx.x;

  output = output + token * inter_size;
  input = input + token * inter_size;
  for (int i = tid; i < inter_size; i += blockDim.x) {
    T fc1_value = input[i];
    output[i] = fc1_value * output[i];
  }
}

template <typename T>
void elementWiseMul(T* output, T const* input, int inter_size, int num_tokens, hipStream_t stream) {
  int const blocks = num_tokens;

  if (inter_size & 3 == 0) {
    using vec_type = typename T4<T>::Type;
    int const threads = std::min(inter_size / 4, 1024);
    elementWiseMulKernel<vec_type><<<blocks, threads, 0, stream>>>(
        reinterpret_cast<vec_type*>(output), reinterpret_cast<vec_type const*>(input), inter_size / 4);
  } else if (inter_size & 1 == 0) {
    using vec_type = typename T2<T>::Type;
    int const threads = std::min(inter_size / 2, 1024);
    elementWiseMulKernel<vec_type><<<blocks, threads, 0, stream>>>(
        reinterpret_cast<vec_type*>(output), reinterpret_cast<vec_type const*>(input), inter_size / 2);
  } else {
    int const threads = std::min(inter_size, 1024);
    elementWiseMulKernel<T><<<blocks, threads, 0, stream>>>(output, input, inter_size);
  }
}

template <typename T, typename WeightType, typename Enable>
void CutlassMoeFCRunner<T, WeightType, Enable>::run_moe_fc(
    const T* input_activations, const T* gating_output, const WeightType* fc1_expert_weights, const T* fc1_scales,
    const T* fc1_expert_biases, ActivationType fc1_activation_type, const WeightType* fc3_expert_weights,
    const T* fc3_scales, const T* fc3_expert_biases, const WeightType* fc2_expert_weights, const T* fc2_scales,
    int num_rows, const int hidden_size, const int inter_size, int num_experts, int local_num_experts,
    int local_experts_start_index, int k, char* workspace_ptr, T* fc2_result, const bool* finished, int active_rows,
    T* expert_scales, int* expanded_source_row_to_expanded_dest_row, int* expert_for_source_row, hipStream_t stream) {
  static constexpr bool scales_required =
      std::is_same<WeightType, uint8_t>::value || std::is_same<WeightType, cutlass::uint4b_t>::value;

  if (scales_required) {
    if (fc1_scales == nullptr) {
      ORT_THROW("[FT Error][Run MoE FC] Scales expected but scale for first matmul is a null pointer");
    } else if (fc2_scales == nullptr) {
      ORT_THROW("[FT Error][Run MoE FC] Scales expected but scale for second matmul is a null pointer");
    }
  } else {
    if (fc1_scales != nullptr) {
      ORT_THROW("[FT Error][Run MoE FC] Scales are ignored for fp32/fp16/bf16 but received scale for FC1");
    } else if (fc2_scales != nullptr) {
      ORT_THROW("[FT Error][Run MoE FC] Scales are ignored for fp32/fp16/bf16 but received scale for FC2");
    }
  }

  configure_ws_ptrs(workspace_ptr, static_cast<size_t>(num_rows), static_cast<size_t>(hidden_size),
                    static_cast<size_t>(inter_size), static_cast<size_t>(num_experts), static_cast<size_t>(k));
  topk_gating_softmax_kernelLauncher<T>(gating_output, finished, expert_scales, softmax_out_, expert_for_source_row,
                                        source_rows_, num_rows, num_experts, k, normalize_routing_weights_, stream);

  const int sorter_ws_size_bytes = static_cast<int>(pad_to_multiple_of_16(sorter_.getWorkspaceSize(k * num_rows)));
  sorter_.run(reinterpret_cast<void*>(fc1_result_), sorter_ws_size_bytes, expert_for_source_row, permuted_experts_,
              source_rows_, permuted_rows_, k * num_rows, stream);

  initialize_moe_routing_kernelLauncher(input_activations, permuted_data_, permuted_rows_,
                                        expanded_source_row_to_expanded_dest_row, num_rows, active_rows, hidden_size, k,
                                        stream);

  const int expanded_active_expert_rows = k * active_rows;
  compute_total_rows_before_expert(permuted_experts_, expanded_active_expert_rows, num_experts,
                                   total_rows_before_expert_, stream);

  if (local_num_experts < num_experts) {
    dispatch_activations(total_rows_before_expert_, num_experts, local_num_experts, local_experts_start_index, stream);
  }

  moe_gemm_runner_.moe_gemm_bias_act(permuted_data_ + total_past_rows_ * hidden_size, fc1_expert_weights, fc1_scales,
                                     fc1_expert_biases, fc1_result_ + total_past_rows_ * inter_size,
                                     total_rows_before_expert_ + local_experts_start_index, expanded_active_expert_rows,
                                     inter_size, hidden_size, local_num_experts, fc1_activation_type, stream);

  if (has_fc3_) {
    if (scales_required) {
      if (fc3_scales == nullptr) {
        ORT_THROW("[FT Error][Run MoE FC] Scales expected but scale for third matmul is a null pointer");
      }
    } else {
      if (fc3_scales != nullptr) {
        ORT_THROW("[FT Error][Run MoE FC] Scales are ignored for fp32/fp16/bf16 but received scale for FC3");
      }
    }
    if (fc3_expert_weights == nullptr) {
      ORT_THROW("[FT Error][Run MoE FC] FC3 weights are null");
    }
    moe_gemm_runner_.moe_gemm(permuted_data_ + total_past_rows_ * hidden_size, fc3_expert_weights, fc3_scales,
                              fc3_expert_biases, fc3_result_ + total_past_rows_ * inter_size,
                              total_rows_before_expert_ + local_experts_start_index, expanded_active_expert_rows,
                              inter_size, hidden_size, local_num_experts, stream);

    elementWiseMul(fc1_result_ + total_past_rows_ * inter_size, fc3_result_ + total_past_rows_ * inter_size,
                   static_cast<int>(inter_size), static_cast<int>(total_covered_rows_), stream);
  }

  moe_gemm_runner_.moe_gemm(fc1_result_ + total_past_rows_ * inter_size, fc2_expert_weights, fc2_scales, nullptr,
                            fc2_result + total_past_rows_ * hidden_size,
                            total_rows_before_expert_ + local_experts_start_index, expanded_active_expert_rows,
                            hidden_size, inter_size, local_num_experts, stream);
}

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 700
template <typename T, typename WeightType, typename Enable>
void CutlassMoeFCRunner<T, WeightType, Enable>::run_moe_fc(const T*, const T*, const WeightType*, const T*, const T*,
                                                           ActivationType, const WeightType*, const T*, const T*,
                                                           const WeightType*, const T*, int, const int, const int, int,
                                                           int, int, int k, char*, T*, T*, int*, int*, hipStream_t) {
  // MoE gemm only supports Volta+ architectures
  ORT_THROW("[FT Error][Run MoE FC] MoE gemm only supports Volta+ architectures");
}
#else
template <typename T, typename WeightType, typename Enable>
void CutlassMoeFCRunner<T, WeightType, Enable>::run_moe_fc(
    const T* input_activations, const T* gating_output, const WeightType* fc1_expert_weights, const T* fc1_scales,
    const T* fc1_expert_biases, ActivationType fc1_activation_type, const WeightType* fc3_expert_weights,
    const T* fc3_scales, const T* fc3_expert_biases, const WeightType* fc2_expert_weights, const T* fc2_scales,
    int num_rows, const int hidden_size, const int inter_size, int num_experts, int local_num_experts,
    int local_experts_start_index, int k, char* workspace_ptr, T* fc2_result, T* expert_scales,
    int* expanded_source_row_to_expanded_dest_row, int* expert_for_source_row, hipStream_t stream) {
  run_moe_fc(input_activations, gating_output, fc1_expert_weights, fc1_scales, fc1_expert_biases, fc1_activation_type,
             fc3_expert_weights, fc3_scales, fc3_expert_biases, fc2_expert_weights, fc2_scales, num_rows, hidden_size,
             inter_size, num_experts, local_num_experts, local_experts_start_index, k, workspace_ptr, fc2_result,
             nullptr, num_rows, expert_scales, expanded_source_row_to_expanded_dest_row, expert_for_source_row, stream);
}
#endif

template <typename T, typename WeightType, typename Enable>
void CutlassMoeFCRunner<T, WeightType, Enable>::compute_total_rows_before_expert(const int* sorted_indices,
                                                                                 const int total_indices,
                                                                                 int num_experts,
                                                                                 int64_t* total_rows_before_expert,
                                                                                 hipStream_t stream) {
  const int threads = std::min(1024, num_experts);
  const int blocks = (num_experts + threads - 1) / threads;

  compute_total_rows_before_expert_kernel<<<blocks, threads, 0, stream>>>(sorted_indices, total_indices, num_experts,
                                                                          total_rows_before_expert);
}

template <typename T, typename WeightType, typename Enable>
void CutlassMoeFCRunner<T, WeightType, Enable>::dispatch_activations(int64_t* total_rows_before_expert, int num_experts,
                                                                     int local_num_experts,
                                                                     int local_experts_start_index,
                                                                     hipStream_t stream) {
  total_rows_before_expert_host_.resize(num_experts);
  hipMemcpyAsync(total_rows_before_expert_host_.data(), total_rows_before_expert, num_experts * sizeof(int64_t),
                  hipMemcpyDeviceToHost, stream);

  const int threads = std::min(1024, num_experts);
  const int blocks = (num_experts + threads - 1) / threads;

  hipEvent_t& copy_event = cuda_event_.Get();
  hipEventCreateWithFlags(&copy_event, hipEventDisableTiming);
  hipEventRecord(copy_event, stream);

  dispatch_activations_kernel<<<blocks, threads, 0, stream>>>(total_rows_before_expert, num_experts, local_num_experts,
                                                              local_experts_start_index);

  get_total_rows_info(local_experts_start_index, local_num_experts, total_past_rows_, total_covered_rows_);
}

template <typename T, typename WeightType, typename Enable>
void CutlassMoeFCRunner<T, WeightType, Enable>::get_total_rows_info(int64_t experts_start_index,
                                                                    int64_t local_num_experts, int64_t& total_past_rows,
                                                                    int64_t& total_covered_rows) {
  int64_t experts_end_index = experts_start_index + local_num_experts - 1;
  total_past_rows = 0;

  hipEventSynchronize(cuda_event_.Get());

  if (experts_start_index > 0) {
    total_past_rows = total_rows_before_expert_host_[experts_start_index - 1];
  }
  total_covered_rows = total_rows_before_expert_host_[experts_end_index] - total_past_rows;
}

// ========================== Permutation things =======================================

// Duplicated and permutes rows for MoE. In addition, reverse the permutation map to help with finalizing routing.

// "expanded_x_row" simply means that the number of values is num_rows x k. It is "expanded" since we will have to
// duplicate some rows in the input matrix to match the dimensions. Duplicates will always get routed to separate
// experts in the end.

// Note that the expanded_dest_row_to_expanded_source_row map referred to here has indices in the range (0,
// k*rows_in_input - 1). However, it is set up so that index 0, rows_in_input, 2*rows_in_input ... (k-1)*rows_in_input
// all map to row 0 in the original matrix. Thus, to know where to read in the source matrix, we simply take the modulus
// of the expanded index.

template <typename T>
__global__ void initialize_moe_routing_kernel(const T* unpermuted_input, T* permuted_output,
                                              const int* expanded_dest_row_to_expanded_source_row,
                                              int* expanded_source_row_to_expanded_dest_row, int num_rows,
                                              int active_rows, int cols) {
  // Reverse permutation map.
  // I do this so that later, we can use the source -> dest map to do the k-way reduction and unpermuting. I need the
  // reverse map for that reduction to allow each threadblock to do 1 k-way reduce without atomics later in MoE. 1
  // thread block will be responsible for all k summations.
  const int expanded_dest_row = blockIdx.x;
  const int expanded_source_row = expanded_dest_row_to_expanded_source_row[expanded_dest_row];
  if (threadIdx.x == 0) {
    expanded_source_row_to_expanded_dest_row[expanded_source_row] = expanded_dest_row;
  }

  if (blockIdx.x < active_rows) {
    // Duplicate and permute rows
    const int source_row = expanded_source_row % num_rows;

    const T* source_row_ptr = unpermuted_input + source_row * cols;
    T* dest_row_ptr = permuted_output + expanded_dest_row * cols;

    for (int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
      dest_row_ptr[tid] = source_row_ptr[tid];
    }
  }
}

template <typename T>
void initialize_moe_routing_kernelLauncher(const T* unpermuted_input, T* permuted_output,
                                           const int* expanded_dest_row_to_expanded_source_row,
                                           int* expanded_source_row_to_expanded_dest_row, int num_rows, int active_rows,
                                           int cols, int k, hipStream_t stream) {
  const int blocks = num_rows * k;
  const int threads = std::min(cols, 1024);
  initialize_moe_routing_kernel<T>
      <<<blocks, threads, 0, stream>>>(unpermuted_input, permuted_output, expanded_dest_row_to_expanded_source_row,
                                       expanded_source_row_to_expanded_dest_row, num_rows, k * active_rows, cols);
}

// Final kernel to unpermute and scale
// This kernel unpermutes the original data, does the k-way reduction and performs the final skip connection.
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 530
template <typename T, int RESIDUAL_NUM>
__global__ void finalize_moe_routing_kernel(const T*, T*, const T*, const T*, const T*, const T*, const int*,
                                            const int*, int, const int) {
  // Does not support pre-Kepler architectures
  ;
}
#else
template <typename T, int RESIDUAL_NUM>
__global__ void finalize_moe_routing_kernel(const T* expanded_permuted_rows, T* reduced_unpermuted_output,
                                            const T* skip_1, const T* skip_2, const T* bias, const T* scales,
                                            const int* expanded_source_row_to_expanded_dest_row,
                                            const int* expert_for_source_row, int cols, int k) {
  const int original_row = blockIdx.x;
  int num_rows = gridDim.x;
  T* reduced_row_ptr = reduced_unpermuted_output + original_row * cols;

  const T* skip_1_row_ptr = nullptr;
  if (RESIDUAL_NUM == 1) {
    skip_1_row_ptr = skip_1 + original_row * cols;
  }
  const T* skip_2_row_ptr = nullptr;
  if (RESIDUAL_NUM == 2) {
    skip_2_row_ptr = skip_2 + original_row * cols;
  }

  for (int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
    T thread_output;
    if (RESIDUAL_NUM == 0) {
      thread_output = T(0);
    } else if (RESIDUAL_NUM == 1) {
      thread_output = skip_1_row_ptr[tid];
    } else if (RESIDUAL_NUM == 2) {
      thread_output = skip_1_row_ptr[tid] + skip_2_row_ptr[tid];
    }
    for (int k_idx = 0; k_idx < k; ++k_idx) {
      const int expanded_original_row = original_row + k_idx * num_rows;
      const int expanded_permuted_row = expanded_source_row_to_expanded_dest_row[expanded_original_row];

      const int64_t k_offset = original_row * k + k_idx;
      const T row_scale = scales[k_offset];
      const T* expanded_permuted_rows_row_ptr = expanded_permuted_rows + expanded_permuted_row * cols;

      const int expert_idx = expert_for_source_row[k_offset];
      const T* bias_ptr = bias ? bias + expert_idx * cols : nullptr;

      thread_output =
          thread_output + row_scale * (expanded_permuted_rows_row_ptr[tid] + (bias_ptr ? bias_ptr[tid] : T(0)));
    }
    reduced_row_ptr[tid] = thread_output;
  }
}
#endif

template <typename T>
void finalize_moe_routing_kernelLauncher(const T* expanded_permuted_rows, T* reduced_unpermuted_output, const T* bias,
                                         const T* scales, const int* expanded_source_row_to_expanded_dest_row,
                                         const int* expert_for_source_row, int num_rows, int cols, int k,
                                         hipStream_t stream) {
  const int blocks = num_rows;
  const int threads = std::min(cols, 1024);
  finalize_moe_routing_kernel<T, 0><<<blocks, threads, 0, stream>>>(
      expanded_permuted_rows, reduced_unpermuted_output, nullptr, nullptr, bias, scales,
      expanded_source_row_to_expanded_dest_row, expert_for_source_row, cols, k);
}

template <typename T>
void finalize_moe_routing_kernelLauncher(const T* expanded_permuted_rows, T* reduced_unpermuted_output, const T* skip,
                                         const T* bias, const T* scales,
                                         const int* expanded_source_row_to_expanded_dest_row,
                                         const int* expert_for_source_row, int num_rows, int cols, int k,
                                         hipStream_t stream) {
  const int blocks = num_rows;
  const int threads = std::min(cols, 1024);
  finalize_moe_routing_kernel<T, 1>
      <<<blocks, threads, 0, stream>>>(expanded_permuted_rows, reduced_unpermuted_output, skip, nullptr, bias, scales,
                                       expanded_source_row_to_expanded_dest_row, expert_for_source_row, cols, k);
}

template <typename T>
void finalize_moe_routing_kernelLauncher(const T* expanded_permuted_rows, T* reduced_unpermuted_output, const T* skip_1,
                                         const T* skip_2, const T* bias, const T* scales,
                                         const int* expanded_source_row_to_expanded_dest_row,
                                         const int* expert_for_source_row, int num_rows, int cols, int k,
                                         hipStream_t stream) {
  const int blocks = num_rows;
  const int threads = std::min(cols, 1024);
  if (skip_2 == nullptr) {
    finalize_moe_routing_kernel<T, 1><<<blocks, threads, 0, stream>>>(
        expanded_permuted_rows, reduced_unpermuted_output, skip_1, skip_2, bias, scales,
        expanded_source_row_to_expanded_dest_row, expert_for_source_row, cols, k);
  } else {
    finalize_moe_routing_kernel<T, 2><<<blocks, threads, 0, stream>>>(
        expanded_permuted_rows, reduced_unpermuted_output, skip_1, skip_2, bias, scales,
        expanded_source_row_to_expanded_dest_row, expert_for_source_row, cols, k);
  }
}

// ========================= TopK Softmax specializations ===========================
template void topk_gating_softmax_kernelLauncher(const float*, const bool*, float*, float*, int*, int*, int, int, int,
                                                 bool, hipStream_t);
template void topk_gating_softmax_kernelLauncher(const half*, const bool*, half*, half*, int*, int*, int, int, int,
                                                 bool, hipStream_t);

// ==================== Variable batched GEMM specializations ==================================
template class CutlassMoeFCRunner<float, float>;
template class CutlassMoeFCRunner<half, half>;
template class CutlassMoeFCRunner<half, cutlass::uint4b_t>;

// ===================== Specializations for init routing =========================
template void initialize_moe_routing_kernelLauncher(const float*, float*, const int*, int*, int, int, int, int,
                                                    hipStream_t);
template void initialize_moe_routing_kernelLauncher(const half*, half*, const int*, int*, int, int, int, int,
                                                    hipStream_t);

// ==================== Specializations for final routing ===================================
template void finalize_moe_routing_kernelLauncher(const float*, float*, const float*, const float*, const int*,
                                                  const int*, int, int, int, hipStream_t);
template void finalize_moe_routing_kernelLauncher(const half*, half*, const half*, const half*, const int*, const int*,
                                                  int, int, int, hipStream_t);
template void finalize_moe_routing_kernelLauncher(const float*, float*, const float*, const float*, const float*,
                                                  const int*, const int*, int, int, int, hipStream_t);
template void finalize_moe_routing_kernelLauncher(const half*, half*, const half*, const half*, const half*, const int*,
                                                  const int*, int, int, int, hipStream_t);
template void finalize_moe_routing_kernelLauncher(const float*, float*, const float*, const float*, const float*,
                                                  const float*, const int*, const int*, int, int, int, hipStream_t);
template void finalize_moe_routing_kernelLauncher(const half*, half*, const half*, const half*, const half*,
                                                  const half*, const int*, const int*, int, int, int, hipStream_t);

}  // namespace ort_fastertransformer
