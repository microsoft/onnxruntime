#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "contrib_ops/cuda/sparse/sparse_attention_impl.h"
#include "contrib_ops/cuda/sparse/block_mask.h"
#include "contrib_ops/cuda/transformers/dump_cuda_tensor.h"
#include "contrib_ops/cuda/bert/rotary_embedding_impl.h"
#include "contrib_ops/cuda/bert/group_query_attention_impl.h"
#include "contrib_ops/cpu/bert/attention_common.h"
#include "contrib_ops/cuda/bert/attention_impl.h"
#include "contrib_ops/cuda/sparse/sparse_attention_v1/sparse_attention_common.h"
#include "contrib_ops/cuda/sparse/sparse_attention_v1/sparse_attention_v1_api.h"
#include "contrib_ops/cuda/sparse/sparse_attention_v2/sparse_attention_v2_api.h"

namespace onnxruntime {
namespace contrib {
namespace cuda {

// Convert total_seq_len_k (total key sequence length excluding paddings) to position_ids for Prompt
__global__ void PositionIdsPrompt(const int32_t* total_seq_len_k,
                                  int64_t* position_ids,
                                  int sequence_length,
                                  int batch_size) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < batch_size * sequence_length) {
    int b = tid / sequence_length;
    int s = tid % sequence_length;
    if (s < total_seq_len_k[b]) {
      position_ids[tid] = s;
    } else {
      // padding
      position_ids[tid] = 1;
    }
  }
}

// Convert total_seq_len_k (total key sequence length excluding paddings) to position_ids for Token Generation
__global__ void PositionIdsToken(const int32_t* total_seq_len_k,
                                 int64_t* position_ids,
                                 int batch_size) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < batch_size) {
    position_ids[tid] = total_seq_len_k[tid] - 1;
  }
}

// Convert total_seq_len_k (total key sequence length excluding paddings) to position_ids
Status FillPositionIds(contrib::SparseAttentionParameters& parameters,
                       const int32_t* total_seq_len_k,
                       int64_t* position_ids,
                       hipStream_t stream,
                       const int max_threads_per_block) {
  const int sequence_length = parameters.sequence_length;
  const int batch_size = parameters.batch_size;
  const int bs = batch_size * sequence_length;

  int threads = max_threads_per_block;
  if (bs <= 64) {
    threads = 64;
  } else if (bs <= 128) {
    threads = 128;
  } else if (bs <= 256) {
    threads = 256;
  } else if (bs <= 512) {
    threads = 512;
  }
  const int blocks = (bs + threads - 1) / threads;

  if (parameters.sequence_length == parameters.total_sequence_length) {  // prompt
    PositionIdsPrompt<<<blocks, threads, 0, stream>>>(total_seq_len_k, position_ids, sequence_length, batch_size);
  } else {
    PositionIdsToken<<<blocks, threads, 0, stream>>>(total_seq_len_k, position_ids, batch_size);
  }

  return CUDA_CALL(hipGetLastError());
}

// Concat new key and value (BSNH format) to kv buffer (BNSH format) in place.
template <typename T>
Status LaunchConcatKVInPlace(contrib::SparseAttentionParameters& parameters,
                             SparseAttentionData<T>& data,
                             const void* new_key,
                             const void* new_value,
                             bool is_new_kv_bnsh_format,
                             hipStream_t stream,
                             const int max_threads_per_block) {
  constexpr bool is_past_kv_bnsh_format = true;
  return LaunchConcatKVInPlace(parameters.batch_size,
                               parameters.kv_num_heads,
                               parameters.head_size,
                               parameters.max_sequence_length,
                               nullptr,
                               data.seqlens_k_total,
                               parameters.sequence_length,
                               reinterpret_cast<const T*>(new_key),
                               reinterpret_cast<const T*>(new_value),
                               data.present_key,
                               data.present_value,
                               is_past_kv_bnsh_format,
                               is_new_kv_bnsh_format,
                               stream,
                               max_threads_per_block);
}

template <typename T>
Status QkvToContext(
    const hipDeviceProp_t& device_prop,
    Stream* ort_stream,
    contrib::SparseAttentionParameters& parameters,
    SparseAttentionData<T>& data) {
  hipStream_t stream = static_cast<hipStream_t>(ort_stream->GetHandle());
  const int max_threads_per_block = device_prop.maxThreadsPerBlock;
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  // const int present_sequence_length = parameters.max_sequence_length;
  const int num_heads = parameters.num_heads;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;

  const void* query;
  const void* key;
  const void* value;

  DUMP_TENSOR_INIT();

  if (!parameters.is_packed_qkv) {
    static_assert(sizeof(T) == 2);
    ORT_RETURN_IF_ERROR(Transpose_BSNH_to_BNSH(
        batch_size, sequence_length, num_heads, head_size,
        reinterpret_cast<const half*>(data.query), reinterpret_cast<half*>(data.transposed_q_buffer),
        stream, max_threads_per_block));
    query = reinterpret_cast<const void*>(data.transposed_q_buffer);
    key = reinterpret_cast<const void*>(data.key);
    value = reinterpret_cast<const void*>(data.value);
  } else {
    size_t q_size = static_cast<size_t>(batch_size * sequence_length * num_heads * head_size);
    size_t k_size = static_cast<size_t>(batch_size * sequence_length * kv_num_heads * head_size);
    auto q = reinterpret_cast<T*>(data.unpacked_qkv_buffer);
    auto k = reinterpret_cast<T*>(data.unpacked_qkv_buffer + q_size);
    auto v = reinterpret_cast<T*>(data.unpacked_qkv_buffer + q_size + k_size);

    Status status = LaunchUnpackQKV<T, LAYOUT_BNSH>(data.query, q, k, v, num_heads, kv_num_heads, head_size,
                                                    sequence_length, batch_size, stream, max_threads_per_block);
    if (status != Status::OK()) {
      return status;
    }

    query = reinterpret_cast<const void*>(q);
    key = reinterpret_cast<const void*>(k);
    value = reinterpret_cast<const void*>(v);
  }

  constexpr bool q_layout = LAYOUT_BNSH;
  bool kv_layout = parameters.is_packed_qkv ? LAYOUT_BNSH : LAYOUT_BSNH;

#if DUMP_TENSOR_LEVEL > 0
  DUMP_TENSOR("query (BNSH)", reinterpret_cast<const T*>(query), batch_size, num_heads, sequence_length, head_size);

  if (LAYOUT_BNSH == kv_layout) {
    DUMP_TENSOR("key (BNSH)", reinterpret_cast<const T*>(key), batch_size, kv_num_heads, sequence_length, head_size);
    DUMP_TENSOR("value (BNSH)", reinterpret_cast<const T*>(value), batch_size, kv_num_heads, sequence_length, head_size);
  } else {
    DUMP_TENSOR("key (BSNH)", reinterpret_cast<const T*>(key), batch_size, sequence_length, kv_num_heads, head_size);
    DUMP_TENSOR("value (BSNH)", reinterpret_cast<const T*>(value), batch_size, sequence_length, kv_num_heads, head_size);
  }
#endif

  if (parameters.do_rotary) {
    size_t bsh = static_cast<size_t>(parameters.batch_size * parameters.sequence_length * parameters.head_size);
    size_t q_size = bsh * static_cast<size_t>(parameters.num_heads);
    size_t k_size = bsh * static_cast<size_t>(parameters.kv_num_heads);
    auto q_buffer = reinterpret_cast<T*>(data.rotary_buffer);
    auto k_buffer = q_buffer + q_size;
    auto position_ids_buff = reinterpret_cast<int64_t*>(k_buffer + k_size);
    ORT_RETURN_IF_ERROR(FillPositionIds(parameters, data.seqlens_k_total, position_ids_buff, stream,
                                        max_threads_per_block));

    DUMP_TENSOR("position_ids", position_ids_buff, batch_size, sequence_length);

    // Launch rotary embedding kernel. This requires separated Q, K and V
    ORT_RETURN_IF_ERROR(LaunchRotaryEmbeddingKernel<T>(stream, q_buffer, reinterpret_cast<const T*>(query),
                                                       position_ids_buff, data.cos_cache, data.sin_cache,
                                                       parameters.batch_size, parameters.sequence_length,
                                                       parameters.num_heads, parameters.head_size,
                                                       parameters.rotary_dim, parameters.max_sequence_length,
                                                       /*position_ids_format*/ 1, parameters.rotary_interleaved,
                                                       max_threads_per_block, q_layout));
    ORT_RETURN_IF_ERROR(LaunchRotaryEmbeddingKernel<T>(stream, k_buffer, reinterpret_cast<const T*>(key),
                                                       position_ids_buff, data.cos_cache, data.sin_cache,
                                                       parameters.batch_size, parameters.sequence_length,
                                                       parameters.kv_num_heads, parameters.head_size,
                                                       parameters.rotary_dim, parameters.max_sequence_length,
                                                       /*position_ids_format*/ 1, parameters.rotary_interleaved,
                                                       max_threads_per_block, kv_layout));
    query = reinterpret_cast<const void*>(q_buffer);
    key = reinterpret_cast<const void*>(k_buffer);

#if DUMP_TENSOR_LEVEL > 0
    DUMP_TENSOR("query after rotary", reinterpret_cast<const T*>(query),
                batch_size, num_heads, sequence_length, head_size);
    if (LAYOUT_BNSH == kv_layout) {
      DUMP_TENSOR("key after rotary", reinterpret_cast<const T*>(key),
                  batch_size, kv_num_heads, sequence_length, head_size);
    } else {
      DUMP_TENSOR("key after rotary", reinterpret_cast<const T*>(key),
                  batch_size, sequence_length, kv_num_heads, head_size);
    }
#endif
  }

  // Concat new key and value to kv buffers (in BNSH format) in place
  ORT_ENFORCE(parameters.past_present_share_buffer);
  ORT_RETURN_IF_ERROR(LaunchConcatKVInPlace(
      parameters, data, key, value, kv_layout, stream, max_threads_per_block));

  // TODO: only dump to total sequence length instead of max sequence length.
#if DUMP_TENSOR_LEVEL > 0
  DUMP_TENSOR("key cache", data.present_key, batch_size, kv_num_heads, parameters.max_sequence_length, head_size);
  DUMP_TENSOR("value cache", data.present_value, batch_size, kv_num_heads, parameters.max_sequence_length, head_size);

  DUMP_TENSOR("block_mask",
              data.kernel_layout.mask,
              data.kernel_layout.num_layout,
              data.kernel_layout.num_rows,
              data.kernel_layout.num_cols);

  DUMP_TENSOR("csr_col_indices",
              data.kernel_layout.csr_col_indices,
              data.kernel_layout.num_layout,
              data.kernel_layout.num_rows,
              data.kernel_layout.num_cols);

  DUMP_TENSOR("csr_row_indices",
              data.kernel_layout.csr_row_indices,
              data.kernel_layout.num_layout,
              data.kernel_layout.num_rows + 1);

  printf(
      "batch_size=%d, sequence_length=%d, num_heads=%d, kv_num_heads=%d head_size=%d, "
      "total_sequence_length=%d, max_sequence_length=%d scale=%f block_size=%d "
      "row_stride=%d col_stride=%d num_layout=%d\n",
      parameters.batch_size,
      parameters.sequence_length,
      parameters.num_heads,
      parameters.kv_num_heads,
      parameters.head_size,
      parameters.total_sequence_length,
      parameters.max_sequence_length,
      parameters.scale,
      data.kernel_layout.block_size,
      data.kernel_layout.num_rows + 1,
      data.kernel_layout.num_rows * data.kernel_layout.num_cols,
      data.kernel_layout.num_layout);
#endif

  int sm = device_prop.major * 10 + device_prop.minor;
  if (data.use_v2_kernel) {
    sparse_attention_v2::SparseAttentionParams params(
        ort_stream,
        sm,
        data.output,
        reinterpret_cast<const void*>(query),
        reinterpret_cast<const void*>(data.present_key),
        reinterpret_cast<const void*>(data.present_value),
        parameters.batch_size,
        parameters.sequence_length,
        parameters.num_heads,
        parameters.kv_num_heads,
        parameters.head_size,
        parameters.total_sequence_length,
        parameters.max_sequence_length,
        parameters.scale,
        data.kernel_layout.block_size,                              // kernel_block_size
        data.kernel_layout.csr_row_indices,                         // skip past_seq_len in row indices
        data.kernel_layout.csr_col_indices,                         // (num_layout, num_rows, num_cols)
        data.kernel_layout.num_rows + 1,                            // stride per head in row indices
        data.kernel_layout.num_rows * data.kernel_layout.num_cols,  // stride per head in col indices
        data.kernel_layout.num_layout,
        data.active_q_blocks,
        data.q_batch_starts,
        data.q_batch_ends,
        data.k_batch_starts,
        data.k_batch_ends,
        data.q_batch_ids,
        data.q_start_sids);

    if constexpr (std::is_same<T, BFloat16>::value) {
      ORT_RETURN_IF_ERROR(sparse_attention_v2::run_sparse_attention_bf16(params));
    } else {
      ORT_RETURN_IF_ERROR(sparse_attention_v2::run_sparse_attention_fp16(params));
    }
  } else {
    sparse_attention_v1::SparseAttentionParams params(
        ort_stream,
        sm,
        data.output,
        reinterpret_cast<const void*>(query),
        reinterpret_cast<const void*>(data.present_key),
        reinterpret_cast<const void*>(data.present_value),
        parameters.batch_size,
        parameters.sequence_length,
        parameters.num_heads,
        parameters.kv_num_heads,
        parameters.head_size,
        parameters.total_sequence_length,
        parameters.max_sequence_length,
        parameters.scale,
        data.kernel_layout.block_size,                              // kernel_block_size
        data.kernel_layout.csr_row_indices,                         // (num_layout, num_rows + 1)
        data.kernel_layout.csr_col_indices,                         // (num_layout, num_rows, num_cols)
        data.kernel_layout.num_rows + 1,                            // stride per head in row indices
        data.kernel_layout.num_rows * data.kernel_layout.num_cols,  // stride per head in col indices
        data.kernel_layout.num_layout);

    if constexpr (std::is_same<T, BFloat16>::value) {
      ORT_RETURN_IF_ERROR(sparse_attention_v1::run_sparse_attention_bf16(params));
    } else {
      ORT_RETURN_IF_ERROR(sparse_attention_v1::run_sparse_attention_fp16(params));
    }
  }

  DUMP_TENSOR("output", reinterpret_cast<const T*>(data.output), batch_size, num_heads, sequence_length, head_size);

  return Status::OK();
}

template Status QkvToContext<half>(
    const hipDeviceProp_t& device_prop,
    Stream* ort_stream,
    contrib::SparseAttentionParameters& parameters,
    SparseAttentionData<half>& data);

template Status QkvToContext<BFloat16>(
    const hipDeviceProp_t& device_prop,
    Stream* ort_stream,
    contrib::SparseAttentionParameters& parameters,
    SparseAttentionData<BFloat16>& data);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
