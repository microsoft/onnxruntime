/*
 * The implementation of this file is based on code provided by https://github.com/NVIDIA/FasterTransformer
 *
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// Modifications Copyright (c) Microsoft.
// Licensed under the MIT License.

#include "decoder_masked_multihead_attention_impl.h"
#include "decoder_masked_multihead_attention_impl_utils.h"

namespace onnxruntime {
namespace contrib {
namespace cuda {

using namespace decoder_masked_self_attention_details;

#define MMHA_LAUNCH_KERNEL(                                                                        \
    T, head_size, THDS_PER_KEY, THDS_PER_VALUE, THDS_PER_BLOCK)                                    \
  size_t dynamic_block_memory = CalcDynamicBlockMemory<T>(params, THDS_PER_VALUE, THDS_PER_BLOCK); \
  dim3 grid(params.num_heads, params.batch_size);                                                  \
  masked_multihead_attention_kernel<T,                                                             \
                                    head_size,                                                     \
                                    THDS_PER_KEY,                                                  \
                                    THDS_PER_VALUE,                                                \
                                    THDS_PER_BLOCK>                                                \
      <<<grid, THDS_PER_BLOCK, dynamic_block_memory, stream>>>(params)

template <typename T, int head_size>
void mmha_launch_kernel(const DecoderMaskedMultiHeadAttentionParams& params, hipStream_t stream) {
  constexpr int THREADS_PER_VALUE = ThreadsPerValue<T, head_size>::value;
  int total_sequence_length = params.total_sequence_length;

  if (total_sequence_length < 32) {
    MMHA_LAUNCH_KERNEL(T, head_size, 4, THREADS_PER_VALUE, 64);
  } else if (total_sequence_length < 2048) {
    MMHA_LAUNCH_KERNEL(T, head_size, 2, THREADS_PER_VALUE, 128);
  } else {
    MMHA_LAUNCH_KERNEL(T, head_size, 1, THREADS_PER_VALUE, 256);
  }
}

// Instantiate templates
template void mmha_launch_kernel<float, 64>(const DecoderMaskedMultiHeadAttentionParams& params, hipStream_t stream);

template void mmha_launch_kernel<uint16_t, 64>(const DecoderMaskedMultiHeadAttentionParams& params, hipStream_t stream);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime