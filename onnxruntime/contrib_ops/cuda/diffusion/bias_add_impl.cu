#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

// The CUDA kernel is modified from SeqLen2Spatial plugin of TensorRT 8.5.
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hipcub/hipcub.hpp>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "contrib_ops/cuda/diffusion/bias_add_impl.h"

using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

template <typename T, int32_t C, int32_t TPB>
__global__ void BiasAddKernel(T const* input, T const* bias, T const* residual, T* output) {
  int32_t base_offset = blockIdx.x * C + threadIdx.x;
  int32_t bias_offset = threadIdx.x;

#pragma unroll
  for (int32_t i = 0; i < C / TPB; ++i) {
    output[base_offset] = input[base_offset] + bias[bias_offset] + residual[base_offset];
    base_offset += TPB;
    bias_offset += TPB;
  }
}

template <typename T, unsigned TPB>
__global__ void BiasAddLargeKernel(
    int32_t const ld, const T* input, const T* bias, const T* residual, T* output) {
  int32_t const offset = blockIdx.x * ld;

  for (int32_t i = threadIdx.x; i < ld; i += TPB) {
    int32_t const base_offset = offset + i;
    output[base_offset] = input[base_offset] + bias[i] + residual[base_offset];
  }
}

template __global__ void BiasAddKernel<float, 320, 320>(float const*, float const*, float const*, float*);
template __global__ void BiasAddKernel<float, 640, 320>(float const*, float const*, float const*, float*);
template __global__ void BiasAddKernel<float, 1280, 320>(float const*, float const*, float const*, float*);
template __global__ void BiasAddKernel<half, 320, 320>(half const*, half const*, half const*, half*);
template __global__ void BiasAddKernel<half, 640, 320>(half const*, half const*, half const*, half*);
template __global__ void BiasAddKernel<half, 1280, 320>(half const*, half const*, half const*, half*);

template <typename T>
void LaunchBiasAddKernel(hipStream_t stream, int32_t grid_size, int32_t num_channels,
                         T const* input, T const* bias, T const* residual, T* output) {
  switch (num_channels) {
    case 320:
      (BiasAddKernel<T, 320, 320>)<<<grid_size, 320, 0, stream>>>(input, bias, residual, output);
      break;
    case 640:
      (BiasAddKernel<T, 640, 320>)<<<grid_size, 320, 0, stream>>>(input, bias, residual, output);
      break;
    case 1280:
      (BiasAddKernel<T, 1280, 320>)<<<grid_size, 320, 0, stream>>>(input, bias, residual, output);
      break;
    default:
      BiasAddLargeKernel<T, 256><<<grid_size, 256, 0, stream>>>(num_channels, input, bias, residual, output);
      break;
  }
}

template void LaunchBiasAddKernel<float>(hipStream_t stream, int32_t grid_size, int32_t num_channels,
                                         float const* input, float const* bias, float const* residual, float* output);

template void LaunchBiasAddKernel<half>(hipStream_t stream, int32_t grid_size, int32_t num_channels,
                                        half const* input, half const* bias, half const* residual, half* output);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
