/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// The CUDA kernel is modified from GroupNorm plugin of TensorRT 8.5
// Modifications: heuristic channels per block; support epsilon; support skip and bias; update coding style.
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <hipcub/hipcub.hpp>
#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "contrib_ops/cuda/diffusion/group_norm_impl.h"
#include "contrib_ops/cuda/utils/dump_cuda_tensor.h"
#include "contrib_ops/cuda/diffusion/group_norm_common_base.h"
#include "contrib_ops/cuda/diffusion/group_norm_impl_kernel.cuh"

using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

template <typename T>
void GroupNormNHWCSum(GroupNormNHWCParams<T> const& params, hipStream_t stream) {
  dim3 grid;

  // The number of blocks to compute all the channels.
  grid.x = DivUp(params.c, params.channels_per_block);

  // The number of blocks to compute all the activations in a given instance.
  grid.y = DivUp(params.hw, params.hw_per_block);

  // The number of instances.
  grid.z = params.n;

#define LAUNCH_GROUPNORM_SUM(ThreadsPerBlock, VecSize)                                               \
  GroupNormNHWCSumKernel<T, ThreadsPerBlock, VecSize>                                                \
      <<<grid, ThreadsPerBlock, 0, stream>>>(                                                        \
          params.skip_workspace, params.group_sum_buffer, params.src, params.skip, params.bias,       \
          params.channels_per_block, params.hw_per_block, params.hw, params.hwc, params.c,           \
          params.channels_per_group, params.groups, params.groups_per_block, params.broadcast_skip); \
  break;

  // Threads_per_block is half of values in kSizes since CHANNELS_PER_THREAD = 2.
  switch (params.threads_per_block) {
    case 256:
      LAUNCH_GROUPNORM_SUM(256, CHANNELS_PER_THREAD)
    case 192:
      LAUNCH_GROUPNORM_SUM(192, CHANNELS_PER_THREAD)
    case 160:
      LAUNCH_GROUPNORM_SUM(160, CHANNELS_PER_THREAD)
    case 128:
      LAUNCH_GROUPNORM_SUM(128, CHANNELS_PER_THREAD)
    case 64:
      LAUNCH_GROUPNORM_SUM(64, CHANNELS_PER_THREAD)
  }
}

template <typename T>
void GroupNormNHWCScale(GroupNormNHWCParams<T> const& params, hipStream_t stream) {
  dim3 grid;

  // The number of blocks to compute all the channels.
  grid.x = DivUp(params.c, params.channels_per_block);
  // The number of blocks to compute all the activations in a given instance.
  grid.y = DivUp(params.hw, params.hw_per_block);
  // The number of instances.
  grid.z = params.n;

#define LAUNCH_GROUPNORM_SCALE(ThreadsPerBlock, VecSize)                                                           \
  GroupNormNHWCScaleKernel<T, VecSize>                                                                             \
      <<<grid, ThreadsPerBlock, 0, stream>>>(                                                                      \
          params.dst, params.src, params.skip, params.gamma, params.beta, params.skip_workspace,                   \
          params.group_sum_buffer, params.epsilon, params.c, params.channels_per_block, params.channels_per_group, \
          params.groups, params.hwc, params.inv_hw_channels_per_group, params.hw, params.hw_per_block,             \
          params.use_silu);                                                                                        \
  break;

  // Threads_per_block is half of values in kSizes since CHANNELS_PER_THREAD = 2.
  switch (params.threads_per_block) {
    case 256:
      LAUNCH_GROUPNORM_SCALE(256, CHANNELS_PER_THREAD)
    case 192:
      LAUNCH_GROUPNORM_SCALE(192, CHANNELS_PER_THREAD)
    case 160:
      LAUNCH_GROUPNORM_SCALE(160, CHANNELS_PER_THREAD)
    case 128:
      LAUNCH_GROUPNORM_SCALE(128, CHANNELS_PER_THREAD)
    case 64:
      LAUNCH_GROUPNORM_SCALE(64, CHANNELS_PER_THREAD)
  }
}

template <typename T>
Status LaunchGroupNormKernel(
    CudaTuningContext* tuning_ctx,
    Stream* ort_stream,
    T* output,
    T* add_out,
    const T* input,
    const T* skip,
    const T* bias,
    const float* gamma,
    const float* beta,
    void* workspace,
    float epsilon,
    int batch_size,
    int num_channels,
    int height,
    int width,
    int num_groups,
    bool use_silu,
    bool broadcast_skip,
    int channels_per_block) {

  // tuning_ctx only used for ROCm EP.
  ORT_UNUSED_PARAMETER(tuning_ctx);

  GroupNormNHWCParams<T> params(output, add_out, input, skip, bias, gamma, beta, reinterpret_cast<float*>(workspace), epsilon,
                                batch_size, num_channels, height, width, num_groups, use_silu,
                                broadcast_skip, channels_per_block);

  if (params.channels_per_block % params.channels_per_group != 0 ||
      params.channels_per_block > kMaxSize ||
      (params.channels_per_group % CHANNELS_PER_THREAD != 0)) {
    return ORT_MAKE_STATUS(ONNXRUNTIME, NOT_IMPLEMENTED,
                           "GroupNorm in CUDA does not support the input: n=", batch_size,
                           " h=", height,
                           " w=", width,
                           " c=", num_channels,
                           " groups=", num_groups);
  }

  auto stream = static_cast<hipStream_t>(ort_stream->GetHandle());
  CUDA_RETURN_IF_ERROR(hipMemsetAsync(
      params.group_sum_buffer, 0, GetGroupNormWorkspaceSizeInBytes(batch_size, num_groups), stream));

  GroupNormNHWCSum<T>(params, stream);
  CUDA_RETURN_IF_ERROR(hipGetLastError());

  DUMP_TENSOR_INIT();
  DUMP_TENSOR("workspace", params.group_sum_buffer, batch_size, 2, num_groups);

  GroupNormNHWCScale<T>(params, stream);
  CUDA_RETURN_IF_ERROR(hipGetLastError());

  return Status::OK();
}

template Status LaunchGroupNormKernel<half>(CudaTuningContext* tuning_ctx, Stream* stream, half* output, half* add_out,
                                            const half* input, const half* skip, const half* bias,
                                            const float* gamma, const float* beta, void* workspace,
                                            float epsilon, int batch_size, int num_channels,
                                            int height, int width, int num_groups, bool silu,
                                            bool broadcast_skip, int channels_per_block);

template Status LaunchGroupNormKernel<float>(CudaTuningContext* tuning_ctx, Stream* stream, float* output, float* add_out,
                                             const float* input, const float* skip, const float* bias,
                                             const float* gamma, const float* beta, void* workspace,
                                             float epsilon, int batch_size, int num_channels,
                                             int height, int width, int num_groups, bool silu,
                                             bool broadcast_skip, int channels_per_block);
}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
