/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cassert>
#include <cmath>
#include <cstdint>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include "core/providers/cuda/cuda_common.h"
#include "contrib_ops/cuda/llm/fpA_intB_gemv/fpA_intB_gemv.h"
#include "contrib_ops/cuda/llm/fpA_intB_gemv/details.h"

namespace onnxruntime::llm {
namespace kernels {
namespace fpA_intB_gemv {

void kernel_launcher(int arch, Params& params, hipStream_t s) {
#define EXEC(KType, A, B, Layout, ConverterInterleave)                                                       \
  if (params.type == KType) {                                                                                \
    select_gs<kernel_type_traits<KType>::isGroupwise, KernelDetails<A, B, Layout, ConverterInterleave, 64>>( \
        params, s);                                                                                          \
    return;                                                                                                  \
  }

// This is not used since there is no alpha for MatMulNBits currently.
#define EXEC_W4A8(KType, A, B, Layout, ConverterInterleave)                                                   \
  if (params.type == KType && params.apply_alpha_in_advance) {                                                \
    select_gs<kernel_type_traits<KType>::isGroupwise, KernelDetails<A, B, Layout, ConverterInterleave, 128>>( \
        params, s);                                                                                           \
    return;                                                                                                   \
  }

  if (arch >= 75 && arch < 80) {
    EXEC(KernelType::FP16Int8Groupwise, FP16DetailsA, Int8DetailsW, ColumnMajorInterleaved, true);
    EXEC(KernelType::FP16Int4Groupwise, FP16DetailsA, Int4DetailsW, ColumnMajorInterleaved, true);
  } else if (arch >= 80 && arch < 90 || arch >= 100) {
    // if (arch == 89 || arch >= 120)
    // {
    //     EXEC_W4A8(KernelType::FP16Int4Groupwise, FP16DetailsA, Int4DetailsW, ColumnMajorInterleaved, true);
    //     EXEC_W4A8(KernelType::BF16Int4Groupwise, BF16DetailsA, Int4DetailsW, ColumnMajorInterleaved, true);
    // }
    EXEC(KernelType::FP16Int8Groupwise, FP16DetailsA, Int8DetailsW, ColumnMajorInterleaved, true);
    EXEC(KernelType::FP16Int4Groupwise, FP16DetailsA, Int4DetailsW, ColumnMajorInterleaved, true);

    EXEC(KernelType::BF16Int8Groupwise, BF16DetailsA, Int8DetailsW, ColumnMajorInterleaved, true);
    EXEC(KernelType::BF16Int4Groupwise, BF16DetailsA, Int4DetailsW, ColumnMajorInterleaved, true);
  } else if (arch >= 90) {
    // Dispatchers for W4A8 groupwise
    // EXEC_W4A8(KernelType::FP16Int4Groupwise, FP16DetailsA, Int4DetailsW, ColumnMajorInterleavedForHopper, true);
    // EXEC_W4A8(KernelType::BF16Int4Groupwise, BF16DetailsA, Int4DetailsW, ColumnMajorInterleavedForHopper, true);

    EXEC(KernelType::FP16Int8Groupwise, FP16DetailsA, Int8DetailsW, ColumnMajorInterleavedForHopper, true);
    EXEC(KernelType::FP16Int4Groupwise, FP16DetailsA, Int4DetailsW, ColumnMajorInterleavedForHopper, true);

    EXEC(KernelType::BF16Int8Groupwise, BF16DetailsA, Int8DetailsW, ColumnMajorInterleavedForHopper, true);
    EXEC(KernelType::BF16Int4Groupwise, BF16DetailsA, Int4DetailsW, ColumnMajorInterleavedForHopper, true);
  }
#undef EXEC_W4A8
#undef EXEC
}

bool is_supported(int arch, KernelType kernel_type) {
#define SUPPORT(Type)      \
  if (kernel_type == Type) \
    return true;

  if (arch >= 75 && arch < 80) {
    SUPPORT(KernelType::FP16Int8Groupwise);
    SUPPORT(KernelType::FP16Int4Groupwise);
  } else if (arch >= 80) {
    SUPPORT(KernelType::FP16Int8Groupwise);
    SUPPORT(KernelType::FP16Int4Groupwise);

    SUPPORT(KernelType::BF16Int8Groupwise);
    SUPPORT(KernelType::BF16Int4Groupwise);
  }
  return false;
#undef SUPPORT
}

}  // namespace fpA_intB_gemv
}  // namespace kernels
}  // namespace onnxruntime::llm
