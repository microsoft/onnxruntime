/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "contrib_ops/cuda/llm/fpA_intB_gemv/dispatcher.h"

namespace onnxruntime::llm {
namespace kernels {
namespace fpA_intB_gemv {

INSTANTIATE_WEIGHT_ONLY_CUDA_DISPATCHERS(
    KernelType::FP16Int4Groupwise, FP16DetailsA, Int4DetailsW, ColumnMajorInterleaved, true, 64);

// KTile=128 for Ada w4a8
// INSTANTIATE_WEIGHT_ONLY_CUDA_DISPATCHERS(
//     KernelType::FP16Int4Groupwise, FP16DetailsA, Int4DetailsW, ColumnMajorInterleaved, true, 128);

}  // namespace fpA_intB_gemv
}  // namespace kernels
}  // namespace onnxruntime::llm
