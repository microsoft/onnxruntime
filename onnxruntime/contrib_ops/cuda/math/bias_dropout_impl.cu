#include "hip/hip_runtime.h"
/**
* Copyright (c) 2016-present, Facebook, Inc.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

/* Modifications Copyright (c) Microsoft. */

#include "core/providers/cuda/cu_inc/common.cuh"
#include "contrib_ops/cuda/math/bias_dropout.h"

#include <hiprand/hiprand_kernel.h>
#include <algorithm>

namespace onnxruntime {
namespace contrib {
namespace cuda {

constexpr int UNROLL = 4;

template <typename T, bool has_same_shape_bias, bool has_residual>
__global__ void BiasDropoutKernel(
    const int64_t N,
    const fast_divmod fdm_dim,
    const float ratio,
    const std::pair<uint64_t, uint64_t> seeds,
    const T* X_data,
    const T* bias_data,
    const T* residual_data,
    T* Y_data,
    bool* mask_data) {
  const float p = 1.0f - ratio;
  const float scale = 1.0f / p;

  CUDA_LONG idx = blockDim.x * blockIdx.x + threadIdx.x;
  CUDA_LONG step_size = gridDim.x * blockDim.x * UNROLL;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seeds.first, idx, seeds.second, &state);

  float4 rand;

  // We ensure every thread generates the same number of random numbers (by rounding
  // up the size) and at the same timestep (by syncing threads).
  // From CUDA hiprand documentation:
  //   The Philox_4x32_10 algorithm is closely tied to the thread and block count.
  //   Each thread computes 4 random numbers in the same time thus the most efficient
  //   use of Philox_4x32_10 is to generate a multiple of 4 times number of threads.
  for (CUDA_LONG id = idx * UNROLL; id < N; id += step_size) {
    rand = hiprand_uniform4(&state);

    // actual computation
    #pragma unroll
    for (int i = 0; i < UNROLL; i++) {
      CUDA_LONG li = id + i;
      if (li < N) {
        float bias;
        if (has_same_shape_bias) {
          bias = float(bias_data[li]);
        } else {
          int offset = fdm_dim.mod(li);
          bias = float(bias_data[offset]);
        }

        mask_data[li] = (&rand.x)[i] < p;
        float output_data = (float(X_data[li]) + bias) * mask_data[li] * scale;
        if (has_residual) {
          output_data += float(residual_data[li]);
        }

        Y_data[li] = T(output_data);
      }
    }

    __syncthreads();
  }

}


template <typename T, bool has_same_shape_bias, bool has_residual>
__global__ void BiasDropoutVectorizedKernel(
    const int64_t N,
    const fast_divmod fdm_dim,
    const float ratio,
    const std::pair<uint64_t, uint64_t> seeds,
    const T* X_data,
    const T* bias_data,
    const T* residual_data,
    T* Y_data,
    bool* mask_data) {
  const float p = 1.0f - ratio;
  const float scale = 1.0f / p;

  CUDA_LONG idx = blockDim.x * blockIdx.x + threadIdx.x;
  CUDA_LONG step_size = gridDim.x * blockDim.x * UNROLL;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seeds.first, idx, seeds.second, &state);

  float4 rand;

  // using vectorized data load/store approach when N % 4 == 0
  // since this is typical case for input shape size
  using LoadT = aligned_vector<T, UNROLL>;
  using MaskLoadT = aligned_vector<bool, UNROLL>;
  using ResidualLoadT = aligned_vector<T, UNROLL>;

  for (CUDA_LONG id = idx * UNROLL; id < N; id += step_size) {
    rand = hiprand_uniform4(&state);

    // vectorized load into storage
    T bias_vec[UNROLL];
    if (has_same_shape_bias) {
      LoadT *value0 = reinterpret_cast<LoadT*>(&bias_vec);
      *value0 = *reinterpret_cast<const LoadT*>(&bias_data[id]);
    }

    T src[UNROLL];
    LoadT *value1 = reinterpret_cast<LoadT*>(&src);
    *value1 = *reinterpret_cast<const LoadT*>(&X_data[id]);

    T residual[UNROLL];
    if (has_residual) {
      ResidualLoadT *value2 = reinterpret_cast<ResidualLoadT*>(&residual);
      *value2 = *reinterpret_cast<const ResidualLoadT*>(&residual_data[id]);
    }

    T r[UNROLL];
    bool mask[UNROLL];

    // actual computation
    #pragma unroll
    for (int ii = 0; ii < UNROLL; ii++) {
      float bias;
      if (has_same_shape_bias) {
        bias = float(bias_vec[ii]);
      } else {
        int offset = fdm_dim.mod(id + ii);
        bias = float(bias_data[offset]);
      }

      mask[ii] = (&rand.x)[ii] < p;
      float output_data = (float(src[ii]) + bias) * mask[ii] * scale;
      if (has_residual) {
        output_data += float(residual[ii]);
      }
      r[ii] = T(output_data);
    }
    // Vectorized writes for mask_data & Y_data
    *(reinterpret_cast<LoadT*>(&Y_data[id])) = *reinterpret_cast<LoadT*>(&r[0]);
    *(reinterpret_cast<MaskLoadT*>(&mask_data[id])) = *reinterpret_cast<MaskLoadT*>(&mask[0]);

    __syncthreads();
  }

}

template <typename T>
void BiasDropoutKernelImpl(
    const hipDeviceProp_t& prop,
    hipStream_t stream,
    const int64_t N,
    const fast_divmod fdm_dim,
    const float ratio,
    PhiloxGenerator& generator,
    const T* X_data,
    const T* bias_data,
    const T* residual_data,
    T* Y_data,
    bool* mask_data,
    bool has_same_shape_bias) {
  const int block_size = 256;
  const int blocks_per_sm = prop.maxThreadsPerMultiProcessor / block_size;
  const int grid_size = std::min(prop.multiProcessorCount * blocks_per_sm, static_cast<int>(CeilDiv(N, block_size * UNROLL)));

  // Compute the number of random numbers generated by each thread, and increment philox generator offset by that amount.
  const uint64_t counter_offset = static_cast<uint64_t>(((N - 1) / (block_size * grid_size * UNROLL) + 1) * UNROLL);
  auto seeds = generator.NextPhiloxSeeds(counter_offset);

  if (N % UNROLL != 0) {
    if (has_same_shape_bias) {
      if (residual_data == nullptr) {
        BiasDropoutKernel<T, true, false><<<grid_size, block_size, 0, stream>>>(N, fdm_dim, ratio, seeds, X_data, bias_data, residual_data, Y_data, mask_data);
      } else {
        BiasDropoutKernel<T, true, true><<<grid_size, block_size, 0, stream>>>(N, fdm_dim, ratio, seeds, X_data, bias_data, residual_data, Y_data, mask_data);
      }
    } else {
      if (residual_data == nullptr) {
        BiasDropoutKernel<T, false, false><<<grid_size, block_size, 0, stream>>>(N, fdm_dim, ratio, seeds, X_data, bias_data, residual_data, Y_data, mask_data);
      } else {
        BiasDropoutKernel<T, false, true><<<grid_size, block_size, 0, stream>>>(N, fdm_dim, ratio, seeds, X_data, bias_data, residual_data, Y_data, mask_data);
      }
    }
  } else {
    if (has_same_shape_bias) {
      if (residual_data == nullptr) {
        BiasDropoutVectorizedKernel<T, true, false><<<grid_size, block_size, 0, stream>>>(N, fdm_dim, ratio, seeds, X_data, bias_data, residual_data, Y_data, mask_data);
      } else {
        BiasDropoutVectorizedKernel<T, true, true><<<grid_size, block_size, 0, stream>>>(N, fdm_dim, ratio, seeds, X_data, bias_data, residual_data, Y_data, mask_data);
      }
    } else {
      if (residual_data == nullptr) {
        BiasDropoutVectorizedKernel<T, false, false><<<grid_size, block_size, 0, stream>>>(N, fdm_dim, ratio, seeds, X_data, bias_data, residual_data, Y_data, mask_data);
      } else {
        BiasDropoutVectorizedKernel<T, false, true><<<grid_size, block_size, 0, stream>>>(N, fdm_dim, ratio, seeds, X_data, bias_data, residual_data, Y_data, mask_data);
      }
    }
  }
}

#define SPECIALIZED_BIAS_DROPOUT_IMPL(T) \
  template void BiasDropoutKernelImpl(  \
      const hipDeviceProp_t& prop,   \
      hipStream_t stream,          \
      const int64_t N,              \
      const fast_divmod fdm_dim,    \
      const float ratio,            \
      PhiloxGenerator& generator,   \
      const T* X_data,              \
      const T* bias_data,           \
      const T* residual_data,       \
      T* Y_data,                    \
      bool* mask_data,              \
      bool has_same_shape_bias);


SPECIALIZED_BIAS_DROPOUT_IMPL(float)
SPECIALIZED_BIAS_DROPOUT_IMPL(double)
SPECIALIZED_BIAS_DROPOUT_IMPL(half)
SPECIALIZED_BIAS_DROPOUT_IMPL(BFloat16)

}  // namespace cuda
}  // namespace contrib {
}  // namespace onnxruntime
