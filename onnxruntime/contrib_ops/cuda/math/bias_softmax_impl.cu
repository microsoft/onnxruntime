#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "contrib_ops/cuda/math/bias_softmax_impl.h"

#include <limits>
#include <algorithm>

#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/cu_inc/binary_elementwise_impl.cuh"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/math/binary_elementwise_ops_impl_functors.cuh"
#include "core/providers/cuda/math/softmax_common.h"
#include "core/providers/cuda/math/softmax_warpwise_impl.cuh"
#include "core/providers/cuda/shared_inc/accumulation_type.h"

using namespace onnxruntime;
using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

// Duplicated softmax_impl.cu here
// So far attempt to use shared kernel with additional template resulted in lost performance

// Note: The intended case for 'input_bias' is the input sequence mask for transformer models
// As an additive mask, it should be zero for preserved tokens and -infty for tokens to screen
// The mask will broadcast from [batch_size, 1, 1, seq_len] to input [batch_size, num_heads, seq_len, seq_len]
// Here element_count = seq_len and bias_broadcast_size_per_batch = num_heads * seq_len

// The softmax + additive mask fusion follows NVIDIA apex's additive_masked_softmax_warp_forward
// see
// https://github.com/NVIDIA/apex/blob/4ef930c1c884fdca5f472ab2ce7cb9b505d26c1a/apex/contrib/csrc/multihead_attn/softmax.h

template <typename input_t, typename output_t, typename acc_t, int log2_elements, bool is_inner_broadcast>
__global__ void BiasSoftmaxWarpForward(output_t* output, const input_t* input, const input_t* input_bias,
                                       int element_count, int batch_count, fast_divmod bias_broadcast_fdm) {
  // "WARP" refers to cooperative threads and might not equal 32 threads of GPU warp
  // thread block is (WARP_SIZE, 128/WARP_SIZE)
  constexpr int next_power_of_two = 1 << log2_elements;
  constexpr int WARP_SIZE = next_power_of_two < GPU_WARP_SIZE ? next_power_of_two : GPU_WARP_SIZE;
  constexpr int WARP_ITERATIONS = next_power_of_two / WARP_SIZE;
#ifdef USE_ROCM
  constexpr int WARP_BATCH = 1;
#else
  constexpr int WARP_BATCH = (next_power_of_two <= 128) ? 2 : 1;
#endif

  // each "WARP" (<=32) processes WARP_BATCH(one of {1,2}) batches
  int first_batch = (blockDim.y * blockIdx.x + threadIdx.y) * WARP_BATCH;

  // last warp may have fewer batches
  int local_batches = batch_count - first_batch;
  if (local_batches > WARP_BATCH) local_batches = WARP_BATCH;

  // thread will process elements (local_index + n * warp_size) within batch
  int local_idx = threadIdx.x;

  // push input, input_bias output pointers to batch we need to process
  input += first_batch * element_count + local_idx;
  output += first_batch * element_count + local_idx;

  // load from global memory and apply bias (likely an additive mask)
  acc_t elements[WARP_BATCH][WARP_ITERATIONS];
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
    // If is_inner_broadcast, input shape is [x, broadcast_size, element_count], bias shape is [x, 1, element_count].
    // Otherwise, input shape is [x, broadcast_size, element_count], bias shape is [1, broadcast_size, element_count].
    int bias_batch_offset =
        is_inner_broadcast ? bias_broadcast_fdm.div(first_batch + i) : bias_broadcast_fdm.mod(first_batch + i);
    int bias_offset = bias_batch_offset * element_count + local_idx;
    int batch_element_count = (i >= local_batches) ? 0 : element_count;
#pragma unroll
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      int element_index = local_idx + it * WARP_SIZE;
      if (element_index < batch_element_count) {
        elements[i][it] =
            (acc_t)input[i * element_count + it * WARP_SIZE] + (acc_t)input_bias[bias_offset + it * WARP_SIZE];
      } else {
        elements[i][it] = -std::numeric_limits<acc_t>::infinity();
      }
    }
  }

  // find maximum value within batch for numerical stability
  acc_t max_value[WARP_BATCH];
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
    max_value[i] = elements[i][0];
#pragma unroll
    for (int it = 1; it < WARP_ITERATIONS; ++it) {
      max_value[i] = (max_value[i] > elements[i][it]) ? max_value[i] : elements[i][it];
    }
  }
  warp_reduce<acc_t, WARP_BATCH, WARP_SIZE, Max>(max_value);

  // normalization factor Z = Sum[ exp(element_i), for element_i in batch ]
  acc_t sum[WARP_BATCH]{acc_t(0.0)};
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
#pragma unroll
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      elements[i][it] = std::exp((acc_t)(elements[i][it] - max_value[i]));
      sum[i] += elements[i][it];
    }
  }
  warp_reduce<acc_t, WARP_BATCH, WARP_SIZE, Add>(sum);

// write back normalized value = exp(element_i)/Z to global memory
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
    if (i >= local_batches) break;
#pragma unroll
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      int element_index = local_idx + it * WARP_SIZE;
      if (element_index < element_count) {
        output[i * element_count + it * WARP_SIZE] = elements[i][it] / sum[i];
      } else {
        break;
      }
    }
  }
}

template <typename T>
Status BiasSoftmaxImpl(hipStream_t stream, hipdnnHandle_t cudnn_handle, T* output_data, const T* input_data,
                       const T* bias_data, int element_count, int batch_count, bool is_inner_broadcast,
                       int bias_broadcast_size) {
  if (element_count == 0) return Status::OK();
  if (element_count <= 1024 && element_count * static_cast<int>(sizeof(T)) <= 4096) {
    typedef AccumulationType_t<T> AccT;
    int log2_elements = log2_ceil(element_count);
    const int next_power_of_two = 1 << log2_elements;

    // This value must match the WARP_SIZE constexpr value computed inside softmax_warp_forward.
    int warp_size = std::min(next_power_of_two, GPU_WARP_SIZE_HOST);

    // This value must match the WARP_BATCH constexpr value computed inside softmax_warp_forward.
#ifdef USE_ROCM
    int batches_per_warp = 1;
    constexpr int threads_per_block = 256;
#else
    int batches_per_warp = (next_power_of_two <= 128) ? 2 : 1;
    constexpr int threads_per_block = 128;
#endif

    int warps_per_block = (threads_per_block / warp_size);
    int batches_per_block = warps_per_block * batches_per_warp;
    int blocks = (batch_count + batches_per_block - 1) / batches_per_block;
    dim3 threads(warp_size, warps_per_block, 1);

    fast_divmod bias_broadcast_fdm = fast_divmod(bias_broadcast_size);

    // Launch code would be more elegant if C++ supported FOR CONSTEXPR
    switch (log2_elements) {
#define LAUNCHE_BIAS_SOFTMAX_KERNEL(log2_elements_value, is_inner_broadcast_value)                                   \
  BiasSoftmaxWarpForward<T, T, AccT, log2_elements_value, is_inner_broadcast_value><<<blocks, threads, 0, stream>>>( \
      output_data, input_data, bias_data, element_count, batch_count, bias_broadcast_fdm)
#define CASE_LOG2_ELEMENTS(log2_elements_value)                \
  case log2_elements_value: {                                  \
    if (is_inner_broadcast) {                                  \
      LAUNCHE_BIAS_SOFTMAX_KERNEL(log2_elements_value, true);  \
    } else {                                                   \
      LAUNCHE_BIAS_SOFTMAX_KERNEL(log2_elements_value, false); \
    }                                                          \
  } break
      CASE_LOG2_ELEMENTS(0);   // 1
      CASE_LOG2_ELEMENTS(1);   // 2
      CASE_LOG2_ELEMENTS(2);   // 4
      CASE_LOG2_ELEMENTS(3);   // 8
      CASE_LOG2_ELEMENTS(4);   // 16
      CASE_LOG2_ELEMENTS(5);   // 32
      CASE_LOG2_ELEMENTS(6);   // 64
      CASE_LOG2_ELEMENTS(7);   // 128
      CASE_LOG2_ELEMENTS(8);   // 256
      CASE_LOG2_ELEMENTS(9);   // 512
      CASE_LOG2_ELEMENTS(10);  // 1024
#undef CASE_LOG2_ELEMENTS
#undef LAUNCHE_BIAS_SOFTMAX_KERNEL
    }
    return Status::OK();
  }

  // For large element count we fall back to explicit Add kernel + CUDA DNN library
  // note: This is an unhappy path! There is no performance benefit for the fusion.
  int output_rank_or_simple_broadcast = 3;
  TArray<int64_t> rhs_strides;
  TArray<fast_divmod> output_fdms;
  const TArray<int64_t>* p_rhs_strides = nullptr;
  const TArray<fast_divmod>* p_output_fdms = nullptr;
  fast_divmod fdm_h(1);
  fast_divmod fdm_c;
  if ((is_inner_broadcast && bias_broadcast_size == 1) || (!is_inner_broadcast && bias_broadcast_size == batch_count)) {
    // input and bias shape is same.
    output_rank_or_simple_broadcast = static_cast<int>(SimpleBroadcast::NoBroadcast);
  } else if (!is_inner_broadcast) {
    output_rank_or_simple_broadcast = static_cast<int>(SimpleBroadcast::RightPerChannelBatchN);
    fdm_c = fast_divmod(element_count * bias_broadcast_size);
  } else {
    rhs_strides.SetSize(3);
    rhs_strides[0] = static_cast<int64_t>(element_count);
    rhs_strides[1] = 0LL;
    rhs_strides[2] = 1LL;
    p_rhs_strides = &rhs_strides;
    output_fdms.SetSize(3);
    output_fdms[0] = fast_divmod(element_count * bias_broadcast_size);
    output_fdms[1] = fast_divmod(element_count);
    output_fdms[2] = fast_divmod(1);
    p_output_fdms = &output_fdms;
  }

  BinaryElementWiseImpl(stream, output_rank_or_simple_broadcast, nullptr, input_data, p_rhs_strides, bias_data,
                        p_output_fdms, fdm_h, fdm_c, output_data, OP_Add<T, T, T>(),
                        static_cast<size_t>(batch_count * element_count));

  // invoke cuda DNN library for Y = softmax(X)
  const int64_t dims[]{batch_count, 1, 1, element_count};
  const auto alpha = Consts<T>::One;
  const auto beta = Consts<T>::Zero;
  CudnnTensor input_tensor, output_tensor;
  ORT_RETURN_IF_ERROR(input_tensor.Set(dims, CudnnTensor::GetDataType<T>()));
  ORT_RETURN_IF_ERROR(output_tensor.Set(dims, CudnnTensor::GetDataType<T>()));
  return SoftmaxForward(cudnn_handle, &alpha, input_tensor, output_data, &beta, output_tensor, output_data);
}

#define SPECIALIZED_BIAS_SOFTMAX_IMPL(T)                                                                          \
  template Status BiasSoftmaxImpl<T>(hipStream_t stream, hipdnnHandle_t cudnn_handle, T * output_data,            \
                                     const T* input_data, const T* bias_data, int element_count, int batch_count, \
                                     bool is_inner_broadcast, int bias_broadcast_size);

// MIOpen doesn't support double so ROCm kernel doesn't have double support for now.
SPECIALIZED_BIAS_SOFTMAX_IMPL(float)
SPECIALIZED_BIAS_SOFTMAX_IMPL(half)
#ifdef USE_CUDA
SPECIALIZED_BIAS_SOFTMAX_IMPL(double)
#endif

#undef SPECIALIZED_BIAS_SOFTMAX_IMPL

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
