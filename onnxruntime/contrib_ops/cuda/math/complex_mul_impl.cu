#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "complex_mul.h"
#include "complex_mul_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/math/binary_elementwise_ops.h"

namespace onnxruntime {
namespace contrib {
namespace cuda {
template <typename T>
__device__ __inline__ void _ComplexMul(T a0, T a1, T b0, T b1, T* output_data, bool is_conj) {
  if (is_conj) {
    T out_real = a0 * b0 + a1 * b1;
    T out_imag = a1 * b0 - a0 * b1;
    output_data[0] = out_real;
    output_data[1] = out_imag;
  } else {
    T out_real = a0 * b0 - a1 * b1;
    T out_imag = a0 * b1 + a1 * b0;
    output_data[0] = out_real;
    output_data[1] = out_imag;
  }
};

// broadcast by computing output coordinate from offset, using fast_divmod
template <typename T, bool lhs_need_compute, bool rhs_need_compute, int NumThreadsPerBlock, int NumElementsPerThread>
__global__ void _ElementWiseWithStrideTwo(
    int32_t output_rank,
    const TArray<int64_t> lhs_padded_strides,
    const T* lhs_data,
    const TArray<int64_t> rhs_padded_strides,
    const T* rhs_data,
    const TArray<fast_divmod> fdm_output_strides,
    T* output_data,
    CUDA_LONG N,
    int64_t lhs_size,
    int64_t rhs_size,
    bool is_conj) {
  CUDA_LONG start = NumElementsPerThread * NumThreadsPerBlock * blockIdx.x + threadIdx.x;
  T a[NumElementsPerThread];
  T b[NumElementsPerThread];
  T c[NumElementsPerThread];
  T d[NumElementsPerThread];

  CUDA_LONG id = start;
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N / 2) {
      CUDA_LONG lhs_index = (lhs_need_compute ? 0 : id);
      CUDA_LONG rhs_index = (rhs_need_compute ? 0 : id);
      // compute indexes with broadcasting rules: https://github.com/onnx/onnx/blob/master/docs/Broadcasting.md
      CUDA_LONG offset = id;
#pragma unroll
      for (auto dim = 0; dim < fdm_output_strides.Capacity(); dim++) {
        if (dim >= output_rank) {
          break;
        }
        int q, r;
        fdm_output_strides[dim].divmod(offset, q, r);
        if (lhs_need_compute) {
          lhs_index += static_cast<int>(lhs_padded_strides[dim]) * q;
        }

        if (rhs_need_compute) {
          rhs_index += static_cast<int>(rhs_padded_strides[dim]) * q;
        }
        offset = r;
      }

      a[i] = lhs_data[(2 * lhs_index) % lhs_size];
      b[i] = lhs_data[(2 * lhs_index + 1) % lhs_size];
      c[i] = rhs_data[(2 * rhs_index) % rhs_size];
      d[i] = rhs_data[(2 * rhs_index + 1) % rhs_size];

      id += NumThreadsPerBlock;
    }
  }

  id = start;
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N / 2) {
      _ComplexMul(a[i], b[i], c[i], d[i], &output_data[2 * id], is_conj);
      id += NumThreadsPerBlock;
    }
  }
};

template <typename T>
void ComplexMul_Impl(
    hipStream_t stream,
    int32_t output_rank_or_simple_broadcast,
    const TArray<int64_t>* lhs_padded_strides,
    const T* lhs_data,
    const TArray<int64_t>* rhs_padded_strides,
    const T* rhs_data,
    const TArray<onnxruntime::cuda::fast_divmod>* fdm_output_strides,
    const onnxruntime::cuda::fast_divmod& fdm_H,
    const onnxruntime::cuda::fast_divmod& fdm_C,
    T* output_data,
    int64_t count,
    int64_t lhs_size,
    int64_t rhs_size,
    bool is_conj) {
  if (count == 0)  // special case where there's a dim value of 0 in the output shape
    return;

  int blocksPerGrid = static_cast<int>(CeilDiv(count, GridDim::maxThreadsPerBlock * GridDim::maxElementsPerThread));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);

  if (lhs_padded_strides && rhs_padded_strides && lhs_padded_strides->Size() && rhs_padded_strides->Size())
    _ElementWiseWithStrideTwo<T, true, true, GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
        output_rank_or_simple_broadcast,
        *lhs_padded_strides,
        lhs_data,
        *rhs_padded_strides,
        rhs_data,
        *fdm_output_strides,
        output_data,
        N,
        lhs_size,
        rhs_size,
        is_conj);
  else if (lhs_padded_strides && lhs_padded_strides->Size())
    _ElementWiseWithStrideTwo<T, true, false, GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
        output_rank_or_simple_broadcast,
        *lhs_padded_strides,
        lhs_data,
        *rhs_padded_strides,
        rhs_data,
        *fdm_output_strides,
        output_data,
        N,
        lhs_size,
        rhs_size,
        is_conj);
  else
    _ElementWiseWithStrideTwo<T, false, true, GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
        output_rank_or_simple_broadcast,
        *lhs_padded_strides,
        lhs_data,
        *rhs_padded_strides,
        rhs_data,
        *fdm_output_strides,
        output_data,
        N,
        lhs_size,
        rhs_size,
        is_conj);
};

#define SPECIALIZE_STACKEDCOMPLEXMUL_IMPL(T)                            \
  template void ComplexMul_Impl<T>(                                     \
      hipStream_t stream,                                              \
      int32_t output_rank_or_simple_broadcast,                          \
      const TArray<int64_t>* lhs_padded_strides,                        \
      const T* lhs_data,                                                \
      const TArray<int64_t>* rhs_padded_strides,                        \
      const T* rhs_data,                                                \
      const TArray<onnxruntime::cuda::fast_divmod>* fdm_output_strides, \
      const onnxruntime::cuda::fast_divmod& fdm_H,                      \
      const onnxruntime::cuda::fast_divmod& fdm_C,                      \
      T* output_data,                                                   \
      int64_t count,                                                    \
      int64_t lhs_size,                                                 \
      int64_t rhs_size,                                                 \
      bool is_conj);

SPECIALIZE_STACKEDCOMPLEXMUL_IMPL(float)
SPECIALIZE_STACKEDCOMPLEXMUL_IMPL(half)

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
