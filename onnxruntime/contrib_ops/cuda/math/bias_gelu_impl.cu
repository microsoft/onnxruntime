#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "contrib_ops/cuda/math/bias_gelu_impl.h"

#include "core/providers/cuda/cu_inc/common.cuh"

using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

namespace {

constexpr int kElementsPerThread = GridDim::maxElementsPerThread;
#ifdef USE_ROCM
constexpr int kThreadsPerBlock = 512;
#else
constexpr int kThreadsPerBlock = GridDim::maxThreadsPerBlock;
#endif

}  // namespace

template <typename T>
__global__ void BiasGeluKernel(int64_t bias_size, const T* X, const T* B, T* Y) {
  const auto kElementsPerBlock = kElementsPerThread * blockDim.x;
  const auto input_base_idx = bias_size * blockIdx.x + kElementsPerBlock * blockIdx.y + threadIdx.x;
  const auto bias_base_idx = kElementsPerBlock * blockIdx.y + threadIdx.x;
  const auto element_stride = blockDim.x;

  T reg_X[kElementsPerThread];
  T reg_B[kElementsPerThread];

  {
    auto input_idx = input_base_idx;
    auto bias_idx = bias_base_idx;
#pragma unroll
    for (int element_idx = 0; element_idx < kElementsPerThread; ++element_idx) {
      if (bias_idx < bias_size) {
        reg_X[element_idx] = X[input_idx];
        reg_B[element_idx] = B[bias_idx];
        input_idx += element_stride;
        bias_idx += element_stride;
      }
    }
  }

  {
    auto input_idx = input_base_idx;
    auto bias_idx = bias_base_idx;
#pragma unroll
    for (int element_idx = 0; element_idx < kElementsPerThread; ++element_idx) {
      if (bias_idx < bias_size) {
        Y[input_idx] = _Gelu(reg_X[element_idx] + reg_B[element_idx]);
        input_idx += element_stride;
        bias_idx += element_stride;
      }
    }
  }
}

template <typename T>
__global__ void VectorizedBiasGeluKernel(int64_t bias_size, const T* X, const T* B, T* Y) {
  const auto kElementsPerBlock = kElementsPerThread * blockDim.x;
  const auto bias_idx = kElementsPerBlock * blockIdx.y + kElementsPerThread * threadIdx.x;
  if (bias_idx >= bias_size) {
    return;
  }

  const auto input_idx = bias_size * blockIdx.x + kElementsPerBlock * blockIdx.y + kElementsPerThread * threadIdx.x;

  using LoadT = aligned_vector<T, kElementsPerThread>;

  T reg_X[kElementsPerThread];
  T reg_B[kElementsPerThread];
  T reg_Y[kElementsPerThread];

  LoadT* value_X = reinterpret_cast<LoadT*>(&reg_X);
  LoadT* value_B = reinterpret_cast<LoadT*>(&reg_B);
  *value_X = *reinterpret_cast<const LoadT*>(&X[input_idx]);
  *value_B = *reinterpret_cast<const LoadT*>(&B[bias_idx]);

#pragma unroll
  for (int element_idx = 0; element_idx < kElementsPerThread; ++element_idx) {
    reg_Y[element_idx] = _Gelu(reg_X[element_idx] + reg_B[element_idx]);
  }

  *(reinterpret_cast<LoadT*>(&Y[input_idx])) = *reinterpret_cast<LoadT*>(&reg_Y[0]);
}

template <typename T>
void LaunchBiasGeluKernel(hipStream_t stream, int64_t input_size, int64_t bias_size, const T* X, const T* B, T* Y) {
  // given a 2D grid of blocks:
  // each grid column handles bias_size elements
  // there are input_size / bias_size columns.
  int num_threads_per_block = std::min<int>(static_cast<int>(CeilDiv(bias_size, kElementsPerThread)), kThreadsPerBlock);
  const auto grid_width = CeilDiv(bias_size, kElementsPerThread * num_threads_per_block);
  const auto grid_height = input_size / bias_size;
  const dim3 grid_dim{static_cast<uint32_t>(grid_height), static_cast<uint32_t>(grid_width)};

  constexpr int vec_alignment = std::alignment_of<aligned_vector<T, kElementsPerThread>>::value;
  if (bias_size % kElementsPerThread == 0 && reinterpret_cast<uint64_t>(X) % vec_alignment == 0 &&
      reinterpret_cast<uint64_t>(B) % vec_alignment == 0 && reinterpret_cast<uint64_t>(Y) % vec_alignment == 0) {
    VectorizedBiasGeluKernel<T><<<grid_dim, num_threads_per_block, 0, stream>>>(bias_size, X, B, Y);
  } else {
    BiasGeluKernel<T><<<grid_dim, num_threads_per_block, 0, stream>>>(bias_size, X, B, Y);
  }
}

// explicit instantiations
#define SPECIALIZED_BIAS_GELU_IMPL(T)                                                                           \
  template void LaunchBiasGeluKernel<T>(hipStream_t stream, int64_t input_size, int64_t bias_size, const T* X, \
                                        const T* B, T* Y)

SPECIALIZED_BIAS_GELU_IMPL(half);
SPECIALIZED_BIAS_GELU_IMPL(float);
SPECIALIZED_BIAS_GELU_IMPL(double);
SPECIALIZED_BIAS_GELU_IMPL(BFloat16);

#undef SPECIALIZED_BIAS_GELU_IMPL

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
