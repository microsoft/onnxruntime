#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "image_scaler_impl.h"

using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

template <typename T, bool batch1>
__global__ void _ImageScalerKernel(
    const T* input_data,
    const float scale,
    const float* bias_data,
    const fast_divmod fdm_C,
    const fast_divmod fdm_HW,
    T* output_data,
    const size_t N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  int n, c;
  if (batch1)
    c = fdm_HW.div(id);
  else
    fdm_C.divmod(fdm_HW.div(id), n, c);
  output_data[id] = input_data[id] * (T)scale + (T)bias_data[c];
}

template <typename T>
void ImageScalerImpl(
    hipStream_t stream,
    const T* input_data,
    const float scale,
    const float* bias_data,
    const int64_t dims[4],  // NCHW
    T* output_data,
    const size_t N) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  fast_divmod fdm_HW((int)(dims[2] * dims[3]));
  fast_divmod fdm_C;
  if (dims[0] == 1) {
    _ImageScalerKernel<T, true><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
        input_data, scale, bias_data, fdm_C, fdm_HW, output_data, N);
  } else {
    fdm_C = fast_divmod((int)dims[1]);
    _ImageScalerKernel<T, false><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
        input_data, scale, bias_data, fdm_C, fdm_HW, output_data, N);
  }
}

#define SPECIALIZED_IMPL(T) \
  template void ImageScalerImpl<T>(hipStream_t stream, const T* input_data, const float scale, const float* bias_data, const int64_t dims[4], T* output_data, const size_t N);

SPECIALIZED_IMPL(float)
SPECIALIZED_IMPL(double)
SPECIALIZED_IMPL(half)

}  // namespace cuda
}  //namespace contrib
}  // namespace onnxruntime
