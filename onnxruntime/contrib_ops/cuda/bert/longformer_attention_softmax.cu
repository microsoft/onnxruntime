#include "hip/hip_runtime.h"
/*
Copyright (c) NVIDIA Corporation and Microsoft Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// This is fast cuda kernels for longformer attention softmax.
// It uses two temporary matrix of BxNxSxS, and consumes more memory when sequence length is large.

#include <hipcub/hipcub.hpp>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "longformer_attention_softmax.h"
#include "attention_impl.h"

using namespace onnxruntime::cuda;
using namespace hipcub;

#define CHECK(expr)         \
  if (!CUBLAS_CALL(expr)) { \
    return false;           \
  }

namespace onnxruntime {
namespace contrib {
namespace cuda {

template <typename T, int blockSize>
__launch_bounds__(blockSize)
    __global__ void LongformerSoftmaxFastKernel(const int* global_attention,
                                                const int* global_index,
                                                const int* batch_global_num,
                                                const T* input,
                                                const T* attention_mask,
                                                T* output,
                                                float scaler,
                                                int dim0,
                                                int sequence_length,
                                                int attention_window) {
  typedef hipcub::BlockReduce<float, blockSize> BlockReduce;
  __shared__ typename BlockReduce::TempStorage block_reduce_temp;
  __shared__ float max_shared;
  __shared__ float sum_shared;

  const T* input_block = input + sequence_length * blockIdx.x;
  T* output_block = output + sequence_length * blockIdx.x;
  const int batch_index = blockIdx.x / dim0;
  const int row_index = blockIdx.x % sequence_length;
  const int global_num = batch_global_num[batch_index];

  // To be consistent with Huggingface Longformer, the row of maksed word are set as zero.
  if ((float)attention_mask[batch_index * sequence_length + row_index] < 0.0f) {
    for (int i = threadIdx.x; i < sequence_length; i += blockSize) {
      output_block[i] = (T)(0);
    }
    return;
  }

  // local attention token
  int col_start = 0;
  int col_end = sequence_length;
  bool is_local_row = (global_attention[batch_index * sequence_length + row_index] == (int)0);
  if (is_local_row) {
    col_start = row_index - attention_window;
    if (col_start < 0) {
      col_start = 0;
    }

    col_end = row_index + attention_window + 1;
    if (col_end > sequence_length) {
      col_end = sequence_length;
    }
  }

  const T* mask_block = attention_mask + sequence_length * batch_index;
  int tid = threadIdx.x;

  // calculate max input
  float max_input = -HIP_INF_F;
  // #pragma unroll 16
  for (int i = tid + col_start; i < col_end; i += blockSize) {
    float x = input_block[i];
    x = x * scaler + (float)mask_block[i];
    if (max_input < x) {
      max_input = x;
    }
  }

  if (is_local_row) {
    for (int g = tid; g < global_num; g += blockSize) {
      int i = global_index[g];
      if (i < col_start || i >= col_end) {
        float x = input_block[i];
        x = x * scaler + (float)mask_block[i];
        if (max_input < x) {
          max_input = x;
        }
      }
    }
  }

  float max_block = BlockReduce(block_reduce_temp).Reduce(max_input, hipcub::Max());
  if (tid == 0) {
    max_shared = max_block;
  }
  __syncthreads();

  float sum_input = 0.f;
  // #pragma unroll 16
  for (int i = tid + col_start; i < col_end; i += blockSize) {
    float x = input_block[i];
    x = expf((x)*scaler + (float)mask_block[i] - max_shared);
    sum_input += x;
  }

  if (is_local_row) {
    for (int g = tid; g < global_num; g += blockSize) {
      int i = global_index[g];
      if (i < col_start || i >= col_end) {
        float x = input_block[i];
        x = expf((x)*scaler + (float)mask_block[i] - max_shared);
        sum_input += x;
      }
    }
  }

  float sum_block = BlockReduce(block_reduce_temp).Reduce(sum_input, hipcub::Sum());
  if (tid == 0) {
    sum_shared = sum_block;
  }
  __syncthreads();
  float recip_sum = 1.f / sum_shared;

  if (is_local_row) {
    // We only need to fill in zeros for blocks that will be used in the matrix multiplication
    // following the Softmax.
    //
    // For now zero-out only [row_index - 2*attention_window, row_index + 2*attention_window],
    // we can even be more agressive and reduce the zeroing out window size since
    // each row has entries in 3 blocks (3*attention_window size instead of 4*attention_window)
    int zero_start = row_index - 2 * attention_window;
    if (zero_start < 0) {
      zero_start = 0;
    }

    int zero_end = row_index + 2 * attention_window;
    if (zero_end > sequence_length) {
      zero_end = sequence_length;
    }

    for (int i = tid + zero_start; i < zero_end; i += blockSize) {
      if (i < col_start || i >= col_end) {
        output_block[i] = (T)(0.);
      }
    }
  }
  __syncthreads();

  if (is_local_row) {
    for (int g = tid; g < global_num; g += blockSize) {
      int i = global_index[g];
      if (i < col_start || i >= col_end) {
        float x = input_block[i];
        x = expf((x)*scaler + (float)mask_block[i] - max_shared);
        output_block[i] = (T)(recip_sum * x);
      }
    }
  }

  // #pragma unroll 16
  for (int i = tid + col_start; i < col_end; i += blockSize) {
    float x = input_block[i];
    x = expf((x)*scaler + (float)mask_block[i] - max_shared);
    output_block[i] = (T)(recip_sum * x);
  }
}

// Launch the softmax kernel for non compact memory.
bool launchSoftmaxFastKernel(
    hipStream_t stream,
    hipblasHandle_t cublas,
    void* workspace,              // softmax space
    const void* q,                // transposed Q with shape (B, N, S, H)
    const void* k,                // transposed K with shape (B, N, S, H)
    const void* v,                // transposed V with shape (B, N, S, H)
    const void* attention_mask,   // attention mask with shape (B, S), with value 0.0 not masked, and -10000.0 masked.
    const void* global_q,         // Q for global tokens with shape (B, N, S, H)
    const void* global_k,         // K for global tokens with shape (B, N, S, H)
    const void* global_v,         // V for global tokens with shape (B, N, S, H)
    const int* global_attention,  // global attention with shape (B, S), with value 0 for local attention and 1 for global attention.
    const int* global_index,      // Global index with shape (B, S)
    const int* batch_global_num,  // Number of global tokens per batch with shape (B, 1)
    void* pinned_buffer,          // Pinned memory in CPU. Number of global tokens per batch with shape (B, 1)
    void* output,                 // output with shape (B, N, S, H)
    float scaler,                 // scalar
    int batch_size,               // batch size
    int sequence_length,          // sequence length
    int num_heads,                // number of heads
    int head_size,                // hidden size per head
    int attention_window,         // one sided windows size
    size_t element_size) {        // size of element: 2 for half, and 4 for float

  bool is_fp16 = (element_size == 2);
  void* scratch1 = reinterpret_cast<char*>(workspace);
  void* scratch2 = reinterpret_cast<char*>(scratch1) + GetAttentionScratchSize(element_size, batch_size, num_heads, sequence_length, sequence_length);

  // setup shared parameters for two strided batched matrix multiplies
  hipDataType Atype;
  hipDataType Btype;
  hipDataType Ctype;
  hipDataType resultType;
  hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;

  __half one_fp16, zero_fp16;
  float one_fp32, zero_fp32;
  void *alpha, *beta_0, *beta_1;

  if (is_fp16) {
    one_fp16 = __float2half(1.f);
    zero_fp16 = __float2half(0.f);
    alpha = static_cast<void*>(&one_fp16);
    beta_0 = static_cast<void*>(&zero_fp16);
    beta_1 = static_cast<void*>(&one_fp16);
    Atype = HIP_R_16F;
    Btype = HIP_R_16F;
    Ctype = HIP_R_16F;
    resultType = HIP_R_16F;
    algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
  } else {
    one_fp32 = 1.f;
    zero_fp32 = 0.f;
    alpha = static_cast<void*>(&one_fp32);
    beta_0 = static_cast<void*>(&zero_fp32);
    beta_1 = static_cast<void*>(&one_fp32);
    Atype = HIP_R_32F;
    Btype = HIP_R_32F;
    Ctype = HIP_R_32F;
    resultType = HIP_R_32F;
  }

  // Strided batch matrix multiply
  //    qk = q * k^T
  // Shapes: q and k = B x N x S x H, qk = B x N x S x S
  // Convert col-major to row-major by swapping q and k in Gemm

  // Local attention part
  // S x S is calculated using sliding block WxW (W is one sided window size) like the following:
  //   [W][W]
  //   [W][W][W]
  //      [W][W][W]
  //         [W][W]
  // The first and last rows have 2 blocks, and the remaining has 3 blocks per row.
  // The calculation are splited into 3 parts. Firstly, fill the middle rows,  then the first row and finally the last row.
  // The results are stored in scratch1.

  int w = attention_window;
  int x_offset = num_heads * sequence_length * head_size;
  int y_offset = num_heads * sequence_length * sequence_length;
  int last_block = (sequence_length / w) - 1;
  int strideA = sequence_length * head_size;
  int strideB = sequence_length * head_size;
  int strideC = sequence_length * sequence_length;

  // When S == 2W, there is no middle rows of blocks:
  //   [W][W]
  //   [W][W]
  // We can use normal matrix multiplication in this case.
  if (sequence_length == 2 * w) {
    CHECK(hipblasGemmStridedBatchedEx(cublas,
                                     HIPBLAS_OP_T,
                                     HIPBLAS_OP_N,
                                     sequence_length,
                                     sequence_length,
                                     head_size,
                                     alpha,
                                     k,
                                     Atype,
                                     head_size,
                                     sequence_length * head_size,
                                     q,
                                     Btype,
                                     head_size,
                                     sequence_length * head_size,
                                     beta_0,
                                     scratch1,
                                     Ctype,
                                     sequence_length,
                                     sequence_length * sequence_length,
                                     batch_size * num_heads,
                                     resultType,
                                     algo));
  } else {  // sequence_length > 2 * w
    for (int i = 0; i < batch_size; ++i) {
      for (int j = 0; j < num_heads; ++j) {
        void* q_head = (char*)q + (i * x_offset + j * sequence_length * head_size + w * head_size) * element_size;
        void* k_head = (char*)k + (i * x_offset + j * sequence_length * head_size) * element_size;
        void* qk_head = (char*)scratch1 + (i * y_offset + j * sequence_length * sequence_length + w * sequence_length) * element_size;
        int count = (sequence_length - 2 * w) / w;
        CHECK(hipblasGemmStridedBatchedEx(cublas,
                                         HIPBLAS_OP_T,
                                         HIPBLAS_OP_N,
                                         3 * w,                    // m
                                         w,                        // n
                                         head_size,                // k
                                         alpha,                    // alpha
                                         k_head,                   // A
                                         Atype,                    // A type
                                         head_size,                // lda
                                         w * head_size,            // strideA
                                         q_head,                   // B
                                         Btype,                    // B type
                                         head_size,                // ldb
                                         w * head_size,            // strideB
                                         beta_0,                   // beta
                                         qk_head,                  // C
                                         Ctype,                    // C type
                                         sequence_length,          // ldc
                                         sequence_length * w + w,  // strideC
                                         count,                    // batch count
                                         resultType,
                                         algo));
      }
    }

    CHECK(hipblasGemmStridedBatchedEx(cublas,
                                     HIPBLAS_OP_T,
                                     HIPBLAS_OP_N,
                                     2 * w,                   // m
                                     w,                       // n
                                     head_size,               // k
                                     alpha,                   // alpha
                                     k,                       // A
                                     Atype,                   // A type
                                     head_size,               // lda
                                     strideA,                 // strideA
                                     q,                       // B
                                     Btype,                   // B type
                                     head_size,               // ldb
                                     strideB,                 // strideB
                                     beta_0,                  // beta
                                     scratch1,                // C
                                     Ctype,                   // C type
                                     sequence_length,         // ldc
                                     strideC,                 // strideC
                                     batch_size * num_heads,  // batch count
                                     resultType,
                                     algo));

    void* q_head = (char*)q + (last_block * w * head_size) * element_size;
    void* k_head = (char*)k + ((last_block - 1) * w * head_size) * element_size;
    void* qk_head = (char*)scratch1 + (last_block * w * sequence_length + (last_block - 1) * w) * element_size;
    CHECK(hipblasGemmStridedBatchedEx(cublas,
                                     HIPBLAS_OP_T,
                                     HIPBLAS_OP_N,
                                     2 * w,
                                     w,
                                     head_size,
                                     alpha,
                                     k_head,
                                     Atype,
                                     head_size,
                                     strideA,
                                     q_head,
                                     Btype,
                                     head_size,
                                     strideB,
                                     beta_0,
                                     qk_head,
                                     Ctype,
                                     sequence_length,
                                     strideC,
                                     batch_size * num_heads,
                                     resultType,
                                     algo));
  }

  const int* batch_global_count = reinterpret_cast<const int*>(pinned_buffer);
  // Global attention part
  for (int i = 0; i < batch_size; ++i) {
    if (batch_global_count[i] > 0) {
      void* q_batch = (char*)q + (i * x_offset) * element_size;
      void* k_batch = (char*)k + (i * x_offset) * element_size;
      void* qk_batch = (char*)scratch1 + (i * y_offset) * element_size;
      // Local tokens attending global tokens
      CHECK(hipblasGemmStridedBatchedEx(cublas,
                                       HIPBLAS_OP_T,
                                       HIPBLAS_OP_N,
                                       batch_global_count[i],
                                       sequence_length,
                                       head_size,
                                       alpha,
                                       k_batch,
                                       Atype,
                                       head_size,
                                       strideA,
                                       q_batch,
                                       Btype,
                                       head_size,
                                       strideB,
                                       beta_0,
                                       qk_batch,
                                       Ctype,
                                       sequence_length,
                                       strideC,
                                       num_heads,
                                       resultType,
                                       algo));

      void* global_q_batch = (char*)global_q + (i * num_heads * sequence_length * head_size) * element_size;
      void* global_k_batch = (char*)global_k + (i * x_offset) * element_size;
      int strideB_global = sequence_length * head_size;

      // Global tokens attending everything
      // This GEMMs need to be last to make sure all global token entries are re-written.
      CHECK(hipblasGemmStridedBatchedEx(cublas,
                                       HIPBLAS_OP_T,
                                       HIPBLAS_OP_N,
                                       sequence_length,
                                       batch_global_count[i],
                                       head_size,
                                       alpha,
                                       global_k_batch,
                                       Atype,
                                       head_size,
                                       strideA,
                                       global_q_batch,
                                       Btype,
                                       head_size,
                                       strideB_global,
                                       beta_0,
                                       qk_batch,
                                       Ctype,
                                       sequence_length,
                                       strideC,
                                       num_heads,
                                       resultType,
                                       algo));
    }
  }

  int dim0 = sequence_length * num_heads;
  int dim1 = sequence_length;
  void* softmax_out = scratch2;

  const int blockSize = 64;
  const int gridSize = batch_size * num_heads * sequence_length;
  if (is_fp16) {
    LongformerSoftmaxFastKernel<__half, blockSize><<<gridSize, blockSize, 0, stream>>>(
        global_attention,
        global_index,
        batch_global_num,
        static_cast<const __half*>(scratch1),
        static_cast<const __half*>(attention_mask),
        static_cast<__half*>(softmax_out), scaler, dim0, dim1, attention_window);
  } else {
    LongformerSoftmaxFastKernel<float, blockSize><<<gridSize, blockSize, 0, stream>>>(
        global_attention,
        global_index,
        batch_global_num,
        static_cast<const float*>(scratch1),
        static_cast<const float*>(attention_mask),
        static_cast<float*>(softmax_out), scaler, dim0, dim1, attention_window);
  }

  // Run the matrix multiply: output = softmax_out * v
  //   softmax_out: B x N x S x S
  //             v: B x N x S x H
  //      attn_out: B x N x S x H
  // Calculation uses full Gemm (S == 2W) or sliding blocks (S > 2W) in a way similar to local attention part.

  if (sequence_length == 2 * w) {
    // convert col-major to row-major by swapping softmax_out and v
    CHECK(hipblasGemmStridedBatchedEx(cublas,
                                     HIPBLAS_OP_N,
                                     HIPBLAS_OP_N,
                                     head_size,
                                     sequence_length,
                                     sequence_length,
                                     alpha,
                                     v,
                                     Atype,
                                     head_size,
                                     sequence_length * head_size,
                                     softmax_out,
                                     Btype,
                                     sequence_length,
                                     sequence_length * sequence_length,
                                     beta_0,
                                     output,
                                     Ctype,
                                     head_size,
                                     sequence_length * head_size,
                                     batch_size * num_heads,
                                     resultType,
                                     algo));
  } else {  // sequence_length > 2 * w
    for (int i = 0; i < batch_size; ++i) {
      for (int j = 0; j < num_heads; ++j) {
        void* v_head = (char*)v + (i * x_offset + j * head_size * sequence_length) * element_size;
        void* prob_head = (char*)softmax_out + (i * y_offset + j * sequence_length * sequence_length + w * sequence_length) * element_size;
        void* out_head = (char*)output + (i * x_offset + j * head_size * sequence_length + w * head_size) * element_size;
        int count = (sequence_length - 2 * w) / w;
        CHECK(hipblasGemmStridedBatchedEx(cublas,
                                         HIPBLAS_OP_N,
                                         HIPBLAS_OP_N,
                                         head_size,
                                         w,
                                         3 * w,
                                         alpha,
                                         v_head,
                                         Atype,
                                         head_size,
                                         w * head_size,
                                         prob_head,
                                         Btype,
                                         sequence_length,
                                         sequence_length * w + w,
                                         beta_0,
                                         out_head,
                                         Ctype,
                                         head_size,
                                         w * head_size,
                                         count,
                                         resultType,
                                         algo));
      }
    }

    CHECK(hipblasGemmStridedBatchedEx(cublas,
                                     HIPBLAS_OP_N,
                                     HIPBLAS_OP_N,
                                     head_size,
                                     w,
                                     2 * w,
                                     alpha,
                                     v,
                                     Atype,
                                     head_size,
                                     sequence_length * head_size,
                                     softmax_out,
                                     Btype,
                                     sequence_length,
                                     sequence_length * sequence_length,
                                     beta_0,
                                     output,
                                     Ctype,
                                     head_size,
                                     sequence_length * head_size,
                                     batch_size * num_heads,
                                     resultType,
                                     algo));

    void* v_head = (char*)v + (last_block - 1) * w * head_size * element_size;
    void* prob_head = (char*)softmax_out + (sequence_length * last_block * w + (last_block - 1) * w) * element_size;
    void* out_head = (char*)output + last_block * w * head_size * element_size;

    CHECK(hipblasGemmStridedBatchedEx(cublas,
                                     HIPBLAS_OP_N,
                                     HIPBLAS_OP_N,
                                     head_size,
                                     w,
                                     2 * w,
                                     alpha,
                                     v_head,
                                     Atype,
                                     head_size,
                                     sequence_length * head_size,
                                     prob_head,
                                     Btype,
                                     sequence_length,
                                     sequence_length * sequence_length,
                                     beta_0,
                                     out_head,
                                     Ctype,
                                     head_size,
                                     sequence_length * head_size,
                                     batch_size * num_heads,
                                     resultType,
                                     algo));
  }

  for (int i = 0; i < batch_size; ++i) {
    if (batch_global_count[i] > 0) {
      int glob_longdim_mm = (last_block - 1) * w;

      void* v_head = (char*)v + (i * x_offset) * element_size;
      void* prob_head = (char*)softmax_out + (i * y_offset + 2 * w * sequence_length) * element_size;
      void* out_head = (char*)output + (i * x_offset + 2 * w * head_size) * element_size;

      CHECK(hipblasGemmStridedBatchedEx(cublas,
                                       HIPBLAS_OP_N,
                                       HIPBLAS_OP_N,
                                       head_size,
                                       glob_longdim_mm,
                                       batch_global_count[i],
                                       alpha,
                                       v_head,
                                       Atype,
                                       head_size,
                                       sequence_length * head_size,
                                       prob_head,
                                       Btype,
                                       sequence_length,
                                       sequence_length * sequence_length,
                                       beta_1,
                                       out_head,
                                       Ctype,
                                       head_size,
                                       sequence_length * head_size,
                                       num_heads,
                                       resultType,
                                       algo));

      // Global tokens
      v_head = (char*)global_v + (i * x_offset) * element_size;
      prob_head = (char*)softmax_out + (i * y_offset) * element_size;
      out_head = (char*)output + (i * x_offset) * element_size;

      CHECK(hipblasGemmStridedBatchedEx(cublas,
                                       HIPBLAS_OP_N,
                                       HIPBLAS_OP_N,
                                       head_size,
                                       batch_global_count[i],
                                       sequence_length,  // Re-write entries completely
                                       alpha,
                                       v_head,
                                       Atype,
                                       head_size,
                                       sequence_length * head_size,
                                       prob_head,
                                       Btype,
                                       sequence_length,
                                       sequence_length * sequence_length,
                                       beta_0,    // Use beta=0 to overwrite
                                       out_head,  // Here assumes global tokens are at the beginning of sequence.
                                       Ctype,
                                       head_size,
                                       sequence_length * head_size,
                                       num_heads,
                                       resultType,
                                       algo));
    }
  }

  return true;
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
