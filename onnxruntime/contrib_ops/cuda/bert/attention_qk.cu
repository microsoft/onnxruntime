#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "contrib_ops/cuda/bert/attention_qk.h"

using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

__global__ void ConvertAndCopyQK(const int count, const float* input, half* output) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < count) {
    output[idx] = __float2half(input[idx]);
  }
}

__global__ void ConvertAndCopyQK(const int count, const half* input, float* output) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < count) {
    output[idx] = __half2float(input[idx]);
  }
}

template <typename T, typename QK>
Status CopyQK(hipStream_t stream,
              const int qk_size,
              const T* input,
              QK* output) {
  const bool half2float = std::is_same<T, half>::value && std::is_same<QK, float>::value;
  const bool float2half = std::is_same<T, float>::value && std::is_same<QK, half>::value;
  ORT_ENFORCE(half2float || float2half);

  int block_size = 256;
  int num_blocks = (qk_size + block_size - 1) / block_size;
  ConvertAndCopyQK<<<num_blocks, block_size, 0, stream>>>(qk_size, input, output);

  return CUDA_CALL(hipGetLastError());
}

template Status CopyQK<float, half>(hipStream_t stream,
                                    const int qk_size,
                                    const float* input,
                                    half* output);

template Status CopyQK<half, float>(hipStream_t stream,
                                    const int qk_size,
                                    const half* input,
                                    float* output);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
