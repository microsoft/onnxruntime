#include "hip/hip_runtime.h"
/*
Copyright (c) Microsoft Corporation.
Licensed under the MIT License.
*/

/*
Kernel implementation for blocking repeated n-grams.
*/

#include "core/providers/cuda/cu_inc/common.cuh"
#include "contrib_ops/cuda/bert/ngram_repeat_block_impl.h"

namespace onnxruntime {
namespace contrib {
namespace cuda {

using namespace onnxruntime::cuda;

// Ban repeated ngrams of length = 'no_repeat_ngram_size'
__global__ void banRepeatedTokens(const int64_t* __restrict__ tokens,
                                  float* __restrict__ lprobs,
                                  int max_predict_len, int vocab_size,
                                  int no_repeat_ngram_size) {
  auto row = blockIdx.x;
  auto col = threadIdx.x;
  auto start = row * (max_predict_len) + col;
  // Each thread compares ngram starting from
  // thread index with final ngram starting from
  // step - no_repeat_ngram_size +2
  auto check_start_pos = blockDim.x;
  auto lprob_start = row * vocab_size;
  bool is_banned = true;
  extern __shared__ int64_t tokens_shm[];
  tokens_shm[col] = tokens[start];
  if (col == blockDim.x - 1) {
    for (int i=1; i<no_repeat_ngram_size; i++){
      if (col+i < max_predict_len){
        tokens_shm[col + i] = tokens[start + i];
      }
    }
  }
  __syncthreads();

  for (int k = 0; k < no_repeat_ngram_size - 1; k++) {
    if (tokens_shm[col + k] != tokens_shm[check_start_pos + k]) {
      is_banned = false;
    }
  }
  if (is_banned == true) {
    auto token_to_be_banned = tokens_shm[col + no_repeat_ngram_size - 1];
    lprobs[lprob_start + token_to_be_banned] = -INFINITY;
  }
}

// Allocate blocks and threads based on
// batch size and sequence length and launch
// kernel
void NGramRepeatBlockImpl(
    hipStream_t stream,
    const int64_t* tokens_ptr,
    float* scores_ptr,
    int bsz,
    int step,
    int max_predict_len,
    int vocab_size,
    int beam_size,
    int no_repeat_ngram_size) {
  int threads = step - no_repeat_ngram_size + 2;
  if (threads <= 0) return;
  int blocks = bsz * beam_size;
  int shared_mem_size = (step + 1) * sizeof(int64_t);

  // Launching N blocks where N is number of samples in a batch (beams*bsz)
  // Launching T threads where T is number of previous ngrams in a sample
  // Allocating shared mem per block for fastser access of input tokens since
  // each token will be accessed N times to compare with current Ngram where
  // N is Ngram size.
  banRepeatedTokens<<<blocks, threads, shared_mem_size, stream>>>(
      tokens_ptr, scores_ptr, max_predict_len, vocab_size, no_repeat_ngram_size);
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
