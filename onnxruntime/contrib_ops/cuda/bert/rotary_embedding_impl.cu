#include "hip/hip_runtime.h"
/*
Copyright (c) Microsoft Corporation.
Licensed under the MIT License.
*/

/*
Kernel implementation for rotary embeddings.
*/

#include "contrib_ops/cuda/bert/rotary_embedding_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include <hip/hip_fp16.h>

using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

template <typename T>
__global__ void RotaryEmbeddingBSNH(T *output,                   // BxSxNxH
                                    const T *input,              // BxSxNxH
                                    const T *cos_cache,          // Mx(H/2)
                                    const T *sin_cache,          // Mx(H/2)
                                    const int64_t *position_ids, // (1) or BxS
                                    const int sequence_length, const int num_heads, const int head_size,
                                    const int rotary_embedding_dim, const int position_ids_format,
                                    const bool interleaved, const int batch_stride, const int seq_stride,
                                    const int head_stride) {
    // B = batch size, S = sequence length, N = num heads, H = head size, M = max sequence length
    // Use .x in innermost loop to access global memory efficiently

    const int b = blockIdx.y;
    const int s = blockIdx.x;
    const int n = blockIdx.z;

    const int i = threadIdx.x;

    if (i >= head_size) {
        return;
    }

    const int block_offset = b * batch_stride + s * seq_stride + n * head_stride;

    const T *input_data = input + block_offset;
    T *output_data = output + block_offset;

    if (i >= rotary_embedding_dim) {
        output_data[i] = input_data[i];
        return;
    }

    // Cache is (M, H/2)
    const int half_rotary_embedding_dim = rotary_embedding_dim / 2;
    const int position_id = (position_ids_format == 0) ? static_cast<int>(position_ids[0]) + s
                                                       : static_cast<int>(position_ids[b * sequence_length + s]);
    const int cache_offset = position_id * half_rotary_embedding_dim;
    const T *cos_data = cos_cache + cache_offset;
    const T *sin_data = sin_cache + cache_offset;

    int cache_idx = 0;
    T sign = 0;
    int j = 0;
    if (interleaved) {
        cache_idx = (i / 2) % half_rotary_embedding_dim;
        sign = (i % 2 == 0) ? -1 : 1;
        j = (i % 2 == 0) ? i + 1 : i - 1; // i - sign
    } else {
        cache_idx = i % half_rotary_embedding_dim;
        sign = (i < half_rotary_embedding_dim) ? -1 : 1;
        j = (i + half_rotary_embedding_dim) % rotary_embedding_dim;
    }
    output_data[i] = input_data[i] * cos_data[cache_idx] + sign * input_data[j] * sin_data[cache_idx];
}

template <typename T>
Status LaunchRotaryEmbeddingKernel(hipStream_t stream, T *output, const T *input, const int64_t *position_ids,
                                   const T *cos_cache, const T *sin_cache, const int batch_size,
                                   const int sequence_length, const int num_heads, const int head_size,
                                   const int rotary_embedding_dim, const int /*max_sequence_length*/,
                                   const int position_ids_format, const bool interleaved,
                                   const int max_threads_per_block, const bool transposed) {
    // Note: Current implementation assumes head_size <= max_threads_per_block
    // because head_size is currently large for LLaMA-2. For smaller head_size
    // and num_heads values, we can create a block as `block(num_heads, head_size, 1)`
    // instead. This will require kernel changes to support.
    ORT_ENFORCE(head_size <= max_threads_per_block, "Rotary embedding dim must be <= max_threads_per_block");

    int tpb = (head_size + 31) / 32 * 32;

    const dim3 block(tpb);
    const dim3 grid(sequence_length, batch_size, num_heads);

    // Default input tensor shape is [batch, seq, hidden_size]
    int head_stride = head_size;
    int seq_stride = num_heads * head_stride;
    int batch_stride = sequence_length * seq_stride;
    if (transposed) {
        // When transposed, input tensor shape is [batch, num_heads, seq, head_size]
        seq_stride = head_size;
        head_stride = sequence_length * seq_stride;
        batch_stride = num_heads * head_stride;
    }

    assert(head_size <= max_threads_per_block);
    RotaryEmbeddingBSNH<<<grid, block, 0, stream>>>(output, input, cos_cache, sin_cache, position_ids, sequence_length,
                                                    num_heads, head_size, rotary_embedding_dim, position_ids_format,
                                                    interleaved, batch_stride, seq_stride, head_stride);

    return CUDA_CALL(hipGetLastError());
}

template Status LaunchRotaryEmbeddingKernel<float>(hipStream_t stream, float *output, const float *input,
                                                   const int64_t *position_ids, const float *cos_cache,
                                                   const float *sin_cache, const int batch_size,
                                                   const int sequence_length, const int num_heads, const int head_size,
                                                   const int rotary_embedding_dim, const int max_sequence_length,
                                                   const int position_ids_format, const bool interleaved,
                                                   const int max_threads_per_block, const bool transposed);

template Status LaunchRotaryEmbeddingKernel<half>(hipStream_t stream, half *output, const half *input,
                                                  const int64_t *position_ids, const half *cos_cache,
                                                  const half *sin_cache, const int batch_size,
                                                  const int sequence_length, const int num_heads, const int head_size,
                                                  const int rotary_embedding_dim, const int max_sequence_length,
                                                  const int position_ids_format, const bool interleaved,
                                                  const int max_threads_per_block, const bool transposed);

template Status LaunchRotaryEmbeddingKernel<BFloat16>(
    hipStream_t stream, BFloat16 *output, const BFloat16 *input, const int64_t *position_ids,
    const BFloat16 *cos_cache, const BFloat16 *sin_cache, const int batch_size, const int sequence_length,
    const int num_heads, const int head_size, const int rotary_embedding_dim, const int max_sequence_length,
    const int position_ids_format, const bool interleaved, const int max_threads_per_block, const bool transposed);

} // namespace cuda
} // namespace contrib
} // namespace onnxruntime
