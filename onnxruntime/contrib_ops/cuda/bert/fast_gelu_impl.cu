#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on gelu plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/
 
Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications: Add (bias) before Gelu is merged into this op to get better performance.
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/shared_inc/cuda_call.h"
#include "contrib_ops/cuda/bert/fast_gelu_impl.h"

using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

// constants for approximating the normal cdf
constexpr float A = 0.5f;

constexpr float B = 0.7978845608028654f;  // sqrt(2.0/M_PI)

constexpr float C = 0.035677408136300125f;  // 0.044715 * sqrt(2.0/M_PI)

template <typename T, unsigned TPB>
__global__ void FastGeluKernel(const T a, const T b, const T c, int input_length, int bias_length, const T* input, const T* bias, T* output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;

  if (idx < input_length) {
    const T x = input[idx];
    const T in = (bias == nullptr) ? x : (T)(x + bias[idx % bias_length]);
    const T cdf = a + a * _Tanh(in * (c * in * in + b));
    output[idx] = in * cdf;
  }
}

template <unsigned TPB>
__global__ void FastGeluKernel2(const half2 a, const half2 b, const half2 c, int input_length, int bias_length, const half2* input, const half2* bias, half2* output) {
// half2 arithmetic functions requires cuda architecture >= 5.3
#if __CUDA_ARCH__ >= 530
  const int idx = blockIdx.x * TPB + threadIdx.x;

  if (idx < input_length) {
    const half2 x = input[idx];
    const half2 in = (bias == nullptr) ? x : (x + bias[idx % bias_length]);
    const half2 cdf = a + a * _Tanh(in * (c * in * in + b));
    output[idx] = in * cdf;
  }
#endif
}

template <>
bool LaunchFastGeluKernel(const hipDeviceProp_t& prop, hipStream_t stream, int input_length, int bias_length, const float* input, const float* bias, float* output) {
  constexpr int blockSize = 256;
  const int gridSize = (input_length + blockSize - 1) / blockSize;
  FastGeluKernel<float, blockSize><<<gridSize, blockSize, 0, stream>>>(A, B, C, input_length, bias_length, input, bias, output);

  return CUDA_CALL(hipPeekAtLastError());
}

template <>
bool LaunchFastGeluKernel(const hipDeviceProp_t& prop, hipStream_t stream, int input_length, int bias_length, const half* input, const half* bias, half* output) {
  constexpr int blockSize = 256;

  if (0 == (bias_length & 1) && prop.major >= 7) {
    const int n = input_length / 2;
    const int gridSize = (n + blockSize - 1) / blockSize;
    const half2 A2 = __floats2half2_rn(A, A);
    const half2 B2 = __floats2half2_rn(B, B);
    const half2 C2 = __floats2half2_rn(C, C);
    const half2* input2 = reinterpret_cast<const half2*>(input);
    const half2* bias2 = reinterpret_cast<const half2*>(bias);
    half2* output2 = reinterpret_cast<half2*>(output);
    FastGeluKernel2<blockSize><<<gridSize, blockSize, 0, stream>>>(A2, B2, C2, n, bias_length / 2, input2, bias2, output2);
  } else {
    const int gridSize = (input_length + blockSize - 1) / blockSize;
    FastGeluKernel<half, blockSize><<<gridSize, blockSize, 0, stream>>>(A, B, C, input_length, bias_length, input, bias, output);
  }

  return CUDA_CALL(hipPeekAtLastError());
}

#if CUDA_VERSION >= 11000 && (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
template <unsigned TPB>
__global__ void FastGeluKernel2(const nv_bfloat162 a, const nv_bfloat162 b, const nv_bfloat162 c,
                                int input_length, int bias_length,
                                const nv_bfloat162* input, const nv_bfloat162* bias, nv_bfloat162* output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;

  if (idx < input_length) {
    const nv_bfloat162 x = input[idx];
    const nv_bfloat162 in = (bias == nullptr) ? x : (x + bias[idx % bias_length]);
    const nv_bfloat162 cdf = a + a * _Tanh(in * (c * in * in + b));
    output[idx] = in * cdf;
  }
}

template <>
bool LaunchFastGeluKernel(const hipDeviceProp_t& prop, hipStream_t stream, int input_length, int bias_length, const hip_bfloat16* input, const hip_bfloat16* bias, hip_bfloat16* output) {
  constexpr int blockSize = 256;

  if (0 == (bias_length & 1) && prop.major >= 7) {
    const int n = input_length / 2;
    const int gridSize = (n + blockSize - 1) / blockSize;
    const nv_bfloat162 A2 = __floats2bfloat162_rn(A, A);
    const nv_bfloat162 B2 = __floats2bfloat162_rn(B, B);
    const nv_bfloat162 C2 = __floats2bfloat162_rn(C, C);
    const nv_bfloat162* input2 = reinterpret_cast<const nv_bfloat162*>(input);
    const nv_bfloat162* bias2 = reinterpret_cast<const nv_bfloat162*>(bias);
    nv_bfloat162* output2 = reinterpret_cast<nv_bfloat162*>(output);
    FastGeluKernel2<blockSize><<<gridSize, blockSize, 0, stream>>>(A2, B2, C2, n, bias_length / 2, input2, bias2, output2);
  } else {
    const int gridSize = (input_length + blockSize - 1) / blockSize;
    FastGeluKernel<hip_bfloat16, blockSize><<<gridSize, blockSize, 0, stream>>>(A, B, C, input_length, bias_length, input, bias, output);
  }

  return CUDA_CALL(hipPeekAtLastError());
}
#endif

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
