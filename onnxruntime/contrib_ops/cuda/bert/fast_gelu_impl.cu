#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on gelu plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/
 
Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications: Add (bias) before Gelu is merged into this op to get better performance.
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/shared_inc/cuda_call.h"
#include "contrib_ops/cuda/bert/fast_gelu_impl.h"

using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

// constants for approximating the normal cdf
constexpr float A = 0.5;

constexpr float B = 0.7978845608028654;  // sqrt(2.0/M_PI)

constexpr float C = 0.035677408136300125;  // 0.044715 * sqrt(2.0/M_PI)

template <typename T, unsigned TPB>
__global__ void FastGeluKernel(const T a, const T b, const T c, int input_length, int bias_length, const T* input, const T* bias, T* output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;

  if (idx < input_length) {
    const T x = input[idx];
    const T in = (bias == nullptr) ? x : (x + bias[idx % bias_length]);
    const T cdf = a + a * _Tanh(in * (c * in * in + b));
    output[idx] = in * cdf;
  }
}

template <unsigned TPB>
__global__ void FastGeluKernel2(const half2 a, const half2 b, const half2 c, int input_length, int bias_length, const half2* input, const half2* bias, half2* output) {
// half2 arithmetic functions requires cuda architecture >= 5.3
#if __CUDA_ARCH__ >= 530
  const int idx = blockIdx.x * TPB + threadIdx.x;

  if (idx < input_length) {
    const half2 x = input[idx];
    const half2 in = (bias == nullptr) ? x : (x + bias[idx % bias_length]);
    const half2 cdf = a + a * _Tanh(in * (c * in * in + b));
    output[idx] = in * cdf;
  }
#endif
}

template <>
bool LaunchFastGeluKernel(const hipDeviceProp_t& prop, hipStream_t stream, int input_length, int bias_length, const float* input, const float* bias, float* output) {
  constexpr int blockSize = 256;
  const int gridSize = (input_length + blockSize - 1) / blockSize;
  FastGeluKernel<float, blockSize><<<gridSize, blockSize, 0, stream>>>(A, B, C, input_length, bias_length, input, bias, output);

  return CUDA_CALL(hipPeekAtLastError());
}

template <>
bool LaunchFastGeluKernel(const hipDeviceProp_t& prop, hipStream_t stream, int input_length, int bias_length, const half* input, const half* bias, half* output) {
  constexpr int blockSize = 256;

  if (0 == (bias_length & 1) && prop.major >= 7) {
    const int n = input_length / 2;
    const int gridSize = (n + blockSize - 1) / blockSize;
    const half2 A2 = __floats2half2_rn(A, A);
    const half2 B2 = __floats2half2_rn(B, B);
    const half2 C2 = __floats2half2_rn(C, C);
    const half2* input2 = reinterpret_cast<const half2*>(input);
    const half2* bias2 = reinterpret_cast<const half2*>(bias);
    half2* output2 = reinterpret_cast<half2*>(output);
    FastGeluKernel2<blockSize><<<gridSize, blockSize, 0, stream>>>(A2, B2, C2, n, bias_length / 2, input2, bias2, output2);
  } else {
    const int gridSize = (input_length + blockSize - 1) / blockSize;
    FastGeluKernel<half, blockSize><<<gridSize, blockSize, 0, stream>>>(A, B, C, input_length, bias_length, input, bias, output);
  }

  return CUDA_CALL(hipPeekAtLastError());
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
