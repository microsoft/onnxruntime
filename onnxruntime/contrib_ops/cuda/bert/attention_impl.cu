#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on qkvToContext plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/

Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications: scaling is moved from masked softmax to the gemm before that.
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_fp16.h>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "attention_impl.h"
#include "attention_softmax.h"

using namespace onnxruntime::cuda;
using namespace hipcub;

namespace onnxruntime {
namespace contrib {
namespace cuda {

static size_t AlignTo(size_t a, size_t b) {
  return CeilDiv(a, b) * b;
}

size_t GetAttentionScratchSize(size_t element_size, int batch_size, int num_heads, int sequence_length, int all_sequence_length) {
  const size_t len = batch_size * num_heads * sequence_length * all_sequence_length;
  const size_t bytes = len * element_size;

  const size_t alignment = 256;
  const size_t bytesAligned = AlignTo(bytes, alignment);
  return bytesAligned;
}

size_t GetAttentionWorkspaceSize(
    size_t element_size,
    int batch_size,
    int num_heads,
    int head_size,
    int sequence_length,
    int past_sequence_length) {
  size_t qkv_size = 3 * batch_size * sequence_length * num_heads * head_size * element_size;
  return qkv_size + 2 * GetAttentionScratchSize(element_size, batch_size, num_heads, sequence_length, past_sequence_length + sequence_length);
}

template <typename T>
bool QkvToContext(
    const hipDeviceProp_t& prop, hipblasHandle_t& cublas, hipStream_t stream,
    const int batch_size, const int sequence_length, const int num_heads, const int head_size, const size_t element_size,
    const T* input, T* output, T* workspace,
    const int* mask_index, const std::vector<int64_t>* mask_index_dims,
    bool is_unidirectional, int past_sequence_length, const T* past, T* present) {
  const int all_sequence_length = past_sequence_length + sequence_length;
  const size_t bytes = GetAttentionScratchSize(element_size, batch_size, num_heads, sequence_length, all_sequence_length);
  T* scratch1 = workspace;
  T* scratch2 = scratch1 + (bytes / element_size);
  T* scratch3 = scratch2 + (bytes / element_size);

  // input should be BxSx3xNxH => scratch3: 3xBxNxSxH
  if (!LaunchTransQkv(stream, sequence_length, batch_size, head_size, num_heads, input, scratch3)) {
    return false;
  }

  // now scratch3 has Q, K, V: each has size BxNxSxH
  const int batches = batch_size * num_heads;
  const int size_per_batch = sequence_length * head_size;
  const int total_size = batches * size_per_batch;

  const T* q = scratch3;
  const T* k = q + total_size;
  const T* v = k + total_size;

  hipblasSetStream(cublas, stream);
  CublasMathModeSetter helper(prop, cublas, HIPBLAS_TENSOR_OP_MATH);

  // Concat past (2xBxNxS'xH) to present (2xBxNxS*xH):
  // past_k (BxNxS'xH) + k (BxNxSxH) => present_k (BxNxS*xH)
  // past_v (BxNxS'xH) + v (BxNxSxH) => present_v (BxNxS*xH)
  const int present_size_per_batch = all_sequence_length * head_size;
  if (nullptr != present) {
    if (!LaunchConcatPastToPresent(stream, all_sequence_length, sequence_length, batch_size, head_size, num_heads, past, k, present)) {
      return false;
    }

    // update pointers to present_k and present_v.
    k = present;
    v = present + batches * present_size_per_batch;
  }

  // Raw attention mask could be 2D (BxS) or 3D (BxSxS*)
  bool use_raw_attention_mask = (nullptr != mask_index && nullptr != mask_index_dims && mask_index_dims->size() >= 2);

  // compute Q*K' (as K'*Q), scaled by 1/sqrt(H) and store in scratch1: BxNxSxS*
  // Q: BxNxSxH, K (present_k): BxNxS*xH, Q*K': BxNxSxS*
  const float rsqrt_head_size = 1.f / sqrt(static_cast<float>(head_size));
  const int temp_matrix_size = sequence_length * all_sequence_length;
  // For raw attention mask, the scalar if 1/sqrt(H) is moved to softmax computation.
  T alpha = (T)(use_raw_attention_mask ? 1.0f : rsqrt_head_size);
  if (!CUBLAS_CALL(CublasGemmStridedBatched(
          cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, all_sequence_length, sequence_length, head_size, alpha, k, head_size, present_size_per_batch,
          q, head_size, size_per_batch, 0.f, scratch1, all_sequence_length, temp_matrix_size, batches))) {
    return false;
  }

  // apply softmax and store result P to scratch2: BxNxSxS*
  if (use_raw_attention_mask) {  // 2d or 3d attention mask
    if (!ComputeSoftmaxWithRawMask<T>(stream, all_sequence_length, sequence_length, batch_size, num_heads, mask_index, scratch1, scratch2, is_unidirectional, rsqrt_head_size, static_cast<int>(mask_index_dims->size()))) {
      return false;
    }
  } else if (nullptr != mask_index) {  // 1d mask index
    ORT_ENFORCE(nullptr != mask_index_dims && mask_index_dims->size() == 1);
    // mask_index has 1D shape: either (batch_size) or (2*batch_size). Only the later one has start postions.
    const int* mask_start = (mask_index_dims->at(0) > batch_size) ? mask_index + batch_size : nullptr;
    if (!ComputeSoftmaxWithMask1D<T>(stream, all_sequence_length, sequence_length, batch_size, num_heads, mask_index, mask_start, scratch1, scratch2, is_unidirectional)) {
      return false;
    }
  } else {  // no mask
    if (!ComputeSoftmax<T>(stream, all_sequence_length, sequence_length, batch_size, num_heads, scratch1, scratch2, is_unidirectional)) {
      return false;
    }
  }

  // compute P*V (as V*P), and store in scratch3: BxNxSxH
  if (!CUBLAS_CALL(CublasGemmStridedBatched(
          cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, head_size, sequence_length, all_sequence_length, 1.f, v, head_size, present_size_per_batch,
          scratch2, all_sequence_length, temp_matrix_size, 0.f, scratch3, head_size, size_per_batch, batches))) {
    return false;
  }

  // scratch3 is BxNxSxH, transpose to output BxSxNxH
  return LaunchTransCtx(stream, sequence_length, batch_size, head_size, num_heads, scratch3, output);
}

bool LaunchAttentionKernel(
    const hipDeviceProp_t& prop,
    hipStream_t stream,
    const void* input,
    const int* mask_index,
    const std::vector<int64_t>* mask_index_dims,
    void* output,
    const int batch_size,
    const int sequence_length,
    const int num_heads,
    const int head_size,
    void* workspace,
    hipblasHandle_t& cublas,
    const size_t element_size,
    bool is_unidirectional,
    int past_sequence_length,
    const void* past,
    void* present) {
  if (element_size == 2) {
    return QkvToContext(prop, cublas, stream,
                        batch_size, sequence_length, num_heads, head_size, element_size,
                        reinterpret_cast<const half*>(input), reinterpret_cast<half*>(output), reinterpret_cast<half*>(workspace),
                        mask_index, mask_index_dims, is_unidirectional,
                        past_sequence_length, reinterpret_cast<const half*>(past), reinterpret_cast<half*>(present));
  } else {
    return QkvToContext(prop, cublas, stream,
                        batch_size, sequence_length, num_heads, head_size, element_size,
                        reinterpret_cast<const float*>(input), reinterpret_cast<float*>(output), reinterpret_cast<float*>(workspace),
                        mask_index, mask_index_dims, is_unidirectional,
                        past_sequence_length, reinterpret_cast<const float*>(past), reinterpret_cast<float*>(present));
  }
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
