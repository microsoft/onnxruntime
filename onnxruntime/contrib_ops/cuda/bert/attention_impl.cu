#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on qkvToContext plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/

Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications: scaling is moved from masked softmax to the gemm before that.
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hipcub/hipcub.hpp>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_math_constants.h>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "attention_impl.h"

using namespace onnxruntime::cuda;
using namespace hipcub;

namespace onnxruntime {
namespace contrib {
namespace cuda {

static size_t AlignTo(size_t a, size_t b) {
  return CeilDiv(a, b) * b;
}

size_t ScratchSize(size_t element_size, int batch_size, int num_heads, int sequence_length, int all_sequence_length) {
  const size_t len = batch_size * num_heads * sequence_length * all_sequence_length;
  const size_t bytes = len * element_size;

  const size_t alignment = 256;
  const size_t bytesAligned = AlignTo(bytes, alignment);
  return bytesAligned;
}

size_t GetAttentionWorkspaceSize(
    size_t element_size,
    int batch_size,
    int num_heads,
    int head_size,
    int sequence_length,
    int past_sequence_length) {
  size_t qkv_size = 3 * batch_size * sequence_length * num_heads * head_size * element_size;
  return qkv_size + 2 * ScratchSize(element_size, batch_size, num_heads, sequence_length, past_sequence_length + sequence_length);
}

template <typename T, unsigned TPB>
__device__ inline void Softmax(const int all_sequence_length,
                               const int sequence_length,
                               const int valid_end,
                               const int valid_start,
                               const T* input,
                               T* output) {
  using BlockReduce = hipcub::BlockReduce<float, TPB>;
  __shared__ typename BlockReduce::TempStorage tmp_storage;

  __shared__ float sum_reverse_block;
  __shared__ float max_block;

  float thread_data_max(-HIP_INF_F);

  // e^x is represented as infinity if x is large enough, like 100.f.
  // Infinity divided by Infinity is a NAN. Thus, softmax gets a NAN if one or more item are large enough.
  // a math transform as below is leveraged to get a stable softmax:
  // e^xi/(e^x1 + ...e^xn) = e^(xi - max) / (e^(x1 - max) + ... + e^(xn - max))
  const int offset = (blockIdx.y * gridDim.x + blockIdx.x) * all_sequence_length;
  for (int i = threadIdx.x; i < valid_end; i += TPB) {
    if (i >= valid_start) {
      const int index = offset + i;
      if (thread_data_max < float(input[index])) {
        thread_data_max = float(input[index]);
      }
    }
  }

  const auto max = BlockReduce(tmp_storage).Reduce(thread_data_max, hipcub::Max());

  // Store max value
  if (threadIdx.x == 0) {
    max_block = max;
  }
  __syncthreads();

  float thread_data_sum(0.f);
  for (int i = threadIdx.x; i < valid_end; i += TPB) {
    if (i >= valid_start) {
      const int index = offset + i;
      const float val = input[index];
      thread_data_sum += expf(val - max_block);
    }
  }

  const auto sum = BlockReduce(tmp_storage).Reduce(thread_data_sum, hipcub::Sum());
  if (threadIdx.x == 0) {
    sum_reverse_block = 1.f / sum;
  }
  __syncthreads();

  for (int i = threadIdx.x; i < all_sequence_length; i += TPB) {
    const int index = offset + i;
    const float val = (i >= valid_start && i < valid_end) ? expf(float(input[index]) - max_block) * sum_reverse_block : 0.f;
    output[index] = T(val);
  }
}

template <typename T, unsigned TPB>
__device__ inline void SoftmaxSmall(const int all_sequence_length,
                                    const int sequence_length,
                                    const int valid_end,
                                    const int valid_start,
                                    const T* input,
                                    T* output,
                                    bool is_unidirectional) {
  using BlockReduce = hipcub::BlockReduce<float, TPB>;
  __shared__ typename BlockReduce::TempStorage tmp_storage;

  __shared__ float sum_reverse_block;
  __shared__ float max_block;

  // Input dimension is BxNxSxS*; blockIdx.y is batch index b; gridDim.x=N*S;  blockIdx.x is index within N*S;
  const int offset = (blockIdx.y * gridDim.x + blockIdx.x) * all_sequence_length;
  const int index = offset + threadIdx.x;

  bool is_valid = false;  // whether it has attention mask == 1.

  // Update end position for unidirectional.
  int end = valid_end;
  if (is_unidirectional) {
    int end_unid = all_sequence_length - sequence_length + (blockIdx.x % sequence_length) + 1;
    if (end_unid <= valid_start) {
      // In this situation, mask of [0, end_unid) and [valid_start, valid_end) has -10000, and [end_unid, valid_start) and [valid_end, all_seq_len) has -20000.
      // So [0, end_unid) will also have value after softmax.
      is_valid = threadIdx.x < end_unid;
    } else {
      end = min(valid_end, end_unid);
    }
  }

  is_valid = is_valid || (threadIdx.x >= valid_start && threadIdx.x < end);

  // e^x is represented as infinity if x is large enough, like 100.f.
  // Infinity divided by Infinity is a NAN. Thus, softmax gets a NAN if one or more item are large enough.
  // a math transform as below is leveraged to get a stable softmax:
  // e^xi/(e^x1 + ...e^xn) = e^(xi - max) / (e^(x1 - max) + ... + e^(xn - max))
  float thread_data_max = is_valid ? float(input[index]) : float(-HIP_INF_F);
  const auto max = BlockReduce(tmp_storage).Reduce(thread_data_max, hipcub::Max(), end);

  // Store max value
  if (threadIdx.x == 0) {
    max_block = max;
  }
  __syncthreads();

  float thread_data_exp(0.f);
  if (is_valid) {
    thread_data_exp = expf(float(input[index]) - max_block);
  }

  const auto sum = BlockReduce(tmp_storage).Reduce(thread_data_exp, hipcub::Sum(), end);

  // Store value of 1.0/sum.
  if (threadIdx.x == 0) {
    sum_reverse_block = (1.f) / sum;
  }
  __syncthreads();

  // threadIdx.x might be larger than all_sequence_length due to alignment to 32x.
  if (threadIdx.x < all_sequence_length) {
    output[index] = T(thread_data_exp * sum_reverse_block);
  }
}

template <typename T, unsigned TPB>
__device__ inline void SoftmaxWithMask2DSmall(const int all_sequence_length,
                                              const int sequence_length,
                                              const int* attention_mask,  // 2D attention mask
                                              const T* input,
                                              T* output,
                                              const bool is_unidirectional,
                                              const float scalar) {
  using BlockReduce = hipcub::BlockReduce<float, TPB>;
  __shared__ typename BlockReduce::TempStorage tmp_storage;

  __shared__ float sum_reverse_block;
  __shared__ float max_block;

  // Input dimension is BxNxSxS*; blockIdx.y is batch index b; gridDim.x=N*S;  blockIdx.x is index within N*S;
  int index = (blockIdx.y * gridDim.x + blockIdx.x) * all_sequence_length + threadIdx.x;

  float thread_data = -HIP_INF_F;
  if (threadIdx.x < all_sequence_length) {
    const int& mask = attention_mask[blockIdx.y * all_sequence_length + threadIdx.x];
    float mask_value = mask > 0 ? 0.0f : -10000.0f;

    if (is_unidirectional) {
      int from_index = all_sequence_length - sequence_length + (blockIdx.x % sequence_length);  // offset of from token in all sequence length.
      if (threadIdx.x > from_index) {
        mask_value += -10000.0f;
      }
    }

    thread_data = float(input[index]) * scalar + mask_value;
  }

  const float max = BlockReduce(tmp_storage).Reduce(thread_data, hipcub::Max(), all_sequence_length);

  // Store max value
  if (threadIdx.x == 0) {
    max_block = max;
  }
  __syncthreads();

  float thread_data_exp = threadIdx.x < all_sequence_length ? expf(thread_data - max_block) : 0.0f;
  const auto sum = BlockReduce(tmp_storage).Reduce(thread_data_exp, hipcub::Sum(), all_sequence_length);

  // Store value of 1.0/sum
  if (threadIdx.x == 0) {
    sum_reverse_block = (1.f) / sum;
  }
  __syncthreads();

  if (threadIdx.x < all_sequence_length) {
    output[index] = T(thread_data_exp * sum_reverse_block);
  }
}

template <typename T, unsigned TPB>
__global__ void SoftmaxKernelSmall(const int all_sequence_length, const int sequence_length, const T* input, T* output, bool is_unidirectional) {
  SoftmaxSmall<T, TPB>(all_sequence_length, sequence_length, all_sequence_length, 0, input, output, is_unidirectional);
}

template <typename T, unsigned TPB>
__global__ void SoftmaxKernel(const int all_sequence_length, const int sequence_length, const T* input, T* output) {
  Softmax<T, TPB>(all_sequence_length, sequence_length, all_sequence_length, 0, input, output);
}

template <typename T>
bool ComputeSoftmax(
    hipStream_t stream, const int all_sequence_length, const int sequence_length, const int batch_size, const int num_heads,
    const T* input, T* output, bool is_unidirectional) {
  const dim3 grid(sequence_length * num_heads, batch_size, 1);
  if (all_sequence_length <= 32) {
    const int blockSize = 32;
    SoftmaxKernelSmall<T, blockSize><<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, input, output, is_unidirectional);
  } else if (all_sequence_length <= 64) {
    const int blockSize = 64;
    SoftmaxKernelSmall<T, blockSize><<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, input, output, is_unidirectional);
  } else if (all_sequence_length <= 128) {
    const int blockSize = 128;
    SoftmaxKernelSmall<T, blockSize><<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, input, output, is_unidirectional);
  } else if (all_sequence_length <= 256) {
    const int blockSize = 256;
    SoftmaxKernelSmall<T, blockSize><<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, input, output, is_unidirectional);
  } else if (all_sequence_length <= 512) {
    const int blockSize = 512;
    SoftmaxKernelSmall<T, blockSize><<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, input, output, is_unidirectional);
  } else if (all_sequence_length <= 1024) {
    const int blockSize = 1024;
    SoftmaxKernelSmall<T, blockSize><<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, input, output, is_unidirectional);
  } else if (!is_unidirectional) {
    const int blockSize = 1024;
    SoftmaxKernel<T, blockSize><<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, input, output);
  } else {
    ORT_THROW("Attention CUDA operator does not support unidirectional with total sequence length > 1024.");
  }

  return CUDA_CALL(hipPeekAtLastError());
}

template <typename T, unsigned TPB>
__global__ void MaskedSoftmaxKernelSmall(const int all_sequence_length, const int sequence_length, const int* mask_end, const int* mask_start, const T* input, T* output, bool is_unidirectional) {
  __shared__ int start_position;
  __shared__ int end_position;

  if (threadIdx.x == 0) {
    const int batch = blockIdx.y;
    start_position = mask_start != nullptr ? max(0, mask_start[batch]) : 0;
    end_position = min(all_sequence_length, mask_end[batch]);

    // Attend to no word has same effect as attend to all words. This is added to get parity with CPU result.
    if (start_position >= end_position) {
      start_position = 0;
      end_position = all_sequence_length;
    }
  }
  __syncthreads();

  SoftmaxSmall<T, TPB>(all_sequence_length, sequence_length, end_position, start_position, input, output, is_unidirectional);
}

template <typename T, unsigned TPB>
__global__ void MaskedSoftmaxKernel(const int all_sequence_length, const int sequence_length, const int* mask_end, const int* mask_start, const T* input, T* output) {
  __shared__ int start_position;
  __shared__ int end_position;

  if (threadIdx.x == 0) {
    const int batch = blockIdx.y;
    start_position = mask_start != nullptr ? max(0, mask_start[batch]) : 0;
    end_position = min(all_sequence_length, mask_end[batch]);

    // Attend to no word has same effect as attend to all words. This is added to get parity with CPU result.
    if (start_position >= end_position) {
      start_position = 0;
      end_position = all_sequence_length;
    }
  }
  __syncthreads();

  Softmax<T, TPB>(all_sequence_length, sequence_length, end_position, start_position, input, output);
}

template <typename T, unsigned TPB>
__global__ void SoftmaxWithMask2DSmallKernel(const int all_sequence_length, const int sequence_length, const int* attention_mask, const T* input, T* output, const bool is_unidirectional, const float scalar) {
  SoftmaxWithMask2DSmall<T, TPB>(all_sequence_length, sequence_length, attention_mask, input, output, is_unidirectional, scalar);
}

template <typename T>
bool ComputeSoftmaxWithMask1D(hipStream_t stream, const int all_sequence_length, const int sequence_length, const int batch_size, const int num_heads,
                              const int* mask_index, const int* mask_start, const T* input, T* output, const bool is_unidirectional) {
  const dim3 grid(sequence_length * num_heads, batch_size, 1);

  if (all_sequence_length <= 32) {
    const int blockSize = 32;
    MaskedSoftmaxKernelSmall<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, mask_index, mask_start, input, output, is_unidirectional);
  } else if (all_sequence_length <= 64) {
    const int blockSize = 64;
    MaskedSoftmaxKernelSmall<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, mask_index, mask_start, input, output, is_unidirectional);
  } else if (all_sequence_length <= 128) {
    const int blockSize = 128;
    MaskedSoftmaxKernelSmall<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, mask_index, mask_start, input, output, is_unidirectional);
  } else if (all_sequence_length <= 256) {
    const int blockSize = 256;
    MaskedSoftmaxKernelSmall<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, mask_index, mask_start, input, output, is_unidirectional);
  } else if (all_sequence_length <= 512) {
    const int blockSize = 512;
    MaskedSoftmaxKernelSmall<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, mask_index, mask_start, input, output, is_unidirectional);
  } else if (all_sequence_length <= 1024) {
    const int blockSize = 1024;
    MaskedSoftmaxKernelSmall<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, mask_index, mask_start, input, output, is_unidirectional);
  } else if (!is_unidirectional) {
    const int blockSize = 1024;
    MaskedSoftmaxKernel<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, mask_index, mask_start, input, output);
  } else {
    ORT_THROW("Attention CUDA operator does not support unidirectional with total sequence length > 1024.");
  }

  return CUDA_CALL(hipPeekAtLastError());
}

template <typename T>
bool ComputeSoftmaxWithMask2D(hipStream_t stream, const int all_sequence_length, const int sequence_length, const int batch_size, const int num_heads,
                              const int* attention_mask, const T* input, T* output, const bool is_unidirectional, const float scalar) {
  const dim3 grid(sequence_length * num_heads, batch_size, 1);

  if (all_sequence_length <= 32) {
    const int blockSize = 32;
    SoftmaxWithMask2DSmallKernel<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, attention_mask, input, output, is_unidirectional, scalar);
  } else if (all_sequence_length <= 64) {
    const int blockSize = 64;
    SoftmaxWithMask2DSmallKernel<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, attention_mask, input, output, is_unidirectional, scalar);
  } else if (all_sequence_length <= 128) {
    const int blockSize = 128;
    SoftmaxWithMask2DSmallKernel<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, attention_mask, input, output, is_unidirectional, scalar);
  } else if (all_sequence_length <= 256) {
    const int blockSize = 256;
    SoftmaxWithMask2DSmallKernel<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, attention_mask, input, output, is_unidirectional, scalar);
  } else if (all_sequence_length <= 512) {
    const int blockSize = 512;
    SoftmaxWithMask2DSmallKernel<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, attention_mask, input, output, is_unidirectional, scalar);
  } else if (all_sequence_length <= 1024) {
    const int blockSize = 1024;
    SoftmaxWithMask2DSmallKernel<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(all_sequence_length, sequence_length, attention_mask, input, output, is_unidirectional, scalar);
  } else {
    ORT_THROW("Attention CUDA operator does not supported 2D attention mask with total sequence length > 1024.");
  }

  return CUDA_CALL(hipPeekAtLastError());
}

template <typename T>
__global__ void TransposeCtx(const int H, const T* input, T* output) {
  // Input:  BxNxSxH
  // Output: BxSxNxH

  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;

  int num_heads = blockDim.y;
  int sequence_length = gridDim.x;

  const int NH = num_heads * H;
  const int NHS = NH * sequence_length;
  const int in_offset = s * H + n * sequence_length * H + b * NHS;
  const int out_offset = n * H + s * NH + b * NHS;

  const int i = threadIdx.x;
  if (i < H) {
    output[out_offset + i] = input[in_offset + i];
  }
}

bool LaunchTransCtx(hipStream_t stream,
                    const int sequence_length, const int batch_size, const int head_size, const int num_heads,
                    const float* input, float* output) {
  const dim3 grid(sequence_length, batch_size, 1);
  if (0 == (head_size & 1)) {
    const int H = head_size / 2;
    const float2* input2 = reinterpret_cast<const float2*>(input);
    float2* output2 = reinterpret_cast<float2*>(output);
    const dim3 block(H, num_heads, 1);
    TransposeCtx<float2><<<grid, block, 0, stream>>>(H, input2, output2);
  } else {
    const dim3 block(head_size, num_heads, 1);
    TransposeCtx<float><<<grid, block, 0, stream>>>(head_size, input, output);
  }
  return CUDA_CALL(hipPeekAtLastError());
}

bool LaunchTransCtx(hipStream_t stream,
                    const int sequence_length, const int batch_size, const int head_size, const int num_heads,
                    const half* input, half* output) {
  const dim3 grid(sequence_length, batch_size, 1);
  if (0 == (head_size % 4)) {
    const int H = head_size / 4;
    const dim3 block(H, num_heads, 1);
    const float2* input2 = reinterpret_cast<const float2*>(input);
    float2* output2 = reinterpret_cast<float2*>(output);
    TransposeCtx<float2><<<grid, block, 0, stream>>>(H, input2, output2);
  } else if (0 == (head_size & 1)) {
    const int H = head_size / 2;
    const dim3 block(H, num_heads, 1);
    const half2* input2 = reinterpret_cast<const half2*>(input);
    half2* output2 = reinterpret_cast<half2*>(output);
    TransposeCtx<half2><<<grid, block, 0, stream>>>(H, input2, output2);
  } else {  // this should be an "odd" case. probably not worth catching it in the half2 kernel.
    const dim3 block(head_size, num_heads, 1);
    TransposeCtx<half><<<grid, block, 0, stream>>>(head_size, input, output);
  }

  return CUDA_CALL(hipPeekAtLastError());
}

template <typename T>
__global__ void TransposeQKV(const int H, const T* input, T* output) {
  // Input:  BxSx3xNxH
  // Output: 3xBxNxSxH

  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;  // matrix id

  const int num_heads = blockDim.y;

  const int sequence_length = gridDim.x;
  const int batch_size = gridDim.y;
  const int NH = num_heads * H;
  const int NHS = NH * sequence_length;
  const int in_offset = n * H + m * NH + s * 3 * NH + b * NHS * 3;
  const int out_offset = s * H + n * sequence_length * H + b * NHS + m * NHS * batch_size;

  const int i = threadIdx.x;
  if (i < H) {
    output[out_offset + i] = input[in_offset + i];
  }
}

bool LaunchTransQkv(hipStream_t stream,
                    const int sequence_length, const int batch_size, const int head_size, const int num_heads,
                    const float* input, float* output) {
  const dim3 grid(sequence_length, batch_size, 3);
  if (0 == (head_size & 1)) {
    const int H = head_size / 2;
    const float2* input2 = reinterpret_cast<const float2*>(input);
    float2* output2 = reinterpret_cast<float2*>(output);
    const dim3 block(H, num_heads, 1);
    TransposeQKV<float2><<<grid, block, 0, stream>>>(H, input2, output2);
  } else {
    const dim3 block(head_size, num_heads, 1);
    TransposeQKV<float><<<grid, block, 0, stream>>>(head_size, input, output);
  }
  return CUDA_CALL(hipPeekAtLastError());
}

bool LaunchTransQkv(hipStream_t stream,
                    const int sequence_length, const int batch_size, const int head_size, const int num_heads,
                    const half* input, half* output) {
  const dim3 grid(sequence_length, batch_size, 3);
  if (0 == (head_size % 4)) {
    const int H = head_size / 4;
    const dim3 block(H, num_heads, 1);
    const float2* input2 = reinterpret_cast<const float2*>(input);
    float2* output2 = reinterpret_cast<float2*>(output);
    TransposeQKV<float2><<<grid, block, 0, stream>>>(H, input2, output2);
  } else if (0 == (head_size & 1)) {
    const int H = head_size / 2;
    const dim3 block(H, num_heads, 1);
    const half2* input2 = reinterpret_cast<const half2*>(input);
    half2* output2 = reinterpret_cast<half2*>(output);
    TransposeQKV<half2><<<grid, block, 0, stream>>>(H, input2, output2);
  } else {  // this should be an "odd" case. probably not worth catching it in the half2 kernel..
    const dim3 block(head_size, num_heads, 1);
    TransposeQKV<half><<<grid, block, 0, stream>>>(head_size, input, output);
  }
  return CUDA_CALL(hipPeekAtLastError());
}

template <typename T>
__global__ void ConcatPastToPresent(const int sequence_length,
                                    const T* past,
                                    const T* k_v,
                                    T* present) {
  const int h = threadIdx.x;
  const int n = threadIdx.y;
  const int s = blockIdx.x;
  const int b = blockIdx.y;
  const int is_v = blockIdx.z;  // 0 for k, 1 for v

  const int all_sequence_length = gridDim.x;
  const int batch_size = gridDim.y;
  const int num_heads = blockDim.y;
  const int H = blockDim.x;

  // past:    2 x BxNxS'xH   (past_k and past_v)
  // k_v:     2 x BxNxSxH    (k and v)
  // present: 2 x BxNxS*xH   (present_k and present_v)
  const int past_sequence_length = all_sequence_length - sequence_length;

  const int present_SH = all_sequence_length * H;
  const int present_NSH = num_heads * present_SH;
  int out_offset = b * present_NSH + n * present_SH + s * H + h + is_v * (present_NSH * batch_size);
  if (s < past_sequence_length) {
    const int past_SH = past_sequence_length * H;
    const int past_NSH = num_heads * past_SH;
    const int in_offset = b * past_NSH + n * past_SH + s * H + h + is_v * (past_NSH * batch_size);
    present[out_offset] = past[in_offset];
  } else if (s < all_sequence_length) {
    const int SH = sequence_length * H;
    const int NSH = num_heads * SH;
    const int in_offset = b * NSH + n * SH + (s - past_sequence_length) * H + h + is_v * (NSH * batch_size);
    present[out_offset] = k_v[in_offset];
  }
}

bool LaunchConcatPastToPresent(hipStream_t stream,
                               const int all_sequence_length,
                               const int sequence_length,
                               const int batch_size,
                               const int head_size,
                               const int num_heads,
                               const float* past,
                               const float* k_v,
                               float* present) {
  const dim3 grid(all_sequence_length, batch_size, 2);
  if (0 == (head_size & 1)) {
    const dim3 block(head_size / 2, num_heads, 1);
    ConcatPastToPresent<float2><<<grid, block, 0, stream>>>(sequence_length, reinterpret_cast<const float2*>(past), reinterpret_cast<const float2*>(k_v), reinterpret_cast<float2*>(present));
  } else {
    const dim3 block(head_size, num_heads, 1);
    ConcatPastToPresent<float><<<grid, block, 0, stream>>>(sequence_length, past, k_v, present);
  }
  return CUDA_CALL(hipPeekAtLastError());
}

bool LaunchConcatPastToPresent(hipStream_t stream,
                               const int all_sequence_length,
                               const int sequence_length,
                               const int batch_size,
                               const int head_size,
                               const int num_heads,
                               const half* past,
                               const half* k_v,
                               half* present) {
  const dim3 grid(all_sequence_length, batch_size, 2);
  if (0 == (head_size % 4)) {
    const dim3 block(head_size / 4, num_heads, 1);
    ConcatPastToPresent<float2><<<grid, block, 0, stream>>>(sequence_length, reinterpret_cast<const float2*>(past), reinterpret_cast<const float2*>(k_v), reinterpret_cast<float2*>(present));
  } else if (0 == (head_size & 1)) {
    const dim3 block(head_size / 2, num_heads, 1);
    ConcatPastToPresent<half2><<<grid, block, 0, stream>>>(sequence_length, reinterpret_cast<const half2*>(past), reinterpret_cast<const half2*>(k_v), reinterpret_cast<half2*>(present));
  } else {  // this should be an "odd" case. probably not worth catching it in the half2 kernel.
    const dim3 block(head_size, num_heads, 1);
    ConcatPastToPresent<half><<<grid, block, 0, stream>>>(sequence_length, past, k_v, present);
  }
  return CUDA_CALL(hipPeekAtLastError());
}

hipblasStatus_t inline CublasGemmStridedBatched(
    hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k, const float alpha,
    const float* A, int lda, long long int strideA, const float* B, int ldb, long long int strideB,
    const float beta, float* C, int ldc, long long int strideC, int batchCount) {
  return hipblasSgemmStridedBatched(
      handle, transa, transb, m, n, k, &alpha, A, lda, strideA, B, ldb, strideB, &beta, C, ldc, strideC, batchCount);
}

hipblasStatus_t inline CublasGemmStridedBatched(
    hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k, const half alpha,
    const half* A, int lda, long long int strideA, const half* B, int ldb, long long int strideB,
    const half beta, half* C, int ldc, long long int strideC, int batchCount) {
  return hipblasHgemmStridedBatched(
      handle, transa, transb, m, n, k, &alpha, A, lda, strideA, B, ldb, strideB, &beta, C, ldc, strideC, batchCount);
}

template <typename T>
bool QkvToContext(
    const hipDeviceProp_t& prop, hipblasHandle_t& cublas, hipStream_t stream,
    const int batch_size, const int sequence_length, const int num_heads, const int head_size, const size_t element_size,
    const T* input, T* output, T* workspace,
    const int* mask_index, const std::vector<int64_t>* mask_index_dims,
    bool is_unidirectional, int past_sequence_length, const T* past, T* present) {
  const int all_sequence_length = past_sequence_length + sequence_length;
  const size_t bytes = ScratchSize(element_size, batch_size, num_heads, sequence_length, all_sequence_length);
  T* scratch1 = workspace;
  T* scratch2 = scratch1 + (bytes / element_size);
  T* scratch3 = scratch2 + (bytes / element_size);

  // input should be BxSx3xNxH => scratch3: 3xBxNxSxH
  if (!LaunchTransQkv(stream, sequence_length, batch_size, head_size, num_heads, input, scratch3)) {
    return false;
  }

  // now scratch3 has Q, K, V: each has size BxNxSxH
  const int batches = batch_size * num_heads;
  const int size_per_batch = sequence_length * head_size;
  const int total_size = batches * size_per_batch;

  const T* q = scratch3;
  const T* k = q + total_size;
  const T* v = k + total_size;

  hipblasSetStream(cublas, stream);
  CublasMathModeSetter helper(prop, cublas, HIPBLAS_TENSOR_OP_MATH);

  // Concat past (2xBxNxS'xH) to present (2xBxNxS*xH):
  // past_k (BxNxS'xH) + k (BxNxSxH) => present_k (BxNxS*xH)
  // past_v (BxNxS'xH) + v (BxNxSxH) => present_v (BxNxS*xH)
  const int present_size_per_batch = all_sequence_length * head_size;
  if (nullptr != present) {
    if (!LaunchConcatPastToPresent(stream, all_sequence_length, sequence_length, batch_size, head_size, num_heads, past, k, present)) {
      return false;
    }

    // update pointers to present_k and present_v.
    k = present;
    v = present + batches * present_size_per_batch;
  }

  bool use_2d_attention_mask = (nullptr != mask_index && nullptr != mask_index_dims && mask_index_dims->size() == 2);

  // compute Q*K' (as K'*Q), scaled by 1/sqrt(H) and store in scratch1: BxNxSxS*
  // Q: BxNxSxH, K (present_k): BxNxS*xH, Q*K': BxNxSxS*
  const float rsqrt_head_size = 1.f / sqrt(static_cast<float>(head_size));
  const int temp_matrix_size = sequence_length * all_sequence_length;
  T alpha = (T)(use_2d_attention_mask ? 1.0f : rsqrt_head_size);
  if (!CUBLAS_CALL(CublasGemmStridedBatched(
          cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, all_sequence_length, sequence_length, head_size, alpha, k, head_size, present_size_per_batch,
          q, head_size, size_per_batch, 0.f, scratch1, all_sequence_length, temp_matrix_size, batches))) {
    return false;
  }

  // apply softmax and store result P to scratch2: BxNxSxS*
  if (use_2d_attention_mask) {  // 2d attention mask
    if (!ComputeSoftmaxWithMask2D<T>(stream, all_sequence_length, sequence_length, batch_size, num_heads, mask_index, scratch1, scratch2, is_unidirectional, rsqrt_head_size)) {
      return false;
    }
  } else if (nullptr != mask_index) {  // 1d mask index
    ORT_ENFORCE(nullptr != mask_index_dims && mask_index_dims->size() == 1);
    // mask_index has 1D shape: either (batch_size) or (2*batch_size). Only the later one has start postions.
    const int* mask_start = (mask_index_dims->at(0) > batch_size) ? mask_index + batch_size : nullptr;
    if (!ComputeSoftmaxWithMask1D<T>(stream, all_sequence_length, sequence_length, batch_size, num_heads, mask_index, mask_start, scratch1, scratch2, is_unidirectional)) {
      return false;
    }
  } else {  // no mask
    if (!ComputeSoftmax<T>(stream, all_sequence_length, sequence_length, batch_size, num_heads, scratch1, scratch2, is_unidirectional)) {
      return false;
    }
  }

  // compute P*V (as V*P), and store in scratch3: BxNxSxH
  if (!CUBLAS_CALL(CublasGemmStridedBatched(
          cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, head_size, sequence_length, all_sequence_length, 1.f, v, head_size, present_size_per_batch,
          scratch2, all_sequence_length, temp_matrix_size, 0.f, scratch3, head_size, size_per_batch, batches))) {
    return false;
  }

  // scratch3 is BxNxSxH, transpose to output BxSxNxH
  return LaunchTransCtx(stream, sequence_length, batch_size, head_size, num_heads, scratch3, output);
}

bool LaunchAttentionKernel(
    const hipDeviceProp_t& prop,
    const void* input,
    const int* mask_index,
    const std::vector<int64_t>* mask_index_dims,
    void* output,
    const int batch_size,
    const int sequence_length,
    const int num_heads,
    const int head_size,
    void* workspace,
    hipblasHandle_t& cublas,
    const size_t element_size,
    bool is_unidirectional,
    int past_sequence_length,
    const void* past,
    void* present) {
  // use default stream
  const hipStream_t stream = nullptr;

  if (element_size == 2) {
    return QkvToContext(prop, cublas, stream,
                        batch_size, sequence_length, num_heads, head_size, element_size,
                        reinterpret_cast<const half*>(input), reinterpret_cast<half*>(output), reinterpret_cast<half*>(workspace),
                        mask_index, mask_index_dims, is_unidirectional,
                        past_sequence_length, reinterpret_cast<const half*>(past), reinterpret_cast<half*>(present));
  } else {
    return QkvToContext(prop, cublas, stream,
                        batch_size, sequence_length, num_heads, head_size, element_size,
                        reinterpret_cast<const float*>(input), reinterpret_cast<float*>(output), reinterpret_cast<float*>(workspace),
                        mask_index, mask_index_dims, is_unidirectional,
                        past_sequence_length, reinterpret_cast<const float*>(past), reinterpret_cast<float*>(present));
  }
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
