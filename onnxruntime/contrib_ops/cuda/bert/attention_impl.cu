#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on qkvToContext plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/

Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications: scaling is moved from masked softmax to the gemm before that.
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_fp16.h>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/shared_inc/fpgeneric.h"
#include "attention_impl.h"
#include "attention_softmax.h"

using namespace onnxruntime::cuda;
using namespace hipcub;

namespace onnxruntime {
namespace contrib {
namespace cuda {

static size_t AlignTo(size_t a, size_t b) {
  return CeilDiv(a, b) * b;
}

size_t GetAttentionScratchSize(size_t element_size, int batch_size, int num_heads, int sequence_length, int all_sequence_length) {
  const size_t len = batch_size * num_heads * sequence_length * all_sequence_length;
  const size_t bytes = len * element_size;

  const size_t alignment = 256;
  const size_t bytesAligned = AlignTo(bytes, alignment);
  return bytesAligned;
}

size_t GetAttentionWorkspaceSize(
    size_t element_size,
    int batch_size,
    int num_heads,
    int head_size,
    int sequence_length,
    int past_sequence_length) {
  size_t qkv_size = 3 * batch_size * sequence_length * num_heads * head_size * element_size;
  return qkv_size + 2 * GetAttentionScratchSize(element_size, batch_size, num_heads, sequence_length, past_sequence_length + sequence_length);
}

template <typename T>
bool QkvToContext(
    const hipDeviceProp_t& prop, hipblasHandle_t& cublas, hipStream_t stream,
    const int batch_size, const int sequence_length, const int num_heads, const int head_size, const size_t element_size,
    const T* input, T* output, T* workspace,
    const int* mask_index, const std::vector<int64_t>* mask_index_dims,
    bool is_unidirectional, int past_sequence_length, const T* past, T* present) {
  const int all_sequence_length = past_sequence_length + sequence_length;
  const size_t bytes = GetAttentionScratchSize(element_size, batch_size, num_heads, sequence_length, all_sequence_length);
  T* scratch1 = workspace;
  T* scratch2 = scratch1 + (bytes / element_size);
  T* scratch3 = scratch2 + (bytes / element_size);

  const int max_threads_per_block(prop.maxThreadsPerBlock);

  // input should be BxSx3xNxH => scratch3: 3xBxNxSxH
  if (!LaunchTransQkv(stream, sequence_length, batch_size, head_size, num_heads, max_threads_per_block, input, scratch3)) {
    return false;
  }

  // now scratch3 has Q, K, V: each has size BxNxSxH
  const int batches = batch_size * num_heads;
  const int size_per_batch = sequence_length * head_size;
  const int total_size = batches * size_per_batch;

  const T* q = scratch3;
  const T* k = q + total_size;
  const T* v = k + total_size;

  hipblasSetStream(cublas, stream);

  // Concat past (2xBxNxS'xH) to present (2xBxNxS*xH):
  // past_k (BxNxS'xH) + k (BxNxSxH) => present_k (BxNxS*xH)
  // past_v (BxNxS'xH) + v (BxNxSxH) => present_v (BxNxS*xH)
  const int present_size_per_batch = all_sequence_length * head_size;
  if (nullptr != present) {
    if (!LaunchConcatPastToPresent(stream, all_sequence_length, sequence_length, batch_size, head_size, num_heads, max_threads_per_block, past, k, present)) {
      return false;
    }

    // update pointers to present_k and present_v.
    k = present;
    v = present + batches * present_size_per_batch;
  }

  // Raw attention mask could be 2D (BxS) or 3D (BxSxS*)
  bool use_raw_attention_mask = (nullptr != mask_index && nullptr != mask_index_dims && mask_index_dims->size() >= 2);

  // compute Q*K' (as K'*Q), scaled by 1/sqrt(H) and store in scratch1: BxNxSxS*
  // Q: BxNxSxH, K (present_k): BxNxS*xH, Q*K': BxNxSxS*
  const float rsqrt_head_size = 1.f / sqrt(static_cast<float>(head_size));
  const int temp_matrix_size = sequence_length * all_sequence_length;
  T one = (T)(1.0f);
  T zero = (T)(0.f);

  // For raw attention mask, the scalar if 1/sqrt(H) is moved to softmax computation.
  // TODO: move scalar to softmax computation since converting 1/Sqrt(H) to half might have loss in precision.
  T alpha = use_raw_attention_mask ? one : (T)(rsqrt_head_size);

  if (!CUBLAS_CALL(cublasGemmStridedBatchedHelper(
          cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, all_sequence_length, sequence_length, head_size, &alpha, k, head_size, present_size_per_batch,
          q, head_size, size_per_batch, &zero, scratch1, all_sequence_length, temp_matrix_size, batches, prop))) {
    return false;
  }

  // apply softmax and store result P to scratch2: BxNxSxS*
  if (use_raw_attention_mask) {  // 2d, 3d or 4d attention mask
    const int mask_dimension = static_cast<int>(mask_index_dims->size());
    const int64_t max_sequence_length = mask_dimension == 4 ? mask_index_dims->at(3) : 0;
    if (!ComputeSoftmaxWithRawMask<T>(stream, all_sequence_length, sequence_length, batch_size, num_heads, mask_index, scratch1, scratch2, is_unidirectional,
                                      rsqrt_head_size, mask_dimension, static_cast<int>(max_sequence_length))) {
      return false;
    }
  } else if (nullptr != mask_index) {  // 1d mask index
    ORT_ENFORCE(nullptr != mask_index_dims && mask_index_dims->size() == 1);
    // mask_index has 1D shape: either (batch_size) or (2*batch_size). Only the later one has start postions.
    const int* mask_start = (mask_index_dims->at(0) > batch_size) ? mask_index + batch_size : nullptr;
    if (!ComputeSoftmaxWithMask1D<T>(stream, all_sequence_length, sequence_length, batch_size, num_heads, mask_index, mask_start, scratch1, scratch2, is_unidirectional)) {
      return false;
    }
  } else {  // no mask
    if (!ComputeSoftmax<T>(stream, all_sequence_length, sequence_length, batch_size, num_heads, scratch1, scratch2, is_unidirectional)) {
      return false;
    }
  }

  // compute P*V (as V*P), and store in scratch3: BxNxSxH
  if (!CUBLAS_CALL(cublasGemmStridedBatchedHelper(
          cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, head_size, sequence_length, all_sequence_length, &one, v, head_size, present_size_per_batch,
          scratch2, all_sequence_length, temp_matrix_size, &zero, scratch3, head_size, size_per_batch, batches, prop))) {
    return false;
  }

  // scratch3 is BxNxSxH, transpose to output BxSxNxH
  return LaunchTransCtx(stream, sequence_length, batch_size, head_size, num_heads, max_threads_per_block, scratch3, output);
}

bool LaunchAttentionKernel(
    const hipDeviceProp_t& prop,
    hipStream_t stream,
    const void* input,
    const int* mask_index,
    const std::vector<int64_t>* mask_index_dims,
    void* output,
    const int batch_size,
    const int sequence_length,
    const int num_heads,
    const int head_size,
    void* workspace,
    hipblasHandle_t& cublas,
    const size_t element_size,
    bool is_unidirectional,
    int past_sequence_length,
    const void* past,
    void* present) {
  if (element_size == 2) {
    return QkvToContext(prop, cublas, stream,
                        batch_size, sequence_length, num_heads, head_size, element_size,
                        reinterpret_cast<const half*>(input), reinterpret_cast<half*>(output), reinterpret_cast<half*>(workspace),
                        mask_index, mask_index_dims, is_unidirectional,
                        past_sequence_length, reinterpret_cast<const half*>(past), reinterpret_cast<half*>(present));
  } else {
    return QkvToContext(prop, cublas, stream,
                        batch_size, sequence_length, num_heads, head_size, element_size,
                        reinterpret_cast<const float*>(input), reinterpret_cast<float*>(output), reinterpret_cast<float*>(workspace),
                        mask_index, mask_index_dims, is_unidirectional,
                        past_sequence_length, reinterpret_cast<const float*>(past), reinterpret_cast<float*>(present));
  }
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
