#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on qkvToContext plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/

Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications:
// (1) support GPT-2 past state, unidirectional mask and 4D attention mask from Megatron
// (2) support 2D attention mask
// (3) allow persistent softmax from PyTorch for debugging purpose.
// (4) support different input hidden size and model hidden size for pruned model
// (5) support different hidden sizes of Q/K and V
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <cassert>
#include <hip/hip_fp16.h>
#include <hipcub/hipcub.hpp>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/shared_inc/fpgeneric.h"
#include "contrib_ops/cuda/bert/attention_impl.h"
#include "contrib_ops/cuda/bert/attention_softmax.h"
#include "contrib_ops/cuda/bert/transformer_common.h"
#include "contrib_ops/cuda/bert/add_bias_transpose.h"
#include "contrib_ops/cuda/bert/tensorrt_fused_multihead_attention/mha_runner.h"
#include "contrib_ops/cuda/bert/tensorrt_fused_multihead_attention/cross_attention/fmha_cross_attention.h"
#include "contrib_ops/cpu/bert/attention_base.h"
#include "contrib_ops/cuda/bert/bert_padding.h"
#include "contrib_ops/cuda/transformers/dump_cuda_tensor.h"
#include "contrib_ops/cuda/bert/cutlass_fmha/memory_efficient_attention.h"

using namespace onnxruntime::cuda;
using namespace onnxruntime::contrib::attention_softmax_cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

constexpr size_t kMemoryAlignment = 256;

static size_t AlignTo(size_t a, size_t b) {
  return CeilDiv(a, b) * b;
}

size_t AlignSize(size_t bytes) {
  const size_t bytesAligned = AlignTo(bytes, kMemoryAlignment);
  return bytesAligned;
}

void CumulatedSequenceLengthCache::Initialize(int32_t sequence_length, hipStream_t stream) {
  if (this->sequence_length != sequence_length) {
    ORT_ENFORCE(buffer.get() != nullptr && this->max_batch_size > 0);
    LaunchTrtSequenceOffset(reinterpret_cast<int32_t*>(buffer.get()), nullptr, this->max_batch_size, sequence_length, stream);
    this->sequence_length = sequence_length;
  }
}

int* GetCumulatedSequenceLength(CumulatedSequenceLengthCache* cache,
                                const int* mask_index,
                                int batch_size,
                                int sequence_length,
                                hipStream_t stream,
                                void* scratch_buffer) {
  if (mask_index == nullptr && cache != nullptr) {
    if (batch_size <= cache->max_batch_size) {
      cache->Initialize(sequence_length, stream);
      return reinterpret_cast<int*>(cache->buffer.get());
    }
  }

  int* sequence_offset = reinterpret_cast<int*>(scratch_buffer);
  LaunchTrtSequenceOffset(sequence_offset, mask_index, batch_size, sequence_length, stream);
  return sequence_offset;
}

size_t GetAttentionScratchSize(
    size_t element_size,
    size_t batch_size,
    size_t num_heads,
    size_t sequence_length,
    size_t total_sequence_length) {
  const size_t bytes = element_size * batch_size * num_heads * sequence_length * total_sequence_length;
  return AlignSize(bytes);
}

size_t GetSequenceOffsetSize(int batch_size, bool has_padding) {
  // There are batch_size + 1 offsets Without padding (or padding removed), and 2 * batch_size + 1 with padding.
  size_t bytes = sizeof(int) * ((has_padding ? 2 * batch_size : batch_size) + 1);
  return AlignSize(bytes);
  ;
}

size_t GetAttentionWorkspaceSize(
    size_t element_size,
    size_t batch_size,
    size_t num_heads,
    size_t qk_head_size,
    size_t v_head_size,
    size_t sequence_length,
    size_t kv_sequence_length,
    size_t total_sequence_length,
    void* fused_runner,
    bool use_fused_cross_attention,
    bool use_memory_efficient_attention) {
  // Note that q, k and v might need alignment for fused attention kernels.
  const size_t qkv_bytes = element_size * batch_size * num_heads *
                           ((sequence_length + kv_sequence_length) * qk_head_size + kv_sequence_length * v_head_size);

#if USE_FLASH_ATTENTION
  if (use_memory_efficient_attention) {
    size_t fmha_buffer_bytes = 0;
    if (MemoryEfficientAttentionParams::need_workspace(v_head_size, element_size == sizeof(float))) {
      fmha_buffer_bytes = batch_size * sequence_length * num_heads * v_head_size * sizeof(float);
    }

    return qkv_bytes + fmha_buffer_bytes;
  }
#else
  ORT_UNUSED_PARAMETER(use_memory_efficient_attention);
#endif

  if (fused_runner != nullptr) {
    return qkv_bytes + GetSequenceOffsetSize(static_cast<int>(batch_size), true);
  }

  if (use_fused_cross_attention) {
    return qkv_bytes + 2 * GetSequenceOffsetSize(static_cast<int>(batch_size), true);
  }

  return qkv_bytes + 2 * GetAttentionScratchSize(element_size, batch_size, num_heads, sequence_length,
                                                 total_sequence_length);
}

template <typename T>
__global__ void AddBiasTransAppendKvToPresentSmall(
    const T* qkv, const T* biases, T* present,
    const int head_size, const int past_sequence_length, const int max_sequence_length) {
  // Input:  BxSxMxNxH  (Format 1)
  // Output: (2, B, N, [P..P+S) of MaxS, H),
  // B is batch_size, S is sequence_length, M is number of matrices, N is num_heads, H is head_size
  const int n = threadIdx.y;
  const int s = blockIdx.x;
  const int b = blockIdx.y;
  const int N = blockDim.y;
  const int S = gridDim.x;
  const int B = gridDim.y;

  constexpr int M = 3;           // Matrix count in qkv
  const int m = blockIdx.z + 1;  // k = 1, v = 2

  const int NH = N * head_size;
  const int NHS = NH * S;

  qkv += (n * head_size + (s * M + m) * NH + b * M * NHS);
  if (biases) {
    biases += (m * NH + n * head_size);
  }

  const int MsH = max_sequence_length * head_size;
  const int NMsH = N * MsH;
  const int BNMsH = B * NMsH;
  present += ((past_sequence_length + s) * head_size + n * MsH + b * NMsH + (m - 1) * BNMsH);

  for (int h = threadIdx.x; h < head_size; h += blockDim.x) {
    T bias = (biases ? biases[h] : (T)0.0f);
    present[h] = qkv[h] + bias;
  }
}

template <typename T>
__global__ void AddBiasTransAppendKvToPresent(
    const T* qkv, const T* biases, T* present,
    const int head_size, const int past_sequence_length, const int max_sequence_length) {
  // Input:  BxSxMxNxH  (Format 1)
  // Output: (2, B, N, [P..P+S) of MaxS, H),
  // B is batch_size, S is sequence_length, M is number of matrices, N is num_heads, H is head_size
  const int n = blockIdx.x;
  const int s = blockIdx.y;
  const int b = (blockIdx.z >> 1);
  const int N = gridDim.x;
  const int S = gridDim.y;
  const int B = (gridDim.z >> 1);

  constexpr int M = 3;                   // Matrix count in qkv
  const int m = (blockIdx.z & 0x1) + 1;  // k = 1, v = 2

  const int NH = N * head_size;
  const int NHS = NH * S;

  qkv += (n * head_size + (s * M + m) * NH + b * M * NHS);
  if (biases) {
    biases += (m * NH + n * head_size);
  }

  const int MsH = max_sequence_length * head_size;
  const int NMsH = N * MsH;
  const int BNMsH = B * NMsH;
  present += ((past_sequence_length + s) * head_size + n * MsH + b * NMsH + (m - 1) * BNMsH);

  for (int h = threadIdx.x; h < head_size; h += blockDim.x) {
    T bias = (biases ? biases[h] : (T)0.0f);
    present[h] = qkv[h] + bias;
  }
}

// qkv buffer is merged tensor of shape (B,S,3,N,H), k v is the second/third of the 3.
// bias is of shape (3, NxH) or nullptr
// append to present of (2, B, N, (P..T) of M, H),
template <typename T>
Status LaunchAddBiasTransAppendKvToPresent(hipStream_t stream,
                                           const int max_sequence_length,
                                           const int past_sequence_length,
                                           const int sequence_length,
                                           const int batch_size,
                                           const int head_size,
                                           const int num_heads,
                                           const int max_threads_per_block,
                                           const T* biases,
                                           const T* qkv_buffer,
                                           T* present) {
  assert(head_size <= (1 << 30));

  int64_t nh = (int64_t)head_size * num_heads;
  if (nh <= max_threads_per_block) {
    const dim3 grid(sequence_length, batch_size, 2);  // 2 for k and v
    const dim3 block(max_threads_per_block / num_heads, num_heads, 1);

    AddBiasTransAppendKvToPresentSmall<T><<<grid, block, 0, stream>>>(
        qkv_buffer, biases, present, head_size, past_sequence_length, max_sequence_length);
  } else {
    const dim3 grid(num_heads, sequence_length, batch_size * 2);  // 2 for k and v
    const dim3 block(std::min(head_size, max_threads_per_block), 1, 1);
    AddBiasTransAppendKvToPresent<T><<<grid, block, 0, stream>>>(
        qkv_buffer, biases, present, head_size, past_sequence_length, max_sequence_length);
  }

  return CUDA_CALL(hipGetLastError());
}

template Status LaunchAddBiasTransAppendKvToPresent(hipStream_t stream,
                                                    const int max_sequence_length,
                                                    const int total_sequence_length,
                                                    const int sequence_length,
                                                    const int batch_size,
                                                    const int head_size,
                                                    const int num_heads,
                                                    const int max_threads_per_block,
                                                    const float* bias,
                                                    const float* qkv_buffer,
                                                    float* present);

template Status LaunchAddBiasTransAppendKvToPresent(hipStream_t stream,
                                                    const int max_sequence_length,
                                                    const int total_sequence_length,
                                                    const int sequence_length,
                                                    const int batch_size,
                                                    const int head_size,
                                                    const int num_heads,
                                                    const int max_threads_per_block,
                                                    const half* bias,
                                                    const half* qkv_buffer,
                                                    half* present);

template <typename T>
Status PrepareQkv(contrib::AttentionParameters& parameters,
                  AttentionData<T>& data,
                  hipStream_t stream,
                  int max_threads_per_block,
                  T* q, T* k, T* v, AttentionQkvFormat& qkv_format) {
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const int kv_sequence_length = parameters.kv_sequence_length;
  const int num_heads = parameters.num_heads;
  const int qk_head_size = parameters.head_size;
  const int v_head_size = parameters.v_head_size;
  const bool past_present_share_buffer = parameters.past_present_share_buffer;
  void* fused_runner = data.fused_runner;
  bool use_memory_efficient_attention = data.use_memory_efficient_attention;

  T* qkv = data.workspace;

  bool use_fused_kernel = (nullptr != fused_runner && !parameters.is_unidirectional);
  bool use_fused_causal = (nullptr != fused_runner && parameters.is_unidirectional);

  // Default format for memory efficient attention.
  // When there is past state, the format shall be BxNxSxH, so we disable memory efficient attention when there is past.
  DUMP_TENSOR_INIT();
  if (nullptr != data.gemm_buffer) {
    if (data.bias == nullptr) {
      assert(nullptr == fused_runner);
      // For quantized attention, bias has been added so only need transpose here.
      // gemm_buffer should be BxSx3xNxH => qkv: 3xBxNxSxH
      assert(qk_head_size == v_head_size);
      int matrix_to_trans = (past_present_share_buffer ? 1 : 3);
      ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, matrix_to_trans, sequence_length, batch_size, qk_head_size, num_heads,
                                         max_threads_per_block, false, data.gemm_buffer, qkv, 3));
      qkv_format = AttentionQkvFormat::Q_K_V_BNSH;
    } else {
      // For fused TRT attention, transpose qkv to BxSxNx3xH (format 2)
      // For memory efficient attention, transpose to 3xBxSxNxH (format 3)
      // For unfused kernel, transpose to 3xBxNxSxH (format 1)
      // For fused causal kernel, use format 1 since we need have K and V to update present state,
      //   at the same time, we update gemm_buffer BxSx3xNxH with bias which is used as input for fused causal kernel.
      const int format = (use_fused_kernel ? 2 : (use_memory_efficient_attention ? 3 : 1));
      qkv_format = use_fused_kernel
                       ? AttentionQkvFormat::QKV_BSN3H
                       : (use_memory_efficient_attention
                              ? AttentionQkvFormat::Q_K_V_BSNH
                              : (use_fused_causal ? AttentionQkvFormat::Q_K_V_BNSH_QKV_BS3NH : AttentionQkvFormat::Q_K_V_BNSH));

      // For fused causal, we will update gemm_buffer with bias directly.
      T* qkv_add_bias = use_fused_causal ? data.gemm_buffer : nullptr;

      int matrix_to_transpose = ((format == AttentionQkvFormat::Q_K_V_BNSH && past_present_share_buffer) ? 1 : 3);
      // format 1: BxSx(NH + NH + NH_v) => BxNxSxH + BxNxSxH + BxNxSxH_v
      // format 2: BxSx(NH + NH + NH) => BxSxNx(H + H + H)
      LaunchAddBiasTranspose(stream, matrix_to_transpose, format, max_threads_per_block,
                             batch_size, sequence_length, num_heads, qk_head_size,
                             data.gemm_buffer, data.bias, qkv, true, v_head_size, qkv_add_bias,
                             3, parameters.do_rotary, parameters.original_past_sequence_length);
    }
  }
  // attention with past/present state
  else if (data.past_key != nullptr || data.present_key != nullptr) {
    if (data.bias == nullptr) {
      // cross attention with past state
      if (data.past_key != nullptr && data.present_key == nullptr) {
        assert(data.past_value != nullptr);
        assert(data.query != nullptr);
        assert(data.key == nullptr);
        assert(data.value == nullptr);
        ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 1, sequence_length, batch_size, qk_head_size, num_heads,
                                          max_threads_per_block, false, data.query, q));
      }
      // cross attention with present state or self attention with present state
      else if (data.past_key == nullptr && data.present_key != nullptr) {
        assert(data.past_value == nullptr);
        assert(data.present_value != nullptr);
        assert(data.query != nullptr);
        assert(data.key != nullptr);
        assert(data.value != nullptr);

        // TODO: supporting packed qkv for self attention may benefit performance
        ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 1, sequence_length, batch_size, qk_head_size, num_heads,
                            max_threads_per_block, false, data.query, q));

        // TODO: supporting packed kv for cross attention may benefit performance
        ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 1, kv_sequence_length, batch_size, qk_head_size, num_heads,
                            max_threads_per_block, false, data.key, data.present_key));
        ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 1, kv_sequence_length, batch_size, v_head_size, num_heads,
                            max_threads_per_block, false, data.value, data.present_value));
      }
      // self attention with past and present state
      else {
        assert(data.past_key != nullptr);
        assert(data.past_value != nullptr);
        assert(data.present_key != nullptr);
        assert(data.present_value != nullptr);
        assert(data.query != nullptr);
        assert(data.key != nullptr);
        assert(data.value != nullptr);
        // TODO: supporting packed qkv for self attention may benefit performance
        ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 1, sequence_length, batch_size, qk_head_size, num_heads,
                            max_threads_per_block, false, data.query, q));
        ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 1, kv_sequence_length, batch_size, qk_head_size, num_heads,
                            max_threads_per_block, false, data.key, k));
        ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 1, kv_sequence_length, batch_size, v_head_size, num_heads,
                            max_threads_per_block, false, data.value, v));
      }
      qkv_format = AttentionQkvFormat::Q_K_V_BNSH;
    }
#if USE_FLASH_ATTENTION
    // When past_key/past_value are inputted directly as key/value and there is no present_key/present_value
    else if (use_memory_efficient_attention && data.past_key != nullptr && data.past_value != nullptr && parameters.pass_past_in_kv) {
      // Transpose past_key and past_value to use memory efficient attention

      // past_key (BxNxSxH) => temp_k_workspace (BxSxNxH)
      ORT_RETURN_IF_ERROR(LaunchTransCtx(stream, kv_sequence_length, batch_size, qk_head_size, num_heads,
                                         max_threads_per_block, false, data.past_key, data.temp_k_workspace));
      // past_value (BxNxSxH_v) => temp_v_workspace (BxSxNxH_v)
      ORT_RETURN_IF_ERROR(LaunchTransCtx(stream, kv_sequence_length, batch_size, qk_head_size, num_heads,
                                         max_threads_per_block, false, data.past_value, data.temp_v_workspace));

      // query => q, temp_k_workspace => k, temp_v_workspace => v
      LaunchAddBias(stream, max_threads_per_block,
              batch_size, sequence_length, kv_sequence_length,
              num_heads, qk_head_size, v_head_size,
              data.bias, data.query, data.temp_k_workspace, data.temp_v_workspace, q, k, v);

      DUMP_TENSOR_D("q(BSNH)", q, batch_size * sequence_length, num_heads, qk_head_size);
      DUMP_TENSOR_D("k(BSNH)", k, batch_size * kv_sequence_length, num_heads, qk_head_size);
      DUMP_TENSOR_D("v(BSNH)", v, batch_size * kv_sequence_length, num_heads, v_head_size);
      qkv_format = AttentionQkvFormat::Q_K_V_BSNH;

      data.past_key = nullptr;
      data.past_value = nullptr;
    }
    // When there is no past_key/past_value and there is present_key/present_value (e.g. get initial kv to use as past_kv in the next iteration)
    else if (use_memory_efficient_attention && data.present_key != nullptr && data.present_value != nullptr) {
      // Use memory efficient attention kernel
      LaunchAddBias(stream, max_threads_per_block,
                    batch_size, sequence_length, kv_sequence_length,
                    num_heads, qk_head_size, v_head_size,
                    data.bias, data.query, data.key, data.value, q, data.temp_k_workspace, data.temp_v_workspace);

      // temp_k_workspace (BxSxNxH) => present_k (BxNxSxH)
      ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 1, kv_sequence_length, batch_size, qk_head_size, num_heads,
                          max_threads_per_block, false, data.temp_k_workspace, data.present_key));

      // temp_v_workspace (BxSxNxH_v) => present_v (BxNxSxH_v)
      ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 1, kv_sequence_length, batch_size, v_head_size, num_heads,
                          max_threads_per_block, false, data.temp_v_workspace, data.present_value));

      DUMP_TENSOR_D("q(BSNH)", q, batch_size * sequence_length, num_heads, qk_head_size);
      DUMP_TENSOR_D("k(BSNH)", data.temp_k_workspace, batch_size * kv_sequence_length, num_heads, qk_head_size);
      DUMP_TENSOR_D("v(BSNH)", data.temp_v_workspace, batch_size * kv_sequence_length, num_heads, v_head_size);
      qkv_format = AttentionQkvFormat::Q_K_V_BSNH;
    }
#endif
    else {
      // Use unfused kernel for Q, use unfused kernel for K and V if needed
      constexpr int format = 0;
      // Query (BxSxNxH) => Q (BxNxSxH)
      LaunchAddBiasTranspose<T>(stream, 1, format, max_threads_per_block,
                                batch_size, sequence_length, num_heads, qk_head_size,
                                data.query, data.bias, q,
                                true, -1);

      if (!parameters.pass_past_in_kv) {
        T* k_dest = (data.past_key == nullptr && data.present_key != nullptr) ? data.present_key : k;
        T* v_dest = (data.past_value == nullptr && data.present_value != nullptr) ? data.present_value : v;

        // Key (BxLxNxH) => K (BxNxLxH)
        LaunchAddBiasTranspose<T>(stream, 1, format, max_threads_per_block,
                                  batch_size, kv_sequence_length, num_heads, qk_head_size,
                                  data.key, data.bias + num_heads * qk_head_size, k_dest,
                                  true, -1);

        // Value (BxLxNxH_v) => V (BxNxLxH_v)
        LaunchAddBiasTranspose<T>(stream, 1, format, max_threads_per_block,
                                  batch_size, kv_sequence_length, num_heads, v_head_size,
                                  data.value, data.bias + 2 * num_heads * qk_head_size, v_dest,
                                  true, -1);

        DUMP_TENSOR_D("q(BNSH)", q, batch_size * num_heads, sequence_length, qk_head_size);
        DUMP_TENSOR_D("k(BNSH)", k_dest, batch_size * num_heads, kv_sequence_length, qk_head_size);
        DUMP_TENSOR_D("v(BNSH)", v_dest, batch_size * num_heads, kv_sequence_length, v_head_size);
      }
      qkv_format = AttentionQkvFormat::Q_K_V_BNSH;
    }
  } else if (data.key == nullptr) {  // gemm_buffer == nullptr and packed qkv
    assert(data.bias == nullptr);
    assert(qk_head_size == v_head_size);

    DUMP_TENSOR_D("packed_qkv", data.query, batch_size * sequence_length, num_heads, 3, qk_head_size);

    if (use_memory_efficient_attention) {
      // unpack qkv to BSNH. Note that there is no bias so we need not output query to q.
      constexpr int format = 4;
      T* qkv_add_bias = nullptr;
      LaunchAddBiasTranspose(stream, 3, format, max_threads_per_block,
                             batch_size, sequence_length, num_heads, qk_head_size,
                             data.query, data.bias, qkv,
                             true, v_head_size, qkv_add_bias, 3);
      DUMP_TENSOR_D("q(BSNH)", q, batch_size * sequence_length, num_heads, qk_head_size);
      DUMP_TENSOR_D("k(BSNH)", k, batch_size * kv_sequence_length, num_heads, qk_head_size);
      DUMP_TENSOR_D("v(BSNH)", v, batch_size * kv_sequence_length, num_heads, v_head_size);
      qkv_format = AttentionQkvFormat::Q_K_V_BSNH;
    } else {
      if (!use_fused_kernel) {
        return ORT_MAKE_STATUS(ONNXRUNTIME, NOT_IMPLEMENTED, "packed QKV format is not implemented for current GPU. Please disable it in fusion options.");
      }

      qkv_format = AttentionQkvFormat::QKV_BSN3H;
    }
  } else if (data.value == nullptr) {  // gemm_buffer == nullptr and packed kv
    // TODO: unpack kv to BNSH for unfused kernel so that we can remove the following constraint.
    // CheckInputs verified this constraint.
    assert(data.bias == nullptr);
    assert(qk_head_size == v_head_size);

    DUMP_TENSOR_D("packed_kv", data.key, batch_size * kv_sequence_length, num_heads, 2, qk_head_size);

    if (use_memory_efficient_attention) {
      // unpack kv to BSNH. Note that there is no bias so we need not output query to q.
      constexpr int format = 4;
      T* qkv_add_bias = nullptr;
      const T* kv_bias = (data.bias == nullptr ? data.bias : data.bias + parameters.hidden_size);
      LaunchAddBiasTranspose(stream, 2, format, max_threads_per_block,
                             batch_size, kv_sequence_length, num_heads, qk_head_size,
                             data.key, kv_bias, k,
                             true, v_head_size, qkv_add_bias, 2);
      DUMP_TENSOR_D("k(BSNH)", k, batch_size * kv_sequence_length, num_heads, qk_head_size);
      DUMP_TENSOR_D("v(BSNH)", v, batch_size * kv_sequence_length, num_heads, v_head_size);
      qkv_format = AttentionQkvFormat::Q_K_V_BSNH;
    } else {
      if (data.fused_cross_attention_kernel == nullptr) {
        return ORT_MAKE_STATUS(ONNXRUNTIME, NOT_IMPLEMENTED, "packed KV format is not implemented for current GPU. Please disable packed kv in fusion options.");
      }

      qkv_format = AttentionQkvFormat::Q_KV_BSNH_BSN2H;
    }
  } else {  // gemm_buffer == nullptr and not packed
    assert(data.query != nullptr && data.key != nullptr && data.value != nullptr);

    DUMP_TENSOR_D("query", data.query, batch_size * sequence_length, num_heads, qk_head_size);
    DUMP_TENSOR_D("key", data.key, batch_size * kv_sequence_length, num_heads, qk_head_size);
    DUMP_TENSOR_D("value", data.value, batch_size * kv_sequence_length, num_heads, v_head_size);

#if DUMP_TENSOR_LEVEL > 1
    if (data.bias != nullptr) {
      DUMP_TENSOR_D("query_bias", data.bias, num_heads, qk_head_size);
      DUMP_TENSOR_D("key_bias", data.bias + num_heads * qk_head_size, num_heads, qk_head_size);
      DUMP_TENSOR_D("value_bias", data.bias + 2 * num_heads * qk_head_size, num_heads, v_head_size);
    }
#endif

    if (data.relative_position_bias != nullptr && parameters.broadcast_res_pos_bias) {
      DUMP_TENSOR_D("relative_position_bias", data.relative_position_bias, num_heads, sequence_length, kv_sequence_length);
    }

    if (data.mask_index != nullptr && parameters.mask_type == AttentionMaskType::MASK_1D_KEY_SEQ_LEN_START) {
      DUMP_TENSOR_D("mask_index", data.mask_index, 3 * batch_size + 2, 1);
    }

    if (data.fused_cross_attention_kernel != nullptr) {
      assert(qk_head_size == v_head_size);

      // For fused cross attention, besides adding bias, K and V needed to be packed:
      //   K (BxSxNxH), V (BxSxNxH) => BxSxNx2xH
      LaunchAddBiasTransposeTrt(
          stream, max_threads_per_block,
          batch_size, sequence_length,
          num_heads, qk_head_size,
          data.bias, data.query, data.key, data.value, qkv, true, kv_sequence_length);

      qkv_format = AttentionQkvFormat::Q_KV_BSNH_BSN2H;
    }
#if USE_FLASH_ATTENTION
    else if (use_memory_efficient_attention) {
      LaunchAddBias(stream, max_threads_per_block,
                    batch_size, sequence_length, kv_sequence_length,
                    num_heads, qk_head_size, v_head_size,
                    data.bias, data.query, data.key, data.value, q, k, v);

      DUMP_TENSOR_D("q(BSNH)", q, batch_size * sequence_length, num_heads, qk_head_size);
      DUMP_TENSOR_D("k(BSNH)", k, batch_size * kv_sequence_length, num_heads, qk_head_size);
      DUMP_TENSOR_D("v(BSNH)", v, batch_size * kv_sequence_length, num_heads, v_head_size);
      qkv_format = AttentionQkvFormat::Q_K_V_BSNH;
    }
#endif
    else if (use_fused_kernel) {
      assert(qk_head_size == v_head_size);

      // Q (BxSxNxH), K (BxSxNxH), V (BxSxNxH) => BxSxNx(H + H + H)
      LaunchAddBiasTransposeTrt(
          stream, max_threads_per_block,
          batch_size, sequence_length,
          num_heads, qk_head_size,
          data.bias, data.query, data.key, data.value, qkv, false, kv_sequence_length);
      DUMP_TENSOR_D("qkv(BSN3H)", qkv, batch_size, sequence_length, num_heads, 2 * qk_head_size + v_head_size);

      qkv_format = AttentionQkvFormat::QKV_BSN3H;
    } else {  // unfused kernel
      ORT_ENFORCE(!use_fused_causal, "MultiHeadAttention has not enabled fused causal");

      // Query (BxSxNxH) => Q (BxNxSxH)
      constexpr int format = 0;
      LaunchAddBiasTranspose<T>(stream, 1, format, max_threads_per_block,
                                batch_size, sequence_length, num_heads, qk_head_size,
                                data.query, data.bias, q,
                                true, -1);

      // Key (BxLxNxH) => K (BxNxLxH)
      LaunchAddBiasTranspose<T>(stream, 1, format, max_threads_per_block,
                                batch_size, kv_sequence_length, num_heads, qk_head_size,
                                data.key, nullptr == data.bias ? nullptr : data.bias + num_heads * qk_head_size, k,
                                true, -1);

      // Value (BxLxNxH_v) => K (BxNxLxH_v)
      LaunchAddBiasTranspose<T>(stream, 1, format, max_threads_per_block,
                                batch_size, kv_sequence_length, num_heads, v_head_size,
                                data.value, nullptr == data.bias ? nullptr : data.bias + 2 * num_heads * qk_head_size, v,
                                true, -1);

      DUMP_TENSOR_D("q(BNSH)", q, batch_size * num_heads, sequence_length, qk_head_size);
      DUMP_TENSOR_D("k(BNSH)", k, batch_size * num_heads, kv_sequence_length, qk_head_size);
      DUMP_TENSOR_D("v(BNSH)", v, batch_size * num_heads, kv_sequence_length, v_head_size);
      qkv_format = AttentionQkvFormat::Q_K_V_BNSH;
    }
  }

  CUDA_RETURN_IF_ERROR(hipGetLastError());
  return Status::OK();
}

template <typename T>
Status QkvToContext(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data) {
  constexpr size_t element_size = sizeof(T);
  const int max_threads_per_block = device_prop.maxThreadsPerBlock;
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const int kv_sequence_length = parameters.kv_sequence_length;
  const int total_sequence_length = parameters.total_sequence_length;
  const int num_heads = parameters.num_heads;
  const int qk_head_size = parameters.head_size;
  const int v_head_size = parameters.v_head_size;
  const bool past_present_share_buffer = parameters.past_present_share_buffer;
  const float mask_filter_value = parameters.mask_filter_value;
  void* fused_runner = data.fused_runner;

  // At most one fused kernel is enabled.
  assert(int(data.use_memory_efficient_attention) + int(fused_runner != nullptr) + int(data.fused_cross_attention_kernel != nullptr) <= 1);

  const int batches = batch_size * num_heads;

  T* qkv = nullptr;
  T* q = nullptr;
  T* k = nullptr;
  T* v = nullptr;
  T* scratch1 = data.workspace;
  if (data.has_qkv_workspace) {
    const int size_per_batch_q = sequence_length * qk_head_size;
    const int size_per_batch_k = kv_sequence_length * qk_head_size;
    const int size_per_batch_v = kv_sequence_length * v_head_size;
    const size_t elements_q = static_cast<size_t>(batches) * static_cast<size_t>(size_per_batch_q);
    const size_t elements_k = static_cast<size_t>(batches) * static_cast<size_t>(size_per_batch_k);
    const size_t elements_v = static_cast<size_t>(batches) * static_cast<size_t>(size_per_batch_v);
    qkv = data.workspace;
    q = qkv;
    k = q + elements_q;
    v = k + elements_k;
    scratch1 = v + elements_v;
  }

  bool use_fused_kernel = (nullptr != fused_runner && !parameters.is_unidirectional);
  bool use_fused_causal = (nullptr != fused_runner && parameters.is_unidirectional);

  AttentionQkvFormat qkv_format = AttentionQkvFormat::Q_K_V_BSNH;
  ORT_RETURN_IF_ERROR(PrepareQkv<T>(parameters, data, stream, max_threads_per_block, q, k, v, qkv_format));

  int present_size_per_batch_k = 0;
  int present_size_per_batch_v = 0;
  if (!past_present_share_buffer) {
    // Concat past key value to present (2xBxNxLxH), where L is kv_sequence_length and T is total_sequence_length.
    // past_k (BxNxPxH) + k (BxNxLxH) => present_k (BxNxTxH)
    // past_v (BxNxPxH) + v (BxNxLxH) => present_v (BxNxTxH)
    // When there is past state, the head size for Q/K/V shall be same: H == H_v.
    present_size_per_batch_k = total_sequence_length * qk_head_size;
    present_size_per_batch_v = total_sequence_length * v_head_size;

    if (nullptr != data.present) {
      assert(qkv_format == AttentionQkvFormat::Q_K_V_BNSH || qkv_format == AttentionQkvFormat::Q_K_V_BNSH_QKV_BS3NH);
      ORT_RETURN_IF_ERROR(
          LaunchConcatPastToPresent(stream, total_sequence_length, sequence_length, batch_size, qk_head_size, num_heads,
                                    max_threads_per_block, data.past, k, data.present));

      // Update pointers to present_k and present_v.
      k = data.present;
      v = data.present + batches * present_size_per_batch_k;
    }

    if (nullptr != data.past_key || nullptr != data.present_key) {
      if (nullptr != data.past_key && nullptr == data.present_key) {
        k = const_cast<T*>(data.past_key);
        v = const_cast<T*>(data.past_value);
      } else if (nullptr == data.past_key && nullptr != data.present_key) {
        if (qkv_format == AttentionQkvFormat::Q_K_V_BNSH) {
          k = data.present_key;
          v = data.present_value;
        }
        else {
          assert(qkv_format == AttentionQkvFormat::Q_K_V_BSNH);
          k = data.temp_k_workspace;
          v = data.temp_v_workspace;
        }
      } else if (parameters.pass_past_in_kv) {
        // past_key and past_value are used directly as key and value in attention computations
        k = const_cast<T*>(data.past_key);
        v = const_cast<T*>(data.past_value);

        // This path has a memory copy from past_key and past_value to present_key and present_value
        // Avoid this path since the memory copy is unnecessary because past_key == present_key and
        // past_value == present_value
        int64_t k_size = (int64_t)batch_size * num_heads * parameters.total_sequence_length * qk_head_size;
        int64_t v_size = (int64_t)batch_size * num_heads * parameters.total_sequence_length * v_head_size;
        hipMemcpyAsync(data.present_key, data.past_key, k_size * sizeof(T), hipMemcpyDeviceToDevice, stream);
        hipMemcpyAsync(data.present_value, data.past_value, v_size * sizeof(T), hipMemcpyDeviceToDevice, stream);
      } else {
        ORT_RETURN_IF_ERROR(
            LaunchConcatTensorToTensor(stream, parameters.total_sequence_length, sequence_length, batch_size, qk_head_size, num_heads,
                                       max_threads_per_block, 1, data.past_key, k, data.present_key));
        ORT_RETURN_IF_ERROR(
            LaunchConcatTensorToTensor(stream, parameters.total_sequence_length, sequence_length, batch_size, v_head_size, num_heads,
                                       max_threads_per_block, 1, data.past_value, v, data.present_value));
        // Update pointers to present_k and present_v.
        k = data.present_key;
        v = data.present_value;
      }
    }
  } else {
    assert(qk_head_size == v_head_size);
    assert(data.fused_cross_attention_kernel == nullptr);
    assert(!use_fused_kernel);
    assert(data.gemm_buffer != nullptr);
    assert(!data.use_memory_efficient_attention);
    assert(data.has_qkv_workspace);

    if (nullptr != data.past_key || nullptr != data.present_key) {
      // TODO: support this case.
      ORT_THROW("buffer sharing for no bias case between past and present is not supported yet.");
    }

    if (data.present != data.past) {
      // For easy testing. Production should better avoid this path.
      int64_t kv_size = 2LL * (int64_t)batch_size * num_heads * parameters.max_sequence_length * qk_head_size;
      hipMemcpyAsync(data.present, data.past, kv_size * sizeof(T), hipMemcpyDeviceToDevice, stream);
    }

    // append last k v to present
    ORT_RETURN_IF_ERROR(LaunchAddBiasTransAppendKvToPresent(
        stream, parameters.max_sequence_length, parameters.past_sequence_length, sequence_length,
        batch_size, qk_head_size, num_heads, max_threads_per_block,
        use_fused_causal ? nullptr : data.bias,  // For fused causal, bias has been added to gemm_buffer
        data.gemm_buffer, data.present));

    present_size_per_batch_k = parameters.max_sequence_length * qk_head_size;
    present_size_per_batch_v = present_size_per_batch_k;
    k = data.present;
    v = data.present + batches * present_size_per_batch_k;
  }

  // Q, K and V are ready now
  DUMP_TENSOR_INIT();

  if (data.fused_cross_attention_kernel != nullptr) {
    assert(qkv_format == AttentionQkvFormat::Q_KV_BSNH_BSN2H);

    // We only enable fused cross attention when there is no key padding mask.
    // Otherwise, key have effective batch size 2 * batch_size, which is different from batch_size of query.
    assert(data.mask_index == nullptr);

    int* q_sequence_offset = GetCumulatedSequenceLength(data.cumulated_sequence_length_q_cache,
                                                        data.mask_index, batch_size, sequence_length, stream,
                                                        scratch1);

    DUMP_TENSOR_D("q_sequence_offset", q_sequence_offset, 1, batch_size + 1);

    int* kv_sequence_offset = q_sequence_offset + (GetSequenceOffsetSize(batch_size, false) / sizeof(int));
    kv_sequence_offset = GetCumulatedSequenceLength(data.cumulated_sequence_length_kv_cache,
                                                    data.mask_index, batch_size, kv_sequence_length, stream,
                                                    kv_sequence_offset);
    CUDA_RETURN_IF_ERROR(hipGetLastError());

    DUMP_TENSOR_D("kv_sequence_offset", kv_sequence_offset, 1, batch_size + 1);

    FusedMultiHeadCrossAttentionKernel const* cross_attention_kernel =
        reinterpret_cast<FusedMultiHeadCrossAttentionKernel const*>(data.fused_cross_attention_kernel);

    // When there is no bias, we can directly use q and packed kv from inputs.
    void const* query = q;
    void const* packed_kv = k;
    if (data.value == nullptr && data.bias == nullptr) {
      query = data.query;
      packed_kv = data.key;
    }

    run_fused_cross_attention(
        query,                   // Q
        packed_kv,               // packed KV
        q_sequence_offset,       // cumulated sequence length of Q
        kv_sequence_offset,      // cumulated sequence length of KV
        data.output,             // output
        cross_attention_kernel,  // kernels
        batch_size,              // batch size
        num_heads,               // number of heads
        qk_head_size,            // head size of Q/K/V
        sequence_length,         // sequence length of Q
        kv_sequence_length,      // sequence length of KV
        stream);

    DUMP_TENSOR("trt cross output", data.output, batch_size * sequence_length, num_heads, v_head_size);
    return Status::OK();
  }

  // Run TRT fused attention.
  if (use_fused_kernel || use_fused_causal) {
    int* sequence_offset = reinterpret_cast<int*>(scratch1);
    if (parameters.mask_type == AttentionMaskType::MASK_2D_KEY_PADDING) {
      DUMP_TENSOR_D("mask", reinterpret_cast<const int*>(data.mask_index), batch_size, sequence_length);
      LaunchTrtSequenceOffset2d(sequence_offset, data.mask_index, batch_size, sequence_length, stream);
    } else {
      sequence_offset = GetCumulatedSequenceLength(data.cumulated_sequence_length_q_cache,
                                                   data.mask_index, batch_size, sequence_length, stream,
                                                   sequence_offset);
    }
    DUMP_TENSOR_D("sequence_offset", sequence_offset, 1, (data.mask_index != nullptr ? 2 : 1) * batch_size + 1);
    CUDA_RETURN_IF_ERROR(hipGetLastError());

    FusedMHARunnerFP16v2* fused_fp16_runner = reinterpret_cast<FusedMHARunnerFP16v2*>(fused_runner);

    const int S = use_fused_causal ? sequence_length : fused_fp16_runner->getSFromMaxSeqLen(sequence_length);

    // B = 2 * batch_size when there is padding in input, and B = batch_size when padding is removed.
    const int B = (nullptr == data.mask_index ? batch_size : 2 * batch_size);

    fused_fp16_runner->setup(S, B);

    if (use_fused_kernel) {
      assert(qkv_format == AttentionQkvFormat::QKV_BSN3H);

      // When there is no bias, we can directly use packed qkv from inputs.
      void const* packed_qkv = qkv;
      if (data.query != nullptr && data.key == nullptr && data.bias == nullptr) {
        packed_qkv = data.query;
      }

      fused_fp16_runner->run(packed_qkv, sequence_offset, data.output, stream);
      DUMP_TENSOR("fused output", data.output, batch_size * sequence_length, num_heads, v_head_size);
    } else {
      assert(qkv_format == AttentionQkvFormat::Q_K_V_BNSH_QKV_BS3NH);
      fused_fp16_runner->run(data.gemm_buffer, sequence_offset, data.output, stream);
      DUMP_TENSOR("fused causal output", data.output, batch_size * sequence_length, num_heads, v_head_size);
    }
    return Status::OK();
  }

  // For raw attention mask, the scalar 1/sqrt(H) is moved to combine with softmax computation.
  const float scale = parameters.scale == 0.0f ? 1.f / sqrt(static_cast<float>(qk_head_size))
                                               : parameters.scale;

#if USE_FLASH_ATTENTION
  if (data.use_memory_efficient_attention) {
    // We only enable fused cross attention when there is no key padding mask.
    // Otherwise, key have effective batch size 2 * batch_size, which is different from batch_size of query.
    assert(qkv_format == AttentionQkvFormat::Q_K_V_BSNH);

    const void* query = q;
    const void* key = k;
    const void* value = v;
    // For packed KV, we can use query input directly.
    if (data.gemm_buffer == nullptr && data.key != nullptr && data.value == nullptr) {
      assert(data.bias == nullptr);
      query = data.query;
    }

    DUMP_TENSOR_D("attention q(BSNH)", q, batch_size * sequence_length, num_heads * qk_head_size);
    DUMP_TENSOR_D("attention k(BSNH)", k, batch_size * sequence_length, num_heads * qk_head_size);
    DUMP_TENSOR_D("attention v(BSNH)", v, batch_size * sequence_length, num_heads * v_head_size);

    MemoryEfficientAttentionParams p;
    p.sm = device_prop.major * 10 + device_prop.minor;
    p.is_half = sizeof(T) == 2;
    p.batch_size = parameters.batch_size;
    p.num_heads = parameters.num_heads;
    p.sequence_length = parameters.sequence_length;
    p.kv_sequence_length = parameters.total_sequence_length;
    p.qk_head_size = parameters.head_size;
    p.v_head_size = parameters.v_head_size;
    p.causal = parameters.is_unidirectional;
    p.scale = scale;
    p.seqlen_k_ptr = nullptr == data.mask_index ? nullptr : const_cast<int32_t*>(reinterpret_cast<const int32_t*>(data.mask_index));
    p.seqstart_q_ptr = nullptr == data.mask_index ? nullptr : const_cast<int32_t*>(reinterpret_cast<const int32_t*>(data.mask_index + batch_size));
    p.seqstart_k_ptr = nullptr == data.mask_index ? nullptr : const_cast<int32_t*>(reinterpret_cast<const int32_t*>(data.mask_index + 2 * batch_size + 1));
    p.query = query;
    p.key = key;
    p.value = value;
    p.attn_bias = nullptr == data.relative_position_bias ? nullptr : data.relative_position_bias;
    p.is_attn_bias_batched = !parameters.broadcast_res_pos_bias;
    p.output = data.output;
    p.workspace = MemoryEfficientAttentionParams::need_workspace(v_head_size, sizeof(T) == sizeof(float)) ? scratch1 : nullptr;
    p.stream = stream;
    run_memory_efficient_attention(p);
    DUMP_TENSOR("attention cutlass output", data.output, batch_size * sequence_length, num_heads, v_head_size);
    return Status::OK();
  }
#endif

  // The following are unfused attention.
  assert(qkv_format == AttentionQkvFormat::Q_K_V_BNSH);
  const int* mask_index = data.mask_index;
  gsl::span<const int64_t>& mask_index_dims = data.mask_index_dims;

  // Raw attention mask could be 2D (BxT) or 3D (BxSxT) or 4D(Bx1xMxM), where M is the max sequence length.
  bool use_raw_attention_mask = (nullptr != mask_index && mask_index_dims.size() >= 2);

  // Compute Q*K' (as K'*Q), scaled by 1/sqrt(H) and store in scratch1: BxNxSxT
  // Q: BxNxSxH, K (present_k): BxNxTxH, Q*K': BxNxSxT
  float one = 1.0f;
  float zero = 0.f;

  float alpha = use_raw_attention_mask ? one : scale;

  hipblasSetStream(cublas, stream);

  DUMP_TENSOR_D("q[BNSH]", q, batch_size, num_heads, sequence_length, qk_head_size);
  DUMP_TENSOR_D("k[BNSH]", k, batch_size, num_heads, total_sequence_length, qk_head_size);
  CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
      cublas, HIPBLAS_OP_T, HIPBLAS_OP_N,
      total_sequence_length, sequence_length, qk_head_size,
      &alpha, k, qk_head_size, present_size_per_batch_k,
      q, qk_head_size, sequence_length * qk_head_size,
      &zero, scratch1, total_sequence_length, sequence_length * total_sequence_length, batches, device_prop));

  DUMP_TENSOR_D("Q", q, batch_size * num_heads, sequence_length, qk_head_size);
  DUMP_TENSOR_D("K", k, batch_size * num_heads, qk_head_size, sequence_length);
  DUMP_TENSOR_D("QK", scratch1, batch_size * num_heads, sequence_length, total_sequence_length);

  const size_t bytes = GetAttentionScratchSize(element_size, batch_size, num_heads,
                                               sequence_length, total_sequence_length);
  T* scratch2 = scratch1 + (bytes / element_size);

  // Apply softmax and store result R to scratch2: BxNxSxT
  if (use_raw_attention_mask) {  // 2d, 3d or 4d attention mask
    const int mask_dimension = static_cast<int>(mask_index_dims.size());

    // For testing, environment variable ORT_TRANSFORMER_OPTIONS=1 could enable persistent softmax used in Torch.
    const TransformerOptions* options = TransformerOptions::GetInstance();
    bool use_persistent_softmax = options->IsPrecisionMode() && !options->DisablePersistentSoftmax();

    T* persistent_softmax_workspace = scratch1;  // replace Q*K' in place with masked score for persistent softmax.
    ORT_RETURN_IF_ERROR(
        ComputeSoftmaxWithRawMask<T>(stream, total_sequence_length, sequence_length, batch_size, num_heads,
                                     mask_index, nullptr, data.relative_position_bias, parameters.broadcast_res_pos_bias,
                                     scratch1, scratch2, parameters.is_unidirectional, scale, mask_dimension,
                                     parameters.max_sequence_length, use_persistent_softmax, persistent_softmax_workspace,
                                     mask_filter_value));
  } else if (nullptr != mask_index) {  // 1d mask index
    assert(mask_index_dims.size() == 1);
    // mask_index has 1D shape: either (batch_size) or (2*batch_size). Only the later one has start postions.
    const int* mask_start = (mask_index_dims[0] > batch_size) ? mask_index + batch_size : nullptr;
    ORT_RETURN_IF_ERROR(ComputeSoftmaxWithMask1D<T>(
        stream, total_sequence_length, sequence_length, batch_size, num_heads,
        mask_index, mask_start, data.relative_position_bias, parameters.broadcast_res_pos_bias,
        scratch1, scratch2, parameters.is_unidirectional));
  } else {  // no mask
    ORT_RETURN_IF_ERROR(
        ComputeSoftmax<T>(stream, total_sequence_length, sequence_length, batch_size, num_heads, data.relative_position_bias,
                          parameters.broadcast_res_pos_bias, scratch1, scratch2, parameters.is_unidirectional));
  }

  DUMP_TENSOR_D("Softmax", scratch2, batch_size * num_heads, sequence_length, total_sequence_length);
  DUMP_TENSOR_D("V", v, batch_size * num_heads, sequence_length, v_head_size);

  // compute R*V (as V*R), and store in temp_output (space used by Q): BxNxSxH_v
  T* temp_output = qkv;
  CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
      cublas, HIPBLAS_OP_N, HIPBLAS_OP_N,
      v_head_size, sequence_length, total_sequence_length,
      &one, v, v_head_size, present_size_per_batch_v,
      scratch2, total_sequence_length, sequence_length * total_sequence_length,
      &zero, temp_output, v_head_size, sequence_length * v_head_size, batches, device_prop));

  // Temp_output is BxNxSxH_v, transpose to output BxSxNxH_v
  Status result = LaunchTransCtx(stream, sequence_length, batch_size, v_head_size, num_heads,
                                 max_threads_per_block, false, temp_output, data.output);
  DUMP_TENSOR("unfused output", data.output, batch_size * sequence_length, num_heads, v_head_size);
  return result;
}

template <typename T>
Status DecoderQkvToContext(
    const hipDeviceProp_t& device_prop,
    hipStream_t stream,
    hipblasHandle_t& cublas,
    const size_t element_size,
    const int batch_size,
    const int sequence_length,
    const int kv_sequence_length,
    const int num_heads,
    const int head_size,
    const bool static_kv,
    const bool use_past,
    const bool has_layer_state,
    const bool has_key_padding_mask,
    const float mask_filter_value,
    const T* gemm_query_buffer,
    const T* gemm_kv_buffer,
    const bool* key_padding_mask,
    const T* key_cache,
    const T* value_cache,
    T* qkv_buffer,
    T* workspace_buffer,
    T* output,
    T* new_key_cache,
    T* new_value_cache) {
  const int max_threads_per_block = device_prop.maxThreadsPerBlock;
  const int BN = batch_size * num_heads;
  const int BHN = BN * head_size;
  const int BNS = BN * sequence_length;
  const int k_buffer_offset = sequence_length * BHN;
  const int v_buffer_offset = (sequence_length + kv_sequence_length) * BHN;

  T* temp_qkv_buffer = workspace_buffer;

  const T* q = qkv_buffer;
  // transpose q and copy them to qkv_buffer
  ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 1, sequence_length, batch_size, head_size, num_heads,
                                     max_threads_per_block, true, gemm_query_buffer, qkv_buffer));

  const T* k = qkv_buffer + k_buffer_offset;
  const T* v = qkv_buffer + v_buffer_offset;
  if (!has_layer_state || !use_past) {
    if (!static_kv) {
      // transpose kv and copy them to qkv_buffer
      ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 2, sequence_length, batch_size, head_size, num_heads,
                                         max_threads_per_block, true, gemm_kv_buffer, qkv_buffer + k_buffer_offset));
    } else {
      // transpose kv and copy them to qkv_buffer
      ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 2, kv_sequence_length, batch_size, head_size, num_heads,
                                         max_threads_per_block, true, gemm_kv_buffer, qkv_buffer + k_buffer_offset));
    }
  } else {
    if (!static_kv) {
      // transpose kv and copy them to temp_buffer
      ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 2, sequence_length, batch_size, head_size, num_heads,
                                         max_threads_per_block, true, gemm_kv_buffer, temp_qkv_buffer));
      // concat cache-k with k and copy to qkv_buffer
      if (nullptr != key_cache) {
        ORT_RETURN_IF_ERROR(LaunchConcatTensorToTensor(stream, kv_sequence_length,
                                                       sequence_length, batch_size, head_size, num_heads,
                                                       max_threads_per_block, 1,
                                                       key_cache,
                                                       temp_qkv_buffer,
                                                       qkv_buffer + k_buffer_offset));
      }
      // concat cache-v with v and copy to qkv_buffer
      if (nullptr != value_cache) {
        ORT_RETURN_IF_ERROR(LaunchConcatTensorToTensor(stream, kv_sequence_length,
                                                       sequence_length, batch_size, head_size, num_heads,
                                                       max_threads_per_block, 1,
                                                       value_cache,
                                                       temp_qkv_buffer + k_buffer_offset,
                                                       qkv_buffer + v_buffer_offset));
      }
    }
  }

  if (has_layer_state) {
    if (use_past && static_kv) {
      CUDA_RETURN_IF_ERROR(hipMemcpyAsync(new_key_cache, key_cache, kv_sequence_length * BHN * sizeof(T),
                                           hipMemcpyDeviceToDevice, stream));
      CUDA_RETURN_IF_ERROR(hipMemcpyAsync(new_value_cache, value_cache, kv_sequence_length * BHN * sizeof(T),
                                           hipMemcpyDeviceToDevice, stream));
    } else {
      CUDA_RETURN_IF_ERROR(hipMemcpyAsync(new_key_cache, k, kv_sequence_length * BHN * sizeof(T),
                                           hipMemcpyDeviceToDevice, stream));
      CUDA_RETURN_IF_ERROR(hipMemcpyAsync(new_value_cache, v, kv_sequence_length * BHN * sizeof(T),
                                           hipMemcpyDeviceToDevice, stream));
    }
  }

  // scratch1: BxNxSxL buffer
  // scratch2: BxNxSxL buffer
  // scratch3: BxNxSxH  buffer
  T* scratch1 = temp_qkv_buffer + 3 * BHN * sequence_length;
  T* scratch2 = scratch1 + BNS * kv_sequence_length;
  T* scratch3 = scratch2 + BNS * kv_sequence_length;

  // compute Q*K' (as K'*Q), scaled by 1/sqrt(H) and store in scratch1: BxNxSxL
  // Q: BxNxSxH, K (present_k): BxNxLxH, Q*K': BxNxSxL
  const float rsqrt_head_size = 1.f / sqrt(static_cast<float>(head_size));
  const int temp_matrix_size = sequence_length * kv_sequence_length;
  float one = 1.0f;
  float zero = 0.f;

  float alpha = rsqrt_head_size;
  const int strideA = kv_sequence_length * head_size;
  const int strideB = sequence_length * head_size;
  if (use_past && static_kv) {
    CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
        cublas, HIPBLAS_OP_T, HIPBLAS_OP_N,
        kv_sequence_length, sequence_length, head_size,
        &alpha, key_cache, head_size, strideA,
        q, head_size, strideB,
        &zero, scratch1, kv_sequence_length, temp_matrix_size, BN, device_prop));
  } else {
    CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
        cublas, HIPBLAS_OP_T, HIPBLAS_OP_N,
        kv_sequence_length, sequence_length, head_size,
        &alpha, k, head_size, strideA,
        q, head_size, strideB,
        &zero, scratch1, kv_sequence_length, temp_matrix_size, BN, device_prop));
  }

  constexpr bool is_unidirectional = false;
  const T* add_before_softmax = nullptr;
  if (has_key_padding_mask) {
    constexpr int mask_dimension = 2;
    constexpr int max_sequence_length = 0;
    ORT_RETURN_IF_ERROR(ComputeSoftmaxWithRawMask<T>(stream, kv_sequence_length, sequence_length, batch_size,
                                                     num_heads, nullptr, key_padding_mask, add_before_softmax,
                                                     false/*broadcast rpb*/, scratch1, scratch2, is_unidirectional,
                                                     1.0f, mask_dimension, max_sequence_length, false, nullptr,
                                                     mask_filter_value));
  } else {
    ORT_RETURN_IF_ERROR(ComputeSoftmax<T>(stream, kv_sequence_length, sequence_length, batch_size, num_heads,
                                          add_before_softmax, false/*broadcast rpb*/, scratch1, scratch2,
                                          is_unidirectional));
  }

  // compute P*V (as V*P), and store in scratch3: BxNxSxH
  if (use_past && static_kv) {
    CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
        cublas, HIPBLAS_OP_N, HIPBLAS_OP_N,
        head_size, sequence_length, kv_sequence_length,
        &one, value_cache, head_size, strideA,
        scratch2, kv_sequence_length, temp_matrix_size,
        &zero, scratch3, head_size, strideB, BN, device_prop));
  } else {
    CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
        cublas, HIPBLAS_OP_N, HIPBLAS_OP_N,
        head_size, sequence_length, kv_sequence_length,
        &one, v, head_size, strideA,
        scratch2, kv_sequence_length, temp_matrix_size,
        &zero, scratch3, head_size, strideB, BN, device_prop));
  }

  // scratch3 is BxNxSxH, transpose to output SxBxNxH
  return LaunchTransCtx(stream, sequence_length, batch_size, head_size, num_heads,
                        max_threads_per_block, true, scratch3, output);
}

Status LaunchDecoderAttentionKernel(
    const hipDeviceProp_t& device_prop,
    hipStream_t stream,
    hipblasHandle_t& cublas,
    const size_t element_size,
    const int batch_size,
    const int sequence_length,
    const int kv_sequence_length,
    const int num_heads,
    const int head_size,
    const bool static_kv,
    const bool use_past,
    const bool has_layer_state,
    const bool has_key_padding_mask,
    const float mask_filter_value,
    const void* gemm_query_buffer,
    const void* gemm_kv_buffer,
    const bool* key_padding_mask,
    const void* key_cache,
    const void* value_cache,
    void* qkv_buffer,
    void* workspace_buffer,
    void* output,
    void* new_key_cache,
    void* new_value_cache) {
  if (element_size == 2) {
    return DecoderQkvToContext(
        device_prop,
        stream,
        cublas,
        element_size,
        batch_size,
        sequence_length,
        kv_sequence_length,
        num_heads,
        head_size,
        static_kv,
        use_past,
        has_layer_state,
        has_key_padding_mask,
        mask_filter_value,
        reinterpret_cast<const half*>(gemm_query_buffer),
        reinterpret_cast<const half*>(gemm_kv_buffer),
        key_padding_mask,
        reinterpret_cast<const half*>(key_cache),
        reinterpret_cast<const half*>(value_cache),
        reinterpret_cast<half*>(qkv_buffer),
        reinterpret_cast<half*>(workspace_buffer),
        reinterpret_cast<half*>(output),
        reinterpret_cast<half*>(new_key_cache),
        reinterpret_cast<half*>(new_value_cache));
  } else {
    return DecoderQkvToContext(
        device_prop,
        stream,
        cublas,
        element_size,
        batch_size,
        sequence_length,
        kv_sequence_length,
        num_heads,
        head_size,
        static_kv,
        use_past,
        has_layer_state,
        has_key_padding_mask,
        mask_filter_value,
        reinterpret_cast<const float*>(gemm_query_buffer),
        reinterpret_cast<const float*>(gemm_kv_buffer),
        key_padding_mask,
        reinterpret_cast<const float*>(key_cache),
        reinterpret_cast<const float*>(value_cache),
        reinterpret_cast<float*>(qkv_buffer),
        reinterpret_cast<float*>(workspace_buffer),
        reinterpret_cast<float*>(output),
        reinterpret_cast<float*>(new_key_cache),
        reinterpret_cast<float*>(new_value_cache));
  }
}

// Template Instantiation
template struct AttentionData<float>;

template struct AttentionData<half>;

template Status QkvToContext<float>(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<float>& data);

template Status QkvToContext<half>(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<half>& data);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
