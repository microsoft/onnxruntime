#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on qkvToContext plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/

Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications: scaling is moved from masked softmax to the gemm before that.
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hipcub/hipcub.hpp>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_math_constants.h>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "attention_impl.h"

using namespace onnxruntime::cuda;
using namespace hipcub;

namespace onnxruntime {
namespace contrib {
namespace cuda {

static size_t AlignTo(size_t a, size_t b) {
  return CeilDiv(a, b) * b;
}

size_t ScratchSize(size_t element_size, int batch_size, int num_heads, int sequence_length) {
  const size_t len = batch_size * num_heads * sequence_length * sequence_length;
  const size_t bytes = len * element_size;

  const size_t alignment = 256;
  const size_t bytesAligned = AlignTo(bytes, alignment);
  return bytesAligned;
}

size_t GetAttentionWorkspaceSize(size_t element_size, int batch_size, int num_heads, int head_size, int sequence_length) {
  size_t qkv_size = 3 * batch_size * sequence_length * num_heads * head_size * element_size;
  return qkv_size + 2 * ScratchSize(element_size, batch_size, num_heads, sequence_length);
}

template <typename T, unsigned TPB>
__device__ inline void Softmax(const int ld, const int num_valid, const T* input, T* output) {
  using BlockReduce = hipcub::BlockReduce<float, TPB>;
  __shared__ typename BlockReduce::TempStorage tmp_storage;

  __shared__ float sum_reverse_block;
  __shared__ float max_block;

  float thread_data_max(-HIP_INF_F);

  // e^x is represented as infinity if x is large enough, like 100.f.
  // Infinity divided by Infinity is a NAN. Thus, softmax gets a NAN if one or more item are large enough.
  // a math transform as below is leveraged to get a stable softmax:
  // e^xi/(e^x1 + ...e^xn) = e^(xi - max) / (e^(x1 - max) + ... + e^(xn - max))
  const int offset = (blockIdx.y * gridDim.x + blockIdx.x) * ld;
  for (int i = threadIdx.x; i < num_valid; i += TPB) {
    const int index = offset + i;
    if (thread_data_max < float(input[index])) {
      thread_data_max = float(input[index]);
    }
  }

  const auto max = BlockReduce(tmp_storage).Reduce(thread_data_max, hipcub::Max());

  // Store max value
  if (threadIdx.x == 0) {
    max_block = max;
  }
  __syncthreads();

  float thread_data_sum(0.f);
  for (int i = threadIdx.x; i < num_valid; i += TPB) {
    const int index = offset + i;
    const float val = input[index];
    thread_data_sum += expf(val - max_block);
  }

  const auto sum = BlockReduce(tmp_storage).Reduce(thread_data_sum, hipcub::Sum());
  if (threadIdx.x == 0) {
    sum_reverse_block = 1.f / sum;
  }
  __syncthreads();

  for (int i = threadIdx.x; i < ld; i += TPB) {
    const int index = offset + i;
    const float val = (i < num_valid) ? expf(float(input[index]) - max_block) * sum_reverse_block : 0.f;
    output[index] = T(val);
  }
}

template <typename T, unsigned TPB>
__device__ inline void SoftmaxSmall(const int ld, const int num_valid, const T* input, T* output) {
  using BlockReduce = hipcub::BlockReduce<float, TPB>;
  __shared__ typename BlockReduce::TempStorage tmp_storage;

  __shared__ float sum_reverse_block;
  __shared__ float max_block;

  const int offset = (blockIdx.y * gridDim.x + blockIdx.x) * ld;
  const int index = offset + threadIdx.x;

  // e^x is represented as infinity if x is large enough, like 100.f.
  // Infinity divided by Infinity is a NAN. Thus, softmax gets a NAN if one or more item are large enough.
  // a math transform as below is leveraged to get a stable softmax:
  // e^xi/(e^x1 + ...e^xn) = e^(xi - max) / (e^(x1 - max) + ... + e^(xn - max))
  float thread_data_max(-HIP_INF_F);
  if (threadIdx.x < num_valid) {
    thread_data_max = input[index];
  }

  const auto max = BlockReduce(tmp_storage).Reduce(thread_data_max, hipcub::Max(), num_valid);

  // Store max value
  if (threadIdx.x == 0) {
    max_block = max;
  }
  __syncthreads();

  float thread_data_exp(0.f);
  if (threadIdx.x < num_valid) {
    const float val = input[index];
    thread_data_exp = expf(val - max_block);
  }

  const auto sum = BlockReduce(tmp_storage).Reduce(thread_data_exp, hipcub::Sum(), num_valid);

  // Store max value
  if (threadIdx.x == 0) {
    sum_reverse_block = (1.f) / sum;
  }
  __syncthreads();

  if (threadIdx.x < ld) {
    // this will be 0 for threadIdx.x >= num_valid
    output[index] = T(thread_data_exp * sum_reverse_block);
  }
}

template <typename T, unsigned TPB>
__global__ void SoftmaxKernelSmall(const int sequence_length, const T* input, T* output) {
  SoftmaxSmall<T, TPB>(sequence_length, sequence_length, input, output);
}

template <typename T, unsigned TPB>
__global__ void SoftmaxKernel(const int sequence_length, const T* input, T* output) {
  Softmax<T, TPB>(sequence_length, sequence_length, input, output);
}

template <typename T>
bool ComputeSoftmax(
    hipStream_t stream, const int sequence_length, const int batch_size, const int num_heads,
    const T* input, T* output) {
  const dim3 grid(sequence_length * num_heads, batch_size, 1);
  if (sequence_length <= 32) {
    const int blockSize = 32;
    SoftmaxKernelSmall<T, blockSize><<<grid, blockSize, 0, stream>>>(sequence_length, input, output);
  } else if (sequence_length <= 128) {
    const int blockSize = 128;
    SoftmaxKernelSmall<T, blockSize><<<grid, blockSize, 0, stream>>>(sequence_length, input, output);
  } else if (sequence_length == 384) {
    const int blockSize = 384;
    SoftmaxKernelSmall<T, blockSize><<<grid, blockSize, 0, stream>>>(sequence_length, input, output);
  } else {
    const int blockSize = 256;
    SoftmaxKernel<T, blockSize><<<grid, blockSize, 0, stream>>>(sequence_length, input, output);
  }

  return CUDA_CALL(hipPeekAtLastError());
}

template <typename T, unsigned TPB>
__global__ void MaskedSoftmaxKernelSmall(const int sequence_length, const int* mask_index, const T* input, T* output) {
  __shared__ int num_valid;

  if (threadIdx.x == 0) {
    num_valid = min(sequence_length, mask_index[blockIdx.y]);
  }
  __syncthreads();

  SoftmaxSmall<T, TPB>(sequence_length, num_valid, input, output);
}

template <typename T, unsigned TPB>
__global__ void MaskedSoftmaxKernel(const int sequence_length, const int* mask_index, const T* input, T* output) {
  __shared__ int num_valid;

  if (threadIdx.x == 0) {
    num_valid = min(sequence_length, mask_index[blockIdx.y]);
  }
  __syncthreads();

  Softmax<T, TPB>(sequence_length, num_valid, input, output);
}

template <typename T>
bool ComputeMaskedSoftmax(hipStream_t stream, const int sequence_length, const int batch_size, const int num_heads,
                          const int* mask_index, const T* input, T* output) {
  // Mask is of length batch_size and assumes the valid region is contiguous starting
  // from the beginning of the sequence

  const dim3 grid(sequence_length * num_heads, batch_size, 1);

  if (sequence_length <= 32) {
    const int blockSize = 32;
    MaskedSoftmaxKernelSmall<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(sequence_length, mask_index, input, output);
  } else if (sequence_length <= 128) {
    const int blockSize = 128;
    MaskedSoftmaxKernelSmall<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(sequence_length, mask_index, input, output);
  } else if (sequence_length == 384) {
    const int blockSize = 384;
    MaskedSoftmaxKernelSmall<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(sequence_length, mask_index, input, output);
  } else {
    const int blockSize = 256;
    MaskedSoftmaxKernel<T, blockSize>
        <<<grid, blockSize, 0, stream>>>(sequence_length, mask_index, input, output);
  }

  return CUDA_CALL(hipPeekAtLastError());
}

template <typename T>
__global__ void TransposeCtx(const int H, const T* input, T* output) {
  // Input:  BxNxSxH
  // Output: BxSxNxH

  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;

  int num_heads = blockDim.y;
  int sequence_length = gridDim.x;

  const int NH = num_heads * H;
  const int NHS = NH * sequence_length;
  const int in_offset = s * H + n * sequence_length * H + b * NHS;
  const int out_offset = n * H + s * NH + b * NHS;

  const int i = threadIdx.x;
  if (i < H) {
    output[out_offset + i] = input[in_offset + i];
  }
}

bool LaunchTransCtx(hipStream_t stream,
                    const int sequence_length, const int batch_size, const int head_size, const int num_heads,
                    const float* input, float* output) {
  const dim3 grid(sequence_length, batch_size, 1);
  if (0 == (head_size & 1)) {
    const int H = head_size / 2;
    const float2* input2 = reinterpret_cast<const float2*>(input);
    float2* output2 = reinterpret_cast<float2*>(output);
    const dim3 block(H, num_heads, 1);
    TransposeCtx<float2><<<grid, block, 0, stream>>>(H, input2, output2);
  } else {
    const dim3 block(head_size, num_heads, 1);
    TransposeCtx<float><<<grid, block, 0, stream>>>(head_size, input, output);
  }
  return CUDA_CALL(hipPeekAtLastError());
}

bool LaunchTransCtx(hipStream_t stream,
                    const int sequence_length, const int batch_size, const int head_size, const int num_heads,
                    const half* input, half* output) {
  const dim3 grid(sequence_length, batch_size, 1);
  if (0 == (head_size % 4)) {
    const int H = head_size / 4;
    const dim3 block(H, num_heads, 1);
    const float2* input2 = reinterpret_cast<const float2*>(input);
    float2* output2 = reinterpret_cast<float2*>(output);
    TransposeCtx<float2><<<grid, block, 0, stream>>>(H, input2, output2);
  } else if (0 == (head_size & 1)) {
    const int H = head_size / 2;
    const dim3 block(H, num_heads, 1);
    const half2* input2 = reinterpret_cast<const half2*>(input);
    half2* output2 = reinterpret_cast<half2*>(output);
    TransposeCtx<half2><<<grid, block, 0, stream>>>(H, input2, output2);
  } else {  // this should be an "odd" case. probably not worth catching it in the half2 kernel.
    const dim3 block(head_size, num_heads, 1);
    TransposeCtx<half><<<grid, block, 0, stream>>>(head_size, input, output);
  }

  return CUDA_CALL(hipPeekAtLastError());
}

template <typename T>
__global__ void TransposeQKV(const int H, const T* input, T* output) {
  // Input:  BxSx3xNxH
  // Output: 3xBxNxSxH

  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;  // matrix id

  const int num_heads = blockDim.y;

  const int sequence_length = gridDim.x;
  const int batch_size = gridDim.y;
  const int NH = num_heads * H;
  const int NHS = NH * sequence_length;
  const int in_offset = n * H + m * NH + s * 3 * NH + b * NHS * 3;
  const int out_offset = s * H + n * sequence_length * H + b * NHS + m * NHS * batch_size;

  const int i = threadIdx.x;
  if (i < H) {
    output[out_offset + i] = input[in_offset + i];
  }
}

bool LaunchTransQkv(hipStream_t stream,
                    const int sequence_length, const int batch_size, const int head_size, const int num_heads,
                    const float* input, float* output) {
  const dim3 grid(sequence_length, batch_size, 3);
  if (0 == (head_size & 1)) {
    const int H = head_size / 2;
    const float2* input2 = reinterpret_cast<const float2*>(input);
    float2* output2 = reinterpret_cast<float2*>(output);
    const dim3 block(H, num_heads, 1);
    TransposeQKV<float2><<<grid, block, 0, stream>>>(H, input2, output2);
  } else {
    const dim3 block(head_size, num_heads, 1);
    TransposeQKV<float><<<grid, block, 0, stream>>>(head_size, input, output);
  }
  return CUDA_CALL(hipPeekAtLastError());
}

bool LaunchTransQkv(hipStream_t stream,
                    const int sequence_length, const int batch_size, const int head_size, const int num_heads,
                    const half* input, half* output) {
  const dim3 grid(sequence_length, batch_size, 3);
  if (0 == (head_size % 4)) {
    const int H = head_size / 4;
    const dim3 block(H, num_heads, 1);
    const float2* input2 = reinterpret_cast<const float2*>(input);
    float2* output2 = reinterpret_cast<float2*>(output);
    TransposeQKV<float2><<<grid, block, 0, stream>>>(H, input2, output2);
  } else if (0 == (head_size & 1)) {
    const int H = head_size / 2;
    const dim3 block(H, num_heads, 1);
    const half2* input2 = reinterpret_cast<const half2*>(input);
    half2* output2 = reinterpret_cast<half2*>(output);
    TransposeQKV<half2><<<grid, block, 0, stream>>>(H, input2, output2);
  } else {  // this should be an "odd" case. probably not worth catching it in the half2 kernel..
    const dim3 block(head_size, num_heads, 1);
    TransposeQKV<half><<<grid, block, 0, stream>>>(head_size, input, output);
  }
  return CUDA_CALL(hipPeekAtLastError());
}

hipblasStatus_t inline CublasGemmStridedBatched(
    hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k, const float alpha,
    const float* A, int lda, long long int strideA, const float* B, int ldb, long long int strideB,
    const float beta, float* C, int ldc, long long int strideC, int batchCount) {
  return hipblasSgemmStridedBatched(
      handle, transa, transb, m, n, k, &alpha, A, lda, strideA, B, ldb, strideB, &beta, C, ldc, strideC, batchCount);
}

hipblasStatus_t inline CublasGemmStridedBatched(
    hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k, const half alpha,
    const half* A, int lda, long long int strideA, const half* B, int ldb, long long int strideB,
    const half beta, half* C, int ldc, long long int strideC, int batchCount) {
  return hipblasHgemmStridedBatched(
      handle, transa, transb, m, n, k, &alpha, A, lda, strideA, B, ldb, strideB, &beta, C, ldc, strideC, batchCount);
}

template <typename T>
bool QkvToContext(
    hipblasHandle_t& cublas, hipStream_t stream,
    const int batch_size, const int sequence_length, const int num_heads, const int head_size, const size_t element_size,
    const T* input, T* output, T* workspace,
    const int* mask_index) {
  const size_t bytes = ScratchSize(element_size, batch_size, num_heads, sequence_length);
  T* scratch1 = workspace;
  T* scratch2 = scratch1 + (bytes / element_size);
  T* scratch3 = scratch2 + (bytes / element_size);

  // input should be BxSx3xNxH => scratch3: 3xBxNxSxH
  if (!LaunchTransQkv(stream, sequence_length, batch_size, head_size, num_heads, input, scratch3)) {
    return false;
  }

  // now scratch3 has Q, K, V: each has size BxNxSxH
  const int batches = batch_size * num_heads;
  const int size_per_batch = sequence_length * head_size;
  const int total_size = batches * size_per_batch;
  const int temp_matrix_size = sequence_length * sequence_length;

  const T* q = scratch3;
  const T* k = q + total_size;
  const T* v = k + total_size;

  hipblasSetStream(cublas, stream);
  CublasMathModeSetter helper(cublas, HIPBLAS_TENSOR_OP_MATH);

  // compute Q*K' (as K'*Q), scaled by 1/sqrt(H) and store in scratch1: BxNxSxS
  const float rsqrt_head_size = 1.f / sqrt(static_cast<float>(head_size));
  if (!CUBLAS_CALL(CublasGemmStridedBatched(
          cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, sequence_length, sequence_length, head_size, rsqrt_head_size, k, head_size, size_per_batch,
          q, head_size, size_per_batch, 0.f, scratch1, sequence_length, temp_matrix_size, batches))) {
    return false;
  }

  // apply softmax and store result P to scratch2: BxNxSxS
  if (nullptr != mask_index) {
    if (!ComputeMaskedSoftmax<T>(stream, sequence_length, batch_size, num_heads, mask_index, scratch1, scratch2)) {
      return false;
    }
  } else {
    if (!ComputeSoftmax<T>(stream, sequence_length, batch_size, num_heads, scratch1, scratch2)) {
      return false;
    }
  }

  // compute P*V (as V*P), and store in scratch3: BxNxSxH
  if (!CUBLAS_CALL(CublasGemmStridedBatched(
          cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, head_size, sequence_length, sequence_length, 1.f, v, head_size, size_per_batch,
          scratch2, sequence_length, temp_matrix_size, 0.f, scratch3, head_size, size_per_batch, batches))) {
    return false;
  }

  // scratch3 is BxNxSxH, transpose to output BxSxNxH
  return LaunchTransCtx(stream, sequence_length, batch_size, head_size, num_heads, scratch3, output);
}

bool LaunchAttentionKernel(
    const void* input,
    const int* mask_index,
    void* output,
    const int batch_size,
    const int sequence_length,
    const int num_heads,
    const int head_size,
    void* workspace,
    hipblasHandle_t& cublas,
    const size_t element_size) {
  // use default stream
  const hipStream_t stream = nullptr;

  if (element_size == 2) {
    return QkvToContext(cublas, stream,
                        batch_size, sequence_length, num_heads, head_size, element_size,
                        reinterpret_cast<const half*>(input), reinterpret_cast<half*>(output), reinterpret_cast<half*>(workspace),
                        mask_index);
  } else {
    return QkvToContext(cublas, stream,
                        batch_size, sequence_length, num_heads, head_size, element_size,
                        reinterpret_cast<const float*>(input), reinterpret_cast<float*>(output), reinterpret_cast<float*>(workspace),
                        mask_index);
  }
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
