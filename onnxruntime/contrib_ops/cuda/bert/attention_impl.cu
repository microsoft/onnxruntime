#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on qkvToContext plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/

Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications:
// (1) support GPT-2 past state, unidirectional mask and 4D attention mask from Megatron
// (2) support 2D attention mask
// (3) allow persistent softmax from PyTorch for debugging purpose.
// (4) support different input hidden size and model hidden size for pruned model
// (5) support different hidden sizes of Q/K and V
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_fp16.h>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/shared_inc/fpgeneric.h"
#include "contrib_ops/cuda/bert/attention_impl.h"
#include "contrib_ops/cuda/bert/attention_softmax.h"
#include "contrib_ops/cuda/bert/transformer_common.h"
#include "contrib_ops/cuda/bert/add_bias_transpose.h"
#include "contrib_ops/cuda/bert/tensorrt_fused_multihead_attention/mha_runner.h"
#include "contrib_ops/cpu/bert/attention_base.h"
#include "contrib_ops/cuda/bert/bert_padding.h"

using namespace onnxruntime::cuda;
using namespace hipcub;

#define CHECK_CUDA(expr) CUDA_RETURN_IF_ERROR(expr)

namespace onnxruntime {
namespace contrib {
namespace cuda {

static size_t AlignTo(size_t a, size_t b) {
  return CeilDiv(a, b) * b;
}

size_t GetAttentionScratchSize(
    size_t element_size,
    size_t batch_size,
    size_t num_heads,
    size_t sequence_length,
    size_t total_sequence_length) {
  const size_t bytes = element_size * batch_size * num_heads * sequence_length * total_sequence_length;

  constexpr size_t alignment = 256;
  const size_t bytesAligned = AlignTo(bytes, alignment);
  return bytesAligned;
}

size_t GetAttentionWorkspaceSize(
    size_t element_size,
    size_t batch_size,
    size_t num_heads,
    size_t qk_head_size,
    size_t v_head_size,
    size_t sequence_length,
    size_t kv_sequence_length,
    size_t total_sequence_length,
    void* fused_runner) {
  const size_t qkv_size = element_size * batch_size * num_heads *
                          ((sequence_length * kv_sequence_length) * qk_head_size + kv_sequence_length * v_head_size);

  if (fused_runner != nullptr) {
    // There are batch_size + 1 offsets Without padding (or padding removed), and 2 * batch_size + 1 with padding.
    size_t sequenceOffsetBytes = sizeof(int) * (2 * batch_size + 1);
    return qkv_size + reinterpret_cast<FusedMHARunnerFP16v2*>(fused_runner)->getWorkspaceSize() + sequenceOffsetBytes;
  }

  return qkv_size + 2 * GetAttentionScratchSize(element_size, batch_size, num_heads, sequence_length,
                                                total_sequence_length);
}

template <typename T>
__global__ void AddBiasTransAppendKvToPresentSmall(
    const T* qkv, const T* biases, T* present,
    const int head_size, const int past_sequence_length, const int max_sequence_length) {
  // Input:  BxSxMxNxH  (Format 1)
  // Output: (2, B, N, [P..P+S) of MaxS, H),
  // B is batch_size, S is sequence_length, M is number of matrices, N is num_heads, H is head_size
  const int n = threadIdx.y;
  const int s = blockIdx.x;
  const int b = blockIdx.y;
  const int N = blockDim.y;
  const int S = gridDim.x;
  const int B = gridDim.y;

  constexpr int M = 3; // Matrix count in qkv
  const int m = blockIdx.z + 1;  // k = 1, v = 2

  const int NH = N * head_size;
  const int NHS = NH * S;

  qkv += (n * head_size + (s * M + m) * NH + b * M * NHS);
  if (biases) {
    biases += (m * NH + n * head_size);
  }

  const int MsH = max_sequence_length * head_size;
  const int NMsH = N * MsH;
  const int BNMsH = B * NMsH;
  present += ((past_sequence_length + s) * head_size + n * MsH + b * NMsH + (m-1) * BNMsH);

  for (int h = threadIdx.x; h < head_size; h += blockDim.x) {
    T bias = (biases ? biases[h] : (T)0.0f);
    present[h] = qkv[h] + bias;
  }
}

template <typename T>
__global__ void AddBiasTransAppendKvToPresent(
    const T* qkv, const T* biases, T* present,
    const int head_size, const int past_sequence_length, const int max_sequence_length) {
  // Input:  BxSxMxNxH  (Format 1)
  // Output: (2, B, N, [P..P+S) of MaxS, H),
  // B is batch_size, S is sequence_length, M is number of matrices, N is num_heads, H is head_size
  const int n = blockIdx.x;
  const int s = blockIdx.y;
  const int b = (blockIdx.z >> 1);
  const int N = gridDim.x;
  const int S = gridDim.y;
  const int B = (gridDim.z >> 1);

  constexpr int M = 3; // Matrix count in qkv
  const int m = (blockIdx.z & 0x1) + 1;  // k = 1, v = 2

  const int NH = N * head_size;
  const int NHS = NH * S;

  qkv += (n * head_size + (s * M + m) * NH + b * M * NHS);
  if (biases) {
    biases += (m * NH + n * head_size);
  }

  const int MsH = max_sequence_length * head_size;
  const int NMsH = N * MsH;
  const int BNMsH = B * NMsH;
  present += ((past_sequence_length + s) * head_size + n * MsH + b * NMsH + (m-1) * BNMsH);

  for (int h = threadIdx.x; h < head_size; h += blockDim.x) {
    T bias = (biases ? biases[h] : (T)0.0f);
    present[h] = qkv[h] + bias;
  }
}

// qkv buffer is merged tensor of shape (B,S,3,N,H), k v is the second/third of the 3.
// bias is of shape (3, NxH) or nullptr
// append to present of (2, B, N, (P..T) of M, H),
template <typename T>
Status LaunchAddBiasTransAppendKvToPresent(hipStream_t stream,
                                           const int max_sequence_length,
                                           const int past_sequence_length,
                                           const int sequence_length,
                                           const int batch_size,
                                           const int head_size,
                                           const int num_heads,
                                           const int max_threads_per_block,
                                           const T* biases,
                                           const T* qkv_buffer,
                                           T* present) {
  ORT_ENFORCE(head_size <= (1 << 30));

  int64_t nh = (int64_t)head_size * num_heads;
  if (nh <= max_threads_per_block) {
    const dim3 grid(sequence_length, batch_size, 2);  // 2 for k and v
    const dim3 block(max_threads_per_block / num_heads, num_heads, 1);

    AddBiasTransAppendKvToPresentSmall<T><<<grid, block, 0, stream>>>(
        qkv_buffer, biases, present, head_size, past_sequence_length, max_sequence_length);
  } else {
    const dim3 grid(num_heads, sequence_length, batch_size * 2);  // 2 for k and v
    const dim3 block(std::min(head_size, max_threads_per_block), 1, 1);
    AddBiasTransAppendKvToPresent<T><<<grid, block, 0, stream>>>(
        qkv_buffer, biases, present, head_size, past_sequence_length, max_sequence_length);
  }

  return CUDA_CALL(hipGetLastError());
}

template Status LaunchAddBiasTransAppendKvToPresent(hipStream_t stream,
                                                    const int max_sequence_length,
                                                    const int total_sequence_length,
                                                    const int sequence_length,
                                                    const int batch_size,
                                                    const int head_size,
                                                    const int num_heads,
                                                    const int max_threads_per_block,
                                                    const float* bias,
                                                    const float* qkv_buffer,
                                                    float* present);

template Status LaunchAddBiasTransAppendKvToPresent(hipStream_t stream,
                                                    const int max_sequence_length,
                                                    const int total_sequence_length,
                                                    const int sequence_length,
                                                    const int batch_size,
                                                    const int head_size,
                                                    const int num_heads,
                                                    const int max_threads_per_block,
                                                    const half* bias,
                                                    const half* qkv_buffer,
                                                    half* present);

template <typename T>
Status QkvToContext(
    const hipDeviceProp_t& prop,
    hipblasHandle_t& cublas,
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data,
    void* fused_runner,
    int past_present_share_buffer) {
  constexpr size_t element_size = sizeof(T);
  const int max_threads_per_block = prop.maxThreadsPerBlock;
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const int kv_sequence_length = parameters.kv_sequence_length;
  const int total_sequence_length = parameters.total_sequence_length;
  const int num_heads = parameters.num_heads;
  const int qk_head_size = parameters.head_size;
  const int v_head_size = parameters.v_head_size;

  T* qkv = data.workspace;
  const int batches = batch_size * num_heads;
  const int size_per_batch_q = sequence_length * qk_head_size;
  const int size_per_batch_k = kv_sequence_length * qk_head_size;
  const int size_per_batch_v = kv_sequence_length * v_head_size;
  const size_t elements_q = static_cast<size_t>(batches) * static_cast<size_t>(size_per_batch_q);
  const size_t elements_k = static_cast<size_t>(batches) * static_cast<size_t>(size_per_batch_k);
  const size_t elements_v = static_cast<size_t>(batches) * static_cast<size_t>(size_per_batch_v);

  // Q, K and V pointers when fused attention if not used
  T* q = qkv;
  T* k = q + elements_q;
  T* v = k + elements_k;

  // For fused TRT attention, qkv need transpose to BxSxNx3xH
  bool use_fused_kernel = (nullptr != fused_runner && data.bias != nullptr);

  if (nullptr != data.gemm_buffer) {
    if (data.bias == nullptr) {
      // gemm_buffer should be BxSx3xNxH => qkv: 3xBxNxSxH
      ORT_ENFORCE(qk_head_size == v_head_size);
      int matrix_to_trans = (past_present_share_buffer ? 1 : 3);
      ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, matrix_to_trans, sequence_length, batch_size, qk_head_size, num_heads,
                                         max_threads_per_block, false, data.gemm_buffer, qkv, 3));
    } else {
      const int format = (use_fused_kernel ? 2 : 1);
      int matrix_to_transpose = ((!use_fused_kernel && past_present_share_buffer) ? 1 : 3);
      // format 1: BxSx(NH + NH + NH_v) => BxNxSxH + BxNxSxH + BxNxSxH_v
      // format 2: BxSx(NH + NH + NH) => BxSxNx(H + H + H)
      LaunchAddBiasTranspose(stream, matrix_to_transpose, format, max_threads_per_block,
                             batch_size, sequence_length, num_heads, qk_head_size,
                             data.gemm_buffer, data.bias, qkv,
                             true, v_head_size, 3);
      CUDA_RETURN_IF_ERROR(hipGetLastError());
    }
  } else {  // gemm_buffer == nullptr
    ORT_ENFORCE(data.query != nullptr && data.key != nullptr && data.value != nullptr && data.bias != nullptr);

    if (use_fused_kernel) {
      ORT_ENFORCE(sequence_length == kv_sequence_length && qk_head_size == v_head_size);

      // Q(BxSxNxH), K (BxSxNxH), V(BxSxNxH) => BxSxNx(H + H + H)
      LaunchAddBiasTransposeTrt(
          stream, max_threads_per_block,
          batch_size, sequence_length,
          num_heads, qk_head_size,
          data.bias, data.query, data.key, data.value, qkv);
    } else {
      // Query(BxSxNxH) => Q (BxNxSxH)
      constexpr int format = 0;
      LaunchAddBiasTranspose<T>(stream, 1, format, max_threads_per_block,
                                batch_size, sequence_length, num_heads, qk_head_size,
                                data.query, data.bias, q,
                                true, -1);

      // Key (BxLxNxH) => K (BxNxLxH)
      LaunchAddBiasTranspose<T>(stream, 1, format, max_threads_per_block,
                                batch_size, kv_sequence_length, num_heads, qk_head_size,
                                data.key, data.bias + num_heads * qk_head_size, k,
                                true, -1);

      // Value (BxLxNxH_v) => K (BxNxLxH_v)
      LaunchAddBiasTranspose<T>(stream, 1, format, max_threads_per_block,
                                batch_size, kv_sequence_length, num_heads, v_head_size,
                                data.value, data.bias + 2 * num_heads * qk_head_size, v,
                                true, -1);
    }

    CUDA_RETURN_IF_ERROR(hipGetLastError());
  }

  T* scratch1;
  scratch1 = qkv + elements_q + elements_k + elements_v;

  T* temp_output = scratch1;
  if (use_fused_kernel) {
    int* sequence_offset = reinterpret_cast<int*>(qkv + elements_q + elements_k + elements_v);
    LaunchTrtSequenceOffset(sequence_offset, data.mask_index, batch_size, sequence_length, stream);
    CUDA_RETURN_IF_ERROR(hipGetLastError());

    FusedMHARunnerFP16v2* fused_fp16_runner = reinterpret_cast<FusedMHARunnerFP16v2*>(fused_runner);

    const int S = fused_fp16_runner->getSFromMaxSeqLen(sequence_length);
    // B = 2 * batch_size when there is padding in input, and B = batch_size when padding is removed.
    const int B = 2 * batch_size;
    fused_fp16_runner->setup(S, B);

    fused_fp16_runner->run(qkv, nullptr, sequence_offset, data.output, nullptr, stream);

    return Status::OK();
  }

  const size_t bytes = GetAttentionScratchSize(element_size, batch_size, num_heads,
                                               sequence_length, total_sequence_length);
  T* scratch2 = scratch1 + (bytes / element_size);

  hipblasSetStream(cublas, stream);

  int present_size_per_batch_k = 0;
  int present_size_per_batch_v = 0;
  if (!past_present_share_buffer) {
    // Concat past key value to present (2xBxNxLxH), where L is kv_sequence_length and T is total_sequence_length.
    // past_k (BxNxPxH) + k (BxNxLxH) => present_k (BxNxTxH)
    // past_v (BxNxPxH) + v (BxNxLxH) => present_v (BxNxTxH)
    // When there is past state, the head size for Q/K/V shall be same: H == H_v.
    present_size_per_batch_k = total_sequence_length * qk_head_size;
    present_size_per_batch_v = total_sequence_length * v_head_size;

    if (nullptr != data.present) {
      ORT_RETURN_IF_ERROR(
          LaunchConcatPastToPresent(stream, total_sequence_length, sequence_length, batch_size, qk_head_size, num_heads,
                                    max_threads_per_block, data.past, k, data.present));

      // Update pointers to present_k and present_v.
      k = data.present;
      v = data.present + batches * present_size_per_batch_k;
    }

  } else {
    ORT_ENFORCE(qk_head_size == v_head_size);
    if (data.present != data.past) {
      // For easy testing. Production should better avoid this path.
      int64_t kv_size = 2LL * (int64_t)batch_size * num_heads * parameters.max_sequence_length * qk_head_size;
      hipMemcpyAsync(data.present, data.past, kv_size * sizeof(T), hipMemcpyDeviceToDevice, stream);
    }
    // append last k v to present
    ORT_RETURN_IF_ERROR(LaunchAddBiasTransAppendKvToPresent(
        stream, parameters.max_sequence_length, parameters.past_sequence_length, sequence_length,
        batch_size, qk_head_size, num_heads, max_threads_per_block,
        data.bias, data.gemm_buffer, data.present));

    present_size_per_batch_k = present_size_per_batch_v = parameters.max_sequence_length * qk_head_size;
    k = data.present;
    v = data.present + batches * present_size_per_batch_k;
  }

  const int* mask_index = data.mask_index;
  gsl::span<const int64_t>& mask_index_dims = data.mask_index_dims;

  // Raw attention mask could be 2D (BxT) or 3D (BxSxT) or 4D(Bx1xMxM), where M is the max sequence length.
  bool use_raw_attention_mask = (nullptr != mask_index && mask_index_dims.size() >= 2);

  // Compute Q*K' (as K'*Q), scaled by 1/sqrt(H) and store in scratch1: BxNxSxT
  // Q: BxNxSxH, K (present_k): BxNxTxH, Q*K': BxNxSxT
  const float rsqrt_head_size = 1.f / sqrt(static_cast<float>(qk_head_size));
  const int temp_matrix_size = sequence_length * total_sequence_length;
  float one = 1.0f;
  float zero = 0.f;

  // For raw attention mask, the scalar 1/sqrt(H) is moved to combine with softmax computation.
  float alpha = use_raw_attention_mask ? one : rsqrt_head_size;

  CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
      cublas, HIPBLAS_OP_T, HIPBLAS_OP_N,
      total_sequence_length, sequence_length, qk_head_size,
      &alpha, k, qk_head_size, present_size_per_batch_k,
      q, qk_head_size, sequence_length * qk_head_size,
      &zero, scratch1, total_sequence_length, temp_matrix_size, batches, prop));

  // Apply softmax and store result R to scratch2: BxNxSxT
  if (use_raw_attention_mask) {  // 2d, 3d or 4d attention mask
    const int mask_dimension = static_cast<int>(mask_index_dims.size());

    // For testing, environment variable ORT_TRANSFORMER_OPTIONS=1 could enable persistent softmax used in Torch.
    const TransformerOptions* options = TransformerOptions::GetInstance();
    bool use_persistent_softmax = options->IsPrecisionMode() && !options->DisablePersistentSoftmax();

    T* persistent_softmax_workspace = scratch1;  // replace Q*K' in place with masked score for persistent softmax.
    ORT_RETURN_IF_ERROR(
        ComputeSoftmaxWithRawMask<T>(stream, total_sequence_length, sequence_length, batch_size, num_heads,
                                     mask_index, nullptr, data.extra_add_qk, scratch1, scratch2,
                                     parameters.is_unidirectional, rsqrt_head_size, mask_dimension,
                                     parameters.max_sequence_length,
                                     use_persistent_softmax, persistent_softmax_workspace));
  } else if (nullptr != mask_index) {  // 1d mask index
    ORT_ENFORCE(mask_index_dims.size() == 1);
    // mask_index has 1D shape: either (batch_size) or (2*batch_size). Only the later one has start postions.
    const int* mask_start = (mask_index_dims[0] > batch_size) ? mask_index + batch_size : nullptr;
    ORT_RETURN_IF_ERROR(ComputeSoftmaxWithMask1D<T>(
        stream, total_sequence_length, sequence_length, batch_size, num_heads,
        mask_index, mask_start, data.extra_add_qk, scratch1, scratch2, parameters.is_unidirectional));
  } else {  // no mask
    ORT_RETURN_IF_ERROR(
        ComputeSoftmax<T>(stream, total_sequence_length, sequence_length, batch_size, num_heads, data.extra_add_qk,
                          scratch1, scratch2, parameters.is_unidirectional));
  }

  // compute R*V (as V*R), and store in temp_output (space used by Q): BxNxSxH_v
  temp_output = qkv;
  CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
      cublas, HIPBLAS_OP_N, HIPBLAS_OP_N,
      v_head_size, sequence_length, total_sequence_length,
      &one, v, v_head_size, present_size_per_batch_v,
      scratch2, total_sequence_length, temp_matrix_size,
      &zero, temp_output, v_head_size, size_per_batch_v, batches, prop));

  // Temp_output is BxNxSxH_v, transpose to output BxSxNxH_v
  return LaunchTransCtx(stream, sequence_length, batch_size, v_head_size, num_heads,
                        max_threads_per_block, false, temp_output, data.output);
}

template <typename T>
Status DecoderQkvToContext(
    const hipDeviceProp_t& prop,
    hipStream_t stream,
    hipblasHandle_t& cublas,
    const size_t element_size,
    const int batch_size,
    const int sequence_length,
    const int kv_sequence_length,
    const int num_heads,
    const int head_size,
    const bool static_kv,
    const bool use_past,
    const bool has_layer_state,
    const bool has_key_padding_mask,
    const T* gemm_query_buffer,
    const T* gemm_kv_buffer,
    const bool* key_padding_mask,
    const T* key_cache,
    const T* value_cache,
    T* qkv_buffer,
    T* workspace_buffer,
    T* output,
    T* new_key_cache,
    T* new_value_cache) {
  const int max_threads_per_block = prop.maxThreadsPerBlock;
  const int BN = batch_size * num_heads;
  const int BHN = BN * head_size;
  const int BNS = BN * sequence_length;
  const int k_buffer_offset = sequence_length * BHN;
  const int v_buffer_offset = (sequence_length + kv_sequence_length) * BHN;

  T* temp_qkv_buffer = workspace_buffer;

  const T* q = qkv_buffer;
  // transpose q and copy them to qkv_buffer
  ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 1, sequence_length, batch_size, head_size, num_heads,
                                     max_threads_per_block, true, gemm_query_buffer, qkv_buffer));

  const T* k = qkv_buffer + k_buffer_offset;
  const T* v = qkv_buffer + v_buffer_offset;
  if (!has_layer_state || !use_past) {
    if (!static_kv) {
      // transpose kv and copy them to qkv_buffer
      ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 2, sequence_length, batch_size, head_size, num_heads,
                                         max_threads_per_block, true, gemm_kv_buffer, qkv_buffer + k_buffer_offset));
    } else {
      // transpose kv and copy them to qkv_buffer
      ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 2, kv_sequence_length, batch_size, head_size, num_heads,
                                         max_threads_per_block, true, gemm_kv_buffer, qkv_buffer + k_buffer_offset));
    }
  } else {
    if (!static_kv) {
      // transpose kv and copy them to temp_buffer
      ORT_RETURN_IF_ERROR(LaunchTransQkv(stream, 2, sequence_length, batch_size, head_size, num_heads,
                                         max_threads_per_block, true, gemm_kv_buffer, temp_qkv_buffer));
      // concat cache-k with k and copy to qkv_buffer
      if (nullptr != key_cache) {
        ORT_RETURN_IF_ERROR(LaunchConcatTensorToTensor(stream, kv_sequence_length,
                                                       sequence_length, batch_size, head_size, num_heads,
                                                       max_threads_per_block, 1,
                                                       key_cache,
                                                       temp_qkv_buffer,
                                                       qkv_buffer + k_buffer_offset));
      }
      // concat cache-v with v and copy to qkv_buffer
      if (nullptr != value_cache) {
        ORT_RETURN_IF_ERROR(LaunchConcatTensorToTensor(stream, kv_sequence_length,
                                                       sequence_length, batch_size, head_size, num_heads,
                                                       max_threads_per_block, 1,
                                                       value_cache,
                                                       temp_qkv_buffer + k_buffer_offset,
                                                       qkv_buffer + v_buffer_offset));
      }
    }
  }

  if (has_layer_state) {
    if (use_past && static_kv) {
      CHECK_CUDA(hipMemcpyAsync(new_key_cache, key_cache, kv_sequence_length * BHN * sizeof(T),
                                 hipMemcpyDeviceToDevice, stream));
      CHECK_CUDA(hipMemcpyAsync(new_value_cache, value_cache, kv_sequence_length * BHN * sizeof(T),
                                 hipMemcpyDeviceToDevice, stream));
    } else {
      CHECK_CUDA(hipMemcpyAsync(new_key_cache, k, kv_sequence_length * BHN * sizeof(T),
                                 hipMemcpyDeviceToDevice, stream));
      CHECK_CUDA(hipMemcpyAsync(new_value_cache, v, kv_sequence_length * BHN * sizeof(T),
                                 hipMemcpyDeviceToDevice, stream));
    }
  }

  // scratch1: BxNxSxL buffer
  // scratch2: BxNxSxL buffer
  // scratch3: BxNxSxH  buffer
  T* scratch1 = temp_qkv_buffer + 3 * BHN * sequence_length;
  T* scratch2 = scratch1 + BNS * kv_sequence_length;
  T* scratch3 = scratch2 + BNS * kv_sequence_length;

  // compute Q*K' (as K'*Q), scaled by 1/sqrt(H) and store in scratch1: BxNxSxL
  // Q: BxNxSxH, K (present_k): BxNxLxH, Q*K': BxNxSxL
  const float rsqrt_head_size = 1.f / sqrt(static_cast<float>(head_size));
  const int temp_matrix_size = sequence_length * kv_sequence_length;
  float one = 1.0f;
  float zero = 0.f;

  float alpha = rsqrt_head_size;
  const int strideA = kv_sequence_length * head_size;
  const int strideB = sequence_length * head_size;
  if (use_past && static_kv) {
    CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
        cublas, HIPBLAS_OP_T, HIPBLAS_OP_N,
        kv_sequence_length, sequence_length, head_size,
        &alpha, key_cache, head_size, strideA,
        q, head_size, strideB,
        &zero, scratch1, kv_sequence_length, temp_matrix_size, BN, prop));
  } else {
    CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
        cublas, HIPBLAS_OP_T, HIPBLAS_OP_N,
        kv_sequence_length, sequence_length, head_size,
        &alpha, k, head_size, strideA,
        q, head_size, strideB,
        &zero, scratch1, kv_sequence_length, temp_matrix_size, BN, prop));
  }

  constexpr bool is_unidirectional = false;
  const T* add_before_softmax = nullptr;
  if (has_key_padding_mask) {
    constexpr int mask_dimension = 2;
    constexpr int max_sequence_length = 0;
    ORT_RETURN_IF_ERROR(ComputeSoftmaxWithRawMask<T>(stream, kv_sequence_length, sequence_length, batch_size, num_heads,
                                                     nullptr, key_padding_mask, add_before_softmax, scratch1, scratch2,
                                                     is_unidirectional, 1.0f, mask_dimension, max_sequence_length,
                                                     false, nullptr));
  } else {
    ORT_RETURN_IF_ERROR(ComputeSoftmax<T>(stream, kv_sequence_length, sequence_length, batch_size, num_heads,
                                          add_before_softmax, scratch1, scratch2, is_unidirectional));
  }

  // compute P*V (as V*P), and store in scratch3: BxNxSxH
  if (use_past && static_kv) {
    CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
        cublas, HIPBLAS_OP_N, HIPBLAS_OP_N,
        head_size, sequence_length, kv_sequence_length,
        &one, value_cache, head_size, strideA,
        scratch2, kv_sequence_length, temp_matrix_size,
        &zero, scratch3, head_size, strideB, BN, prop));
  } else {
    CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
        cublas, HIPBLAS_OP_N, HIPBLAS_OP_N,
        head_size, sequence_length, kv_sequence_length,
        &one, v, head_size, strideA,
        scratch2, kv_sequence_length, temp_matrix_size,
        &zero, scratch3, head_size, strideB, BN, prop));
  }

  // scratch3 is BxNxSxH, transpose to output SxBxNxH
  return LaunchTransCtx(stream, sequence_length, batch_size, head_size, num_heads,
                        max_threads_per_block, true, scratch3, output);
}

Status LaunchDecoderAttentionKernel(
    const hipDeviceProp_t& prop,
    hipStream_t stream,
    hipblasHandle_t& cublas,
    const size_t element_size,
    const int batch_size,
    const int sequence_length,
    const int kv_sequence_length,
    const int num_heads,
    const int head_size,
    const bool static_kv,
    const bool use_past,
    const bool has_layer_state,
    const bool has_key_padding_mask,
    const void* gemm_query_buffer,
    const void* gemm_kv_buffer,
    const bool* key_padding_mask,
    const void* key_cache,
    const void* value_cache,
    void* qkv_buffer,
    void* workspace_buffer,
    void* output,
    void* new_key_cache,
    void* new_value_cache) {
  if (element_size == 2) {
    return DecoderQkvToContext(
        prop,
        stream,
        cublas,
        element_size,
        batch_size,
        sequence_length,
        kv_sequence_length,
        num_heads,
        head_size,
        static_kv,
        use_past,
        has_layer_state,
        has_key_padding_mask,
        reinterpret_cast<const half*>(gemm_query_buffer),
        reinterpret_cast<const half*>(gemm_kv_buffer),
        key_padding_mask,
        reinterpret_cast<const half*>(key_cache),
        reinterpret_cast<const half*>(value_cache),
        reinterpret_cast<half*>(qkv_buffer),
        reinterpret_cast<half*>(workspace_buffer),
        reinterpret_cast<half*>(output),
        reinterpret_cast<half*>(new_key_cache),
        reinterpret_cast<half*>(new_value_cache));
  } else {
    return DecoderQkvToContext(
        prop,
        stream,
        cublas,
        element_size,
        batch_size,
        sequence_length,
        kv_sequence_length,
        num_heads,
        head_size,
        static_kv,
        use_past,
        has_layer_state,
        has_key_padding_mask,
        reinterpret_cast<const float*>(gemm_query_buffer),
        reinterpret_cast<const float*>(gemm_kv_buffer),
        key_padding_mask,
        reinterpret_cast<const float*>(key_cache),
        reinterpret_cast<const float*>(value_cache),
        reinterpret_cast<float*>(qkv_buffer),
        reinterpret_cast<float*>(workspace_buffer),
        reinterpret_cast<float*>(output),
        reinterpret_cast<float*>(new_key_cache),
        reinterpret_cast<float*>(new_value_cache));
  }
}

// Template Instantiation
template struct AttentionData<float>;

template struct AttentionData<half>;

template Status QkvToContext<float>(
    const hipDeviceProp_t& prop,
    hipblasHandle_t& cublas,
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<float>& data,
    void* fused_runner,
    int past_present_share_buffer);

template Status QkvToContext<half>(
    const hipDeviceProp_t& prop,
    hipblasHandle_t& cublas,
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<half>& data,
    void* fused_runner,
    int past_present_share_buffer);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
