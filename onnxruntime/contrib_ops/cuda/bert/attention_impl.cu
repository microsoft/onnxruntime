#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on qkvToContext plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/

Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications:
// (1) support GPT-2 past state, unidirectional mask and 4D attention mask from Megatron
// (2) support 2D attention mask
// (3) allow persistent softmax from PyTorch for debugging purpose.
// (4) support different input hidden size and model hidden size for pruned model
// (5) support different hidden sizes of Q/K and V
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/shared_inc/fpgeneric.h"
#include "core/platform/env_var_utils.h"
#include "contrib_ops/cpu/bert/attention_base.h"
#include "contrib_ops/cpu/bert/attention_common.h"
#include "contrib_ops/cpu/bert/attention_parameters.h"
#include "contrib_ops/cuda/bert/attention_impl.h"
#include "contrib_ops/cuda/bert/attention_kv_cache.h"
#include "contrib_ops/cuda/bert/attention_qk.h"
#include "contrib_ops/cuda/bert/attention_softmax.h"
#include "contrib_ops/cuda/bert/bert_padding.h"
#include "contrib_ops/cuda/bert/cutlass_fmha/memory_efficient_attention.h"
#include "contrib_ops/cuda/bert/cudnn_fmha/cudnn_flash_attention.h"
#include "contrib_ops/cuda/bert/flash_attention/flash_api.h"
#include "contrib_ops/cuda/bert/lean_attention/lean_api.h"
#include "contrib_ops/cuda/bert/tensorrt_fused_multihead_attention/mha_runner.h"
#include "contrib_ops/cuda/bert/tensorrt_fused_multihead_attention/cross_attention/fmha_cross_attention.h"
#include "contrib_ops/cuda/bert/transformer_common.h"
#include "contrib_ops/cuda/utils/dump_cuda_tensor.h"

using namespace onnxruntime::cuda;
using namespace onnxruntime::contrib::attention_softmax_cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

constexpr size_t kMemoryAlignment = 256;

static size_t AlignTo(size_t a, size_t b) {
  return CeilDiv(a, b) * b;
}

size_t AlignSize(size_t bytes) {
  const size_t bytesAligned = AlignTo(bytes, kMemoryAlignment);
  return bytesAligned;
}

const int32_t* CumulatedSequenceLengthCache::TryGet(int batch_size, int32_t seq_len, hipStream_t stream) {
  if (this->sequence_length == 0 && seq_len > 0) {
    // Initialize only once with sequence length in the first request.
    std::call_once(init_once_flag_, [&]() {
      ORT_ENFORCE(buffer.get() != nullptr && this->max_batch_size > 0);
      LaunchTrtSequenceOffset(reinterpret_cast<int32_t*>(buffer.get()), nullptr,
                              this->max_batch_size, seq_len, stream);
      // Syncronize to ensure thread-safe since other thread will not wait for the above kernel finish.
      // Otherwise, the data might be consumed by other threads before it is ready and causes data race issue.
      hipStreamSynchronize(stream);
      this->sequence_length = seq_len;
    });
  }

  if (this->sequence_length == seq_len && batch_size <= this->max_batch_size) {
    return reinterpret_cast<const int32_t*>(buffer.get());
  }

  return nullptr;
}

size_t GetAttentionScratchSize(
    size_t element_size,
    size_t batch_size,
    size_t num_heads,
    size_t sequence_length,
    size_t total_sequence_length) {
  const size_t bytes = element_size * batch_size * num_heads * sequence_length * total_sequence_length;
  return AlignSize(bytes);
}

size_t GetSequenceOffsetSize(int batch_size, bool has_padding) {
  // There are batch_size + 1 offsets Without padding (or padding removed), and 2 * batch_size + 1 with padding.
  size_t bytes = sizeof(int) * ((has_padding ? 2 * batch_size : batch_size) + 1);
  return AlignSize(bytes);
  ;
}

size_t GetAttentionWorkspaceSize(
    size_t element_size,
    size_t batch_size,
    size_t num_heads,
    size_t qk_head_size,
    size_t v_head_size,
    size_t sequence_length,
    size_t kv_sequence_length,
    size_t total_sequence_length,
    void* fused_runner,
    bool use_flash_attention,
    bool use_lean_attention,
    bool use_fused_cross_attention,
    bool use_memory_efficient_attention,
    bool use_cudnn_flash_attention,
    bool no_qkv_workspace) {
  // Note that q, k and v might need alignment for fused attention kernels.
  const size_t qkv_size = element_size * batch_size * num_heads *
                          ((sequence_length + kv_sequence_length) * qk_head_size + kv_sequence_length * v_head_size);
  const size_t qkv_bytes = no_qkv_workspace ? 0 : qkv_size;

#if USE_FLASH_ATTENTION
  if (use_flash_attention) {
    return qkv_bytes;
  }
#else
  ORT_UNUSED_PARAMETER(use_flash_attention);
#endif

#if USE_LEAN_ATTENTION
  if (use_lean_attention) {
    return qkv_bytes;
  }
#else
  ORT_UNUSED_PARAMETER(use_lean_attention);
#endif

#if USE_MEMORY_EFFICIENT_ATTENTION
  if (use_memory_efficient_attention) {
    size_t fmha_buffer_bytes = 0;
    if (MemoryEfficientAttentionParams::need_workspace(v_head_size, element_size == sizeof(float))) {
      fmha_buffer_bytes = batch_size * sequence_length * num_heads * v_head_size * sizeof(float);
    }

    return qkv_bytes + fmha_buffer_bytes;
  }
#else
  ORT_UNUSED_PARAMETER(use_memory_efficient_attention);
#endif

  if (fused_runner != nullptr) {
    return qkv_bytes + GetSequenceOffsetSize(static_cast<int>(batch_size), true);
  }

  if (use_fused_cross_attention) {
    return qkv_bytes + 2 * GetSequenceOffsetSize(static_cast<int>(batch_size), true);
  }

  if (use_cudnn_flash_attention) {
    return qkv_bytes;
  }

  return qkv_bytes + 2 * GetAttentionScratchSize(element_size, batch_size, num_heads, sequence_length,
                                                 total_sequence_length);
}

template <typename T>
Status FusedTrtCrossAttention(
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data) {
  assert(data.qkv_format == AttentionQkvFormat::Q_KV_BSNH_BSN2H);

  // We only enable fused cross attention when there is no key padding mask.
  // Otherwise, key have effective batch size 2 * batch_size, which is different from batch_size of query.
  assert(data.mask_index == nullptr);
  assert(data.scratch != nullptr);
  assert(data.q != nullptr);
  assert(data.k != nullptr);

#ifndef NDEBUG
  char* scratch_end = reinterpret_cast<char*>(data.scratch) + 2 * GetSequenceOffsetSize(parameters.batch_size, false);
  char* buffer_end = reinterpret_cast<char*>(data.workspace) + data.workspace_bytes;
  assert(scratch_end <= buffer_end);
#endif
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;

  int32_t* q_sequence_offset = const_cast<int32_t*>(data.cumulated_sequence_length_q_cache);
  if (q_sequence_offset == nullptr) {
    q_sequence_offset = reinterpret_cast<int*>(data.scratch);
    LaunchTrtSequenceOffset(q_sequence_offset, data.mask_index, batch_size, sequence_length, stream);
  }

  CUDA_RETURN_IF_ERROR(hipGetLastError());
  DUMP_TENSOR_INIT();
  DUMP_TENSOR_D("q_sequence_offset", q_sequence_offset, 1, batch_size + 1);

  int32_t* kv_sequence_offset = const_cast<int32_t*>(data.cumulated_sequence_length_kv_cache);
  if (kv_sequence_offset == nullptr) {
    int* scratch = reinterpret_cast<int*>(data.scratch) + (GetSequenceOffsetSize(batch_size, false) / sizeof(int));
    kv_sequence_offset = reinterpret_cast<int*>(scratch);
    LaunchTrtSequenceOffset(kv_sequence_offset, data.mask_index, batch_size, parameters.kv_sequence_length, stream);
  }

  CUDA_RETURN_IF_ERROR(hipGetLastError());
  DUMP_TENSOR_D("kv_sequence_offset", kv_sequence_offset, 1, batch_size + 1);

  FusedMultiHeadCrossAttentionKernel const* cross_attention_kernel =
      reinterpret_cast<FusedMultiHeadCrossAttentionKernel const*>(data.fused_cross_attention_kernel);

  run_fused_cross_attention(
      data.q,                         // Q
      data.k,                         // packed KV
      q_sequence_offset,              // cumulated sequence length of Q
      kv_sequence_offset,             // cumulated sequence length of KV
      data.output,                    // output
      cross_attention_kernel,         // kernels
      batch_size,                     // batch size
      parameters.num_heads,           // number of heads
      parameters.head_size,           // head size of Q/K/V
      sequence_length,                // sequence length of Q
      parameters.kv_sequence_length,  // sequence length of KV
      stream);

  return Status::OK();
}

template <>
Status FusedTrtCrossAttention<float>(
    hipStream_t /*stream*/,
    contrib::AttentionParameters& /*parameters*/,
    AttentionData<float>& /*data*/) {
  return ORT_MAKE_STATUS(ONNXRUNTIME, StatusCode::NOT_IMPLEMENTED,
                         "Trt fused cross attention does not support float tensor");
}

template <typename T>
Status FusedTrtSelfAttention(
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data) {
  assert(data.scratch != nullptr);
#ifndef NDEBUG
  char* scratch_end = reinterpret_cast<char*>(data.scratch) + GetSequenceOffsetSize(parameters.batch_size, false);
  char* buffer_end = reinterpret_cast<char*>(data.workspace) + data.workspace_bytes;
  assert(scratch_end <= buffer_end);
#endif

  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const bool causal = parameters.is_unidirectional;

  const int32_t* sequence_offset = data.cumulated_sequence_length_q_cache;
  if (parameters.mask_type == AttentionMaskType::MASK_2D_KEY_PADDING) {
    LaunchTrtSequenceOffset2d(reinterpret_cast<int*>(data.scratch), data.mask_index, batch_size, sequence_length, stream);
    sequence_offset = reinterpret_cast<const int*>(data.scratch);
  } else {
    if (sequence_offset == nullptr) {
      LaunchTrtSequenceOffset(reinterpret_cast<int*>(data.scratch), data.mask_index, batch_size, sequence_length, stream);
      sequence_offset = reinterpret_cast<const int*>(data.scratch);
    }
  }

  CUDA_RETURN_IF_ERROR(hipGetLastError());

  DUMP_TENSOR_INIT();
  DUMP_TENSOR_D("sequence_offset", sequence_offset, 1, (data.mask_index != nullptr ? 2 : 1) * batch_size + 1);

  FusedMHARunnerFP16v2* fused_fp16_runner = reinterpret_cast<FusedMHARunnerFP16v2*>(data.fused_runner);

  const int s = causal ? sequence_length : fused_fp16_runner->NormalizeSequenceLength(sequence_length);

  // B = 2 * batch_size when there is padding in input, and B = batch_size when padding is removed.
  const int b = (nullptr == data.mask_index ? batch_size : 2 * batch_size);

  if (!causal) {
    assert(data.qkv_format == AttentionQkvFormat::QKV_BSN3H);
    fused_fp16_runner->Run(b, s, data.q, sequence_offset, data.output, stream);
  } else {
    assert(data.qkv_format == AttentionQkvFormat::Q_K_V_BNSH_QKV_BS3NH);
    fused_fp16_runner->Run(b, s, data.gemm_buffer, sequence_offset, data.output, stream);
  }

  return Status::OK();
}

// Template Specialization for float type
template <>
Status FusedTrtSelfAttention<float>(
    hipStream_t /*stream*/,
    contrib::AttentionParameters& /*parameters*/,
    AttentionData<float>& /*data*/) {
  return ORT_MAKE_STATUS(ONNXRUNTIME, StatusCode::NOT_IMPLEMENTED,
                         "Trt fused attention does not support float tensor");
}

#if USE_FLASH_ATTENTION
template <typename T>
Status FlashAttention(
    const hipDeviceProp_t& device_prop,
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data,
    float scale) {
  assert(data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH ||
         data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH_BNSH_BNSH);
  assert(nullptr == data.mask_index);
  assert(nullptr == data.attention_bias);
  assert(parameters.head_size == parameters.v_head_size);

  constexpr bool is_bf16 = false;
  ORT_RETURN_IF_ERROR(onnxruntime::flash::mha_fwd(
      device_prop, stream, data.q, data.k, data.v, data.output, reinterpret_cast<void*>(data.softmax_lse),
      parameters.batch_size, parameters.num_heads, parameters.num_heads, parameters.head_size,
      parameters.sequence_length, parameters.total_sequence_length, scale, 0.0, parameters.is_unidirectional, is_bf16,
      false, data.num_splits, reinterpret_cast<void*>(data.softmax_lse_accum),
      reinterpret_cast<void*>(data.out_accum), data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH));

  return Status::OK();
}

template <>
Status FlashAttention(
    const hipDeviceProp_t& device_prop,
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<float>& data,
    float scale) {
  ORT_UNUSED_PARAMETER(device_prop);
  ORT_UNUSED_PARAMETER(stream);
  ORT_UNUSED_PARAMETER(parameters);
  ORT_UNUSED_PARAMETER(data);
  ORT_UNUSED_PARAMETER(scale);
  return ORT_MAKE_STATUS(ONNXRUNTIME, StatusCode::NOT_IMPLEMENTED, "flash attention does not support float tensor");
}
#endif

#if USE_LEAN_ATTENTION
template <typename T>
Status LeanAttention(
    const hipDeviceProp_t& device_prop,
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data,
    float scale) {
  assert(data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH ||
         data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH_BNSH_BNSH);
  assert(nullptr == data.mask_index);
  assert(nullptr == data.attention_bias);
  assert(parameters.head_size == parameters.v_head_size);

  constexpr bool is_bf16 = false;

  ORT_RETURN_IF_ERROR(onnxruntime::lean::mha_fwd_kvcache(
    device_prop, stream,
    data.q,
    data.k, // k_cache
    data.v, // v_cache
    nullptr,  // new_k (we have appended new_k to k_cache)
    nullptr,  // new_v (we have appended new_v to k_cache)
    data.output,
    reinterpret_cast<void*>(data.softmax_lse),
    nullptr, // seqlens_k
    nullptr, // cos_cache
    nullptr, // sin_cache
    nullptr, // block_table
    parameters.batch_size,
    parameters.num_heads,
    parameters.num_heads, // num_heads_k
    parameters.head_size,
    parameters.sequence_length, // seqlen_q
    parameters.total_sequence_length, // seqlen_k
    0, // seqlen_k_new
    0, // rotary_dim
    scale, // softmax_scale
    parameters.is_unidirectional,
    is_bf16,
    false, // past_bsnh
    data.num_splits,
    data.grid_dim_z,
    data.max_tiles_per_tb,
    data.high_load_tbs,
    data.tiles_per_head,
    reinterpret_cast<void*>(data.softmax_lse_accum),
    reinterpret_cast<void*>(data.out_accum),
    data.lean_sync_flag,
    -1, // local_window_size
    false, // is_rotary_interleaved
    false // is_packed_qkv
    ));

  return Status::OK();
}

template <>
Status LeanAttention(
    const hipDeviceProp_t& device_prop,
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<float>& data,
    float scale) {
  ORT_UNUSED_PARAMETER(device_prop);
  ORT_UNUSED_PARAMETER(stream);
  ORT_UNUSED_PARAMETER(parameters);
  ORT_UNUSED_PARAMETER(data);
  ORT_UNUSED_PARAMETER(scale);
  return ORT_MAKE_STATUS(ONNXRUNTIME, StatusCode::NOT_IMPLEMENTED, "lean attention does not support float tensor");
}
#endif



template <typename T>
Status CudnnFlashAttention(
    hipdnnHandle_t cudnn_handle,
    Stream* ort_stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data,
    float scale) {
  assert(data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH ||
         data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH_BNSH_BNSH ||
         data.qkv_format == AttentionQkvFormat::Q_K_V_BNSH);
  assert(parameters.mask_type == AttentionMaskType::MASK_NONE ||
         parameters.mask_type == AttentionMaskType::MASK_1D_KEY_SEQ_LEN);
  constexpr bool is_bf16 = false;

  T* attention_bias = const_cast<T*>(data.attention_bias);
  int* mask_sequence_lengths_kv = const_cast<int*>(data.mask_index);

  cudnn_sdpa::run(
      data.output,
      data.q,
      data.k,
      data.v,
      attention_bias,
      nullptr,                                 // (optional) mask_sequence_lengths_q
      mask_sequence_lengths_kv,                // (optional) mask_sequence_lengths_kv
      parameters.batch_size,
      parameters.num_heads,                    // num_heads_q,
      parameters.num_heads,                    // num_heads_kv,
      parameters.head_size,                    // head_size_qk
      parameters.v_head_size,                  // head_size_v
      parameters.sequence_length,              // sequence_length_q
      parameters.total_sequence_length,        // sequence_length_kv
      scale,                                   // scaling factor applied prior softmax
      parameters.is_unidirectional,            // causal
      is_bf16,                                 // True if bfloat16, otherwise float16
      parameters.broadcast_attn_bias_dim_0,    // broadcast attention bias dimension 0 or not
      parameters.broadcast_attn_bias_dim_1,    // broadcast attention bias dimension 1 or not
      0,                                       // sliding window length. 0 means no sliding window.
      data.qkv_format,
      cudnn_handle,
      ort_stream,
      data.allocator);

  return Status::OK();
}

template <>
Status CudnnFlashAttention(
    hipdnnHandle_t cudnn_handle,
    Stream* ort_stream,
    contrib::AttentionParameters& parameters,
    AttentionData<float>& data,
    float scale) {
  ORT_UNUSED_PARAMETER(cudnn_handle);
  ORT_UNUSED_PARAMETER(ort_stream);
  ORT_UNUSED_PARAMETER(parameters);
  ORT_UNUSED_PARAMETER(data);
  ORT_UNUSED_PARAMETER(scale);
  return ORT_MAKE_STATUS(ONNXRUNTIME, StatusCode::NOT_IMPLEMENTED,
                         "cudnn flash attention does not support float tensor");
}

#if USE_MEMORY_EFFICIENT_ATTENTION
template <typename T>
Status EfficientAttention(
    const hipDeviceProp_t& device_prop,
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data,
    float scale) {
  // We only enable fused cross attention when there is no key padding mask.
  // Otherwise, key have effective batch size 2 * batch_size, which is different from batch_size of query.
  assert(data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH ||
         data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH_BNSH_BNSH);
  assert(parameters.mask_type == AttentionMaskType::MASK_NONE ||
         parameters.mask_type == AttentionMaskType::MASK_1D_KEY_SEQ_LEN_START);

  MemoryEfficientAttentionParams p;
  p.sm = device_prop.major * 10 + device_prop.minor;
  p.is_half = sizeof(T) == 2;
  p.batch_size = parameters.batch_size;
  p.num_heads = parameters.num_heads;
  p.sequence_length = parameters.sequence_length;
  p.kv_sequence_length = parameters.total_sequence_length;
  p.max_sequence_length = parameters.total_sequence_length;
  p.qk_head_size = parameters.head_size;
  p.v_head_size = parameters.v_head_size;
  p.causal = parameters.is_unidirectional;
  p.scale = scale;
  p.use_smooth_softmax = false;

  if (nullptr == data.mask_index) {
    p.seqlen_k_ptr = nullptr;
    p.seqstart_q_ptr = nullptr;
    p.seqstart_k_ptr = nullptr;
  } else {
    p.seqlen_k_ptr = const_cast<int32_t*>(reinterpret_cast<const int32_t*>(data.mask_index));
    p.seqstart_q_ptr = p.seqlen_k_ptr + parameters.batch_size;
    p.seqstart_k_ptr = p.seqlen_k_ptr + 2 * parameters.batch_size + 1;
  }

  p.query = data.q;
  p.key = data.k;
  p.value = data.v;

  p.attn_bias = data.attention_bias;
  p.broadcast_attn_bias_dim_0 = parameters.broadcast_attn_bias_dim_0;
  p.broadcast_attn_bias_dim_1 = parameters.broadcast_attn_bias_dim_1;

  p.output = data.output;
  p.is_kv_bsnh = data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH;
  p.workspace = MemoryEfficientAttentionParams::need_workspace(parameters.v_head_size, sizeof(T) == sizeof(float))
                    ? data.scratch
                    : nullptr;
  p.stream = stream;
  p.has_custom_right_padding = false;
  run_memory_efficient_attention(p);

  return Status::OK();
}
#endif

template <typename T, typename QK>
Status LaunchDecoderMaskedMultiHeadAttention(
  const DecoderMaskedMultiHeadAttentionParameters& parameters,
  hipStream_t stream,
  const int head_size) {

  DUMP_STRING_INIT();
  DUMP_STRING("DMMHA parameters...");
  DUMP_STRING("is_mha = ", (parameters.is_mha == true));
  DUMP_STRING("is_cross_attention = ", (parameters.is_cross_attention == true));
  DUMP_STRING("is_packed_qkv = ", (parameters.is_packed_qkv == true));
  DUMP_STRING("kv_data_in_flight = ", (parameters.kv_data_in_flight == true));

  DUMP_STRING("Batch size = ", parameters.batch_size);
  DUMP_STRING("Sequence length = ", parameters.sequence_length);
  DUMP_STRING("Num heads = ", parameters.num_heads);
  DUMP_STRING("Head size = ", parameters.head_size);
  DUMP_STRING("Hidden size = ", parameters.hidden_size);

  DUMP_STRING("Past sequence length = ", parameters.past_sequence_length);
  DUMP_STRING("KV sequence length = ", parameters.kv_sequence_length);
  DUMP_STRING("Total sequence length = ", parameters.total_sequence_length);
  DUMP_STRING("Max sequence length = ", parameters.max_sequence_length);

  DUMP_STRING("parameters.k is null = ", (parameters.k == nullptr));
  DUMP_STRING("parameters.v is null = ", (parameters.v == nullptr));
  DUMP_STRING("parameters.k_cache is null = ", (parameters.k_cache == nullptr));
  DUMP_STRING("parameters.v_cache is null = ", (parameters.v_cache == nullptr));

  DUMP_STRING("parameters.q_bias is null = ", (parameters.q_bias == nullptr));
  DUMP_STRING("parameters.k_bias is null = ", (parameters.k_bias == nullptr));
  DUMP_STRING("parameters.v_bias is null = ", (parameters.v_bias == nullptr));

  DUMP_STRING("parameters.attention_bias is null = ", (parameters.attention_bias == nullptr));
  DUMP_STRING("Scale = ", parameters.scale);
  DUMP_STRING("Mask is null = ", (parameters.mask == nullptr));
  DUMP_STRING("Mask filter value = ", parameters.mask_filter_value);

  DUMP_STRING("Beam width = ", parameters.beam_width);
  DUMP_STRING("parameters.cache_indir is null = ", (parameters.cache_indir == nullptr));
  DUMP_STRING("parameters.out_qk is null = ", (parameters.out_qk == nullptr));

  switch (head_size) {
    case 32:
      mmha_launch_kernel<T, QK, 32>(parameters, stream);
      break;

    case 64:
      mmha_launch_kernel<T, QK, 64>(parameters, stream);
      break;

    case 128:
      mmha_launch_kernel<T, QK, 128>(parameters, stream);
      break;

    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, NOT_IMPLEMENTED,
                             "Unsupported head size in DecoderMaskedMultiHeadAttention. Got head size: ",
                             head_size);
  }

  return Status::OK();
}

template <typename T, typename QK>
Status DecoderMaskedMultiHeadAttention(
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data,
    float scale) {
  assert(data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH ||
         data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH_BNSH_BNSH);
  assert(parameters.mask_type == AttentionMaskType::MASK_NONE ||
         parameters.mask_type == AttentionMaskType::MASK_2D_KEY_PADDING);
  assert(parameters.head_size == parameters.v_head_size);

  DecoderMaskedMultiHeadAttentionParameters p;
  p.is_mha = true;
  p.is_cross_attention = (data.past_key == nullptr && data.present_key == nullptr);
  p.is_packed_qkv = false;
  p.kv_data_in_flight = ParseEnvironmentVariableWithDefault<bool>(attention::kDecoderMaskedAttentionLoadKVDataInFlight, false);

  p.batch_size = parameters.batch_size;
  p.sequence_length = parameters.sequence_length;
  p.num_heads = parameters.num_heads;
  p.head_size = parameters.head_size;
  p.hidden_size = parameters.hidden_size;

  p.past_sequence_length = parameters.past_sequence_length;
  p.kv_sequence_length = parameters.kv_sequence_length;
  p.total_sequence_length = p.is_cross_attention ? parameters.kv_sequence_length : parameters.total_sequence_length;
  p.max_sequence_length = p.is_cross_attention ? parameters.kv_sequence_length : parameters.max_sequence_length;

  p.q = data.q;
  p.k = p.is_cross_attention ? nullptr : data.k;
  p.v = p.is_cross_attention ? nullptr : data.v;
  p.k_cache = p.is_cross_attention ? data.k : data.present_key;
  p.v_cache = p.is_cross_attention ? data.v : data.present_value;

  p.q_bias = data.q_bias;
  p.k_bias = data.k_bias;
  p.v_bias = data.v_bias;

  p.attention_bias = const_cast<T*>(data.attention_bias);
  p.broadcast_attn_bias_dim_0 = parameters.broadcast_attn_bias_dim_0;
  p.broadcast_attn_bias_dim_1 = parameters.broadcast_attn_bias_dim_1;

  p.scale = scale;
  p.mask = data.mask_index;
  p.mask_filter_value = parameters.mask_filter_value;

  p.beam_width = parameters.beam_width;
  p.cache_indir = data.cache_indirection;

  p.out = data.output;
  p.out_qk = data.output_qk;

  // DecoderMaskedMultiHeadAttention(T, QK) is defined for:
  // T = float, QK = float
  // T = float, QK = half
  // T = uint16_t, QK = float
  // T = uint16_t, QK = half
  if (std::is_same<T, float>::value) {
    return LaunchDecoderMaskedMultiHeadAttention<float, QK>(p, stream, parameters.head_size);
  }
  if (std::is_same<T, half>::value) {
    return LaunchDecoderMaskedMultiHeadAttention<uint16_t, QK>(p, stream, parameters.head_size);
  }
  return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "DecoderMaskedMultiHeadAttention is only implemented for float32 and float16.");
}

template <typename T, typename QK>
Status UnfusedAttention(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    Stream* ort_stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data,
    float scale) {
  assert(data.qkv_format == AttentionQkvFormat::Q_K_V_BNSH);

  auto stream = static_cast<hipStream_t>(ort_stream->GetHandle());

  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const int total_sequence_length = parameters.total_sequence_length;
  const int num_heads = parameters.num_heads;
  const int qk_head_size = parameters.head_size;
  const int v_head_size = parameters.v_head_size;
  const int batches = batch_size * num_heads;

  const int* mask_index = data.mask_index;
  gsl::span<const int64_t>& mask_index_dims = data.mask_index_dims;

  // Raw attention mask could be 2D (BxT) or 3D (BxSxT) or 4D(Bx1xMxM), where M is the max sequence length.
  bool use_raw_attention_mask = (nullptr != mask_index && mask_index_dims.size() >= 2);

  // Compute Q*K' (as K'*Q), scaled by 1/sqrt(H) and store in scratch: BxNxSxT
  // Q: BxNxSxH, K (present_k): BxNxTxH, Q*K': BxNxSxT
  float one = 1.0f;
  float zero = 0.f;

  float alpha = use_raw_attention_mask ? one : scale;

  hipblasSetStream(cublas, stream);

  const int present_sequence_length = parameters.past_present_share_buffer
                                          ? parameters.max_sequence_length
                                          : total_sequence_length;
  const int present_size_per_batch_k = present_sequence_length * qk_head_size;
  const int present_size_per_batch_v = present_sequence_length * v_head_size;

  DUMP_TENSOR_INIT();
  DUMP_TENSOR_D("q", data.q, batch_size, num_heads, sequence_length, qk_head_size);
  DUMP_TENSOR_D("k", data.k, batch_size, num_heads, total_sequence_length, qk_head_size);
  DUMP_TENSOR_D("v", data.v, batch_size, num_heads, total_sequence_length, v_head_size);
  DUMP_TENSOR_D("mask_index", mask_index, mask_index_dims);

  CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
      cublas, HIPBLAS_OP_T, HIPBLAS_OP_N,
      total_sequence_length, sequence_length, qk_head_size,
      &alpha, data.k, qk_head_size, present_size_per_batch_k,
      data.q, qk_head_size, sequence_length * qk_head_size,
      &zero, data.scratch, total_sequence_length, sequence_length * total_sequence_length, batches,
      device_prop, parameters.use_tf32));

  DUMP_TENSOR_D("QK", data.scratch, batch_size, num_heads, sequence_length, total_sequence_length);

  constexpr size_t element_size = sizeof(T);
  const size_t bytes = GetAttentionScratchSize(element_size, batch_size, num_heads,
                                               sequence_length, total_sequence_length);
  T* scratch2 = data.scratch + (bytes / element_size);

  const bool broadcast_attn_bias_dim_0 = parameters.broadcast_attn_bias_dim_0;
  const bool broadcast_attn_bias_dim_1 = parameters.broadcast_attn_bias_dim_1;

  // Apply softmax and store result R to scratch2: BxNxSxT
  if (use_raw_attention_mask) {  // 2d, 3d or 4d attention mask
    const int mask_dimension = static_cast<int>(mask_index_dims.size());

    // For testing, environment variable ORT_TRANSFORMER_OPTIONS=1 could enable persistent softmax used in Torch.
    const TransformerOptions* options = TransformerOptions::GetInstance();
    bool use_persistent_softmax = options->IsPrecisionMode() && !options->DisablePersistentSoftmax();

    // replace Q*K' in place with masked score for persistent softmax.
    T* persistent_softmax_workspace = data.scratch;
    ORT_RETURN_IF_ERROR(
        ComputeSoftmaxWithRawMask<T>(
            ort_stream, total_sequence_length, sequence_length, batch_size, num_heads,
            mask_index, nullptr, data.attention_bias, broadcast_attn_bias_dim_0, broadcast_attn_bias_dim_1,
            data.scratch, scratch2, parameters.is_unidirectional, scale, mask_dimension,
            parameters.max_sequence_length, use_persistent_softmax, persistent_softmax_workspace,
            parameters.mask_filter_value));
  } else if (nullptr != mask_index) {  // 1d mask index
    assert(mask_index_dims.size() == 1);
    // mask_index has 1D shape: either (batch_size) or (2*batch_size). Only the later one has start postions.
    const int* mask_start = (mask_index_dims[0] > batch_size) ? mask_index + batch_size : nullptr;
    ORT_RETURN_IF_ERROR(ComputeSoftmaxWithMask1D<T>(
        stream, total_sequence_length, sequence_length, batch_size, num_heads,
        mask_index, mask_start, data.attention_bias, broadcast_attn_bias_dim_0, broadcast_attn_bias_dim_1,
        data.scratch, scratch2, parameters.is_unidirectional));
  } else {  // no mask
    if (nullptr != data.output_qk) {
      int64_t qk_size = (int64_t)batch_size * num_heads * sequence_length * total_sequence_length;
      if (std::is_same<T, QK>::value) {
        hipMemcpyAsync(data.output_qk, data.scratch, qk_size * sizeof(QK), hipMemcpyDeviceToDevice, stream);
      } else {
        ORT_RETURN_IF_ERROR(
          (CopyQK<T, QK>(stream, static_cast<int>(qk_size), data.scratch, reinterpret_cast<QK*>(data.output_qk))));
      }
    }
    ORT_RETURN_IF_ERROR(
        ComputeSoftmax<T>(
            stream, total_sequence_length, sequence_length, batch_size, num_heads,
            data.attention_bias, broadcast_attn_bias_dim_0, broadcast_attn_bias_dim_1,
            data.scratch, scratch2, parameters.is_unidirectional));
  }

  DUMP_TENSOR_D("Softmax", scratch2, batch_size, num_heads, sequence_length, total_sequence_length);

  // compute R*V (as V*R), and store in temp_output (space used by Q): BxNxSxH_v
  T* temp_output = data.q;
  CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
      cublas, HIPBLAS_OP_N, HIPBLAS_OP_N,
      v_head_size, sequence_length, total_sequence_length,
      &one, data.v, v_head_size, present_size_per_batch_v,
      scratch2, total_sequence_length, sequence_length * total_sequence_length,
      &zero, temp_output, v_head_size, sequence_length * v_head_size, batches, device_prop, parameters.use_tf32));

  // Temp_output is BxNxSxH_v, transpose to output BxSxNxH_v
  Status result = LaunchTransCtx(stream, sequence_length, batch_size, v_head_size, num_heads,
                                 device_prop.maxThreadsPerBlock, false, temp_output, data.output);
  DUMP_TENSOR_D("Attention Output", data.output, batch_size, sequence_length, num_heads, v_head_size);
  return result;
}

#ifndef USE_ROCM  // exclude the following from hipify since they are not used in ROCM EP

template <typename T>
Status ConcatPastToPresent(int batch_size, int num_heads, int qk_head_size, int v_head_size,
                           int sequence_length, int total_sequence_length,
                           hipStream_t stream, int max_threads_per_block,
                           AttentionData<T>& data) {
  // Concat past key value to present (2xBxNxLxH), where L is kv_sequence_length and T is total_sequence_length.
  // past_k (BxNxPxH) + k (BxNxLxH) => present_k (BxNxTxH)
  // past_v (BxNxPxH) + v (BxNxLxH) => present_v (BxNxTxH)
  // When there is past state, the head size for Q/K/V shall be same: H == H_v.

  if (nullptr != data.present) { // Attention op
    assert(data.qkv_format == AttentionQkvFormat::Q_K_V_BNSH ||
           data.qkv_format == AttentionQkvFormat::Q_K_V_BNSH_QKV_BS3NH);

    ORT_RETURN_IF_ERROR(
        LaunchConcatTensorToTensor(
            stream, total_sequence_length, sequence_length, batch_size, qk_head_size, num_heads,
            max_threads_per_block, 2, data.past, data.k, data.present));



    // Update pointers to present_k and present_v.
    data.k = data.present;
    data.v = data.present + batch_size * num_heads * total_sequence_length * qk_head_size;
  } else  { // MultiHeadAttention op
    if (nullptr != data.present_key) {
      ORT_ENFORCE(data.qkv_format == AttentionQkvFormat::Q_K_V_BNSH ||
                  data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH_BNSH_BNSH);
      if (nullptr != data.past_key) {
        assert(data.past_key != data.k);
        assert(data.past_value != data.v);

        ORT_RETURN_IF_ERROR(
            LaunchConcatTensorToTensor(stream, total_sequence_length, sequence_length,
                                      batch_size, qk_head_size, num_heads,
                                      max_threads_per_block, 1, data.past_key, data.k, data.present_key));
        ORT_RETURN_IF_ERROR(
            LaunchConcatTensorToTensor(stream, total_sequence_length, sequence_length,
                                      batch_size, v_head_size, num_heads,
                                      max_threads_per_block, 1, data.past_value, data.v, data.present_value));
        // Update pointers to present_k and present_v.
        data.k = data.present_key;
        data.v = data.present_value;
      } else { // nullptr == data.past_key && nullptr != data.present_key
        if (data.k != data.present_key) {
          int64_t k_size = (int64_t)batch_size * num_heads * total_sequence_length * qk_head_size;
          hipMemcpyAsync(data.present_key, data.k, k_size * sizeof(T), hipMemcpyDeviceToDevice, stream);
        }

        if (data.v != data.present_value) {
          int64_t v_size = (int64_t)batch_size * num_heads * total_sequence_length * v_head_size;
          hipMemcpyAsync(data.present_value, data.v, v_size * sizeof(T), hipMemcpyDeviceToDevice, stream);
        }
      }
    }
  }

  return CUDA_CALL(hipGetLastError());
}

// Template Instantiation
template Status ConcatPastToPresent<float>(int batch_size, int num_heads, int qk_head_size, int v_head_size,
                                           int sequence_length, int total_sequence_length,
                                           hipStream_t stream,
                                           int max_threads_per_block,
                                           AttentionData<float>& data);

template Status ConcatPastToPresent<half>(int batch_size, int num_heads, int qk_head_size, int v_head_size,
                                          int sequence_length, int total_sequence_length,
                                          hipStream_t stream,
                                          int max_threads_per_block,
                                          AttentionData<half>& data);
#endif

template <typename T>
Status PastPresentBufferShare(int batch_size, int num_heads, int qk_head_size, int v_head_size,
                              int sequence_length, void* fused_runner,
                              contrib::AttentionParameters& parameters,
                              AttentionData<T>& data,
                              hipStream_t stream,
                              int max_threads_per_block) {
  ORT_ENFORCE(qk_head_size == v_head_size);
  assert(data.fused_cross_attention_kernel == nullptr);
  assert(nullptr == fused_runner || parameters.is_unidirectional);
  assert(!data.use_memory_efficient_attention);
  assert(!data.use_flash_attention);
  assert(data.has_qkv_workspace);

  bool combined_key_value = nullptr != data.present;
  bool separate_key_value = nullptr != data.past_key && nullptr != data.present_key &&
                            nullptr != data.past_value && nullptr != data.present_value;

  // Return early if buffer sharing is not possible
  if (!combined_key_value && !separate_key_value) {
    return Status::OK();
  }

  if (combined_key_value) { // Attention op
    assert(data.gemm_buffer != nullptr);

    if (data.present != data.past) {
      // For easy testing. Production should better avoid this path.
      int64_t kv_size = 2LL * (int64_t)batch_size * num_heads * parameters.max_sequence_length * qk_head_size;
      hipMemcpyAsync(data.present, data.past, kv_size * sizeof(T), hipMemcpyDeviceToDevice, stream);
    }

    // For fused causal, bias has been added to gemm_buffer.
    const T* bias = (nullptr != fused_runner && parameters.is_unidirectional) ? nullptr : data.bias;

    // append last k v to present
    ORT_RETURN_IF_ERROR(LaunchAddBiasTransAppendKvToPresent(
        stream, parameters.max_sequence_length, parameters.past_sequence_length, sequence_length,
        batch_size, qk_head_size, num_heads, max_threads_per_block,
        bias, data.gemm_buffer, data.present));

    data.k = data.present;
    data.v = data.present + batch_size * num_heads * parameters.max_sequence_length * qk_head_size;
  } else if (data.use_decoder_masked_multihead_attention) {  // DecoderMaskedMultiHeadAttention op
    assert(data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH ||
           data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH_BNSH_BNSH);

    // DecoderMaskedMultiHeadAttention kernel manages the KV caches
    // so this case is empty
  } else {  // MultiHeadAttention op
    assert(data.qkv_format == AttentionQkvFormat::Q_K_V_BNSH ||
           data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH_BNSH_BNSH);
    assert(data.seqlens_k_total);

    // Using BNSH since AddBiasTranspose has already been applied
    constexpr bool is_past_kv_bnsh_format = true;
    constexpr bool is_new_kv_bnsh_format = true;
    ORT_RETURN_IF_ERROR(LaunchConcatKVInPlace(
      batch_size, num_heads, qk_head_size, parameters.max_sequence_length,
      data.seqlens_k_total, nullptr, parameters.sequence_length, data.k, data.v, data.present_key, data.present_value,
      is_past_kv_bnsh_format, is_new_kv_bnsh_format, stream, max_threads_per_block));

    data.k = data.present_key;
    data.v = data.present_value;
  }

  return CUDA_CALL(hipGetLastError());
}

template Status PastPresentBufferShare<float>(int batch_size, int num_heads, int qk_head_size, int v_head_size,
                                              int sequence_length, void* fused_runner,
                                              contrib::AttentionParameters& parameters,
                                              AttentionData<float>& data,
                                              hipStream_t stream,
                                              int max_threads_per_block);

template Status PastPresentBufferShare<half>(int batch_size, int num_heads, int qk_head_size, int v_head_size,
                                             int sequence_length, void* fused_runner,
                                             contrib::AttentionParameters& parameters,
                                             AttentionData<half>& data,
                                             hipStream_t stream,
                                             int max_threads_per_block);

template <typename T, typename QK>
Status QkvToContext(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    hipdnnHandle_t& cudnn,
    Stream* ort_stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data) {
  auto stream = static_cast<hipStream_t>(ort_stream->GetHandle());
  const int max_threads_per_block = device_prop.maxThreadsPerBlock;
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const int total_sequence_length = parameters.total_sequence_length;
  const int num_heads = parameters.num_heads;
  const int qk_head_size = parameters.head_size;
  const int v_head_size = parameters.v_head_size;
  void* fused_runner = data.fused_runner;

  // At most one fused kernel is enabled.
  assert((static_cast<int>(data.use_flash_attention) +
          static_cast<int>(data.use_memory_efficient_attention) +
          static_cast<int>(data.use_decoder_masked_multihead_attention) +
          static_cast<int>(fused_runner != nullptr) +
          static_cast<int>(data.fused_cross_attention_kernel != nullptr) +
          static_cast<int>(data.kernel_type == AttentionKernelType::AttentionKernel_CudnnFlashAttention)) <= 1);

  DUMP_STRING_INIT();
  DUMP_STRING("Preparing Q, K, V");
  ORT_RETURN_IF_ERROR(PrepareQkv<T>(parameters, data, stream, max_threads_per_block));

  if (!parameters.past_present_share_buffer) {
    ORT_RETURN_IF_ERROR(ConcatPastToPresent<T>(batch_size, num_heads, qk_head_size, v_head_size,
                                            sequence_length, total_sequence_length,
                                            stream, max_threads_per_block, data));

  } else {  // past_present_share_buffer
    ORT_RETURN_IF_ERROR(PastPresentBufferShare<T>(batch_size, num_heads, qk_head_size, v_head_size,
                                               sequence_length, fused_runner,
                                               parameters, data, stream, max_threads_per_block));
  }

  // Q, K and V are ready now
  if (data.fused_cross_attention_kernel != nullptr) {
    DUMP_STRING("FusedTrtCrossAttention");
    return FusedTrtCrossAttention<T>(stream, parameters, data);
  }

  // Run TRT fused attention.
  if (nullptr != fused_runner) {
    DUMP_STRING("FusedTrtSelfAttention");
    return FusedTrtSelfAttention<T>(stream, parameters, data);
  }

  // For raw attention mask, the scalar 1/sqrt(H) is moved to combine with softmax computation.
  const float scale = parameters.scale == 0.0f ? 1.f / sqrt(static_cast<float>(qk_head_size))
                                               : parameters.scale;
#if USE_LEAN_ATTENTION
  if (data.use_lean_attention) {
    DUMP_STRING("LeanAttention");
    return LeanAttention<T>(device_prop, stream, parameters, data, scale);
  }
#endif

#if USE_FLASH_ATTENTION
  if (data.use_flash_attention) {
    DUMP_STRING("FlashAttention");
    return FlashAttention<T>(device_prop, stream, parameters, data, scale);
  }
#endif

  if (data.kernel_type == AttentionKernelType::AttentionKernel_CudnnFlashAttention) {
    DUMP_STRING("CudnnFlashAttention");
    return CudnnFlashAttention<T>(cudnn, ort_stream, parameters, data, scale);
  }

#if USE_MEMORY_EFFICIENT_ATTENTION
  if (data.use_memory_efficient_attention) {
    DUMP_STRING("EfficientAttention");
    return EfficientAttention<T>(device_prop, stream, parameters, data, scale);
  }
#endif

  if (data.use_decoder_masked_multihead_attention) {
    DUMP_STRING("DecoderMaskedMHA");
    return DecoderMaskedMultiHeadAttention<T, QK>(stream, parameters, data, scale);
  }

  DUMP_STRING("UnfusedAttention");
  return UnfusedAttention<T, QK>(device_prop, cublas, ort_stream, parameters, data, scale);
}

// Template Instantiation
template struct AttentionData<float>;

template struct AttentionData<half>;

template Status QkvToContext<float>(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    hipdnnHandle_t& cudnn,
    Stream* ort_stream,
    contrib::AttentionParameters& parameters,
    AttentionData<float>& data);

template Status QkvToContext<half>(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    hipdnnHandle_t& cudnn,
    Stream* ort_stream,
    contrib::AttentionParameters& parameters,
    AttentionData<half>& data);

template Status QkvToContext<float, half>(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    hipdnnHandle_t& cudnn,
    Stream* ort_stream,
    contrib::AttentionParameters& parameters,
    AttentionData<float>& data);

template Status QkvToContext<half, float>(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    hipdnnHandle_t& cudnn,
    Stream* ort_stream,
    contrib::AttentionParameters& parameters,
    AttentionData<half>& data);

template Status LaunchDecoderMaskedMultiHeadAttention<float, float>(
  const DecoderMaskedMultiHeadAttentionParameters& parameters,
  hipStream_t stream,
  const int head_size);

template Status LaunchDecoderMaskedMultiHeadAttention<float, half>(
  const DecoderMaskedMultiHeadAttentionParameters& parameters,
  hipStream_t stream,
  const int head_size);

template Status LaunchDecoderMaskedMultiHeadAttention<uint16_t, float>(
  const DecoderMaskedMultiHeadAttentionParameters& parameters,
  hipStream_t stream,
  const int head_size);

template Status LaunchDecoderMaskedMultiHeadAttention<uint16_t, half>(
  const DecoderMaskedMultiHeadAttentionParameters& parameters,
  hipStream_t stream,
  const int head_size);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
