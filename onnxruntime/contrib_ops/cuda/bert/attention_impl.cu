#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on qkvToContext plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/

Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications:
// (1) support GPT-2 past state, unidirectional mask and 4D attention mask from Megatron
// (2) support 2D attention mask
// (3) allow persistent softmax from PyTorch for debugging purpose.
// (4) support different input hidden size and model hidden size for pruned model
// (5) support different hidden sizes of Q/K and V
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/shared_inc/fpgeneric.h"
#include "contrib_ops/cuda/bert/attention_softmax.h"
#include "contrib_ops/cuda/bert/transformer_common.h"
#include "contrib_ops/cuda/bert/tensorrt_fused_multihead_attention/mha_runner.h"
#include "contrib_ops/cuda/bert/tensorrt_fused_multihead_attention/cross_attention/fmha_cross_attention.h"
#include "contrib_ops/cpu/bert/attention_base.h"
#include "contrib_ops/cuda/bert/bert_padding.h"
#include "contrib_ops/cuda/transformers/dump_cuda_tensor.h"
#include "contrib_ops/cuda/bert/cutlass_fmha/memory_efficient_attention.h"
#include "contrib_ops/cuda/bert/flash_attention/flash_api.h"
#include "contrib_ops/cuda/bert/attention_impl.h"

using namespace onnxruntime::cuda;
using namespace onnxruntime::contrib::attention_softmax_cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

constexpr size_t kMemoryAlignment = 256;

static size_t AlignTo(size_t a, size_t b) {
  return CeilDiv(a, b) * b;
}

size_t AlignSize(size_t bytes) {
  const size_t bytesAligned = AlignTo(bytes, kMemoryAlignment);
  return bytesAligned;
}

void CumulatedSequenceLengthCache::Initialize(int32_t sequence_length, hipStream_t stream) {
  if (this->sequence_length != sequence_length) {
    ORT_ENFORCE(buffer.get() != nullptr && this->max_batch_size > 0);
    LaunchTrtSequenceOffset(reinterpret_cast<int32_t*>(buffer.get()), nullptr,
                            this->max_batch_size, sequence_length, stream);
    this->sequence_length = sequence_length;
  }
}

int* GetCumulatedSequenceLength(CumulatedSequenceLengthCache* cache,
                                const int* mask_index,
                                int batch_size,
                                int sequence_length,
                                hipStream_t stream,
                                void* scratch_buffer) {
  if (mask_index == nullptr && cache != nullptr) {
    if (batch_size <= cache->max_batch_size) {
      cache->Initialize(sequence_length, stream);
      return reinterpret_cast<int*>(cache->buffer.get());
    }
  }

  int* sequence_offset = reinterpret_cast<int*>(scratch_buffer);
  LaunchTrtSequenceOffset(sequence_offset, mask_index, batch_size, sequence_length, stream);
  return sequence_offset;
}

size_t GetAttentionScratchSize(
    size_t element_size,
    size_t batch_size,
    size_t num_heads,
    size_t sequence_length,
    size_t total_sequence_length) {
  const size_t bytes = element_size * batch_size * num_heads * sequence_length * total_sequence_length;
  return AlignSize(bytes);
}

size_t GetSequenceOffsetSize(int batch_size, bool has_padding) {
  // There are batch_size + 1 offsets Without padding (or padding removed), and 2 * batch_size + 1 with padding.
  size_t bytes = sizeof(int) * ((has_padding ? 2 * batch_size : batch_size) + 1);
  return AlignSize(bytes);
  ;
}

size_t GetAttentionWorkspaceSize(
    size_t element_size,
    size_t batch_size,
    size_t num_heads,
    size_t qk_head_size,
    size_t v_head_size,
    size_t sequence_length,
    size_t kv_sequence_length,
    size_t total_sequence_length,
    void* fused_runner,
    bool use_flash_attention,
    bool use_fused_cross_attention,
    bool use_memory_efficient_attention) {
  // Note that q, k and v might need alignment for fused attention kernels.
  const size_t qkv_bytes = element_size * batch_size * num_heads *
                           ((sequence_length + kv_sequence_length) * qk_head_size + kv_sequence_length * v_head_size);

#if USE_FLASH_ATTENTION
  if (use_flash_attention) {
    return qkv_bytes + onnxruntime::flash::get_softmax_lse_size(sequence_length, batch_size, num_heads);
  }
#else
  ORT_UNUSED_PARAMETER(use_flash_attention);
#endif

#if USE_MEMORY_EFFICIENT_ATTENTION
  if (use_memory_efficient_attention) {
    size_t fmha_buffer_bytes = 0;
    if (MemoryEfficientAttentionParams::need_workspace(v_head_size, element_size == sizeof(float))) {
      fmha_buffer_bytes = batch_size * sequence_length * num_heads * v_head_size * sizeof(float);
    }

    return qkv_bytes + fmha_buffer_bytes;
  }
#else
  ORT_UNUSED_PARAMETER(use_memory_efficient_attention);
#endif

  if (fused_runner != nullptr) {
    return qkv_bytes + GetSequenceOffsetSize(static_cast<int>(batch_size), true);
  }

  if (use_fused_cross_attention) {
    return qkv_bytes + 2 * GetSequenceOffsetSize(static_cast<int>(batch_size), true);
  }

  return qkv_bytes + 2 * GetAttentionScratchSize(element_size, batch_size, num_heads, sequence_length,
                                                 total_sequence_length);
}

template <typename T>
Status FusedTrtCrossAttention(
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data) {
  assert(data.qkv_format == AttentionQkvFormat::Q_KV_BSNH_BSN2H);

  // We only enable fused cross attention when there is no key padding mask.
  // Otherwise, key have effective batch size 2 * batch_size, which is different from batch_size of query.
  assert(data.mask_index == nullptr);

  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  int* q_sequence_offset = GetCumulatedSequenceLength(data.cumulated_sequence_length_q_cache,
                                                      data.mask_index, batch_size,
                                                      sequence_length, stream,
                                                      data.scratch);

  DUMP_TENSOR_INIT();
  DUMP_TENSOR_D("q_sequence_offset", q_sequence_offset, 1, batch_size + 1);

  int* kv_sequence_offset = q_sequence_offset + (GetSequenceOffsetSize(batch_size, false) / sizeof(int));
  kv_sequence_offset = GetCumulatedSequenceLength(data.cumulated_sequence_length_kv_cache,
                                                  data.mask_index, batch_size, parameters.kv_sequence_length, stream,
                                                  kv_sequence_offset);
  CUDA_RETURN_IF_ERROR(hipGetLastError());

  DUMP_TENSOR_D("kv_sequence_offset", kv_sequence_offset, 1, batch_size + 1);

  FusedMultiHeadCrossAttentionKernel const* cross_attention_kernel =
      reinterpret_cast<FusedMultiHeadCrossAttentionKernel const*>(data.fused_cross_attention_kernel);

  // When there is no bias, we can directly use q and packed kv from inputs.
  void const* query = data.q;
  void const* packed_kv = data.k;
  if (data.value == nullptr && data.bias == nullptr) {
    query = data.query;
    packed_kv = data.key;
  }

  run_fused_cross_attention(
      query,                          // Q
      packed_kv,                      // packed KV
      q_sequence_offset,              // cumulated sequence length of Q
      kv_sequence_offset,             // cumulated sequence length of KV
      data.output,                    // output
      cross_attention_kernel,         // kernels
      batch_size,                     // batch size
      parameters.num_heads,           // number of heads
      parameters.head_size,           // head size of Q/K/V
      sequence_length,                // sequence length of Q
      parameters.kv_sequence_length,  // sequence length of KV
      stream);

  DUMP_TENSOR("trt cross output", data.output,
              batch_size, sequence_length, parameters.num_heads, parameters.v_head_size);
  return Status::OK();
}

template <>
Status FusedTrtCrossAttention<float>(
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<float>& data) {
  return ORT_MAKE_STATUS(ONNXRUNTIME, StatusCode::NOT_IMPLEMENTED,
                         "Trt fused cross attention does not support float tensor");
}

template <typename T>
Status FusedTrtSelfAttention(
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data) {
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const bool causal = parameters.is_unidirectional;

  int* sequence_offset = reinterpret_cast<int*>(data.scratch);

  DUMP_TENSOR_INIT();
  if (parameters.mask_type == AttentionMaskType::MASK_2D_KEY_PADDING) {
    DUMP_TENSOR_D("mask", reinterpret_cast<const int*>(data.mask_index), batch_size, sequence_length);
    LaunchTrtSequenceOffset2d(sequence_offset, data.mask_index, batch_size, sequence_length, stream);
  } else {
    sequence_offset = GetCumulatedSequenceLength(data.cumulated_sequence_length_q_cache,
                                                 data.mask_index, batch_size, sequence_length, stream,
                                                 sequence_offset);
  }
  DUMP_TENSOR_D("sequence_offset", sequence_offset, 1, (data.mask_index != nullptr ? 2 : 1) * batch_size + 1);
  CUDA_RETURN_IF_ERROR(hipGetLastError());

  FusedMHARunnerFP16v2* fused_fp16_runner = reinterpret_cast<FusedMHARunnerFP16v2*>(data.fused_runner);

  const int S = causal ? sequence_length : fused_fp16_runner->getSFromMaxSeqLen(sequence_length);

  // B = 2 * batch_size when there is padding in input, and B = batch_size when padding is removed.
  const int B = (nullptr == data.mask_index ? batch_size : 2 * batch_size);

  fused_fp16_runner->setup(S, B);

  if (!causal) {
    assert(data.qkv_format == AttentionQkvFormat::QKV_BSN3H);

    // When there is no bias, we can directly use packed qkv from inputs.
    void const* packed_qkv = data.q;
    if (data.query != nullptr && data.key == nullptr && data.bias == nullptr) {
      packed_qkv = data.query;
    }

    fused_fp16_runner->run(packed_qkv, sequence_offset, data.output, stream);
    DUMP_TENSOR("fused output", data.output,
                batch_size, sequence_length, parameters.num_heads, parameters.v_head_size);
  } else {
    assert(data.qkv_format == AttentionQkvFormat::Q_K_V_BNSH_QKV_BS3NH);
    fused_fp16_runner->run(data.gemm_buffer, sequence_offset, data.output, stream);
    DUMP_TENSOR("fused causal output", data.output,
                batch_size, sequence_length, parameters.num_heads, parameters.v_head_size);
  }
  return Status::OK();
}

// Template Specialization for float type
template <>
Status FusedTrtSelfAttention<float>(
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<float>& data) {
  return ORT_MAKE_STATUS(ONNXRUNTIME, StatusCode::NOT_IMPLEMENTED,
                         "Trt fused attention does not support float tensor");
}

#if USE_FLASH_ATTENTION
template <typename T>
Status FlashAttention(
    const hipDeviceProp_t& device_prop,
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data,
    float scale) {
  assert(data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH);
  assert(nullptr == data.mask_index);
  assert(nullptr == data.relative_position_bias);
  assert(parameters.head_size == parameters.v_head_size);

  void* query = reinterpret_cast<void*>(data.q);
  void* key = reinterpret_cast<void*>(data.k);
  void* value = reinterpret_cast<void*>(data.v);
  // For packed KV, we can use query input directly.
  if (data.gemm_buffer == nullptr && data.key != nullptr && data.value == nullptr && data.bias == nullptr) {
    query = reinterpret_cast<void*>(const_cast<T*>(data.query));
  }

  DUMP_TENSOR_INIT();
  DUMP_TENSOR_D("q(BSNH)", reinterpret_cast<const T*>(query),
                parameters.batch_size, parameters.sequence_length, parameters.num_heads, parameters.head_size);
  DUMP_TENSOR_D("k(BSNH)", data.k,
                parameters.batch_size, parameters.total_sequence_length, parameters.num_heads, parameters.head_size);
  DUMP_TENSOR_D("v(BSNH)", data.v,
                parameters.batch_size, parameters.total_sequence_length,
                parameters.num_heads, parameters.v_head_size);

  ORT_RETURN_IF_ERROR(onnxruntime::flash::mha_fwd(
      device_prop, stream, query, key, value, data.output, reinterpret_cast<void*>(data.scratch),
      parameters.batch_size, parameters.num_heads, parameters.num_heads, parameters.head_size,
      parameters.sequence_length, parameters.total_sequence_length, scale, parameters.is_unidirectional));

  DUMP_TENSOR("flash attention output", data.output,
              parameters.batch_size, parameters.sequence_length, parameters.num_heads, parameters.v_head_size);

  return Status::OK();
}

template <>
Status FlashAttention(
    const hipDeviceProp_t& device_prop,
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<float>& data,
    float scale) {
  return ORT_MAKE_STATUS(ONNXRUNTIME, StatusCode::NOT_IMPLEMENTED, "flash attention does not support float tensor");
}
#endif

#if USE_MEMORY_EFFICIENT_ATTENTION
template <typename T>
Status EfficientAttention(
    const hipDeviceProp_t& device_prop,
    hipStream_t stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data,
    float scale) {
  // We only enable fused cross attention when there is no key padding mask.
  // Otherwise, key have effective batch size 2 * batch_size, which is different from batch_size of query.
  assert(data.qkv_format == AttentionQkvFormat::Q_K_V_BSNH);

  const void* query = data.q;
  const void* key = data.k;
  const void* value = data.v;
  // For packed KV, we can use query input directly.
  if (data.gemm_buffer == nullptr && data.key != nullptr && data.value == nullptr) {
    assert(data.bias == nullptr);
    query = data.query;
  }

  DUMP_TENSOR_INIT();
  DUMP_TENSOR_D("q(BSNH)", reinterpret_cast<const T*>(query),
                parameters.batch_size, parameters.sequence_length, parameters.num_heads, parameters.head_size);
  DUMP_TENSOR_D("k(BSNH)", data.k,
                parameters.batch_size, parameters.total_sequence_length, parameters.num_heads, parameters.head_size);
  DUMP_TENSOR_D("v(BSNH)", data.v,
                parameters.batch_size, parameters.total_sequence_length,
                parameters.num_heads, parameters.v_head_size);

  MemoryEfficientAttentionParams p;
  p.sm = device_prop.major * 10 + device_prop.minor;
  p.is_half = sizeof(T) == 2;
  p.batch_size = parameters.batch_size;
  p.num_heads = parameters.num_heads;
  p.sequence_length = parameters.sequence_length;
  p.kv_sequence_length = parameters.total_sequence_length;
  p.qk_head_size = parameters.head_size;
  p.v_head_size = parameters.v_head_size;
  p.causal = parameters.is_unidirectional;
  p.scale = scale;
  p.seqlen_k_ptr = nullptr == data.mask_index
                       ? nullptr
                       : const_cast<int32_t*>(reinterpret_cast<const int32_t*>(data.mask_index));
  p.seqstart_q_ptr = nullptr == data.mask_index
                         ? nullptr
                         : const_cast<int32_t*>(reinterpret_cast<const int32_t*>(
                               data.mask_index + parameters.batch_size));
  p.seqstart_k_ptr = nullptr == data.mask_index
                         ? nullptr
                         : const_cast<int32_t*>(reinterpret_cast<const int32_t*>(
                               data.mask_index + 2 * parameters.batch_size + 1));
  p.query = query;
  p.key = key;
  p.value = value;
  p.attn_bias = nullptr == data.relative_position_bias ? nullptr : data.relative_position_bias;
  p.is_attn_bias_batched = !parameters.broadcast_res_pos_bias;
  p.output = data.output;
  p.workspace = MemoryEfficientAttentionParams::need_workspace(parameters.v_head_size, sizeof(T) == sizeof(float))
                    ? data.scratch
                    : nullptr;
  p.stream = stream;
  run_memory_efficient_attention(p);
  DUMP_TENSOR("efficient attention output", data.output,
              parameters.batch_size, parameters.sequence_length, parameters.num_heads, parameters.v_head_size);

  return Status::OK();
}
#endif

template <typename T>
Status UnfusedAttention(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    Stream* ort_stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data,
    float scale) {
  assert(data.qkv_format == AttentionQkvFormat::Q_K_V_BNSH);

  auto stream = static_cast<hipStream_t>(ort_stream->GetHandle());

  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const int total_sequence_length = parameters.total_sequence_length;
  const int num_heads = parameters.num_heads;
  const int qk_head_size = parameters.head_size;
  const int v_head_size = parameters.v_head_size;
  const int batches = batch_size * num_heads;

  const int* mask_index = data.mask_index;
  gsl::span<const int64_t>& mask_index_dims = data.mask_index_dims;

  // Raw attention mask could be 2D (BxT) or 3D (BxSxT) or 4D(Bx1xMxM), where M is the max sequence length.
  bool use_raw_attention_mask = (nullptr != mask_index && mask_index_dims.size() >= 2);

  // Compute Q*K' (as K'*Q), scaled by 1/sqrt(H) and store in scratch: BxNxSxT
  // Q: BxNxSxH, K (present_k): BxNxTxH, Q*K': BxNxSxT
  float one = 1.0f;
  float zero = 0.f;

  float alpha = use_raw_attention_mask ? one : scale;

  hipblasSetStream(cublas, stream);

  DUMP_TENSOR_INIT();
  DUMP_TENSOR_D("q[BNSH]", data.q, batch_size, num_heads, sequence_length, qk_head_size);
  DUMP_TENSOR_D("k[BNSH]", data.k, batch_size, num_heads, total_sequence_length, qk_head_size);

  const int present_sequence_length = parameters.past_present_share_buffer
                                          ? parameters.max_sequence_length
                                          : total_sequence_length;
  const int present_size_per_batch_k = present_sequence_length * qk_head_size;
  const int present_size_per_batch_v = present_sequence_length * v_head_size;

  CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
      cublas, HIPBLAS_OP_T, HIPBLAS_OP_N,
      total_sequence_length, sequence_length, qk_head_size,
      &alpha, data.k, qk_head_size, present_size_per_batch_k,
      data.q, qk_head_size, sequence_length * qk_head_size,
      &zero, data.scratch, total_sequence_length, sequence_length * total_sequence_length, batches, device_prop));

  DUMP_TENSOR_D("Q", data.q, batch_size, num_heads, sequence_length, qk_head_size);
  DUMP_TENSOR_D("K", data.k, batch_size, num_heads, qk_head_size, sequence_length);
  DUMP_TENSOR_D("QK", data.scratch, batch_size, num_heads, sequence_length, total_sequence_length);

  constexpr size_t element_size = sizeof(T);
  const size_t bytes = GetAttentionScratchSize(element_size, batch_size, num_heads,
                                               sequence_length, total_sequence_length);
  T* scratch2 = data.scratch + (bytes / element_size);

  // Apply softmax and store result R to scratch2: BxNxSxT
  if (use_raw_attention_mask) {  // 2d, 3d or 4d attention mask
    const int mask_dimension = static_cast<int>(mask_index_dims.size());

    // For testing, environment variable ORT_TRANSFORMER_OPTIONS=1 could enable persistent softmax used in Torch.
    const TransformerOptions* options = TransformerOptions::GetInstance();
    bool use_persistent_softmax = options->IsPrecisionMode() && !options->DisablePersistentSoftmax();

    // replace Q*K' in place with masked score for persistent softmax.
    T* persistent_softmax_workspace = data.scratch;
    ORT_RETURN_IF_ERROR(
        ComputeSoftmaxWithRawMask<T>(
            ort_stream, total_sequence_length, sequence_length, batch_size, num_heads,
            mask_index, nullptr, data.relative_position_bias, parameters.broadcast_res_pos_bias,
            data.scratch, scratch2, parameters.is_unidirectional, scale, mask_dimension,
            parameters.max_sequence_length, use_persistent_softmax, persistent_softmax_workspace,
            parameters.mask_filter_value));
  } else if (nullptr != mask_index) {  // 1d mask index
    assert(mask_index_dims.size() == 1);
    // mask_index has 1D shape: either (batch_size) or (2*batch_size). Only the later one has start postions.
    const int* mask_start = (mask_index_dims[0] > batch_size) ? mask_index + batch_size : nullptr;
    ORT_RETURN_IF_ERROR(ComputeSoftmaxWithMask1D<T>(
        stream, total_sequence_length, sequence_length, batch_size, num_heads,
        mask_index, mask_start, data.relative_position_bias, parameters.broadcast_res_pos_bias,
        data.scratch, scratch2, parameters.is_unidirectional));
  } else {  // no mask
    ORT_RETURN_IF_ERROR(
        ComputeSoftmax<T>(
            stream, total_sequence_length, sequence_length, batch_size, num_heads, data.relative_position_bias,
            parameters.broadcast_res_pos_bias, data.scratch, scratch2, parameters.is_unidirectional));
  }

  DUMP_TENSOR_D("Softmax", scratch2, batch_size, num_heads, sequence_length, total_sequence_length);
  DUMP_TENSOR_D("V", data.v, batch_size, num_heads, sequence_length, v_head_size);

  // compute R*V (as V*R), and store in temp_output (space used by Q): BxNxSxH_v
  T* temp_output = data.q;
  CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
      cublas, HIPBLAS_OP_N, HIPBLAS_OP_N,
      v_head_size, sequence_length, total_sequence_length,
      &one, data.v, v_head_size, present_size_per_batch_v,
      scratch2, total_sequence_length, sequence_length * total_sequence_length,
      &zero, temp_output, v_head_size, sequence_length * v_head_size, batches, device_prop));

  // Temp_output is BxNxSxH_v, transpose to output BxSxNxH_v
  Status result = LaunchTransCtx(stream, sequence_length, batch_size, v_head_size, num_heads,
                                 device_prop.maxThreadsPerBlock, false, temp_output, data.output);
  DUMP_TENSOR("unfused output", data.output, batch_size, sequence_length, num_heads, v_head_size);
  return result;
}

template <typename T>
Status QkvToContext(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    Stream* ort_stream,
    contrib::AttentionParameters& parameters,
    AttentionData<T>& data) {
  auto stream = static_cast<hipStream_t>(ort_stream->GetHandle());
  const int max_threads_per_block = device_prop.maxThreadsPerBlock;
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const int total_sequence_length = parameters.total_sequence_length;
  const int num_heads = parameters.num_heads;
  const int qk_head_size = parameters.head_size;
  const int v_head_size = parameters.v_head_size;
  void* fused_runner = data.fused_runner;

  // At most one fused kernel is enabled.
  assert((int(data.use_flash_attention) +
          int(data.use_memory_efficient_attention) +
          int(fused_runner != nullptr) +
          int(data.fused_cross_attention_kernel != nullptr)) <= 1);

  ORT_RETURN_IF_ERROR(PrepareQkv<T>(parameters, data, stream, max_threads_per_block));

  if (!parameters.past_present_share_buffer) {
    ORT_RETURN_IF_ERROR(ConcatPastToPresent(batch_size, num_heads, qk_head_size, v_head_size,
                                            sequence_length, total_sequence_length, parameters.pass_past_in_kv,
                                            stream, max_threads_per_block, data));

  } else {  // past_present_share_buffer
    assert(qk_head_size == v_head_size);
    assert(data.fused_cross_attention_kernel == nullptr);
    assert(nullptr == fused_runner || parameters.is_unidirectional);
    assert(data.gemm_buffer != nullptr);
    assert(!data.use_memory_efficient_attention);
    assert(!data.use_flash_attention);
    assert(data.has_qkv_workspace);

    if (nullptr != data.past_key || nullptr != data.present_key) {
      // TODO: support this case.
      ORT_THROW("buffer sharing for no bias case between past and present is not supported yet.");
    }

    if (data.present != data.past) {
      // For easy testing. Production should better avoid this path.
      int64_t kv_size = 2LL * (int64_t)batch_size * num_heads * parameters.max_sequence_length * qk_head_size;
      hipMemcpyAsync(data.present, data.past, kv_size * sizeof(T), hipMemcpyDeviceToDevice, stream);
    }

    // For fused causal, bias has been added to gemm_buffer.
    const T* bias = (nullptr != fused_runner && parameters.is_unidirectional) ? nullptr : data.bias;

    // append last k v to present
    ORT_RETURN_IF_ERROR(LaunchAddBiasTransAppendKvToPresent(
        stream, parameters.max_sequence_length, parameters.past_sequence_length, sequence_length,
        batch_size, qk_head_size, num_heads, max_threads_per_block,
        bias, data.gemm_buffer, data.present));

    data.k = data.present;
    data.v = data.present + batch_size * num_heads * parameters.max_sequence_length * qk_head_size;
  }

  // Q, K and V are ready now
  if (data.fused_cross_attention_kernel != nullptr) {
    return FusedTrtCrossAttention(stream, parameters, data);
  }

  // Run TRT fused attention.
  if (nullptr != fused_runner) {
    return FusedTrtSelfAttention(stream, parameters, data);
  }

  // For raw attention mask, the scalar 1/sqrt(H) is moved to combine with softmax computation.
  const float scale = parameters.scale == 0.0f ? 1.f / sqrt(static_cast<float>(qk_head_size))
                                               : parameters.scale;

#if USE_FLASH_ATTENTION
  if (data.use_flash_attention) {
    return FlashAttention(device_prop, stream, parameters, data, scale);
  }
#endif

#if USE_MEMORY_EFFICIENT_ATTENTION
  if (data.use_memory_efficient_attention) {
    return EfficientAttention(device_prop, stream, parameters, data, scale);
  }
#endif

  return UnfusedAttention(device_prop, cublas, ort_stream, parameters, data, scale);
}

// Template Instantiation
template struct AttentionData<float>;

template struct AttentionData<half>;

template Status QkvToContext<float>(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    Stream* ort_stream,
    contrib::AttentionParameters& parameters,
    AttentionData<float>& data);

template Status QkvToContext<half>(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    Stream* ort_stream,
    contrib::AttentionParameters& parameters,
    AttentionData<half>& data);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
