#include "hip/hip_runtime.h"
/*
 * The implementation of this file is based on code provided by https://github.com/NVIDIA/FasterTransformer
 *
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// Modifications Copyright (c) Microsoft.
// Licensed under the MIT License.

// Modifications:
// (1) Removed some code paths from the original implementation that had features which is not supported by
//  corresponding ORT kernel - for example- CrossAttention support, FP8, INT8, supports, etc.
// (2) When dealing with masked tokens, this kernel implementation deviates from FasterTransformer by applying
// mask filter values. Appropriate commentary exists in the code below.

#include "contrib_ops/cuda/bert/rotary_embedding_util.h"
#include "decoder_masked_multihead_attention_impl.h"
#include "decoder_masked_multihead_attention_impl_utils.h"
#include <cfloat>

namespace onnxruntime {
namespace contrib {
namespace cuda {

using namespace decoder_masked_self_attention_details;

template <
    // The type of the inputs. Supported types: float and half.
    typename T,
    // The hidden dimension per head.
    int head_size,
    // The number of threads per key.
    int THREADS_PER_KEY,
    // The number of threads per value.
    int THREADS_PER_VALUE,
    // The number of threads in a threadblock.
    int THREADS_PER_BLOCK>
__global__ void masked_multihead_attention_kernel(DecoderMaskedMultiHeadAttentionParams params) {
  // This kernel contains some code that cannot be compiled on CUDA ARCH 5.3 or lower
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 530
  (void)(params);
#else

  // Make sure the hidden dimension per head is a multiple of the number of threads per key.
  static_assert(head_size % THREADS_PER_KEY == 0, "");

  // Make sure the hidden dimension per head is a multiple of the number of threads per value.
  static_assert(head_size % THREADS_PER_VALUE == 0, "");

  // The size of a warp.
  constexpr int WARP_SIZE = 32;

  // The number of warps in a threadblock.
  constexpr int WARPS_PER_BLOCK = THREADS_PER_BLOCK / WARP_SIZE;

  extern __shared__ char smem_[];

  // The shared memory for the Q*K^T values and partial logits in softmax.
  float* qk_smem = reinterpret_cast<float*>(smem_);

  // The shared memory for the logits. For FP32, that's the same buffer as qk_smem.
  char* logits_smem_ = smem_;

  if (sizeof(T) != 4) {
    // For fp16, we have allocated separate memory for logits - use it
    logits_smem_ += (((params.total_sequence_length + 3) / 4) * 16);
  }

  T* logits_smem = reinterpret_cast<T*>(logits_smem_);

  // The shared memory to do the final reduction for the output values. Reuse qk_smem.
  T* out_smem = reinterpret_cast<T*>(smem_);

  // The shared memory buffers for the block-wide reductions. One for max, one for sum.
  __shared__ float red_smem[WARPS_PER_BLOCK * 2];

  // A vector of Q or K elements for the current timestep.
  using Qk_vec_k = typename Qk_vec_k_<T, head_size>::Type;  // with kernel-used precision
  using Qk_vec_m = typename Qk_vec_m_<T, head_size>::Type;  // with memory-used precision

  // Use alignment for safely casting the shared buffers as Qk_vec_k.
  // Shared memory to store Q inputs.
  __shared__ __align__(sizeof(Qk_vec_k)) T q_smem[head_size];

  // The number of elements per vector.
  constexpr int QK_VEC_SIZE = sizeof(Qk_vec_m) / sizeof(T);

  // Make sure the hidden size per head is a multiple of the vector size.
  static_assert(head_size % QK_VEC_SIZE == 0, "");

  constexpr int QK_VECS_PER_WARP = head_size / QK_VEC_SIZE;

  // The layout of the cache is [B, H, head_size/x, L, x] with x == 4/8/16 for FP32/FP16/FP8. Since each thread
  // owns x elements, we have to decompose the linear index into chunks of x values and the posi-
  // tion of the thread in that chunk.

  // The number of elements in a chunk of 16B (that's the x in the above formula).
  constexpr int QK_ELTS_IN_16B = 16 / sizeof(T);

  // The number of K vectors in 16B.
  constexpr int QK_VECS_IN_16B = 16 / sizeof(Qk_vec_m);

  // The batch/beam idx
  const int bi = blockIdx.y;

  // The beam idx
  // const int beami = bi % params.beam_width;

  // The "beam-aware" batch idx
  const int bbi = bi / params.beam_width;

  // The head.
  const int hi = blockIdx.x;

  // Combine the batch and the head indices.
  const int bhi = bi * params.num_heads + hi;

  // Combine the "beam-aware" batch idx and the head indices.
  const int bbhi = bbi * params.beam_width * params.num_heads + hi;

  // The thread in the block.
  const int tidx = threadIdx.x;

  // While doing the product Q*K^T for the different keys we track the max.
  float qk_max = -FLT_MAX;

  float qk = 0.0F;

  int qkv_base_offset = params.is_mha && !params.is_packed_qkv
                            ? bi * params.hidden_size + hi * head_size
                            : bi * (3 * params.hidden_size) + hi * head_size;

  const size_t bi_total_seq_length = bi * params.total_sequence_length;

  const size_t bi_max_seq_length = bi * params.max_sequence_length;

  int tlength = params.is_cross_attention ? params.kv_sequence_length : params.past_sequence_length;

  // First QK_VECS_PER_WARP load Q and K + the bias values for the current timestep.
  const bool is_masked = tidx >= QK_VECS_PER_WARP;

  // The offset in the Q and K buffer also accounts for the batch.
  int qk_offset = qkv_base_offset + tidx * QK_VEC_SIZE;

  // Trigger the loads from the Q and K buffers.
  Qk_vec_k q;
  zero(q);

  if (!is_masked) {
    q = vec_conversion<Qk_vec_k, Qk_vec_m>(*reinterpret_cast<const Qk_vec_m*>(&reinterpret_cast<T*>(params.q)[qk_offset]));
  }

  // The offset in the bias buffer.
  int qk_bias_offset = hi * head_size + tidx * QK_VEC_SIZE;

  // Trigger the loads from the Q and K bias buffers.
  if (params.q_bias && !is_masked) {
    Qk_vec_k q_bias;

    q_bias = vec_conversion<Qk_vec_k, Qk_vec_m>(*reinterpret_cast<const Qk_vec_m*>(&reinterpret_cast<T*>(params.q_bias)[qk_bias_offset]));

    q = add_vec(q, q_bias);
  }


  T* params_k_cache = reinterpret_cast<T*>(params.k_cache);

  const float inv_sqrt_dh = params.scale;

  if (!is_masked) {
    // Store the Q values to shared memory.
    *reinterpret_cast<Qk_vec_k*>(&q_smem[tidx * QK_VEC_SIZE]) = q;
  }

  if (!params.is_cross_attention) {
    Qk_vec_k k;

    zero(k);

    if (!is_masked) {
      k = vec_conversion<Qk_vec_k, Qk_vec_m>(*reinterpret_cast<const Qk_vec_m*>(&reinterpret_cast<T*>(params.k)[qk_offset]));

      if (params.k_bias) {
        Qk_vec_k k_bias;

        k_bias = vec_conversion<Qk_vec_k, Qk_vec_m>(*reinterpret_cast<const Qk_vec_m*>(&reinterpret_cast<T*>(params.k_bias)[qk_bias_offset]));

        k = add_vec(k, k_bias);
      }
    }

    if (params.rotary_embedding_dim > 0) {
      const bool do_rotary = !is_masked && QK_VEC_SIZE * tidx < params.rotary_embedding_dim;

      T* q_smem = reinterpret_cast<T*>(smem_);
      T* k_smem = q_smem + params.rotary_embedding_dim;

      const int half_rotary_dim = params.rotary_embedding_dim / 2;
      const int half_idx = (tidx * QK_VEC_SIZE) / half_rotary_dim;
      const int intra_half_idx = (tidx * QK_VEC_SIZE) % half_rotary_dim;
      const int smem_pitch = half_rotary_dim;

      assert(half_rotary_dim % QK_VEC_SIZE == 0);

      if (do_rotary) {
        *reinterpret_cast<Qk_vec_k*>(q_smem + half_idx * smem_pitch + intra_half_idx) = q;
        *reinterpret_cast<Qk_vec_k*>(k_smem + half_idx * smem_pitch + intra_half_idx) = k;
      }

      __syncthreads();

      const int transpose_idx = half_idx * (half_rotary_dim / 2) + intra_half_idx / 2;
      constexpr int tidx_factor = (QK_VEC_SIZE > 1) ? QK_VEC_SIZE / 2 : 1;

      if (do_rotary) {
        vec_from_smem_transpose(q, q_smem, transpose_idx, smem_pitch);
        vec_from_smem_transpose(k, k_smem, transpose_idx, smem_pitch);

        apply_rotary_embedding(
            q, k, transpose_idx / tidx_factor, params.rotary_embedding_dim, params.t_step);

        write_smem_transpose(k, k_smem, transpose_idx, smem_pitch);
        write_smem_transpose(q, q_smem, transpose_idx, smem_pitch);
      }

      __syncthreads();

      if (do_rotary) {
        q = *reinterpret_cast<Qk_vec_k*>(q_smem + half_idx * smem_pitch + intra_half_idx);
        k = *reinterpret_cast<Qk_vec_k*>(k_smem + half_idx * smem_pitch + intra_half_idx);
      }

      __syncthreads();
    }

    if (!is_masked) {
      // Write the K values to the global memory cache.
      // NOTE: The stores are uncoalesced as we have multiple chunks of 16B spread across the memory
      // system. We designed it this way as it allows much better memory loads (and there are many
      // more loads) + the stores are really "write and forget" since we won't need the ack before
      // the end of the kernel. There's plenty of time for the transactions to complete.

      // The 16B chunk written by the thread.
      int co = tidx / QK_VECS_IN_16B;

      // The position of the thread in that 16B chunk.
      int ci = tidx % QK_VECS_IN_16B * QK_VEC_SIZE;

      // Two chunks are separated by L * x elements. A thread write QK_VEC_SIZE elements.
      int offset = bhi * params.max_sequence_length * head_size + co * params.max_sequence_length * QK_ELTS_IN_16B +
                   tlength * QK_ELTS_IN_16B + ci;

      // Trigger the stores to global memory.
      *reinterpret_cast<Qk_vec_m*>(&params_k_cache[offset]) = vec_conversion<Qk_vec_m, Qk_vec_k>(k);

      // Compute \sum_i Q[i] * K^T[i] for the current timestep.
      using Qk_vec_acum = Qk_vec_k;
      qk = dot<Qk_vec_acum, Qk_vec_k>(q, k);

      if (QK_VECS_PER_WARP <= WARP_SIZE) {
#pragma unroll
        for (int mask = QK_VECS_PER_WARP / 2; mask >= 1; mask /= 2) {
          qk += __shfl_xor_sync(shfl_mask(QK_VECS_PER_WARP), qk, mask);
        }
      }
    }

    if (QK_VECS_PER_WARP > WARP_SIZE) {
      constexpr int WARPS_PER_RED = (QK_VECS_PER_WARP + WARP_SIZE - 1) / WARP_SIZE;
      qk = block_sum<WARPS_PER_RED>(&red_smem[WARPS_PER_RED], qk);
    }

    // Store that value in shared memory. Keep the Q*K^T value in register for softmax.
    if (tidx == 0) {
      // Normalize qk.
      qk *= inv_sqrt_dh;
      if (params.relative_attention_bias != nullptr) {
        qk = add_vec(qk,
                     reinterpret_cast<T*>(params.relative_attention_bias)[hi * params.sequence_length * params.total_sequence_length + tlength]);
      }
      qk_max = qk;
      qk_smem[tlength] = qk;
    }
  }

  // Make sure the data is in shared memory.
  __syncthreads();

  // The type of queries and keys for the math in the Q*K^T product.
  using K_vec_k = typename K_vec_k_<T, THREADS_PER_KEY>::Type;
  using K_vec_m = typename K_vec_m_<T, THREADS_PER_KEY>::Type;

  // The number of elements per vector.
  constexpr int K_VEC_SIZE = sizeof(K_vec_m) / sizeof(T);

  // Make sure the hidden size per head is a multiple of the vector size.
  static_assert(head_size % K_VEC_SIZE == 0, "");

  // The number of elements per thread.
  constexpr int K_ELTS_PER_THREAD = head_size / THREADS_PER_KEY;

  // The number of vectors per thread.
  constexpr int K_VECS_PER_THREAD = K_ELTS_PER_THREAD / K_VEC_SIZE;

  // The position the first key loaded by each thread from the cache buffer (for this B * H).
  int ko = tidx / THREADS_PER_KEY;

  // The position of the thread in the chunk of keys.
  int ki = tidx % THREADS_PER_KEY * K_VEC_SIZE;

  static_assert(head_size == THREADS_PER_KEY * K_VEC_SIZE * K_VECS_PER_THREAD);

  // Load the Q values from shared memory. The values are reused during the loop on K.
  K_vec_k q_vec[K_VECS_PER_THREAD];
#pragma unroll
  for (int ii = 0; ii < K_VECS_PER_THREAD; ++ii) {
    q_vec[ii] = *reinterpret_cast<const K_vec_k*>(&q_smem[ki + ii * THREADS_PER_KEY * K_VEC_SIZE]);
  }

  // The number of timesteps loaded per iteration.
  constexpr int K_PER_ITER = THREADS_PER_BLOCK / THREADS_PER_KEY;

  // The number of keys per warp.
  constexpr int K_PER_WARP = WARP_SIZE / THREADS_PER_KEY;

  // Base pointer for the beam's batch, before offsetting with indirection buffer
  T* k_cache_batch = &params_k_cache[bbhi * params.max_sequence_length * head_size + ki];

  // Pick a number of keys to make sure all the threads of a warp enter (due to shfl_sync).
  int ti_end = ((tlength + K_PER_WARP - 1) / K_PER_WARP) * K_PER_WARP;

  // Iterate over the keys/timesteps to compute the various (Q*K^T)_{ti} values.
  bool has_beams = params.cache_indir != nullptr && !params.is_cross_attention;
  const int* beam_indices = has_beams ? &params.cache_indir[bi_max_seq_length] : nullptr;

  for (int ti = ko; ti < ti_end; ti += K_PER_ITER) {
    bool is_masked = (params.mask != nullptr) && (params.mask[bi_total_seq_length + ti] == 0);

    // The keys loaded from the key cache.
    K_vec_k k_vec[K_VECS_PER_THREAD];

    if (has_beams) {
#pragma unroll
      for (int ii = 0; ii < K_VECS_PER_THREAD; ++ii) {
        int jj = ii * params.max_sequence_length + ti;

        if (ti < tlength) {
          const int beam_offset = beam_indices[ti] * params.num_heads * params.max_sequence_length * head_size;
          k_vec[ii] = vec_conversion<K_vec_k, K_vec_m>(
              (*reinterpret_cast<const K_vec_m*>(&k_cache_batch[beam_offset + jj * QK_ELTS_IN_16B])));
        }
      }
    } else {
#pragma unroll
      for (int ii = 0; ii < K_VECS_PER_THREAD; ++ii) {
        int jj = ii * params.max_sequence_length + ti;

        if (ti < tlength) {
          k_vec[ii] = vec_conversion<K_vec_k, K_vec_m>(
              (*reinterpret_cast<const K_vec_m*>(&k_cache_batch[jj * QK_ELTS_IN_16B])));
        }
      }
    }

    // Perform the dot product and normalize qk.
    // WARNING: ALL THE THREADS OF A WARP MUST ENTER!!!
    float qk = Qk_dot<T, THREADS_PER_KEY>::dot(q_vec, k_vec) * inv_sqrt_dh;

    // This is a deviation from FasterTransformer kernel implementation
    // but this aligns with ORT's other Attention kernels which strives to
    // mimic PyTorch when dealing with mask filter values
    if (is_masked) {
      qk += params.mask_filter_value;
    }

    // Store the product to shared memory. There's one qk value per timestep. Update the max.
    if (ti < tlength && tidx % THREADS_PER_KEY == 0) {
      if (params.relative_attention_bias != nullptr) {
        qk = add_vec(qk,
                     reinterpret_cast<T*>(params.relative_attention_bias)[hi * params.sequence_length * params.total_sequence_length + ti]);
      }
      qk_max = fmaxf(qk_max, qk);
      qk_smem[ti] = qk;
    }
  }

  // Perform the final reduction to compute the max inside each warp.
  //
  // NOTE: In a group of THREADS_PER_KEY threads, the leader already has the max value for the
  // group so it's not needed to run the reduction inside the group (again).
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= THREADS_PER_KEY; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }

  // Decompose the thread index into warp and lane.
  const int warp = tidx / WARP_SIZE;
  const int lane = tidx % WARP_SIZE;

  // The warp leader writes the max to shared memory.
  if (lane == 0) {
    red_smem[warp] = qk_max;
  }

  // Make sure the products are in shared memory.
  __syncthreads();

  // The warps finalize the reduction.
  qk_max = lane < WARPS_PER_BLOCK ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = WARPS_PER_BLOCK / 2; mask >= 1; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }

  // Broadcast to all the threads in the warp.
  qk_max = __shfl_sync(uint32_t(-1), qk_max, 0);

  // Compute the logits and start the sum.
  float sum = 0.f;
  int sum_tlength = params.is_cross_attention ? tlength - 1 : tlength;
  for (int ti = tidx; ti <= sum_tlength; ti += THREADS_PER_BLOCK) {
    // This is a deviation from FasterTransformer kernel implementation
    // but this aligns with ORT's other Attention kernels which strives to
    // mimic PyTorch when dealing with mask filter values
    float logit = __expf(qk_smem[ti] - qk_max);
    sum += logit;
    qk_smem[ti] = logit;
  }

  // Compute the sum.
  sum = block_sum<WARPS_PER_BLOCK>(&red_smem[WARPS_PER_BLOCK], sum);

  // Normalize the logits.
  float inv_sum = __fdividef(1.f, sum + 1.e-6f);
  for (int ti = tidx; ti <= sum_tlength; ti += THREADS_PER_BLOCK) {
    float logit = qk_smem[ti] * inv_sum;
    ConvertFromFloat(logits_smem[ti], logit);
  }

  // Put Values part below so we leverage __syncthreads
  // from the previous step

  // The number of elements per vector.
  constexpr int V_VEC_SIZE = head_size / THREADS_PER_VALUE;

  // A vector of V elements for the current timestep.
  using V_vec_k = typename V_vec_k_<T, V_VEC_SIZE>::Type;
  using V_vec_m = typename V_vec_m_<T, V_VEC_SIZE>::Type;

  // The value computed by this thread.
  int vo = tidx / THREADS_PER_VALUE;

  // The hidden dimensions computed by this particular thread.
  int vi = tidx % THREADS_PER_VALUE * V_VEC_SIZE;

  // The base pointer for the value in the cache buffer.
  T* params_v_cache = reinterpret_cast<T*>(params.v_cache);

  T* v_cache = &params_v_cache[bhi * params.max_sequence_length * head_size + vi];

  // Base pointer for the beam's batch, before offsetting with indirection buffer
  T* v_cache_batch = &params_v_cache[bbhi * params.max_sequence_length * head_size + vi];

  // The number of values processed per iteration of the loop.
  constexpr int V_PER_ITER = THREADS_PER_BLOCK / THREADS_PER_VALUE;

  // One group of threads computes the product(s) for the current timestep.
  V_vec_k v_bias;
  if (params.v_bias && !params.is_cross_attention) {
    zero(v_bias);

    T* params_v_bias = reinterpret_cast<T*>(params.v_bias);

    if (vo == tlength % V_PER_ITER) {
      v_bias = vec_conversion<V_vec_k, V_vec_m>(*reinterpret_cast<const V_vec_m*>(&params_v_bias[hi * head_size + vi]));
    }
  }

  // From previous, before values, step
  // Also make sure the logits are in shared memory.
  __syncthreads();

  using V_vec_acum = typename V_vec_acum_fp32_<V_vec_k>::Type;

  // The partial outputs computed by each thread.
  V_vec_acum out;
  zero(out);

  // Loop over the timesteps to compute the partial outputs.
  for (int ti = vo; ti < tlength; ti += V_PER_ITER) {
    // Fetch offset based on cache_indir when beam sampling
    const int beam_src = has_beams ? params.cache_indir[bi_max_seq_length + ti] : 0;
    const int beam_offset = has_beams ? beam_src * params.num_heads * params.max_sequence_length * head_size : 0;

    // Load the values from the cache.
    V_vec_k v = vec_conversion<V_vec_k, V_vec_m>(*reinterpret_cast<const V_vec_m*>(&v_cache_batch[beam_offset + ti * head_size]));

    // Load the logits from shared memory.
    T logit = logits_smem[ti];
    out = fma(logit, v, out);
  }

  // One group of threads computes the product(s) for the current timestep.
  if (vo == tlength % V_PER_ITER && !params.is_cross_attention) {
    const auto v_offset = qkv_base_offset + vi;

    V_vec_k v;
    v = vec_conversion<V_vec_k, V_vec_m>(*reinterpret_cast<const V_vec_m*>(&reinterpret_cast<T*>(params.v)[v_offset]));
    if (params.v_bias) {
      v = add_vec(v, v_bias);
    }

    // Store the values with bias back to global memory in the cache for V.
    *reinterpret_cast<V_vec_m*>(&v_cache[tlength * head_size]) = vec_conversion<V_vec_m, V_vec_k>(v);

    // Initialize the output value with the current timestep.
    out = fma(logits_smem[tlength], v, out);
  }

  // Make sure we can start writing to shared memory.
  __syncthreads();

  // Run the final reduction amongst the different groups computing different partial outputs.
#pragma unroll
  for (int active_groups = V_PER_ITER; active_groups >= 2; active_groups /= 2) {
    // The midpoint in the number of active groups.
    int midpoint = active_groups / 2;

    // The upper part of active threads store to shared memory.
    if (vo >= midpoint && vo < active_groups) {
      ConvertFromFloat(*reinterpret_cast<V_vec_k*>(&out_smem[(vo - midpoint) * head_size + vi]), out);
    }
    __syncthreads();

    // The bottom warps update their values.
    if (vo < midpoint) {
      out = add_vec(*reinterpret_cast<const V_vec_k*>(&out_smem[vo * head_size + vi]), out);
    }
    __syncthreads();
  }

  // Output the final values.
  T* params_out = reinterpret_cast<T*>(params.out);
  if (vo == 0) {
    ConvertFromFloat(*reinterpret_cast<V_vec_m*>(&params_out[bhi * head_size + vi]), out);
  }
#endif
}

// Template instantiation(s)

// fp32 + head size = 32
template void __global__ masked_multihead_attention_kernel<float, 32, 4, 8, 64>(DecoderMaskedMultiHeadAttentionParams params);

template void __global__ masked_multihead_attention_kernel<float, 32, 2, 8, 128>(DecoderMaskedMultiHeadAttentionParams params);

template void __global__ masked_multihead_attention_kernel<float, 32, 1, 8, 256>(DecoderMaskedMultiHeadAttentionParams params);

// fp16 + head size = 32
template void __global__ masked_multihead_attention_kernel<uint16_t, 32, 4, 4, 64>(DecoderMaskedMultiHeadAttentionParams params);

template void __global__ masked_multihead_attention_kernel<uint16_t, 32, 2, 4, 128>(DecoderMaskedMultiHeadAttentionParams params);

template void __global__ masked_multihead_attention_kernel<uint16_t, 32, 1, 4, 256>(DecoderMaskedMultiHeadAttentionParams params);

// fp32 + head size = 64
template void __global__ masked_multihead_attention_kernel<float, 64, 4, 16, 64>(DecoderMaskedMultiHeadAttentionParams params);

template void __global__ masked_multihead_attention_kernel<float, 64, 2, 16, 128>(DecoderMaskedMultiHeadAttentionParams params);

template void __global__ masked_multihead_attention_kernel<float, 64, 1, 16, 256>(DecoderMaskedMultiHeadAttentionParams params);

// fp16 + head size = 64
template void __global__ masked_multihead_attention_kernel<uint16_t, 64, 4, 8, 64>(DecoderMaskedMultiHeadAttentionParams params);

template void __global__ masked_multihead_attention_kernel<uint16_t, 64, 2, 8, 128>(DecoderMaskedMultiHeadAttentionParams params);

template void __global__ masked_multihead_attention_kernel<uint16_t, 64, 1, 8, 256>(DecoderMaskedMultiHeadAttentionParams params);

// fp32 + head size = 128
template void __global__ masked_multihead_attention_kernel<float, 128, 4, 32, 64>(DecoderMaskedMultiHeadAttentionParams params);

template void __global__ masked_multihead_attention_kernel<float, 128, 2, 32, 128>(DecoderMaskedMultiHeadAttentionParams params);

template void __global__ masked_multihead_attention_kernel<float, 128, 1, 32, 256>(DecoderMaskedMultiHeadAttentionParams params);

// fp16 + head size = 128
template void __global__ masked_multihead_attention_kernel<uint16_t, 128, 4, 16, 64>(DecoderMaskedMultiHeadAttentionParams params);

template void __global__ masked_multihead_attention_kernel<uint16_t, 128, 2, 16, 128>(DecoderMaskedMultiHeadAttentionParams params);

template void __global__ masked_multihead_attention_kernel<uint16_t, 128, 1, 16, 256>(DecoderMaskedMultiHeadAttentionParams params);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
