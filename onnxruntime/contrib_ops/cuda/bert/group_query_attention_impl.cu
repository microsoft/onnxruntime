#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on our Multi-Head Attention impl.cu file,
 which is based on qkvToContext plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/

Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications:
// (1) support GPT-2 past state, unidirectional mask (causal)
// (2) use flash attention kernel from (https://github.com/Dao-AILab/flash-attention)
// (3) support different number of heads for Q and KV
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <cassert>
#include <hip/hip_fp16.h>
#include <hipcub/hipcub.hpp>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/shared_inc/fpgeneric.h"
#include "contrib_ops/cuda/bert/attention_softmax.h"
#include "contrib_ops/cuda/bert/transformer_common.h"
#include "contrib_ops/cuda/bert/add_bias_transpose.h"
#include "contrib_ops/cpu/bert/attention_base.h"
#include "contrib_ops/cuda/bert/bert_padding.h"
#include "contrib_ops/cuda/transformers/dump_cuda_tensor.h"
#include "contrib_ops/cuda/bert/cutlass_fmha/memory_efficient_attention.h"
#include "contrib_ops/cuda/bert/flash_attention/flash_api.h"
#include "contrib_ops/cuda/bert/group_query_attention_impl.h"
#include "contrib_ops/cuda/bert/attention_impl.h"
#include "core/providers/cuda/shared_inc/cuda_call.h"
#include "contrib_ops/cuda/bert/rotary_embedding_impl.h"
#include <hipblas.h>

using namespace onnxruntime::cuda;

// Macro to help compute index of flatten 4D matrix, note that dim1 is not used so it is excluded.
#define INDEX_4D(dim2, dim3, dim4, i, j, k, l) ((i) * (dim2) * (dim3) * (dim4) + (j) * (dim3) * (dim4) + (k) * (dim4) + (l))

namespace onnxruntime {
namespace contrib {
namespace cuda {

////////// Auxiliary Kernels for KV prep

// Kernel for seqlens_k
__global__ void repeat_seqlen(int32_t* seqlens_k, int32_t seqlen, int batch_size) {
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < batch_size) seqlens_k[id] = seqlen;
}

// Kernel to append new and past kv in either BSNH or BNSH format
// Adapted from ConcatTensorToTensor kernel in attention_kv_cache.cu file
template <typename T>
__global__ void ConcatNewToPastKV(const int new_seqlen,
                                  const int past_buffer_seqlen,
                                  const T* past_kv,
                                  const T* new_kv,
                                  T* present_kv,
                                  const int* seqlens_k,
                                  const bool is_bsnh) {  // refers to past; otherwise bnsh
  const int h = threadIdx.x;
  const int n = threadIdx.y;
  const int s = blockIdx.x;
  const int b = blockIdx.y;

  const int present_buffer_seqlen = gridDim.x;
  const int num_heads = blockDim.y;
  const int H = blockDim.x;

  const int present_batch_stride = present_buffer_seqlen * num_heads * H;
  const int row_stride = is_bsnh ? num_heads * H : H;
  const int present_head_stride = is_bsnh ? H : present_buffer_seqlen * H;

  // past_kv:     BPNH or BNPH
  // new_kv:      BLNH
  // present_kv:  BTNH or BNTH, where T = P + L
  const int past_seqlen = seqlens_k == nullptr ? 0 : seqlens_k[b];

  int out_offset = b * present_batch_stride + s * row_stride + n * present_head_stride + h;
  if (s < past_seqlen) {
    const int past_batch_stride = past_buffer_seqlen * num_heads * H;
    const int past_head_stride = is_bsnh ? H : past_buffer_seqlen * H;
    const int in_offset = b * past_batch_stride + s * row_stride + n * past_head_stride + h;
    present_kv[out_offset] = past_kv[in_offset];
  } else if (s < past_seqlen + new_seqlen) {
    // Note: new KV always BSNH
    const int new_batch_stride = new_seqlen * num_heads * H;
    const int new_row_stride = num_heads * H;
    const int new_head_stride = H;
    const int in_offset = b * new_batch_stride + (s - past_seqlen) * new_row_stride + n * new_head_stride + h;
    present_kv[out_offset] = new_kv[in_offset];
  }
}

// Use when (H*)*num_heads > 1024
template <typename T>
__global__ void ConcatNewToPastKVLarge(const int new_seqlen,
                                       const int past_buffer_seqlen,
                                       const int H,
                                       const int num_heads,
                                       const T* past_kv,
                                       const T* new_kv,
                                       T* present_kv,
                                       const int* seqlens_k,
                                       const bool is_bsnh) {
  int i = threadIdx.x + (blockDim.x * blockIdx.x);
  if (i < H * num_heads) {
    const int h = i % H;
    const int n = i / H;
    const int s = blockIdx.y;
    const int b = blockIdx.z;
    const int present_buffer_seqlen = gridDim.y;

    const int present_batch_stride = present_buffer_seqlen * num_heads * H;
    const int row_stride = is_bsnh ? num_heads * H : H;
    const int present_head_stride = is_bsnh ? H : present_buffer_seqlen * H;

    // past_kv:     BPNH or BNPH
    // new_kv:      BLNH
    // present_kv:  BTNH or BNTH, where T = P + L
    const int past_seqlen = seqlens_k == nullptr ? 0 : seqlens_k[b];

    int out_offset = b * present_batch_stride + s * row_stride + n * present_head_stride + h;
    if (s < past_seqlen) {
      const int past_batch_stride = past_buffer_seqlen * num_heads * H;
      const int past_head_stride = is_bsnh ? H : past_buffer_seqlen * H;
      const int in_offset = b * past_batch_stride + s * row_stride + n * past_head_stride + h;
      present_kv[out_offset] = past_kv[in_offset];
    } else if (s < past_seqlen + new_seqlen) {
      const int new_batch_stride = new_seqlen * num_heads * H;
      const int new_row_stride = num_heads * H;
      const int new_head_stride = H;
      const int in_offset = b * new_batch_stride + (s - past_seqlen) * new_row_stride + n * new_head_stride + h;
      present_kv[out_offset] = new_kv[in_offset];
    }
  }
}

// Concat new to past in present. Supports past BSNH or past BNSH
template <typename T>
Status LaunchConcatNewToPastKV(contrib::GroupQueryAttentionParameters& parameters,
                               GroupQueryAttentionData<T>& data,
                               const void* new_key,
                               const void* new_value,
                               hipStream_t stream,
                               const int max_threads_per_block,
                               const bool past_only = false) {
  const int batch_size = parameters.batch_size;
  const int kv_sequence_length = past_only ? 0 : parameters.sequence_length;
  const int past_sequence_length = parameters.seqlen_past_kv_cache;
  const int present_sequence_length = parameters.seqlen_present_kv_cache;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;
  const int* seqlens_k = parameters.is_prompt ? nullptr : reinterpret_cast<const int*>(data.seqlens_k);

  AttentionQkvFormat past_kv_format = parameters.past_kv_format;

  assert(past_kv_format == AttentionQkvFormat::Q_K_V_BSNH || past_kv_format == AttentionQkvFormat::Q_K_V_BNSH);
  const int H = head_size / 4;  // divide by 4 so kernel can operate on 4 float16 elements at a time.
  if (H * kv_num_heads <= max_threads_per_block) {
    const dim3 grid(present_sequence_length, batch_size, 1);
    const dim3 block(H, kv_num_heads, 1);
    ConcatNewToPastKV<float2><<<grid, block, 0, stream>>>(kv_sequence_length,
                                                          past_sequence_length,
                                                          reinterpret_cast<const float2*>(data.past_key),
                                                          reinterpret_cast<const float2*>(new_key),
                                                          reinterpret_cast<float2*>(data.present_key),
                                                          seqlens_k,
                                                          past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
    ConcatNewToPastKV<float2><<<grid, block, 0, stream>>>(kv_sequence_length,
                                                          past_sequence_length,
                                                          reinterpret_cast<const float2*>(data.past_value),
                                                          reinterpret_cast<const float2*>(new_value),
                                                          reinterpret_cast<float2*>(data.present_value),
                                                          seqlens_k,
                                                          past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
  } else {
    int steps = (H * kv_num_heads + 255) / 256;
    const dim3 grid(steps, present_sequence_length, batch_size);
    const dim3 block(256, 1, 1);
    ConcatNewToPastKVLarge<float2><<<grid, block, 0, stream>>>(kv_sequence_length,
                                                               past_sequence_length,
                                                               H,
                                                               kv_num_heads,
                                                               reinterpret_cast<const float2*>(data.past_key),
                                                               reinterpret_cast<const float2*>(new_key),
                                                               reinterpret_cast<float2*>(data.present_key),
                                                               seqlens_k,
                                                               past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
    ConcatNewToPastKVLarge<float2><<<grid, block, 0, stream>>>(kv_sequence_length,
                                                               past_sequence_length,
                                                               H,
                                                               kv_num_heads,
                                                               reinterpret_cast<const float2*>(data.past_value),
                                                               reinterpret_cast<const float2*>(new_value),
                                                               reinterpret_cast<float2*>(data.present_value),
                                                               seqlens_k,
                                                               past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
  }
  return CUDA_CALL(hipGetLastError());
}

// Kernel to append new kv to kv buffer in place
template <typename T>
__global__ void ConcatKVInPlace(const int max_seqlen,
                                T* kv_buff,
                                const T* new_kv,
                                const int* past_seqlens_k,
                                const int* total_seqlens_k,
                                const bool is_past_kv_bnsh_format,
                                const bool is_new_kv_bnsh_format) {
  const int h = threadIdx.x;
  const int n = threadIdx.y;
  const int s = blockIdx.x;
  const int b = blockIdx.y;

  const int new_seqlen = gridDim.x;
  const int kv_num_heads = blockDim.y;
  const int H = blockDim.x;

  const int past_seq_len = (total_seqlens_k != nullptr)
                               ? (total_seqlens_k[b] - new_seqlen)
                               : (past_seqlens_k == nullptr ? 0 : past_seqlens_k[b]);

  int out_offset = is_past_kv_bnsh_format
                       ? INDEX_4D(kv_num_heads, max_seqlen, H, b, n, s + past_seq_len, h)
                       : INDEX_4D(max_seqlen, kv_num_heads, H, b, s + past_seq_len, n, h);

  int in_offset = is_new_kv_bnsh_format
                      ? INDEX_4D(kv_num_heads, new_seqlen, H, b, n, s, h)
                      : INDEX_4D(new_seqlen, kv_num_heads, H, b, s, n, h);

  kv_buff[out_offset] = new_kv[in_offset];
}

template <typename T>
__global__ void ConcatKVInPlaceLarge(const int max_seqlen,
                                     const int H,
                                     const int kv_num_heads,
                                     T* kv_buff,
                                     const T* new_kv,
                                     const int* past_seqlens_k,
                                     const int* total_seqlens_k,
                                     const bool is_past_kv_bnsh_format,
                                     const bool is_new_kv_bnsh_format) {  // refers to kv buff; otherwise bnsh
  int i = threadIdx.x + (blockDim.x * blockIdx.x);
  if (i < H * kv_num_heads) {
    const int h = i % H;
    const int n = i / H;
    const int s = blockIdx.y;
    const int b = blockIdx.z;
    const int new_seqlen = gridDim.y;
    const int past_seq_len = (total_seqlens_k != nullptr)
                                 ? (total_seqlens_k[b] - new_seqlen)
                                 : (past_seqlens_k == nullptr ? 0 : past_seqlens_k[b]);

    int out_offset = is_past_kv_bnsh_format
                         ? INDEX_4D(kv_num_heads, max_seqlen, H, b, n, s + past_seq_len, h)
                         : INDEX_4D(max_seqlen, kv_num_heads, H, b, s + past_seq_len, n, h);

    int in_offset = is_new_kv_bnsh_format
                        ? INDEX_4D(kv_num_heads, new_seqlen, H, b, n, s, h)
                        : INDEX_4D(new_seqlen, kv_num_heads, H, b, s, n, h);

    kv_buff[out_offset] = new_kv[in_offset];
  }
}

// Concat new to kv buffer in place
template <typename T>
Status LaunchConcatKVInPlace(int batch_size,
                             int kv_num_heads,
                             int head_size,
                             int max_sequence_length,
                             const int* past_seqlens_k,
                             const int* total_seqlens_k,
                             int new_seq_len,
                             const T* new_key,
                             const T* new_value,
                             T* present_key,
                             T* present_value,
                             bool is_past_kv_bnsh_format,
                             bool is_new_kv_bnsh_format,
                             hipStream_t stream,
                             const int max_threads_per_block) {
  static_assert(sizeof(T) == 2);
  assert(head_size % 4 == 0);

  const int H = head_size / 4;
  if (H * kv_num_heads <= max_threads_per_block) {
    const dim3 grid(new_seq_len, batch_size, 1);
    const dim3 block(H, kv_num_heads, 1);
    ConcatKVInPlace<float2><<<grid, block, 0, stream>>>(max_sequence_length,
                                                        reinterpret_cast<float2*>(present_key),
                                                        reinterpret_cast<const float2*>(new_key),
                                                        past_seqlens_k,
                                                        total_seqlens_k,
                                                        is_past_kv_bnsh_format,
                                                        is_new_kv_bnsh_format);
    ConcatKVInPlace<float2><<<grid, block, 0, stream>>>(max_sequence_length,
                                                        reinterpret_cast<float2*>(present_value),
                                                        reinterpret_cast<const float2*>(new_value),
                                                        past_seqlens_k,
                                                        total_seqlens_k,
                                                        is_past_kv_bnsh_format,
                                                        is_new_kv_bnsh_format);
  } else {
    int steps = int(ceil(float(H * kv_num_heads) / 256.0));
    const dim3 grid(steps, new_seq_len, batch_size);
    const dim3 block(256, 1, 1);
    ConcatKVInPlaceLarge<float2><<<grid, block, 0, stream>>>(max_sequence_length,
                                                             H,
                                                             kv_num_heads,
                                                             reinterpret_cast<float2*>(present_key),
                                                             reinterpret_cast<const float2*>(new_key),
                                                             past_seqlens_k,
                                                             total_seqlens_k,
                                                             is_past_kv_bnsh_format,
                                                             is_new_kv_bnsh_format);
    ConcatKVInPlaceLarge<float2><<<grid, block, 0, stream>>>(max_sequence_length,
                                                             H,
                                                             kv_num_heads,
                                                             reinterpret_cast<float2*>(present_value),
                                                             reinterpret_cast<const float2*>(new_value),
                                                             past_seqlens_k,
                                                             total_seqlens_k,
                                                             is_past_kv_bnsh_format,
                                                             is_new_kv_bnsh_format);
  }
  return CUDA_CALL(hipGetLastError());
}

// Concat new to kv buffer in place
template <typename T>
Status LaunchConcatKVInPlace(contrib::GroupQueryAttentionParameters& parameters,
                             GroupQueryAttentionData<T>& data,
                             const void* new_key,
                             const void* new_value,
                             bool is_new_kv_bnsh_format,
                             hipStream_t stream,
                             const int max_threads_per_block) {
  const int max_sequence_length = parameters.seqlen_present_kv_cache;
  const int* past_seqlens_k = parameters.is_prompt ? nullptr : reinterpret_cast<const int*>(data.seqlens_k);

  assert(parameters.past_kv_format == AttentionQkvFormat::Q_K_V_BSNH ||
         parameters.past_kv_format == AttentionQkvFormat::Q_K_V_BNSH);
  bool is_past_kv_bnsh_format = (parameters.past_kv_format == AttentionQkvFormat::Q_K_V_BNSH);

  return LaunchConcatKVInPlace(parameters.batch_size,
                               parameters.kv_num_heads,
                               parameters.head_size,
                               max_sequence_length,
                               past_seqlens_k,
                               nullptr,  // total_seqlens_k is not available
                               parameters.sequence_length,
                               reinterpret_cast<const T*>(new_key),
                               reinterpret_cast<const T*>(new_value),
                               data.present_key,
                               data.present_value,
                               is_past_kv_bnsh_format,
                               is_new_kv_bnsh_format,
                               stream,
                               max_threads_per_block);
}

// Kernel for use with memory efficient kernel... kv_in is grouped and of bnsh or bsnh... kv_out is ungrouped and bsnh
template <typename T>
__global__ void Ungroup(const T* kv_in,
                        T* kv_out,
                        const int in_seqlen,
                        const int kv_num_heads,
                        const bool is_bsnh) {
  const int h = threadIdx.x;
  const int out_n = threadIdx.y;
  const int s = blockIdx.x;
  const int b = blockIdx.y;

  const int out_seqlen = gridDim.x;
  const int q_num_heads = blockDim.y;
  const int H = blockDim.x;

  const int q_kv_head_ratio = q_num_heads / kv_num_heads;
  const int out_batch_stride = out_seqlen * q_num_heads * H;
  const int out_row_stride = is_bsnh ? q_num_heads * H : H;
  const int out_head_stride = is_bsnh ? H : out_seqlen * H;

  const int in_batch_stride = in_seqlen * kv_num_heads * H;
  const int in_row_stride = is_bsnh ? kv_num_heads * H : H;
  const int in_head_stride = is_bsnh ? H : in_seqlen * H;
  const int in_n = out_n / q_kv_head_ratio;

  const int out_offset = out_batch_stride * b + out_row_stride * s + out_head_stride * out_n + h;
  const int in_offset = in_batch_stride * b + in_row_stride * s + in_head_stride * in_n + h;
  kv_out[out_offset] = kv_in[in_offset];
}

template <typename T>
__global__ void UngroupLarge(const T* kv_in,
                             T* kv_out,
                             const int H,
                             const int in_seqlen,
                             const int q_num_heads,
                             const int kv_num_heads,
                             const bool is_bsnh) {
  int i = threadIdx.x + (blockDim.x * blockIdx.x);  // index along H * q_num_heads elements
  if (i < H * q_num_heads) {
    const int out_seqlen = gridDim.y;
    const int s = blockIdx.y;
    const int b = blockIdx.z;

    const int q_kv_head_ratio = q_num_heads / kv_num_heads;
    const int out_batch_stride = out_seqlen * q_num_heads * H;
    const int out_row_stride = is_bsnh ? q_num_heads * H : H;
    const int out_head_stride = is_bsnh ? H : out_seqlen * H;

    const int in_batch_stride = in_seqlen * kv_num_heads * H;
    const int in_row_stride = is_bsnh ? kv_num_heads * H : H;
    const int in_head_stride = is_bsnh ? H : in_seqlen * H;

    const int h = i % H;
    const int out_n = i / H;
    const int in_n = out_n / q_kv_head_ratio;
    const int out_offset = out_batch_stride * b + out_row_stride * s + out_head_stride * out_n + h;
    const int in_offset = in_batch_stride * b + in_row_stride * s + in_head_stride * in_n + h;
    kv_out[out_offset] = kv_in[in_offset];
  }
}

// Ungroup kv or present kv for use in Memory Efficient kernel. If present kv is not null and is BNSH, transposes it.
Status LaunchUngroup(contrib::GroupQueryAttentionParameters& parameters,
                     float2* k_buff, float2* v_buff,
                     const float2* k_og, const float2* v_og,
                     const int buff_seqlen, const int og_seqlen,
                     const bool is_bsnh,
                     hipStream_t stream,
                     const int max_threads_per_block) {
  const int batch_size = parameters.batch_size;
  const int num_heads = parameters.num_heads;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;

  const int H = head_size / 4;
  if (H * num_heads <= max_threads_per_block) {
    const dim3 grid(buff_seqlen, batch_size, 1);
    const dim3 block(H, num_heads, 1);
    Ungroup<float2><<<grid, block, 0, stream>>>(k_og,
                                                k_buff,
                                                og_seqlen,
                                                kv_num_heads,
                                                is_bsnh);
    Ungroup<float2><<<grid, block, 0, stream>>>(v_og,
                                                v_buff,
                                                og_seqlen,
                                                kv_num_heads,
                                                is_bsnh);
  } else {
    int steps = int(ceil(float(H * num_heads) / 256.0));
    const dim3 grid(steps, buff_seqlen, batch_size);
    const dim3 block(256, 1, 1);
    UngroupLarge<float2><<<grid, block, 0, stream>>>(k_og,
                                                     k_buff,
                                                     H,
                                                     og_seqlen,
                                                     num_heads,
                                                     kv_num_heads,
                                                     is_bsnh);
    UngroupLarge<float2><<<grid, block, 0, stream>>>(v_og,
                                                     v_buff,
                                                     H,
                                                     og_seqlen,
                                                     num_heads,
                                                     kv_num_heads,
                                                     is_bsnh);
  }
  return CUDA_CALL(hipGetLastError());
}

__global__ void PastToTotalSeqlen(int32_t* seqlens_k,
                                  int32_t* seqlens_k_buff,
                                  const int add_seqlen) {
  seqlens_k_buff[threadIdx.x] = seqlens_k[threadIdx.x] + add_seqlen;
}

// Convert Past to Total sequence length tensor
Status LaunchGetSeqlenBuff(contrib::GroupQueryAttentionParameters& parameters, int32_t* seqlens_k,
                           int32_t* seqlens_k_buff, bool is_total, hipStream_t stream,
                           const int /*threads_per_block*/) {
  if (parameters.is_prompt) {
    return Status::OK();
  }
  const int batch_size = parameters.batch_size;
  const int add_seqlen = is_total ? parameters.sequence_length : 0;

  const dim3 grid(1, 1, 1);
  // TODO(aciddelgado): unlikely but could have a bigger batch_size than max_threads
  const dim3 block(batch_size, 1, 1);

  // TODO(aciddelgado): small version
  PastToTotalSeqlen<<<grid, block, 0, stream>>>(seqlens_k, seqlens_k_buff, add_seqlen);

  return CUDA_CALL(hipGetLastError());
}

// Kernel to unpack qkv from packed qkv
template <typename T, bool output_bnsh>
__global__ void UnpackQKV(const T* packed_qkv, T* unpacked_q, T* unpacked_k, T* unpacked_v, const int num_heads,
                          const int kv_num_heads, const int head_size, const int sequence_length,
                          const int batch_size) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int d = (num_heads + 2 * kv_num_heads) * head_size;
  const int qkv_size = batch_size * sequence_length * d;
  const int q_hidden = num_heads * head_size;
  const int k_hidden = kv_num_heads * head_size;
  if (tid < qkv_size) {
    int b = tid / (d * sequence_length);
    int s = (tid % (d * sequence_length)) / d;
    int offset = tid % d;
    if (output_bnsh) {  // output BNSH
      int head_count = kv_num_heads;
      T* unpacked;
      if (offset < q_hidden) {
        unpacked = unpacked_q;
        head_count = num_heads;
      } else if (offset < q_hidden + k_hidden) {
        unpacked = unpacked_k;
        offset -= q_hidden;
      } else {
        unpacked = unpacked_v;
        offset -= (q_hidden + k_hidden);
      }
      int n = offset / head_size;
      int h = offset % head_size;

      int unpacked_i = INDEX_4D(head_count, sequence_length, head_size, b, n, s, h);
      unpacked[unpacked_i] = packed_qkv[tid];
    } else {  // output BSNH
      if (offset < q_hidden) {
        int unpacked_i = b * sequence_length * num_heads * head_size + s * num_heads * head_size + offset;
        unpacked_q[unpacked_i] = packed_qkv[tid];
      } else if (offset < q_hidden + k_hidden) {
        int unpacked_i = b * sequence_length * kv_num_heads * head_size +
                         s * kv_num_heads * head_size + (offset - q_hidden);
        unpacked_k[unpacked_i] = packed_qkv[tid];
      } else {
        int unpacked_i = b * sequence_length * kv_num_heads * head_size +
                         s * kv_num_heads * head_size + (offset - q_hidden - k_hidden);
        unpacked_v[unpacked_i] = packed_qkv[tid];
      }
    }
  }
}

// Unpack packed qkv
template <typename T, bool output_bnsh>
Status LaunchUnpackQKV(const T* packed_qkv, T* unpacked_q, T* unpacked_k, T* unpacked_v, const int num_heads,
                       const int kv_num_heads, const int head_size, const int sequence_length, const int batch_size,
                       hipStream_t stream, const int max_threads_per_block) {
  const int threads = max_threads_per_block;
  const int blocks = (batch_size * sequence_length * (num_heads + 2 * kv_num_heads) * head_size + threads - 1) / threads;
  UnpackQKV<T, output_bnsh><<<blocks, threads, 0, stream>>>(
      packed_qkv, unpacked_q, unpacked_k, unpacked_v, num_heads, kv_num_heads, head_size, sequence_length, batch_size);
  return CUDA_CALL(hipGetLastError());
}

// Kernel to convert seqlens_k to position_ids
__global__ void SeqlensToPosIdsPrompt(int32_t* seqlens_k, int64_t* position_ids, const int seqlen,
                                      const int batch_size) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int b = tid / seqlen;
  int s = tid % seqlen;
  if (b < batch_size) {
    if (s < seqlens_k[b] + 1) {
      position_ids[tid] = s;
    } else {
      position_ids[tid] = 1;
    }
  }
}

// Kernel to convert seqlens_k to position_ids
__global__ void SeqlensToPosIdsToken(int32_t* seqlens_k, int64_t* position_ids, const int batch_size) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < batch_size) {
    position_ids[tid] = seqlens_k[tid];
  }
}

// Convert seqlens_k to position_ids
Status LaunchSeqlensToPosIds(contrib::GroupQueryAttentionParameters& parameters, int32_t* seqlens_k,
                             int64_t* position_ids, hipStream_t stream, const int max_threads_per_block) {
  const int seqlen = parameters.sequence_length;
  const int batch_size = parameters.batch_size;
  const int threads = max_threads_per_block;
  const int blocks = (batch_size * seqlen + threads - 1) / threads;
  if (parameters.is_prompt) {
    SeqlensToPosIdsPrompt<<<blocks, threads, 0, stream>>>(seqlens_k, position_ids, seqlen, batch_size);
  } else {
    SeqlensToPosIdsToken<<<blocks, threads, 0, stream>>>(seqlens_k, position_ids, batch_size);
  }
  return CUDA_CALL(hipGetLastError());
}

////////// Launch Kernels

#if USE_FLASH_ATTENTION
template <typename T>
Status FlashAttention(
    const hipDeviceProp_t& device_prop,
    hipStream_t stream,
    contrib::GroupQueryAttentionParameters& parameters,
    GroupQueryAttentionData<T>& data,
    float scale) {
  const int max_threads_per_block = device_prop.maxThreadsPerBlock;
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const int kv_sequence_length = parameters.sequence_length;
  const int num_heads = parameters.num_heads;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;
  AttentionQkvFormat past_kv_format = parameters.past_kv_format;
  bool is_causal = parameters.is_unidirectional;
  bool is_bf16 = std::is_same<T, BFloat16>::value;

  void* query = reinterpret_cast<void*>(const_cast<T*>(data.query));
  void* key;
  void* value;

  if (!parameters.is_packed_qkv) {
    key = reinterpret_cast<void*>(const_cast<T*>(data.key));
    value = reinterpret_cast<void*>(const_cast<T*>(data.value));
  } else {
    const size_t key_offset = static_cast<size_t>(num_heads * head_size);
    const size_t value_offset = static_cast<size_t>(kv_num_heads * head_size);
    key = reinterpret_cast<T*>(query) + key_offset;
    value = reinterpret_cast<T*>(key) + value_offset;
  }

  void* seqlens_k = reinterpret_cast<void*>(data.seqlens_k);
  if (parameters.is_prompt) {
    // set seqlens_k to zeros... flash api uses seqlens_k to indicate where to append key and value
    // user should use seqlens_k to index into output to get new tokens
    if (batch_size <= parameters.zeros_count) {
      seqlens_k = parameters.zero_ptr;
    } else {
      // Launch kernel to create larger seqlen tensor when batch_size > 256
      constexpr int thr_per_blk = 256;
      int blk_in_grid = (batch_size + thr_per_blk - 1) / thr_per_blk;
      repeat_seqlen<<<blk_in_grid, thr_per_blk, 0, stream>>>(data.seqlens_k_total, 0, batch_size);
      seqlens_k = data.seqlens_k_total;
    }
  } else if (!parameters.kv_share_buffer) {  // copy past kv to present kv
    ORT_RETURN_IF_ERROR(LaunchConcatNewToPastKV(parameters, data, nullptr, nullptr, stream, max_threads_per_block,
                                                true));
  }

  void* present_key = reinterpret_cast<void*>(const_cast<T*>(data.present_key));
  void* present_value = reinterpret_cast<void*>(const_cast<T*>(data.present_value));
  void* cos_cache = reinterpret_cast<void*>(const_cast<T*>(data.cos_cache));
  void* sin_cache = reinterpret_cast<void*>(const_cast<T*>(data.sin_cache));

  bool past_bsnh = past_kv_format == AttentionQkvFormat::Q_K_V_BSNH;
  ORT_RETURN_IF_ERROR(onnxruntime::flash::mha_fwd_kvcache(
      device_prop, stream, query, present_key, present_value, key, value, data.output,
      reinterpret_cast<void*>(data.softmax_lse), seqlens_k, cos_cache, sin_cache,
      batch_size, num_heads, kv_num_heads, head_size, sequence_length,
      parameters.seqlen_present_kv_cache, kv_sequence_length, parameters.rotary_dim,
      scale, is_causal, is_bf16, past_bsnh, parameters.num_splits, reinterpret_cast<void*>(data.softmax_lse_accum),
      reinterpret_cast<void*>(data.out_accum), parameters.local_window_size, parameters.rotary_interleaved,
      parameters.is_packed_qkv));

  // if (parameters.left_padding && parameters.is_prompt) {
  //   ORT_RETURN_IF_ERROR(LaunchLeftPadLast(parameters, data, stream, device_prop.maxThreadsPerBlock));
  // }

  DUMP_TENSOR_INIT();
  DUMP_TENSOR("flash attention output", data.output, batch_size, sequence_length, num_heads, head_size);

  return Status::OK();
}
#endif

#if USE_MEMORY_EFFICIENT_ATTENTION
template <typename T>
Status EfficientAttention(
    const hipDeviceProp_t& device_prop,
    hipStream_t stream,
    contrib::GroupQueryAttentionParameters& parameters,
    GroupQueryAttentionData<T>& data,
    float scale) {
  const int max_threads_per_block = device_prop.maxThreadsPerBlock;
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const int present_sequence_length = parameters.seqlen_present_kv_cache;
  const int num_heads = parameters.num_heads;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;
  AttentionQkvFormat past_kv_format = parameters.past_kv_format;

  const void* query;
  const void* key;
  const void* value;

  if (!parameters.is_packed_qkv) {
    query = reinterpret_cast<const void*>(data.query);
    key = reinterpret_cast<const void*>(data.key);
    value = reinterpret_cast<const void*>(data.value);
  } else {
    size_t q_size = static_cast<size_t>(batch_size * sequence_length * num_heads * head_size);
    size_t k_size = static_cast<size_t>(batch_size * sequence_length * kv_num_heads * head_size);
    auto q = reinterpret_cast<T*>(data.unpacked_qkv_buffer);
    auto k = reinterpret_cast<T*>(data.unpacked_qkv_buffer + q_size);
    auto v = reinterpret_cast<T*>(data.unpacked_qkv_buffer + q_size + k_size);

    Status status = LaunchUnpackQKV<T, LAYOUT_BSNH>(
        reinterpret_cast<const T*>(data.query), q, k, v, num_heads, kv_num_heads,
        head_size, sequence_length, batch_size, stream, max_threads_per_block);
    if (status != Status::OK()) {
      return status;
    }

    query = reinterpret_cast<const void*>(q);
    key = reinterpret_cast<const void*>(k);
    value = reinterpret_cast<const void*>(v);
  }

  if (parameters.do_rotary) {
    size_t q_size = static_cast<size_t>(batch_size * sequence_length * num_heads * head_size);
    size_t k_size = static_cast<size_t>(batch_size * sequence_length * kv_num_heads * head_size);
    auto q_buffer = reinterpret_cast<T*>(data.rotary_buffer);
    auto k_buffer = q_buffer + q_size;
    auto position_ids_buff = reinterpret_cast<int64_t*>(k_buffer + k_size);
    ORT_RETURN_IF_ERROR(LaunchSeqlensToPosIds(parameters, data.seqlens_k, position_ids_buff, stream,
                                              max_threads_per_block));
    DUMP_TENSOR_INIT();
    DUMP_TENSOR("position_ids", position_ids_buff, batch_size, sequence_length);
    // Launch rotary embedding kernel
    ORT_RETURN_IF_ERROR(LaunchRotaryEmbeddingKernel<T>(stream, q_buffer, reinterpret_cast<const T*>(query),
                                                       position_ids_buff, data.cos_cache, data.sin_cache,
                                                       parameters.batch_size, parameters.sequence_length,
                                                       parameters.num_heads, parameters.head_size,
                                                       parameters.rotary_dim, parameters.seqlen_present_kv_cache,
                                                       /*position_ids_format*/ 1, parameters.rotary_interleaved,
                                                       device_prop.maxThreadsPerBlock, /*transposed*/ false));
    ORT_RETURN_IF_ERROR(LaunchRotaryEmbeddingKernel<T>(stream, k_buffer, reinterpret_cast<const T*>(key),
                                                       position_ids_buff, data.cos_cache, data.sin_cache,
                                                       parameters.batch_size, parameters.sequence_length,
                                                       parameters.kv_num_heads, parameters.head_size,
                                                       parameters.rotary_dim, parameters.seqlen_present_kv_cache,
                                                       /*position_ids_format*/ 1, parameters.rotary_interleaved,
                                                       device_prop.maxThreadsPerBlock, /*transposed*/ false));
    query = reinterpret_cast<const void*>(q_buffer);
    key = reinterpret_cast<const void*>(k_buffer);
  }

  if (parameters.is_prompt) {
    // Launch kernel to copy seqlen
    constexpr int thr_per_blk = 256;
    int blk_in_grid = (batch_size + thr_per_blk - 1) / thr_per_blk;
    repeat_seqlen<<<blk_in_grid, thr_per_blk, 0, stream>>>(data.seqlens_k_total, parameters.sequence_length,
                                                           batch_size);
  } else {
    ORT_RETURN_IF_ERROR(LaunchGetSeqlenBuff(parameters, data.seqlens_k, data.seqlens_k_total, true, stream, 256));
  }

  if (parameters.kv_share_buffer) {
    // Share buffer case
    if (data.past_key == nullptr || data.past_key != data.present_key) {
      return ORT_MAKE_STATUS(ONNXRUNTIME, INVALID_ARGUMENT,
                             "Past and present kv shall share the same tensor when kv_share_buffer is on.");
    }
    // Concatenate new kv in place
    constexpr bool is_new_kv_bnsh_format = false;
    ORT_RETURN_IF_ERROR(LaunchConcatKVInPlace(
        parameters, data, key, value, is_new_kv_bnsh_format, stream, max_threads_per_block));
  } else {
    // Not share buffer case
    if (data.past_key != nullptr && data.past_key == data.present_key) {
      return ORT_MAKE_STATUS(ONNXRUNTIME, INVALID_ARGUMENT,
                             "Past and present kv share the same tensor but kv_share_buffer is not on.");
    }
    // Copy past and concat new KV to present buffer
    ORT_RETURN_IF_ERROR(LaunchConcatNewToPastKV(parameters, data, key, value, stream, max_threads_per_block));
  }

  // Ungroup if grouped, otherwise use present kv directly
  const bool is_bsnh = past_kv_format == AttentionQkvFormat::Q_K_V_BSNH;
  if (num_heads == kv_num_heads) {
    // Use present kv directly if not grouped
    key = reinterpret_cast<const void*>(data.present_key);
    value = reinterpret_cast<const void*>(data.present_value);
  } else {
    // Otherwise we use intermediate buffers to run memory efficient attention... best avoid this path
    float2* k_buff = reinterpret_cast<float2*>(data.k);
    float2* v_buff = reinterpret_cast<float2*>(data.v);
    const float2* k_og = reinterpret_cast<const float2*>(data.present_key);
    const float2* v_og = reinterpret_cast<const float2*>(data.present_value);
    ORT_RETURN_IF_ERROR(LaunchUngroup(parameters, k_buff, v_buff, k_og, v_og, present_sequence_length,
                                      present_sequence_length, is_bsnh, stream, max_threads_per_block));
    key = reinterpret_cast<const void*>(data.k);
    value = reinterpret_cast<const void*>(data.v);
  }

  DUMP_TENSOR_INIT();
  DUMP_TENSOR("seqlens_k", data.seqlens_k_total, batch_size, 1);

  MemoryEfficientAttentionParams p;
  p.sm = device_prop.major * 10 + device_prop.minor;
  p.is_half = sizeof(T) == 2;
  p.batch_size = batch_size;
  p.num_heads = num_heads;
  p.sequence_length = sequence_length;
  p.kv_sequence_length = present_sequence_length;  // TOTALLY UNNECESSARY IF WE HAVE SEQLENS_K, maybe remove
  p.max_sequence_length = present_sequence_length;
  p.qk_head_size = head_size;
  p.v_head_size = head_size;
  p.causal = true;
  p.scale = scale;
  p.seqlen_k_ptr = data.seqlens_k_total;  // Note: seqlens_k is total sequence length for efficient
  p.seqstart_q_ptr = nullptr;
  p.seqstart_k_ptr = nullptr;
  p.query = query;
  p.key = key;
  p.value = value;
  p.attn_bias = nullptr;
  p.is_attn_bias_batched = false;
  p.is_kv_bsnh = past_kv_format == AttentionQkvFormat::Q_K_V_BSNH;
  p.output = data.output;
  p.workspace = MemoryEfficientAttentionParams::need_workspace(p.v_head_size, sizeof(T) == sizeof(float))
                    ? data.fmha_buffer
                    : nullptr;
  p.stream = stream;
  p.has_custom_right_padding = true;
  run_memory_efficient_attention(p);

  DUMP_TENSOR("efficient attention output", data.output, batch_size, sequence_length, num_heads, head_size);

  return Status::OK();
}
#endif

////////// API Functions

template <typename T>
Status QkvToContext(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& /*cublas*/,
    Stream* ort_stream,
    contrib::GroupQueryAttentionParameters& parameters,
    GroupQueryAttentionData<T>& data) {
  auto stream = static_cast<hipStream_t>(ort_stream->GetHandle());
  const float scale = parameters.scale == 0.0f ? 1.f / sqrt(static_cast<float>(parameters.head_size)) : parameters.scale;

#if USE_FLASH_ATTENTION
  if (data.use_flash_attention) {
    return FlashAttention(device_prop, stream, parameters, data, scale);
  }
#endif

#if USE_MEMORY_EFFICIENT_ATTENTION
  if (data.use_memory_efficient_attention) {
    return EfficientAttention(device_prop, stream, parameters, data, scale);
  }
#endif

  return ORT_MAKE_STATUS(ONNXRUNTIME, INVALID_ARGUMENT, "Unfused Group Query Attention not implemented yet.");
}

template struct GroupQueryAttentionData<half>;

template Status QkvToContext<half>(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    Stream* ort_stream,
    contrib::GroupQueryAttentionParameters& parameters,
    GroupQueryAttentionData<half>& data);

template struct GroupQueryAttentionData<BFloat16>;

template Status QkvToContext<BFloat16>(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    Stream* ort_stream,
    contrib::GroupQueryAttentionParameters& parameters,
    GroupQueryAttentionData<BFloat16>& data);

template Status LaunchUnpackQKV<half, LAYOUT_BNSH>(
    const half* packed_qkv, half* unpacked_q, half* unpacked_k, half* unpacked_v, const int num_heads,
    const int kv_num_heads, const int head_size, const int sequence_length, const int batch_size,
    hipStream_t stream, const int max_threads_per_block);

template Status LaunchUnpackQKV<BFloat16, LAYOUT_BNSH>(
    const BFloat16* packed_qkv, BFloat16* unpacked_q, BFloat16* unpacked_k, BFloat16* unpacked_v, const int num_heads,
    const int kv_num_heads, const int head_size, const int sequence_length, const int batch_size,
    hipStream_t stream, const int max_threads_per_block);

template Status LaunchConcatKVInPlace<half>(int batch_size,
                                            int kv_num_heads,
                                            int head_size,
                                            int max_sequence_length,
                                            const int* past_seqlens_k,
                                            const int* total_seqlens_k,
                                            int new_seq_len,
                                            const half* new_key,
                                            const half* new_value,
                                            half* present_key,
                                            half* present_value,
                                            bool is_past_kv_bnsh_format,
                                            bool is_new_kv_bnsh_format,
                                            hipStream_t stream,
                                            const int max_threads_per_block);

template Status LaunchConcatKVInPlace<BFloat16>(int batch_size,
                                                int kv_num_heads,
                                                int head_size,
                                                int max_sequence_length,
                                                const int* past_seqlens_k,
                                                const int* total_seqlens_k,
                                                int new_seq_len,
                                                const BFloat16* new_key,
                                                const BFloat16* new_value,
                                                BFloat16* present_key,
                                                BFloat16* present_value,
                                                bool is_past_kv_bnsh_format,
                                                bool is_new_kv_bnsh_format,
                                                hipStream_t stream,
                                                const int max_threads_per_block);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime

#undef OFFSET_BNSH
#undef OFFSET_BSNH
