#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on our Multi-Head Attention impl.cu file,
 which is based on qkvToContext plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/

Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications:
// (1) support GPT-2 past state, unidirectional mask (causal)
// (2) use flash attention kernel from (https://github.com/Dao-AILab/flash-attention)
// (3) support different number of heads for Q and KV
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <cassert>
#include <hip/hip_fp16.h>
#include <hipcub/hipcub.hpp>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/shared_inc/fpgeneric.h"
#include "contrib_ops/cuda/bert/attention_softmax.h"
#include "contrib_ops/cuda/bert/transformer_common.h"
#include "contrib_ops/cuda/bert/add_bias_transpose.h"
#include "contrib_ops/cpu/bert/attention_base.h"
#include "contrib_ops/cuda/bert/bert_padding.h"
#include "contrib_ops/cuda/transformers/dump_cuda_tensor.h"
#include "contrib_ops/cuda/bert/cutlass_fmha/memory_efficient_attention.h"
#include "contrib_ops/cuda/bert/flash_attention/flash_api.h"
#include "contrib_ops/cuda/bert/group_query_attention_impl.h"
#include "contrib_ops/cuda/bert/attention_impl.h"
#include "core/providers/cuda/shared_inc/cuda_call.h"
#include <hipblas.h>

using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

////////// Auxiliary Kernels for KV prep

// Kernel for seqlens_k
__global__ void repeat_seqlen(int32_t* seqlens_k, int32_t seqlen, int batch_size) {
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < batch_size) seqlens_k[id] = seqlen;
}

// Kernel to append new and past kv in either BSNH or BNSH format
// Adapted from ConcatTensorToTensor kernel in attention_kv_cache.cu file
template <typename T>
__global__ void ConcatNewToPastKV(const int new_seqlen,
                                  const int past_buffer_seqlen,
                                  const T* past_kv,
                                  const T* new_kv,
                                  T* present_kv,
                                  const int* seqlens_k,
                                  const bool is_bsnh) {  // refers to past; otherwise bnsh
  const int h = threadIdx.x;
  const int n = threadIdx.y;
  const int s = blockIdx.x;
  const int b = blockIdx.y;

  const int present_buffer_seqlen = gridDim.x;
  const int num_heads = blockDim.y;
  const int H = blockDim.x;

  const int present_batch_stride = present_buffer_seqlen * num_heads * H;
  const int row_stride = is_bsnh ? num_heads * H : H;
  const int present_head_stride = is_bsnh ? H : present_buffer_seqlen * H;

  // past_kv:     BPNH or BNPH
  // new_kv:      BLNH
  // present_kv:  BTNH or BNTH, where T = P + L
  const int past_seqlen = seqlens_k == nullptr ? 0 : seqlens_k[b];

  int out_offset = b * present_batch_stride + s * row_stride + n * present_head_stride + h;
  if (s < past_seqlen) {
    const int past_batch_stride = past_buffer_seqlen * num_heads * H;
    const int past_head_stride = is_bsnh ? H : past_buffer_seqlen * H;
    const int in_offset = b * past_batch_stride + s * row_stride + n * past_head_stride + h;
    present_kv[out_offset] = past_kv[in_offset];
  } else if (s < past_seqlen + new_seqlen) {
    // Note: new KV always BSNH
    const int new_batch_stride = new_seqlen * num_heads * H;
    const int new_row_stride = num_heads * H;
    const int new_head_stride = H;
    const int in_offset = b * new_batch_stride + (s - past_seqlen) * new_row_stride + n * new_head_stride + h;
    present_kv[out_offset] = new_kv[in_offset];
  }
}

// Use when (H*)*num_heads > 1024
template <typename T>
__global__ void ConcatNewToPastKVLarge(const int new_seqlen,
                                       const int past_buffer_seqlen,
                                       const int H,
                                       const int num_heads,
                                       const T* past_kv,
                                       const T* new_kv,
                                       T* present_kv,
                                       const int* seqlens_k,
                                       const bool is_bsnh) {
  int i = threadIdx.x + (blockDim.x * blockIdx.x);
  if (i < H * num_heads) {
    const int h = i % H;
    const int n = i / H;
    const int s = blockIdx.y;
    const int b = blockIdx.z;
    const int present_buffer_seqlen = gridDim.y;

    const int present_batch_stride = present_buffer_seqlen * num_heads * H;
    const int row_stride = is_bsnh ? num_heads * H : H;
    const int present_head_stride = is_bsnh ? H : present_buffer_seqlen * H;

    // past_kv:     BPNH or BNPH
    // new_kv:      BLNH
    // present_kv:  BTNH or BNTH, where T = P + L
    const int past_seqlen = seqlens_k == nullptr ? 0 : seqlens_k[b];

    int out_offset = b * present_batch_stride + s * row_stride + n * present_head_stride + h;
    if (s < past_seqlen) {
      const int past_batch_stride = past_buffer_seqlen * num_heads * H;
      const int past_head_stride = is_bsnh ? H : past_buffer_seqlen * H;
      const int in_offset = b * past_batch_stride + s * row_stride + n * past_head_stride + h;
      present_kv[out_offset] = past_kv[in_offset];
    } else if (s < past_seqlen + new_seqlen) {
      const int new_batch_stride = new_seqlen * num_heads * H;
      const int new_row_stride = num_heads * H;
      const int new_head_stride = H;
      const int in_offset = b * new_batch_stride + (s - past_seqlen) * new_row_stride + n * new_head_stride + h;
      present_kv[out_offset] = new_kv[in_offset];
    }
  }
}

// Concat new to past in present. Supports past BSNH or past BNSH
template <typename T>
Status LaunchConcatNewToPastKV(contrib::GroupQueryAttentionParameters& parameters,
                               GroupQueryAttentionData<T>& data,
                               hipStream_t stream,
                               const int max_threads_per_block) {
  const int batch_size = parameters.batch_size;
  const int kv_sequence_length = parameters.sequence_length;
  const int past_sequence_length = parameters.seqlen_past_kv_cache;
  const int present_sequence_length = parameters.seqlen_present_kv_cache;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;
  const int* seqlens_k = parameters.is_prompt ? nullptr : reinterpret_cast<const int*>(data.seqlens_k);

  AttentionQkvFormat past_kv_format = parameters.past_kv_format;

  assert(past_kv_format == AttentionQkvFormat::Q_K_V_BSNH || past_kv_format == AttentionQkvFormat::Q_K_V_BNSH);
  const int H = head_size / 4;  // divide by 4 so kernel can operate on 4 float16 elements at a time.
  if (H * kv_num_heads <= max_threads_per_block) {
    const dim3 grid(present_sequence_length, batch_size, 1);
    const dim3 block(H, kv_num_heads, 1);
    ConcatNewToPastKV<float2><<<grid, block, 0, stream>>>(kv_sequence_length,
                                                          past_sequence_length,
                                                          reinterpret_cast<const float2*>(data.past_key),
                                                          reinterpret_cast<const float2*>(data.key),
                                                          reinterpret_cast<float2*>(data.present_key),
                                                          seqlens_k,
                                                          past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
    ConcatNewToPastKV<float2><<<grid, block, 0, stream>>>(kv_sequence_length,
                                                          past_sequence_length,
                                                          reinterpret_cast<const float2*>(data.past_value),
                                                          reinterpret_cast<const float2*>(data.value),
                                                          reinterpret_cast<float2*>(data.present_value),
                                                          seqlens_k,
                                                          past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
  } else {
    int steps = (H * kv_num_heads + 255) / 256;
    const dim3 grid(steps, present_sequence_length, batch_size);
    const dim3 block(256, 1, 1);
    ConcatNewToPastKVLarge<float2><<<grid, block, 0, stream>>>(kv_sequence_length,
                                                               past_sequence_length,
                                                               H,
                                                               kv_num_heads,
                                                               reinterpret_cast<const float2*>(data.past_key),
                                                               reinterpret_cast<const float2*>(data.key),
                                                               reinterpret_cast<float2*>(data.present_key),
                                                               seqlens_k,
                                                               past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
    ConcatNewToPastKVLarge<float2><<<grid, block, 0, stream>>>(kv_sequence_length,
                                                               past_sequence_length,
                                                               H,
                                                               kv_num_heads,
                                                               reinterpret_cast<const float2*>(data.past_value),
                                                               reinterpret_cast<const float2*>(data.value),
                                                               reinterpret_cast<float2*>(data.present_value),
                                                               seqlens_k,
                                                               past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
  }
  return CUDA_CALL(hipGetLastError());
}

// Kernel to append new kv to kv buffer in place
template <typename T>
__global__ void ConcatKVInPlace(const int max_seqlen,
                                T* kv_buff,
                                const T* new_kv,
                                const int* seqlens_k,
                                const bool is_bsnh) {  // refers to kv buff; otherwise bnsh
  const int h = threadIdx.x;
  const int n = threadIdx.y;
  const int s = blockIdx.x;
  const int b = blockIdx.y;

  const int new_seqlen = gridDim.x;
  const int num_heads = blockDim.y;
  const int H = blockDim.x;

  const int present_batch_stride = max_seqlen * num_heads * H;
  const int present_row_stride = is_bsnh ? num_heads * H : H;
  const int present_head_stride = is_bsnh ? H : max_seqlen * H;

  // kv_buff:     BTNH or BNTH with buffered memory for new
  // new_kv:      BLNH

  const int past_seq_len = seqlens_k == nullptr ? 0 : seqlens_k[b];

  int out_offset = b * present_batch_stride + (s + past_seq_len) * present_row_stride + n * present_head_stride + h;
  // Note: new KV always BSNH
  const int new_batch_stride = new_seqlen * num_heads * H;
  const int new_row_stride = num_heads * H;
  const int new_head_stride = H;
  const int in_offset = b * new_batch_stride + s * new_row_stride + n * new_head_stride + h;
  kv_buff[out_offset] = new_kv[in_offset];
}

template <typename T>
__global__ void ConcatKVInPlaceLarge(const int max_seqlen,
                                     const int H,
                                     const int num_heads,
                                     T* kv_buff,
                                     const T* new_kv,
                                     const int* seqlens_k,
                                     const bool is_bsnh) {  // refers to kv buff; otherwise bnsh
  int i = threadIdx.x + (blockDim.x * blockIdx.x);
  if (i < H * num_heads) {
    const int h = i % H;
    const int n = i / H;
    const int s = blockIdx.y;
    const int b = blockIdx.z;
    const int new_seqlen = gridDim.y;

    const int present_batch_stride = max_seqlen * num_heads * H;
    const int present_row_stride = is_bsnh ? num_heads * H : H;
    const int present_head_stride = is_bsnh ? H : max_seqlen * H;

    // kv_buff:     BTNH or BNTH with buffered memory for new
    // new_kv:      BLNH

    const int past_seq_len = seqlens_k == nullptr ? 0 : seqlens_k[b];

    int out_offset = b * present_batch_stride + (s + past_seq_len) * present_row_stride + n * present_head_stride + h;
    // Note: new KV always BSNH
    const int new_batch_stride = new_seqlen * num_heads * H;
    const int new_row_stride = num_heads * H;
    const int new_head_stride = H;
    const int in_offset = b * new_batch_stride + s * new_row_stride + n * new_head_stride + h;
    kv_buff[out_offset] = new_kv[in_offset];
  }
}

// Concat new to kv buffer in place
template <typename T>
Status LaunchConcatKVInPlace(contrib::GroupQueryAttentionParameters& parameters,
                             GroupQueryAttentionData<T>& data,
                             hipStream_t stream,
                             const int max_threads_per_block) {
  const int batch_size = parameters.batch_size;
  const int kv_sequence_length = parameters.sequence_length;
  const int present_sequence_length = parameters.seqlen_present_kv_cache;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;

  // Indicates past sequence_length of each sequence
  const int* seqlens_k = parameters.is_prompt ? nullptr : reinterpret_cast<const int*>(data.seqlens_k);

  AttentionQkvFormat past_kv_format = parameters.past_kv_format;
  assert(past_kv_format == AttentionQkvFormat::Q_K_V_BSNH || past_kv_format == AttentionQkvFormat::Q_K_V_BNSH);
  const int H = head_size / 4;
  if (H * kv_num_heads <= max_threads_per_block) {
    const dim3 grid(kv_sequence_length, batch_size, 1);
    const dim3 block(H, kv_num_heads, 1);
    ConcatKVInPlace<float2><<<grid, block, 0, stream>>>(present_sequence_length,
                                                        reinterpret_cast<float2*>(data.present_key),
                                                        reinterpret_cast<const float2*>(data.key),
                                                        seqlens_k,
                                                        past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
    ConcatKVInPlace<float2><<<grid, block, 0, stream>>>(present_sequence_length,
                                                        reinterpret_cast<float2*>(data.present_value),
                                                        reinterpret_cast<const float2*>(data.value),
                                                        seqlens_k,
                                                        past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
  } else {
    int steps = int(ceil(float(H * kv_num_heads) / 256.0));
    const dim3 grid(steps, kv_sequence_length, batch_size);
    const dim3 block(256, 1, 1);
    ConcatKVInPlaceLarge<float2><<<grid, block, 0, stream>>>(present_sequence_length,
                                                             H,
                                                             kv_num_heads,
                                                             reinterpret_cast<float2*>(data.present_key),
                                                             reinterpret_cast<const float2*>(data.key),
                                                             seqlens_k,
                                                             past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
    ConcatKVInPlaceLarge<float2><<<grid, block, 0, stream>>>(present_sequence_length,
                                                             H,
                                                             kv_num_heads,
                                                             reinterpret_cast<float2*>(data.present_value),
                                                             reinterpret_cast<const float2*>(data.value),
                                                             seqlens_k,
                                                             past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
  }
  return CUDA_CALL(hipGetLastError());
}

// Kernel for use with memory efficient kernel... kv_in is grouped and of bnsh or bsnh... kv_out is ungrouped and bsnh
template <typename T>
__global__ void Ungroup(const T* kv_in,
                        T* kv_out,
                        const int in_seqlen,
                        const int kv_num_heads,
                        const bool is_bsnh) {
  const int h = threadIdx.x;
  const int out_n = threadIdx.y;
  const int s = blockIdx.x;
  const int b = blockIdx.y;

  const int out_seqlen = gridDim.x;
  const int q_num_heads = blockDim.y;
  const int H = blockDim.x;

  const int q_kv_head_ratio = q_num_heads / kv_num_heads;
  const int out_batch_stride = out_seqlen * q_num_heads * H;
  const int out_row_stride = is_bsnh ? q_num_heads * H : H;
  const int out_head_stride = is_bsnh ? H : out_seqlen * H;

  const int in_batch_stride = in_seqlen * kv_num_heads * H;
  const int in_row_stride = is_bsnh ? kv_num_heads * H : H;
  const int in_head_stride = is_bsnh ? H : in_seqlen * H;
  const int in_n = out_n / q_kv_head_ratio;

  const int out_offset = out_batch_stride * b + out_row_stride * s + out_head_stride * out_n + h;
  const int in_offset = in_batch_stride * b + in_row_stride * s + in_head_stride * in_n + h;
  kv_out[out_offset] = kv_in[in_offset];
}

template <typename T>
__global__ void UngroupLarge(const T* kv_in,
                             T* kv_out,
                             const int H,
                             const int in_seqlen,
                             const int q_num_heads,
                             const int kv_num_heads,
                             const bool is_bsnh) {
  int i = threadIdx.x + (blockDim.x * blockIdx.x);  // index along H * q_num_heads elements
  if (i < H * q_num_heads) {
    const int out_seqlen = gridDim.y;
    const int s = blockIdx.y;
    const int b = blockIdx.z;

    const int q_kv_head_ratio = q_num_heads / kv_num_heads;
    const int out_batch_stride = out_seqlen * q_num_heads * H;
    const int out_row_stride = is_bsnh ? q_num_heads * H : H;
    const int out_head_stride = is_bsnh ? H : out_seqlen * H;

    const int in_batch_stride = in_seqlen * kv_num_heads * H;
    const int in_row_stride = is_bsnh ? kv_num_heads * H : H;
    const int in_head_stride = is_bsnh ? H : in_seqlen * H;

    const int h = i % H;
    const int out_n = i / H;
    const int in_n = out_n / q_kv_head_ratio;
    const int out_offset = out_batch_stride * b + out_row_stride * s + out_head_stride * out_n + h;
    const int in_offset = in_batch_stride * b + in_row_stride * s + in_head_stride * in_n + h;
    kv_out[out_offset] = kv_in[in_offset];
  }
}

// Ungroup kv or present kv for use in Memory Efficient kernel. If present kv is not null and is BNSH, transposes it.
Status LaunchUngroup(contrib::GroupQueryAttentionParameters& parameters,
                     float2* k_buff, float2* v_buff,
                     const float2* k_og, const float2* v_og,
                     const int buff_seqlen, const int og_seqlen,
                     const bool is_bsnh,
                     hipStream_t stream,
                     const int max_threads_per_block) {
  const int batch_size = parameters.batch_size;
  const int num_heads = parameters.num_heads;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;

  const int H = head_size / 4;
  if (H * num_heads <= max_threads_per_block) {
    const dim3 grid(buff_seqlen, batch_size, 1);
    const dim3 block(H, num_heads, 1);
    Ungroup<float2><<<grid, block, 0, stream>>>(k_og,
                                                k_buff,
                                                og_seqlen,
                                                kv_num_heads,
                                                is_bsnh);
    Ungroup<float2><<<grid, block, 0, stream>>>(v_og,
                                                v_buff,
                                                og_seqlen,
                                                kv_num_heads,
                                                is_bsnh);
  } else {
    int steps = int(ceil(float(H * num_heads) / 256.0));
    const dim3 grid(steps, buff_seqlen, batch_size);
    const dim3 block(256, 1, 1);
    UngroupLarge<float2><<<grid, block, 0, stream>>>(k_og,
                                                     k_buff,
                                                     H,
                                                     og_seqlen,
                                                     num_heads,
                                                     kv_num_heads,
                                                     is_bsnh);
    UngroupLarge<float2><<<grid, block, 0, stream>>>(v_og,
                                                     v_buff,
                                                     H,
                                                     og_seqlen,
                                                     num_heads,
                                                     kv_num_heads,
                                                     is_bsnh);
  }
  return CUDA_CALL(hipGetLastError());
}


__global__ void PastToTotalSeqlen(int32_t* seqlens_k,
                                  int32_t* seqlens_k_buff,
                                  const int add_seqlen) {
  seqlens_k_buff[threadIdx.x] = seqlens_k[threadIdx.x] + add_seqlen;
}

// Convert Past to Total sequence length tensor
Status LaunchGetSeqlenBuff(contrib::GroupQueryAttentionParameters& parameters, int32_t* seqlens_k,
                           int32_t* seqlens_k_buff, bool is_total, hipStream_t stream,
                               const int threads_per_block) {
  if (parameters.is_prompt) {
    return Status::OK();
  }
  const int batch_size = parameters.batch_size;
  const int add_seqlen = is_total ? parameters.sequence_length : 0;

  const dim3 grid(1, 1, 1);
  // TODO(aciddelgado): unlikely but could have a bigger batch_size than max_threads
  const dim3 block(batch_size, 1, 1);

  // TODO(aciddelgado): small version
  PastToTotalSeqlen<<<grid, block, 0, stream>>>(seqlens_k, seqlens_k_buff, add_seqlen);

  return CUDA_CALL(hipGetLastError());
}

// // Kernel to append new kv to kv buffer in place
// template <typename T>
// __global__ void LeftPadLast(const int max_seqlen,
//                             T* kv_buff,
//                             const int* seqlens_k) {  // refers to kv buff; otherwise bnsh
//   const int h = threadIdx.x;
//   const int n = blockIdx.x;
//   const int b = blockIdx.y;

//   const int num_heads = gridDim.x;
//   const int H = blockDim.x;

//   const int present_batch_stride = max_seqlen * num_heads * H;
//   const int present_row_stride = num_heads * H;
//   const int present_head_stride = H;

//   // kv_buff:     BTNH or BNTH with buffered memory for new
//   // new_kv:      BLNH

//   const int s = seqlens_k[b];

//   const int in_offset = b * present_batch_stride + s * present_row_stride + n * present_head_stride + h;
//   const int out_offset = b * present_batch_stride + (max_seqlen - 1) * present_row_stride + n * present_head_stride + h;
//   kv_buff[out_offset] = kv_buff[in_offset];
// }

// // Concat new to kv buffer in place
// template <typename T>
// Status LaunchLeftPadLast(contrib::GroupQueryAttentionParameters& parameters,
//                              GroupQueryAttentionData<T>& data,
//                              hipStream_t stream,
//                              const int max_threads_per_block) {
//   const int batch_size = parameters.batch_size;
//   const int sequence_length = parameters.sequence_length;
//   const int num_heads = parameters.num_heads;
//   const int head_size = parameters.head_size;

//   // Indicates past sequence_length of each sequence
//   const int* seqlens_k = reinterpret_cast<const int*>(data.seqlens_k);

//   const int H = head_size / 4;
//   const dim3 grid(num_heads, batch_size, 1);
//   const dim3 block(H, 1, 1);
//   LeftPadLast<float2><<<grid, block, 0, stream>>>(sequence_length,
//                                                   reinterpret_cast<float2*>(data.output),
//                                                   seqlens_k);
//   return CUDA_CALL(hipGetLastError());
// }

////////// Launch Kernels

#if USE_FLASH_ATTENTION
template <typename T>
Status FlashAttention(
    const hipDeviceProp_t& device_prop,
    hipStream_t stream,
    contrib::GroupQueryAttentionParameters& parameters,
    GroupQueryAttentionData<T>& data,
    float scale) {
  const int max_threads_per_block = device_prop.maxThreadsPerBlock;
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const int kv_sequence_length = parameters.sequence_length;
  const int present_sequence_length = parameters.seqlen_present_kv_cache;
  const int num_heads = parameters.num_heads;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;
  AttentionQkvFormat past_kv_format = parameters.past_kv_format;

  void* query = reinterpret_cast<void*>(const_cast<T*>(data.query));
  void* key = reinterpret_cast<void*>(const_cast<T*>(data.key));
  void* value = reinterpret_cast<void*>(const_cast<T*>(data.value));

  bool is_causal = parameters.is_unidirectional;

  // Note: seqlens_k is past sequence length for flash
  if (parameters.is_prompt) {
    // Launch kernel to copy seqlen
    constexpr int thr_per_blk = 256;
    int blk_in_grid = (batch_size + thr_per_blk -1) / thr_per_blk;
    repeat_seqlen<<<blk_in_grid, thr_per_blk, 0, stream>>>(data.seqlens_k_total, parameters.sequence_length, batch_size);
  }

  void* seqlens_k = reinterpret_cast<void*>(data.seqlens_k);

  if (parameters.kv_share_buffer) {
    // Share buffer case
    if (data.past_key == nullptr || data.past_key != data.present_key) {
      return ORT_MAKE_STATUS(ONNXRUNTIME, INVALID_ARGUMENT,
                             "Past and present kv shall share the same tensor when kv_share_buffer is on.");
    }

    if (parameters.is_prompt) {
      ORT_RETURN_IF_ERROR(LaunchConcatKVInPlace(parameters, data, stream, max_threads_per_block));
      key = nullptr;
      value = nullptr;
      seqlens_k = reinterpret_cast<void*>(data.seqlens_k_total);
    }

    void* present_key = reinterpret_cast<void*>(const_cast<T*>(data.present_key));
    void* present_value = reinterpret_cast<void*>(const_cast<T*>(data.present_value));

    DUMP_TENSOR_INIT();
    DUMP_TENSOR("seqlens_k", reinterpret_cast<int*>(seqlens_k), batch_size, 1);

    bool past_bsnh = past_kv_format == AttentionQkvFormat::Q_K_V_BSNH;
    ORT_RETURN_IF_ERROR(onnxruntime::flash::mha_fwd_kvcache(
        device_prop, stream, query, present_key, present_value, key, value, data.output, reinterpret_cast<void*>(data.softmax_lse),
        seqlens_k, batch_size, num_heads, kv_num_heads,
        head_size, sequence_length, present_sequence_length, kv_sequence_length,
        scale, is_causal, past_bsnh, parameters.num_splits, reinterpret_cast<void*>(data.softmax_lse_accum),
        reinterpret_cast<void*>(data.out_accum)));
  } else {
    // Not share buffer case
    // Note that Flash Attention kv-caching operates in place on a buffer... therefore this path is inneficient
    if (data.past_key != nullptr && data.past_key == data.present_key) {
      return ORT_MAKE_STATUS(ONNXRUNTIME, INVALID_ARGUMENT,
                             "Past and present kv share the same tensor but kv_share_buffer is not on.");
    }

    ORT_RETURN_IF_ERROR(LaunchConcatNewToPastKV(parameters, data, stream, max_threads_per_block));

    if (!parameters.is_prompt) {
      ORT_RETURN_IF_ERROR(LaunchGetSeqlenBuff(parameters, data.seqlens_k, data.seqlens_k_total, true, stream, 256));
    }

    seqlens_k = reinterpret_cast<void*>(data.seqlens_k_total);

    void* present_key = reinterpret_cast<void*>(const_cast<T*>(data.present_key));
    void* present_value = reinterpret_cast<void*>(const_cast<T*>(data.present_value));

    DUMP_TENSOR_INIT();
    DUMP_TENSOR("seqlens_k", reinterpret_cast<int*>(seqlens_k), batch_size, 1);
    DUMP_TENSOR("Q", data.query, batch_size, sequence_length, num_heads, head_size);
    DUMP_TENSOR("K", data.present_key, batch_size, kv_num_heads, present_sequence_length, head_size);
    DUMP_TENSOR("V", data.present_value, batch_size, kv_num_heads, present_sequence_length, head_size);

    bool past_bsnh = past_kv_format == AttentionQkvFormat::Q_K_V_BSNH;
    ORT_RETURN_IF_ERROR(onnxruntime::flash::mha_fwd_kvcache(
        device_prop, stream, query, present_key, present_value, nullptr, nullptr, data.output, reinterpret_cast<void*>(data.softmax_lse),
        seqlens_k, batch_size, num_heads, kv_num_heads,
        head_size, sequence_length, present_sequence_length, 0,
        scale, is_causal, past_bsnh, parameters.num_splits, reinterpret_cast<void*>(data.softmax_lse_accum),
        reinterpret_cast<void*>(data.out_accum)));
  }

  // if (parameters.left_padding && parameters.is_prompt) {
  //   ORT_RETURN_IF_ERROR(LaunchLeftPadLast(parameters, data, stream, device_prop.maxThreadsPerBlock));
  // }

  DUMP_TENSOR_INIT();
  DUMP_TENSOR("flash attention output", data.output, batch_size, sequence_length, num_heads, head_size);

  return Status::OK();
}
#endif

#if USE_MEMORY_EFFICIENT_ATTENTION
template <typename T>
Status EfficientAttention(
    const hipDeviceProp_t& device_prop,
    hipStream_t stream,
    contrib::GroupQueryAttentionParameters& parameters,
    GroupQueryAttentionData<T>& data,
    float scale) {
  const int max_threads_per_block = device_prop.maxThreadsPerBlock;
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const int present_sequence_length = parameters.seqlen_present_kv_cache;
  const int num_heads = parameters.num_heads;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;
  AttentionQkvFormat past_kv_format = parameters.past_kv_format;

  const void* query = reinterpret_cast<const void*>(data.query);
  const void* key = reinterpret_cast<const void*>(data.key);
  const void* value = reinterpret_cast<const void*>(data.value);

  if (parameters.is_prompt) {
    // Launch kernel to copy seqlen
    constexpr int thr_per_blk = 256;
    int blk_in_grid = (batch_size + thr_per_blk - 1) / thr_per_blk;
    repeat_seqlen<<<blk_in_grid, thr_per_blk, 0, stream>>>(data.seqlens_k_total, parameters.sequence_length, batch_size);
  } else {
    ORT_RETURN_IF_ERROR(LaunchGetSeqlenBuff(parameters, data.seqlens_k, data.seqlens_k_total, true, stream, 256));
  }

  if (parameters.kv_share_buffer) {
    // Share buffer case
    if (data.past_key == nullptr || data.past_key != data.present_key) {
      return ORT_MAKE_STATUS(ONNXRUNTIME, INVALID_ARGUMENT,
                             "Past and present kv shall share the same tensor when kv_share_buffer is on.");
    }
    // Concatenate new kv in place
    ORT_RETURN_IF_ERROR(LaunchConcatKVInPlace(parameters, data, stream, max_threads_per_block));
  } else {
    // Not share buffer case
    if (data.past_key != nullptr && data.past_key == data.present_key) {
      return ORT_MAKE_STATUS(ONNXRUNTIME, INVALID_ARGUMENT,
                             "Past and present kv share the same tensor but kv_share_buffer is not on.");
    }
    // Copy past and concat new KV to present buffer
    ORT_RETURN_IF_ERROR(LaunchConcatNewToPastKV(parameters, data, stream, max_threads_per_block));
  }

  // Ungroup if grouped, otherwise use present kv directly
  const bool is_bsnh = past_kv_format == AttentionQkvFormat::Q_K_V_BSNH;
  if (num_heads == kv_num_heads) {
    // Use present kv directly if not grouped
    key = reinterpret_cast<const void*>(data.present_key);
    value = reinterpret_cast<const void*>(data.present_value);
  } else {
    // Otherwise we use intermediate buffers to run memory efficient attention... best avoid this path
    float2* k_buff = reinterpret_cast<float2*>(data.k);
    float2* v_buff = reinterpret_cast<float2*>(data.v);
    const float2* k_og = reinterpret_cast<const float2*>(data.present_key);
    const float2* v_og = reinterpret_cast<const float2*>(data.present_value);
    ORT_RETURN_IF_ERROR(LaunchUngroup(parameters, k_buff, v_buff, k_og, v_og, present_sequence_length,
                                      present_sequence_length, is_bsnh, stream, max_threads_per_block));
    key = reinterpret_cast<const void*>(data.k);
    value = reinterpret_cast<const void*>(data.v);
  }

  DUMP_TENSOR_INIT();
  DUMP_TENSOR("seqlens_k", data.seqlens_k_total, batch_size, 1);

  MemoryEfficientAttentionParams p;
  p.sm = device_prop.major * 10 + device_prop.minor;
  p.is_half = sizeof(T) == 2;
  p.batch_size = batch_size;
  p.num_heads = num_heads;
  p.sequence_length = sequence_length;
  p.kv_sequence_length = present_sequence_length;  // TOTALLY UNNECESSARY IF WE HAVE SEQLENS_K, maybe remove
  p.max_sequence_length = present_sequence_length;
  p.qk_head_size = head_size;
  p.v_head_size = head_size;
  p.causal = parameters.is_unidirectional;
  p.scale = scale;
  p.seqlen_k_ptr = data.seqlens_k_total;  // Note: seqlens_k is total sequence length for efficient
  p.query = query;
  p.key = key;
  p.value = value;
  p.attn_bias = nullptr;
  p.is_attn_bias_batched = false;
  p.is_kv_bsnh = past_kv_format == AttentionQkvFormat::Q_K_V_BSNH;
  p.output = data.output;
  p.workspace = MemoryEfficientAttentionParams::need_workspace(p.v_head_size, sizeof(T) == sizeof(float))
                    ? data.fmha_buffer
                    : nullptr;
  p.stream = stream;
  p.has_custom_right_padding = true;
  run_memory_efficient_attention(p);

  // if (parameters.left_padding && parameters.is_prompt) {
  //   ORT_RETURN_IF_ERROR(LaunchLeftPadLast(parameters, data, stream, device_prop.maxThreadsPerBlock));
  // }

  DUMP_TENSOR_INIT();
  DUMP_TENSOR("efficient attention output", data.output, batch_size, sequence_length, num_heads, head_size);

  return Status::OK();
}
#endif

////////// API Functions

template <typename T>
Status QkvToContext(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    Stream* ort_stream,
    contrib::GroupQueryAttentionParameters& parameters,
    GroupQueryAttentionData<T>& data) {
  auto stream = static_cast<hipStream_t>(ort_stream->GetHandle());
  const float scale = parameters.scale == 0.0f ? 1.f / sqrt(static_cast<float>(parameters.head_size)) : parameters.scale;

#if USE_FLASH_ATTENTION
  if (data.use_flash_attention) {
    return FlashAttention(device_prop, stream, parameters, data, scale);
  }
#endif

#if USE_MEMORY_EFFICIENT_ATTENTION
  if (data.use_memory_efficient_attention) {
    return EfficientAttention(device_prop, stream, parameters, data, scale);
  }
#endif

  return ORT_MAKE_STATUS(ONNXRUNTIME, INVALID_ARGUMENT, "Unfused Group Query Attention not implemented yet.");
}

template struct GroupQueryAttentionData<half>;

template Status QkvToContext<half>(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    Stream* ort_stream,
    contrib::GroupQueryAttentionParameters& parameters,
    GroupQueryAttentionData<half>& data);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
