#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on our Multi-Head Attention impl.cu file,
 which is based on qkvToContext plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/

Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications:
// (1) support GPT-2 past state, unidirectional mask (causal)
// (2) use flash attention kernel from (https://github.com/Dao-AILab/flash-attention)
// (3) support different number of heads for Q and KV
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <cassert>
#include <hip/hip_fp16.h>
#include <hipcub/hipcub.hpp>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/shared_inc/fpgeneric.h"
#include "contrib_ops/cuda/bert/attention_softmax.h"
#include "contrib_ops/cuda/bert/transformer_common.h"
#include "contrib_ops/cuda/bert/add_bias_transpose.h"
#include "contrib_ops/cpu/bert/attention_base.h"
#include "contrib_ops/cuda/bert/bert_padding.h"
#include "contrib_ops/cuda/transformers/dump_cuda_tensor.h"
#include "contrib_ops/cuda/bert/cutlass_fmha/memory_efficient_attention.h"
#include "contrib_ops/cuda/bert/flash_attention/flash_api.h"
#include "contrib_ops/cuda/bert/group_query_attention_impl.h"
#include "contrib_ops/cuda/bert/attention_impl.h"

using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

////////// Auxiliary Kernels for KV prep

// Kernel for seqlens_k
__global__ void repeat_seqlen(int32_t* seqlens_k, int32_t seqlen, int batch_size) {
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < batch_size) seqlens_k[id] = seqlen;
}

// Kernel to append new and past kv in either BSNH or BNSH format
// Adapted from ConcatTensorToTensor kernel in attention_kv_cache.cu file
template <typename T>
__global__ void ConcatNewToPastKV(const int new_seqlen,
                                  const T* past_kv,
                                  const T* new_kv,
                                  T* present_kv,
                                  const bool is_bsnh) {  // refers to past; otherwise bnsh
  const int h = threadIdx.x;
  const int n = threadIdx.y;
  const int s = blockIdx.x;
  const int b = blockIdx.y;

  const int present_seqlen = gridDim.x;
  const int num_heads = blockDim.y;
  const int H = blockDim.x;

  const int present_batch_stride = present_seqlen * num_heads * H;
  const int row_stride = is_bsnh ? num_heads * H : H;
  const int present_head_stride = is_bsnh ? H : present_seqlen * H;

  // past_kv:     BPNH or BNPH
  // new_kv:      BLNH
  // present_kv:  BTNH or BNTH, where T = P + L
  const int past_seqlen = present_seqlen - new_seqlen;

  int out_offset = b * present_batch_stride + s * row_stride + n * present_head_stride + h;
  if (s < past_seqlen) {
    const int past_batch_stride = past_seqlen * num_heads * H;
    const int past_head_stride = is_bsnh ? H : past_seqlen * H;
    const int in_offset = b * past_batch_stride + s * row_stride + n * past_head_stride + h;
    present_kv[out_offset] = past_kv[in_offset];
  } else if (s < present_seqlen) {
    // Note: new KV always BSNH
    const int new_batch_stride = new_seqlen * num_heads * H;
    const int new_row_stride = num_heads * H;
    const int new_head_stride = H;
    const int in_offset = b * new_batch_stride + (s - past_seqlen) * new_row_stride + n * new_head_stride + h;
    present_kv[out_offset] = new_kv[in_offset];
  }
}

// Use when (H*)*num_heads > 1024
template <typename T>
__global__ void ConcatNewToPastKVLarge(const int new_seqlen,
                                       const int H,
                                       const int num_heads,
                                       const T* past_kv,
                                       const T* new_kv,
                                       T* present_kv,
                                       const bool is_bsnh) {
  int i = threadIdx.x + (blockDim.x * blockIdx.x);
  if (i < H * num_heads) {
    const int h = i % H;
    const int n = i / H;
    const int s = blockIdx.y;
    const int b = blockIdx.z;
    const int present_seqlen = gridDim.y;

    const int present_batch_stride = present_seqlen * num_heads * H;
    const int row_stride = is_bsnh ? num_heads * H : H;
    const int present_head_stride = is_bsnh ? H : present_seqlen * H;

    // past_kv:     BPNH or BNPH
    // new_kv:      BLNH
    // present_kv:  BTNH or BNTH, where T = P + L
    const int past_seqlen = present_seqlen - new_seqlen;

    int out_offset = b * present_batch_stride + s * row_stride + n * present_head_stride + h;
    if (s < past_seqlen) {
      const int past_batch_stride = past_seqlen * num_heads * H;
      const int past_head_stride = is_bsnh ? H : past_seqlen * H;
      const int in_offset = b * past_batch_stride + s * row_stride + n * past_head_stride + h;
      present_kv[out_offset] = past_kv[in_offset];
    } else if (s < present_seqlen) {
      const int new_batch_stride = new_seqlen * num_heads * H;
      const int new_row_stride = num_heads * H;
      const int new_head_stride = H;
      const int in_offset = b * new_batch_stride + (s - past_seqlen) * new_row_stride + n * new_head_stride + h;
      present_kv[out_offset] = new_kv[in_offset];
    }
  }
}

// Concat new to past in present. Supports past BSNH or past BNSH
template <typename T>
Status LaunchConcatNewToPastKV(contrib::GroupQueryAttentionParameters& parameters,
                               GroupQueryAttentionData<T>& data,
                               hipStream_t stream,
                               const int max_threads_per_block) {
  const int batch_size = parameters.batch_size;
  const int kv_sequence_length = parameters.kv_sequence_length;
  const int present_sequence_length = parameters.present_sequence_length;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;
  AttentionQkvFormat past_kv_format = parameters.past_kv_format;

  assert(past_kv_format == AttentionQkvFormat::Q_K_V_BSNH || past_kv_format == AttentionQkvFormat::Q_K_V_BNSH);
  const int H = head_size / 4;  // divide by 4 so kernel can operate on 4 float16 elements at a time.
  if (H * kv_num_heads <= max_threads_per_block) {
    const dim3 grid(present_sequence_length, batch_size, 1);
    const dim3 block(H, kv_num_heads, 1);
    ConcatNewToPastKV<float2><<<grid, block, 0, stream>>>(kv_sequence_length,
                                                          reinterpret_cast<const float2*>(data.past_key),
                                                          reinterpret_cast<const float2*>(data.key),
                                                          reinterpret_cast<float2*>(data.present_key),
                                                          past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
    ConcatNewToPastKV<float2><<<grid, block, 0, stream>>>(kv_sequence_length,
                                                          reinterpret_cast<const float2*>(data.past_value),
                                                          reinterpret_cast<const float2*>(data.value),
                                                          reinterpret_cast<float2*>(data.present_value),
                                                          past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
  } else {
    int steps = (H * kv_num_heads + 255) / 256;
    const dim3 grid(steps, present_sequence_length, batch_size);
    const dim3 block(256, 1, 1);
    ConcatNewToPastKVLarge<float2><<<grid, block, 0, stream>>>(kv_sequence_length,
                                                               H,
                                                               kv_num_heads,
                                                               reinterpret_cast<const float2*>(data.past_key),
                                                               reinterpret_cast<const float2*>(data.key),
                                                               reinterpret_cast<float2*>(data.present_key),
                                                               past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
    ConcatNewToPastKVLarge<float2><<<grid, block, 0, stream>>>(kv_sequence_length,
                                                               H,
                                                               kv_num_heads,
                                                               reinterpret_cast<const float2*>(data.past_value),
                                                               reinterpret_cast<const float2*>(data.value),
                                                               reinterpret_cast<float2*>(data.present_value),
                                                               past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
  }
  return CUDA_CALL(hipGetLastError());
}

// Kernel to append new kv to kv buffer in place
template <typename T>
__global__ void ConcatKVInPlace(const int past_seqlen,
                                const int present_seqlen,
                                T* kv_buff,
                                const T* new_kv,
                                const bool is_bsnh) {  // refers to kv buff; otherwise bnsh
  const int h = threadIdx.x;
  const int n = threadIdx.y;
  const int s = blockIdx.x;
  const int b = blockIdx.y;

  const int new_seqlen = gridDim.x;
  const int num_heads = blockDim.y;
  const int H = blockDim.x;

  const int present_batch_stride = present_seqlen * num_heads * H;
  const int present_row_stride = is_bsnh ? num_heads * H : H;
  const int present_head_stride = is_bsnh ? H : present_seqlen * H;

  // kv_buff:     BTNH or BNTH with buffered memory for new
  // new_kv:      BLNH

  int out_offset = b * present_batch_stride + (s + past_seqlen) * present_row_stride + n * present_head_stride + h;
  // Note: new KV always BSNH
  const int new_batch_stride = new_seqlen * num_heads * H;
  const int new_row_stride = num_heads * H;
  const int new_head_stride = H;
  const int in_offset = b * new_batch_stride + s * new_row_stride + n * new_head_stride + h;
  kv_buff[out_offset] = new_kv[in_offset];
}

template <typename T>
__global__ void ConcatKVInPlaceLarge(const int past_seqlen,
                                     const int present_seqlen,
                                     const int H,
                                     const int num_heads,
                                     T* kv_buff,
                                     const T* new_kv,
                                     const bool is_bsnh) {  // refers to kv buff; otherwise bnsh
  int i = threadIdx.x + (blockDim.x * blockIdx.x);
  if (i < H * num_heads) {
    const int h = i % H;
    const int n = i / H;
    const int s = blockIdx.y;
    const int b = blockIdx.z;
    const int new_seqlen = gridDim.y;

    const int present_batch_stride = present_seqlen * num_heads * H;
    const int present_row_stride = is_bsnh ? num_heads * H : H;
    const int present_head_stride = is_bsnh ? H : present_seqlen * H;

    // kv_buff:     BTNH or BNTH with buffered memory for new
    // new_kv:      BLNH

    int out_offset = b * present_batch_stride + (s + past_seqlen) * present_row_stride + n * present_head_stride + h;
    // Note: new KV always BSNH
    const int new_batch_stride = new_seqlen * num_heads * H;
    const int new_row_stride = num_heads * H;
    const int new_head_stride = H;
    const int in_offset = b * new_batch_stride + s * new_row_stride + n * new_head_stride + h;
    kv_buff[out_offset] = new_kv[in_offset];
  }
}

// Concat new to kv buffer in place
template <typename T>
Status LaunchConcatKVInPlace(contrib::GroupQueryAttentionParameters& parameters,
                             GroupQueryAttentionData<T>& data,
                             hipStream_t stream,
                             const int max_threads_per_block) {
  const int batch_size = parameters.batch_size;
  const int kv_sequence_length = parameters.kv_sequence_length;
  const int present_sequence_length = parameters.present_sequence_length;
  const int past_sequence_length = parameters.past_sequence_length;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;
  AttentionQkvFormat past_kv_format = parameters.past_kv_format;
  assert(past_kv_format == AttentionQkvFormat::Q_K_V_BSNH || past_kv_format == AttentionQkvFormat::Q_K_V_BNSH);
  const int H = head_size / 4;
  if (H * kv_num_heads <= max_threads_per_block) {
    const dim3 grid(kv_sequence_length, batch_size, 1);
    const dim3 block(H, kv_num_heads, 1);
    ConcatKVInPlace<float2><<<grid, block, 0, stream>>>(past_sequence_length,
                                                        present_sequence_length,
                                                        reinterpret_cast<float2*>(data.present_key),
                                                        reinterpret_cast<const float2*>(data.key),
                                                        past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
    ConcatKVInPlace<float2><<<grid, block, 0, stream>>>(past_sequence_length,
                                                        present_sequence_length,
                                                        reinterpret_cast<float2*>(data.present_value),
                                                        reinterpret_cast<const float2*>(data.value),
                                                        past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
  } else {
    int steps = int(ceil(float(H * kv_num_heads) / 256.0));
    const dim3 grid(steps, kv_sequence_length, batch_size);
    const dim3 block(256, 1, 1);
    ConcatKVInPlaceLarge<float2><<<grid, block, 0, stream>>>(past_sequence_length,
                                                             present_sequence_length,
                                                             H,
                                                             kv_num_heads,
                                                             reinterpret_cast<float2*>(data.present_key),
                                                             reinterpret_cast<const float2*>(data.key),
                                                             past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
    ConcatKVInPlaceLarge<float2><<<grid, block, 0, stream>>>(past_sequence_length,
                                                             present_sequence_length,
                                                             H,
                                                             kv_num_heads,
                                                             reinterpret_cast<float2*>(data.present_value),
                                                             reinterpret_cast<const float2*>(data.value),
                                                             past_kv_format == AttentionQkvFormat::Q_K_V_BSNH);
  }
  return CUDA_CALL(hipGetLastError());
}

// Kernel for use with memory efficient kernel... kv_in is grouped and of bnsh or bsnh... kv_out is ungrouped and bsnh
template <typename T>
__global__ void Ungroup(const T* kv_in,
                        T* kv_out,
                        const int in_seqlen,
                        const int kv_num_heads,
                        const bool is_bsnh) {
  const int h = threadIdx.x;
  const int out_n = threadIdx.y;
  const int s = blockIdx.x;
  const int b = blockIdx.y;

  const int out_seqlen = gridDim.x;
  const int q_num_heads = blockDim.y;
  const int H = blockDim.x;

  const int q_kv_head_ratio = q_num_heads / kv_num_heads;
  const int out_batch_stride = out_seqlen * q_num_heads * H;
  const int out_row_stride = is_bsnh ? q_num_heads * H : H;
  const int out_head_stride = is_bsnh ? H : out_seqlen * H;

  const int in_batch_stride = in_seqlen * kv_num_heads * H;
  const int in_row_stride = is_bsnh ? kv_num_heads * H : H;
  const int in_head_stride = is_bsnh ? H : in_seqlen * H;
  const int in_n = out_n / q_kv_head_ratio;

  const int out_offset = out_batch_stride * b + out_row_stride * s + out_head_stride * out_n + h;
  const int in_offset = in_batch_stride * b + in_row_stride * s + in_head_stride * in_n + h;
  kv_out[out_offset] = kv_in[in_offset];
}

template <typename T>
__global__ void UngroupLarge(const T* kv_in,
                             T* kv_out,
                             const int H,
                             const int in_seqlen,
                             const int q_num_heads,
                             const int kv_num_heads,
                             const bool is_bsnh) {
  int i = threadIdx.x + (blockDim.x * blockIdx.x);  // index along H * q_num_heads elements
  if (i < H * q_num_heads) {
    const int out_seqlen = gridDim.y;
    const int s = blockIdx.y;
    const int b = blockIdx.z;

    const int q_kv_head_ratio = q_num_heads / kv_num_heads;
    const int out_batch_stride = out_seqlen * q_num_heads * H;
    const int out_row_stride = is_bsnh ? q_num_heads * H : H;
    const int out_head_stride = is_bsnh ? H : out_seqlen * H;

    const int in_batch_stride = in_seqlen * kv_num_heads * H;
    const int in_row_stride = is_bsnh ? kv_num_heads * H : H;
    const int in_head_stride = is_bsnh ? H : in_seqlen * H;

    const int h = i % H;
    const int out_n = i / H;
    const int in_n = out_n / q_kv_head_ratio;
    const int out_offset = out_batch_stride * b + out_row_stride * s + out_head_stride * out_n + h;
    const int in_offset = in_batch_stride * b + in_row_stride * s + in_head_stride * in_n + h;
    kv_out[out_offset] = kv_in[in_offset];
  }
}

// Ungroup kv or present kv for use in Memory Efficient kernel. If present kv is not null and is BNSH, transposes it.
Status LaunchUngroup(contrib::GroupQueryAttentionParameters& parameters,
                     float2* k_buff, float2* v_buff,
                     const float2* k_og, const float2* v_og,
                     const int buff_seqlen, const int og_seqlen,
                     const bool is_bsnh,
                     hipStream_t stream,
                     const int max_threads_per_block) {
  const int batch_size = parameters.batch_size;
  const int num_heads = parameters.num_heads;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;

  const int H = head_size / 4;
  if (H * num_heads <= max_threads_per_block) {
    const dim3 grid(buff_seqlen, batch_size, 1);
    const dim3 block(H, num_heads, 1);
    Ungroup<float2><<<grid, block, 0, stream>>>(k_og,
                                                k_buff,
                                                og_seqlen,
                                                kv_num_heads,
                                                is_bsnh);
    Ungroup<float2><<<grid, block, 0, stream>>>(v_og,
                                                v_buff,
                                                og_seqlen,
                                                kv_num_heads,
                                                is_bsnh);
  } else {
    int steps = int(ceil(float(H * num_heads) / 256.0));
    const dim3 grid(steps, buff_seqlen, batch_size);
    const dim3 block(256, 1, 1);
    UngroupLarge<float2><<<grid, block, 0, stream>>>(k_og,
                                                     k_buff,
                                                     H,
                                                     og_seqlen,
                                                     num_heads,
                                                     kv_num_heads,
                                                     is_bsnh);
    UngroupLarge<float2><<<grid, block, 0, stream>>>(v_og,
                                                     v_buff,
                                                     H,
                                                     og_seqlen,
                                                     num_heads,
                                                     kv_num_heads,
                                                     is_bsnh);
  }
  return CUDA_CALL(hipGetLastError());
}

////////// Launch Kernels

#if USE_FLASH_ATTENTION
template <typename T>
Status FlashAttention(
    const hipDeviceProp_t& device_prop,
    hipStream_t stream,
    contrib::GroupQueryAttentionParameters& parameters,
    GroupQueryAttentionData<T>& data,
    float scale) {
  const int max_threads_per_block = device_prop.maxThreadsPerBlock;
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const int kv_sequence_length = parameters.kv_sequence_length;
  const int present_sequence_length = parameters.present_sequence_length;
  const int num_heads = parameters.num_heads;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;
  AttentionQkvFormat past_kv_format = parameters.past_kv_format;

  void* query = reinterpret_cast<void*>(const_cast<T*>(data.query));
  void* key = reinterpret_cast<void*>(const_cast<T*>(data.key));
  void* value = reinterpret_cast<void*>(const_cast<T*>(data.value));

  bool is_causal = parameters.is_unidirectional;

  if (data.past_key != nullptr && data.past_key == data.present_key) {
    // Share buffer case
    void* present_key = reinterpret_cast<void*>(const_cast<T*>(data.present_key));
    void* present_value = reinterpret_cast<void*>(const_cast<T*>(data.present_value));

    // Launch kernel to copy seqlen
    int thr_per_blk = 256;
    int blk_in_grid = ceil(float(batch_size) / thr_per_blk);
    repeat_seqlen<<<blk_in_grid, thr_per_blk, 0, stream>>>(data.seqlens_k, parameters.past_sequence_length, batch_size);

    bool past_bsnh = past_kv_format == AttentionQkvFormat::Q_K_V_BSNH;
    ORT_RETURN_IF_ERROR(onnxruntime::flash::mha_fwd_kvcache(
        device_prop, stream, query, present_key, present_value, key, value, data.output, reinterpret_cast<void*>(data.softmax_lse),
        reinterpret_cast<void*>(data.seqlens_k), batch_size, num_heads, kv_num_heads,
        head_size, sequence_length, present_sequence_length, kv_sequence_length,
        scale, is_causal, past_bsnh, parameters.num_splits, reinterpret_cast<void*>(data.softmax_lse_accum),
        reinterpret_cast<void*>(data.out_accum)));

  } else {
    // Not share buffer or no past (prompt generation)
    // Note that Flash Attention kv-caching operates in place on a buffer... therefore this path is inneficient
    ORT_RETURN_IF_ERROR(LaunchConcatNewToPastKV(parameters, data, stream, max_threads_per_block));

    void* present_key = reinterpret_cast<void*>(const_cast<T*>(data.present_key));
    void* present_value = reinterpret_cast<void*>(const_cast<T*>(data.present_value));

    bool past_bsnh = past_kv_format == AttentionQkvFormat::Q_K_V_BSNH;
    ORT_RETURN_IF_ERROR(onnxruntime::flash::mha_fwd(
        device_prop, stream, query, present_key, present_value, data.output, reinterpret_cast<void*>(data.softmax_lse),
        batch_size, num_heads, kv_num_heads, head_size,
        sequence_length, present_sequence_length, scale, is_causal, parameters.num_splits,
        reinterpret_cast<void*>(data.softmax_lse_accum), reinterpret_cast<void*>(data.out_accum), past_bsnh));
  }

  DUMP_TENSOR_INIT();
  DUMP_TENSOR("flash attention output", data.output, batch_size, sequence_length, num_heads, head_size);

  return Status::OK();
}
#endif

#if USE_MEMORY_EFFICIENT_ATTENTION
template <typename T>
Status EfficientAttention(
    const hipDeviceProp_t& device_prop,
    hipStream_t stream,
    contrib::GroupQueryAttentionParameters& parameters,
    GroupQueryAttentionData<T>& data,
    float scale) {
  const int max_threads_per_block = device_prop.maxThreadsPerBlock;
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const int kv_sequence_length = parameters.kv_sequence_length;
  const int past_sequence_length = parameters.past_sequence_length;
  const int present_sequence_length = parameters.present_sequence_length;
  const int num_heads = parameters.num_heads;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;
  AttentionQkvFormat past_kv_format = parameters.past_kv_format;

  const void* query = reinterpret_cast<const void*>(data.query);
  const void* key = reinterpret_cast<const void*>(data.key);
  const void* value = reinterpret_cast<const void*>(data.value);
  if (data.past_key != nullptr) {
    // Past key case
    // concatenate new kv to past kv
    if (data.past_key == data.present_key) {
      ORT_RETURN_IF_ERROR(LaunchConcatKVInPlace(parameters, data, stream, max_threads_per_block));
    } else {
      ORT_RETURN_IF_ERROR(LaunchConcatNewToPastKV(parameters, data, stream, max_threads_per_block));
    }
    const bool is_bsnh = past_kv_format == AttentionQkvFormat::Q_K_V_BSNH;
    if (num_heads == kv_num_heads) {
      // Use present kv directly if not grouped
      key = reinterpret_cast<const void*>(data.present_key);
      value = reinterpret_cast<const void*>(data.present_value);
    } else {
      // Otherwise we use intermediate buffers to run memory efficient attention... best avoid this path
      float2* k_buff = reinterpret_cast<float2*>(data.k);
      float2* v_buff = reinterpret_cast<float2*>(data.v);
      const float2* k_og = reinterpret_cast<const float2*>(data.present_key);
      const float2* v_og = reinterpret_cast<const float2*>(data.present_value);
      ORT_RETURN_IF_ERROR(LaunchUngroup(parameters, k_buff, v_buff, k_og, v_og, past_sequence_length + kv_sequence_length,
                                        present_sequence_length, is_bsnh, stream, max_threads_per_block));
      key = reinterpret_cast<const void*>(data.k);
      value = reinterpret_cast<const void*>(data.v);
    }
  } else if (num_heads == kv_num_heads) {
    // no past or present and no need to ungroup... still copy kv into present buffer
    ORT_RETURN_IF_ERROR(LaunchConcatNewToPastKV(parameters, data, stream, max_threads_per_block));
    key = reinterpret_cast<const void*>(data.present_key);
    value = reinterpret_cast<const void*>(data.present_value);
  } else {
    // intermediate buffer so q and kv have same num heads... still copy kv into present buffer
    ORT_RETURN_IF_ERROR(LaunchConcatNewToPastKV(parameters, data, stream, max_threads_per_block));
    float2* k_buff = reinterpret_cast<float2*>(data.k);
    float2* v_buff = reinterpret_cast<float2*>(data.v);
    const float2* k_og = reinterpret_cast<const float2*>(data.present_key);
    const float2* v_og = reinterpret_cast<const float2*>(data.present_value);
    ORT_RETURN_IF_ERROR(LaunchUngroup(parameters, k_buff, v_buff, k_og, v_og, kv_sequence_length,
                                      kv_sequence_length, past_kv_format == AttentionQkvFormat::Q_K_V_BSNH, stream,
                                      max_threads_per_block));
    key = reinterpret_cast<const void*>(data.k);
    value = reinterpret_cast<const void*>(data.v);
  }

  MemoryEfficientAttentionParams p;
  p.sm = device_prop.major * 10 + device_prop.minor;
  p.is_half = sizeof(T) == 2;
  p.batch_size = batch_size;
  p.num_heads = num_heads;
  p.sequence_length = sequence_length;
  p.kv_sequence_length = past_sequence_length + kv_sequence_length;
  p.max_sequence_length = (num_heads == kv_num_heads) ? present_sequence_length : past_sequence_length + kv_sequence_length;
  p.qk_head_size = head_size;
  p.v_head_size = head_size;
  p.causal = parameters.is_unidirectional;
  p.scale = scale;
  p.seqlen_k_ptr = nullptr;
  p.seqstart_q_ptr = nullptr;
  p.seqstart_k_ptr = nullptr;
  p.query = query;
  p.key = key;
  p.value = value;
  p.attn_bias = nullptr;
  p.is_attn_bias_batched = false;
  p.is_kv_bsnh = past_kv_format == AttentionQkvFormat::Q_K_V_BSNH;
  p.output = data.output;
  p.workspace = MemoryEfficientAttentionParams::need_workspace(p.v_head_size, sizeof(T) == sizeof(float))
                    ? data.fmha_buffer
                    : nullptr;
  p.stream = stream;
  run_memory_efficient_attention(p);

  DUMP_TENSOR_INIT();
  DUMP_TENSOR("efficient attention output", data.output, batch_size, sequence_length, num_heads, head_size);

  return Status::OK();
}
#endif

////////// API Functions

template <typename T>
Status QkvToContext(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    Stream* ort_stream,
    contrib::GroupQueryAttentionParameters& parameters,
    GroupQueryAttentionData<T>& data) {
  auto stream = static_cast<hipStream_t>(ort_stream->GetHandle());
  const float scale = parameters.scale == 0.0f ? 1.f / sqrt(static_cast<float>(parameters.head_size)) : parameters.scale;

#if USE_FLASH_ATTENTION
  if (data.use_flash_attention) {
    return FlashAttention(device_prop, stream, parameters, data, scale);
  }
#endif

#if USE_MEMORY_EFFICIENT_ATTENTION
  if (data.use_memory_efficient_attention) {
    return EfficientAttention(device_prop, stream, parameters, data, scale);
  }
#endif

  return ORT_MAKE_STATUS(ONNXRUNTIME, INVALID_ARGUMENT, "Unfused Group Query Attention not implemented yet.");
}

template struct GroupQueryAttentionData<half>;

template Status QkvToContext<half>(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    Stream* ort_stream,
    contrib::GroupQueryAttentionParameters& parameters,
    GroupQueryAttentionData<half>& data);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
