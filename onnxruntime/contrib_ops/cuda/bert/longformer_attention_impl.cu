#include "hip/hip_runtime.h"
/*
Copyright (c) NVIDIA Corporation and Microsoft Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Limitations of current Longformer Attention CUDA Kernels:
// (1) Does not support global tokens in the middle. All global tokens shall be in the beginning of sequence.
// (2) Maximum number of global tokens <= one-sided attention window

#include <hipcub/hipcub.hpp>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <hip/library_types.h>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "longformer_attention_impl.h"
#include "attention_impl.h"
#include "longformer_attention_softmax.h"

using namespace onnxruntime::cuda;
using namespace hipcub;

#define CHECK(expr)         \
  if (!CUBLAS_CALL(expr)) { \
    return false;           \
  }

#define CHECK_CUDA(expr)  \
  if (!CUDA_CALL(expr)) { \
    return false;         \
  }

namespace onnxruntime {
namespace contrib {
namespace cuda {

// Denote: batch size (B), sequence length (S), number of heads (N), dimension per head (H), max number of global tokens (G)
//
// Workspace layout (default data type T is float or half):
//   [SoftmaxSpace: see below] [Q:BxNxSxH] [K:BxNxSxH] [V:BxNxSxH] [Global_Q:BxNxSxH] [Global_K:BxNxSxH] [Global_V:BxNxSxH]
// where Global_Q, Global_K and Global_V are optional. They are not allocated when there is no global token.
//
// SoftmaxSpace layout:
//    [scratch1: (5S-3W)*W*N*B][scratch2: size_t 20]
// Scratch1 has 5 buffers for local and global attention calculation.
// Scratch2 has 5 input pointers, 5 output pointers, 5 buffer sizes and 5 strides related to scratch1.

size_t GetScratch1Size(size_t element_size, int batch_size, int num_heads, int sequence_length, int window) {
  return (5 * sequence_length - 3 * window) * window * num_heads * batch_size * element_size;
}

constexpr size_t GetScratch2Size() {
  return 10 * (sizeof(void*) + sizeof(size_t));
}

size_t GetLongformerSoftmaxWorkspaceSize(
    size_t element_size,
    int batch_size,
    int num_heads,
    int sequence_length,
    int window,
    bool use_fast_kernel) {
  if (!use_fast_kernel) {
      size_t scratch1_size = GetScratch1Size(element_size, batch_size, num_heads, sequence_length, window);
      size_t scratch2_size = 10 * (sizeof(void*) + sizeof(size_t));
      return scratch1_size + scratch2_size;
  } else {
    // Non-compact layout when environment variable ORT_LONGFORMER_COMPACT_MEMORY=0 is set.
    //    [scratch1: BxNxSxS] [scratch2: BxNxSxS]
    return 2 * GetAttentionScratchSize(element_size, batch_size, num_heads, sequence_length, sequence_length);
  }
}

size_t GetLongformerAttentionWorkspaceSize(
    size_t element_size,
    int batch_size,
    int num_heads,
    int head_size,
    int sequence_length,
    int max_num_global,
    int window,
    bool use_fast_kernel) {
  size_t softmax_size = GetLongformerSoftmaxWorkspaceSize(element_size, batch_size, num_heads, sequence_length, window, use_fast_kernel);
  size_t qkv_size = 3 * batch_size * sequence_length * num_heads * head_size * element_size;
  size_t global_qkv_size = max_num_global > 0 ? qkv_size : 0;
  return softmax_size + qkv_size + global_qkv_size;
}

// Size of buffer of pinned memory in CPU. The buffer is used to copy memory between CPU and GPU.
// The buffer includes two parts: [global_count (copy of batch_global_num): int Bx1] [copy of scratch2]
size_t GetPinnedBufferSize(int batch_size) {
  return sizeof(int) * batch_size + GetScratch2Size();
}

// Softmax kernel for compact format
template <typename T, int blockSize>
__launch_bounds__(blockSize)
    __global__ void LongformerSoftmaxKernel(const int* global_attention,
                                            const int* global_index,
                                            const int* batch_global_num,
                                            void* input_pointers,
                                            const T* attention_mask,
                                            float scaler,
                                            int dim0,
                                            int sequence_length,
                                            int window,
                                            int num_heads) {
  typedef hipcub::BlockReduce<float, blockSize> BlockReduce;
  __shared__ typename BlockReduce::TempStorage block_reduce_temp;
  __shared__ float max_shared;
  __shared__ float sum_shared;

  int tid = threadIdx.x;
  const int batch_index = blockIdx.x / dim0;
  const int row_index = blockIdx.x % sequence_length;
  const int head_index = (blockIdx.x / sequence_length) % num_heads;

  // Adjust the pointers for the batch
  const T* mask_block = attention_mask + sequence_length * batch_index;
  const int* global_index_block = global_index + sequence_length * batch_index;
  const int global_num = batch_global_num[batch_index];

  size_t* p_inputs = (size_t*)(input_pointers);
  size_t* p_outputs = (size_t*)(input_pointers) + 5;
  size_t* input_sizes = (size_t*)(input_pointers) + 10;
  size_t* input_strides = (size_t*)(input_pointers) + 15;

  const T* inputs[5];
  T* outputs[5];
  for (int i = 0; i < 5; ++i) {
    inputs[i] = (T*)p_inputs[i] + batch_index * num_heads * input_sizes[i];
    outputs[i] = (T*)p_outputs[i] + batch_index * num_heads * input_sizes[i];
  }

  // Local attention token
  int col_start = 0;
  int col_end = sequence_length;
  bool is_local_row = (global_attention[batch_index * sequence_length + row_index] == static_cast<int>(0));
  if (is_local_row) {
    col_start = row_index - window;
    if (col_start < 0) {
      col_start = 0;
    }

    col_end = row_index + window + 1;
    if (col_end > sequence_length) {
      col_end = sequence_length;
    }
  }

  // If mask is set then set everything to zero to match huggingface transformers implementation
  if ((float)mask_block[row_index] != 0.f) {
    if (is_local_row) {
      T* output_block = nullptr;
      T* output_global = nullptr;
      int local_offset = row_index % window;
      int local_start = 0;
      int local_end = 3 * window;
      if (row_index < window) {
        local_start = 0;
        local_end = 2 * window;
        output_block = outputs[0] + row_index * input_strides[0] + head_index * input_sizes[0];
      } else if (row_index < sequence_length - window) {
        output_block = outputs[1] + (row_index - window) * input_strides[1] + head_index * input_sizes[1];
      } else {
        local_start = 0;
        local_end = 2 * window;
        output_block = outputs[2] + local_offset * input_strides[2] + head_index * input_sizes[2];
      }

      for (int i = local_start + tid; i < local_end; i += blockSize) {
        output_block[i] = 0;
      }

      if ((row_index - 2 * window) >= 0) {
        output_global = outputs[3] + (row_index - window) * input_strides[3] + head_index * input_sizes[3];
      }

      if (output_global != nullptr) {
        for (int i = tid; i < global_num; i += blockSize) {
          output_global[i] = 0;
        }
      }

    } else {
      T* output_block = outputs[4];
      for (int i = tid; i < sequence_length; i += blockSize)
        output_block[i] = 0;
    }
    return;
  }

  float sum_input = 0.;

  // Calculate max input
  float max_input = -HIP_INF_F;

  if (is_local_row) {
    const T* input_block = nullptr;
    T* output_block = nullptr;
    T* output_global = nullptr;
    int local_offset = row_index % window;
    int local_start = local_offset;
    int local_end = local_start + 2 * window + 1;
    int zero_start = 0;
    int zero_end = 3 * window;
    if (row_index < window) {
      local_start = 0;
      local_end = local_offset + window + 1;
      zero_end = 2 * window;

      input_block = inputs[0] + row_index * input_strides[0] + head_index * input_sizes[0];
      output_block = outputs[0] + row_index * input_strides[0] + head_index * input_sizes[0];
    } else if (row_index < sequence_length - window) {
      input_block = inputs[1] + (row_index - window) * input_strides[1] + head_index * input_sizes[1];
      output_block = outputs[1] + (row_index - window) * input_strides[1] + head_index * input_sizes[1];
    } else {
      local_start = local_offset;
      local_end = 2 * window;
      zero_end = 2 * window;

      input_block = inputs[2] + local_offset * input_strides[2] + head_index * input_sizes[2];
      output_block = outputs[2] + local_offset * input_strides[2] + head_index * input_sizes[2];
    }

    const T* input_global = nullptr;
    int local_global = row_index - window;
    if (local_global > global_num) local_global = global_num;
    if (local_global > 0) {
      input_global = inputs[3] + (row_index - window) * input_strides[3] + head_index * input_sizes[3];
    }

    if (row_index < window) {
      output_global = (T*)outputs[0] + row_index * input_strides[0] + head_index * input_sizes[0];
    } else if (row_index < 2 * window) {
      output_global = outputs[1] + (row_index - window) * input_strides[1] + head_index * input_sizes[1];
    } else {
      output_global = outputs[3] + (row_index - window) * input_strides[3] + head_index * input_sizes[3];
    }

    for (int i = local_start + tid, j = col_start + tid; i < local_end; i += blockSize, j += blockSize) {
      float x = input_block[i];
      x = x * scaler + (float)mask_block[j];
      if (max_input < x)
        max_input = x;
    }

    if (input_global != nullptr) {
      for (int i = tid; i < local_global; i += blockSize) {
        float x = input_global[global_index_block[i]];
        x = x * scaler + (float)mask_block[global_index_block[i]];
        if (max_input < x)
          max_input = x;
      }
    }

    float max_block = BlockReduce(block_reduce_temp).Reduce(max_input, hipcub::Max());
    if (tid == 0) {
      max_shared = max_block;
    }
    __syncthreads();

    for (int i = local_start + tid, j = col_start + tid; i < local_end; i += blockSize, j += blockSize) {
      float x = input_block[i];
      x = expf((x)*scaler + (float)mask_block[j] - max_shared);
      sum_input += x;
    }

    if (input_global != nullptr) {
      for (int i = tid, j = col_start + tid; i < local_global; i += blockSize, j += blockSize) {
        float x = input_global[global_index_block[i]];
        x = expf((x)*scaler + (float)mask_block[j] - max_shared);
        sum_input += x;
      }
    }

    float sum_block = BlockReduce(block_reduce_temp).Reduce(sum_input, hipcub::Sum());
    if (tid == 0) {
      sum_shared = sum_block;
    }
    __syncthreads();
    float recip_sum = 1.f / sum_shared;

    for (int i = tid + zero_start; i < local_start; i += blockSize) {
      output_block[i] = (T)(0.);
    }

    for (int i = tid + local_end; i < zero_end; i += blockSize) {
      output_block[i] = (T)(0.);
    }

    __syncthreads();

    for (int i = local_start + tid, j = col_start + tid; i < local_end; i += blockSize, j += blockSize) {
      float x = input_block[i];
      x = expf((x)*scaler + (float)mask_block[j] - max_shared);
      output_block[i] = (T)(recip_sum * x);
    }

    if (input_global != nullptr) {
      for (int i = tid; i < local_global; i += blockSize) {
        float x = input_global[global_index_block[i]];
        x = expf((x)*scaler + (float)mask_block[global_index_block[i]] - max_shared);
        output_global[i] = (T)(recip_sum * x);
      }
    }
  } else {
    // Global tokens
    const T* input_block = inputs[4] + row_index * input_strides[4] + head_index * input_sizes[4];
    T* output_block = outputs[4] + row_index * input_strides[4] + head_index * input_sizes[4];

    for (int i = tid; i < sequence_length; i += blockSize) {
      float x = input_block[i];
      x = x * scaler + (float)mask_block[i];
      if (max_input < x)
        max_input = x;
    }

    float max_block = BlockReduce(block_reduce_temp).Reduce(max_input, hipcub::Max());
    if (tid == 0) {
      max_shared = max_block;
    }
    __syncthreads();

    for (int i = tid; i < sequence_length; i += blockSize) {
      float x = input_block[i];
      x = expf((x)*scaler + (float)mask_block[i] - max_shared);
      sum_input += x;
    }

    float sum_block = BlockReduce(block_reduce_temp).Reduce(sum_input, hipcub::Sum());
    if (tid == 0) {
      sum_shared = sum_block;
    }
    __syncthreads();
    float recip_sum = 1.f / sum_shared;

    for (int i = tid; i < sequence_length; i += blockSize) {
      float x = input_block[i];
      x = expf((x)*scaler + (float)mask_block[i] - max_shared);
      output_block[i] = (T)(recip_sum * x);
    }
  }
}

bool launchSoftmaxKernel(
    hipStream_t stream,
    hipblasHandle_t cublas,
    void* workspace,
    const void* q,                // transposed Q with shape (B, N, S, H)
    const void* k,                // transposed K with shape (B, N, S, H)
    const void* v,                // transposed V with shape (B, N, S, H)
    const void* attention_mask,   // attention mask with shape (B, S), with value 0 not masked and -10000 masked.
    const void* global_q,         // Q for global tokens with shape (B, N, S, H).
    const void* global_k,         // K for global tokens with shape (B, N, S, H)
    const void* global_v,         // V for global tokens with shape (B, N, S, H)
    const int* global_attention,  // global attention with shape (B, S), with value 0 for local attention and 1 for global attention.
    const int* global_index,      // Global index with shape (B, S)
    const int* batch_global_num,  // Number of global tokens per batch with shape (B, 1)
    void* pinned_buffer,          // Pinned memory in CPU. It has two parts: Number of global tokens per batch with shape (B, 1), and a buffer to copy data to scratch2
    void* output,                 // output with shape (B, N, S, H)
    float scaler,                 // scalar
    int batch_size,               // batch size
    int sequence_length,          // sequence length
    int num_heads,                // number of heads
    int head_size,                // hidden size per head
    int window,                   // one sided window size
    size_t element_size) {        // size of element: 2 for half, and 4 for float
  const int* global_count = reinterpret_cast<const int*>(pinned_buffer);

  bool is_fp16 = (element_size == 2);
  void* scratch1 = reinterpret_cast<char*>(workspace);
  char* scratch2 = (char*)scratch1 + GetScratch1Size(element_size, batch_size, num_heads, sequence_length, window);

  // Setup shared parameters for two strided batched matrix multiplies
  hipDataType Atype;
  hipDataType Btype;
  hipDataType Ctype;
  hipDataType resultType;
  hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;

  __half one_fp16, zero_fp16;
  float one_fp32, zero_fp32;
  void *alpha, *beta_0, *beta_1;

  if (is_fp16) {
    one_fp16 = __float2half(1.f);
    zero_fp16 = __float2half(0.f);
    alpha = static_cast<void*>(&one_fp16);
    beta_0 = static_cast<void*>(&zero_fp16);
    beta_1 = static_cast<void*>(&one_fp16);
    Atype = HIP_R_16F;
    Btype = HIP_R_16F;
    Ctype = HIP_R_16F;
    resultType = HIP_R_16F;
    algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
  } else {
    one_fp32 = 1.f;
    zero_fp32 = 0.f;
    alpha = static_cast<void*>(&one_fp32);
    beta_0 = static_cast<void*>(&zero_fp32);
    beta_1 = static_cast<void*>(&one_fp32);
    Atype = HIP_R_32F;
    Btype = HIP_R_32F;
    Ctype = HIP_R_32F;
    resultType = HIP_R_32F;
  }

  // Strided batch matrix multiply
  //    qk = q * k^T
  // Shapes: q and k = B x N x S x H, qk = B x N x S x S
  // Convert col-major to row-major by swapping q and k in Gemm
  int elements_per_batch = num_heads * sequence_length * head_size;
  int stride_per_head = sequence_length * head_size;  // stride for Q, K, V and output

  // Local attention part
  // S x S is calculated using sliding block WxW (W is one sided window size) like the following:
  //   [W][W]
  //   [W][W][W]
  //      [W][W][W]
  //         [W][W]
  // The first and last rows have 2 blocks per row, and the remaining has 3 blocks per row.
  // The calculation are splited into 3 parts. Firstly, fill the middle rows,  then the first row and finally the last row.
  // To save space, we do not store the whole matrix. Instead, we only allocate space for these blocks.
  //
  // For global attention part, we have two assumptions:
  // (1) Global tokens are at the beginging of sequence
  // (2) Number of global tokens <= attention window
  //
  // The results are stored in scratch1 buffer:
  //   Number of elements for local attention are (3*S/W-2)*W*W*N*B, or (3S-2W)*W*N*B
  //   Number of elements for local attends to global are (S-W)*W*N*B
  //   Number of elements for global attends to everything are S*W*N*B
  // Total elements (FP16 or FP32) are (5S-3W)*W*N*B

  const int w = window;
  const int middle_count = (sequence_length - 2 * w) / w;
  int last_block = (sequence_length / w) - 1;

  // Determine the non-zero block dimensions and pointers

  // Buffer size per head for a single batch
  size_t buffer_sizes[5] = {
      static_cast<size_t>(w * w * 2),                  // first row of blocks has 2 WxW blocks
      static_cast<size_t>(w * w * middle_count * 3),   // middle rows of blocks have 3 WxW blocks per row
      static_cast<size_t>(w * w * 2),                  // last row of blocks has 2 WxW blocks
      static_cast<size_t>(w * (sequence_length - w)),  // local attends to global: global tokens are assumed to be smaller than window size
      static_cast<size_t>(w * sequence_length)};       // global attends to everything.

  size_t buffer_strides[5] = {
      static_cast<size_t>(w * 2),
      static_cast<size_t>(w * 3),
      static_cast<size_t>(w * 2),
      static_cast<size_t>(w),  // global tokens are assumed to be smaller than window size
      static_cast<size_t>(sequence_length)};

  void* input_pointers[5];
  void* output_pointers[5];

  char* current_pointer = (char*)scratch1;
  for (int i = 0; i < 5; ++i) {
    input_pointers[i] = (void*)current_pointer;
    output_pointers[i] = (void*)current_pointer;  // output pointer is same as input
    current_pointer += buffer_sizes[i] * num_heads * batch_size * element_size;
  }
  assert(current_pointer == scratch2);

  // Copy to a continues buffer first so that we only need call hipMemcpyAsync once

  constexpr size_t totalBytes = 10 * (sizeof(size_t) + sizeof(void*));
  char* temp_buffer = reinterpret_cast<char*>(pinned_buffer) + sizeof(int) * batch_size;
  memcpy(temp_buffer, &input_pointers[0], 5 * sizeof(void*));
  memcpy(temp_buffer + 5 * sizeof(void*), &output_pointers[0], 5 * sizeof(void*));
  memcpy(temp_buffer + 10 * sizeof(void*), &buffer_sizes[0], 5 * sizeof(size_t));
  memcpy(temp_buffer + 10 * sizeof(void*) + 5 * sizeof(size_t), &buffer_strides[0], 5 * sizeof(size_t));
  CHECK_CUDA(hipMemcpyAsync(scratch2, temp_buffer, totalBytes, hipMemcpyHostToDevice, stream));

  // Local attention part
  {
    if (middle_count > 0) {
      for (int i = 0; i < batch_size; ++i) {
        for (int j = 0; j < num_heads; ++j) {
          void* q_head = (char*)q + (i * elements_per_batch + j * sequence_length * head_size + w * head_size) * element_size;
          void* k_head = (char*)k + (i * elements_per_batch + j * sequence_length * head_size) * element_size;
          void* qk_head = (char*)input_pointers[1] + (i * num_heads + j) * buffer_sizes[1] * element_size;
          CHECK(hipblasGemmStridedBatchedEx(cublas,
                                           HIPBLAS_OP_T,
                                           HIPBLAS_OP_N,
                                           3 * w,          // m
                                           w,              // n
                                           head_size,      // k
                                           alpha,          // alpha
                                           k_head,         // A
                                           Atype,          // A type
                                           head_size,      // lda
                                           w * head_size,  // strideA
                                           q_head,         // B
                                           Btype,          // B type
                                           head_size,      // ldb
                                           w * head_size,  // strideB
                                           beta_0,         // beta
                                           qk_head,        // C
                                           Ctype,          // C type
                                           3 * w,          // ldc
                                           3 * w * w,      // strideC
                                           middle_count,   // batch count
                                           resultType,
                                           algo));
        }
      }
    }

    CHECK(hipblasGemmStridedBatchedEx(cublas,
                                     HIPBLAS_OP_T,
                                     HIPBLAS_OP_N,
                                     2 * w,                   // m
                                     w,                       // n
                                     head_size,               // k
                                     alpha,                   // alpha
                                     k,                       // A
                                     Atype,                   // A type
                                     head_size,               // lda
                                     stride_per_head,         // strideA
                                     q,                       // B
                                     Btype,                   // B type
                                     head_size,               // ldb
                                     stride_per_head,         // strideB
                                     beta_0,                  // beta
                                     input_pointers[0],       // C
                                     Ctype,                   // C type
                                     2 * w,                   // ldc
                                     buffer_sizes[0],         // strideC
                                     batch_size * num_heads,  // batch count
                                     resultType,
                                     algo));

    void* q_head = (char*)q + (last_block * w * head_size) * element_size;
    void* k_head = (char*)k + ((last_block - 1) * w * head_size) * element_size;

    CHECK(hipblasGemmStridedBatchedEx(cublas,
                                     HIPBLAS_OP_T,
                                     HIPBLAS_OP_N,
                                     2 * w,                   // m
                                     w,                       // n
                                     head_size,               // k
                                     alpha,                   // alpha
                                     k_head,                  // A
                                     Atype,                   // A type
                                     head_size,               // lda
                                     stride_per_head,         // strideA
                                     q_head,                  // B
                                     Btype,                   // B type
                                     head_size,               // ldb
                                     stride_per_head,         // strideB
                                     beta_0,                  // beta
                                     input_pointers[2],       // C
                                     Ctype,                   // C type
                                     2 * w,                   // ldc
                                     buffer_sizes[2],         // strideC
                                     batch_size * num_heads,  // batch count
                                     resultType,
                                     algo));
  }

  // Global attention part
  for (int i = 0; i < batch_size; ++i) {
    if (global_count[i] > 0) {
      void* q_batch = (char*)q + (i * elements_per_batch + w * head_size) * element_size;
      void* k_batch = (char*)k + (i * elements_per_batch) * element_size;
      void* qk_batch = (char*)input_pointers[3] + (i * buffer_sizes[3]) * num_heads * element_size;

      // Local tokens attending global tokens
      CHECK(hipblasGemmStridedBatchedEx(cublas,
                                       HIPBLAS_OP_T,
                                       HIPBLAS_OP_N,
                                       global_count[i],      // m
                                       sequence_length - w,  // n
                                       head_size,            // k
                                       alpha,                // alpha
                                       k_batch,              // A
                                       Atype,                // A type
                                       head_size,            // lda
                                       stride_per_head,      // strideA
                                       q_batch,              // B
                                       Btype,                // B type
                                       head_size,            // ldb
                                       stride_per_head,      // strideB
                                       beta_0,               // beta
                                       qk_batch,             // C
                                       Ctype,                // C type
                                       w,                    // ldc
                                       buffer_sizes[3],      // strideC
                                       num_heads,            // batch count
                                       resultType,
                                       algo));

      // It is feasible to use compact format for Global_Q with shape BxNxGxH to save space.
      // In that case, elements_per_batch is num_heads * max_num_global * head_size, and stride_per_head is max_num_global * head_size.

      void* global_q_batch = (char*)global_q + (i * elements_per_batch) * element_size;  
      void* global_k_batch = (char*)global_k + (i * elements_per_batch) * element_size;
      qk_batch = (char*)input_pointers[4] + (i * buffer_sizes[4] * num_heads) * element_size;

      // Global tokens attending everything
      // This GEMMs need to be last to make sure all global token entries are re-written.
      CHECK(hipblasGemmStridedBatchedEx(cublas,
                                       HIPBLAS_OP_T,
                                       HIPBLAS_OP_N,
                                       sequence_length,  // m
                                       global_count[i],  // n
                                       head_size,        // k
                                       alpha,            // alpha
                                       global_k_batch,   // A
                                       Atype,            // A type
                                       head_size,        // lda
                                       stride_per_head,  // strideA
                                       global_q_batch,   // B
                                       Btype,            // B type
                                       head_size,        // ldb
                                       stride_per_head,  // strideB.
                                       beta_0,           // beta
                                       qk_batch,         // C
                                       Ctype,            // C type
                                       sequence_length,  // ldc
                                       buffer_sizes[4],  // strideC
                                       num_heads,        // batch count
                                       resultType,
                                       algo));
    }
  }

  int dim0 = sequence_length * num_heads;
  int dim1 = sequence_length;

  const int blockSize = 64;
  const int gridSize = batch_size * num_heads * sequence_length;
  if (is_fp16) {
    LongformerSoftmaxKernel<__half, blockSize><<<gridSize, blockSize, 0, stream>>>(
        global_attention,
        global_index,
        batch_global_num,
        scratch2,
        static_cast<const __half*>(attention_mask),
        scaler, dim0, dim1, window, num_heads);
  } else {
    LongformerSoftmaxKernel<float, blockSize><<<gridSize, blockSize, 0, stream>>>(
        global_attention,
        global_index,
        batch_global_num,
        scratch2,
        static_cast<const float*>(attention_mask),
        scaler, dim0, dim1, window, num_heads);
  }

  // Run the matrix multiply: output = softmax_out * v
  //   softmax_out: B x N x S x S
  //             v: B x N x S x H
  //      attn_out: B x N x S x H
  // Calculation uses sliding blocks in a way similar to local attention part.

  {
    if (middle_count > 0) {
      for (int i = 0; i < batch_size; ++i) {
        for (int j = 0; j < num_heads; ++j) {
          void* v_head = (char*)v + (i * elements_per_batch + j * head_size * sequence_length) * element_size;
          void* prob_head = (char*)output_pointers[1] + (i * num_heads * buffer_sizes[1] + j * buffer_sizes[1]) * element_size;
          void* out_head = (char*)output + (i * elements_per_batch + j * head_size * sequence_length + w * head_size) * element_size;
          CHECK(hipblasGemmStridedBatchedEx(cublas,
                                           HIPBLAS_OP_N,
                                           HIPBLAS_OP_N,
                                           head_size,               // m
                                           w,                       // n
                                           3 * w,                   // k
                                           alpha,                   // alpha
                                           v_head,                  // A
                                           Atype,                   // A type
                                           head_size,               // lda
                                           w * head_size,           // strideA
                                           prob_head,               // B
                                           Btype,                   // B type
                                           (int)buffer_strides[1],  // ldb
                                           3 * w * w,               // strideB
                                           beta_0,                  // beta
                                           out_head,                // C
                                           Ctype,                   // C type
                                           head_size,               // ldc
                                           w * head_size,           // strideC
                                           middle_count,            // batch count
                                           resultType,
                                           algo));
        }
      }
    }

    CHECK(hipblasGemmStridedBatchedEx(cublas,
                                     HIPBLAS_OP_N,
                                     HIPBLAS_OP_N,
                                     head_size,               // m
                                     w,                       // n
                                     2 * w,                   // k
                                     alpha,                   // alpha
                                     v,                       // A
                                     Atype,                   // A type
                                     head_size,               // lda
                                     stride_per_head,         // strideA
                                     output_pointers[0],      // B
                                     Btype,                   // B type
                                     (int)buffer_strides[0],  // ldb
                                     buffer_sizes[0],         // strideB
                                     beta_0,                  // beta
                                     output,                  // C
                                     Ctype,                   // C type
                                     head_size,               // ldc
                                     stride_per_head,         // strideC
                                     batch_size * num_heads,  // batch count
                                     resultType,
                                     algo));

    void* v_head = (char*)v + (last_block - 1) * w * head_size * element_size;
    void* out_head = (char*)output + last_block * w * head_size * element_size;

    CHECK(hipblasGemmStridedBatchedEx(cublas,
                                     HIPBLAS_OP_N,
                                     HIPBLAS_OP_N,
                                     head_size,               // m
                                     w,                       // n
                                     2 * w,                   // k
                                     alpha,                   // alpha
                                     v_head,                  // A
                                     Atype,                   // A type
                                     head_size,               // lda
                                     stride_per_head,         // strideA
                                     output_pointers[2],      // B
                                     Btype,                   // B type
                                     (int)buffer_strides[2],  // ldb
                                     buffer_sizes[2],         // strideB
                                     beta_0,                  // beta
                                     out_head,                // C
                                     Ctype,                   // C type
                                     head_size,               // ldc
                                     stride_per_head,         // strideC
                                     batch_size * num_heads,  // batch count
                                     resultType,
                                     algo));
  }

  for (int i = 0; i < batch_size; ++i) {
    if (global_count[i] > 0) {
      int glob_longdim_mm = sequence_length - 2 * w;

      void* v_head = (char*)v + (i * elements_per_batch) * element_size;
      void* prob_head = (char*)output_pointers[3] + (i * buffer_sizes[3] * num_heads + w * buffer_strides[3]) * element_size;
      void* out_head = (char*)output + (i * elements_per_batch + 2 * w * head_size) * element_size;

      CHECK(hipblasGemmStridedBatchedEx(cublas,
                                       HIPBLAS_OP_N,
                                       HIPBLAS_OP_N,
                                       head_size,               // m
                                       glob_longdim_mm,         // n
                                       global_count[i],         // k
                                       alpha,                   // alpha
                                       v_head,                  // A
                                       Atype,                   // A type
                                       head_size,               // lda
                                       stride_per_head,         // strideA
                                       prob_head,               // B
                                       Btype,                   // B type
                                       (int)buffer_strides[3],  // ldb
                                       buffer_sizes[3],         // strideB
                                       beta_1,                  // beta
                                       out_head,                // C
                                       Ctype,                   // C type
                                       head_size,               // ldc
                                       stride_per_head,         // strideC
                                       num_heads,               // batch count
                                       resultType,
                                       algo));

      // Global tokens
      v_head = (char*)global_v + (i * elements_per_batch) * element_size;
      prob_head = (char*)output_pointers[4] + (i * buffer_sizes[4] * num_heads) * element_size;
      out_head = (char*)output + (i * elements_per_batch) * element_size;

      CHECK(hipblasGemmStridedBatchedEx(cublas,
                                       HIPBLAS_OP_N,
                                       HIPBLAS_OP_N,
                                       head_size,               // m
                                       global_count[i],         // n
                                       sequence_length,         // k: re-write entries completely
                                       alpha,                   // alpha
                                       v_head,                  // A
                                       Atype,                   // A type
                                       head_size,               // lda
                                       stride_per_head,         // strideA
                                       prob_head,               // B
                                       Btype,                   // B type
                                       (int)buffer_strides[4],  // ldb
                                       buffer_sizes[4],         // strideB
                                       beta_0,                  // beta: overwrite
                                       out_head,                // C: assumes global tokens are at the beginning of sequence
                                       Ctype,                   // C type
                                       head_size,               // ldc
                                       stride_per_head,         // strideC
                                       num_heads,               // batch count
                                       resultType,
                                       algo));
    }
  }

  return true;
}

template <typename T>
bool LongformerQkvToContext(
    hipblasHandle_t& cublas, hipStream_t stream,
    const int batch_size, const int sequence_length, const int num_heads, const int head_size,
    const int window, const size_t element_size,
    const T* input, const T* attention_mask,
    const T* global_input, const int* global_attention,
    const int* global_index, const int* batch_global_num, const int max_num_global,
    void* pinned_buffer, T* workspace,
    T* output,
    size_t softmax_workspace_size,
    bool use_fast_kernel) {
  T* qkv = reinterpret_cast<T*>((char*)workspace + softmax_workspace_size);

  // Number of elements in Q, K, V, Global_Q, Global_K or Global_V are same: BxNxSxH
  const int elements = batch_size * num_heads * sequence_length * head_size;

  // Input should be BxSx3xNxH => qkv: 3xBxNxSxH
  if (!LaunchTransQkv(stream, sequence_length, batch_size, head_size, num_heads, input, qkv)) {
    return false;
  }

  // Input 'global_input' should be BxSx3xNxH => global_qkv: 3xBxNxSxH
  T* global_qkv = qkv + 3 * elements;

  // When there is no global token, no need to process global Q, K and V
  if (max_num_global > 0 && nullptr != global_input) {
    if (!LaunchTransQkv(stream, sequence_length, batch_size, head_size, num_heads, global_input, global_qkv)) {
      return false;
    }
  }

  // Now qkv has Q, K, V: each has size BxNxSxH
  const T* q = qkv;
  const T* k = q + elements;
  const T* v = k + elements;

  const T* global_q = global_qkv;
  const T* global_k = global_q + elements;
  const T* global_v = global_k + elements;

  // Q*K' are scaled by 1/sqrt(H)
  const float rsqrt_head_size = 1.f / sqrt(static_cast<float>(head_size));

  T* temp_output = qkv;  // Q will be overwritten

  if (use_fast_kernel) {
    if (!launchSoftmaxFastKernel(
            stream,
            cublas,
            workspace,         // softmax space
            q,                 // transposed Q with shape (B, N, S, H)
            k,                 // transposed K with shape (B, N, S, H)
            v,                 // transposed V with shape (B, N, S, H)
            attention_mask,    // attention mask with shape (B, S), with value 0.0 not masked, and -10000.0 masked.
            global_q,          // Q for global tokens with shape (B, N, S, H)
            global_k,          // K for global tokens with shape (B, N, S, H)
            global_v,          // V for global tokens with shape (B, N, S, H)
            global_attention,  // global attention with shape (B, S), with value 0 for local attention and 1 for global attention.
            global_index,      // Global index with shape (B, S)
            batch_global_num,  // Number of global tokens per batch with shape (B, 1)
            pinned_buffer,     // Pinned memory in CPU. Number of global tokens per batch with shape (B, 1)
            temp_output,       // output with shape (B, N, S, H)
            rsqrt_head_size,   // scalar
            batch_size,        // batch size
            sequence_length,   // sequence length
            num_heads,         // number of heads
            head_size,         // hidden size per head
            window,            // Half (one-sided) window size
            element_size)) {
      return false;
    }
  } else {
    assert(max_num_global <= window);
    if (!launchSoftmaxKernel(
            stream,
            cublas,
            workspace,         // softmax space
            q,                 // Transposed Q with shape B x N x S x H
            k,                 // Transposed K with shape B x N x S x H
            v,                 // Transposed V with shape B x N x S x H
            attention_mask,    // Attention mask flags with shape B x S. Value -10000.0 means masked, and 0.0 not mased.
            global_q,          // Transposed global Q with shape B x N x S x H.
            global_k,          // Transposed global K with shape B x N x S x H
            global_v,          // Transposed global V with shape B x N x S x H
            global_attention,  // Global attention flags with shape B x S
            global_index,      // Global index with shape B x S
            batch_global_num,  // Number of global token per batch with shape B x 1
            pinned_buffer,     // Pinned Memory Buffer
            temp_output,       // Output with shape B x N x S x H
            rsqrt_head_size,   // Scaler
            batch_size,        // Batch size
            sequence_length,   // Sequence length
            num_heads,         // Number of attention heads
            head_size,         // Hidden size per head
            window,            // Half (one-sided) window size
            element_size)) {
      return false;
    }
  }
  

  // The temp_output is BxNxSxH, transpose it to final output BxSxNxH
  return LaunchTransCtx(stream, sequence_length, batch_size, head_size, num_heads, temp_output, output);
}

bool LaunchLongformerAttentionKernel(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    hipStream_t stream,
    const void* input,
    const void* attention_mask,
    const void* global_input,
    const int* global_attention,
    const int* global_index,
    const int* batch_global_num,
    void* pinned_buffer,
    void* workspace,
    void* output,
    int batch_size,
    int sequence_length,
    int num_heads,
    int head_size,
    int window,
    int max_num_global,
    const size_t element_size,
    bool use_fast_kernel) {
  CublasMathModeSetter helper(device_prop, cublas, HIPBLAS_TENSOR_OP_MATH);
  size_t softmax_workspace_size = GetLongformerSoftmaxWorkspaceSize(element_size, batch_size, num_heads, sequence_length, window, use_fast_kernel);
  if (element_size == 2) {
    return LongformerQkvToContext(cublas, stream,
                                  batch_size, sequence_length, num_heads, head_size, window, element_size,
                                  reinterpret_cast<const half*>(input),
                                  reinterpret_cast<const half*>(attention_mask),
                                  reinterpret_cast<const half*>(global_input),
                                  global_attention,
                                  global_index,
                                  batch_global_num,
                                  max_num_global,
                                  pinned_buffer,
                                  reinterpret_cast<half*>(workspace),
                                  reinterpret_cast<half*>(output),
                                  softmax_workspace_size,
                                  use_fast_kernel);
  } else {
    return LongformerQkvToContext(cublas, stream,
                                  batch_size, sequence_length, num_heads, head_size, window, element_size,
                                  reinterpret_cast<const float*>(input),
                                  reinterpret_cast<const float*>(attention_mask),
                                  reinterpret_cast<const float*>(global_input),
                                  global_attention,
                                  global_index,
                                  batch_global_num,
                                  max_num_global,
                                  pinned_buffer,
                                  reinterpret_cast<float*>(workspace),
                                  reinterpret_cast<float*>(output),
                                  softmax_workspace_size,
                                  use_fast_kernel);
  }
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
