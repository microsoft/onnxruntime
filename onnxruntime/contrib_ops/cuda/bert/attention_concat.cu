#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cuda_common.h"
#include "attention_impl.h"

using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

template <typename T>
__global__ void ConcatTensorToTensor(const int tensor_add_sequence_length,
                                     const T* tensor_in,
                                     const T* tensor_add,
                                     T* tensor_out) {
  const int h = threadIdx.x;
  const int n = threadIdx.y;
  const int s = blockIdx.x;
  const int b = blockIdx.y;
  const int chunk_id = blockIdx.z;

  const int all_sequence_length = gridDim.x;
  const int batch_size = gridDim.y;
  const int num_heads = blockDim.y;
  const int H = blockDim.x;

  // K: number of identical tensors
  // tensor_in:    K x BxNxS'xH
  // tensor_add:   K x BxNxSxH
  // tensor_out:   K x BxNx(S'+S)xH
  const int tensor_in_sequence_length = all_sequence_length - tensor_add_sequence_length;

  const int present_SH = all_sequence_length * H;
  const int present_NSH = num_heads * present_SH;
  int out_offset = b * present_NSH + n * present_SH + s * H + h + chunk_id * (present_NSH * batch_size);
  if (s < tensor_in_sequence_length) {
    const int past_SH = tensor_in_sequence_length * H;
    const int past_NSH = num_heads * past_SH;
    const int in_offset = b * past_NSH + n * past_SH + s * H + h + chunk_id * (past_NSH * batch_size);
    tensor_out[out_offset] = tensor_in[in_offset];
  } else if (s < all_sequence_length) {
    const int SH = tensor_add_sequence_length * H;
    const int NSH = num_heads * SH;
    const int in_offset = b * NSH + n * SH + (s - tensor_in_sequence_length) * H + h + chunk_id * (NSH * batch_size);
    tensor_out[out_offset] = tensor_add[in_offset];
  }
}

template <typename T>
__global__ void ConcatTensorToTensorLarge(const int tensor_add_sequence_length,
                                          const int H,
                                          const T* tensor_in,
                                          const T* tensor_add,
                                          T* tensor_out) {
  // Use when (H*)*num_heads > 1024
  int h = threadIdx.x;
  const int n = threadIdx.y;
  const int s = blockIdx.x;
  const int b = blockIdx.y;
  const int chunk_id = blockIdx.z;

  const int all_sequence_length = gridDim.x;
  const int batch_size = gridDim.y;
  const int num_heads = blockDim.y;
  const int stride = blockDim.x;

  // K: number of identical tensor
  // tensor_in:    K x BxNxS'xH
  // tensor_add:   K x BxNxSxH
  // tensor_out:   K x BxNx(S'+S)xH
  const int tensor_in_sequence_length = all_sequence_length - tensor_add_sequence_length;

  const int present_SH = all_sequence_length * H;
  const int present_NSH = num_heads * present_SH;
  while (h < H) {
    int out_offset = b * present_NSH + n * present_SH + s * H + h + chunk_id * (present_NSH * batch_size);
    if (s < tensor_in_sequence_length) {
      const int past_SH = tensor_in_sequence_length * H;
      const int past_NSH = num_heads * past_SH;
      const int in_offset = b * past_NSH + n * past_SH + s * H + h + chunk_id * (past_NSH * batch_size);
      tensor_out[out_offset] = tensor_in[in_offset];
    } else if (s < all_sequence_length) {
      const int SH = tensor_add_sequence_length * H;
      const int NSH = num_heads * SH;
      const int in_offset = b * NSH + n * SH + (s - tensor_in_sequence_length) * H + h + chunk_id * (NSH * batch_size);
      tensor_out[out_offset] = tensor_add[in_offset];
    }

    h += stride;
  }
}


bool LaunchConcatTensorToTensor(hipStream_t stream,
                                const int all_sequence_length,
                                const int sequence_length,
                                const int batch_size,
                                const int head_size,
                                const int num_heads,
                                const int max_threads_per_block,
                                const int matrix_num,
                                const float* tensor_in,
                                const float* tensor_add,
                                float* tensor_out) {
  const dim3 grid(all_sequence_length, batch_size, matrix_num);
  if (0 == (head_size & 1)) {
    const int H = head_size / 2;
    if (H * num_heads <= max_threads_per_block) {
      const dim3 block(H, num_heads, 1);
      ConcatTensorToTensor<float2><<<grid, block, 0, stream>>>(sequence_length, reinterpret_cast<const float2*>(tensor_in), reinterpret_cast<const float2*>(tensor_add), reinterpret_cast<float2*>(tensor_out));
    } else {
      const dim3 block(max_threads_per_block / num_heads, num_heads, 1);
      ConcatTensorToTensorLarge<float2><<<grid, block, 0, stream>>>(sequence_length, H, reinterpret_cast<const float2*>(tensor_in), reinterpret_cast<const float2*>(tensor_add), reinterpret_cast<float2*>(tensor_out));
    }
  } else {
    if (head_size * num_heads <= max_threads_per_block) {
      const dim3 block(head_size, num_heads, 1);
      ConcatTensorToTensor<float><<<grid, block, 0, stream>>>(sequence_length, tensor_in, tensor_add, tensor_out);
    } else {
      const dim3 block(max_threads_per_block / num_heads, num_heads, 1);
      ConcatTensorToTensorLarge<float><<<grid, block, 0, stream>>>(sequence_length, head_size, tensor_in, tensor_add, tensor_out);
    }

  }
  return CUDA_CALL(hipPeekAtLastError());
}

bool LaunchConcatTensorToTensor(hipStream_t stream,
                                const int all_sequence_length,
                                const int sequence_length,
                                const int batch_size,
                                const int head_size,
                                const int num_heads,
                                const int max_threads_per_block,
                                const int matrix_num,
                                const half* tensor_in,
                                const half* tensor_add,
                                half* tensor_out) {
  const dim3 grid(all_sequence_length, batch_size, matrix_num);
  if (0 == (head_size % 4)) {
    const int H = head_size / 4;
    if (H * num_heads <= max_threads_per_block) {
      const dim3 block(H, num_heads, 1);
      ConcatTensorToTensor<float2><<<grid, block, 0, stream>>>(sequence_length, reinterpret_cast<const float2*>(tensor_in), reinterpret_cast<const float2*>(tensor_add), reinterpret_cast<float2*>(tensor_out));
    } else {
      const dim3 block(max_threads_per_block / num_heads, num_heads, 1);
      ConcatTensorToTensorLarge<float2><<<grid, block, 0, stream>>>(sequence_length, H, reinterpret_cast<const float2*>(tensor_in), reinterpret_cast<const float2*>(tensor_add), reinterpret_cast<float2*>(tensor_out));
    }
  } else if (0 == (head_size & 1)) {
    const int H = head_size / 2;
    if (H * num_heads <= max_threads_per_block) {
      const dim3 block(H, num_heads, 1);
      ConcatTensorToTensor<half2><<<grid, block, 0, stream>>>(sequence_length, reinterpret_cast<const half2*>(tensor_in), reinterpret_cast<const half2*>(tensor_add), reinterpret_cast<half2*>(tensor_out));
    } else {
      const dim3 block(max_threads_per_block / num_heads, num_heads, 1);
      ConcatTensorToTensorLarge<half2><<<grid, block, 0, stream>>>(sequence_length, H, reinterpret_cast<const half2*>(tensor_in), reinterpret_cast<const half2*>(tensor_add), reinterpret_cast<half2*>(tensor_out));
    }
  } else {  // this should be an "odd" case. probably not worth catching it in the half2 kernel.
    if (head_size * num_heads <= max_threads_per_block) {
      const dim3 block(head_size, num_heads, 1);
      ConcatTensorToTensor<half><<<grid, block, 0, stream>>>(sequence_length, tensor_in, tensor_add, tensor_out);
    } else {
      const dim3 block(max_threads_per_block / num_heads, num_heads, 1);
      ConcatTensorToTensorLarge<half><<<grid, block, 0, stream>>>(sequence_length, head_size, tensor_in, tensor_add, tensor_out);
    }
  }
  return CUDA_CALL(hipPeekAtLastError());
}

bool LaunchConcatPastToPresent(hipStream_t stream,
                               const int all_sequence_length,
                               const int sequence_length,
                               const int batch_size,
                               const int head_size,
                               const int num_heads,
                               const int max_threads_per_block,
                               const float* past,
                               const float* k_v,
                               float* present) {
  return LaunchConcatTensorToTensor(
    stream,
    all_sequence_length,
    sequence_length,
    batch_size,
    head_size,
    num_heads,
    max_threads_per_block,
    2,
    past,
    k_v,
    present);
}

bool LaunchConcatPastToPresent(hipStream_t stream,
                               const int all_sequence_length,
                               const int sequence_length,
                               const int batch_size,
                               const int head_size,
                               const int num_heads,
                               const int max_threads_per_block,
                               const half* past,
                               const half* k_v,
                               half* present) {
  return LaunchConcatTensorToTensor(
    stream,
    all_sequence_length,
    sequence_length,
    batch_size,
    head_size,
    num_heads,
    max_threads_per_block,
    2,
    past,
    k_v,
    present);
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
