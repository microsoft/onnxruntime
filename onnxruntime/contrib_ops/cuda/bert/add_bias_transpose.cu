#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.
#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "contrib_ops/cuda/bert/add_bias_transpose.h"

namespace onnxruntime {
namespace cuda {

struct __align__(8) Half4 {
  half2 x;
  half2 y;
};

__device__ __forceinline__ Half4 operator+(const Half4& a, const Half4& b) {
  Half4 r;
  r.x = a.x + b.x;
  r.y = a.y + b.y;
  return r;
}

__device__ __forceinline__ float2 operator+(const float2& a, const float2& b) {
  return make_float2(a.x + b.x, a.y + b.y);
}

__device__ __forceinline__ float4 operator+(const float4& a, const float4& b) {
  return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

}  // namespace cuda
}  // namespace onnxruntime

using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

template <typename T>
__global__ void AddBiasTransposeTrt(const T* input, const T* biases, T* output) {
  // Format 2 for TensorRT fused attention (N*H <= 1024)
  //     Input:  BxSxMxNxH
  //     Output: BxSxNxMxH
  // B is batch_size, S is sequence_length, M is number of matrices, N is num_heads, H is head_size
  // This kernel could support hidden size up to 4 * 1024 when T is Half4 and input is half.

  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;  // matrix id

  const int H = blockDim.x;
  const int N = blockDim.y;
  const int S = gridDim.x;
  const int M = gridDim.z;

  const int NH = N * H;
  const int offset = (b * S + s) * M * NH;
  const int in_offset = offset + m * NH + n * H;
  const int out_offset = offset + (n * M + m) * H;

  const int h = threadIdx.x;
  if (h < H) {
    output[out_offset + h] = input[in_offset + h] + biases[m * NH + n * H + h];
  }
}

template <typename T>
__global__ void AddBiasTransposeTrtLarge(const int head_size, const T* input, const T* biases, T* output) {
  // Format 2 for TensorRT fused attention (N*H > 1024)
  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;

  const int stride = blockDim.x;
  const int H = head_size;
  const int N = blockDim.y;
  const int S = gridDim.x;
  const int M = gridDim.z;

  const int NH = N * H;
  const int offset = (b * S + s) * M * NH;
  const int in_offset = offset + m * NH + n * H;
  const int out_offset = offset + (n * M + m) * H;

  int h = threadIdx.x;
  while (h < H) {
    output[out_offset + h] = input[in_offset + h] + biases[m * NH + n * H + h];
    h += stride;
  }
}

template <typename T>
__global__ void AddBiasTransposeTrt(const T* query, const T* key, const T* value, const T* biases, T* output) {
  // Separated Q/K/V inputs for TensorRT fused attention (N*H <= 1024)
  //     Q:  BxSxNxH
  //     K:  BxSxNxH
  //     V:  BxSxNxH
  //     Output: BxSxNxMxH
  // B is batch_size, S is sequence_length, M is number of matrices (3), N is num_heads, H is head_size

  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;  // matrix id

  const int H = blockDim.x;
  const int N = blockDim.y;
  const int S = gridDim.x;
  const int M = gridDim.z;

  const T* input = (m == 0 ? query : (m == 1 ? key : value));
  const int NH = N * H;
  const int in_offset = (b * S + s) * NH + n * H;
  const int out_offset = (b * S + s) * M * NH + (n * M + m) * H;

  const int h = threadIdx.x;
  if (h < H) {
    output[out_offset + h] = input[in_offset + h] + biases[m * NH + n * H + h];
  }
}

template <typename T>
__global__ void AddBiasTransposeTrtLarge(const int head_size,
                                         const T* query, const T* key, const T* value, const T* biases, T* output) {
  // Separated Q/K/V inputs for TensorRT fused attention (N*H > 1024)
  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;  // matrix id

  const int stride = blockDim.x;
  const int H = head_size;
  const int N = blockDim.y;
  const int S = gridDim.x;
  const int M = gridDim.z;

  const T* input = (m == 0 ? query : (m == 1 ? key : value));
  const int NH = N * H;
  const int in_offset = (b * S + s) * NH + n * H;
  const int out_offset = (b * S + s) * M * NH + (n * M + m) * H;

  int h = threadIdx.x;
  if (h < H) {
    output[out_offset + h] = input[in_offset + h] + biases[m * NH + n * H + h];
    h += stride;
  }
}

template <typename T>
__global__ void AddBiasTransposeTrtKV(const T* key, const T* value, const T* biases, T* output) {
  // Separated K/V inputs for TensorRT fused cross attention (N*H <= 1024)
  //     K:  BxSxNxH
  //     V:  BxSxNxH
  //     Output: BxSxNxMxH (packed KV, requires H = H_v)
  // B is batch_size, S is sequence_length, M is number of matrices (2), N is num_heads, H is head_size

  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;  // matrix id

  const int H = blockDim.x;
  const int N = blockDim.y;
  const int S = gridDim.x;
  const int NH = N * H;

  const int in_offset = (b * S + s) * NH + n * H;
  const T* input = (m == 0 ? key : value);

  constexpr int M = 2;
  const int out_offset = (b * S + s) * M * NH + (n * M + m) * H;

  const int h = threadIdx.x;
  if (h < H) {
    output[out_offset + h] = input[in_offset + h] + biases[(m + 1) * NH + n * H + h];
  }
}

template <typename T>
__global__ void AddBiasTransposeTrtKVLarge(const int head_size,
                                           const T* key, const T* value, const T* biases,
                                           T* output) {
  // Separated K/V inputs for TensorRT fused cross attention (N*H > 1024)
  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;  // matrix id

  const int stride = blockDim.x;
  const int H = head_size;
  const int N = blockDim.y;
  const int S = gridDim.x;
  const int NH = N * H;

  const int in_offset = (b * S + s) * NH + n * H;
  const T* input = (m == 0 ? key : value);

  constexpr int M = 2;
  const int out_offset = (b * S + s) * M * NH + (n * M + m) * H;

  int h = threadIdx.x;
  while (h < H) {
    output[out_offset + h] = input[in_offset + h] + biases[(m + 1) * NH + n * H + h];
    h += stride;
  }
}

template <typename T>
__global__ void AddBiasTransposeQKV(int M, const T* input, const T* biases, T* output, T* qkv_add_bias) {
  // Format 1 for unfused attention, or fused causal attention
  //     Input:  BxSxMxNxH
  //     Output: MxBxNxSxH
  //     qkv_add_bias: BxSxMxNxH
  // B is batch_size, S is sequence_length, M is number of matrices, N is num_heads, H is head_size
  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;  // matrix id

  const int head_size = blockDim.x;
  const int num_heads = blockDim.y;

  const int sequence_length = gridDim.x;
  const int batch_size = gridDim.y;
  const int H = head_size;
  const int NH = num_heads * head_size;
  const int NHS = NH * sequence_length;

  int in_offset = n * head_size + (m + s * M) * NH + b * NHS * M;
  const int out_offset = s * head_size + n * sequence_length * H + b * NHS + m * NHS * batch_size;

  const int h = threadIdx.x;
  if (h < head_size) {
    output[out_offset + h] = input[in_offset + h] + biases[m * NH + n * H + h];
    if (nullptr != qkv_add_bias) {
      qkv_add_bias[in_offset + h] = input[in_offset + h] + biases[m * NH + n * H + h];
    }
  }
}

// this suppose 3 matrix in total
template <typename T>
__global__ void AddBiasTransposeQKV(const T* input, const T* biases, T* output, int v_head_size) {
  // Format 1 for unfused attention
  //     Input:  BxSx(NxH + NxH + NxH_v)  (Packed QKV where K and V has different hidden sizes)
  //     Output: BxNxSxH + BxNxSxH + BxNxSxH_v
  // B is batch_size, S is sequence_length, N is num_heads, H is qk_head_size, H_v is v_head_size
  int n = threadIdx.y;        // head_num_id
  int s = blockIdx.x;         // sequence_id
  int b = blockIdx.y;         // batch_id
  int m = blockIdx.z;         // matrix id (Q=0, K=1, V=2)
  const int h = threadIdx.x;  // head_element_id

  const int qk_head_size = blockDim.x;
  const int num_heads = blockDim.y;

  const int sequence_length = gridDim.x;
  const int batch_size = gridDim.y;

  const int head_size = (m == 2 ? v_head_size : qk_head_size);

  const int total_head_size = num_heads * (qk_head_size + qk_head_size + v_head_size);

  int in_offset;
  int out_offset;
  int bias_offset;
  in_offset = b * (total_head_size * sequence_length) +  // B
              s * (total_head_size) +                    // S
              m * (qk_head_size * num_heads) +           // M
              n * head_size +                            // N
              h;                                         // H

  out_offset = m * (num_heads * qk_head_size * sequence_length * batch_size) +  // M
               b * (num_heads * head_size * sequence_length) +                  // B
               n * (sequence_length * head_size) +                              // N
               s * (head_size) +                                                // S
               h;                                                               // H

  bias_offset = m * (num_heads * qk_head_size) +  // M
                n * (head_size) +                 // N
                h;                                // H

  if (h < head_size) {
    output[out_offset] = input[in_offset] + biases[bias_offset];
  }
}

template <typename T>
__global__ void AddBiasTransposeQKVLarge(const int head_size, const T* input, const T* biases, T* output,
                                         T* qkv_add_bias, const int M) {
  // Format 1 for unfused attention (N*H > 1024), or fused causal attention
  //     Input:  BxSxMxNxH (Packed QKV)
  //     Output: MxBxNxSxH
  //     qkv_add_bias: BxSxMxNxH
  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;  // matrix id

  const int stride = blockDim.x;
  const int num_heads = blockDim.y;

  const int sequence_length = gridDim.x;
  const int batch_size = gridDim.y;
  const int H = head_size;
  const int NH = num_heads * H;
  const int NHS = NH * sequence_length;
  int in_offset = n * H + (m + s * M) * NH + b * NHS * M;
  const int out_offset = s * H + n * sequence_length * H + b * NHS + m * NHS * batch_size;

  int h = threadIdx.x;
  while (h < H) {
    output[out_offset + h] = input[in_offset + h] + biases[m * NH + n * H + h];
    if (nullptr != qkv_add_bias) {
      qkv_add_bias[in_offset + h] = input[in_offset + h] + biases[m * NH + n * H + h];
    }
    h += stride;
  }
}

template <typename T>
__global__ void AddBiasTransposeCutlass(const T* input, const T* biases, T* output, int v_head_size) {
  // Format 3 for cutlass memory efficient attention
  //     Input:  BxSx(NxH + NxH + NxH_v)  (Packed QKV where K and V has different hidden sizes)
  //     Output: BxNxSxH + BxNxSxH + BxNxSxH_v
  // B is batch_size, S is sequence_length, N is num_heads, H is qk_head_size, H_v is v_head_size
  int n = threadIdx.y;        // head_num_id
  int s = blockIdx.x;         // sequence_id
  int b = blockIdx.y;         // batch_id
  int m = blockIdx.z;         // matrix id (Q=0, K=1, V=2)
  const int h = threadIdx.x;  // head_element_id

  const int qk_head_size = blockDim.x;
  const int num_heads = blockDim.y;

  const int sequence_length = gridDim.x;
  const int batch_size = gridDim.y;

  const int head_size = (m == 2 ? v_head_size : qk_head_size);

  const int total_head_size = num_heads * (qk_head_size + qk_head_size + v_head_size);

  int in_offset;
  int out_offset;
  int bias_offset;
  in_offset = b * (total_head_size * sequence_length) +  // B
              s * (total_head_size) +                    // S
              m * (qk_head_size * num_heads) +           // M
              n * head_size +                            // N
              h;                                         // H

  out_offset = m * (num_heads * qk_head_size * sequence_length * batch_size) +  // M
               b * (num_heads * head_size * sequence_length) +                  // B
               s * (num_heads * head_size) +                                    // S
               n * (head_size) +                                                // N
               h;                                                               // H

  bias_offset = m * (num_heads * qk_head_size) +  // M
                n * (head_size) +                 // N
                h;                                // H

  if (h < head_size) {
    output[out_offset] = input[in_offset] + biases[bias_offset];
  }
}

template <typename T>
__global__ void AddBiasUnpack(int M, const T* input, const T* biases, T* output) {
  // Format 4 to unpack TRT packed input format for memory efficient attention.
  //     Input:  BxSxNxMxH
  //     Output: MxBxSxNxH
  // B is batch_size, S is sequence_length, M is number of matrices, N is num_heads, H is head_size
  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;  // matrix id

  const int head_size = blockDim.x;
  const int num_heads = blockDim.y;

  const int sequence_length = gridDim.x;
  const int batch_size = gridDim.y;
  const int H = head_size;
  const int NH = num_heads * head_size;
  const int NHS = NH * sequence_length;

  int in_offset = m * head_size + n * M * H + (s * NH + b * NHS) * M;
  const int out_offset = n * head_size + s * NH + b * NHS + m * NHS * batch_size;

  const int h = threadIdx.x;
  if (h < head_size) {
    if (biases != nullptr) {
      output[out_offset + h] = input[in_offset + h] + biases[m * NH + n * H + h];
    } else {
      output[out_offset + h] = input[in_offset + h];
    }
  }
}

template <typename T>
__global__ void AddBiasTransposeCutlass(int M, const T* input, const T* biases, T* output) {
  // Format 3 for cutlass memory efficient attention
  //     Input:  BxSxMxNxH
  //     Output: MxBxSxNxH
  // B is batch_size, S is sequence_length, M is number of matrices, N is num_heads, H is head_size
  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;  // matrix id

  const int head_size = blockDim.x;
  const int num_heads = blockDim.y;

  const int sequence_length = gridDim.x;
  const int batch_size = gridDim.y;
  const int H = head_size;
  const int NH = num_heads * head_size;
  const int NHS = NH * sequence_length;

  int in_offset = n * head_size + (m + s * M) * NH + b * NHS * M;
  const int out_offset = n * head_size + s * NH + b * NHS + m * NHS * batch_size;

  const int h = threadIdx.x;
  if (h < head_size) {
    output[out_offset + h] = input[in_offset + h] + biases[m * NH + n * H + h];
  }
}

template <typename T>
__global__ void AddBiasTransposeCutlassLarge(const int head_size, const T* input, const T* biases, T* output,
                                             const int M) {
  // Format 3 for cutlass memory efficient attention
  //     Input:  BxSxMxNxH (Packed QKV)
  //     Output: MxBxSxNxH
  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;  // matrix id

  const int stride = blockDim.x;
  const int num_heads = blockDim.y;

  const int sequence_length = gridDim.x;
  const int batch_size = gridDim.y;
  const int H = head_size;
  const int NH = num_heads * H;
  const int NHS = NH * sequence_length;
  int in_offset = n * H + (m + s * M) * NH + b * NHS * M;
  const int out_offset = n * H + s * NH + b * NHS + m * NHS * batch_size;

  int h = threadIdx.x;
  while (h < H) {
    output[out_offset + h] = input[in_offset + h] + biases[m * NH + n * H + h];
    h += stride;
  }
}

template <typename T>
__global__ void AddBiasTranspose(const T* input, const T* biases, T* output) {
  // Format 0 for Separated Q, K, V (N*H <= 1024)
  //    Input:  MxBxSxNxH
  //    Output: MxBxNxSxH
  // B is batch_size, S is sequence_length, M is number of matrices, N is num_heads, H is head_size
  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;  // matrix id

  const int head_size = blockDim.x;
  const int num_heads = blockDim.y;
  const int sequence_length = gridDim.x;
  const int batch_size = gridDim.y;

  const int H = head_size;
  const int NH = num_heads * head_size;
  const int NHS = NH * sequence_length;

  int in_offset = n * H + s * NH + (b + m * batch_size) * NHS;
  const int out_offset = (s + n * sequence_length) * H + (b + m * batch_size) * NHS;

  const int h = threadIdx.x;
  if (h < head_size) {
    output[out_offset + h] = input[in_offset + h] + biases[m * NH + n * H + h];
  }
}

template <typename T>
__global__ void AddBiasTransposeLarge(const int head_size, const T* input, const T* biases, T* output) {
  // Format 0 for Separated Q, K, V (N*H > 1024)
  int n = threadIdx.y;
  int s = blockIdx.x;
  int b = blockIdx.y;
  int m = blockIdx.z;  // matrix id

  const int stride = blockDim.x;
  const int num_heads = blockDim.y;

  const int sequence_length = gridDim.x;
  const int batch_size = gridDim.y;

  const int H = head_size;
  const int NH = num_heads * H;
  const int NHS = NH * sequence_length;

  int in_offset = n * H + s * NH + (b + m * batch_size) * NHS;
  const int out_offset = (s + n * sequence_length) * H + (b + m * batch_size) * NHS;

  int h = threadIdx.x;
  while (h < H) {
    output[out_offset + h] = input[in_offset + h] + biases[m * NH + n * H + h];
    h += stride;
  }
}

template <typename T>
void InvokeAddBiasTranspose(
    hipStream_t stream, const int num_matrices, const int format, const int max_threads_per_block,
    const int batch_size, const int sequence_length, const int num_heads, const int qk_head_size,
    const T* input, const T* biases, T* output, T* qkv_add_bias, const int v_head_size, int total_matrix_count) {
  assert(num_heads <= max_threads_per_block);
  const dim3 grid(sequence_length, batch_size, num_matrices);
  if (qk_head_size * num_heads <= max_threads_per_block) {
    const dim3 block(qk_head_size, num_heads, 1);
    if (format == 2) {
      AddBiasTransposeTrt<T><<<grid, block, 0, stream>>>(input, biases, output);
    } else if (format == 1) {
      if (v_head_size == -1 || qk_head_size == v_head_size) {
        AddBiasTransposeQKV<T><<<grid, block, 0, stream>>>(total_matrix_count, input, biases, output, qkv_add_bias);
      } else {
        ORT_ENFORCE(total_matrix_count == 3);
        AddBiasTransposeQKV<T><<<grid, block, 0, stream>>>(input, biases, output, v_head_size);
      }
    } else if (format == 3) {
      if (v_head_size == -1 || qk_head_size == v_head_size) {
        AddBiasTransposeCutlass<T><<<grid, block, 0, stream>>>(total_matrix_count, input, biases, output);
      } else {
        ORT_ENFORCE(total_matrix_count == 3);
        AddBiasTransposeCutlass<T><<<grid, block, 0, stream>>>(input, biases, output, v_head_size);
      }
    } else if (format == 4) {  // format == 4
      AddBiasUnpack<T><<<grid, block, 0, stream>>>(total_matrix_count, input, biases, output);
    } else {  // format == 0
      AddBiasTranspose<T><<<grid, block, 0, stream>>>(input, biases, output);
    }
  } else {
    const dim3 block(max_threads_per_block / num_heads, num_heads, 1);
    if (format == 2) {
      AddBiasTransposeTrtLarge<T><<<grid, block, 0, stream>>>(qk_head_size, input, biases, output);
    } else if (format == 1) {
      if (v_head_size == -1 || qk_head_size == v_head_size) {
        AddBiasTransposeQKVLarge<T><<<grid, block, 0, stream>>>(qk_head_size, input, biases, output,
                                                                qkv_add_bias, total_matrix_count);
      } else {
        // It is rare for hidden size > 4096 (for half precision) and qk_head_size != v_head_size.
        ORT_THROW("AddBiasTranspose (format 1) not implemented for hidden_size > max_threads_per_block when qk_head_size != v_head_size");
      }
    } else if (format == 3) {
      if (v_head_size == -1 || qk_head_size == v_head_size) {
        AddBiasTransposeCutlassLarge<T><<<grid, block, 0, stream>>>(qk_head_size, input, biases, output,
                                                                    total_matrix_count);
      } else {
        ORT_THROW("AddBiasTranspose (format 3) not implemented for hidden_size > max_threads_per_block when qk_head_size != v_head_size");
      }
    } else if (format == 4) {  // format == 4
      ORT_THROW("AddBiasTranspose (format 4) not implemented for hidden_size > max_threads_per_block");
    } else {  // format 0
      AddBiasTransposeLarge<T><<<grid, block, 0, stream>>>(qk_head_size, input, biases, output);
    }
  }
}

template <>
void LaunchAddBiasTranspose(
    hipStream_t stream, const int num_matrices, const int format, const int max_threads_per_block,
    const int batch_size, const int sequence_length, const int num_heads, const int qk_head_size,
    const half* input, const half* biases, half* output,
    bool enable_half4, const int v_head_size, half* qkv_add_bias, int total_matrix_count) {
  total_matrix_count = std::max(num_matrices, total_matrix_count);
  if (enable_half4 && 0 == (qk_head_size % 4) && (v_head_size == -1 || 0 == (v_head_size % 4))) {
    const int H = qk_head_size / 4;
    const int H_v = v_head_size / 4;
    const Half4* input2 = reinterpret_cast<const Half4*>(input);
    const Half4* biases2 = reinterpret_cast<const Half4*>(biases);
    Half4* output2 = reinterpret_cast<Half4*>(output);
    Half4* qkv_add_bias2 = reinterpret_cast<Half4*>(qkv_add_bias);
    InvokeAddBiasTranspose<Half4>(stream, num_matrices, format, max_threads_per_block,
                                  batch_size, sequence_length, num_heads, H, input2, biases2, output2,
                                  qkv_add_bias2, H_v, total_matrix_count);
  } else if (0 == (qk_head_size & 1) && (v_head_size == -1 || 0 == (v_head_size & 1))) {
    const int H = qk_head_size / 2;
    const int H_v = v_head_size / 2;
    const half2* input2 = reinterpret_cast<const half2*>(input);
    const half2* biases2 = reinterpret_cast<const half2*>(biases);
    half2* output2 = reinterpret_cast<half2*>(output);
    half2* qkv_add_bias2 = reinterpret_cast<half2*>(qkv_add_bias);
    InvokeAddBiasTranspose<half2>(stream, num_matrices, format, max_threads_per_block,
                                  batch_size, sequence_length, num_heads, H, input2, biases2, output2,
                                  qkv_add_bias2, H_v, total_matrix_count);
  } else {
    InvokeAddBiasTranspose<half>(
        stream, num_matrices, format, max_threads_per_block,
        batch_size, sequence_length, num_heads, qk_head_size, input, biases, output,
        qkv_add_bias, v_head_size, total_matrix_count);
  }
}

template <>
void LaunchAddBiasTranspose(
    hipStream_t stream, const int num_matrices, const int format, const int max_threads_per_block,
    const int batch_size, const int sequence_length, const int num_heads, const int qk_head_size,
    const float* input, const float* biases, float* output,
    bool /*enable_half4*/, const int v_head_size, float* qkv_add_bias, int total_matrix_count) {
  total_matrix_count = std::max(num_matrices, total_matrix_count);
  if (0 == (qk_head_size % 4) && (v_head_size == -1 || 0 == (v_head_size % 4))) {
    const int H = qk_head_size / 4;
    const float4* input2 = reinterpret_cast<const float4*>(input);
    const float4* biases2 = reinterpret_cast<const float4*>(biases);
    float4* output2 = reinterpret_cast<float4*>(output);
    float4* qkv_add_bias2 = reinterpret_cast<float4*>(qkv_add_bias);
    InvokeAddBiasTranspose<float4>(
        stream, num_matrices, format, max_threads_per_block,
        batch_size, sequence_length, num_heads, H, input2, biases2, output2,
        qkv_add_bias2, v_head_size / 4, total_matrix_count);
  } else if (0 == (qk_head_size & 1) && (v_head_size == -1 || 0 == (v_head_size & 1))) {
    const int H = qk_head_size / 2;
    const float2* input2 = reinterpret_cast<const float2*>(input);
    const float2* biases2 = reinterpret_cast<const float2*>(biases);
    float2* output2 = reinterpret_cast<float2*>(output);
    float2* qkv_add_bias2 = reinterpret_cast<float2*>(qkv_add_bias);
    InvokeAddBiasTranspose<float2>(
        stream, num_matrices, format, max_threads_per_block,
        batch_size, sequence_length, num_heads, H, input2, biases2, output2,
        qkv_add_bias2, v_head_size / 2, total_matrix_count);
  } else {
    InvokeAddBiasTranspose<float>(
        stream, num_matrices, format, max_threads_per_block,
        batch_size, sequence_length, num_heads, qk_head_size, input, biases, output,
        qkv_add_bias, v_head_size, total_matrix_count);
  }
}

template <typename T>
void InvokeAddBiasTransposeTrt(
    hipStream_t stream, const int max_threads_per_block,
    const int batch_size, const int sequence_length, const int num_heads, const int head_size,
    const T* biases, const T* query, const T* key, const T* value, T* output,
    bool is_cross_attention, int kv_sequence_length) {
  if (!is_cross_attention) {
    ORT_ENFORCE(sequence_length == kv_sequence_length);
    constexpr int num_matrices = 3;
    const dim3 grid(sequence_length, batch_size, num_matrices);
    if (head_size * num_heads <= max_threads_per_block) {
      const dim3 block(head_size, num_heads, 1);
      AddBiasTransposeTrt<T><<<grid, block, 0, stream>>>(query, key, value, biases, output);
    } else {
      const dim3 block(max_threads_per_block / num_heads, num_heads, 1);
      AddBiasTransposeTrtLarge<T><<<grid, block, 0, stream>>>(head_size, query, key, value, biases, output);
    }
  } else {  // cross attention
    // Q: add bias
    {
      constexpr int num_matrices = 1;
      const dim3 grid(sequence_length, batch_size, num_matrices);
      if (head_size * num_heads <= max_threads_per_block) {
        const dim3 block(head_size, num_heads, 1);
        AddBiasTransposeTrt<T><<<grid, block, 0, stream>>>(query, biases, output);
      } else {
        const dim3 block(max_threads_per_block / num_heads, num_heads, 1);
        AddBiasTransposeTrtLarge<T><<<grid, block, 0, stream>>>(head_size, query, biases, output);
      }
    }

    // KV: add bias and pack kv
    {
      constexpr int num_matrices = 2;
      const dim3 grid(kv_sequence_length, batch_size, num_matrices);
      T* packed_kv = output + batch_size * sequence_length * num_heads * head_size;
      if (head_size * num_heads <= max_threads_per_block) {
        const dim3 block(head_size, num_heads, 1);
        AddBiasTransposeTrtKV<T><<<grid, block, 0, stream>>>(key, value, biases, packed_kv);
      } else {
        const dim3 block(max_threads_per_block / num_heads, num_heads, 1);
        AddBiasTransposeTrtKVLarge<T><<<grid, block, 0, stream>>>(head_size, key, value, biases, packed_kv);
      }
    }
  }
}

template <>
void LaunchAddBiasTransposeTrt(
    hipStream_t stream, const int max_threads_per_block,
    const int batch_size, const int sequence_length,
    const int num_heads, const int head_size,
    const float* biases, const float* query, const float* key, const float* value, float* output,
    bool is_cross_attention, int kv_sequence_length) {
  ORT_ENFORCE(false, "Shall not call this since fused kernel does not support float input.");
}

template <>
void LaunchAddBiasTransposeTrt(
    hipStream_t stream, const int max_threads_per_block,
    const int batch_size, const int sequence_length,
    const int num_heads, const int head_size,
    const half* biases, const half* query, const half* key, const half* value, half* output,
    bool is_cross_attention, int kv_sequence_length) {
  if (0 == (head_size % 4)) {
    const int H = head_size / 4;
    const Half4* query2 = reinterpret_cast<const Half4*>(query);
    const Half4* key2 = reinterpret_cast<const Half4*>(key);
    const Half4* value2 = reinterpret_cast<const Half4*>(value);
    const Half4* biases2 = reinterpret_cast<const Half4*>(biases);
    Half4* output2 = reinterpret_cast<Half4*>(output);
    InvokeAddBiasTransposeTrt<Half4>(stream, max_threads_per_block,
                                     batch_size, sequence_length, num_heads, H,
                                     biases2, query2, key2, value2, output2, is_cross_attention, kv_sequence_length);
  } else if (0 == (head_size & 1)) {
    const int H = head_size / 2;
    const half2* query2 = reinterpret_cast<const half2*>(query);
    const half2* key2 = reinterpret_cast<const half2*>(key);
    const half2* value2 = reinterpret_cast<const half2*>(value);
    const half2* biases2 = reinterpret_cast<const half2*>(biases);
    half2* output2 = reinterpret_cast<half2*>(output);
    InvokeAddBiasTransposeTrt<half2>(stream, max_threads_per_block,
                                     batch_size, sequence_length, num_heads, H,
                                     biases2, query2, key2, value2, output2, is_cross_attention, kv_sequence_length);
  } else {
    InvokeAddBiasTransposeTrt<half>(stream, max_threads_per_block,
                                    batch_size, sequence_length, num_heads, head_size,
                                    biases, query, key, value, output, is_cross_attention, kv_sequence_length);
  }
}

template <typename T>
void InvokeAddBias(
    hipStream_t stream, const int max_threads_per_block,
    const int batch_size, const int sequence_length, const int kv_sequence_length,
    const int num_heads, const int head_size, const int v_head_size,
    const T* biases, const T* query, const T* key, const T* value, T* q, T* k, T* v) {
  assert(num_heads <= max_threads_per_block);
  constexpr int num_matrices = 1;
  // Q
  {
    const dim3 grid(sequence_length, batch_size, num_matrices);
    if (head_size * num_heads <= max_threads_per_block) {
      const dim3 block(head_size, num_heads, 1);
      AddBiasTransposeTrt<T><<<grid, block, 0, stream>>>(query, biases, q);
    } else {
      const dim3 block(max_threads_per_block / num_heads, num_heads, 1);
      AddBiasTransposeTrtLarge<T><<<grid, block, 0, stream>>>(head_size, query, biases, q);
    }
  }
  // K
  {
    const dim3 grid(kv_sequence_length, batch_size, num_matrices);
    const T* biases_k = biases + num_heads * head_size;

    if (head_size * num_heads <= max_threads_per_block) {
      const dim3 block(head_size, num_heads, 1);
      AddBiasTransposeTrt<T><<<grid, block, 0, stream>>>(key, biases_k, k);
    } else {
      const dim3 block(max_threads_per_block / num_heads, num_heads, 1);
      AddBiasTransposeTrtLarge<T><<<grid, block, 0, stream>>>(head_size, key, biases_k, k);
    }
  }

  // V
  {
    const dim3 grid(kv_sequence_length, batch_size, num_matrices);

    const T* biases_v = biases + 2 * num_heads * head_size;
    if (v_head_size * num_heads <= max_threads_per_block) {
      const dim3 block(v_head_size, num_heads, 1);
      AddBiasTransposeTrt<T><<<grid, block, 0, stream>>>(value, biases_v, v);
    } else {
      const dim3 block(max_threads_per_block / num_heads, num_heads, 1);
      AddBiasTransposeTrtLarge<T><<<grid, block, 0, stream>>>(v_head_size, value, biases_v, v);
    }
  }
}

template <>
void LaunchAddBias(
    hipStream_t stream, const int max_threads_per_block,
    const int batch_size, const int sequence_length, const int kv_sequence_length,
    const int num_heads, const int head_size, const int v_head_size,
    const float* biases, const float* query, const float* key, const float* value, float* q, float* k, float* v) {
  if (0 == (head_size % 4) && 0 == (v_head_size % 4)) {
    const int H = head_size / 4;
    const int H_v = v_head_size / 4;
    const float4* query2 = reinterpret_cast<const float4*>(query);
    const float4* key2 = reinterpret_cast<const float4*>(key);
    const float4* value2 = reinterpret_cast<const float4*>(value);
    const float4* biases2 = reinterpret_cast<const float4*>(biases);
    float4* q2 = reinterpret_cast<float4*>(q);
    float4* k2 = reinterpret_cast<float4*>(k);
    float4* v2 = reinterpret_cast<float4*>(v);
    InvokeAddBias<float4>(stream, max_threads_per_block,
                          batch_size, sequence_length, kv_sequence_length, num_heads, H, H_v,
                          biases2, query2, key2, value2, q2, k2, v2);
  } else if (0 == (head_size & 1) && 0 == (v_head_size & 1)) {
    const int H = head_size / 2;
    const int H_v = v_head_size / 2;
    const float2* query2 = reinterpret_cast<const float2*>(query);
    const float2* key2 = reinterpret_cast<const float2*>(key);
    const float2* value2 = reinterpret_cast<const float2*>(value);
    const float2* biases2 = reinterpret_cast<const float2*>(biases);
    float2* q2 = reinterpret_cast<float2*>(q);
    float2* k2 = reinterpret_cast<float2*>(k);
    float2* v2 = reinterpret_cast<float2*>(v);
    InvokeAddBias<float2>(stream, max_threads_per_block,
                          batch_size, sequence_length, kv_sequence_length, num_heads, H, H_v,
                          biases2, query2, key2, value2, q2, k2, v2);
  } else {
    InvokeAddBias<float>(stream, max_threads_per_block,
                         batch_size, sequence_length, kv_sequence_length, num_heads, head_size, v_head_size,
                         biases, query, key, value, q, k, v);
  }
}

template <>
void LaunchAddBias(
    hipStream_t stream, const int max_threads_per_block,
    const int batch_size, const int sequence_length, const int kv_sequence_length,
    const int num_heads, const int head_size, const int v_head_size,
    const half* biases, const half* query, const half* key, const half* value, half* q, half* k, half* v) {
  if (0 == (head_size % 4) && 0 == (v_head_size % 4)) {
    const int H = head_size / 4;
    const int H_v = v_head_size / 4;
    const Half4* query2 = reinterpret_cast<const Half4*>(query);
    const Half4* key2 = reinterpret_cast<const Half4*>(key);
    const Half4* value2 = reinterpret_cast<const Half4*>(value);
    const Half4* biases2 = reinterpret_cast<const Half4*>(biases);
    Half4* q2 = reinterpret_cast<Half4*>(q);
    Half4* k2 = reinterpret_cast<Half4*>(k);
    Half4* v2 = reinterpret_cast<Half4*>(v);
    InvokeAddBias<Half4>(stream, max_threads_per_block,
                         batch_size, sequence_length, kv_sequence_length, num_heads, H, H_v,
                         biases2, query2, key2, value2, q2, k2, v2);
  } else if (0 == (head_size & 1) && 0 == (v_head_size & 1)) {
    const int H = head_size / 2;
    const int H_v = v_head_size / 2;
    const half2* query2 = reinterpret_cast<const half2*>(query);
    const half2* key2 = reinterpret_cast<const half2*>(key);
    const half2* value2 = reinterpret_cast<const half2*>(value);
    const half2* biases2 = reinterpret_cast<const half2*>(biases);
    half2* q2 = reinterpret_cast<half2*>(q);
    half2* k2 = reinterpret_cast<half2*>(k);
    half2* v2 = reinterpret_cast<half2*>(v);
    InvokeAddBias<half2>(stream, max_threads_per_block,
                         batch_size, sequence_length, kv_sequence_length, num_heads, H, H_v,
                         biases2, query2, key2, value2, q2, k2, v2);
  } else {
    InvokeAddBias<half>(stream, max_threads_per_block,
                        batch_size, sequence_length, kv_sequence_length, num_heads, head_size, v_head_size,
                        biases, query, key, value, q, k, v);
  }
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
