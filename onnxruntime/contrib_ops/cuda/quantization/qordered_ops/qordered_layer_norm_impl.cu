#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "contrib_ops/cuda/quantization/qordered_ops/qordered_layer_norm_impl.h"
#include "contrib_ops/cuda/quantization/qordered_ops/qordered_common.cuh"

namespace onnxruntime {
namespace contrib {
namespace cuda {

static __device__ inline float ToFloat(const __half h) { return __half2float(h); }

static __device__ inline float ToFloat(const float f) { return f; }

constexpr static unsigned QORDER_LAYERNORM_ROWS_PER_BLOCK = 8;

template <typename T>
__global__ void QOrderedLayerNormRowKernel(const int8_t* __restrict__ src, const float src_scale,
                                           int8_t* __restrict__ dst, const float dst_scale,
                                           const T* __restrict__ gamma, const T* __restrict__ beta, const float epsilon,
                                           const unsigned rows, const unsigned cols) {
  int32_t sum = 0;
  int32_t square_sum = 0;

  unsigned r = blockIdx.x * QORDER_LAYERNORM_ROWS_PER_BLOCK + threadIdx.y;

  if (rows <= r) {
    return;
  }

  const size_t batch_row_index = static_cast<size_t>(blockIdx.y) * (rows * cols) + r * cols;
  src += batch_row_index;
  dst += batch_row_index;
  for (unsigned c = threadIdx.x << 2; c < cols; c += 128) {
    char4 ch4 = __ldg(reinterpret_cast<const char4*>(src + c));
    sum += (static_cast<short>(ch4.x) + static_cast<short>(ch4.y) +
            static_cast<short>(ch4.z) + static_cast<short>(ch4.w));
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 610
    square_sum = __dp4a(ch4, ch4, square_sum);
#else
    square_sum = Dp4a_Defined(ch4, ch4);
#endif
  }

  sum = WarpReduceSum<int32_t>(sum);
  square_sum = WarpReduceSum<int32_t>(square_sum);

  const float mean = __double2float_rn(src_scale * (double)sum / cols);

  const float rvar = rsqrtf(src_scale * src_scale * __double2float_rn(static_cast<double>(square_sum) - (static_cast<double>(sum) * static_cast<double>(sum) / static_cast<double>(cols))) / cols + epsilon);

  const float dst_rscale = 1.0f / dst_scale;

  float4 f4;
  for (unsigned c = threadIdx.x << 2; c < cols; c += 128) {
    char4 ch4 = __ldg(reinterpret_cast<const char4*>(src + c));

    f4.x = (src_scale * ch4.x - mean) * rvar * ToFloat(gamma[c]);
    f4.y = (src_scale * ch4.y - mean) * rvar * ToFloat(gamma[c + 1]);
    f4.z = (src_scale * ch4.z - mean) * rvar * ToFloat(gamma[c + 2]);
    f4.w = (src_scale * ch4.w - mean) * rvar * ToFloat(gamma[c + 3]);

    if (beta) {
      f4.x += ToFloat(beta[c]);
      f4.y += ToFloat(beta[c + 1]);
      f4.z += ToFloat(beta[c + 2]);
      f4.w += ToFloat(beta[c + 3]);
    }

    *reinterpret_cast<char4*>(dst + c) = QuantizeFloat4Char4(f4, dst_rscale);
  }
}

template <typename T>
Status QOrderedLayerNorm(hipStream_t stream, const hipDeviceProp_t& /*device_prop*/, hipblasLtOrder_t order,
                       const int8_t* src, const float src_scale, int8_t* dst, const float dst_scale,
                       const T* gamma, const T* beta, const float epsilon,
                       const unsigned batch, const unsigned rows, const unsigned cols) {
  // The implementation only supports Row major tensor data ordering for now
  ORT_RETURN_IF(order != HIPBLASLT_ORDER_ROW, "Order current not supported!");

  dim3 threads(32, QORDER_LAYERNORM_ROWS_PER_BLOCK, 1);

  dim3 blocks(static_cast<unsigned>(rows + QORDER_LAYERNORM_ROWS_PER_BLOCK - 1) / QORDER_LAYERNORM_ROWS_PER_BLOCK,
              static_cast<unsigned>(batch), 1);

  QOrderedLayerNormRowKernel<T><<<blocks, threads, 0, stream>>>(
      src, src_scale, dst, dst_scale, gamma, beta, epsilon, rows, cols);

  return CUDA_CALL(hipGetLastError());  
}

template Status QOrderedLayerNorm<float>(hipStream_t stream, const hipDeviceProp_t& /*device_prop*/, hipblasLtOrder_t order,
                                       const int8_t* src, const float src_scale, int8_t* dst, const float dst_scale,
                                       const float* gamma, const float* beta, const float epsilon,
                                       const unsigned batch, const unsigned rows, const unsigned cols);

template Status QOrderedLayerNorm<__half>(hipStream_t stream, const hipDeviceProp_t& /*device_prop*/, hipblasLtOrder_t order,
                                        const int8_t* src, const float src_scale, int8_t* dst, const float dst_scale,
                                        const __half* gamma, const __half* beta, const float epsilon,
                                        const unsigned batch, const unsigned rows, const unsigned cols);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
