#include "hip/hip_runtime.h"
// Modifications: scaling is moved from masked softmax to the gemm before that.
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hipcub/hipcub.hpp>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <cmath>
#include <hip/hip_math_constants.h>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "dequantize_blockwise.cuh"

using namespace onnxruntime::cuda;
using namespace hipcub;

namespace onnxruntime {
namespace contrib {
namespace cuda {

__device__ __forceinline__ void DequantizeEightElements(uint32_t values_quant, half scale, half zp, half* output) {
  half2 scale_half2 = {scale, scale};
  half zp_adjust = -scale * __short2half_rn(zp);
  half2 zp_adjust2 = {zp_adjust, zp_adjust};

  alignas(16) half2 results[4];
  half v0 = __uint2half_rn(values_quant & 0xF);
  half v1 = __uint2half_rn((values_quant >> 4) & 0xF);
  results[0] = __halves2half2(v0, v1) * scale_half2 + zp_adjust2;

  half v2 = __uint2half_rn((values_quant >> 8) & 0xF);
  half v3 = __uint2half_rn((values_quant >> 12) & 0xF);
  results[1] = __halves2half2(v2, v3) * scale_half2 + zp_adjust2;

  half v4 = __uint2half_rn((values_quant >> 16) & 0xF);
  half v5 = __uint2half_rn((values_quant >> 20) & 0xF);
  results[2] = __halves2half2(v4, v5) * scale_half2 + zp_adjust2;

  half v6 = __uint2half_rn((values_quant >> 24) & 0xF);
  half v7 = __uint2half_rn((values_quant >> 28) & 0xF);
  results[3] = __halves2half2(v6, v7) * scale_half2 + zp_adjust2;
  *(reinterpret_cast<float4*>(output)) = *(reinterpret_cast<float4*>(results));
}

__device__ __forceinline__ void DequantizeEightElements(uint32_t values_quant, float scale, float zp, float* output) {
  float zp_adjust = -scale * zp;
  output[0] = float(values_quant & 0xF) * scale + zp_adjust;
  output[1] = float((values_quant >> 4) & 0xF) * scale + zp_adjust;
  output[2] = float((values_quant >> 8) & 0xF) * scale + zp_adjust;
  output[3] = float((values_quant >> 12) & 0xF) * scale + zp_adjust;
  output[4] = float((values_quant >> 16) & 0xF) * scale + zp_adjust;
  output[5] = float((values_quant >> 20) & 0xF) * scale + zp_adjust;
  output[6] = float((values_quant >> 24) & 0xF) * scale + zp_adjust;
  output[7] = float((values_quant >> 28) & 0xF) * scale + zp_adjust;
}

template <class T>
__global__ void Dequantize4BitsKernel(
    T* output,
    const uint8_t* quant_data,
    const T* scale_data,
    const uint8_t* zero_points,
    int block_size,
    int blocks_per_threadblock,
    int shift) {
  int block_id = blockIdx.x * blocks_per_threadblock + ((threadIdx.x * 8) >> shift);
  int element_offset = block_id * block_size + ((threadIdx.x * 8) & ((1 << shift) - 1));
  uint32_t quant_value = *(reinterpret_cast<const uint32_t*>(quant_data + element_offset / 2));
  T scale = *(scale_data + block_id);
  uint8_t zp = 8;
  if (zero_points) {
    zp = (block_id & 0x01) ? (zero_points[block_id / 2] >> 4) : (zero_points[block_id / 2] & 0x0f);
  }

  output = output + element_offset;
  DequantizeEightElements(quant_value, scale, static_cast<T>(zp), output);
}

template <class T>
Status Dequantize4Bits(
    T* output,
    const uint8_t* quant_data,
    const T* scales_data,
    const uint8_t* zero_points,  // shape: [N, (block_per_K + 1)/2]
    int k,
    int n,
    int block_size,
    hipStream_t stream) {
  // k is padded and equal to block_per_K * block_size
  ORT_ENFORCE(k % block_size == 0, "k must be a multiplier of block_size");
  constexpr int element_per_thread = 8;
  int blocks_per_threadblock = GridDim::maxThreadsPerBlock * element_per_thread / block_size;
  int blocks_per_K = k / block_size;
  int blocks_per_grid = static_cast<int>(CeilDiv(n * blocks_per_K, blocks_per_threadblock));
  int shift = static_cast<int>(log2f(float(block_size)));

  Dequantize4BitsKernel<<<blocks_per_grid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      output,
      quant_data,
      scales_data,
      zero_points,
      block_size,
      blocks_per_threadblock,
      shift);

  return Status::OK();
}

template Status Dequantize4Bits<float>(
    float* output,
    const uint8_t* quant_data,
    const float* scales_data,
    const uint8_t* zero_points,
    int k,
    int n,
    int block_size,
    hipStream_t stream);

template Status Dequantize4Bits<half>(
    half* output,
    const uint8_t* quant_data,
    const half* scales_data,
    const uint8_t* zero_points,
    int k,
    int n,
    int block_size,
    hipStream_t stream);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
