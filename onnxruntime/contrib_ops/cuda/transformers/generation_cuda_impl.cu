#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "cub/util_type.cuh"
#include <hipcub/hipcub.hpp>
#include <cub/device/device_segmented_radix_sort.cuh>
#include "contrib_ops/cuda/bert/utils.cuh"
#include "contrib_ops/cuda/transformers/generation_cuda_impl.h"

namespace onnxruntime {
namespace contrib {
namespace cuda {
__global__ void InitKernel(float* beam_scores,
                           int num_beams,
                           int total_elements) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < total_elements) {
    int beam_index = index % num_beams;
    beam_scores[index] = beam_index > 0 ? static_cast<float>(-1e9) : 0.0f;
  }
}

void LaunchInitKernel(
    float* beam_scores,
    int batch_size,
    int num_beams,
    hipStream_t stream) {
  int total_elements = batch_size * num_beams;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  InitKernel<<<gridSize, blockSize, 0, stream>>>(beam_scores, num_beams, total_elements);
}

__global__ void NextTokenKernel(const int64_t* next_token_indices,
                                int32_t* next_indices,
                                int32_t* next_tokens,
                                int vocab_size,
                                int total_elements) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < total_elements) {
    next_indices[index] = next_token_indices[index] / vocab_size;
    next_tokens[index] = next_token_indices[index] % vocab_size;
  }
}

void LaunchNextTokenKernel(const int64_t* next_token_indices,
                           int32_t* next_indices,
                           int32_t* next_tokens,
                           int batch_size,
                           int top_k,
                           int vocab_size,
                           hipStream_t stream) {
  int total_elements = batch_size * top_k;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  NextTokenKernel<<<gridSize, blockSize, 0, stream>>>(next_token_indices,
                                                      next_indices,
                                                      next_tokens,
                                                      vocab_size,
                                                      total_elements);
}

template <typename T>
__global__ void LogitsProcessKernel(
    T* next_token_scores,
    const int* vocab_mask,
    const int* prefix_vocab_mask,
    const int* presence_mask,
    float presence_penalty,
    float temperature,
    int num_beams,
    int vocab_size,
    int padded_vocab_size,
    int total_elements,
    int demote_token_id,
    const int32_t* sequences,
    int max_sequence_length,
    int current_sequence_length,
    float repetition_penalty,
    int no_repeat_ngram_size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < total_elements) {
    int batch_beam_index = index / padded_vocab_size;
    int word_id = index % padded_vocab_size;

    if (word_id >= vocab_size) {
      // Set any value within the padding region to the lowest value so that it isn't picked
      next_token_scores[index] = hipcub::FpLimits<T>::Lowest();
    } else {
      // RepetitionPenaltyLogitsProcessor
      if (repetition_penalty != 1.0f) {
        const int32_t* current_sequence = sequences + batch_beam_index * max_sequence_length;
        bool found = false;
        for (int i = 0; i < current_sequence_length; i++) {
          if (current_sequence[i] == word_id) {
            found = true;
            break;
          }
        }
        if (found) {
          float score = (float)next_token_scores[index];
          next_token_scores[index] = (T)(score < 0 ? score * repetition_penalty : score / repetition_penalty);
        }
      }

      // NoRepeatNGramLogitsProcessor
      if (no_repeat_ngram_size > 0 && current_sequence_length >= no_repeat_ngram_size) {
        const int32_t* current_sequence = sequences + batch_beam_index * max_sequence_length;
        bool found = false;
        for (int i = no_repeat_ngram_size - 1; i < current_sequence_length; i++) {
          if (current_sequence[i] == word_id) {  // last token of n-gram matched
            found = true;
            for (int j = 0; j < no_repeat_ngram_size - 1; j++) {  // match the remaining N-1 tokens
              if (current_sequence[i - j - 1] != current_sequence[current_sequence_length - 1 - j]) {
                found = false;
                break;
              }
            }
            if (found) {
              break;
            }
          }
        }

        if (found) {
          next_token_scores[index] = hipcub::FpLimits<T>::Lowest();
          return;
        }
      }

      // VocabMaskLogitsProcessor
      if (vocab_mask != nullptr && vocab_mask[word_id] == 0) {
        next_token_scores[index] = hipcub::FpLimits<T>::Lowest();
        return;
      }

      // PrefixVocabMaskLogitsProcessor
      int batch_id = batch_beam_index / num_beams;
      if (prefix_vocab_mask != nullptr && prefix_vocab_mask[batch_id * vocab_size + word_id] == 0) {
        next_token_scores[index] = hipcub::FpLimits<T>::Lowest();
        return;
      }

      // MinLengthLogitsProcessor
      if (word_id == demote_token_id) {
        next_token_scores[index] = hipcub::FpLimits<T>::Lowest();
      }

      // PresencePenaltyLogitsProcessor
      if (presence_mask != nullptr && presence_mask[index] == 1) {
        float score = (float)next_token_scores[index] - presence_penalty;
        next_token_scores[index] = (T)score;
      }

      // TemperatureLogitsProcessor
      if (temperature != 1.0f) {
        float score = (float)(next_token_scores[index]);
        next_token_scores[index] = (T)(score / temperature);
      }
    }
  }
}

template <typename T>
void LaunchLogitsProcessKernel(
    T* next_token_scores,
    const int* vocab_mask,
    const int* prefix_vocab_mask,
    int* presence_mask,
    float presence_penalty,
    float temperature,
    int batch_size,
    int num_beams,
    int vocab_size,
    int padded_vocab_size,
    int demote_token_id,
    const int32_t* sequences,
    int max_sequence_length,
    int current_sequence_length,
    float repetition_penalty,
    int no_repeat_ngram_size,
    hipStream_t stream) {
  int total_elements = batch_size * num_beams * padded_vocab_size;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  LogitsProcessKernel<T><<<gridSize, blockSize, 0, stream>>>(
      next_token_scores,
      vocab_mask,
      prefix_vocab_mask,
      presence_mask,
      presence_penalty,
      temperature,
      num_beams,
      vocab_size,
      padded_vocab_size,
      total_elements,
      demote_token_id,
      sequences,
      max_sequence_length,
      current_sequence_length,
      repetition_penalty,
      no_repeat_ngram_size);
}

// Instantiation
template void LaunchLogitsProcessKernel(
    float* next_token_scores,
    const int* vocab_mask,
    const int* prefix_vocab_mask,
    int* presence_mask,
    float presence_penalty,
    float temperature,
    int batch_size,
    int num_beams,
    int vocab_size,
    int padded_vocab_size,
    int demote_token_id,
    const int32_t* sequences,
    int max_sequence_length,
    int current_sequence_length,
    float repetition_penalty,
    int no_repeat_ngram_size,
    hipStream_t stream);

template void LaunchLogitsProcessKernel(
    half* next_token_scores,
    const int* vocab_mask,
    const int* prefix_vocab_mask,
    int* presence_mask,
    float presence_penalty,
    float temperature,
    int batch_size,
    int num_beams,
    int vocab_size,
    int padded_vocab_size,
    int demote_token_id,
    const int32_t* sequences,
    int max_sequence_length,
    int current_sequence_length,
    float repetition_penalty,
    int no_repeat_ngram_size,
    hipStream_t stream);

__global__ void InitializeBeamHypotheses(BeamHypotheses* beam_hyps, int beam_hyps_count, float length_penalty, HypothesisScore* beams, int num_beams) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= beam_hyps_count)
    return;

  BeamHypotheses& beam_hyp = beam_hyps[index];
  beam_hyp.beams_ = beams + index * num_beams;
  beam_hyp.beams_count_ = num_beams;
  beam_hyp.beams_used_ = 0;
  beam_hyp.length_penalty_ = length_penalty;
  beam_hyp.done_ = false;
}

// For counts that are typically far less than 256, this will round up the count to the next multiple of 32
// If this winds up being >256 then it uses a block size of 256 and calculates the appropriate grid_size
struct GridBlock32 {
  GridBlock32(int count) {
    block_size_ = (count + 31) & ~31;  // Round up to nearest multiple of 32
    if (block_size_ > 256) {
      grid_size_ = (block_size_ + 255) / 256;
      block_size_ = 256;
    }
  }

  int grid_size_{1};
  int block_size_;
};

void LaunchInitializeBeamHypotheses(gsl::span<BeamHypotheses> beam_hyps,
                                    float length_penalty,
                                    gsl::span<HypothesisScore> beams,
                                    int num_beams,
                                    hipStream_t stream) {
  GridBlock32 gb32{static_cast<int>(beam_hyps.size())};
  InitializeBeamHypotheses<<<gb32.grid_size_, gb32.block_size_, 0, stream>>>(beam_hyps.data(),
                                                                             static_cast<int>(beam_hyps.size()),
                                                                             length_penalty,
                                                                             beams.data(),
                                                                             num_beams);
}

__device__ void BeamHypotheses::Add(const int32_t* hypothesis, int hypothesis_length, float sum_logprobs) {
  float score = sum_logprobs / pow(static_cast<float>(hypothesis_length), length_penalty_);

  size_t index = beams_used_;
  // If the array is full, don't add unless it's better than the worst element
  if (index == beams_count_) {
    if (score <= beams_[--index].score)
      return;
  } else
    beams_used_++;

  // Rotate existing elements over while the new element scores higher
  for (; index > 0 && score > beams_[index - 1].score; index--)
    beams_[index] = beams_[index - 1];

  beams_[index] = HypothesisScore{hypothesis, hypothesis_length, score};
}

__device__ bool BeamHypotheses::CanImprove(float best_sum_logprobs, int current_length) const {
  float current_score = best_sum_logprobs / pow(static_cast<float>(current_length), length_penalty_);
  return beams_[beams_count_ - 1].score < current_score;
}

__device__ void BeamHypotheses::Output(
    int top_k,
    int max_length,
    int pad_token_id,
    int32_t* sequences,       // buffer of shape (num_return_sequences, max_length)
    float* sequences_scores)  // buffer of shape (num_return_sequences) or empty
{
  // Copy the top_k beams into the sequences
  for (int index = 0; index < top_k; index++) {
    auto& item = beams_[index];
    int32_t* target = sequences + index * max_length;

    // Note that word_ids might be less than max_length.
    for (int i = 0; i < item.hypothesis_length; i++)
      target[i] = item.hypothesis[i];
    // Pad remaining values with pad token id
    for (int i = item.hypothesis_length; i < max_length; i++)
      target[i] = pad_token_id;

    if (sequences_scores)
      sequences_scores[index] = item.score;
  }
}

__global__ void BeamSearchScorer_Process(BeamScorerState& state_cpu,
                                         BeamScorerState& state,
                                         const int32_t* sequences_buffer,
                                         int sequence_length,
                                         BeamHypotheses* beam_hyps_,
                                         float* next_beam_scores_,
                                         int32_t* next_beam_tokens_,
                                         int32_t* next_beam_indices_,
                                         int32_t* hypothesis_buffer_,
                                         const float* next_scores,
                                         const int32_t* next_tokens,
                                         const int32_t* next_indices) {
  // Sequences shape is (batch_size * num_beams, total_sequence_length)
  // It contains word ID of whole sequence generated so far.
  // It is different from subgraph input_ids, which only need one word when past state is not empty.

  int batch = threadIdx.x;
  int batch_start = batch * state.num_beams_;

  cuda::BeamHypotheses& beam_hyp = beam_hyps_[batch];
  if (!beam_hyp.done_) {
    // Next tokens for this sentence.
    size_t beam_idx = 0;
    size_t top_k = 2 * state.num_beams_;
    for (size_t j = 0; j < top_k; j++) {
      int32_t next_token = next_tokens[batch * top_k + j];
      float next_score = next_scores[batch * top_k + j];
      int32_t next_index = next_indices[batch * top_k + j];

      int batch_beam_idx = batch_start + next_index;
      // Add to generated hypotheses if end of sentence.
      if ((state.eos_token_id_ >= 0) && (next_token == state.eos_token_id_)) {
        bool is_beam_token_worse_than_top_num_beams = (j >= state.num_beams_);
        if (is_beam_token_worse_than_top_num_beams) {
          continue;
        }

        // Clone the sequence and append to buffer.
        const int32_t* src = sequences_buffer + batch_beam_idx * state.max_length_;
        auto clone = hypothesis_buffer_ + atomicAdd(&state.hypothesis_buffer_used_, sequence_length);

        for (unsigned i = 0; i < sequence_length; i++)
          clone[i] = src[i];
        beam_hyp.Add(clone, sequence_length, next_score);
      } else {
        // Add next predicted token since it is not eos_token.
        next_beam_scores_[batch_start + beam_idx] = next_score;
        next_beam_tokens_[batch_start + beam_idx] = next_token;
        next_beam_indices_[batch_start + beam_idx] = batch_beam_idx;
        ++beam_idx;
      }

      // Once the beam for next step is full, don't add more tokens to it.
      if (beam_idx == state.num_beams_)
        break;
    }

    //  Check if we are done so that we can save a pad step if all(done)
    if (beam_hyp.beams_used_ == state.num_beams_) {
      if (state.early_stopping_ || !beam_hyp.CanImprove(*std::max_element(next_scores + batch_start, next_scores + batch_start + top_k), sequence_length)) {
        beam_hyp.done_ = true;
        if (atomicAdd(&state.not_done_count_, -1) == 1)
          state_cpu.not_done_count_ = 0;  // Update the CPU side
      }
    }
  } else {
    // Pad the batch.
    for (size_t beam_idx = 0; beam_idx < state.num_beams_; beam_idx++) {
      next_beam_scores_[batch_start + beam_idx] = 0.0f;
      next_beam_tokens_[batch_start + beam_idx] = state.pad_token_id_;
      next_beam_indices_[batch_start + beam_idx] = 0;
    }
  }
}

void LaunchBeamSearchScorer_Process(BeamScorerState& state_cpu,
                                    BeamScorerState& state,
                                    gsl::span<const int32_t> sequences,
                                    int sequence_length,
                                    gsl::span<BeamHypotheses> beam_hyps,
                                    gsl::span<float> next_beam_scores,
                                    gsl::span<int32_t> next_beam_tokens,
                                    gsl::span<int32_t> next_beam_indices,
                                    gsl::span<int32_t> hypothesis_buffer,
                                    gsl::span<const float> next_scores,
                                    gsl::span<const int32_t> next_tokens,
                                    gsl::span<const int32_t> next_indices,
                                    hipStream_t stream) {
  BeamSearchScorer_Process<<<1, state_cpu.batch_size_, 0, stream>>>(state_cpu,
                                                                    state,
                                                                    sequences.data(),
                                                                    sequence_length,
                                                                    beam_hyps.data(),
                                                                    next_beam_scores.data(),
                                                                    next_beam_tokens.data(),
                                                                    next_beam_indices.data(),
                                                                    hypothesis_buffer.data(),
                                                                    next_scores.data(),
                                                                    next_tokens.data(),
                                                                    next_indices.data());
}

__global__ void BeamSearchScorer_AppendNextTokenToSequences1(BeamScorerState& state,
                                                             int batch_beam_size,
                                                             const int32_t* sequences_buffer,
                                                             int32_t* next_sequences,
                                                             int sequence_length,
                                                             int32_t* next_beam_indices_) {
  int beam_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (beam_idx >= batch_beam_size)
    return;
  int sequence_index = threadIdx.y + blockIdx.y * blockDim.y;
  if (sequence_index >= sequence_length)
    return;

  int beam_index = next_beam_indices_[beam_idx];
  next_sequences[beam_idx * state.max_length_ + sequence_index] = sequences_buffer[beam_index * state.max_length_ + sequence_index];
}

__global__ void BeamSearchScorer_AppendNextTokenToSequences2(BeamScorerState& state,
                                                             int32_t* next_sequences,
                                                             int sequence_length,
                                                             const int32_t* next_beam_tokens_) {
  int beam_idx = threadIdx.x;
  next_sequences[beam_idx * state.max_length_ + sequence_length] = next_beam_tokens_[beam_idx];
}

void LaunchBeamSearchScorer_AppendNextTokenToSequences(BeamScorerState& state_cpu,
                                                       BeamScorerState& state,
                                                       gsl::span<const int32_t> sequences,
                                                       gsl::span<int32_t> next_sequences,
                                                       int sequence_length,
                                                       gsl::span<int32_t> next_beam_tokens,
                                                       gsl::span<int32_t> next_beam_indices,
                                                       hipStream_t stream) {
  const int max_threads = 512;
  int batch_beam_size = state_cpu.batch_size_ * state_cpu.num_beams_;
  dim3 block_size;
  dim3 grid_size;
  if (batch_beam_size * sequence_length <= max_threads) {  // Can fit into a single thread block
    block_size.x = batch_beam_size;
    block_size.y = sequence_length;
  } else {
    if (sequence_length <= max_threads)  // Sequence length fits into thread block, but batch_beam_size does not, so chunk it
    {
      block_size.x = max_threads / sequence_length;
      block_size.y = sequence_length;

      grid_size.x = (batch_beam_size + block_size.x - 1) / block_size.x;
    } else {  // Exceed max_threads in every dimension, so divide into max_thread chunks
      block_size.x = 1;
      block_size.y = max_threads;

      grid_size.x = batch_beam_size;
      grid_size.y = (sequence_length + block_size.y - 1) / block_size.y;
    }
  }
  BeamSearchScorer_AppendNextTokenToSequences1<<<grid_size, block_size, 0, stream>>>(state,
                                                                                     batch_beam_size,
                                                                                     sequences.data(),
                                                                                     next_sequences.data(),
                                                                                     sequence_length,
                                                                                     next_beam_indices.data());

  BeamSearchScorer_AppendNextTokenToSequences2<<<1, batch_beam_size, 0, stream>>>(state,
                                                                                  next_sequences.data(),
                                                                                  sequence_length,
                                                                                  next_beam_tokens.data());
}

__global__ void BeamSearchScorer_Finalize(BeamScorerState& state,
                                          const int32_t* sequences_buffer,
                                          int sequence_length,
                                          BeamHypotheses* beam_hyps_,
                                          const float* final_beam_scores,
                                          int32_t* output,
                                          float* sequence_scores) {
  int batch_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (batch_index >= state.batch_size_)
    return;

  // Finalize all open beam hypotheses and add to generated hypotheses.
  cuda::BeamHypotheses& beam_hyp = beam_hyps_[batch_index];
  if (!beam_hyp.done_) {
    for (size_t beam_index = 0; beam_index < state.num_beams_; beam_index++) {
      size_t batch_beam_index = batch_index * state.num_beams_ + beam_index;
      float final_score = final_beam_scores[batch_beam_index];
      auto final_tokens = sequences_buffer + batch_beam_index * state.max_length_;
      beam_hyp.Add(final_tokens, sequence_length, final_score);
    }
  }

  // Select the best hypotheses according to number of sequences to return.
  auto batch_output = output + batch_index * state.num_return_sequences_ * state.max_length_;

  beam_hyp.Output(
      state.num_return_sequences_,
      state.max_length_,
      state.pad_token_id_,
      batch_output,
      sequence_scores ? sequence_scores + batch_index * state.num_return_sequences_ : nullptr);
}

void LaunchBeamSearchScorer_Finalize(int batch_size,
                                     BeamScorerState& state,
                                     gsl::span<const int32_t> sequences,
                                     int sequence_length,
                                     gsl::span<BeamHypotheses> beam_hyps,
                                     gsl::span<const float> final_beam_scores,
                                     gsl::span<int32_t> output,
                                     gsl::span<float> sequence_scores,
                                     hipStream_t stream) {
  BeamSearchScorer_Finalize<<<1, batch_size, 0, stream>>>(state,
                                                          sequences.data(),
                                                          sequence_length,
                                                          beam_hyps.data(),
                                                          final_beam_scores.data(),
                                                          output.data(),
                                                          sequence_scores.data());
}

__global__ void AddProbsKernel(float* log_probs,
                               float* cum_log_probs,
                               const int vocab_size,
                               const int total_elements) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int batch_beam_index = index / vocab_size;

  if (index < total_elements)
    log_probs[index] += cum_log_probs[batch_beam_index];
}

template <typename T>
void LaunchAddProbsKernel(T* log_probs,
                          T* cum_log_probs,
                          const int batch_size,
                          const int num_beams,
                          const int vocab_size,
                          hipStream_t stream) {
  int total_elements = batch_size * num_beams * vocab_size;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  AddProbsKernel<<<gridSize, blockSize, 0, stream>>>(log_probs, cum_log_probs, vocab_size, total_elements);
}

template void LaunchAddProbsKernel(
    float* log_probs,
    float* cum_log_probs,
    const int batch_size,
    const int num_beams,
    const int vocab_size,
    hipStream_t stream);

template <typename T>
__global__ void UpdateGptInputsKernel(const T* old_mask_data,
                                      T* mask_data,
                                      int32_t* next_positions,
                                      int batch_beam_size,
                                      int current_length) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < batch_beam_size * current_length) {
    // Update attention mask.
    int i = index / current_length;
    int j = index % current_length;
    mask_data[index] = (j < current_length - 1) ? old_mask_data[i * (current_length - 1) + j] : static_cast<T>(1);

    if (next_positions != nullptr) {
      // Update sequence length (or next positions).
      if (index < batch_beam_size) {
        next_positions[index]++;
      }
    }
  }
}

void LaunchUpdateGptKernel(const int32_t* old_mask_data,
                           int32_t* mask_data,
                           int32_t* next_positions,
                           int batch_beam_size,
                           int current_length,
                           hipStream_t stream) {
  assert(current_length > 0);
  int total_elements = batch_beam_size * current_length;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  UpdateGptInputsKernel<int32_t><<<gridSize, blockSize, 0, stream>>>(
      old_mask_data, mask_data, next_positions, batch_beam_size, current_length);
}

template <typename T>
void GetTempStorageSize(const T* d_keys_in,
                        const int* d_values_in,
                        int* d_offsets,
                        int num_items,
                        int num_segments,
                        hipStream_t stream,
                        bool is_descending,
                        size_t& temp_storage_bytes) {
  if (is_descending) {
    CUDA_CALL_THROW(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr,
                                                                       temp_storage_bytes,
                                                                       d_keys_in,
                                                                       (T*)nullptr,
                                                                       d_values_in,
                                                                       (int*)nullptr,
                                                                       num_items,
                                                                       num_segments,
                                                                       d_offsets,
                                                                       d_offsets + 1,
                                                                       0,
                                                                       sizeof(T) * 8,
                                                                       stream));
  } else {
    CUDA_CALL_THROW(hipcub::DeviceSegmentedRadixSort::SortPairs(nullptr,
                                                             temp_storage_bytes,
                                                             d_keys_in,
                                                             (T*)nullptr,
                                                             d_values_in,
                                                             (int*)nullptr,
                                                             num_items,
                                                             num_segments,
                                                             d_offsets,
                                                             d_offsets + 1,
                                                             0,
                                                             sizeof(T) * 8,
                                                             stream));
  }
}

template void GetTempStorageSize(
    const float* d_keys_in,
    const int* d_values_in,
    int* d_offsets,
    int num_items,
    int num_segments,
    hipStream_t stream,
    bool is_descending,
    size_t& temp_storage_bytes);

template void GetTempStorageSize(
    const half* d_keys_in,
    const int* d_values_in,
    int* d_offsets,
    int num_items,
    int num_segments,
    hipStream_t stream,
    bool is_descending,
    size_t& temp_storage_bytes);

// TODO: merge to one kernel
__global__ void SetupParamsKernel(int* d_values_in,
                                  int* d_offsets,
                                  int batch_size,
                                  int vocab_size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total_elements = batch_size * vocab_size;
  if (index < total_elements) {
    d_values_in[index] = index % vocab_size;
  }
  if (index < batch_size + 1) {
    d_offsets[index] = index * vocab_size;
  }
}

void LaunchSetupParamsKernel(int* d_values_in,
                             int* d_offsets,
                             int batch_size,
                             int vocab_size,
                             hipStream_t stream) {
  int total_elements = batch_size * vocab_size;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  SetupParamsKernel<<<gridSize, blockSize, 0, stream>>>(d_values_in,
                                                        d_offsets,
                                                        batch_size,
                                                        vocab_size);
}

template <typename T>
void LaunchSortPairs(void* d_temp_storage,
                     size_t temp_storage_bytes,
                     const T* d_keys_in,
                     T* d_keys_out,
                     const int* d_values_in,
                     int* d_values_out,
                     int num_items,
                     int num_segments,
                     int* d_offsets,
                     hipStream_t stream,
                     bool is_descending) {
  if (is_descending) {
    CUDA_CALL_THROW(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(d_temp_storage,
                                                                       temp_storage_bytes,
                                                                       d_keys_in,
                                                                       d_keys_out,
                                                                       d_values_in,
                                                                       d_values_out,
                                                                       num_items,
                                                                       num_segments,
                                                                       d_offsets,
                                                                       d_offsets + 1,
                                                                       0,
                                                                       sizeof(T) * 8,
                                                                       stream));
  } else {
    CUDA_CALL_THROW(hipcub::DeviceSegmentedRadixSort::SortPairs(d_temp_storage,
                                                             temp_storage_bytes,
                                                             d_keys_in,
                                                             d_keys_out,
                                                             d_values_in,
                                                             d_values_out,
                                                             num_items,
                                                             num_segments,
                                                             d_offsets,
                                                             d_offsets + 1,
                                                             0,
                                                             sizeof(T) * 8,
                                                             stream));
  }
}

template void LaunchSortPairs(void* d_temp_storage,
                              size_t temp_storage_bytes,
                              const float* d_keys_in,
                              float* d_keys_out,
                              const int* d_values_in,
                              int* d_values_out,
                              int num_items,
                              int num_segments,
                              int* d_offsets,
                              hipStream_t stream,
                              bool is_descending);

template void LaunchSortPairs(void* d_temp_storage,
                              size_t temp_storage_bytes,
                              const half* d_keys_in,
                              half* d_keys_out,
                              const int* d_values_in,
                              int* d_values_out,
                              int num_items,
                              int num_segments,
                              int* d_offsets,
                              hipStream_t stream,
                              bool is_descending);

// A stateful callback functor that maintains a running prefix to be applied
// during consecutive scan operations.
struct BlockPrefixCallbackOp {
  float running_total;  // running prefix

  __device__ BlockPrefixCallbackOp(float running_total) : running_total(running_total) {}
  // Callback operator to be entered by the first warp of threads in the block.
  // Thread-0 is responsible for returning a value for seeding the block-wide scan.
  __device__ float operator()(float block_aggregate) {
    float old_prefix = running_total;
    running_total += block_aggregate;
    return old_prefix;
  }
};

template <typename T, int kBlockSize>
__global__ void FilterLogitsKernelCustom(float* d_sorted_logits_in,
                                         const int* d_sorted_indices,
                                         T* d_logits_in_out,
                                         float top_p_threshold,
                                         float filter_value,
                                         int batch_size,
                                         int vocab_size) {
  int vocab_idx = threadIdx.x;
  int batch_id = blockIdx.x;
  int offset = batch_id * vocab_size;

  typedef hipcub::BlockScan<float, kBlockSize> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;
  BlockPrefixCallbackOp prefix_op(0);

  for (int idx = vocab_idx; idx < vocab_size; idx += kBlockSize) {
    float sum = d_sorted_logits_in[offset + idx];
    BlockScan(temp_storage).ExclusiveSum(sum, sum, prefix_op);

    __syncthreads();
    if (sum >= top_p_threshold) {
      int original_index = offset + d_sorted_indices[offset + idx];
      d_logits_in_out[original_index] = (T)filter_value;
    }
  }
}

template <typename T, int kBlockSize>
__global__ void FilterLogitsKernel(float* d_sorted_logits_in,
                                   const int* d_sorted_indices,
                                   T* d_logits_in_out,
                                   float top_p_threshold,
                                   float filter_value,
                                   int min_tokens_to_keep,
                                   int batch_size,
                                   int vocab_size) {
  int vocab_idx = threadIdx.x;
  int batch_id = blockIdx.x;
  int offset = batch_id * vocab_size;

  typedef hipcub::BlockScan<float, kBlockSize> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;
  BlockPrefixCallbackOp prefix_op(0);

  for (int idx = vocab_idx; idx < vocab_size; idx += kBlockSize) {
    float sum = d_sorted_logits_in[offset + idx];
    BlockScan(temp_storage).InclusiveSum(sum, sum, prefix_op);

    __syncthreads();

    if (sum <= top_p_threshold) {
      if (idx + min_tokens_to_keep < vocab_size) {
        int original_index = offset + d_sorted_indices[offset + idx];
        d_logits_in_out[original_index] = (T)filter_value;
      }
    }
  }
}

template <typename T>
void LaunchFilterLogitsKernel(float* d_sorted_logits_in,
                              const int* d_sorted_indices,
                              T* d_logits_in_out,
                              float top_p,
                              float filter_value,
                              int min_tokens_to_keep,
                              int batch_size,
                              int vocab_size,
                              hipStream_t stream,
                              bool is_descending) {
  constexpr int kBlockSize = 256;

  if (is_descending) {
    FilterLogitsKernelCustom<T, kBlockSize><<<batch_size, kBlockSize, 0, stream>>>(d_sorted_logits_in,
                                                                                   d_sorted_indices,
                                                                                   d_logits_in_out,
                                                                                   top_p,
                                                                                   filter_value,
                                                                                   batch_size,
                                                                                   vocab_size);
  } else {
    FilterLogitsKernel<T, kBlockSize><<<batch_size, kBlockSize, 0, stream>>>(d_sorted_logits_in,
                                                                             d_sorted_indices,
                                                                             d_logits_in_out,
                                                                             1 - top_p,
                                                                             filter_value,
                                                                             min_tokens_to_keep,
                                                                             batch_size,
                                                                             vocab_size);
  }
}

template void LaunchFilterLogitsKernel(float* d_sorted_logits_in,
                                       const int* d_sorted_indices,
                                       float* d_logits_in_out,
                                       float top_p,
                                       float filter_value,
                                       int min_tokens_to_keep,
                                       int batch_size,
                                       int vocab_size,
                                       hipStream_t stream,
                                       bool is_descending);

template void LaunchFilterLogitsKernel(float* d_sorted_logits_in,
                                       const int* d_sorted_indices,
                                       half* d_logits_in_out,
                                       float top_p,
                                       float filter_value,
                                       int min_tokens_to_keep,
                                       int batch_size,
                                       int vocab_size,
                                       hipStream_t stream,
                                       bool is_descending);

// Ref: https://github.com/pytorch/pytorch/blob/release/1.13/aten/src/ATen/native/cuda/MultinomialKernel.cu
template <typename scalar_t, typename accscalar_t>
__global__ void sampleMultinomialOnce(int32_t* dest,
                                      int distributions,
                                      int categories,
                                      scalar_t* sampled,
                                      scalar_t* dist,
                                      int stride_dist,        // dist->stride(0)
                                      int stride_categories,  // dist->stride(1)
                                      int* d_presence_mask) {
  extern __shared__ unsigned char my_smem[];
  __shared__ bool found;
  __shared__ unsigned foundPos;
  accscalar_t* smem = reinterpret_cast<accscalar_t*>(my_smem);
  accscalar_t accZero = static_cast<accscalar_t>(0);
  scalar_t zero = static_cast<scalar_t>(0);
  for (int curDist = blockIdx.x;
       curDist < distributions; curDist += gridDim.x) {
    // Assume sum = 1 in Top P sampling as the input is softmaxed.
    accscalar_t sum = 1;

    // Broadcast sum and sample value
    if (threadIdx.x == 0) {
      // Make sure the sum of our distribution didn't overflow
      // CUDA_KERNEL_ASSERT(!_isinf(val));
      // CUDA_KERNEL_ASSERT(sum > accZero);
      foundPos = 0;
      smem[0] = sum;
      smem[1] = sampled[curDist];
    }
    __syncthreads();
    sum = smem[0];
    scalar_t sample = static_cast<scalar_t>(smem[1]);
    __syncthreads();
    if (sum == accZero) {
      // Choose the first element
      if (threadIdx.x == 0) {
        dest[curDist] = 0;
      }
      continue;
    }
    int chunks = (categories + (int)blockDim.x - 1) / blockDim.x;
    accscalar_t prevHighProb = accZero;
    found = false;
    for (int chunk = 0; chunk < chunks && !found; ++chunk) {
      // All threads in bounds load a value
      int cat = chunk * blockDim.x + threadIdx.x;
      accscalar_t dist_val = cat < categories ? static_cast<accscalar_t>(dist[curDist * stride_dist + cat * stride_categories]) / sum : accZero;
      smem[threadIdx.x] = dist_val;
      __syncthreads();
      // Perform an inclusive prefix sum of the shared memory contents
      for (int offset = 1; offset < blockDim.x; offset *= 2) {
        accscalar_t val = accZero;
        if (threadIdx.x >= offset) {
          val = smem[threadIdx.x - offset] + smem[threadIdx.x];
        }
        __syncthreads();
        if (threadIdx.x >= offset) {
          smem[threadIdx.x] = val;
        }
        __syncthreads();
      }
      // Each thread will check to see if the sample falls in its bucket
      scalar_t curBucket =
          static_cast<scalar_t>(smem[threadIdx.x] + prevHighProb);
      scalar_t prevBucket = static_cast<scalar_t>(
          threadIdx.x == 0 ? prevHighProb
                           : smem[threadIdx.x - 1] + prevHighProb);
      bool inBucket =
          (cat < categories) &&
          (!(sample >= curBucket) &&
           (sample >= prevBucket) &&
           (dist_val > zero));
      if (inBucket) {
        // We're done; we have the sample
        // Torch indices are 1-based
        atomicMax(&foundPos, cat);
        found = true;
      }
      // Store the previous scan's high value for future use
      prevHighProb = prevHighProb + smem[blockDim.x - 1];
      __syncthreads();
    }
    if (threadIdx.x == 0) {
      if (found) {
        dest[curDist] = foundPos;
      } else {
        // This should address a rare bug where we don't select a valid index. This likely occurs when
        // due to floating point arithmetic rounding errors, our cumulative sum does not add up to 1, but
        // and our uniform sample is greater than this value. In this case we likely have unitialized memory
        // in dest[curDist]. So basically we will loop through the distribution and pick the largest index
        // where the distribution is non-zero. This is obviously terribly inefficient, but due to the
        // rarity in which this occurs, this should not be an issue.
        for (int cat = categories - 1; cat >= 0; --cat) {
          if (dist[curDist * stride_dist + cat * stride_categories] > zero) {
            dest[curDist] = cat;
            break;
          }
        }
      }
    }
  }

  // update presence mask
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= distributions * categories) {
    return;
  }
  int dist_idx = index / categories;
  int cat_idx = index % categories;
  if (dest[dist_idx] == cat_idx) {
    d_presence_mask[index] = 1;
  }
}

// Only support n_sample = 1
void TorchMultinomialKernelLauncher(float* d_input,
                                    float* d_sampled,
                                    int32_t* d_output,
                                    int batch_size,
                                    int vocab_size,
                                    int* d_presence_mask,
                                    hipStream_t stream) {
  // Store the props in class variables
  int device;
  CUDA_CALL_THROW(hipGetDevice(&device));
  hipDeviceProp_t props;
  CUDA_CALL_THROW(hipGetDeviceProperties(&props, device));

  int numSM = props.multiProcessorCount;
  int maxThreads = props.maxThreadsPerBlock;
  int warp_size = props.warpSize;
  int requiredWarps = (vocab_size + warp_size - 1) / warp_size;
  int requiredThreads = std::min(maxThreads, requiredWarps * warp_size);
  int requiredShared = requiredThreads * sizeof(float);

  dim3 block(requiredThreads);
  dim3 grid(std::min(batch_size, numSM * 4));

  sampleMultinomialOnce<float, float>
      <<<grid, block, requiredShared, stream>>>(d_output,
                                                batch_size,
                                                vocab_size,
                                                d_sampled,
                                                d_input,
                                                vocab_size,
                                                1,
                                                d_presence_mask);
}

__global__ void UpdateDecoderMaskedMultiHeadAttentionCacheIndirectionKernel(int32_t* tgt_indir_cache,
                                                                            const int32_t* src_indir_cache,
                                                                            const int32_t* beam_ids,
                                                                            int batch_size,
                                                                            int beam_width,
                                                                            int input_seq_length,
                                                                            int max_seq_length,
                                                                            int current_length) {
  int time_step = threadIdx.x + blockIdx.x * blockDim.x;
  int bb_id = threadIdx.y + blockIdx.y * blockDim.y;
  const int batch_id = bb_id / beam_width;
  const int beam_id = bb_id % beam_width;

  if (bb_id >= beam_width * batch_size || time_step >= current_length) {
    return;
  }

  const int src_beam = beam_ids[batch_id * beam_width + beam_id] % beam_width;

  const int tgt_offset = batch_id * beam_width * max_seq_length + beam_id * max_seq_length + time_step;

  if (time_step < input_seq_length) {
    // For time steps that correspond to the input sequence,
    // the beam that it comes from is always 0.
    tgt_indir_cache[tgt_offset] = static_cast<int32_t>(0);
  } else if (time_step == (current_length - 1)) {
    // For the final (newly generated) time step,
    // the beam that it comes from is always the beam that we
    // are currently processing (i.e.) from this point on, these time-steps
    // form the new beams.
    tgt_indir_cache[tgt_offset] = static_cast<int32_t>(beam_id);
  } else {
    // For all other time-steps, we look up the source indirection, to
    // see which beam it came from based on the `src_beam`.
    const int src_offset = batch_id * beam_width * max_seq_length + src_beam * max_seq_length + time_step;
    tgt_indir_cache[tgt_offset] = src_indir_cache[src_offset];
  }
}

void UpdateDecoderMaskedMultiHeadAttentionCacheIndirection(int32_t* tgt_indir_cache,
                                                           const int32_t* src_indir_cache,
                                                           const int32_t* beam_ids,
                                                           int batch_size,
                                                           int beam_width,
                                                           int input_seq_length,
                                                           int max_seq_length,
                                                           int current_length,
                                                           hipStream_t stream) {
  const dim3 block(32);
  const dim3 grid((current_length + block.x - 1) / block.x, batch_size * beam_width);
  UpdateDecoderMaskedMultiHeadAttentionCacheIndirectionKernel<<<grid, block, 0, stream>>>(tgt_indir_cache,
                                                                                          src_indir_cache,
                                                                                          beam_ids,
                                                                                          batch_size,
                                                                                          beam_width,
                                                                                          input_seq_length,
                                                                                          max_seq_length,
                                                                                          current_length);
}

#ifndef USE_ROCM
namespace {
template <typename T, size_t size>
struct TypeMapper : public V_vec_m_<T, size> {};

template <>
struct TypeMapper<int32_t, 2> {
  using Type = uint2;
};

template <>
struct TypeMapper<int32_t, 4> {
  using Type = uint4;
};
}  // namespace
#endif

template <typename T>
__global__ void KeyCacheExpansionKernel(const T* input,
                                        T* output,
                                        int beam_width,
                                        int max_seq_length,
                                        int head_size) {
  const int num_heads = gridDim.y;
  const int sequence_length = gridDim.z;

  const int bbid = blockIdx.x;
  const int batch_id = bbid / beam_width;
  const int head_id = blockIdx.y;
  const int s = blockIdx.z;
  const int tidx = threadIdx.x;

  const int input_offset = ((batch_id * num_heads + head_id) * sequence_length + s) * head_size + tidx;
  const int output_offset = ((bbid * num_heads + head_id) * max_seq_length + s) * head_size + tidx;

  if (tidx < head_size) {
    output[output_offset] = input[input_offset];
  }
}

template <typename T>
void KeyCacheExpansionKernelLauncher(const T* key_cache,
                                     T* key_cache_expanded,
                                     int batch_size,
                                     int beam_width,
                                     int num_heads,
                                     int sequence_length,
                                     int max_seq_length,
                                     int head_size,
                                     hipStream_t stream) {
  const dim3 grid(batch_size * beam_width, num_heads, sequence_length);

  int equiv_head_size = (head_size & 1) == 0 ? (head_size >> 1) : head_size;
  equiv_head_size = (equiv_head_size & 1) == 0 ? (equiv_head_size >> 1) : equiv_head_size;

  // Here we know head_size is smaller than max_thread_num_per_block
  int tpb = std::max(GPU_WARP_SIZE_HOST, equiv_head_size);

  // round up tpb to power of 2
  --tpb;
  tpb |= (tpb >> 1);
  tpb |= (tpb >> 2);
  tpb |= (tpb >> 4);
  tpb |= (tpb >> 8);
  tpb |= (tpb >> 16);
  tpb++;

#ifndef USE_ROCM
  if ((head_size % 4) == 0) {
    using vec_type = typename TypeMapper<T, 4>::Type;
    const dim3 block(tpb);
    KeyCacheExpansionKernel<<<grid, block, 0, stream>>>(reinterpret_cast<const vec_type*>(key_cache),
                                                        reinterpret_cast<vec_type*>(key_cache_expanded),
                                                        beam_width,
                                                        max_seq_length,
                                                        equiv_head_size);
  } else if ((head_size & 1) == 0) {
    using vec_type = typename TypeMapper<T, 2>::Type;
    const dim3 block(tpb);
    KeyCacheExpansionKernel<<<grid, block, 0, stream>>>(reinterpret_cast<const vec_type*>(key_cache),
                                                        reinterpret_cast<vec_type*>(key_cache_expanded),
                                                        beam_width,
                                                        max_seq_length,
                                                        equiv_head_size);
  } else {
#endif
    const dim3 block(tpb);
    KeyCacheExpansionKernel<<<grid, block, 0, stream>>>(key_cache,
                                                        key_cache_expanded,
                                                        beam_width,
                                                        max_seq_length,
                                                        head_size);
#ifndef USE_ROCM
  }
#endif
}

template void KeyCacheExpansionKernelLauncher(const float* key_cache,
                                              float* key_cache_expanded,
                                              int batch_size,
                                              int beam_width,
                                              int num_heads,
                                              int sequence_length,
                                              int max_seq_length,
                                              int head_size,
                                              hipStream_t stream);

template void KeyCacheExpansionKernelLauncher(const half* key_cache,
                                              half* key_cache_expanded,
                                              int batch_size,
                                              int beam_width,
                                              int num_heads,
                                              int sequence_length,
                                              int max_seq_length,
                                              int head_size,
                                              hipStream_t stream);

template void KeyCacheExpansionKernelLauncher(const int32_t* key_cache,
                                              int32_t* key_cache_expanded,
                                              int batch_size,
                                              int beam_width,
                                              int num_heads,
                                              int sequence_length,
                                              int max_seq_length,
                                              int head_size,
                                              hipStream_t stream);

template <typename T>
__global__ void BufferExpansionKernel(const T* input,
                                      T* output,
                                      int chunk_size) {
  const int batch_id = blockIdx.x;
  const int beam_id = blockIdx.y;
  const int tidx = threadIdx.x;
  const int beam_size = gridDim.y;
  const int idx = blockIdx.z * blockDim.x + tidx;

  const int input_offset = batch_id * chunk_size + idx;
  const int output_offset = batch_id * beam_size * chunk_size + beam_id * chunk_size + idx;

  if (idx < chunk_size) {
    output[output_offset] = input[input_offset];
  }
}

template <typename T>
void BufferExpansionKernelLauncher(const T* input,
                                   T* output,
                                   int batch_size,
                                   int beam_width,
                                   int chunk_size,
                                   hipStream_t stream) {
  const dim3 block(128);

#ifndef USE_ROCM
  if ((chunk_size % 4) == 0) {
    using vec_type = typename TypeMapper<T, 4>::Type;
    const dim3 grid(batch_size, beam_width, (chunk_size / 4 + block.x - 1) / block.x);
    BufferExpansionKernel<<<grid, block, 0, stream>>>(reinterpret_cast<const vec_type*>(input),
                                                      reinterpret_cast<vec_type*>(output),
                                                      chunk_size / 4);
  } else if ((chunk_size & 1) == 0) {
    using vec_type = typename TypeMapper<T, 2>::Type;
    const dim3 grid(batch_size, beam_width, (chunk_size / 2 + block.x - 1) / block.x);
    BufferExpansionKernel<<<grid, block, 0, stream>>>(reinterpret_cast<const vec_type*>(input),
                                                      reinterpret_cast<vec_type*>(output),
                                                      chunk_size / 2);
  } else {
#endif
    const dim3 grid(batch_size, beam_width, (chunk_size + block.x - 1) / block.x);
    BufferExpansionKernel<<<grid, block, 0, stream>>>(input,
                                                      output,
                                                      chunk_size);
#ifndef USE_ROCM
  }
#endif
}

template void BufferExpansionKernelLauncher(const float* input,
                                            float* output,
                                            int batch_size,
                                            int beam_width,
                                            int chunk_size,
                                            hipStream_t stream);

template void BufferExpansionKernelLauncher(const half* input,
                                            half* output,
                                            int batch_size,
                                            int beam_width,
                                            int chunk_size,
                                            hipStream_t stream);

template void BufferExpansionKernelLauncher(const int32_t* input,
                                            int32_t* output,
                                            int batch_size,
                                            int beam_width,
                                            int chunk_size,
                                            hipStream_t stream);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
