#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "cub/util_type.cuh"
#include <hipcub/hipcub.hpp>
#include <cub/device/device_segmented_radix_sort.cuh>
#include "contrib_ops/cuda/transformers/generation_cuda_impl.h"

namespace onnxruntime {
namespace contrib {
namespace cuda {
__global__ void InitKernel(float* beam_scores,
                           int num_beams,
                           int total_elements) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < total_elements) {
    int beam_index = index % num_beams;
    beam_scores[index] = beam_index > 0 ? static_cast<float>(-1e9) : 0.0f;
  }
}

void LaunchInitKernel(
    float* beam_scores,
    int batch_size,
    int num_beams,
    hipStream_t stream) {
  int total_elements = batch_size * num_beams;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  InitKernel<<<gridSize, blockSize, 0, stream>>>(beam_scores, num_beams, total_elements);
}

__global__ void NextTokenKernel(const int64_t* next_token_indices,
                                int32_t* next_indices,
                                int32_t* next_tokens,
                                int vocab_size,
                                int total_elements) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < total_elements) {
    next_indices[index] = next_token_indices[index] / vocab_size;
    next_tokens[index] = next_token_indices[index] % vocab_size;
  }
}

void LaunchNextTokenKernel(const int64_t* next_token_indices,
                           int32_t* next_indices,
                           int32_t* next_tokens,
                           int batch_size,
                           int top_k,
                           int vocab_size,
                           hipStream_t stream) {
  int total_elements = batch_size * top_k;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  NextTokenKernel<<<gridSize, blockSize, 0, stream>>>(next_token_indices,
                                                      next_indices,
                                                      next_tokens,
                                                      vocab_size,
                                                      total_elements);
}

template <typename T>
__global__ void LogitsProcessKernel(
    T* next_token_scores,
    const int* vocab_mask,
    const int* prefix_vocab_mask,
    const int* presence_mask,
    float presence_penalty,
    float temperature,
    int num_beams,
    int vocab_size,
    int padded_vocab_size,
    int total_elements,
    int demote_token_id,
    int32_t* sequences,
    int max_sequence_length,
    int current_sequence_length,
    float repetition_penalty,
    int no_repeat_ngram_size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < total_elements) {
    int batch_beam_index = index / padded_vocab_size;
    int word_id = index % padded_vocab_size;

    if (word_id >= vocab_size) {
      // Set any value within the padding region to the lowest value so that it isn't picked
      next_token_scores[index] = hipcub::FpLimits<T>::Lowest();
    } else {
      // RepetitionPenaltyLogitsProcessor
      if (repetition_penalty != 1.0f) {
        int32_t* current_sequence = sequences + batch_beam_index * max_sequence_length;
        bool found = false;
        for (int i = 0; i < current_sequence_length; i++) {
          if (current_sequence[i] == word_id) {
            found = true;
            break;
          }
        }
        if (found) {
          float score = (float)next_token_scores[index];
          next_token_scores[index] = (T)(score < 0 ? score * repetition_penalty : score / repetition_penalty);
        }
      }

      // NoRepeatNGramLogitsProcessor
      if (no_repeat_ngram_size > 0 && current_sequence_length >= no_repeat_ngram_size) {
        int32_t* current_sequence = sequences + batch_beam_index * max_sequence_length;
        bool found = false;
        for (int i = no_repeat_ngram_size - 1; i < current_sequence_length; i++) {
          if (current_sequence[i] == word_id) {  // last token of n-gram matched
            found = true;
            for (int j = 0; j < no_repeat_ngram_size - 1; j++) {  // match the remaining N-1 tokens
              if (current_sequence[i - j - 1] != current_sequence[current_sequence_length - 1 - j]) {
                found = false;
                break;
              }
            }
            if (found) {
              break;
            }
          }
        }

        if (found) {
          next_token_scores[index] = hipcub::FpLimits<T>::Lowest();
          return;
        }
      }

      // VocabMaskLogitsProcessor
      if (vocab_mask != nullptr && vocab_mask[word_id] == 0) {
        next_token_scores[index] = hipcub::FpLimits<T>::Lowest();
        return;
      }

      // PrefixVocabMaskLogitsProcessor
      int batch_id = batch_beam_index / num_beams;
      if (prefix_vocab_mask != nullptr && prefix_vocab_mask[batch_id * vocab_size + word_id] == 0) {
        next_token_scores[index] = hipcub::FpLimits<T>::Lowest();
        return;
      }

      // MinLengthLogitsProcessor
      if (word_id == demote_token_id) {
        next_token_scores[index] = hipcub::FpLimits<T>::Lowest();
      }

      // PresencePenaltyLogitsProcessor
      if (presence_mask != nullptr && presence_mask[index] == 1) {
        float score = (float)next_token_scores[index] - presence_penalty;
        next_token_scores[index] = (T)score;
      }

      // TemperatureLogitsProcessor
      if (temperature != 1.0f) {
        float score = (float)(next_token_scores[index]);
        next_token_scores[index] = (T)(score / temperature);
      }
    }
  }
}

template <typename T>
void LaunchLogitsProcessKernel(
    T* next_token_scores,
    const int* vocab_mask,
    const int* prefix_vocab_mask,
    int* presence_mask,
    float presence_penalty,
    float temperature,
    int batch_size,
    int num_beams,
    int vocab_size,
    int padded_vocab_size,
    int demote_token_id,
    int32_t* sequences,
    int max_sequence_length,
    int current_sequence_length,
    float repetition_penalty,
    int no_repeat_ngram_size,
    hipStream_t stream) {
  int total_elements = batch_size * num_beams * padded_vocab_size;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  LogitsProcessKernel<T><<<gridSize, blockSize, 0, stream>>>(
      next_token_scores,
      vocab_mask,
      prefix_vocab_mask,
      presence_mask,
      presence_penalty,
      temperature,
      num_beams,
      vocab_size,
      padded_vocab_size,
      total_elements,
      demote_token_id,
      sequences,
      max_sequence_length,
      current_sequence_length,
      repetition_penalty,
      no_repeat_ngram_size);
}

// Instantiation
template void LaunchLogitsProcessKernel(
    float* next_token_scores,
    const int* vocab_mask,
    const int* prefix_vocab_mask,
    int* presence_mask,
    float presence_penalty,
    float temperature,
    int batch_size,
    int num_beams,
    int vocab_size,
    int padded_vocab_size,
    int demote_token_id,
    int32_t* sequences,
    int max_sequence_length,
    int current_sequence_length,
    float repetition_penalty,
    int no_repeat_ngram_size,
    hipStream_t stream);

template void LaunchLogitsProcessKernel(
    half* next_token_scores,
    const int* vocab_mask,
    const int* prefix_vocab_mask,
    int* presence_mask,
    float presence_penalty,
    float temperature,
    int batch_size,
    int num_beams,
    int vocab_size,
    int padded_vocab_size,
    int demote_token_id,
    int32_t* sequences,
    int max_sequence_length,
    int current_sequence_length,
    float repetition_penalty,
    int no_repeat_ngram_size,
    hipStream_t stream);

__global__ void AddProbsKernel(float* log_probs,
                               float* cum_log_probs,
                               const int vocab_size,
                               const int total_elements) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int batch_beam_index = index / vocab_size;

  if (index < total_elements)
    log_probs[index] += cum_log_probs[batch_beam_index];
}

template <typename T>
void LaunchAddProbsKernel(T* log_probs,
                          T* cum_log_probs,
                          const int batch_size,
                          const int num_beams,
                          const int vocab_size,
                          hipStream_t stream) {
  int total_elements = batch_size * num_beams * vocab_size;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  AddProbsKernel<<<gridSize, blockSize, 0, stream>>>(log_probs, cum_log_probs, vocab_size, total_elements);
}

template void LaunchAddProbsKernel(
    float* log_probs,
    float* cum_log_probs,
    const int batch_size,
    const int num_beams,
    const int vocab_size,
    hipStream_t stream);

template <typename T>
__global__ void UpdateGptInputsKernel(const T* old_mask_data,
                                      T* mask_data,
                                      int32_t* next_positions,
                                      int batch_beam_size,
                                      int current_length) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < batch_beam_size * current_length) {
    // Update attention mask.
    int i = index / current_length;
    int j = index % current_length;
    mask_data[index] = (j < current_length - 1) ? old_mask_data[i * (current_length - 1) + j] : static_cast<T>(1);

    if (next_positions != nullptr) {
      // Update sequence length (or next positions).
      if (index < batch_beam_size) {
        next_positions[index]++;
      }
    }
  }
}

void LaunchUpdateGptKernel(const int32_t* old_mask_data,
                           int32_t* mask_data,
                           int32_t* next_positions,
                           int batch_beam_size,
                           int current_length,
                           hipStream_t stream) {
  assert(current_length > 0);
  int total_elements = batch_beam_size * current_length;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  UpdateGptInputsKernel<int32_t><<<gridSize, blockSize, 0, stream>>>(
      old_mask_data, mask_data, next_positions, batch_beam_size, current_length);
}

template <typename T>
void GetTempStorageSize(const T* d_keys_in,
                        const int* d_values_in,
                        int* d_offsets,
                        int num_items,
                        int num_segments,
                        hipStream_t stream,
                        bool is_descending,
                        size_t& temp_storage_bytes) {
  if (is_descending) {
    hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr,
                                                       temp_storage_bytes,
                                                       d_keys_in,
                                                       (T*)nullptr,
                                                       d_values_in,
                                                       (int*)nullptr,
                                                       num_items,
                                                       num_segments,
                                                       d_offsets,
                                                       d_offsets + 1,
                                                       0,
                                                       sizeof(T) * 8,
                                                       stream);
  } else {
    hipcub::DeviceSegmentedRadixSort::SortPairs(nullptr,
                                             temp_storage_bytes,
                                             d_keys_in,
                                             (T*)nullptr,
                                             d_values_in,
                                             (int*)nullptr,
                                             num_items,
                                             num_segments,
                                             d_offsets,
                                             d_offsets + 1,
                                             0,
                                             sizeof(T) * 8,
                                             stream);
  }
}

template void GetTempStorageSize(
    const float* d_keys_in,
    const int* d_values_in,
    int* d_offsets,
    int num_items,
    int num_segments,
    hipStream_t stream,
    bool is_descending,
    size_t& temp_storage_bytes);

template void GetTempStorageSize(
    const half* d_keys_in,
    const int* d_values_in,
    int* d_offsets,
    int num_items,
    int num_segments,
    hipStream_t stream,
    bool is_descending,
    size_t& temp_storage_bytes);

// TODO: merge to one kernel
__global__ void SetupParamsKernel(int* d_values_in,
                                  int* d_offsets,
                                  int batch_size,
                                  int vocab_size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total_elements = batch_size * vocab_size;
  if (index < total_elements) {
    d_values_in[index] = index % vocab_size;
  }
  if (index < batch_size + 1) {
    d_offsets[index] = index * vocab_size;
  }
}

void LaunchSetupParamsKernel(int* d_values_in,
                             int* d_offsets,
                             int batch_size,
                             int vocab_size,
                             hipStream_t stream) {
  int total_elements = batch_size * vocab_size;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  SetupParamsKernel<<<gridSize, blockSize, 0, stream>>>(d_values_in,
                                                        d_offsets,
                                                        batch_size,
                                                        vocab_size);
}

template <typename T>
void LaunchSortPairs(void* d_temp_storage,
                     size_t temp_storage_bytes,
                     const T* d_keys_in,
                     T* d_keys_out,
                     const int* d_values_in,
                     int* d_values_out,
                     int num_items,
                     int num_segments,
                     int* d_offsets,
                     hipStream_t stream,
                     bool is_descending) {
  if (is_descending) {
    hipcub::DeviceSegmentedRadixSort::SortPairsDescending(d_temp_storage,
                                                       temp_storage_bytes,
                                                       d_keys_in,
                                                       d_keys_out,
                                                       d_values_in,
                                                       d_values_out,
                                                       num_items,
                                                       num_segments,
                                                       d_offsets,
                                                       d_offsets + 1,
                                                       0,
                                                       sizeof(T) * 8,
                                                       stream);
  } else {
    hipcub::DeviceSegmentedRadixSort::SortPairs(d_temp_storage,
                                             temp_storage_bytes,
                                             d_keys_in,
                                             d_keys_out,
                                             d_values_in,
                                             d_values_out,
                                             num_items,
                                             num_segments,
                                             d_offsets,
                                             d_offsets + 1,
                                             0,
                                             sizeof(T) * 8,
                                             stream);
  }
}

template void LaunchSortPairs(void* d_temp_storage,
                              size_t temp_storage_bytes,
                              const float* d_keys_in,
                              float* d_keys_out,
                              const int* d_values_in,
                              int* d_values_out,
                              int num_items,
                              int num_segments,
                              int* d_offsets,
                              hipStream_t stream,
                              bool is_descending);

template void LaunchSortPairs(void* d_temp_storage,
                              size_t temp_storage_bytes,
                              const half* d_keys_in,
                              half* d_keys_out,
                              const int* d_values_in,
                              int* d_values_out,
                              int num_items,
                              int num_segments,
                              int* d_offsets,
                              hipStream_t stream,
                              bool is_descending);

// A stateful callback functor that maintains a running prefix to be applied
// during consecutive scan operations.
struct BlockPrefixCallbackOp {
  float running_total; // running prefix

  __device__ BlockPrefixCallbackOp(float running_total) : running_total(running_total) {}
  // Callback operator to be entered by the first warp of threads in the block.
  // Thread-0 is responsible for returning a value for seeding the block-wide scan.
  __device__ float operator()(float block_aggregate) {
    float old_prefix = running_total;
    running_total += block_aggregate;
    return old_prefix;
  }
};

template <typename T, int kBlockSize>
__global__ void FilterLogitsKernelCustom(float* d_sorted_logits_in,
                                         const int* d_sorted_indices,
                                         T* d_logits_in_out,
                                         float top_p_threshold,
                                         float filter_value,
                                         int batch_size,
                                         int vocab_size) {
  int vocab_idx = threadIdx.x;
  int batch_id = blockIdx.x;
  int offset = batch_id * vocab_size;

  typedef hipcub::BlockScan<float, kBlockSize> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;
  BlockPrefixCallbackOp prefix_op(0);

  for (int idx = vocab_idx; idx < vocab_size; idx += kBlockSize) {
    float sum = d_sorted_logits_in[offset + idx];
    BlockScan(temp_storage).ExclusiveSum(sum, sum, prefix_op);

    __syncthreads();
    if (sum >= top_p_threshold) {
      int original_index = offset + d_sorted_indices[offset + idx];
      d_logits_in_out[original_index] = (T)filter_value;
    }
  }
}

template <typename T, int kBlockSize>
__global__ void FilterLogitsKernel(float* d_sorted_logits_in,
                                   const int* d_sorted_indices,
                                   T* d_logits_in_out,
                                   float top_p_threshold,
                                   float filter_value,
                                   int min_tokens_to_keep,
                                   int batch_size,
                                   int vocab_size) {
  int vocab_idx = threadIdx.x;
  int batch_id = blockIdx.x;
  int offset = batch_id * vocab_size;

  typedef hipcub::BlockScan<float, kBlockSize> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;
  BlockPrefixCallbackOp prefix_op(0);

  for (int idx = vocab_idx; idx < vocab_size; idx += kBlockSize) {
    float sum = d_sorted_logits_in[offset + idx];
    BlockScan(temp_storage).InclusiveSum(sum, sum, prefix_op);

    __syncthreads();

    if (sum <= top_p_threshold) {
      if (idx + min_tokens_to_keep < vocab_size) {
        int original_index = offset + d_sorted_indices[offset + idx];
        d_logits_in_out[original_index] = (T)filter_value;
      }
    }
  }
}

template <typename T>
void LaunchFilterLogitsKernel(float* d_sorted_logits_in,
                              const int* d_sorted_indices,
                              T* d_logits_in_out,
                              float top_p,
                              float filter_value,
                              int min_tokens_to_keep,
                              int batch_size,
                              int vocab_size,
                              hipStream_t stream,
                              bool is_descending) {
  constexpr int kBlockSize = 256;

  if (is_descending) {
    FilterLogitsKernelCustom<T, kBlockSize><<<batch_size, kBlockSize, 0, stream>>>(d_sorted_logits_in,
                                                                                   d_sorted_indices,
                                                                                   d_logits_in_out,
                                                                                   top_p,
                                                                                   filter_value,
                                                                                   batch_size,
                                                                                   vocab_size);
  } else {
    FilterLogitsKernel<T, kBlockSize><<<batch_size, kBlockSize, 0, stream>>>(d_sorted_logits_in,
                                                                             d_sorted_indices,
                                                                             d_logits_in_out,
                                                                             1 - top_p,
                                                                             filter_value,
                                                                             min_tokens_to_keep,
                                                                             batch_size,
                                                                             vocab_size);
  }
}

template void LaunchFilterLogitsKernel(float* d_sorted_logits_in,
                                       const int* d_sorted_indices,
                                       float* d_logits_in_out,
                                       float top_p,
                                       float filter_value,
                                       int min_tokens_to_keep,
                                       int batch_size,
                                       int vocab_size,
                                       hipStream_t stream,
                                       bool is_descending);

template void LaunchFilterLogitsKernel(float* d_sorted_logits_in,
                                       const int* d_sorted_indices,
                                       half* d_logits_in_out,
                                       float top_p,
                                       float filter_value,
                                       int min_tokens_to_keep,
                                       int batch_size,
                                       int vocab_size,
                                       hipStream_t stream,
                                       bool is_descending);

// Ref: https://github.com/pytorch/pytorch/blob/release/1.13/aten/src/ATen/native/cuda/MultinomialKernel.cu
template <typename scalar_t, typename accscalar_t>
__global__ void sampleMultinomialOnce(int32_t* dest,
                                      int distributions,
                                      int categories,
                                      scalar_t* sampled,
                                      scalar_t* dist,
                                      int stride_dist,        // dist->stride(0)
                                      int stride_categories,  // dist->stride(1)
                                      int* d_presence_mask) {
  extern __shared__ unsigned char my_smem[];
  __shared__ bool found;
  __shared__ unsigned foundPos;
  accscalar_t* smem = reinterpret_cast<accscalar_t*>(my_smem);
  accscalar_t accZero = static_cast<accscalar_t>(0);
  scalar_t zero = static_cast<scalar_t>(0);
  for (int curDist = blockIdx.x;
       curDist < distributions; curDist += gridDim.x) {
    // Assume sum = 1 in Top P sampling as the input is softmaxed.
    accscalar_t sum = 1;

    // Broadcast sum and sample value
    if (threadIdx.x == 0) {
      // Make sure the sum of our distribution didn't overflow
      // CUDA_KERNEL_ASSERT(!_isinf(val));
      // CUDA_KERNEL_ASSERT(sum > accZero);
      foundPos = 0;
      smem[0] = sum;
      smem[1] = sampled[curDist];
    }
    __syncthreads();
    sum = smem[0];
    scalar_t sample = static_cast<scalar_t>(smem[1]);
    __syncthreads();
    if (sum == accZero) {
      // Choose the first element
      if (threadIdx.x == 0) {
        dest[curDist] = 0;
      }
      continue;
    }
    int chunks = (categories + (int)blockDim.x - 1) / blockDim.x;
    accscalar_t prevHighProb = accZero;
    found = false;
    for (int chunk = 0; chunk < chunks && !found; ++chunk) {
      // All threads in bounds load a value
      int cat = chunk * blockDim.x + threadIdx.x;
      accscalar_t dist_val = cat < categories ? static_cast<accscalar_t>(dist[curDist * stride_dist + cat * stride_categories]) / sum : accZero;
      smem[threadIdx.x] = dist_val;
      __syncthreads();
      // Perform an inclusive prefix sum of the shared memory contents
      for (int offset = 1; offset < blockDim.x; offset *= 2) {
        accscalar_t val = accZero;
        if (threadIdx.x >= offset) {
          val = smem[threadIdx.x - offset] + smem[threadIdx.x];
        }
        __syncthreads();
        if (threadIdx.x >= offset) {
          smem[threadIdx.x] = val;
        }
        __syncthreads();
      }
      // Each thread will check to see if the sample falls in its bucket
      scalar_t curBucket =
          static_cast<scalar_t>(smem[threadIdx.x] + prevHighProb);
      scalar_t prevBucket = static_cast<scalar_t>(
          threadIdx.x == 0 ? prevHighProb
                           : smem[threadIdx.x - 1] + prevHighProb);
      bool inBucket =
          (cat < categories) &&
          (!(sample >= curBucket) &&
           (sample >= prevBucket) &&
           (dist_val > zero));
      if (inBucket) {
        // We're done; we have the sample
        // Torch indices are 1-based
        atomicMax(&foundPos, cat);
        found = true;
      }
      // Store the previous scan's high value for future use
      prevHighProb = prevHighProb + smem[blockDim.x - 1];
      __syncthreads();
    }
    if (threadIdx.x == 0) {
      if (found) {
        dest[curDist] = foundPos;
      } else {
        // This should address a rare bug where we don't select a valid index. This likely occurs when
        // due to floating point arithmetic rounding errors, our cumulative sum does not add up to 1, but
        // and our uniform sample is greater than this value. In this case we likely have unitialized memory
        // in dest[curDist]. So basically we will loop through the distribution and pick the largest index
        // where the distribution is non-zero. This is obviously terribly inefficient, but due to the
        // rarity in which this occurs, this should not be an issue.
        for (int cat = categories - 1; cat >= 0; --cat) {
          if (dist[curDist * stride_dist + cat * stride_categories] > zero) {
            dest[curDist] = cat;
            break;
          }
        }
      }
    }
  }

  // update presence mask
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= distributions * categories) {
    return;
  }
  int dist_idx = index / categories;
  int cat_idx = index % categories;
  if (dest[dist_idx] == cat_idx) {
    d_presence_mask[index] = 1;
  }
}

// Only support n_sample = 1
void TorchMultinomialKernelLauncher(float* d_input,
                                    float* d_sampled,
                                    int32_t* d_output,
                                    int batch_size,
                                    int vocab_size,
                                    int* d_presence_mask,
                                    hipStream_t stream) {
  // Store the props in class variables
  int device;
  hipGetDevice(&device);
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, device);

  int numSM = props.multiProcessorCount;
  int maxThreads = props.maxThreadsPerBlock;
  int warp_size = 32;  // at::cuda::warp_size();
  int requiredWarps = (vocab_size + warp_size - 1) / warp_size;
  int requiredThreads = std::min(maxThreads, requiredWarps * warp_size);
  int requiredShared = requiredThreads * sizeof(float);

  dim3 block(requiredThreads);
  dim3 grid(std::min(batch_size, numSM * 4));

  sampleMultinomialOnce<float, float>
      <<<grid, block, requiredShared, stream>>>(d_output,
                                                batch_size,
                                                vocab_size,
                                                d_sampled,
                                                d_input,
                                                vocab_size,
                                                1,
                                                d_presence_mask);
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
