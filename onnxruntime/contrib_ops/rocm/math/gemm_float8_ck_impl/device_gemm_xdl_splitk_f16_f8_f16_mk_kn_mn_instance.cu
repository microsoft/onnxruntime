// SPDX-License-Identifier: MIT
// Modifications Copyright (c) Microsoft.
// Copyright (c) 2018-2023, Advanced Micro Devices, Inc. All rights reserved.


#include <hip/hip_runtime.h>
#include <cstdlib>

#if defined(USE_COMPOSABLE_KERNEL) && !defined(DISABLE_FLOAT8_TYPES)

#include "ck/ck.hpp"
#include "ck/tensor_operation/gpu/device/tensor_layout.hpp"
#include "ck/tensor_operation/gpu/device/gemm_specialization.hpp"
#include "ck/tensor_operation/gpu/device/impl/device_gemm_xdl_splitk_c_shuffle.hpp"

#include "ck/library/tensor_operation_instance/add_device_operation_instance.hpp"

#include "contrib_ops/rocm/math/gemm_float8_ck.cuh"

namespace onnxruntime {
namespace rocm {
namespace tunable {
namespace blas {
namespace internal {

template <ck::index_t... Is>
using S = ck::Sequence<Is...>;

using PassThrough = ck::tensor_operation::element_wise::PassThrough;

static constexpr auto GemmMNPadding = ck::tensor_operation::device::GemmSpecialization::MNPadding;
static constexpr auto GemmMNKPadding = ck::tensor_operation::device::GemmSpecialization::MNKPadding;

using ck::tensor_operation::device::DeviceGemmXdlSplitKCShuffle;

template <typename ScaleElemT>
using device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances_generic = std::tuple<
    // clang-format off
        //#########################|AData| BData| CData| AccData| ALayout| BLayout| CLayout|           A|                 B|           C|           GEMM| Block|  MPer|  NPer| K0Per| K1| MPer| NPer| MXdl| NXdl|  ABlockTransfer| ABlockTransfer| ABlockTransfer| ABlockTransfer| ABlockTransfer| ABlockTransfer| ABlockLds|  BBlockTransfer| BBlockTransfer| BBlockTransfer| BlockTransfer| BBlockTransfer| BBlockTransfer| BBlockLds|    CShuffle|    CShuffle|     CBlockTransferClusterLengths|  CBlockTransfer|
        //#########################| Type|  Type|  Type|    Type|        |        |        | Elementwise|       Elementwise| Elementwise| Specialization|  Size| Block| Block| Block|   |  XDL|  XDL|  Per|  Per|   ThreadCluster|  ThreadCluster| SrcAccessOrder|   SrcVectorDim|      SrcScalar|      DstScalar| AddExtraM|   ThreadCluster|  ThreadCluster| SrcAccessOrder|  SrcVectorDim|      SrcScalar|      DstScalar| AddExtraN| MXdlPerWave| NXdlPerWave| _MBlock_MXdlPerWave_MWaveMPerXdl| ScalarPerVector|
        //#########################|     |      |      |        |        |        |        |   Operation|         Operation|   Operation|               |      |      |      |      |   |     |     | Wave| Wave| Lengths_K0_M_K1|   ArrangeOrder|               |               |      PerVector|   PerVector_K1|          | Lengths_K0_N_K1|   ArrangeOrder|               |              |      PerVector|   PerVector_K1|          |  PerShuffle|  PerShuffle| _NBlock_NXdlPerWave_NWaveNPerXdl|   _NWaveNPerXdl|
        //#########################|     |      |      |        |        |        |        |            |                  |            |               |      |      |      |      |   |     |     |     |     |                |               |               |               |               |               |          |                |               |               |              |               |               |          |            |            |                                 |                |
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNKPadding,   128,   128,   128,     4,  8,   32,   32,    4,    2,  S<1, 4, 32, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              1,              8,      true,  S<1, 4, 32, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              1,              8,      true,           1,           1,                   S<1, 16, 1, 8>,               2>,
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNKPadding,    64,    32,    32,     4,  8,   32,   32,    1,    1,  S<1, 2, 32, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              1,              8,      true,  S<1, 4, 16, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              1,              8,      true,           1,           1,                   S<1, 16, 1, 4>,               2>
    // clang-format on
    >;

// The derived version is simply double BBlockTransferSrcScalarPerVector and adjust other values correspondingly
template <typename ScaleElemT>
using device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances_ort = std::tuple<
    // clang-format off
        //#########################|AData| BData| CData| AccData| ALayout| BLayout| CLayout|           A|                 B|           C|          GEMM| Block|  MPer|  NPer| K0Per| K1| MPer| NPer| MXdl| NXdl|  ABlockTransfer| ABlockTransfer| ABlockTransfer| ABlockTransfer| ABlockTransfer| ABlockTransfer| ABlockLds|  BBlockTransfer| BBlockTransfer| BBlockTransfer| BlockTransfer| BBlockTransfer| BBlockTransfer| BBlockLds|    CShuffle|    CShuffle|     CBlockTransferClusterLengths|  CBlockTransfer| Compute|
        //#########################| Type|  Type|  Type|    Type|        |        |        | Elementwise|       Elementwise| Elementwise|Specialization|  Size| Block| Block| Block|   |  XDL|  XDL|  Per|  Per|   ThreadCluster|  ThreadCluster| SrcAccessOrder|   SrcVectorDim|      SrcScalar|      DstScalar| AddExtraM|   ThreadCluster|  ThreadCluster| SrcAccessOrder|  SrcVectorDim|      SrcScalar|      DstScalar| AddExtraN| MXdlPerWave| NXdlPerWave| _MBlock_MXdlPerWave_MWaveMPerXdl| ScalarPerVector|    Type|
        //#########################|     |      |      |        |        |        |        |   Operation|         Operation|   Operation|              |      |      |      |      |   |     |     | Wave| Wave| Lengths_K0_M_K1|   ArrangeOrder|               |               |      PerVector|   PerVector_K1|          | Lengths_K0_N_K1|   ArrangeOrder|               |              |      PerVector|   PerVector_K1|          |  PerShuffle|  PerShuffle| _NBlock_NXdlPerWave_NWaveNPerXdl|   _NWaveNPerXdl|        |
        //#########################|     |      |      |        |        |        |        |            |                  |            |              |      |      |      |      |   |     |     |     |     |                |               |               |               |               |               |          |                |               |               |              |               |               |          |            |            |                                 |                |        |
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNPadding,   256,   256,   128,     8,  4,   32,   32,    4,    2,  S<1, 8, 32, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              4,              4,      true,  S<1, 8, 32, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              4,              4,      true,           1,           1,                   S<1, 32, 1, 8>,               8,    F16>,
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNPadding,   256,   128,   256,     8,  4,   32,   32,    2,    4,  S<1, 8, 32, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              4,              4,      true,  S<1, 8, 32, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              8,              4,      true,           1,           1,                   S<1, 32, 1, 8>,               8,    F16>,
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNPadding,   128,   128,   128,     8,  4,   32,   32,    4,    2,  S<1, 8, 16, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              4,              4,      true,  S<1, 8, 16, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              8,              4,      true,           1,           1,                   S<1, 16, 1, 8>,               8,    F16>,
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNPadding,   256,    64,   192,     8,  4,   32,   32,    1,    3,  S<1, 8, 32, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              4,              4,      true,  S<1, 8, 24, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              4,              4,      true,           1,           1,                   S<1, 32, 1, 8>,               8,    F16>,
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNPadding,   256,   192,    64,     8,  4,   32,   32,    3,    1,  S<1, 8, 32, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              4,              4,      true,  S<1, 8, 16, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              4,              4,      true,           1,           1,                   S<1, 32, 1, 8>,               8,    F16>,
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNPadding,   256,   128,   128,     8,  4,   32,   32,    2,    2,  S<1, 8, 32, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              4,              4,      true,  S<1, 8, 32, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              4,              4,      true,           1,           1,                   S<1, 32, 1, 8>,               8,    F16>,
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNPadding,   128,   128,    64,     8,  4,   32,   32,    2,    2,  S<1, 8, 16, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              4,              4,      true,  S<1, 8, 16, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              4,              4,      true,           1,           1,                   S<1, 32, 1, 4>,               8,    F16>,
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNPadding,   128,    64,   128,     8,  4,   32,   32,    2,    2,  S<1, 8, 16, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              4,              4,      true,  S<1, 8, 16, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              8,              4,      true,           1,           1,                   S<1, 16, 1, 8>,               8,    F16>,
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNPadding,   256,   128,    64,     8,  4,   32,   32,    2,    1,  S<1, 8, 32, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              4,              4,      true,  S<1, 8, 32, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              2,              4,      true,           1,           1,                   S<1, 16, 1, 4>,               8,    F16>,
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNPadding,   256,    64,   128,     8,  4,   32,   32,    1,    2,  S<1, 8, 32, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              4,              4,      true,  S<1, 8, 32, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              4,              4,      true,           1,           1,                   S<1, 32, 1, 8>,               8,    F16>,
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNPadding,   128,    32,   192,     8,  4,   32,   32,    1,    3,  S<1, 8, 16, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              4,              4,      true,  S<1, 8, 12, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,             16,              4,      true,           1,           1,                   S<1, 16, 1, 8>,               8,    F16>,
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNPadding,   128,   192,    32,     8,  4,   32,   32,    3,    1,  S<1, 8, 16, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              4,              4,      true,  S<1, 8, 16, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              2,              4,      true,           1,           1,                   S<1, 32, 1, 4>,               8,    F16>,
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNPadding,   128,    32,    64,     8,  4,   32,   32,    1,    1,  S<1, 8, 16, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              4,              4,      true,  S<1, 8, 16, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              4,              4,      true,           1,           1,                   S<1, 16, 1, 8>,               8,    F16>,
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNPadding,   128,    64,    32,     8,  4,   32,   32,    1,    1,  S<1, 8, 16, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              4,              4,      true,  S<1, 8, 16, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              2,              4,      true,           1,           1,                   S<1, 32, 1, 4>,               8,    F16>,
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNPadding,   128,    32,   128,     8,  4,   32,   32,    1,    2,  S<1, 8, 16, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              4,              4,      true,  S<1, 8, 16, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              8,              4,      true,           1,           1,                   S<1, 16, 1, 8>,               8,    F16>,
        DeviceGemmXdlSplitKCShuffle<  F16,    F8,   F16,     F32,     Row,      Row,    Row, PassThrough, Scale<ScaleElemT>, PassThrough, GemmMNPadding,   128,   128,    32,     8,  4,   32,   32,    2,    1,  S<1, 8, 16, 1>,  S<0, 2, 1, 3>,  S<0, 2, 1, 3>,              3,              4,              4,      true,  S<1, 8, 16, 1>,  S<0, 1, 3, 2>,  S<0, 1, 3, 2>,             2,              2,              4,      true,           1,           1,                   S<1, 32, 1, 4>,               8,    F16>
    // clang-format on
    >;

void add_device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances_ort(
    std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
        Row, Row, Row, F16, F8, F16, PassThrough, Scale<Float8E4M3FN>, PassThrough>>>& instances) {
  ck::tensor_operation::device::instance::add_device_operation_instances(
      instances, device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances_ort<Float8E4M3FN>{});
  ck::tensor_operation::device::instance::add_device_operation_instances(
      instances, device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances_generic<Float8E4M3FN>{});
}

void add_device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances_ort(
    std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
        Row, Row, Row, F16, F8, F16, PassThrough, Scale<Float8E4M3FNUZ>, PassThrough>>>& instances) {
  ck::tensor_operation::device::instance::add_device_operation_instances(
      instances, device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances_ort<Float8E4M3FNUZ>{});
  ck::tensor_operation::device::instance::add_device_operation_instances(
      instances, device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances_generic<Float8E4M3FNUZ>{});
}

}  // namespace internal
}  // namespace blas
}  // namespace tunable
}  // namespace rocm
}  // namespace onnxruntime

#endif
