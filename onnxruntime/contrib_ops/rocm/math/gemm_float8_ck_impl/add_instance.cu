// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.


#include <hip/hip_runtime.h>
#include <cstdlib>

#if defined(USE_COMPOSABLE_KERNEL) && !defined(DISABLE_FLOAT8_TYPES)

#include "ck/ck.hpp"
#include "ck/tensor_operation/gpu/device/tensor_layout.hpp"
#include "ck/tensor_operation/gpu/device/impl/device_gemm_xdl_splitk_c_shuffle.hpp"

#include "contrib_ops/rocm/math/gemm_float8_ck.cuh"

namespace onnxruntime {
namespace rocm {
namespace tunable {
namespace blas {

using F8 = ck::f8_t;
using F16 = ck::half_t;
using F32 = float;

using Row = ck::tensor_layout::gemm::RowMajor;
using Col = ck::tensor_layout::gemm::ColumnMajor;

template <ck::index_t... Is>
using S = ck::Sequence<Is...>;

using PassThrough = ck::tensor_operation::element_wise::PassThrough;

namespace internal {
void add_device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances_ck(
    std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
        Row, Row, Row, F16, F8, F16, PassThrough, Scale<Float8E4M3FN>, PassThrough>>>& instances);

void add_device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances_ck(
    std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
        Row, Row, Row, F16, F8, F16, PassThrough, Scale<Float8E4M3FNUZ>, PassThrough>>>& instances);

void add_device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances_ort(
    std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
        Row, Row, Row, F16, F8, F16, PassThrough, Scale<Float8E4M3FN>, PassThrough>>>& instances);

void add_device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances_ort(
    std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
        Row, Row, Row, F16, F8, F16, PassThrough, Scale<Float8E4M3FNUZ>, PassThrough>>>& instances);
}  // namespace internal

void add_device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances(
    std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
        Row, Row, Row, F16, F8, F16, PassThrough, Scale<Float8E4M3FN>, PassThrough>>>& instances) {
  internal::add_device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances_ck(instances);
  internal::add_device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances_ort(instances);
}

void add_device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances(
    std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
        Row, Row, Row, F16, F8, F16, PassThrough, Scale<Float8E4M3FNUZ>, PassThrough>>>& instances) {
  internal::add_device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances_ck(instances);
  internal::add_device_gemm_xdl_splitk_f16_f8_f16_mk_kn_mn_instances_ort(instances);
}

namespace internal {
void add_device_gemm_xdl_splitk_f8_f16_f16_mk_kn_mn_instances_ck(
    std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
        Row, Row, Row, F8, F16, F16, Scale<Float8E4M3FN>, PassThrough, PassThrough>>>& instances);

void add_device_gemm_xdl_splitk_f8_f16_f16_mk_kn_mn_instances_ck(
    std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
        Row, Row, Row, F8, F16, F16, Scale<Float8E4M3FNUZ>, PassThrough, PassThrough>>>& instances);

// TODO: The first try of derivation does not going well due to various constraints.
// void add_device_gemm_xdl_splitk_f8_f16_f16_mk_kn_mn_instances_ort(
//     std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
//         Row, Row, Row, F8, F16, F16, Scale<Float8E4M3FN>, PassThrough, PassThrough>>>& instances);

// TODO: The first try of derivation does not going well due to various constraints.
// void add_device_gemm_xdl_splitk_f8_f16_f16_mk_kn_mn_instances_ort(
//     std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
//         Row, Row, Row, F8, F16, F16, Scale<Float8E4M3FNUZ>, PassThrough, PassThrough>>>& instances);
}  // namespace internal

void add_device_gemm_xdl_splitk_f8_f16_f16_mk_kn_mn_instances(
    std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
        Row, Row, Row, F8, F16, F16, Scale<Float8E4M3FN>, PassThrough, PassThrough>>>& instances) {
  internal::add_device_gemm_xdl_splitk_f8_f16_f16_mk_kn_mn_instances_ck(instances);
  //   internal::add_device_gemm_xdl_splitk_f8_f16_f16_mk_kn_mn_instances_ort(instances);  // TODO:
}

void add_device_gemm_xdl_splitk_f8_f16_f16_mk_kn_mn_instances(
    std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
        Row, Row, Row, F8, F16, F16, Scale<Float8E4M3FNUZ>, PassThrough, PassThrough>>>& instances) {
  internal::add_device_gemm_xdl_splitk_f8_f16_f16_mk_kn_mn_instances_ck(instances);
  //   internal::add_device_gemm_xdl_splitk_f8_f16_f16_mk_kn_mn_instances_ort(instances);  // TODO:
}

namespace internal {
void add_device_gemm_xdl_splitk_f16_f8_f16_mk_nk_mn_instances_ck(
    std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
        Row, Col, Row, F16, F8, F16, PassThrough, Scale<Float8E4M3FN>, PassThrough>>>& instances);

void add_device_gemm_xdl_splitk_f16_f8_f16_mk_nk_mn_instances_ck(
    std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
        Row, Col, Row, F16, F8, F16, PassThrough, Scale<Float8E4M3FNUZ>, PassThrough>>>& instances);
}  // namespace internal

void add_device_gemm_xdl_splitk_f16_f8_f16_mk_nk_mn_instances(
    std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
        Row, Col, Row, F16, F8, F16, PassThrough, Scale<Float8E4M3FN>, PassThrough>>>& instances) {
  internal::add_device_gemm_xdl_splitk_f16_f8_f16_mk_nk_mn_instances_ck(instances);
}

void add_device_gemm_xdl_splitk_f16_f8_f16_mk_nk_mn_instances(
    std::vector<std::unique_ptr<ck::tensor_operation::device::DeviceGemmSplitK<
        Row, Col, Row, F16, F8, F16, PassThrough, Scale<Float8E4M3FNUZ>, PassThrough>>>& instances) {
  internal::add_device_gemm_xdl_splitk_f16_f8_f16_mk_nk_mn_instances_ck(instances);
}

}  // namespace blas
}  // namespace tunable
}  // namespace rocm
}  // namespace onnxruntime

#endif
