// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "contrib_ops/rocm/bert/gemm_fast_gelu_impl.h"

#include <hip/hip_fp16.h>

#include "contrib_ops/rocm/bert/gemm_fast_gelu_tunable_op.h"
#include "core/providers/rocm/tunable/gemm_common.h"

using onnxruntime::rocm::tunable::blas::BlasOp;

namespace onnxruntime {
namespace contrib {
namespace rocm {

// See it as row-major
template <typename T>
Status LaunchGemmFastGeluKernel(bool tuning,
                                hipStream_t stream,
                                rocblas_handle handle,
                                bool transa,
                                bool transb,
                                int64_t m,
                                int64_t n,
                                int64_t k,
                                const T alpha,
                                const T* a,
                                int64_t lda,
                                const T* b,
                                int64_t ldb,
                                const T* bias,
                                const T beta,
                                T* c,
                                int64_t ldc) {
  GemmFastGeluParams<T> params;
  params.tuning = tuning;
  params.stream = stream;
  params.handle = handle;
  params.opa = transa ? BlasOp::Trans : BlasOp::NonTrans;
  params.opb = transb ? BlasOp::Trans : BlasOp::NonTrans;

  params.m = m;
  params.n = n;
  params.k = k;
  params.alpha = alpha;
  params.a = a;
  params.lda = lda;
  params.b = b;
  params.ldb = ldb;
  params.bias = bias;
  params.beta = beta;
  params.c = c;
  params.ldc = ldc;

  if (tuning) {
    static GemmFastGeluTunableOp<T> op;
    op.EnableTuning();
    return op(&params);
  }

  return GemmFastGeluUnfused(&params);
}

#define SPECIALIZED_IMPL(T)                                                                     \
  template Status LaunchGemmFastGeluKernel<T>(bool tuning,                                      \
                                              hipStream_t stream, rocblas_handle handle,        \
                                              bool transa, bool transb,                         \
                                              int64_t m, int64_t n, int64_t k, const T alpha,   \
                                              const T* a, int64_t lda, const T* b, int64_t ldb, \
                                              const T* bias, const T beta, T* c, int64_t ldc);

SPECIALIZED_IMPL(float)
SPECIALIZED_IMPL(half)
SPECIALIZED_IMPL(BFloat16)

}  // namespace rocm
}  // namespace contrib
}  // namespace onnxruntime
