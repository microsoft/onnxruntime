#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on qkvToContext plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/

Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications: scaling is moved from masked softmax to the gemm before that.
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_fp16.h>
#include "core/providers/rocm/cu_inc/common.cuh"
#include "core/providers/rocm/rocm_common.h"
#include "core/providers/rocm/shared_inc/fpgeneric.h"
#include "contrib_ops/rocm/bert/attention_impl.h"
#include "contrib_ops/rocm/bert/attention_softmax.h"
#include "contrib_ops/rocm/bert/transformer_common.h"

using namespace onnxruntime::rocm;
using namespace hipcub;

#define CHECK_ROCM(expr)  \
  if (!HIP_CALL(expr)) { \
    return false;         \
  }

namespace onnxruntime {
namespace contrib {
namespace rocm {

static size_t AlignTo(size_t a, size_t b) {
  return CeilDiv(a, b) * b;
}

size_t GetAttentionScratchSize(size_t element_size, int batch_size, int num_heads, int sequence_length, int all_sequence_length) {
  const size_t len = batch_size * num_heads * sequence_length * all_sequence_length;
  const size_t bytes = len * element_size;

  const size_t alignment = 256;
  const size_t bytesAligned = AlignTo(bytes, alignment);
  return bytesAligned;
}

size_t GetAttentionWorkspaceSize(
    size_t element_size,
    int batch_size,
    int num_heads,
    int head_size,
    int sequence_length,
    int past_sequence_length) {
  size_t qkv_size = 3 * batch_size * sequence_length * num_heads * head_size * element_size;
  return qkv_size + 2 * GetAttentionScratchSize(element_size, batch_size, num_heads, sequence_length, past_sequence_length + sequence_length);
}

template <typename T>
bool QkvToContext(
    const hipDeviceProp_t& prop, rocblas_handle& rocblas, hipStream_t stream,
    const int batch_size, const int sequence_length, const int num_heads, const int head_size, const size_t element_size,
    const T* input, T* output, T* workspace,
    const int* mask_index, gsl::span<const int64_t> mask_index_dims,
    bool is_unidirectional, int past_sequence_length, const T* past, const T* extra_add_qk, T* present, bool use_persistent_softmax) {
  const int all_sequence_length = past_sequence_length + sequence_length;
  const size_t bytes = GetAttentionScratchSize(element_size, batch_size, num_heads, sequence_length, all_sequence_length);
  T* scratch1 = workspace;
  T* scratch2 = scratch1 + (bytes / element_size);
  T* scratch3 = scratch2 + (bytes / element_size);

  const int max_threads_per_block = prop.maxThreadsPerBlock;

  // input should be BxSx3xNxH => scratch3: 3xBxNxSxH
  if (!LaunchTransQkv(stream, 3, sequence_length, batch_size, head_size, num_heads, max_threads_per_block, false, input, scratch3)) {
    return false;
  }

  // now scratch3 has Q, K, V: each has size BxNxSxH
  const int batches = batch_size * num_heads;
  const int size_per_batch = sequence_length * head_size;
  const int total_size = batches * size_per_batch;

  const T* q = scratch3;
  const T* k = q + total_size;
  const T* v = k + total_size;

  rocblas_set_stream(rocblas, stream);

  // Concat past (2xBxNxS'xH) to present (2xBxNxS*xH):
  // past_k (BxNxS'xH) + k (BxNxSxH) => present_k (BxNxS*xH)
  // past_v (BxNxS'xH) + v (BxNxSxH) => present_v (BxNxS*xH)
  const int present_size_per_batch = all_sequence_length * head_size;
  if (nullptr != present) {
    if (!LaunchConcatPastToPresent(stream, all_sequence_length, sequence_length, batch_size, head_size, num_heads, max_threads_per_block, past, k, present)) {
      return false;
    }

    // update pointers to present_k and present_v.
    k = present;
    v = present + batches * present_size_per_batch;
  }

  // Raw attention mask could be 2D (BxS) or 3D (BxSxS*) or 4D(Bx1xMxM), where M is the max sequence length.
  bool use_raw_attention_mask = (nullptr != mask_index && mask_index_dims.size() >= 2);

  // compute Q*K' (as K'*Q), scaled by 1/sqrt(H) and store in scratch1: BxNxSxS*
  // Q: BxNxSxH, K (present_k): BxNxS*xH, Q*K': BxNxSxS*
  const float rsqrt_head_size = 1.f / sqrt(static_cast<float>(head_size));
  const int temp_matrix_size = sequence_length * all_sequence_length;

  typedef typename ToHipType<T>::MappedType HipT;

  //float one = 1.0f;
  //float zero = 0.f;
  const HipT one = ToHipType<T>::FromFloat(1.0f);
  const HipT zero = ToHipType<T>::FromFloat(0.f);

  // For raw attention mask, the scalar if 1/sqrt(H) is moved to softmax computation.
  //float temp_alpha = use_raw_attention_mask ? one : rsqrt_head_size;
  const HipT alpha = use_raw_attention_mask ? one : ToHipType<T>::FromFloat(rsqrt_head_size);

  if (!ROCBLAS_CALL(rocblasGemmStridedBatchedHelper(
          rocblas, rocblas_operation_transpose, rocblas_operation_none, all_sequence_length, sequence_length, head_size, &alpha, k, head_size, present_size_per_batch,
          q, head_size, size_per_batch, &zero, scratch1, all_sequence_length, temp_matrix_size, batches))) {
    return false;
  }

  // apply softmax and store result P to scratch2: BxNxSxS*
  if (use_raw_attention_mask) {  // 2d, 3d or 4d attention mask
    const int mask_dimension = static_cast<int>(mask_index_dims.size());
    const int64_t max_sequence_length = mask_dimension == 4 ? mask_index_dims.at(3) : 0;

    T* persistent_softmax_workspace = scratch1; // replace Q*K' in place with masked score if persistent softmax is selected.
    if (!ComputeSoftmaxWithRawMask<T>(stream, all_sequence_length, sequence_length, batch_size, num_heads, mask_index, nullptr, extra_add_qk, scratch1, scratch2,
                                      is_unidirectional, rsqrt_head_size, mask_dimension, static_cast<int>(max_sequence_length),
                                      use_persistent_softmax, persistent_softmax_workspace)) {
      return false;
    }
  } else if (nullptr != mask_index) {  // 1d mask index
    ORT_ENFORCE(mask_index_dims.size() == 1);
    // mask_index has 1D shape: either (batch_size) or (2*batch_size). Only the later one has start postions.
    const int* mask_start = (mask_index_dims.at(0) > batch_size) ? mask_index + batch_size : nullptr;
    if (!ComputeSoftmaxWithMask1D<T>(stream, all_sequence_length, sequence_length, batch_size, num_heads, mask_index, mask_start, extra_add_qk, scratch1, scratch2, is_unidirectional)) {
      return false;
    }
  } else {  // no mask
    if (!ComputeSoftmax<T>(stream, all_sequence_length, sequence_length, batch_size, num_heads, extra_add_qk, scratch1, scratch2, is_unidirectional)) {
      return false;
    }
  }

  // compute P*V (as V*P), and store in scratch3: BxNxSxH
  if (!ROCBLAS_CALL(rocblasGemmStridedBatchedHelper(
          rocblas, rocblas_operation_none, rocblas_operation_none, head_size, sequence_length, all_sequence_length, &one, v, head_size, present_size_per_batch,
          scratch2, all_sequence_length, temp_matrix_size, &zero, scratch3, head_size, size_per_batch, batches))) {
    return false;
  }

  // scratch3 is BxNxSxH, transpose to output BxSxNxH
  return LaunchTransCtx(stream, sequence_length, batch_size, head_size, num_heads, max_threads_per_block, false, scratch3, output);
}

bool LaunchAttentionKernel(
    const hipDeviceProp_t& prop,
    hipStream_t stream,
    const void* input,
    const int* mask_index,
    gsl::span<const int64_t> mask_index_dims,
    void* output,
    const int batch_size,
    const int sequence_length,
    const int num_heads,
    const int head_size,
    void* workspace,
    rocblas_handle& rocblas,
    const size_t element_size,
    bool is_unidirectional,
    int past_sequence_length,
    const void* past,
    const void* extra_add_qk,
    void* present) {
  // For testing, environment variable ORT_TRANSFORMER_OPTIONS=1 could enable persistent softmax
  const TransformerOptions* options = TransformerOptions::GetInstance();
  bool use_persistent_softmax = options->IsPrecisionMode() && !options->DisablePersistentSoftmax();
  if (element_size == 2) {
    return QkvToContext(prop, rocblas, stream,
                        batch_size, sequence_length, num_heads, head_size, element_size,
                        reinterpret_cast<const __half*>(input), reinterpret_cast<__half*>(output), reinterpret_cast<__half*>(workspace),
                        mask_index, mask_index_dims, is_unidirectional,
                        past_sequence_length, reinterpret_cast<const __half*>(past), reinterpret_cast<const __half*>(extra_add_qk),
                        reinterpret_cast<__half*>(present), use_persistent_softmax);
  } else {
    return QkvToContext(prop, rocblas, stream,
                        batch_size, sequence_length, num_heads, head_size, element_size,
                        reinterpret_cast<const float*>(input), reinterpret_cast<float*>(output), reinterpret_cast<float*>(workspace),
                        mask_index, mask_index_dims, is_unidirectional,
                        past_sequence_length, reinterpret_cast<const float*>(past), reinterpret_cast<const float*>(extra_add_qk),
                        reinterpret_cast<float*>(present), use_persistent_softmax);
  }
}
}  // namespace rocm
}  // namespace contrib
}  // namespace onnxruntime
