#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on gelu plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/

Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications: Add (bias) before Gelu is merged into this op to get better performance.
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/rocm/rocm_common.h"
#include "core/providers/rocm/cu_inc/common.cuh"
#include "core/providers/rocm/shared_inc/rocm_call.h"
#include "contrib_ops/rocm/bert/fast_gelu_impl.h"

using namespace onnxruntime::rocm;

namespace onnxruntime {
namespace contrib {
namespace rocm {

// constants for approximating the normal cdf
constexpr float A = 0.5;

constexpr float B = 0.7978845608028654;  // sqrt(2.0/M_PI)

constexpr float C = 0.035677408136300125;  // 0.044715 * sqrt(2.0/M_PI)

constexpr float one = 1.0;
constexpr float two = 2.0;

template <typename T, unsigned TPB>
__global__ void FastGeluKernel(const T a, const T b, const T c, const T oneT, const T twoT,
                               int input_length, int bias_length, const T* input, const T* bias, T* output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;

  if (idx < input_length) {
    const T x = input[idx];
    const T in = (bias == nullptr) ? x : (x + bias[idx % bias_length]);

    // const T cdf = a + a * _Tanh(in * (c * in * in + b));
    const T u = twoT * in * (c * in * in + b);
    const T emu = __expf(-u);
    const T cdf = a + a * (twoT/(oneT + emu) - oneT);

    output[idx] = in * cdf;
  }
}

template <unsigned TPB>
__global__ void FastGeluKernel2(const half2 a, const half2 b, const half2 c, const half2 one2, const half2 two2,
                                int input_length, int bias_length, const half2* input, const half2* bias,
                                half2* output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;

  if (idx < input_length) {
    const half2 x = input[idx];
    const half2 in = (bias == nullptr) ? x : (x + bias[idx % bias_length]);

    // const half2 cdf = a + a * _Tanh(in * (c * in * in + b));
    const half2 u = two2 * in * (c * in * in + b);
    const half2 emu = h2exp(-u);
    const half2 cdf = a + a * (two2/(one2 + emu) - one2);

    output[idx] = in * cdf;
  }
}

template <unsigned TPB>
__global__ void FastGeluKernel4Bias(const half2 a, const half2 b, const half2 c, const half2 one2, const half2 two2,
                                    int input_length, int bias_length, const float2* input, const float2* bias,
                                    float2* output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;

  if (idx < input_length) {
    float2 input_vec = input[idx];
    float2 bias_vec = bias[idx % bias_length];
    float2 output_vec = output[idx];

    half2* input_half = reinterpret_cast<half2*>(&input_vec);
    half2* bias_half = reinterpret_cast<half2*>(&bias_vec);
    half2* output_half = reinterpret_cast<half2*>(&output_vec);

    half2 lo_data = input_half[0];
    half2 hi_data = input_half[1];
    half2 lo_bias = bias_half[0];
    half2 hi_bias = bias_half[1];

    lo_data += lo_bias;
    hi_data += hi_bias;

    const half2 lo_u = two2 * lo_data * (c * lo_data * lo_data + b);
    const half2 hi_u = two2 * hi_data * (c * hi_data * hi_data + b);
    const half2 lo_emu = h2exp(-lo_u);
    const half2 hi_emu = h2exp(-hi_u);
    const half2 lo_cdf = a + a * (two2/(one2 + lo_emu) - one2);
    const half2 hi_cdf = a + a * (two2/(one2 + hi_emu) - one2);

    output_half[0] = lo_data * lo_cdf;
    output_half[1] = hi_data * hi_cdf;

    output[idx] = output_vec;
  }
}

template <unsigned TPB>
__global__ void FastGeluKernel4(const half2 a, const half2 b, const half2 c, const half2 one2, const half2 two2,
                                int input_length, const float2* input, float2* output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;

  if (idx < input_length) {
    float2 input_vec = input[idx];
    float2 output_vec = output[idx];

    half2* input_half = reinterpret_cast<half2*>(&input_vec);
    half2* output_half = reinterpret_cast<half2*>(&output_vec);

    half2 lo_data = input_half[0];
    half2 hi_data = input_half[1];

    const half2 lo_u = two2 * lo_data * (c * lo_data * lo_data + b);
    const half2 hi_u = two2 * hi_data * (c * hi_data * hi_data + b);
    const half2 lo_emu = h2exp(-lo_u);
    const half2 hi_emu = h2exp(-hi_u);
    const half2 lo_cdf = a + a * (two2/(one2 + lo_emu) - one2);
    const half2 hi_cdf = a + a * (two2/(one2 + hi_emu) - one2);

    output_half[0] = lo_data * lo_cdf;
    output_half[1] = hi_data * hi_cdf;

    output[idx] = output_vec;
  }
}

template <>
bool LaunchFastGeluKernel(const hipDeviceProp_t& prop, hipStream_t stream, int input_length, int bias_length,
                          const float* input, const float* bias, float* output, bool /*use_half2*/) {
  constexpr int blockSize = 256;
  const int gridSize = (input_length + blockSize - 1) / blockSize;
  hipLaunchKernelGGL(HIP_KERNEL_NAME(FastGeluKernel<float, blockSize>), dim3(gridSize), dim3(blockSize), 0,
                     stream, A, B, C, one, two, input_length, bias_length, input, bias, output);

  return HIP_CALL(hipPeekAtLastError());
}

template <>
bool LaunchFastGeluKernel(const hipDeviceProp_t& prop, hipStream_t stream, int input_length, int bias_length,
                          const half* input, const half* bias, half* output, bool use_half2) {
  constexpr int blockSize = 256;
  if (use_half2 && prop.major >= 7 && (0 == (bias_length % 4) || 0 == (bias_length & 1))) {
    const half2 A2 = __float2half2_rn(A);
    const half2 B2 = __float2half2_rn(B);
    const half2 C2 = __float2half2_rn(C);
    const half2 one2 = __float2half2_rn(one);
    const half2 two2 = __float2half2_rn(two);
    if (0 == (bias_length % 4)) {
      const int n = input_length / 4;
      const int gridSize = (n + blockSize - 1) / blockSize;
      const float2* input4 = reinterpret_cast<const float2*>(input);
      const float2* bias4 = reinterpret_cast<const float2*>(bias);
      float2* output4 = reinterpret_cast<float2*>(output);
      if (bias == nullptr)
        hipLaunchKernelGGL(HIP_KERNEL_NAME(FastGeluKernel4<blockSize>), dim3(gridSize), dim3(blockSize), 0,
                           stream, A2, B2, C2, one2, two2, n, input4, output4);
      else
        hipLaunchKernelGGL(HIP_KERNEL_NAME(FastGeluKernel4Bias<blockSize>), dim3(gridSize), dim3(blockSize), 0,
                           stream, A2, B2, C2, one2, two2, n, bias_length / 4, input4, bias4, output4);
    } else {
      const int n = input_length / 2;
      const int gridSize = (n + blockSize - 1) / blockSize;
      const half2* input2 = reinterpret_cast<const half2*>(input);
      const half2* bias2 = reinterpret_cast<const half2*>(bias);
      half2* output2 = reinterpret_cast<half2*>(output);
      hipLaunchKernelGGL(HIP_KERNEL_NAME(FastGeluKernel2<blockSize>), dim3(gridSize), dim3(blockSize), 0,
                         stream, A2, B2, C2, one2, two2, n, bias_length / 2, input2, bias2, output2);
    }
  } else {
    const int gridSize = (input_length + blockSize - 1) / blockSize;
    const half oneT = half(one);
    const half twoT = half(two);
    hipLaunchKernelGGL(HIP_KERNEL_NAME(FastGeluKernel<half, blockSize>), dim3(gridSize), dim3(blockSize), 0,
                       stream, A, B, C, oneT, twoT, input_length, bias_length, input, bias, output);
  }

  return HIP_CALL(hipPeekAtLastError());
}

template <>
bool LaunchFastGeluKernel(const hipDeviceProp_t& prop, hipStream_t stream, int input_length, int bias_length,
                          const BFloat16* input, const BFloat16* bias, BFloat16* output, bool /*use_half2*/) {
  constexpr int blockSize = 256;
  const int gridSize = (input_length + blockSize - 1) / blockSize;
  const BFloat16 oneT = BFloat16(one);
  const BFloat16 twoT = BFloat16(two);
  hipLaunchKernelGGL(HIP_KERNEL_NAME(FastGeluKernel<BFloat16, blockSize>), dim3(gridSize), dim3(blockSize), 0,
                     stream, A, B, C, oneT, twoT, input_length, bias_length, input, bias, output);
  return HIP_CALL(hipPeekAtLastError());
}

}  // namespace rocm
}  // namespace contrib
}  // namespace onnxruntime
