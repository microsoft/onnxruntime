#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on gelu plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/
 
Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications: Add (bias) before Gelu is merged into this op to get better performance.
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/rocm/rocm_common.h"
#include "core/providers/rocm/cu_inc/common.cuh"
#include "core/providers/rocm/shared_inc/rocm_call.h"
#include "contrib_ops/rocm/bert/fast_gelu_impl.h"

using namespace onnxruntime::rocm;

namespace onnxruntime {
namespace contrib {
namespace rocm {

// constants for approximating the normal cdf
constexpr float A = 0.5;

constexpr float B = 0.7978845608028654;  // sqrt(2.0/M_PI)

constexpr float C = 0.035677408136300125;  // 0.044715 * sqrt(2.0/M_PI)

constexpr float one = 1.0;
constexpr float two = 2.0;

template <typename T, unsigned TPB>
__global__ void FastGeluKernel(const T a, const T b, const T c, int input_length, int bias_length, const T* input, const T* bias, T* output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;

  const T twoT = T(two);
  const T oneT = T(one);

  if (idx < input_length) {
    const T x = input[idx];
    const T in = (bias == nullptr) ? x : (x + bias[idx % bias_length]);

    // const T cdf = a + a * _Tanh(in * (c * in * in + b));
    const T u = twoT * in * (c * in * in + b);
    const T emu = __expf(-u);
    const T cdf = a + a * (twoT/(oneT + emu) - oneT);

    output[idx] = in * cdf;
  }
}

template <unsigned TPB>
__global__ void FastGeluKernel2(const half2 a, const half2 b, const half2 c, int input_length, int bias_length, const half2* input, const half2* bias, half2* output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;

  const half2 two2 = __floats2half2_rn(two, two);
  const half2 one2 = __floats2half2_rn(one, one);

  if (idx < input_length) {
    const half2 x = input[idx];
    const half2 in = (bias == nullptr) ? x : (x + bias[idx % bias_length]);

    // const half2 cdf = a + a * _Tanh(in * (c * in * in + b));
    const half2 u = two2 * in * (c * in * in + b);
    const half2 emu = h2exp(-u);
    const half2 cdf = a + a * (two2/(one2 + emu) - one2);

    output[idx] = in * cdf;
  }
}

template <>
bool LaunchFastGeluKernel(const hipDeviceProp_t& prop, hipStream_t stream, int input_length, int bias_length, const float* input, const float* bias, float* output) {
  constexpr int blockSize = 256;
  const int gridSize = (input_length + blockSize - 1) / blockSize;
  hipLaunchKernelGGL(HIP_KERNEL_NAME(FastGeluKernel<float, blockSize>), dim3(gridSize), dim3(blockSize), 0, stream, A, B, C, input_length, bias_length, input, bias, output);

  return HIP_CALL(hipPeekAtLastError());
}

template <>
bool LaunchFastGeluKernel(const hipDeviceProp_t& prop, hipStream_t stream, int input_length, int bias_length, const half* input, const half* bias, half* output) {
  constexpr int blockSize = 256;

  if (0 == (bias_length & 1) && prop.major >= 7) {
    const int n = input_length / 2;
    const int gridSize = (n + blockSize - 1) / blockSize;
    const half2 A2 = __floats2half2_rn(A, A);
    const half2 B2 = __floats2half2_rn(B, B);
    const half2 C2 = __floats2half2_rn(C, C);
    const half2* input2 = reinterpret_cast<const half2*>(input);
    const half2* bias2 = reinterpret_cast<const half2*>(bias);
    half2* output2 = reinterpret_cast<half2*>(output);
    hipLaunchKernelGGL(HIP_KERNEL_NAME(FastGeluKernel2<blockSize>), dim3(gridSize), dim3(blockSize), 0, stream, A2, B2, C2, n, bias_length / 2, input2, bias2, output2);
  } else {
    const int gridSize = (input_length + blockSize - 1) / blockSize;
    hipLaunchKernelGGL(HIP_KERNEL_NAME(FastGeluKernel<half, blockSize>), dim3(gridSize), dim3(blockSize), 0, stream, A, B, C, input_length, bias_length, input, bias, output);
  }

  return HIP_CALL(hipPeekAtLastError());
}

}  // namespace rocm
}  // namespace contrib
}  // namespace onnxruntime
