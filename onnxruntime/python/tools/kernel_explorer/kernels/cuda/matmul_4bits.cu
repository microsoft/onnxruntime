// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

// This file serve as a simple example for adding a tunable op to onnxruntime.

#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <pybind11/pybind11.h>

#include <string>

#include "core/providers/cuda/tunable/cuda_tunable.h"
#include "python/tools/kernel_explorer/kernel_explorer_interface.h"
#include "python/tools/kernel_explorer/kernels/vector_add_kernel.cuh"
#include "contrib_ops/cuda/quantization/matmul_nbits.cuh"

namespace py = pybind11;

namespace onnxruntime {

// Extend the OpParams so that all specializations have the same parameter passing interface
template <typename T>
struct MatrixFloatInt4Params : cuda::tunable::OpParams {
  std::string Signature() const override { return std::to_string(n_); }

  T* output_;
  const T* a_;
  const uint8_t* b_;
  const T* scales_;
  const uint8_t* zero_points_;
  int m_;
  int n_;
  int k_;
};

template <typename T>
class MatrixFloatInt4 : public IKernelExplorer {
 public:
  MatrixFloatInt4(DeviceArray& output,
                  DeviceArray& a,
                  DeviceArray& b,
                  DeviceArray& scales,
                  int m, int n, int k) {
    params_.tuning_ctx = TuningContext();
    params_.stream = Stream();
    params_.output_ = static_cast<T*>(output.ptr());
    params_.a_ = static_cast<T*>(a.ptr());
    params_.b_ = static_cast<uint8_t*>(b.ptr());
    params_.scales_ = static_cast<T*>(scales.ptr());
    params_.zero_points_ = nullptr;
    params_.m_ = m;
    params_.n_ = n;
    params_.k_ = k;

    CUDA_CALL_THROW(hipGetDeviceProperties(&device_prop_, 0));
  }

  MatrixFloatInt4(DeviceArray& output,
                  DeviceArray& a,
                  DeviceArray& b,
                  DeviceArray& scales,
                  DeviceArray& zeropoints,
                  int m, int n, int k) : MatrixFloatInt4(output, a, b, scales, m, n, k) {
    params_.zero_points_ = static_cast<uint8_t*>(zeropoints.ptr());
  }

  void Run() override {
    contrib::cuda::TryMatMul4Bits<T>(
        params_.output_,
        params_.a_,
        params_.b_,
        params_.scales_,
        params_.zero_points_,
        params_.m_,
        params_.n_,
        params_.k_,
        32,
        static_cast<int>(device_prop_.sharedMemPerBlock),
        params_.StreamHandle());
  }

 private:
  // A VectorAddOp<T> is a callable that can process const VectorAddParams<T>*
  using ParamsT = MatrixFloatInt4Params<T>;
  ParamsT params_{};
  hipDeviceProp_t device_prop_;
};

#define REGISTER_OP(name, type)                                                                             \
  py::class_<name<type>>(m, #name "_" #type)                                                                \
      .def(py::init<DeviceArray&, DeviceArray&, DeviceArray&, DeviceArray&, int, int, int>())               \
      .def(py::init<DeviceArray&, DeviceArray&, DeviceArray&, DeviceArray&, DeviceArray&, int, int, int>()) \
      .def("SetRepeats", &name<type>::SetRepeats)                                                           \
      .def("Profile", &name<type>::Profile)                                                                 \
      .def("Run", &name<type>::Run);

KE_REGISTER(m) {
  REGISTER_OP(MatrixFloatInt4, half);
  REGISTER_OP(MatrixFloatInt4, float);
}

}  // namespace onnxruntime
