// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

// This file serve as a simple example for adding a tunable op to onnxruntime.

#if USE_CUDA
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#elif USE_ROCM
#include <hip/hip_fp16.h>
#endif
#include <pybind11/pybind11.h>

#include <string>

#if USE_CUDA
#include "core/providers/cuda/tunable/cuda_tunable.h"
#elif USE_ROCM
#include "core/providers/rocm/tunable/rocm_tunable.h"
#endif
#include "python/tools/kernel_explorer/kernel_explorer_interface.h"
#include "python/tools/kernel_explorer/kernels/vector_add_kernel.cuh"

namespace py = pybind11;

namespace onnxruntime {

// #####################################################################################################################
//  In practice, VectorAddParam, VectorAddOp and VectorAddTunableOp should be tightly integrated to onnxruntime.
//  We place them here purely for demo purpose.
// #####################################################################################################################

// Extend the OpParams so that all specializations have the same parameter passing interface
template <typename T>
struct VectorAddParams :
#if USE_CUDA
    cuda::tunable::OpParams
#elif USE_ROCM
    rocm::tunable::OpParams
#endif
{
  std::string Signature() const override { return std::to_string(n); }

  T* x;
  T* y;
  T* z;
  int n;
};

// Wrap the kernel function, so that we have a unified launch interface. If the kernel has state, the state can also
// be managed at this level via a functor
template <typename T, int TPB, int Vec>
Status VectorAddOp(const VectorAddParams<T>* params) {
  return LaunchVectorAdd<T, TPB, Vec>(
      params->StreamHandle(),
      params->x,
      params->y,
      params->z,
      params->n);
}

#define ADD_OP(threads_per_block)                         \
  this->RegisterOp(VectorAddOp<T, threads_per_block, 1>); \
  this->RegisterOp(VectorAddOp<T, threads_per_block, 2>); \
  this->RegisterOp(VectorAddOp<T, threads_per_block, 4>); \
  this->RegisterOp(VectorAddOp<T, threads_per_block, 8>);

// A Tunable VectorAddOp is a collection of non-tunable VectorAddOps implementations that have variable performance
// characteristics. Those implementations may be put into a C++ container for tuner to select.
template <typename T>
class VectorAddTunableOp :
#if USE_CUDA
    public cuda::tunable::TunableOp<VectorAddParams<T>>
#elif USE_ROCM
    public rocm::tunable::TunableOp<VectorAddParams<T>>
#endif
{
 public:
  VectorAddTunableOp() {
    ADD_OP(64);
    ADD_OP(128);
    ADD_OP(192);
    ADD_OP(256);
    ADD_OP(320);
    ADD_OP(384);
    ADD_OP(448);
    ADD_OP(512);
  }
};

#undef ADD_OP

// #####################################################################################################################
//  Following code just wraps our kernel implementation and expose them as python interface. This is the code that
//  should be in the kernel_explorer directory.
// #####################################################################################################################

template <typename T, int TPB, int Vec>
class VectorAdd : public IKernelExplorer {
 public:
  VectorAdd(DeviceArray& x, DeviceArray& y, DeviceArray& z, int n) {
    params_.tuning_ctx = TuningContext();
    params_.stream = Stream();
    params_.x = static_cast<T*>(x.ptr());
    params_.y = static_cast<T*>(y.ptr());
    params_.z = static_cast<T*>(z.ptr());
    params_.n = n;
  }

  void Run() override {
    ORT_THROW_IF_ERROR((VectorAddOp<T, TPB, Vec>(&params_)));
  }

 private:
  // A VectorAddOp<T> is a callable that can process const VectorAddParams<T>*
  using ParamsT = VectorAddParams<T>;
  ParamsT params_{};
};

template <typename T>
class VectorAddTunable : public IKernelExplorer {
 public:
  VectorAddTunable(DeviceArray& x, DeviceArray& y, DeviceArray& z, int n) {
    params_.tuning_ctx = TuningContext();
    params_.stream = Stream();
    params_.x = static_cast<T*>(x.ptr());
    params_.y = static_cast<T*>(y.ptr());
    params_.z = static_cast<T*>(z.ptr());
    params_.n = n;

    params_.TuningContext()->EnableTunableOpAndTuning();
  }

  void Run() override {
    ORT_THROW_IF_ERROR(impl_(&params_));
  }

 private:
  using ParamsT = VectorAddParams<T>;
  ParamsT params_;

  // tunable is stateful, store it as an instance
  VectorAddTunableOp<T> impl_;
};

#define REGISTER_OP(name, type, threads_per_block, vec_size)                                              \
  py::class_<name<type, threads_per_block, vec_size>>(m, #name"_"#type"_"#threads_per_block"_"#vec_size)  \
    .def(py::init<DeviceArray&, DeviceArray&, DeviceArray&, int>())                                       \
    .def("SetRepeats", &name<type, threads_per_block, vec_size>::SetRepeats)                              \
    .def("Profile", &name<type, threads_per_block, vec_size>::Profile)                                    \
    .def("Run", &name<type, threads_per_block, vec_size>::Run);

#define REGISTER_OP_FOR_ALL_VEC_SIZE(name, type, threads_per_block)  \
  REGISTER_OP(name, type, threads_per_block, 1)                      \
  REGISTER_OP(name, type, threads_per_block, 2)                      \
  REGISTER_OP(name, type, threads_per_block, 4)                      \
  REGISTER_OP(name, type, threads_per_block, 8)

#define REGISTER_OP_FOR_ALL_THREADS_PER_BLOCK(name, type)  \
  REGISTER_OP_FOR_ALL_VEC_SIZE(name, type, 64)             \
  REGISTER_OP_FOR_ALL_VEC_SIZE(name, type, 128)            \
  REGISTER_OP_FOR_ALL_VEC_SIZE(name, type, 192)            \
  REGISTER_OP_FOR_ALL_VEC_SIZE(name, type, 256)            \
  REGISTER_OP_FOR_ALL_VEC_SIZE(name, type, 320)            \
  REGISTER_OP_FOR_ALL_VEC_SIZE(name, type, 384)            \
  REGISTER_OP_FOR_ALL_VEC_SIZE(name, type, 448)            \
  REGISTER_OP_FOR_ALL_VEC_SIZE(name, type, 512)

#define REGISTER_TUNABLE_OP(type)                                      \
  py::class_<VectorAddTunable<type>>(m, "VectorAdd_" #type "_Tunable") \
      .def(py::init<DeviceArray&, DeviceArray&, DeviceArray&, int>())  \
      .def("SetRepeats", &VectorAddTunable<type>::SetRepeats)          \
      .def("Profile", &VectorAddTunable<type>::Profile)                \
      .def("Run", &VectorAddTunable<type>::Run);

KE_REGISTER(m) {
  REGISTER_OP_FOR_ALL_THREADS_PER_BLOCK(VectorAdd, half);
  REGISTER_OP_FOR_ALL_THREADS_PER_BLOCK(VectorAdd, float);

  REGISTER_TUNABLE_OP(half);
  REGISTER_TUNABLE_OP(float)
}

}  // namespace onnxruntime
