#include <algorithm>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <vector>

int main(int argc, char* argv[])
{
    int n_devices = 0;
    int rc = hipGetDeviceCount(&n_devices);
    if (rc != hipSuccess)
    {
        hipError_t error = hipGetLastError();
        std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return rc;
    }

    std::vector<std::pair<int, int>> arch(n_devices);
    for (int cd = 0; cd < n_devices; ++cd)
    {
        hipDeviceProp_t dev;
        int rc = hipGetDeviceProperties(&dev, cd);
        if (rc != hipSuccess)
        {
            hipError_t error = hipGetLastError();
            std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
            return rc;
        }
        else
        {
            arch[cd] = {dev.major, dev.minor};
        }
    }

    std::pair<int, int> best_cc = *std::max_element(begin(arch), end(arch));
    std::cout << best_cc.first << best_cc.second;

    return 0;
}
