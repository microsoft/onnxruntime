#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// This file is adapted from microsoft/DeepSpeed
// multi_tensor_adam.cu

/* Copyright 2020 The Microsoft DeepSpeed Team
   Copyright NVIDIA/apex
   This file is adapted from fused adam in NVIDIA/apex, commit a109f85
*/

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "multi_tensor_apply.cuh"
#include "type_shim.h"
#include <cmath>

#define BLOCK_SIZE 512
#define ILP 4

typedef enum {
    ADAM_MODE_0 = 0,   // L2 regularization mode
    ADAM_MODE_1 = 1,   // Decoupled weight decay mode (AdamW) as implemented in transformers/AdamW
    ADAM_MODE_2 = 2    // Decoupled weight decay mode (AdamW) as implemented in pytorch/AdamW
} adamMode_t;

using MATH_T = float;

template <typename T>
struct AdamFunctor {
    __device__ __forceinline__ void operator()(int chunk_size,
                                               volatile int* noop_gmem,
                                               TensorListMetadata<4>& tl,
                                               const float beta1,
                                               const float beta2,
                                               const float epsilon,
                                               const float lr,
                                               const float lr_corrected,
                                               const float bias_correction1,
                                               const float bias_correction2,
                                               adamMode_t mode,
                                               const float decay)
    {
        int tensor_loc = tl.block_to_tensor[blockIdx.x];

        int chunk_idx = tl.block_to_chunk[blockIdx.x];
        int n = tl.sizes[tensor_loc];

        T* g = (T*)tl.addresses[0][tensor_loc];
        g += chunk_idx * chunk_size;

        T* p = (T*)tl.addresses[1][tensor_loc];
        p += chunk_idx * chunk_size;

        T* m = (T*)tl.addresses[2][tensor_loc];
        m += chunk_idx * chunk_size;

        T* v = (T*)tl.addresses[3][tensor_loc];
        v += chunk_idx * chunk_size;

        n -= chunk_idx * chunk_size;

        // see note in multi_tensor_scale_kernel.cu
        for (int i_start = 0; i_start < n && i_start < chunk_size; i_start += blockDim.x * ILP) {
            MATH_T r_g[ILP];
            MATH_T r_p[ILP];
            MATH_T r_m[ILP];
            MATH_T r_v[ILP];
#pragma unroll
            for (int ii = 0; ii < ILP; ii++) {
                int i = i_start + threadIdx.x + ii * blockDim.x;
                if (i < n && i < chunk_size) {
                    r_g[ii] = g[i];
                    r_p[ii] = p[i];
                    r_m[ii] = m[i];
                    r_v[ii] = v[i];
                } else {
                    r_g[ii] = MATH_T(0);
                    r_p[ii] = MATH_T(0);
                    r_m[ii] = MATH_T(0);
                    r_v[ii] = MATH_T(0);
                }
            }
#pragma unroll
            for (int ii = 0; ii < ILP; ii++) {
                if (mode == ADAM_MODE_0) {  // L2
                    r_g[ii] = r_g[ii] + (decay * r_p[ii]);
                    r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
                    r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
                    MATH_T denom = sqrtf(r_v[ii]) + epsilon;
                    r_p[ii] = r_p[ii] - (lr_corrected * r_m[ii] / denom);
                } else if (mode == ADAM_MODE_1) {  // weight decay
                    // Adapted to be mathematically equivalent to transformers AdamW
                    r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
                    r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
                    MATH_T denom = sqrtf(r_v[ii]) + epsilon;
                    r_p[ii] = r_p[ii] - (lr_corrected * r_m[ii] / denom);
                    r_p[ii] = r_p[ii] - (lr * decay * r_p[ii]);
                } else if (mode == ADAM_MODE_2) {
                    // Adapted to be mathematically equivalent to torch AdamW
                    r_p[ii] = r_p[ii] -  (r_p[ii] * lr * decay);
                    r_m[ii] = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
                    r_v[ii] = beta2 * r_v[ii] + (1 - beta2) * r_g[ii] * r_g[ii];
                    MATH_T denom = (sqrtf(r_v[ii]) / sqrtf(bias_correction2)) + epsilon;
                    r_p[ii] = r_p[ii] - (lr * r_m[ii]) / (bias_correction1 * denom);
                }
            }
#pragma unroll
            for (int ii = 0; ii < ILP; ii++) {
                int i = i_start + threadIdx.x + ii * blockDim.x;
                if (i < n && i < chunk_size) {
                    p[i] = r_p[ii];
                    m[i] = r_m[ii];
                    v[i] = r_v[ii];
                }
            }
        }
    }
};

void multi_tensor_adam_cuda(int chunk_size,
                            at::Tensor noop_flag,
                            std::vector<std::vector<at::Tensor>> tensor_lists,
                            const float lr,
                            const float beta1,
                            const float beta2,
                            const float epsilon,
                            const int step,
                            const int mode,
                            const int bias_correction,
                            const float weight_decay)
{
    using namespace at;

    // Handle bias correction mode
    float bias_correction1 = 1.0, bias_correction2 = 1.0;
    float lr_corrected = lr;
    if (bias_correction == 1) {
        bias_correction1 = 1 - std::pow(beta1, step);
        bias_correction2 = 1 - std::pow(beta2, step);
        lr_corrected *= std::sqrt(bias_correction2) / bias_correction1;
    }

    // Assume single type across p,g,m1,m2 now
    DISPATCH_DOUBLE_FLOAT_AND_HALF(tensor_lists[0][0].scalar_type(),
                                   0,
                                   "adam",
                                   multi_tensor_apply<4>(BLOCK_SIZE,
                                                         chunk_size,
                                                         noop_flag,
                                                         tensor_lists,
                                                         AdamFunctor<scalar_t_0>(),
                                                         beta1,
                                                         beta2,
                                                         epsilon,
                                                         lr,
                                                         lr_corrected,
                                                         bias_correction1,
                                                         bias_correction2,
                                                         (adamMode_t)mode,
                                                         weight_decay);)

    AT_CUDA_CHECK(hipGetLastError());
}
