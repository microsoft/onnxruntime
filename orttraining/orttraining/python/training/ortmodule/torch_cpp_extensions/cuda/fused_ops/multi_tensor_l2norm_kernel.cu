#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Copyright NVIDIA/apex
// This file is adapted from NVIDIA/apex, commit 3ff1a10f72ec07067c4e44759442329804ac5162

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
#include <c10/cuda/CUDAGuard.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "type_shim.h"
#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 512
#define ILP 4

template <typename T>
__device__ __forceinline__ bool is_aligned(T* p) {
  return ((uint64_t)p) % (ILP * sizeof(T)) == 0;
}

template <typename T>
__device__ __forceinline__ void load_store(T* dst, T* src, int dst_offset, int src_offset) {
  typedef typename std::aligned_storage<ILP * sizeof(T), ILP * alignof(T)>::type LT;
  ((LT*)dst)[dst_offset] = ((LT*)src)[src_offset];
}

template <typename x_t>
struct L2NormFunctor {
  __device__ __forceinline__ void operator()(int chunk_size, volatile int* noop_gmem, TensorListMetadata<1>& tl,
                                             float* output, float* output_per_tensor, bool per_tensor,
                                             int max_chunks_per_tensor) {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    x_t* x = (x_t*)tl.addresses[0][tensor_loc];
    x += chunk_idx * chunk_size;

    n -= chunk_idx * chunk_size;

    __shared__ float s_vals[512];

    float vals[ILP];  // = {0}; // this probably works too but I want to be sure...
    x_t r_x[ILP];
    for (int i = 0; i < ILP; i++) {
      vals[i] = 0.f;
      r_x[i] = 0;
    }

    // to make things simple, we put aligned case in a different code path
    if (n % ILP == 0 && chunk_size % ILP == 0 && is_aligned(x)) {
      for (int i_start = threadIdx.x; i_start * ILP < n && i_start * ILP < chunk_size; i_start += blockDim.x) {
        // load
        load_store(r_x, x, 0, i_start);
#pragma unroll
        for (int ii = 0; ii < ILP; ii++) {
          float next = static_cast<float>(r_x[ii]);
          vals[ii] += next * next;
        }
      }
    } else {
      for (int i_start = 0; i_start < n && i_start < chunk_size; i_start += blockDim.x * ILP) {
#pragma unroll
        for (int ii = 0; ii < ILP; ii++) {
          int i = i_start + threadIdx.x + ii * blockDim.x;
          if (i < n && i < chunk_size) {
            float next = static_cast<float>(x[i]);
            vals[ii] += next * next;
          }
        }
      }
    }

    float val = 0.f;
    for (int i = 0; i < ILP; i++) val += vals[i];

    float final = reduce_block_into_lanes(s_vals, val);

    if (threadIdx.x == 0) {
      if (!isfinite(final)) *noop_gmem = 1;  // Blindly fire off a write.  These will race but that's ok.
      output[blockIdx.x] += final;
      if (per_tensor)
        output_per_tensor[(tl.start_tensor_this_launch + tensor_loc) * max_chunks_per_tensor + chunk_idx] = final;
    }
  }
};

__global__ void cleanup(float* output, float* output_per_tensor, float* ret, float* ret_per_tensor, bool per_tensor,
                        int max_chunks_per_tensor) {
  __shared__ float vals[512];

  if (blockIdx.x == 0) {
    float val = 0;
    if (threadIdx.x < 320) val = output[threadIdx.x];

    float final = reduce_block_into_lanes(vals, val);

    if (threadIdx.x == 0) *ret = sqrt(final);
  }

  if (per_tensor) {
    float* output_this_tensor = output_per_tensor + blockIdx.x * max_chunks_per_tensor;

    float val = 0;
    for (int i = threadIdx.x; i < max_chunks_per_tensor; i += blockDim.x) val += output_this_tensor[i];

    float final = reduce_block_into_lanes(vals, val);

    if (threadIdx.x == 0) ret_per_tensor[blockIdx.x] = sqrt(final);
  }
}

std::tuple<at::Tensor, at::Tensor> multi_tensor_l2norm_cuda(int chunk_size, at::Tensor noop_flag,
                                                            std::vector<std::vector<at::Tensor>> tensor_lists,
                                                            at::optional<bool> per_tensor_python) {
  bool per_tensor = per_tensor_python.has_value() ? per_tensor_python.value() : false;

  auto float_options = tensor_lists[0][0].options().dtype(at::kFloat);
  auto output = at::zeros({320}, float_options);

  at::Tensor output_per_tensor;
  at::Tensor ret_per_tensor;

  int ntensors = tensor_lists[0].size();
  int max_chunks_per_tensor = -1;

  if (per_tensor) {
    for (int t = 0; t < ntensors; t++) {
      int max_chunks_this_tensor = (tensor_lists[0][t].numel() + chunk_size - 1) / chunk_size;
      if (max_chunks_this_tensor > max_chunks_per_tensor) max_chunks_per_tensor = max_chunks_this_tensor;
    }
    output_per_tensor = at::zeros({ntensors * max_chunks_per_tensor}, float_options);
    ret_per_tensor = at::empty({ntensors}, float_options);
  } else {
    ret_per_tensor = at::empty({0}, float_options);
  }

  DISPATCH_DOUBLE_FLOAT_AND_HALF(
      tensor_lists[0][0].scalar_type(), 0, "multi_tensor_l2norm_cuda",
      multi_tensor_apply<1>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists, L2NormFunctor<scalar_t_0>(),
                            output.data_ptr<float>(), per_tensor ? output_per_tensor.data_ptr<float>() : nullptr,
                            per_tensor, max_chunks_per_tensor);)

  AT_CUDA_CHECK(hipGetLastError());
  // AT_CUDA_CHECK(hipDeviceSynchronize());

  // This involves one more small kernel launches, but will be negligible end to end.
  // I could get rid of these by hacking the functor + multi tensor harness with persistence
  // logic, but keeping it simple for now
  auto ret = at::empty({1}, output.options());
  const at::cuda::OptionalCUDAGuard device_guard(device_of(output));
  auto stream = at::cuda::getCurrentCUDAStream();
  cleanup<<<per_tensor ? ntensors : 1, 512, 0, stream>>>(
      output.data_ptr<float>(), per_tensor ? output_per_tensor.data_ptr<float>() : nullptr, ret.data_ptr<float>(),
      per_tensor ? ret_per_tensor.data_ptr<float>() : nullptr, per_tensor, max_chunks_per_tensor);

  return std::tuple<at::Tensor, at::Tensor>(ret, ret_per_tensor);
}
