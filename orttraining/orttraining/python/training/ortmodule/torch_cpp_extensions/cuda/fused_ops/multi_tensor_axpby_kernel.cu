#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Copyright NVIDIA/apex
// This file is adapted from NVIDIA/apex, commit 0c7d8e3fa9a095a1641a2290877436d0314b69c6

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "type_shim.h"
#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 512
#define ILP 4

template<typename T>
__device__ __forceinline__ bool is_aligned(T* p){
  return ((uint64_t)p) % (ILP*sizeof(T)) == 0;
}

template<typename T>
__device__ __forceinline__ void load_store(T* dst, T* src, int dst_offset, int src_offset){
  typedef typename std::aligned_storage<ILP*sizeof(T), ILP*alignof(T)>::type LT;
  ((LT*)dst)[dst_offset] = ((LT*)src)[src_offset];
}

template<typename x_t, typename y_t, typename out_t>
struct AxpbyFunctor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorListMetadata<3>& tl,
    float a,
    float b,
    int arg_to_check)
  {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    x_t* x = (x_t*)tl.addresses[0][tensor_loc];
    x += chunk_idx*chunk_size;

    y_t* y = (y_t*)tl.addresses[1][tensor_loc];
    y += chunk_idx*chunk_size;

    out_t* out = (out_t*)tl.addresses[2][tensor_loc];
    out += chunk_idx*chunk_size;

    n -= chunk_idx*chunk_size;

    bool finite = true;
    x_t r_x[ILP];
    y_t r_y[ILP];
    out_t r_out[ILP];

    // to make things simple, we put aligned case in a different code path
    if(n % ILP == 0 && chunk_size % ILP == 0 && is_aligned(x) && is_aligned(y) && is_aligned(out))
    {
      for(int i_start = threadIdx.x; i_start*ILP < n && i_start*ILP < chunk_size; i_start += blockDim.x)
      {
        // load
        load_store(r_x, x, 0 , i_start);
        load_store(r_y, y, 0 , i_start);
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          r_out[ii] = a*static_cast<float>(r_x[ii]) + b*static_cast<float>(r_y[ii]);
          if(arg_to_check == -1)
            finite = finite && (isfinite(r_x[ii]) && isfinite(r_y[ii]));
          if(arg_to_check == 0)
            finite = finite && isfinite(r_x[ii]);
          if(arg_to_check == 1)
            finite = finite && isfinite(r_y[ii]);
        }
        // store
        load_store(out, r_out, i_start , 0);
      }
    }
    else
    {
      // Non-divergent exit condition for __syncthreads, not necessary here
      for(int i_start = 0; i_start < n && i_start < chunk_size; i_start += blockDim.x*ILP)
      {
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          r_x[ii] = 0;
          r_y[ii] = 0;
          int i = i_start + threadIdx.x + ii*blockDim.x;
          if(i < n && i < chunk_size)
          {
            r_x[ii] = x[i];
            r_y[ii] = y[i];
          }
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          r_out[ii] = a*static_cast<float>(r_x[ii]) + b*static_cast<float>(r_y[ii]);
          if(arg_to_check == -1)
            finite = finite && (isfinite(r_x[ii]) && isfinite(r_y[ii]));
          if(arg_to_check == 0)
            finite = finite && isfinite(r_x[ii]);
          if(arg_to_check == 1)
            finite = finite && isfinite(r_y[ii]);
        }
        // see note in multi_tensor_scale_kernel.cu
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          int i = i_start + threadIdx.x + ii*blockDim.x;
          if(i < n && i < chunk_size)
            out[i] = r_out[ii];
        }
      }
    }
    if(!finite)
      *noop_gmem = 1; // Blindly fire off a write.  These will race but that's ok.
  }
};

void multi_tensor_axpby_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>>& tensor_lists,
  float a,
  float b,
  int arg_to_check)
{
  using namespace at;
  // The output (downscaled) type is always float.
  // If build times suffer, think about where to put this dispatch,
  // and what logic should be moved out of multi_tensor_apply.

  DISPATCH_DOUBLE_FLOAT_AND_HALF(tensor_lists[0][0].scalar_type(), 0, "multi_tensor_axpby_cuda",
    DISPATCH_DOUBLE_FLOAT_AND_HALF(tensor_lists[1][0].scalar_type(), 1, "multi_tensor_axpby_cuda",
      DISPATCH_DOUBLE_FLOAT_AND_HALF(tensor_lists[2][0].scalar_type(), 2, "multi_tensor_axpby_cuda",
           multi_tensor_apply<3>(
             BLOCK_SIZE,
             chunk_size,
             noop_flag,
             tensor_lists,
             AxpbyFunctor<scalar_t_0, scalar_t_1, scalar_t_2>(),
             a,
             b,
             arg_to_check); )))

  AT_CUDA_CHECK(hipGetLastError());

  // AT_CUDA_CHECK(hipDeviceSynchronize());
}
