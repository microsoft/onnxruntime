#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "orttraining/training_ops/rocm/tensor/gather_grad_impl.h"
#include "core/providers/rocm/cu_inc/common.cuh"
#include "core/providers/rocm/shared_inc/rocm_call.h"

#include <hipcub/hipcub.hpp>


namespace onnxruntime {
namespace rocm {

template <typename T>
__global__ void _Iota(
    hipcub::CountingInputIterator<T> input,
    size_t length,
    T* output) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(idx, length);
  output[idx] = input[idx];
}

template <typename T, typename Tin>
__global__ void _GatherGradImpl(
    const Tin* input,
    const Tin* indices,
    const T* grad_output,
    T* grad_weight,
    int64_t numel,
    int64_t input_numel,
    int64_t param_itrs,
    int64_t stride) {
  int idx = blockIdx.x * 4 + threadIdx.y;

  const int SZ = 4;
  if (idx < numel && (idx == 0 || input[idx] != input[idx - 1])) {
    do {
      for (int itr = 0; itr < param_itrs; ++itr) {
        const int start_feature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
        const int weight_row = itr * input_numel + ((int)input[idx]) * stride;  //the offset of the input
        const int grad_row = (itr * numel + ((int)indices[idx])) * stride;      //the offset of the gradient

        float gradient[SZ];
        float weight[SZ];

#pragma unroll
        for (int ii = 0; ii < SZ; ii++) {
          int feature_dim = start_feature + ii * GPU_WARP_SIZE;
          if (feature_dim < stride) {
            gradient[ii] = static_cast<float>(grad_output[grad_row + feature_dim]);
            weight[ii] = static_cast<float>(grad_weight[weight_row + feature_dim]);
          }
        }

#pragma unroll
        for (int ii = 0; ii < SZ; ii++) {
          weight[ii] += gradient[ii];
        }

#pragma unroll
        for (int ii = 0; ii < SZ; ii++) {
          int feature_dim = start_feature + ii * GPU_WARP_SIZE;
          if (feature_dim < stride) {
            grad_weight[weight_row + feature_dim] = static_cast<T>(weight[ii]);
          }
        }
      }
      idx++;
    } while (idx < numel && input[idx] == input[idx - 1]);
  }
}

template <typename T, typename Tin>
void GatherGradImpl(
    const RocmKernel& rocm_kernel,
    const T* grad_data,
    const Tin* indices_data,
    const int64_t num_indices,
    const int64_t num_weights,
    const int64_t stride,
    T* output_data,
    const int64_t num_inputs,  //The number of input elements starting from the gathering dimension
    const int64_t param_itrs   //The size of dimensions of the data before gathering dimension
    ) {
  // allocate intermediate buffers
  auto original_indices = rocm_kernel.template GetScratchBuffer<Tin>(num_indices);

  // initialize original_indices with [0, num_indices)
  {
    const auto blocks_per_grid = CeilDiv(num_indices, GridDim::maxThreadsPerBlock);
    hipcub::CountingInputIterator<Tin> counting_input(Tin{});
    hipLaunchKernelGGL(_Iota, dim3(blocks_per_grid), dim3(GridDim::maxThreadsPerBlock), 0, 0, 
        counting_input, num_indices, original_indices.get());
  }

  auto indices_data_sorted = rocm_kernel.template GetScratchBuffer<Tin>(num_indices);
  auto original_indices_sorted = rocm_kernel.template GetScratchBuffer<Tin>(num_indices);

  // sort indices and original indices
  size_t sort_temp_storage_size_bytes = 0;
  HIP_CALL_THROW(hipcub::DeviceRadixSort::SortPairs(
      nullptr, sort_temp_storage_size_bytes,
      indices_data, indices_data_sorted.get(),
      original_indices.get(), original_indices_sorted.get(),
      num_indices));

  auto sort_temp_storage = rocm_kernel.GetScratchBuffer<void>(sort_temp_storage_size_bytes);

  HIP_CALL_THROW(hipcub::DeviceRadixSort::SortPairs(
      sort_temp_storage.get(), sort_temp_storage_size_bytes,
      indices_data, indices_data_sorted.get(),
      original_indices.get(), original_indices_sorted.get(),
      num_indices));

  dim3 block(GPU_WARP_SIZE, 4);
  dim3 grid(CeilDiv(num_indices, 4), CeilDiv(stride, 128));

  hipLaunchKernelGGL(_GatherGradImpl, dim3(grid), dim3(block), 0, 0, 
      indices_data_sorted.get(),
      original_indices_sorted.get(),
      grad_data,
      output_data,
      num_indices,
      num_inputs,
      param_itrs,
      stride);
}

#define SPECIALIZED_GRAD_IMPL2(T)           \
  template void GatherGradImpl<T, int64_t>( \
      const RocmKernel& rocm_kernel,        \
      const T* grad_data,                   \
      const int64_t* indices_data,          \
      const int64_t num_indices,            \
      const int64_t num_weights,            \
      const int64_t stride,                 \
      T* output_data,                       \
      const int64_t num_inputs,             \
      const int64_t param_itrs);            \
  template void GatherGradImpl<T, int32_t>( \
      const RocmKernel& rocm_kernel,        \
      const T* grad_data,                   \
      const int32_t* indices_data,          \
      const int64_t num_indices,            \
      const int64_t num_weights,            \
      const int64_t stride,                 \
      T* output_data,                       \
      const int64_t num_inputs,             \
      const int64_t param_itrs);

SPECIALIZED_GRAD_IMPL2(float)
SPECIALIZED_GRAD_IMPL2(half)

}  // namespace rocm
}  // namespace onnxruntime
