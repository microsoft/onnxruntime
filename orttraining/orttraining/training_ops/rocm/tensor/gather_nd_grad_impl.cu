#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "orttraining/training_ops/rocm/tensor/gather_nd_grad_impl.h"

#include "core/providers/rocm/cu_inc/common.cuh"
#include "core/providers/rocm/atomic/common.cuh"

namespace onnxruntime {
namespace rocm {

template <typename T>
__global__ void _GatherNDGradKernel(
    const size_t num_slices,
    const T* update_data,
    T* output_data,
    const size_t slice_size,
    const int64_t* slice_offsets) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(i, num_slices * slice_size);
  uint64_t slice_offset = slice_offsets[i / slice_size];
  size_t j = i % slice_size;
  atomic_add(output_data + slice_offset + j, update_data[i]);
};

template <typename T>
void GatherNDGradImpl(
    hipStream_t stream,
    const size_t num_slices,
    const void* update_data,
    void* output_data,
    const size_t slice_size,
    const int64_t* input_slice_offsets_data) {
  const auto blocks_per_grid = CeilDiv(num_slices * slice_size, GridDim::maxThreadsPerBlock);
  hipLaunchKernelGGL(HIP_KERNEL_NAME(_GatherNDGradKernel<T>), dim3(blocks_per_grid), dim3(GridDim::maxThreadsPerBlock), 0, stream, 
      num_slices, static_cast<const T*>(update_data), static_cast<T*>(output_data), slice_size, input_slice_offsets_data);
}

#define SPECIALIZED_GRAD_IMPL(T) \
  template void GatherNDGradImpl<T>(hipStream_t stream, const size_t num_slices, const void* update_data, void* output_data, const size_t slice_size, const int64_t* input_slice_offsets_data)

SPECIALIZED_GRAD_IMPL(float);
SPECIALIZED_GRAD_IMPL(half);
SPECIALIZED_GRAD_IMPL(double);

}  // namespace rocm
}  // namespace onnxruntime
