#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "orttraining/training_ops/cuda/optimizer/common.cuh"
#include "orttraining/training_ops/cuda/optimizer/adam.h"
#include "orttraining/training_ops/cuda/optimizer/common.h"

namespace onnxruntime {
namespace cuda {
template <typename T1, typename T3, typename T4, typename T_GRAD, typename T_GRAD_NORM, typename T_MIXED_PRECISION_FP>
__global__ void _AdamOptimizer_mode0(
    const T1* eta,
    const T3* weights,
    const T_GRAD* grads,
    const T4* moment_1,
    const T4* moment_2,
    const T3* loss_scale,
    const T_GRAD_NORM* grad_norm,
    const float alpha,
    const float beta,
    const float lambda,
    const float epsilon,
    const float max_norm,
    const float alpha_correction,
    const float beta_correction,
    T4* moment_1_out,
    T4* moment_2_out,
    T3* weights_out,
    T_GRAD* grads_out,
    T_MIXED_PRECISION_FP* mixed_precision_weights_out,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  const float actual_scale = _ComputeGradScale<T3, T_GRAD_NORM, float>(loss_scale, grad_norm, max_norm);

  // Gradient scaling/clipping.
  const float g = static_cast<float>(grads[id]) / actual_scale;
  // A shared constant.
  const float one = 1.0f;

  // Compute exponentially-averaged historical gradient.
  const float m1o = alpha * static_cast<float>(moment_1[id]) + (one - alpha) * g;
  const float m1o_corrected = m1o / alpha_correction;

  // Compute exponentially-averaged historical squared gradient.
  const float m2o = beta * static_cast<float>(moment_2[id]) + (one - beta) * g * g;
  const float m2o_corrected = m2o / beta_correction;

  // Compute weight update.
  const float denom = _Sqrt(m2o_corrected) + epsilon;
  const float update = (m1o_corrected / denom) + (lambda * weights[id]);

  const float delta = -static_cast<float>(*eta) * update;

  // Compute the new gradient.
  if (grads_out) {
    grads_out[id] = T_GRAD(delta);
  }

  // Compute the new weight.
  if (weights_out) {
    weights_out[id] = weights[id] + T3(delta);

    if (mixed_precision_weights_out) {
      mixed_precision_weights_out[id] = static_cast<T_MIXED_PRECISION_FP>(weights_out[id]);
    }
  }

  moment_1_out[id] = m1o;
  moment_2_out[id] = m2o;
}

template <typename T1, typename T3, typename T4, typename T_GRAD, typename T_GRAD_NORM, typename T_MIXED_PRECISION_FP>
__global__ void _AdamOptimizer_mode1(
    const T1* eta,
    const T3* weights,
    const T_GRAD* grads,
    const T4* moment_1,
    const T4* moment_2,
    const T3* loss_scale,
    const T_GRAD_NORM* grad_norm,
    const float alpha,
    const float beta,
    const float lambda,
    const float epsilon,
    const float max_norm,
    const float alpha_correction,
    const float beta_correction,
    T4* moment_1_out,
    T4* moment_2_out,
    T3* weights_out,
    T_GRAD* grads_out,
    T_MIXED_PRECISION_FP* mixed_precision_weights_out,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  const float actual_scale = _ComputeGradScale<T3, T_GRAD_NORM, float>(loss_scale, grad_norm, max_norm);

  // Gradient scaling/clipping.
  const float g = static_cast<float>(grads[id]) / actual_scale;
  // A shared constant.
  const float one = 1.0f;

  // Compute exponentially-averaged historical gradient.
  const float m1o = alpha * static_cast<float>(moment_1[id]) + (one - alpha) * g;

  // Compute exponentially-averaged historical squared gradient.
  const float m2o = beta * static_cast<float>(moment_2[id]) + (one - beta) * g * g;

  const float denom = _Sqrt(m2o) + epsilon;

  // Apply bias correction terms on learning rate
  const float step_size = static_cast<float>(*eta) * _Sqrt(beta_correction) / alpha_correction;

  // Huggingface updates weights in the following logic:
  // param' = param - step_size * m1o / denom
  // param_out = param' - original_lr * lambda * param'
  // then param_out = param - step_size * m1o / denom - original_lr * lambda * (param - step_size * m1o / denom)
  // so delta = -step_size * m1o / denom - original_lr * lambda * (param - step_size * m1o / denom)
  const float delta = -step_size * m1o / denom - static_cast<float>(*eta) * lambda * (weights[id] - step_size * m1o / denom);

  // Compute the new gradient.
  if (grads_out) {
    grads_out[id] = T_GRAD(delta);
  }
  
  // Compute the new weight.
  if (weights_out) {
    weights_out[id] = weights[id] + T3(delta);

    if (mixed_precision_weights_out) {
      mixed_precision_weights_out[id] = static_cast<T_MIXED_PRECISION_FP>(weights_out[id]);
    }
  }

  moment_1_out[id] = m1o;
  moment_2_out[id] = m2o;
}

template <typename T1, typename T2, typename T3, typename T4, typename T_GRAD, typename T_GRAD_NORM, typename T_MIXED_PRECISION_FP>
void AdamOptimizerImpl(
    hipStream_t stream,
    const T1* eta,
    const T2 update_count,
    const T3* weights,
    const T_GRAD* grads,
    const T4* moment_1,
    const T4* moment_2,
    const T3* loss_scale,
    const T_GRAD_NORM* grad_norm,
    const float alpha,
    const float beta,
    const float lambda,
    const float epsilon,
    const float max_norm,
    const bool do_bias_correction,
    const int64_t weight_decay_mode,
    T4* moment_1_out,
    T4* moment_2_out,
    T3* weights_out,
    T_GRAD* grads_out,
    T_MIXED_PRECISION_FP* mixed_precision_weights_out,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  // If bias correction coefficients are set to 1s, it's equivalent to disabling bias correction. 
  const float alpha_correction = do_bias_correction ?
    onnxruntime::contrib::compute_bias_correction_coefficient(alpha, update_count) : 1.f;
  const float beta_correction = do_bias_correction ?
    onnxruntime::contrib::compute_bias_correction_coefficient(beta, update_count) : 1.f;
  
  // Currently two modes of Adamw are supported:
  // Mode 0: Pytorch https://pytorch.org/docs/stable/_modules/torch/optim/adamw.html#AdamW,
  //         bias correction is applied on m and v individually,
  //         weight decay is applied before weight is updated.
  // Mode 1: Huggingface https://huggingface.co/transformers/_modules/transformers/optimization.html#AdamW.,
  //         bias correction is applied on learning rate,
  //         weight decay is applied after weight is updated.
  if (weight_decay_mode == 0) {
    _AdamOptimizer_mode0<T1, T3, T4, T_GRAD, T_GRAD_NORM, T_MIXED_PRECISION_FP><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      eta,
      weights,
      grads,
      moment_1,
      moment_2,
      loss_scale,
      grad_norm,
      alpha,
      beta,
      lambda,
      epsilon,
      max_norm,
      alpha_correction,
      beta_correction,

      moment_1_out,
      moment_2_out,
      weights_out,
      grads_out,
      mixed_precision_weights_out,
      N);
  }
  else if (weight_decay_mode == 1) {
    _AdamOptimizer_mode1<T1, T3, T4, T_GRAD, T_GRAD_NORM, T_MIXED_PRECISION_FP><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      eta,
      weights,
      grads,
      moment_1,
      moment_2,
      loss_scale,
      grad_norm,
      alpha,
      beta,
      lambda,
      epsilon,
      max_norm,
      alpha_correction,
      beta_correction,
      moment_1_out,
      moment_2_out,
      weights_out,
      grads_out,
      mixed_precision_weights_out,
      N);
  }
  else {
    // Shouldn't reach here
    ORT_THROW("Unsupported Adamw optimizer mode.");
  }
}

#define SPECIALIZED_AdamOptimizerImpl(T1, T2, T3, T4, T_GRAD, T_GRAD_NORM, T_MIXED_PRECISION_FP)  \
  template void AdamOptimizerImpl(                                                                \
      hipStream_t stream,                                                                        \
      const T1* eta,                                                                              \
      const T2 update_count,                                                                      \
      const T3* weights,                                                                          \
      const T_GRAD* grads,                                                                        \
      const T4* moment_1,                                                                         \
      const T4* moment_2,                                                                         \
      const T3* loss_scale,                                                                       \
      const T_GRAD_NORM* grad_norm,                                                               \
      const float alpha,                                                                          \
      const float beta,                                                                           \
      const float lambda,                                                                         \
      const float epsilon,                                                                        \
      const float max_norm,                                                                       \
      const bool do_bias_correction,                                                              \
      const int64_t weight_decay_mode,                                                            \
      T4* moment_1_out,                                                                           \
      T4* moment_2_out,                                                                           \
      T3* weights_out,                                                                            \
      T_GRAD* grads_out,                                                                          \
      T_MIXED_PRECISION_FP* mixed_precision_weights_out,                                          \
      size_t count);

SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, float, float, float, half)
SPECIALIZED_AdamOptimizerImpl(half, int64_t, float, half, float, float, half)
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, half, float, float, half)
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, float, half, half, half)
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, float, half, float, half)
SPECIALIZED_AdamOptimizerImpl(half, int64_t, float, half, half, half, half)
SPECIALIZED_AdamOptimizerImpl(half, int64_t, float, half, half, float, half)
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, half, half, half, half)
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, half, half, float, half)

#if CUDA_VERSION >= 11000 && (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, float, float, float, hip_bfloat16)
SPECIALIZED_AdamOptimizerImpl(hip_bfloat16, int64_t, float, hip_bfloat16, float, float, hip_bfloat16)
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, hip_bfloat16, float, float, hip_bfloat16)
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, float, hip_bfloat16, hip_bfloat16, hip_bfloat16)
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, float, hip_bfloat16, float, hip_bfloat16)
SPECIALIZED_AdamOptimizerImpl(hip_bfloat16, int64_t, float, hip_bfloat16, hip_bfloat16, hip_bfloat16, hip_bfloat16)
SPECIALIZED_AdamOptimizerImpl(hip_bfloat16, int64_t, float, hip_bfloat16, hip_bfloat16, float, hip_bfloat16)
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, hip_bfloat16, hip_bfloat16, hip_bfloat16, hip_bfloat16)
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, hip_bfloat16, hip_bfloat16, float, hip_bfloat16)
#endif

}  // namespace cuda
}  // namespace onnxruntime
