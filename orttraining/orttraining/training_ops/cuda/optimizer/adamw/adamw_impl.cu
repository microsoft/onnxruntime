#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <algorithm>

#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/cu_inc/common.cuh"

#include "orttraining/training_ops/cuda/optimizer/adamw/adamw_impl.h"
#include "orttraining/training_ops/cuda/optimizer/common.cuh"
#include "orttraining/training_ops/cuda/optimizer/common.h"

namespace onnxruntime {
namespace cuda {

template <typename T_WEIGHT, typename T_GRAD, typename T_MOMENTUM>
__device__ void PrepareMTAData(
    const ChunkGroup<MTA_ADAMW_GROUP_SIZE>& chunks,
    const int& block_idx,
    T_WEIGHT*& weight_chunk_ptr,
    T_GRAD*& grad_chunk_ptr,
    T_MOMENTUM*& momentum_1_chunk_ptr,
    T_MOMENTUM*& momentum_2_chunk_ptr,
    int& chunk_size) {
  const int tensor_idx = chunks.block_index_to_tensor_group_index[block_idx];
  const int tensor_size = chunks.tensor_sizes[tensor_idx];
  T_WEIGHT* weight_tensor_ptr = static_cast<T_WEIGHT*>(chunks.tensor_ptrs[0][tensor_idx]);
  T_GRAD* grad_tensor_ptr = static_cast<T_GRAD*>(chunks.tensor_ptrs[1][tensor_idx]);
  T_MOMENTUM* momentum_1_tensor_ptr = static_cast<T_MOMENTUM*>(chunks.tensor_ptrs[2][tensor_idx]);
  T_MOMENTUM* momentum_2_tensor_ptr = static_cast<T_MOMENTUM*>(chunks.tensor_ptrs[3][tensor_idx]);
  const int chunk_start_idx = chunks.block_index_to_chunk_start_index[block_idx];
  // chunk_size is chunks.chunk_size if the loaded chunk is full. Otherwise (this
  // chunk is the last one in the source tensor), the actual size is determined
  // by the bound of the source tensor.
  chunk_size = min(tensor_size, chunk_start_idx + chunks.chunk_size) - chunk_start_idx;

  weight_chunk_ptr = weight_tensor_ptr + chunk_start_idx;
  grad_chunk_ptr = grad_tensor_ptr + chunk_start_idx;
  momentum_1_chunk_ptr = momentum_1_tensor_ptr + chunk_start_idx;
  momentum_2_chunk_ptr = momentum_2_tensor_ptr + chunk_start_idx;
}

// Torch Adam equivalence.
template <typename T_WEIGHT, typename T_GRAD, typename T_MOMENTUM>
__global__ void AdamWComputeMode0(
    ChunkGroup<MTA_ADAMW_GROUP_SIZE> chunks,
    const float alpha,
    const float beta,
    const float epsilon,
    const float lr,
    const float alpha_correction,
    const float beta_correction,
    const float decay) {
  const int block_idx = blockIdx.x;

  T_WEIGHT* weight_chunk_ptr;
  T_GRAD* grad_chunk_ptr;
  T_MOMENTUM* momentum_1_chunk_ptr;
  T_MOMENTUM* momentum_2_chunk_ptr;

  // TODO(pengwa): unroll this one for better perf.
  int chunk_size;

  PrepareMTAData(chunks, block_idx, weight_chunk_ptr, grad_chunk_ptr,
                 momentum_1_chunk_ptr, momentum_2_chunk_ptr, chunk_size);

#pragma unroll 4
  for (int i = threadIdx.x; i < chunk_size; i += blockDim.x) {
    float w = static_cast<float>(weight_chunk_ptr[i]);
    float g = static_cast<float>(grad_chunk_ptr[i]);
    float m1 = static_cast<float>(momentum_1_chunk_ptr[i]);
    float m2 = static_cast<float>(momentum_2_chunk_ptr[i]);

    // Perform weight decay.
    w = w - (w * lr * decay);

    // Compute exponentially-averaged historical gradient.
    m1 = alpha * m1 + (1.f - alpha) * g;

    // Compute exponentially-averaged historical squared gradient.
    m2 = beta * m2 + (1.f - beta) * g * g;

    // Compute the new weight.
    const float denom = (_Sqrt(m2) / _Sqrt(beta_correction)) + epsilon;
    w = w - (lr * m1) / (alpha_correction * denom);

    // Update the new weight and momentums.
    weight_chunk_ptr[i] = static_cast<T_WEIGHT>(w);
    momentum_1_chunk_ptr[i] = static_cast<T_MOMENTUM>(m1);
    momentum_2_chunk_ptr[i] = static_cast<T_MOMENTUM>(m2);
  }
}

// Huggingface AdamW equivalence.
template <typename T_WEIGHT, typename T_GRAD, typename T_MOMENTUM>
__global__ void AdamWComputeMode1(
    ChunkGroup<MTA_ADAMW_GROUP_SIZE> chunks,
    const float alpha,
    const float beta,
    const float epsilon,
    const float lr,
    const float lr_corrected,
    const float decay) {
  const int block_idx = blockIdx.x;

  T_WEIGHT* weight_chunk_ptr;
  T_GRAD* grad_chunk_ptr;
  T_MOMENTUM* momentum_1_chunk_ptr;
  T_MOMENTUM* momentum_2_chunk_ptr;
  int chunk_size;

  PrepareMTAData(chunks, block_idx, weight_chunk_ptr, grad_chunk_ptr,
                 momentum_1_chunk_ptr, momentum_2_chunk_ptr, chunk_size);

#pragma unroll 4
  for (int i = threadIdx.x; i < chunk_size; i += blockDim.x) {
    float w = static_cast<float>(weight_chunk_ptr[i]);
    float g = static_cast<float>(grad_chunk_ptr[i]);
    float m1 = static_cast<float>(momentum_1_chunk_ptr[i]);
    float m2 = static_cast<float>(momentum_2_chunk_ptr[i]);

    // Compute exponentially-averaged historical gradient.
    m1 = alpha * m1 + (1.f - alpha) * g;

    // Compute exponentially-averaged historical squared gradient.
    m2 = beta * m2 + (1.f - beta) * g * g;

    float denom = _Sqrt(m2) + epsilon;
    w = w - (lr_corrected * m1 / denom);

    // Perform weight decay.
    w = w - (lr * decay * w);

    // Update the new weight and momentums.
    weight_chunk_ptr[i] = static_cast<T_WEIGHT>(w);
    momentum_1_chunk_ptr[i] = static_cast<T_MOMENTUM>(m1);
    momentum_2_chunk_ptr[i] = static_cast<T_MOMENTUM>(m2);
  }
}

template <typename T_WEIGHT, typename T_GRAD, typename T_MOMENTUM>
void AdamWMTAFunctor<T_WEIGHT, T_GRAD, T_MOMENTUM>::operator()(
    hipStream_t stream,
    ChunkGroup<MTA_ADAMW_GROUP_SIZE> chunks,
    const float alpha,
    const float beta,
    const float epsilon,
    const float lr,
    const float decay,
    const int64_t adam_mode,
    const int64_t correct_bias,
    const int64_t update_count) {
  const int block_count = chunks.chunk_count;
  const int thread_count = ChunkGroup<MTA_ADAMW_GROUP_SIZE>::thread_count_per_block;

  float alpha_correction = 1.f, beta_correction = 1.f;
  float lr_corrected = lr;
  if (correct_bias == 1) {
    // Notes:
    // > there is a minor difference compared with Apex's implementation,
    //   which uses double storing corrections before casting to float passing to kernels.
    // > std::pow(float, int) return double since C++11, so we cast back to float.
    alpha_correction = 1.f - static_cast<float>(std::pow(alpha, update_count));
    beta_correction = 1.f - static_cast<float>(std::pow(beta, update_count));
    lr_corrected *= std::sqrt(beta_correction) / alpha_correction;
  }

  // Currently two kinds of AdamW supported:
  // Mode 0: Pytorch https://pytorch.org/docs/stable/_modules/torch/optim/adamw.html#AdamW,
  //         bias correction is applied on m and v individually,
  //         weight decay is applied before weight is updated.
  // Mode 1: Huggingface https://github.com/huggingface/transformers/blob/d91841315aab55cf1347f4eb59332858525fad0f/
  //         src/transformers/optimization.py,
  //         bias correction is applied on learning rate, then use lr_corrected for subsequent computations.
  //         weight decay is applied after weight is updated.
  if (adam_mode == 0) {
    AdamWComputeMode0<T_WEIGHT, T_GRAD, T_MOMENTUM><<<block_count, thread_count, 0, stream>>>(
        chunks, alpha, beta, epsilon, lr, alpha_correction, beta_correction, decay);
  } else if (adam_mode == 1) {
    AdamWComputeMode1<T_WEIGHT, T_GRAD, T_MOMENTUM><<<block_count, thread_count, 0, stream>>>(
        chunks, alpha, beta, epsilon, lr, lr_corrected, decay);
  } else {
    ORT_THROW("Unsupported Adamw optimizer mode.");
  }
}

#define INSTANTIATE_ADAMMTA_FUNCTOR(T_WEIGHT, T_GRAD, T_MOMENTUM)           \
  template void AdamWMTAFunctor<T_WEIGHT, T_GRAD, T_MOMENTUM>::operator()(  \
      hipStream_t stream,                                                  \
      ChunkGroup<MTA_ADAMW_GROUP_SIZE> chunks,                              \
      const float alpha,                                                    \
      const float beta,                                                     \
      const float epsilon,                                                  \
      const float lr,                                                       \
      const float decay,                                                    \
      const int64_t adam_mode,                                              \
      const int64_t correct_bias,                                           \
      const int64_t update_count);                                          \
                                                                            \
  template __global__ void AdamWComputeMode0<T_WEIGHT, T_GRAD, T_MOMENTUM>( \
      ChunkGroup<MTA_ADAMW_GROUP_SIZE> chunks,                              \
      const float alpha,                                                    \
      const float beta,                                                     \
      const float epsilon,                                                  \
      const float lr,                                                       \
      const float alpha_correction,                                         \
      const float beta_correction,                                          \
      const float decay);                                                   \
                                                                            \
  template __global__ void AdamWComputeMode1<T_WEIGHT, T_GRAD, T_MOMENTUM>( \
      ChunkGroup<MTA_ADAMW_GROUP_SIZE> chunks,                              \
      const float alpha,                                                    \
      const float beta,                                                     \
      const float epsilon,                                                  \
      const float lr,                                                       \
      const float lr_corrected,                                             \
      const float decay);

INSTANTIATE_ADAMMTA_FUNCTOR(float, float, float);

#undef INSTANTIATE_ADAMMTA_FUNCTOR

}  // namespace cuda
}  // namespace onnxruntime
