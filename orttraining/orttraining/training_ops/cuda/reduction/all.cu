#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "orttraining/training_ops/cuda/reduction/all.h"

#include <thrust/logical.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#ifdef _WIN32
#pragma warning(disable : 4244)
#endif
namespace onnxruntime {
namespace cuda {

__global__ void assign_true(bool* ptr) {
  *ptr = true;
}

__global__ void assign_false(bool* ptr) {
  *ptr = false;
}

template<>
void LaunchAllKernel(hipStream_t stream, const bool* data, const int size, bool* output) {
  if(thrust::all_of(thrust::cuda::par.on(stream), data, data + size, thrust::identity<bool>())) {
    assign_true<<<1, 1, 0, stream>>>(output);
  }
  else
  {
    assign_false<<<1, 1, 0, stream>>>(output);
  }
}

}  // namespace cuda
}  // namespace onnxruntime
