#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "orttraining/training_ops/cuda/reduction/reduction_all.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/atomic/common.cuh"
#include "core/providers/cuda/reduction/reduction_utils.cuh"
#include "core/providers/cuda/shared_inc/accumulation_type.h"

namespace onnxruntime {
namespace cuda {

template <typename Tin, typename Tout>
__global__ void ScalarSqrtKernel(Tin* input, Tout* output) {
  *output = (Tout)_Sqrt(*input);
}

template <typename Tin, typename Tout>
void ScalarSqrt(Tin* input, Tout* output) {
  ScalarSqrtKernel<<<1, 1, 0>>>(input, output);
}

template void ScalarSqrt(float* input, float* output);
template void ScalarSqrt(half* input, half* output);
template void ScalarSqrt(float* input, half* output);
#if CUDA_VERSION >= 11000 && (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
template void ScalarSqrt(hip_bfloat16* input, hip_bfloat16* output);
template void ScalarSqrt(float* input, hip_bfloat16* output);
#endif

template <typename TIn, typename TOut, typename TBuf, typename TInOp, typename TOutOp>
__launch_bounds__(ChunkGroup<1>::thread_count_per_block)
__global__ void MultiTensorReduceKernel(ChunkGroup<1> chunk_group, TOut* output) {
  const int group_index = chunk_group.block_index_to_tensor_group_index[blockIdx.x];
  const int tensor_size = chunk_group.tensor_sizes[group_index];
  const int chunk_size = chunk_group.chunk_size;
  const int chunk_start = chunk_group.block_index_to_chunk_start_index[blockIdx.x];
  const TIn* w = reinterpret_cast<const TIn*>(chunk_group.tensor_ptrs[0][group_index]) + chunk_start;
  TOut* w_norm = output;

  TBuf w_sum = TBuf(0.f);
  constexpr int load_count_per_thread = 4;
  for (int i = threadIdx.x; i < chunk_size && i + chunk_start < tensor_size; i += blockDim.x * load_count_per_thread) {
#pragma unroll
    for (int j = 0; j < load_count_per_thread; ++j) {
      const int index_in_chunk = i + j * blockDim.x;
      const int index_in_tensor = chunk_start + index_in_chunk;
      if (index_in_chunk < chunk_size && index_in_tensor < tensor_size) {
        const TBuf w_element = TBuf(w[index_in_chunk]);
        w_sum += TInOp()(w_element);
      }
    }
  }

// Thread count in a block must be a multiple of GPU_WARP_SIZE.
#pragma unroll
  for (int stride = GPU_WARP_SIZE / 2; stride > 0; stride /= 2) {
    w_sum += WARP_SHFL_DOWN(w_sum, stride);
  }

  const int warp_count_in_block = blockDim.x / GPU_WARP_SIZE;
  const int lid = threadIdx.x % GPU_WARP_SIZE;
  const int wid = threadIdx.x / GPU_WARP_SIZE;

  // Shape is 2 x warp_count_in_block.
  extern __shared__ unsigned char shared_memory_[];
  TBuf* shared_memory = reinterpret_cast<TBuf*>(shared_memory_);

  if (lid == 0) {
    shared_memory[wid] = w_sum;
  }

  __syncthreads();

#pragma unroll
  for (int stride = warp_count_in_block / 2; stride > 0; stride /= 2) {
    if (threadIdx.x < stride) {
      shared_memory[threadIdx.x] += shared_memory[threadIdx.x + stride];
    }
    __syncthreads();
  }

  if (threadIdx.x == 0) {
    atomic_add(w_norm, TOutOp()(TOut(shared_memory[0])));
  }
}

template <typename TIn, typename TOut, typename TBuf, typename TInOp, typename TOutOp>
void MultiTensorReduce(ChunkGroup<1> chunk_group, TOut* output) {
  // thread count per block.
  constexpr int thread_count = ChunkGroup<1>::thread_count_per_block;
  // shared memory's size per block.
  const int shared_memory_size = thread_count / GPU_WARP_SIZE * sizeof(TBuf);

  // Enforce assumptions used inside this reduction CUDA kernel.
  ORT_ENFORCE(thread_count % GPU_WARP_SIZE == 0);
  ORT_ENFORCE((thread_count & (thread_count - 1)) == 0);

  MultiTensorReduceKernel<TIn, TOut, TBuf, TInOp, TOutOp><<<chunk_group.chunk_count, thread_count, shared_memory_size>>>(chunk_group, output);
}

template <typename TIn, typename TOut>
void MultiTensorReduceL2<TIn, TOut>::operator()(ChunkGroup<1> chunk_group, TOut* output) {
  using TBuf = AccumulationType_t<TIn>;
  MultiTensorReduce<TIn, TOut, TBuf, Square, Identity>(chunk_group, output);
}

#define INSTANTIATE_MULTI_TENSOR_REDUCTION_L2_FUNCTOR(TIn, TOut) \
  template void MultiTensorReduceL2<TIn, TOut>::operator()(ChunkGroup<1> chunk_group, TOut* output);

INSTANTIATE_MULTI_TENSOR_REDUCTION_L2_FUNCTOR(double, float)
INSTANTIATE_MULTI_TENSOR_REDUCTION_L2_FUNCTOR(float, float)
INSTANTIATE_MULTI_TENSOR_REDUCTION_L2_FUNCTOR(half, float)
INSTANTIATE_MULTI_TENSOR_REDUCTION_L2_FUNCTOR(float, half)
INSTANTIATE_MULTI_TENSOR_REDUCTION_L2_FUNCTOR(half, half)
#if CUDA_VERSION >= 11000 && (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
INSTANTIATE_MULTI_TENSOR_REDUCTION_L2_FUNCTOR(hip_bfloat16, float)
INSTANTIATE_MULTI_TENSOR_REDUCTION_L2_FUNCTOR(float, hip_bfloat16)
INSTANTIATE_MULTI_TENSOR_REDUCTION_L2_FUNCTOR(hip_bfloat16, hip_bfloat16)
#endif

}  // namespace cuda
}  // namespace onnxruntime