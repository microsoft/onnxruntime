#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "orttraining/training_ops/cuda/activation/bias_gelu_grad_impl.h"

#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/shared_inc/fast_divmod.h"
#include "orttraining/training_ops/cpu/activation/gelu_computation_mode.h"
#include "orttraining/training_ops/cuda/activation/gelu_grad_impl_common.cuh"

namespace onnxruntime {
namespace cuda {

template <typename T, typename GeluComputationMode, int num_consecutive_elements_per_group, int num_groups_per_thread>
__global__ void BiasGeluGradDxKernel(int64_t bias_size, const T* dY, const T* X, const T* B, T* dX) {
  const int64_t input_base_idx = bias_size * blockIdx.x + num_consecutive_elements_per_group * threadIdx.x;
  const int64_t bias_base_idx = num_consecutive_elements_per_group * threadIdx.x;
  const int64_t group_stride = num_consecutive_elements_per_group * blockDim.x;

#pragma unroll
  for (int group_idx = 0; group_idx < num_groups_per_thread; ++group_idx) {
#pragma unroll
    for (int element_idx = 0; element_idx < num_consecutive_elements_per_group; ++element_idx) {
      const auto offset = group_stride * group_idx + element_idx;
      const auto input_idx = input_base_idx + offset, bias_idx = bias_base_idx + offset;
      if (bias_idx < bias_size) {
        dX[input_idx] = ComputeGeluGradScalar(dY[input_idx], X[input_idx] + B[bias_idx], GeluComputationMode{});
      }
    }
  }
}

template <typename T, typename GeluComputationMode>
void LaunchBiasGeluGradDxKernel(
    int64_t input_size, int64_t bias_size,
    const T* dY, const T* X, const T* B, T* dX) {
  // each block handles bias_size elements
  // there are input_size / bias_size blocks
  constexpr int num_consecutive_elements_per_group = 4;
  constexpr int num_groups_per_thread = 4;

  const auto num_threads_per_block = CeilDiv(bias_size, num_consecutive_elements_per_group * num_groups_per_thread);
  const auto num_blocks_per_grid = input_size / bias_size;

  BiasGeluGradDxKernel<T, GeluComputationMode, num_consecutive_elements_per_group, num_groups_per_thread>
      <<<num_blocks_per_grid, num_threads_per_block>>>(bias_size, dY, X, B, dX);
}

// explicit instantiations
#define SPECIALIZED_BIAS_GELU_GRAD_IMPL(T, GeluComputationMode)     \
  template void LaunchBiasGeluGradDxKernel<T, GeluComputationMode>( \
      int64_t input_size, int64_t bias_size,                        \
      const T* dY, const T* X, const T* B, T* dX)

SPECIALIZED_BIAS_GELU_GRAD_IMPL(half, gelu_computation_mode::Default);
SPECIALIZED_BIAS_GELU_GRAD_IMPL(float, gelu_computation_mode::Default);
SPECIALIZED_BIAS_GELU_GRAD_IMPL(double, gelu_computation_mode::Default);

SPECIALIZED_BIAS_GELU_GRAD_IMPL(half, gelu_computation_mode::Approximation);
SPECIALIZED_BIAS_GELU_GRAD_IMPL(float, gelu_computation_mode::Approximation);
SPECIALIZED_BIAS_GELU_GRAD_IMPL(double, gelu_computation_mode::Approximation);

#undef SPECIALIZED_BIAS_GELU_GRAD_IMPL

}  // namespace cuda
}  // namespace onnxruntime
