#include "hip/hip_runtime.h"
/**
* Copyright (c) 2016-present, Facebook, Inc.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

/* Modifications Copyright (c) Microsoft. */

#include "core/providers/cuda/cu_inc/common.cuh"
#include "orttraining/training_ops/cuda/nn/dropout_impl.h"
#include <hiprand/hiprand_kernel.h>
#include <algorithm>

namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void DropoutGradientKernel(
    const int64_t N,
    const T* dY_data,
    const bool* mask_data,
    const T scale,
    T* dX_data) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  dX_data[id] = dY_data[id] * T(mask_data[id]) * scale;
}

template <typename T>
void DropoutGradientKernelImpl(
    const int64_t N,
    const T* dY_data,
    const bool* mask_data,
    const float ratio,
    T* dX_data) {
  if (ratio == 0.0f) {
    if (dY_data != dX_data) {
      CUDA_CALL_THROW(hipMemcpyAsync(dX_data, dY_data, N * sizeof(T), hipMemcpyDeviceToDevice));
    }
  } else {
    const float scale = 1.f / (1.f - ratio);
    const int blocksPerGrid = (N + GridDim::maxThreadsPerBlock - 1) / GridDim::maxThreadsPerBlock;
    DropoutGradientKernel<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(N, dY_data, mask_data, T(scale), dX_data);
  }
}

#define SPECIALIZED_DROPOUT_GRAD_IMPL(T)   \
  template void DropoutGradientKernelImpl( \
      const int64_t N,                     \
      const T* dY_data,                    \
      const bool* mask_data,               \
      const float scale,                   \
      T* dX_data);

SPECIALIZED_DROPOUT_GRAD_IMPL(float)
SPECIALIZED_DROPOUT_GRAD_IMPL(double)
SPECIALIZED_DROPOUT_GRAD_IMPL(half)

}  // namespace cuda
}  // namespace onnxruntime
