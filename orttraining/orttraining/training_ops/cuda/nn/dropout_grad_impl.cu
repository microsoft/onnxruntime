#include "hip/hip_runtime.h"
/**
* Copyright (c) 2016-present, Facebook, Inc.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

/* Modifications Copyright (c) Microsoft. */

#include "core/providers/cuda/cu_inc/common.cuh"
#include "orttraining/training_ops/cuda/nn/dropout_grad_impl.h"
#include <hiprand/hiprand_kernel.h>
#include <algorithm>

namespace onnxruntime {
namespace cuda {

template <typename T, int NumThreadsPerBlock, int NumElementsPerThread>
__global__ void DropoutGradientKernel(
    const int64_t N,
    const T* dY_data,
    const bool* mask_data,
    const float scale,
    T* dX_data) {
  CUDA_LONG id = NumElementsPerThread * NumThreadsPerBlock * blockIdx.x + threadIdx.x;
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      dX_data[id] = T(float(dY_data[id]) * mask_data[id] * scale);
      id += NumThreadsPerBlock;
    }
  }
}

template <typename T>
void DropoutGradientKernelImpl(
    hipStream_t stream,
    const int64_t N,
    const T* dY_data,
    const bool* mask_data,
    const float ratio,
    T* dX_data) {
  if (ratio == 0.0f) {
    if (dY_data != dX_data) {
      CUDA_CALL_THROW(hipMemcpyAsync(dX_data, dY_data, N * sizeof(T), hipMemcpyDeviceToDevice, stream));
    }
  } else {
    const float scale = 1.f / (1.f - ratio);
    const int blocksPerGrid = static_cast<int>(CeilDiv(N, GridDim::maxThreadsPerBlock * GridDim::maxElementsPerThread));
    DropoutGradientKernel<T, GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread>
                         <<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(N, dY_data, mask_data, scale, dX_data);
  }
}

#define SPECIALIZED_DROPOUT_GRAD_IMPL(T)   \
  template void DropoutGradientKernelImpl( \
      hipStream_t stream,           \
      const int64_t N,                     \
      const T* dY_data,                    \
      const bool* mask_data,               \
      const float scale,                   \
      T* dX_data);

SPECIALIZED_DROPOUT_GRAD_IMPL(float)
SPECIALIZED_DROPOUT_GRAD_IMPL(double)
SPECIALIZED_DROPOUT_GRAD_IMPL(half)
#if CUDA_VERSION >= 11000 && (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
SPECIALIZED_DROPOUT_GRAD_IMPL(hip_bfloat16)
#endif

}  // namespace cuda
}  // namespace onnxruntime
