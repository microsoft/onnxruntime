#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_fp16.h>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "orttraining/training_ops/cuda/math/isfinite.cuh"

namespace onnxruntime {
namespace cuda {

template <typename TSrc>
__global__ void _IsFinite(const TSrc* input, bool* output, CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  output[id] = _IsFiniteScalar(input[id]);
}

template <typename TSrc>
void IsFinite(hipStream_t stream, const TSrc* input, bool* output, size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _IsFinite<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(input, output, N);
}

#define SPECIALIZE_ISFINITE_IMPL(T) \
template void IsFinite(hipStream_t stream, const T* input, bool* output, size_t count);

SPECIALIZE_ISFINITE_IMPL(half)
SPECIALIZE_ISFINITE_IMPL(float)
SPECIALIZE_ISFINITE_IMPL(double)

template <typename TSrc>
__global__ void IsAllFiniteMultiTensorImpl(ChunkGroup<1> chunks, bool* output) {
  const int block_idx = blockIdx.x;
  const int tensor_idx = chunks.block_index_to_tensor_group_index[block_idx];
  const int tensor_size = chunks.tensor_sizes[tensor_idx];
  const TSrc* tensor_ptr = static_cast<TSrc*>(chunks.tensor_ptrs[0][tensor_idx]);
  const int chunk_start_idx = chunks.block_index_to_chunk_start_index[block_idx];
  // chunk_size is chunks.chunk_size if the loaded chunk is full. Otherwise (this
  // chunk is the last one in the source tensor), the actual size is determined
  // by the bound of the source tensor.
  const int chunk_size = min(tensor_size, chunk_start_idx + chunks.chunk_size) - chunk_start_idx;

  const TSrc* chunk_ptr = tensor_ptr + chunk_start_idx;
  bool result = true;
#pragma unroll(4)
  for (int i = threadIdx.x; i < chunk_size; i += blockDim.x) {
    result &= _IsFiniteScalar(chunk_ptr[i]);
  }

  if (!result) {
    *output = false;
  }
}

template <typename T>
void IsAllFiniteFunctor<T>::operator()(hipStream_t stream, ChunkGroup<1> chunks, bool* output) {
  const int block_count = chunks.chunk_count;
  const int thread_count = ChunkGroup<1>::thread_count_per_block;
  IsAllFiniteMultiTensorImpl<T><<<block_count, thread_count, 0, stream>>>(chunks, output);
}

#define INSTANTIATE_ISALLFINITE_FUNCTOR(T) \
  template void IsAllFiniteFunctor<T>::operator()(hipStream_t stream, ChunkGroup<1> chunks, bool* output);

INSTANTIATE_ISALLFINITE_FUNCTOR(half)
INSTANTIATE_ISALLFINITE_FUNCTOR(float)
INSTANTIATE_ISALLFINITE_FUNCTOR(double)

}  // namespace cuda
}  // namespace onnxruntime