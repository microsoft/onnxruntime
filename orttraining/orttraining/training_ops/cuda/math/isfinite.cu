#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_fp16.h>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "contrib_ops/cuda/math/isfinite.cuh"

namespace onnxruntime {
namespace cuda {

template <typename TSrc>
__global__ void _IsFinite(const TSrc* input, bool* output, CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  output[id] = IsFiniteScalar(input[id]);
}

template <typename TSrc>
void IsFinite(hipStream_t stream, const TSrc* input, bool* output, size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _IsFinite<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(input, output, N);
}

#define SPECIALIZE_ISFINITE_IMPL(T) \
  template void IsFinite(hipStream_t stream, const T* input, bool* output, size_t count);

SPECIALIZE_ISFINITE_IMPL(half)
SPECIALIZE_ISFINITE_IMPL(float)
SPECIALIZE_ISFINITE_IMPL(double)

}  // namespace cuda
}  // namespace onnxruntime