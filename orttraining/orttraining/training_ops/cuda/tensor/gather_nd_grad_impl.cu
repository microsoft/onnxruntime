#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "orttraining/training_ops/cuda/tensor/gather_nd_grad_impl.h"

#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/atomic/common.cuh"

namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void _GatherNDGradKernel(
    const size_t num_slices,
    const T* update_data,
    T* output_data,
    const size_t slice_size,
    const int64_t* slice_offsets) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(i, num_slices * slice_size);
  uint64_t slice_offset = slice_offsets[i / slice_size];
  size_t j = i % slice_size;
  atomic_add(output_data + slice_offset + j, update_data[i]);
};

template <typename T>
void GatherNDGradImpl(
    const size_t num_slices,
    const void* update_data,
    void* output_data,
    const size_t slice_size,
    const int64_t* input_slice_offsets_data) {
  const auto blocks_per_grid = CeilDiv(num_slices * slice_size, GridDim::maxThreadsPerBlock);
  _GatherNDGradKernel<T><<<blocks_per_grid, GridDim::maxThreadsPerBlock, 0>>>(
      num_slices, static_cast<const T*>(update_data), static_cast<T*>(output_data), slice_size, input_slice_offsets_data);
}

#define SPECIALIZED_GRAD_IMPL(T) \
  template void GatherNDGradImpl<T>(const size_t num_slices, const void* update_data, void* output_data, const size_t slice_size, const int64_t* input_slice_offsets_data)

SPECIALIZED_GRAD_IMPL(float);
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
SPECIALIZED_GRAD_IMPL(half);
SPECIALIZED_GRAD_IMPL(double);
#endif
#if CUDA_VERSION >= 11000 && (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
SPECIALIZED_GRAD_IMPL(hip_bfloat16);
#endif

}  // namespace cuda
}  // namespace onnxruntime
