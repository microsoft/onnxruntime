#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.
#ifdef _WIN32
#pragma warning(disable : 4244)
#endif
#include "orttraining/training_ops/cuda/tensor/gather_nd_grad_impl.h"

#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/atomic/common.cuh"

namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void _GatherNDGradKernel(
    const size_t num_slices,
    const T* update_data,
    T* output_data,
    const size_t slice_size,
    const int64_t* slice_offsets) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(i, num_slices * slice_size);
  uint64_t slice_offset = slice_offsets[i / slice_size];
  size_t j = i % slice_size;
  atomic_add(output_data + slice_offset + j, update_data[i]);
};

template <typename T>
void GatherNDGradImpl(
    hipStream_t stream,
    const size_t num_slices,
    const void* update_data,
    void* output_data,
    const size_t slice_size,
    const int64_t* input_slice_offsets_data) {
  const unsigned int blocks_per_grid = static_cast<unsigned int>(CeilDiv(num_slices * slice_size, GridDim::maxThreadsPerBlock));
  _GatherNDGradKernel<T><<<blocks_per_grid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      num_slices, static_cast<const T*>(update_data), static_cast<T*>(output_data), slice_size, input_slice_offsets_data);
}

#define SPECIALIZED_GRAD_IMPL(T) \
  template void GatherNDGradImpl<T>(hipStream_t stream, const size_t num_slices, const void* update_data, void* output_data, const size_t slice_size, const int64_t* input_slice_offsets_data)

SPECIALIZED_GRAD_IMPL(float);
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
SPECIALIZED_GRAD_IMPL(half);
SPECIALIZED_GRAD_IMPL(double);
#endif
#if CUDA_VERSION >= 11000 && (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
SPECIALIZED_GRAD_IMPL(hip_bfloat16);
#endif

}  // namespace cuda
}  // namespace onnxruntime
