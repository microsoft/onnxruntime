#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "orttraining/training_ops/cuda/tensor/pad_and_unflatten_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"

namespace onnxruntime {
namespace cuda {

constexpr int kBlockSize = 256;
constexpr int kNumUnroll = 4;

template <typename T>
__global__ void FillOutputWithIndexKernel(const CUDA_LONG N,
                                          const fast_divmod output_element_stride_fdm,
                                          const int64_t index_value_upper_bound,
                                          const T* input_data,
                                          const int64_t* indices_data,
                                          T* output_data) {
  CUDA_LONG idx = blockDim.x * blockIdx.x + threadIdx.x;
  CUDA_LONG id = idx * kNumUnroll;

  T input[kNumUnroll];
  if (id < N) {
#pragma unroll
    for (int i = 0; i < kNumUnroll; ++i) {
      CUDA_LONG li = id + i;
      if (li < N) {
        input[i] = input_data[li];
      }
    }
  }

#pragma unroll
  for (int i = 0; i < kNumUnroll; ++i) {
    CUDA_LONG li = id + i;
    if (li < N) {
      int row_index, col_index;
      output_element_stride_fdm.divmod(li, row_index, col_index);
      assert(indices_data[row_index] < index_value_upper_bound);
      output_data[indices_data[row_index] * output_element_stride_fdm.d_ + col_index] = input[i];
    }
  }
}

template <typename T>
void PadAndUnflattenImpl(hipStream_t stream,
                         const int64_t total_element_count,
                         const fast_divmod output_element_stride_fdm,
                         const int64_t index_value_upper_bound,
                         const T* input_data,
                         const int64_t* indices_data,
                         T* output_data) {
  const int blocksPerGrid = static_cast<int>(CeilDiv(total_element_count, kBlockSize * kNumUnroll));
  FillOutputWithIndexKernel<T><<<blocksPerGrid, kBlockSize, 0, stream>>>(
      static_cast<CUDA_LONG>(total_element_count),
      output_element_stride_fdm,
      index_value_upper_bound,
      input_data,
      indices_data,
      output_data);
}

#define SPECIALIZED_RESTORE_FROM_MASK_IMPL(T)                                       \
  template void PadAndUnflattenImpl<T>(hipStream_t stream,                         \
                                       const int64_t total_element_count,           \
                                       const fast_divmod output_element_stride_fdm, \
                                       const int64_t index_value_upper_bound,       \
                                       const T* input_data,                         \
                                       const int64_t* indices_data,                 \
                                       T* output_data);

SPECIALIZED_RESTORE_FROM_MASK_IMPL(float)
SPECIALIZED_RESTORE_FROM_MASK_IMPL(double)
SPECIALIZED_RESTORE_FROM_MASK_IMPL(half)
SPECIALIZED_RESTORE_FROM_MASK_IMPL(BFloat16)

#undef SPECIALIZED_RESTORE_FROM_MASK_IMPL

}  // namespace cuda
}  // namespace onnxruntime
