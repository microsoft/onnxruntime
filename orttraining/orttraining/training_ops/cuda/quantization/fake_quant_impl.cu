#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "orttraining/training_ops/cuda/quantization/fake_quant_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"

namespace onnxruntime {
namespace cuda {

namespace {
constexpr int NumElementsPerThread = GridDim::maxElementsPerThread;
constexpr int NumThreadsPerBlock = GridDim::maxThreadsPerBlock;
}  // namespace

template <typename T>
__global__ void FakeQuantPerTensorImpl(const int64_t num_elements, const T* input_data, const T quant_scale,
                                       const T quant_zero_point, const int64_t quant_min, const int64_t quant_max,
                                       T* fake_quantized_data, bool* quantization_mask_data) {
  CUDA_LONG start = NumElementsPerThread * NumThreadsPerBlock * blockIdx.x + threadIdx.x;

  T values[NumElementsPerThread];
  T fake_quantized_values[NumElementsPerThread];
  bool mask_values[NumElementsPerThread];

  CUDA_LONG idx = start;
  // Load
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (idx < num_elements) {
      values[i] = input_data[idx];
      idx += NumThreadsPerBlock;
    }
  }

  // Compute
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    // Quantize
    const auto quantized_value = std::nearbyint(values[i] / quant_scale) + quant_zero_point;
    // Clamp and De-Quantize
    fake_quantized_values[i] =
        (fminf(quant_max, fmaxf(quant_min, quantized_value)) - quant_zero_point) * quant_scale;
    // Compute mask
    mask_values[i] = (quant_min <= quantized_value && quantized_value <= quant_max);
  }

  // Write
  idx = start;
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (idx < num_elements) {
      fake_quantized_data[idx] = fake_quantized_values[i];
      quantization_mask_data[idx] = mask_values[i];
      idx += NumThreadsPerBlock;
    }
  }
}

template <typename T>
void FakeQuantPerTensor(hipStream_t stream, const int64_t num_elements, const T* input_data, const T quant_scale,
                        const T quant_zero_point, const int64_t quant_min, const int64_t quant_max,
                        T* fake_quantized_data, bool* quantization_mask_data) {
  int blocksPerGrid =
      static_cast<int>(CeilDiv(num_elements, NumThreadsPerBlock * NumElementsPerThread));
  FakeQuantPerTensorImpl<T><<<blocksPerGrid, NumThreadsPerBlock, 0, stream>>>(
      num_elements, input_data, quant_scale, quant_zero_point,
      quant_min, quant_max, fake_quantized_data, quantization_mask_data);
}

#define SPECIALIZED_FAKEQUANT_IMPL(T)                                                    \
  template void FakeQuantPerTensor<T>(hipStream_t stream, const int64_t num_elements,   \
                                      const T* input_data, const T quant_scale,          \
                                      const T quant_zero_point, const int64_t quant_min, \
                                      const int64_t quant_max, T* fake_quantized_data,   \
                                      bool* quantization_mask_data);

SPECIALIZED_FAKEQUANT_IMPL(float)

#undef SPECIALIZED_FAKEQUANT_IMPL

}  // namespace cuda
}  // namespace onnxruntime
