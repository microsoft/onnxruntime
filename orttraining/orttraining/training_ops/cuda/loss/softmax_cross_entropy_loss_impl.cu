#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "orttraining/training_ops/cuda/loss/softmax_cross_entropy_loss_impl.h"
#include "core/providers/cuda/cuda_common.h"

namespace onnxruntime {
namespace cuda {

template <typename T, typename Tin>
__global__ void _ComputeWeightsSoftmaxCrossEntropy(
    T* weight_data_nd,
    const Tin* label_data,
    const T* weight_data,
    CUDA_LONG N_D,
    CUDA_LONG C,
    CUDA_LONG ignore_index) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(i, N_D);
  if (label_data[i] != ignore_index) {
    CUDA_KERNEL_ASSERT(label_data[i] >= 0 && label_data[i] < C);
    weight_data_nd[i] = weight_data != nullptr ? weight_data[label_data[i]] : 1;
  }
}

template <typename T, typename Tin>
void ComputeWeightsSoftmaxCrossEntropyImpl(
    hipStream_t stream,
    const Tin* label,
    const T* weight,
    size_t count,
    size_t label_depth,
    int64_t ignore_index,
    T* weight_data_nd) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N_D = static_cast<CUDA_LONG>(count);
  CUDA_LONG C = static_cast<CUDA_LONG>(label_depth);
  CUDA_LONG II = static_cast<CUDA_LONG>(ignore_index);
  _ComputeWeightsSoftmaxCrossEntropy<T, Tin><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      weight_data_nd,
      label,
      weight,
      N_D,
      C,
      II);
}

template <typename T, typename Tin>
__global__ void _WeightedSoftmaxCrossEntropyLoss(
    const T* log_prob_data,
    const Tin* label_data,
    const T* weight_data,
    const T* normalize_factor_data,
    T* output_data,
    CUDA_LONG N_D,
    CUDA_LONG C,
    CUDA_LONG II) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(i, N_D);
  if (II == label_data[i]) {
    output_data[i] = 0;
  } else {
    CUDA_KERNEL_ASSERT(label_data[i] >= 0 && label_data[i] < C);
    output_data[i] = -log_prob_data[i * C + label_data[i]] * weight_data[i] / (*normalize_factor_data);
  }
}

template <typename T, typename Tin>
void SoftmaxCrossEntropyLossImpl(
    hipStream_t stream,
    const T* log_prob,
    const Tin* label,
    const T* weight,
    const T* normalize_factor,
    size_t count,
    size_t label_depth,
    int64_t ignore_index,
    T* output_data) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N_D = static_cast<CUDA_LONG>(count);
  CUDA_LONG C = static_cast<CUDA_LONG>(label_depth);
  CUDA_LONG II = static_cast<CUDA_LONG>(ignore_index);
  _WeightedSoftmaxCrossEntropyLoss<T, Tin><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      log_prob,
      label,
      weight,
      normalize_factor,
      output_data,
      N_D,
      C,
      II);
}

#define SPECIALIZED_IMPL_SoftMaxEntropyLossImpl(T, Tin) \
  template void SoftmaxCrossEntropyLossImpl(            \
      hipStream_t stream,                              \
      const T* log_prob,                                \
      const Tin* label,                                 \
      const T* weight,                                  \
      const T* normalize_factor,                        \
      size_t count,                                     \
      size_t label_depth,                               \
      int64_t ignore_index,                             \
      T* output_data);

SPECIALIZED_IMPL_SoftMaxEntropyLossImpl(float, int32_t)
SPECIALIZED_IMPL_SoftMaxEntropyLossImpl(float, int64_t)

template <typename T, typename Tin>
__global__ void _WeightedSoftmaxCrossEntropyLossGrad(
    const T* dY,
    const T* log_prob,
    const Tin* label,
    const T* weight,
    const T* normalize_factor,
    T* output_data,
    CUDA_LONG N_D,
    CUDA_LONG C) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(i, N_D * C);

  int row = i / C;
  int d = i % C;
  CUDA_KERNEL_ASSERT(weight[row] == 0 || (label[row] >= 0 && label[row] < C));
  if(0 == *normalize_factor){
    // normalize_factor is sum of labels' weights. Because zero 
    // sum implies all weights are 0, the loss function should 
    // be constant 0 and its corresponding gradient should be 0 as well.
    output_data[i] = 0;
  } else {
    output_data[i] = (*dY) * weight[row] * (_Exp(log_prob[i]) - 1.0 * (d == label[row])) / (*normalize_factor);
  }
}

template <typename T, typename Tin>
__global__ void _WeightedReductionNoneSoftmaxCrossEntropyLossGrad(
    const T* dY,
    const T* log_prob,
    const Tin* label,
    const T* weight,
    const T* normalize_factor,
    T* output_data,
    CUDA_LONG N_D,
    CUDA_LONG C) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(i, N_D * C);

  int row = i / C;
  int d = i % C;
  CUDA_KERNEL_ASSERT(weight[row] == 0 || (label[row] >= 0 && label[row] < C));
  if(0 == *normalize_factor){
    // normalize_factor is sum of labels' weights. Because zero 
    // sum implies all weights are 0, the loss function should 
    // be constant 0 and its corresponding gradient should be 0 as well.
    output_data[i] = 0;
  } else {
    output_data[i] = dY[row] * weight[row] * (_Exp(log_prob[i]) - 1.0 * (d == label[row])) / (*normalize_factor);
  }
}

template <typename T, typename Tin>
void SoftmaxCrossEntropyLossGradImpl(
    hipStream_t stream,
    const T* dY,
    const T* log_prob,
    const Tin* label,
    const T* weight,
    const T* normalize_factor,
    size_t count,
    size_t label_depth,
    bool reduction_none,
    T* output_data) {
  CUDA_LONG N_D = static_cast<CUDA_LONG>(count);
  CUDA_LONG C = static_cast<CUDA_LONG>(label_depth);
  int blocksPerGrid = (int)(ceil(static_cast<float>(N_D * C) / GridDim::maxThreadsPerBlock));
  if (reduction_none) {
    _WeightedReductionNoneSoftmaxCrossEntropyLossGrad<T, Tin><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
        dY,
        log_prob,
        label,
        weight,
        normalize_factor,
        output_data,
        N_D,
        C);
  } else {
    _WeightedSoftmaxCrossEntropyLossGrad<T, Tin><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
        dY,
        log_prob,
        label,
        weight,
        normalize_factor,
        output_data,
        N_D,
        C);
  }
}

#define SPECIALIZED_IMPL_SoftMaxEntropyLossGradImpl(T, Tin) \
  template void SoftmaxCrossEntropyLossGradImpl(            \
      hipStream_t stream,                                  \
      const T* dY,                                          \
      const T* log_prob,                                    \
      const Tin* label,                                     \
      const T* weight,                                      \
      const T* normalize_factor,                            \
      size_t count,                                         \
      size_t label_depth,                                   \
      bool reducation_none,                                 \
      T* output_data);

SPECIALIZED_IMPL_SoftMaxEntropyLossGradImpl(float, int32_t)
SPECIALIZED_IMPL_SoftMaxEntropyLossGradImpl(float, int64_t)

#define SPECIALIZED_IMPL_ComputeWeightsSoftmaxCrossEntropyImpl(T, Tin) \
  template void ComputeWeightsSoftmaxCrossEntropyImpl(                 \
      hipStream_t stream,                                             \
      const Tin* label,                                                \
      const T* weight,                                                 \
      size_t count,                                                    \
      size_t label_depth,                                              \
      int64_t ignore_index,                                            \
      T* weight_data_nd);

SPECIALIZED_IMPL_ComputeWeightsSoftmaxCrossEntropyImpl(float, int32_t)
SPECIALIZED_IMPL_ComputeWeightsSoftmaxCrossEntropyImpl(float, int64_t)

}  // namespace cuda
}  // namespace onnxruntime